#include "hip/hip_runtime.h"
#include "PerSplatLoss.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#define TensorView _Slang_TensorView
#include "generated/slang_all.cu"
#undef TensorView

#include "common.cuh"

__global__ void per_splat_losses_forward_kernel(
    bool is_3dgs,
    const size_t num_points,
    const float* __restrict__ scales_buffer,
    const float* __restrict__ opacities_buffer,
    const float* __restrict__ quats_buffer,
    float* __restrict__ out_losses,
    float max_gauss_ratio,
    float scale_regularization_weight,
    float mcmc_opacity_reg_weight,
    float mcmc_scale_reg_weight,
    float erank_reg_weight,
    float erank_reg_weight_s3,
    float quat_norm_reg_weight
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    FixedArray<float, kNumPerSplatLosses> losses;

    bool inside = idx < num_points;
    if (inside) {
        float3 scale;
        if (is_3dgs) scale = { scales_buffer[3*idx+0], scales_buffer[3*idx+1], scales_buffer[3*idx+2] };
        else scale = { scales_buffer[2*idx+0], scales_buffer[2*idx+1], 0.0f };
        float opacity = opacities_buffer[idx];
        float4 quat = { quats_buffer[4*idx+0], quats_buffer[4*idx+1], quats_buffer[4*idx+2], quats_buffer[4*idx+3] };
        per_splat_losses(
            is_3dgs,
            scale, opacity, quat,
            max_gauss_ratio,
            scale_regularization_weight,
            mcmc_opacity_reg_weight,
            mcmc_scale_reg_weight,
            erank_reg_weight,
            erank_reg_weight_s3,
            quat_norm_reg_weight,
            &losses
        );
    }

    auto block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    for (int i = 0; i < kNumPerSplatLosses; i++) {
        float loss = inside ? losses[i] : 0.0f;
        float loss_reduced = cg::reduce(warp, loss, cg::plus<float>());
        loss_reduced = loss_reduced / (float)num_points;
        if (warp.thread_rank() == 0) {
            atomicAdd(&out_losses[i], loss_reduced);
        }
    }
}


__global__ void per_splat_losses_backward_kernel(
    bool is_3dgs,
    const size_t num_points,
    const float* __restrict__ scales_buffer,
    const float* __restrict__ opacities_buffer,
    const float* __restrict__ quats_buffer,
    const float* __restrict__ v_out_losses,
    float* __restrict__ v_scales_buffer,
    float* __restrict__ v_opacities_buffer,
    float* __restrict__ v_quats_buffer,
    float max_gauss_ratio,
    float scale_regularization_weight,
    float mcmc_opacity_reg_weight,
    float mcmc_scale_reg_weight,
    float erank_reg_weight,
    float erank_reg_weight_s3,
    float quat_norm_reg_weight
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    bool inside = idx < num_points;
    if (!inside) return;

    FixedArray<float, kNumPerSplatLosses> v_losses;
    for (int i = 0; i < kNumPerSplatLosses; i++)
        v_losses[i] = v_out_losses[i] / (float)num_points;

    float3 scale;
    if (is_3dgs) scale = { scales_buffer[3*idx+0], scales_buffer[3*idx+1], scales_buffer[3*idx+2] };
    else scale = { scales_buffer[2*idx+0], scales_buffer[2*idx+1], 0.0f };
    float opacity = opacities_buffer[idx];
    float4 quat = { quats_buffer[4*idx+0], quats_buffer[4*idx+1], quats_buffer[4*idx+2], quats_buffer[4*idx+3] };

    float3 v_scale;
    float v_opacity;
    float4 v_quat;

    per_splat_losses_bwd(
        is_3dgs,
        scale, opacity, quat,
        v_losses,
        &v_scale, &v_opacity, &v_quat,
        max_gauss_ratio,
        scale_regularization_weight,
        mcmc_opacity_reg_weight,
        mcmc_scale_reg_weight,
        erank_reg_weight,
        erank_reg_weight_s3,
        quat_norm_reg_weight
    );

    if (is_3dgs) v_scales_buffer[3*idx+0] = v_scale.x, v_scales_buffer[3*idx+1] = v_scale.y, v_scales_buffer[3*idx+2] = v_scale.z;
    else v_scales_buffer[2*idx+0] = v_scale.x, v_scales_buffer[2*idx+1] = v_scale.y;
    v_opacities_buffer[idx] = v_opacity;
    v_quats_buffer[4*idx+0] = v_quat.x, v_quats_buffer[4*idx+1] = v_quat.y, v_quats_buffer[4*idx+2] = v_quat.z, v_quats_buffer[4*idx+3] = v_quat.w;
}



torch::Tensor compute_per_splat_losses_forward_tensor(
    torch::Tensor &scales,  // [N, 3] or [N, 2]
    torch::Tensor &opacities,  // [N, 1]
    torch::Tensor &quats,  // [N, 4]
    float mcmc_opacity_reg_weight,
    float mcmc_scale_reg_weight,
    float max_gauss_ratio,
    float scale_regularization_weight,
    float erank_reg_weight,
    float erank_reg_weight_s3,
    float quat_norm_reg_weight
) {
    DEVICE_GUARD(scales);
    CHECK_INPUT(scales);
    CHECK_INPUT(opacities);
    CHECK_INPUT(quats);

    const size_t num_points = opacities.size(0);
    const bool is_3dgs = (scales.size(-1) == 3);

    if (scales.ndimension() != 2 || scales.size(0) != num_points ||
        (scales.size(1) != 3 && scales.size(1) != 2))
        AT_ERROR("scales shape must be (n, 2) or (n, 3)");
    if (opacities.ndimension() != 2 || opacities.size(0) != num_points || opacities.size(1) != 1)
        AT_ERROR("opacities shape must be (n, 1)");
    if (quats.ndimension() != 2 || quats.size(0) != num_points || quats.size(1) != 4)
        AT_ERROR("quats shape must be (n, 4)");

    torch::Tensor loss = torch::zeros({kNumPerSplatLosses}, opacities.options());

    per_splat_losses_forward_kernel<<<_LAUNGH_ARGS_1D(num_points)>>>(
        is_3dgs,
        num_points,
        scales.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        quats.contiguous().data_ptr<float>(),
        loss.data_ptr<float>(),
        mcmc_opacity_reg_weight,
        mcmc_scale_reg_weight,
        max_gauss_ratio,
        scale_regularization_weight,
        erank_reg_weight,
        erank_reg_weight_s3,
        quat_norm_reg_weight
    );

    return loss;
}


std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
compute_per_splat_losses_backward_tensor(
    torch::Tensor &scales,  // [N, 3] or [N, 2]
    torch::Tensor &opacities,  // [N, 1]
    torch::Tensor &quats,  // [N, 4]
    torch::Tensor &v_losses,  // [kNumPerSplatLosses]
    float mcmc_opacity_reg_weight,
    float mcmc_scale_reg_weight,
    float max_gauss_ratio,
    float scale_regularization_weight,
    float erank_reg_weight,
    float erank_reg_weight_s3,
    float quat_norm_reg_weight
) {
    DEVICE_GUARD(scales);
    CHECK_INPUT(scales);
    CHECK_INPUT(opacities);
    CHECK_INPUT(quats);
    CHECK_INPUT(v_losses);

    const size_t num_points = opacities.size(0);
    const bool is_3dgs = (scales.size(-1) == 3);

    if (scales.ndimension() != 2 || scales.size(0) != num_points ||
        (scales.size(1) != 3 && scales.size(1) != 2))
        AT_ERROR("scales shape must be (n, 2) or (n, 3)");
    if (opacities.ndimension() != 2 || opacities.size(0) != num_points || opacities.size(1) != 1)
        AT_ERROR("opacities shape must be (n, 1)");
    if (quats.ndimension() != 2 || quats.size(0) != num_points || quats.size(1) != 4)
        AT_ERROR("quats shape must be (n, 4)");
    if (v_losses.ndimension() != 1 || v_losses.size(0) != kNumPerSplatLosses)
        AT_ERROR("v_losses shape must be (kNumPerSplatLosses,)");

    torch::Tensor v_scales = torch::empty_like(scales);
    torch::Tensor v_opacities = torch::empty_like(opacities);
    torch::Tensor v_quats = torch::empty_like(quats);

    per_splat_losses_backward_kernel<<<_LAUNGH_ARGS_1D(num_points)>>>(
        is_3dgs,
        num_points,
        scales.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        quats.contiguous().data_ptr<float>(),
        v_losses.data_ptr<float>(),
        v_scales.contiguous().data_ptr<float>(),
        v_opacities.contiguous().data_ptr<float>(),
        v_quats.contiguous().data_ptr<float>(),
        mcmc_opacity_reg_weight,
        mcmc_scale_reg_weight,
        max_gauss_ratio,
        scale_regularization_weight,
        erank_reg_weight,
        erank_reg_weight_s3,
        quat_norm_reg_weight
    );

    return std::make_tuple(v_scales, v_opacities, v_quats);
}


