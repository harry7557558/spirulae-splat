#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "projection.cuh"
#include "rasterization.cuh"
#include "sh.cuh"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <math.h>
#include <torch/extension.h>
#include <tuple>

namespace cg = cooperative_groups;



torch::Tensor compute_sh_forward_tensor(
    const std::string &method,
    const unsigned num_points,
    const unsigned degree,
    const unsigned degrees_to_use,
    torch::Tensor &viewdirs,  // [..., 3]
    torch::Tensor &coeffs   // [..., K, 3]
) {
    DEVICE_GUARD(viewdirs);
    unsigned num_bases = num_sh_bases(degree);
    if (coeffs.ndimension() != 3 || coeffs.size(0) != num_points ||
        coeffs.size(1) != num_bases || coeffs.size(2) != 3) {
        AT_ERROR("coeffs must have dimensions (N, D, 3)");
    }
    torch::Tensor colors = torch::empty({num_points, 3}, coeffs.options());    
    if (method == "poly") {
        compute_sh_forward_kernel<SHType::Poly><<<
            (num_points + N_THREADS - 1) / N_THREADS,
            N_THREADS>>>(
            num_points,
            degree,
            degrees_to_use,
            (float3 *)viewdirs.contiguous().data_ptr<float>(),
            coeffs.contiguous().data_ptr<float>(),
            colors.contiguous().data_ptr<float>()
        );
    } else if (method == "fast") {
        compute_sh_forward_kernel<SHType::Fast><<<
            (num_points + N_THREADS - 1) / N_THREADS,
            N_THREADS>>>(
            num_points,
            degree,
            degrees_to_use,
            (float3 *)viewdirs.contiguous().data_ptr<float>(),
            coeffs.contiguous().data_ptr<float>(),
            colors.contiguous().data_ptr<float>()
        );
    } else {
        AT_ERROR("Invalid method: ", method);
    }
    return colors;
}


torch::Tensor compute_sh_backward_tensor(
    const std::string &method,
    const unsigned num_points,
    const unsigned degree,
    const unsigned degrees_to_use,
    torch::Tensor &viewdirs,  // [..., 3]
    torch::Tensor &v_colors  // [..., 3]
) {
    DEVICE_GUARD(viewdirs);
    if (viewdirs.ndimension() != 2 || viewdirs.size(0) != num_points ||
        viewdirs.size(1) != 3) {
        AT_ERROR("viewdirs must have dimensions (N, 3)");
    }
    if (v_colors.ndimension() != 2 || v_colors.size(0) != num_points ||
        v_colors.size(1) != 3) {
        AT_ERROR("v_colors must have dimensions (N, 3)");
    }
    unsigned num_bases = num_sh_bases(degree);
    torch::Tensor v_coeffs =
        torch::zeros({num_points, num_bases, 3}, v_colors.options());
    if (method == "poly") {
        compute_sh_backward_kernel<SHType::Poly><<<
            (num_points + N_THREADS - 1) / N_THREADS,
            N_THREADS>>>(
            num_points,
            degree,
            degrees_to_use,
            (float3 *)viewdirs.contiguous().data_ptr<float>(),
            v_colors.contiguous().data_ptr<float>(),
            v_coeffs.contiguous().data_ptr<float>()
        );
    } else if (method == "fast") {
        compute_sh_backward_kernel<SHType::Fast><<<
            (num_points + N_THREADS - 1) / N_THREADS,
            N_THREADS>>>(
            num_points,
            degree,
            degrees_to_use,
            (float3 *)viewdirs.contiguous().data_ptr<float>(),
            v_colors.contiguous().data_ptr<float>(),
            v_coeffs.contiguous().data_ptr<float>()
        );
    } else {
        AT_ERROR("Invalid method: ", method);
    }
    return v_coeffs;
}


std::tuple<
    torch::Tensor,  // bounds, [N, 4], int
    torch::Tensor,  // num_tiles_hit, [N]
    torch::Tensor,  // positions, [N, 3]
    torch::Tensor,  // axes_u, [N, 3]
    torch::Tensor,  // axes_v, [N, 4]
    torch::Tensor  // depth_gradsm, [N, 2]
> project_gaussians_forward_tensor(
    const int num_points,
    torch::Tensor &means3d,  // [N, 3]
    torch::Tensor &scales,  // [N, 2]
    torch::Tensor &quats,  // [N, 4]
    torch::Tensor &viewmat,  // [3|4, 4]
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const unsigned img_height,
    const unsigned img_width,
    const unsigned block_width,
    const float clip_thresh
) {
    DEVICE_GUARD(means3d);

    dim3 tile_bounds_dim3;
    tile_bounds_dim3.x = int((img_width + block_width - 1) / block_width);
    tile_bounds_dim3.y = int((img_height + block_width - 1) / block_width);
    tile_bounds_dim3.z = 1;

    float4 intrins = {fx, fy, cx, cy};

    auto int32 = means3d.options().dtype(torch::kInt32);
    auto float32 = means3d.options().dtype(torch::kFloat32);
    torch::Tensor bounds_d = torch::zeros({num_points, 4}, int32);
    torch::Tensor num_tiles_hit_d = torch::zeros({num_points}, int32);
    torch::Tensor positions_d = torch::zeros({num_points, 3}, float32);
    torch::Tensor axes_u_d = torch::zeros({num_points, 3}, float32);
    torch::Tensor axes_v_d = torch::zeros({num_points, 3}, float32);
    torch::Tensor depth_grads_d = torch::zeros({num_points, 2}, float32);

    project_gaussians_forward_kernel<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        (float3 *)means3d.contiguous().data_ptr<float>(),
        (float2 *)scales.contiguous().data_ptr<float>(),
        (float4 *)quats.contiguous().data_ptr<float>(),
        viewmat.contiguous().data_ptr<float>(),
        intrins,
        tile_bounds_dim3,
        block_width,
        clip_thresh,
        // Outputs.
        (int4 *)bounds_d.contiguous().data_ptr<int32_t>(),
        num_tiles_hit_d.contiguous().data_ptr<int32_t>(),
        (float3 *)positions_d.contiguous().data_ptr<float>(),
        (float3 *)axes_u_d.contiguous().data_ptr<float>(),
        (float3 *)axes_v_d.contiguous().data_ptr<float>(),
        (float2 *)depth_grads_d.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        bounds_d, num_tiles_hit_d,
        positions_d, axes_u_d, axes_v_d,
        depth_grads_d
    );
}

std::tuple<
    torch::Tensor,  // v_means3d
    torch::Tensor,  // v_scales
    torch::Tensor,  // v_quats
    torch::Tensor  // v_viewmat, [4, 4]
> project_gaussians_backward_tensor(
    const int num_points,
    torch::Tensor &means3d,  // [N, 3]
    torch::Tensor &scales,  // [N, 2]
    torch::Tensor &quats,  // [N, 4]
    torch::Tensor &viewmat,  // [3|4, 4]
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    torch::Tensor &num_tiles_hit,  // [N], int
    torch::Tensor &v_positions,  // [N, 3]
    torch::Tensor &v_axes_u,  // [N, 3]
    torch::Tensor &v_axes_v,  // [N, 3]
    torch::Tensor &v_depth_grads  // [N, 2]
) {
    DEVICE_GUARD(means3d);

    float4 intrins = {fx, fy, cx, cy};

    auto int32 = means3d.options().dtype(torch::kInt32);
    auto float32 = means3d.options().dtype(torch::kFloat32);
    torch::Tensor v_means3d = torch::zeros({num_points, 3}, float32);
    torch::Tensor v_scales = torch::zeros({num_points, 2}, float32);
    torch::Tensor v_quats = torch::zeros({num_points, 4}, float32);
    torch::Tensor v_viewmat = torch::zeros({4, 4}, float32);

    project_gaussians_backward_kernel<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        (float3 *)means3d.contiguous().data_ptr<float>(),
        (float2 *)scales.contiguous().data_ptr<float>(),
        (float4 *)quats.contiguous().data_ptr<float>(),
        viewmat.contiguous().data_ptr<float>(),
        intrins,
        num_tiles_hit.contiguous().data_ptr<int32_t>(),
        (float3 *)v_positions.contiguous().data_ptr<float>(),
        (float3 *)v_axes_u.contiguous().data_ptr<float>(),
        (float3 *)v_axes_v.contiguous().data_ptr<float>(),
        (float2 *)v_depth_grads.contiguous().data_ptr<float>(),
        // Outputs.
        (float3 *)v_means3d.contiguous().data_ptr<float>(),
        (float2 *)v_scales.contiguous().data_ptr<float>(),
        (float4 *)v_quats.contiguous().data_ptr<float>(),
        (float *)v_viewmat.contiguous().data_ptr<float>()
    );

    return std::make_tuple(v_means3d, v_scales, v_quats, v_viewmat);
}


std::tuple<
    torch::Tensor,
    torch::Tensor
> map_gaussian_to_intersects_tensor(
    const int num_points,
    const int num_intersects,
    const torch::Tensor &positions,
    const torch::Tensor &bounds,
    const torch::Tensor &cum_tiles_hit,
    const std::tuple<int, int, int> tile_bounds,
    const unsigned block_width
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(positions);
    CHECK_INPUT(bounds);
    CHECK_INPUT(cum_tiles_hit);

    dim3 tile_bounds_dim3;
    tile_bounds_dim3.x = std::get<0>(tile_bounds);
    tile_bounds_dim3.y = std::get<1>(tile_bounds);
    tile_bounds_dim3.z = std::get<2>(tile_bounds);

    auto int32 = positions.options().dtype(torch::kInt32);
    auto int64 = positions.options().dtype(torch::kInt64);
    torch::Tensor gaussian_ids_unsorted =
        torch::zeros({num_intersects}, int32);
    torch::Tensor isect_ids_unsorted =
        torch::zeros({num_intersects}, int64);

    map_gaussian_to_intersects<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        (float3 *)positions.contiguous().data_ptr<float>(),
        (int4 *)bounds.contiguous().data_ptr<int32_t>(),
        cum_tiles_hit.contiguous().data_ptr<int32_t>(),
        tile_bounds_dim3,
        block_width,
        // Outputs.
        isect_ids_unsorted.contiguous().data_ptr<int64_t>(),
        gaussian_ids_unsorted.contiguous().data_ptr<int32_t>()
    );

    return std::make_tuple(isect_ids_unsorted, gaussian_ids_unsorted);
}


torch::Tensor get_tile_bin_edges_tensor(
    int num_intersects, const torch::Tensor &isect_ids_sorted, 
    const std::tuple<int, int, int> tile_bounds
) {
    DEVICE_GUARD(isect_ids_sorted);
    CHECK_INPUT(isect_ids_sorted);
    int num_tiles = std::get<0>(tile_bounds) * std::get<1>(tile_bounds);
    torch::Tensor tile_bins = torch::zeros(
        {num_tiles, 2}, isect_ids_sorted.options().dtype(torch::kInt32)
    );
    get_tile_bin_edges<<<
        (num_intersects + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_intersects,
        isect_ids_sorted.contiguous().data_ptr<int64_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>()
    );
    return tile_bins;
}



std::tuple<
    torch::Tensor,  // new_opacities
    torch::Tensor  // new_scales
> compute_relocation_tensor(
    torch::Tensor &opacities,
    torch::Tensor &scales,
    torch::Tensor &ratios,
    torch::Tensor &binoms,
    const int n_max
) {
    DEVICE_GUARD(opacities);
    CHECK_INPUT(opacities);
    CHECK_INPUT(scales);
    CHECK_INPUT(ratios);
    CHECK_INPUT(binoms);
    torch::Tensor new_opacities = torch::empty_like(opacities);
    torch::Tensor new_scales = torch::empty_like(scales);

    uint32_t N = opacities.size(0);
    uint32_t num_scales = scales.size(1);
    if (N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        compute_relocation_kernel<<<(N + N_THREADS - 1) / N_THREADS, N_THREADS, 0, stream>>>(
            (int)N, (int)num_scales,
            opacities.data_ptr<float>(),
            scales.data_ptr<float>(),
            ratios.data_ptr<int>(),
            binoms.data_ptr<float>(),
            n_max,
            new_opacities.data_ptr<float>(),
            new_scales.data_ptr<float>()
        );
    }
    return std::make_tuple(new_opacities, new_scales);
}

std::tuple<
    torch::Tensor,  // new_position_offsets
    torch::Tensor,  // new_opacities
    torch::Tensor  // new_scales
> compute_relocation_split_tensor(
    torch::Tensor &positions,
    torch::Tensor &quats,
    torch::Tensor &opacities,
    torch::Tensor &scales
) {
    DEVICE_GUARD(opacities);
    CHECK_INPUT(opacities);
    CHECK_INPUT(scales);
    CHECK_INPUT(positions);
    CHECK_INPUT(quats);

    torch::Tensor new_position_offsets = torch::empty_like(positions);
    torch::Tensor new_opacities = torch::empty_like(opacities);
    torch::Tensor new_scales = torch::empty_like(scales);

    uint32_t N = opacities.size(0);
    uint32_t num_scales = scales.size(1);
    if (N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        compute_relocation_split_kernel<<<(N + N_THREADS - 1) / N_THREADS, N_THREADS, 0, stream>>>(
            (int)N,
            (float3*)positions.data_ptr<float>(),
            (float4*)quats.data_ptr<float>(),
            opacities.data_ptr<float>(),
            (float2*)scales.data_ptr<float>(),
            (float3*)new_position_offsets.data_ptr<float>(),
            new_opacities.data_ptr<float>(),
            (float2*)new_scales.data_ptr<float>()
        );
    }
    return std::make_tuple(
        new_position_offsets,
        new_opacities, new_scales
    );
}





std::tuple<
    torch::Tensor,  // final_index
    torch::Tensor,  // out_img
    torch::Tensor  // out_alpha
> rasterize_simple_forward_tensor(
    const std::tuple<int, int, int> tile_bounds,
    const std::tuple<int, int, int> block,
    const std::tuple<int, int, int> img_size,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const torch::Tensor &gaussian_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const torch::Tensor &opacities,
    const torch::Tensor &anisotropies,
    const torch::Tensor &background
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(gaussian_ids_sorted);
    CHECK_INPUT(tile_bins);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(anisotropies);
    CHECK_INPUT(background);

    dim3 tile_bounds_dim3;
    tile_bounds_dim3.x = std::get<0>(tile_bounds);
    tile_bounds_dim3.y = std::get<1>(tile_bounds);
    tile_bounds_dim3.z = std::get<2>(tile_bounds);

    dim3 block_dim3;
    block_dim3.x = std::get<0>(block);
    block_dim3.y = std::get<1>(block);
    block_dim3.z = std::get<2>(block);

    dim3 img_size_dim3;
    img_size_dim3.x = std::get<0>(img_size);
    img_size_dim3.y = std::get<1>(img_size);
    img_size_dim3.z = std::get<2>(img_size);

    const int channels = colors.size(1);
    const int img_width = img_size_dim3.x;
    const int img_height = img_size_dim3.y;

    float4 intrins = {fx, fy, cx, cy};

    auto int32 = positions.options().dtype(torch::kInt32);
    auto float32 = positions.options().dtype(torch::kFloat32);
    torch::Tensor final_idx = torch::zeros(
        {img_height, img_width}, int32
    );
    torch::Tensor out_img = torch::zeros(
        {img_height, img_width, channels}, float32
    );
    torch::Tensor out_alpha = torch::zeros(
        {img_height, img_width}, float32
    );

    rasterize_simple_forward_kernel<<<tile_bounds_dim3, block_dim3>>>(
        tile_bounds_dim3,
        img_size_dim3,
        intrins,
        gaussian_ids_sorted.contiguous().data_ptr<int32_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        (float2 *)anisotropies.contiguous().data_ptr<float>(),
        *(float3 *)background.contiguous().data_ptr<float>(),
        // outputs
        final_idx.contiguous().data_ptr<int>(),
        (float3 *)out_img.contiguous().data_ptr<float>(),
        out_alpha.contiguous().data_ptr<float>()
    );

    return std::make_tuple(final_idx, out_img, out_alpha);
}

std::tuple<
    torch::Tensor, // v_positions
    torch::Tensor, // v_positions_xy_abs
    torch::Tensor, // v_axes_u
    torch::Tensor, // v_axes_v
    torch::Tensor, // v_colors
    torch::Tensor, // v_opacities
    torch::Tensor  // v_anisotropies
> rasterize_simple_backward_tensor(
    const unsigned img_height,
    const unsigned img_width,
    const unsigned block_width,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const torch::Tensor &gaussians_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const torch::Tensor &opacities,
    const torch::Tensor &anisotropies,
    const torch::Tensor &background,
    const torch::Tensor &final_idx,
    const torch::Tensor &output_alpha,
    const torch::Tensor &v_output,
    const torch::Tensor &v_output_alpha
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(final_idx);
    CHECK_INPUT(output_alpha);
    CHECK_INPUT(v_output);
    CHECK_INPUT(v_output_alpha);

    if (positions.ndimension() != 2 || positions.size(1) != 3) {
        AT_ERROR("positions must have dimensions (num_points, 3)");
    }

    if (colors.ndimension() != 2 || colors.size(1) != 3) {
        AT_ERROR("colors must have 2 dimensions");
    }

    const int num_points = positions.size(0);
    const dim3 tile_bounds = {
        (img_width + block_width - 1) / block_width,
        (img_height + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {img_width, img_height, 1};
    const int channels = colors.size(1);
    float4 intrins = {fx, fy, cx, cy};

    auto options = positions.options();
    torch::Tensor v_positions = torch::zeros({num_points, 3}, options);
    torch::Tensor v_positions_xy_abs = torch::zeros({num_points, 2}, options);
    torch::Tensor v_axes_u = torch::zeros({num_points, 3}, options);
    torch::Tensor v_axes_v = torch::zeros({num_points, 3}, options);
    torch::Tensor v_colors = torch::zeros({num_points, channels}, options);
    torch::Tensor v_opacities = torch::zeros({num_points, 1}, options);
    torch::Tensor v_anisotropies = torch::zeros({num_points, 2}, options);

    rasterize_simple_backward_kernel<<<tile_bounds, block>>>(
        tile_bounds,
        img_size,
        intrins,
        gaussians_ids_sorted.contiguous().data_ptr<int>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        (float2 *)anisotropies.contiguous().data_ptr<float>(),
        *(float3 *)background.contiguous().data_ptr<float>(),
        final_idx.contiguous().data_ptr<int>(),
        output_alpha.contiguous().data_ptr<float>(),
        (float3 *)v_output.contiguous().data_ptr<float>(),
        v_output_alpha.contiguous().data_ptr<float>(),
        // outputs
        (float3 *)v_positions.contiguous().data_ptr<float>(),
        (float2 *)v_positions_xy_abs.contiguous().data_ptr<float>(),
        (float3 *)v_axes_u.contiguous().data_ptr<float>(),
        (float3 *)v_axes_v.contiguous().data_ptr<float>(),
        (float3 *)v_colors.contiguous().data_ptr<float>(),
        v_opacities.contiguous().data_ptr<float>(),
        (float2 *)v_anisotropies.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        v_positions, v_positions_xy_abs,
        v_axes_u, v_axes_v,
        v_colors, v_opacities, v_anisotropies
    );
}



std::tuple<
    torch::Tensor,  // final_index
    torch::Tensor,  // out_img
    torch::Tensor  // out_visibility
> rasterize_depth_forward_tensor(
    const int depth_mode,
    const std::tuple<int, int, int> tile_bounds,
    const std::tuple<int, int, int> block,
    const std::tuple<int, int, int> img_size,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const torch::Tensor &gaussian_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &opacities,
    const torch::Tensor &anisotropies
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(gaussian_ids_sorted);
    CHECK_INPUT(tile_bins);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(opacities);
    CHECK_INPUT(anisotropies);

    dim3 tile_bounds_dim3;
    tile_bounds_dim3.x = std::get<0>(tile_bounds);
    tile_bounds_dim3.y = std::get<1>(tile_bounds);
    tile_bounds_dim3.z = std::get<2>(tile_bounds);

    dim3 block_dim3;
    block_dim3.x = std::get<0>(block);
    block_dim3.y = std::get<1>(block);
    block_dim3.z = std::get<2>(block);

    dim3 img_size_dim3;
    img_size_dim3.x = std::get<0>(img_size);
    img_size_dim3.y = std::get<1>(img_size);
    img_size_dim3.z = std::get<2>(img_size);

    const int img_width = img_size_dim3.x;
    const int img_height = img_size_dim3.y;

    float4 intrins = {fx, fy, cx, cy};

    auto dtype = positions.dtype();
    if (dtype != torch::kFloat32 && dtype != torch::kFloat16)
        AT_ERROR("Must be FP32 or FP16");
    if (axes_u.dtype() != positions.dtype() ||
        axes_v.dtype() != positions.dtype() ||
        opacities.dtype() != positions.dtype() ||
        anisotropies.dtype() != positions.dtype()
    ) AT_ERROR("dtype must match");

    auto int32 = positions.options().dtype(torch::kInt32);
    auto floatt = positions.options();
    torch::Tensor final_idx = torch::zeros(
        {img_height, img_width}, int32
    );
    torch::Tensor out_depth = torch::zeros(
        {img_height, img_width, 1}, floatt
    );
    torch::Tensor out_visibility = torch::zeros(
        {img_height, img_width, 2}, floatt
    );

    if (positions.dtype() == torch::kFloat32)
        rasterize_depth_forward_kernel<float><<<tile_bounds_dim3, block_dim3>>>(
            depth_mode,
            tile_bounds_dim3,
            img_size_dim3,
            intrins,
            gaussian_ids_sorted.contiguous().data_ptr<int32_t>(),
            (int2 *)tile_bins.contiguous().data_ptr<int>(),
            (vec3<float> *)positions.contiguous().data_ptr<float>(),
            (vec3<float> *)axes_u.contiguous().data_ptr<float>(),
            (vec3<float> *)axes_v.contiguous().data_ptr<float>(),
            opacities.contiguous().data_ptr<float>(),
            (vec2<float> *)anisotropies.contiguous().data_ptr<float>(),
            // outputs
            final_idx.contiguous().data_ptr<int>(),
            out_depth.contiguous().data_ptr<float>(),
            (vec2<float> *)out_visibility.contiguous().data_ptr<float>()
        );

    if (positions.dtype() == torch::kFloat16)
        rasterize_depth_forward_kernel<halfc><<<tile_bounds_dim3, block_dim3>>>(
            depth_mode,
            tile_bounds_dim3,
            img_size_dim3,
            intrins,
            gaussian_ids_sorted.contiguous().data_ptr<int32_t>(),
            (int2 *)tile_bins.contiguous().data_ptr<int>(),
            (vec3<halfc> *)positions.contiguous().data_ptr<at::Half>(),
            (vec3<halfc> *)axes_u.contiguous().data_ptr<at::Half>(),
            (vec3<halfc> *)axes_v.contiguous().data_ptr<at::Half>(),
            (halfc *)opacities.contiguous().data_ptr<at::Half>(),
            (vec2<halfc> *)anisotropies.contiguous().data_ptr<at::Half>(),
            // outputs
            final_idx.contiguous().data_ptr<int>(),
            (halfc *)out_depth.contiguous().data_ptr<at::Half>(),
            (vec2<halfc> *)out_visibility.contiguous().data_ptr<at::Half>()
        );

    return std::make_tuple(final_idx, out_depth, out_visibility);
}

std::tuple<
    torch::Tensor, // v_positions
    torch::Tensor, // v_positions_xy_abs
    torch::Tensor, // v_axes_u
    torch::Tensor, // v_axes_v
    torch::Tensor, // v_opacities
    torch::Tensor  // v_anisotropies
> rasterize_depth_backward_tensor(
    const int depth_mode,
    const unsigned img_height,
    const unsigned img_width,
    const unsigned block_width,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const torch::Tensor &gaussians_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &opacities,
    const torch::Tensor &anisotropies,
    const torch::Tensor &final_idx,
    const torch::Tensor &output_depth,
    const torch::Tensor &output_visibility,
    const torch::Tensor &v_output_depth
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(opacities);
    CHECK_INPUT(final_idx);
    CHECK_INPUT(output_depth);
    CHECK_INPUT(output_visibility);
    CHECK_INPUT(v_output_depth);

    if (positions.ndimension() != 2 || positions.size(1) != 3) {
        AT_ERROR("positions must have dimensions (num_points, 3)");
    }
    if (output_visibility.ndimension() != 3 || output_visibility.size(2) != 2) {
        AT_ERROR("output_visibility must have dimensions (h, w, 2)");
    }

    const int num_points = positions.size(0);
    const dim3 tile_bounds = {
        (img_width + block_width - 1) / block_width,
        (img_height + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {img_width, img_height, 1};
    float4 intrins = {fx, fy, cx, cy};

    // rewritten to test if FP16 is faster (answer: no)

    auto dtype = positions.dtype();
    if (dtype != torch::kFloat32 && dtype != torch::kFloat16)
        AT_ERROR("Must be FP32 or FP16");
    if (axes_u.dtype() != positions.dtype() ||
        axes_v.dtype() != positions.dtype() ||
        opacities.dtype() != positions.dtype() ||
        anisotropies.dtype() != positions.dtype() ||
        output_depth.dtype() != positions.dtype() ||
        output_visibility.dtype() != positions.dtype() ||
        v_output_depth.dtype() != positions.dtype()
    ) AT_ERROR("dtype must match");

    auto options = positions.options();
    torch::Tensor v_positions = torch::zeros({num_points, 3}, options);
    torch::Tensor v_positions_xy_abs = torch::zeros({num_points, 2}, options);
    torch::Tensor v_axes_u = torch::zeros({num_points, 3}, options);
    torch::Tensor v_axes_v = torch::zeros({num_points, 3}, options);
    torch::Tensor v_opacities = torch::zeros({num_points, 1}, options);
    torch::Tensor v_anisotropies = torch::zeros({num_points, 2}, options);

    if (positions.dtype() == torch::kFloat32)
        rasterize_depth_backward_kernel<float><<<tile_bounds, block>>>(
            depth_mode,
            tile_bounds,
            img_size,
            intrins,
            gaussians_ids_sorted.contiguous().data_ptr<int>(),
            (int2 *)tile_bins.contiguous().data_ptr<int>(),
            (vec3<float> *)positions.contiguous().data_ptr<float>(),
            (vec3<float> *)axes_u.contiguous().data_ptr<float>(),
            (vec3<float> *)axes_v.contiguous().data_ptr<float>(),
            opacities.contiguous().data_ptr<float>(),
            (vec2<float> *)anisotropies.contiguous().data_ptr<float>(),
            final_idx.contiguous().data_ptr<int>(),
            output_depth.contiguous().data_ptr<float>(),
            (vec2<float> *)output_visibility.contiguous().data_ptr<float>(),
            v_output_depth.contiguous().data_ptr<float>(),
            // outputs
            (vec3<float> *)v_positions.contiguous().data_ptr<float>(),
            (vec2<float> *)v_positions_xy_abs.contiguous().data_ptr<float>(),
            (vec3<float> *)v_axes_u.contiguous().data_ptr<float>(),
            (vec3<float> *)v_axes_v.contiguous().data_ptr<float>(),
            v_opacities.contiguous().data_ptr<float>(),
            (vec2<float> *)v_anisotropies.contiguous().data_ptr<float>()
        );

    if (positions.dtype() == torch::kFloat16)
        rasterize_depth_backward_kernel<halfc><<<tile_bounds, block>>>(
            depth_mode,
            tile_bounds,
            img_size,
            intrins,
            gaussians_ids_sorted.contiguous().data_ptr<int>(),
            (int2 *)tile_bins.contiguous().data_ptr<int>(),
            (vec3<halfc> *)positions.contiguous().data_ptr<at::Half>(),
            (vec3<halfc> *)axes_u.contiguous().data_ptr<at::Half>(),
            (vec3<halfc> *)axes_v.contiguous().data_ptr<at::Half>(),
            (halfc *)opacities.contiguous().data_ptr<at::Half>(),
            (vec2<halfc> *)anisotropies.contiguous().data_ptr<at::Half>(),
            final_idx.contiguous().data_ptr<int>(),
            (halfc *)output_depth.contiguous().data_ptr<at::Half>(),
            (vec2<halfc> *)output_visibility.contiguous().data_ptr<at::Half>(),
            (halfc *)v_output_depth.contiguous().data_ptr<at::Half>(),
            // outputs
            (vec3<halfc> *)v_positions.contiguous().data_ptr<at::Half>(),
            (vec2<halfc> *)v_positions_xy_abs.contiguous().data_ptr<at::Half>(),
            (vec3<halfc> *)v_axes_u.contiguous().data_ptr<at::Half>(),
            (vec3<halfc> *)v_axes_v.contiguous().data_ptr<at::Half>(),
            (halfc *)v_opacities.contiguous().data_ptr<at::Half>(),
            (vec2<halfc> *)v_anisotropies.contiguous().data_ptr<at::Half>()
        );

    return std::make_tuple(
        v_positions, v_positions_xy_abs,
        v_axes_u, v_axes_v,
        v_opacities, v_anisotropies
    );
}



std::tuple<
    torch::Tensor, // final_idx
    torch::Tensor,  // out_alpha
    torch::Tensor,  // out_img
    torch::Tensor,  // out_depth
    torch::Tensor,  // out_reg_depth
    torch::Tensor  // out_reg_normal
> rasterize_forward_tensor(
    const std::tuple<int, int, int> tile_bounds,
    const std::tuple<int, int, int> block,
    const std::tuple<int, int, int> img_size,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const float depth_reg_pairwise_factor,
    const torch::Tensor &gaussian_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const torch::Tensor &ch_coeffs,
    const torch::Tensor &opacities,
    const torch::Tensor &anisotropies,
    // const torch::Tensor &background,
    const torch::Tensor &depth_grads,
    const torch::Tensor &depth_ref_im
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(gaussian_ids_sorted);
    CHECK_INPUT(tile_bins);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(ch_coeffs);
    CHECK_INPUT(opacities);
    CHECK_INPUT(anisotropies);
    // CHECK_INPUT(background);
    CHECK_INPUT(depth_grads);
    CHECK_INPUT(depth_ref_im);

    dim3 tile_bounds_dim3;
    tile_bounds_dim3.x = std::get<0>(tile_bounds);
    tile_bounds_dim3.y = std::get<1>(tile_bounds);
    tile_bounds_dim3.z = std::get<2>(tile_bounds);

    dim3 block_dim3;
    block_dim3.x = std::get<0>(block);
    block_dim3.y = std::get<1>(block);
    block_dim3.z = std::get<2>(block);

    dim3 img_size_dim3;
    img_size_dim3.x = std::get<0>(img_size);
    img_size_dim3.y = std::get<1>(img_size);
    img_size_dim3.z = std::get<2>(img_size);

    const int channels = colors.size(1);
    const int img_width = img_size_dim3.x;
    const int img_height = img_size_dim3.y;

    float4 intrins = {fx, fy, cx, cy};

    auto int32 = positions.options().dtype(torch::kInt32);
    auto float32 = positions.options().dtype(torch::kFloat32);
    torch::Tensor final_idx = torch::zeros(
        {img_height, img_width}, int32
    );
    torch::Tensor out_alpha = torch::zeros(
        {img_height, img_width}, float32
    );
    torch::Tensor out_img = torch::zeros(
        {img_height, img_width, channels}, float32
    );
    torch::Tensor out_depth_grad = torch::zeros(
        {img_height, img_width, 4}, float32
    );
    torch::Tensor out_reg_depth = torch::zeros(
        {img_height, img_width}, float32
    );
    torch::Tensor out_reg_normal = torch::zeros(
        {img_height, img_width}, float32
    );

    rasterize_forward_kernel<<<tile_bounds_dim3, block_dim3>>>(
        tile_bounds_dim3,
        img_size_dim3,
        intrins,
        depth_reg_pairwise_factor,
        gaussian_ids_sorted.contiguous().data_ptr<int32_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        (unsigned)ch_degree_r, (unsigned)ch_degree_r_to_use,
        (unsigned)ch_degree_phi, (unsigned)ch_degree_phi_to_use,
        (float3 *)ch_coeffs.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        (float2 *)anisotropies.contiguous().data_ptr<float>(),
        // *(float3 *)background.contiguous().data_ptr<float>(),
        (float2 *)depth_grads.contiguous().data_ptr<float>(),
        (float3 *)depth_ref_im.contiguous().data_ptr<float>(),
        // outputs
        final_idx.contiguous().data_ptr<int>(),
        out_alpha.contiguous().data_ptr<float>(),
        (float3 *)out_img.contiguous().data_ptr<float>(),
        (float4 *)out_depth_grad.contiguous().data_ptr<float>(),
        out_reg_depth.contiguous().data_ptr<float>(),
        out_reg_normal.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        final_idx, out_alpha,
        out_img, out_depth_grad,
        out_reg_depth, out_reg_normal
    );
}


std::tuple<
    torch::Tensor, // v_positions
    torch::Tensor, // v_positions_xy_abs
    torch::Tensor, // v_axes_u
    torch::Tensor, // v_axes_v
    torch::Tensor, // v_colors
    torch::Tensor, // v_ch_coeffs
    // torch::Tensor, // v_ch_coeffs_abs
    torch::Tensor, // v_opacities
    torch::Tensor, // v_anisotropies
    // torch::Tensor, // v_background
    torch::Tensor, // v_depth_grad
    torch::Tensor  // v_depth_ref_im
> rasterize_backward_tensor(
    const unsigned img_height,
    const unsigned img_width,
    const unsigned block_width,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float depth_reg_pairwise_factor,
    const torch::Tensor &gaussians_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const torch::Tensor &ch_coeffs,
    const torch::Tensor &opacities,
    const torch::Tensor &anisotropies,
    // const torch::Tensor &background,
    const torch::Tensor &depth_grads,
    const torch::Tensor &depth_ref_im,
    const torch::Tensor &final_idx,
    const torch::Tensor &output_alpha,
    const torch::Tensor &output_depth_grad,
    const torch::Tensor &v_output_alpha,
    const torch::Tensor &v_output,
    const torch::Tensor &v_output_depth_grad,
    const torch::Tensor &v_output_reg_depth,
    const torch::Tensor &v_output_reg_normal
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(ch_coeffs);
    CHECK_INPUT(opacities);
    CHECK_INPUT(anisotropies);
    CHECK_INPUT(final_idx);
    CHECK_INPUT(output_alpha);
    CHECK_INPUT(output_depth_grad);
    CHECK_INPUT(v_output_alpha);
    CHECK_INPUT(v_output);
    CHECK_INPUT(v_output_depth_grad);
    CHECK_INPUT(v_output_reg_depth);
    CHECK_INPUT(v_output_reg_normal);

    if (positions.ndimension() != 2 || positions.size(1) != 3) {
        AT_ERROR("xys must have dimensions (num_points, 2)");
    }
    if (colors.ndimension() != 2 || colors.size(1) != 3) {
        AT_ERROR("colors must have 2 dimensions");
    }
    if (output_depth_grad.ndimension() != 3 || output_depth_grad.size(2) != 4) {
        AT_ERROR("output_depth_grad must have 3 dimensions");
    }
    if (v_output_depth_grad.ndimension() != 3 || v_output_depth_grad.size(2) != 4) {
        AT_ERROR("v_output_depth_grad must have 3 dimensions");
    }

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);

    const int num_points = positions.size(0);
    const dim3 tile_bounds = {
        (img_width + block_width - 1) / block_width,
        (img_height + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {img_width, img_height, 1};
    const int channels = colors.size(1);
    float4 intrins = {fx, fy, cx, cy};

    auto options = positions.options();
    torch::Tensor v_positions = torch::zeros({num_points, 3}, options);
    torch::Tensor v_positions_xy_abs = torch::zeros({num_points, 2}, options);
    torch::Tensor v_axes_u = torch::zeros({num_points, 3}, options);
    torch::Tensor v_axes_v = torch::zeros({num_points, 3}, options);
    torch::Tensor v_colors = torch::zeros({num_points, channels}, options);
    torch::Tensor v_ch_coeffs = torch::zeros({num_points, dim_ch, channels}, options);
    // torch::Tensor v_ch_coeffs_abs = torch::zeros({num_points, 1}, options);
    torch::Tensor v_opacities = torch::zeros({num_points, 1}, options);
    torch::Tensor v_anisotropies = torch::zeros({num_points, 2}, options);
    // torch::Tensor v_background = torch::zeros({3}, options);
    torch::Tensor v_depth_grad = torch::zeros({num_points, 2}, options);
    torch::Tensor v_depth_ref_im = torch::zeros({img_height, img_width, 3}, options);

    rasterize_backward_kernel<<<tile_bounds, block>>>(
        tile_bounds, img_size, intrins,
        (unsigned)ch_degree_r, (unsigned)ch_degree_r_to_use,
        (unsigned)ch_degree_phi, (unsigned)ch_degree_phi_to_use,
        depth_reg_pairwise_factor,
        gaussians_ids_sorted.contiguous().data_ptr<int>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        (float3 *)ch_coeffs.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        (float2 *)anisotropies.contiguous().data_ptr<float>(),
        // *(float3 *)background.contiguous().data_ptr<float>(),
        (float2 *)depth_grads.contiguous().data_ptr<float>(),
        (float3 *)depth_ref_im.contiguous().data_ptr<float>(),
        final_idx.contiguous().data_ptr<int>(),
        output_alpha.contiguous().data_ptr<float>(),
        (float4 *)output_depth_grad.contiguous().data_ptr<float>(),
        v_output_alpha.contiguous().data_ptr<float>(),
        (float3 *)v_output.contiguous().data_ptr<float>(),
        (float4 *)v_output_depth_grad.contiguous().data_ptr<float>(),
        v_output_reg_depth.contiguous().data_ptr<float>(),
        v_output_reg_normal.contiguous().data_ptr<float>(),
        // outputs
        (float3 *)v_positions.contiguous().data_ptr<float>(),
        (float2 *)v_positions_xy_abs.contiguous().data_ptr<float>(),
        (float3 *)v_axes_u.contiguous().data_ptr<float>(),
        (float3 *)v_axes_v.contiguous().data_ptr<float>(),
        (float3 *)v_colors.contiguous().data_ptr<float>(),
        (float3 *)v_ch_coeffs.contiguous().data_ptr<float>(),
        // v_ch_coeffs_abs.contiguous().data_ptr<float>(),
        v_opacities.contiguous().data_ptr<float>(),
        (float2 *)v_anisotropies.contiguous().data_ptr<float>(),
        // (float3 *)v_background.contiguous().data_ptr<float>(),
        (float2 *)v_depth_grad.contiguous().data_ptr<float>(),
        (float3 *)v_depth_ref_im.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        v_positions, v_positions_xy_abs,
        v_axes_u, v_axes_v,
        v_colors, v_ch_coeffs, //v_ch_coeffs_abs,
        v_opacities, v_anisotropies,
        // v_background,
        v_depth_grad, v_depth_ref_im
    );
}



torch::Tensor render_background_sh_forward_tensor(
    const unsigned w,
    const unsigned h,
    const unsigned block_width,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const torch::Tensor &rotation,
    const unsigned sh_degree,
    const torch::Tensor &sh_coeffs
) {
    DEVICE_GUARD(sh_coeffs);
    CHECK_INPUT(sh_coeffs);
    CHECK_INPUT(rotation);

    if (rotation.numel() != 9) {
        AT_ERROR("rotation must be 3x3");
    }
    if (sh_coeffs.ndimension() != 2 ||
        sh_coeffs.size(0) != sh_degree*sh_degree ||
        sh_coeffs.size(1) != 3) {
        AT_ERROR("sh_coeffs must be (sh_regree**2, 3)");
    }

    const dim3 tile_bounds = {
        (w + block_width - 1) / block_width,
        (h + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {w, h, 1};

    auto options = sh_coeffs.options();
    torch::Tensor out_color = torch::empty({h, w, 3}, options);

    render_background_sh_forward_kernel<<<tile_bounds, block>>>(
        tile_bounds, img_size,
        fx, fy, cx, cy,
        rotation.contiguous().data_ptr<float>(),
        sh_degree,
        (float3 *)sh_coeffs.contiguous().data_ptr<float>(),
        (float3 *)out_color.contiguous().data_ptr<float>()
    );

    return out_color;
}


std::tuple<
    torch::Tensor,  // v_rotation
    torch::Tensor  // v_sh_coeffs
> render_background_sh_backward_tensor(
    const unsigned w,
    const unsigned h,
    const unsigned block_width,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const torch::Tensor &rotation,
    const unsigned sh_degree,
    const torch::Tensor &sh_coeffs,
    const torch::Tensor &out_color,
    const torch::Tensor &v_out_color
) {
    DEVICE_GUARD(sh_coeffs);
    CHECK_INPUT(sh_coeffs);
    CHECK_INPUT(rotation);
    CHECK_INPUT(v_out_color);

    if (rotation.numel() != 9) {
        AT_ERROR("rotation must be 3x3");
    }
    if (sh_coeffs.ndimension() != 2 ||
        sh_coeffs.size(0) != sh_degree*sh_degree ||
        sh_coeffs.size(1) != 3) {
        AT_ERROR("sh_coeffs shape must be (sh_regree**2, 3)");
    }
    if (out_color.ndimension() != 3 ||
        out_color.size(0) != h ||
        out_color.size(1) != w ||
        out_color.size(2) != 3) {
        AT_ERROR("out_color shape must be (h, w, 3)");
    }
    if (v_out_color.ndimension() != 3 ||
        v_out_color.size(0) != h ||
        v_out_color.size(1) != w ||
        v_out_color.size(2) != 3) {
        AT_ERROR("v_out_color shape must be (h, w, 3)");
    }

    const dim3 tile_bounds = {
        (w + block_width - 1) / block_width,
        (h + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {w, h, 1};

    auto options = sh_coeffs.options();
    torch::Tensor v_rotation = torch::zeros({3, 3}, options);
    torch::Tensor v_sh_coeffs = torch::zeros({sh_degree*sh_degree, 3}, options);

    render_background_sh_backward_kernel<<<tile_bounds, block>>>(
        tile_bounds, img_size,
        fx, fy, cx, cy,
        rotation.contiguous().data_ptr<float>(),
        sh_degree,
        (float3 *)sh_coeffs.contiguous().data_ptr<float>(),
        (float3 *)out_color.contiguous().data_ptr<float>(),
        (float3 *)v_out_color.contiguous().data_ptr<float>(),
        v_rotation.contiguous().data_ptr<float>(),
        (float3 *)v_sh_coeffs.contiguous().data_ptr<float>()
    );

    return std::make_tuple(v_rotation, v_sh_coeffs);
}
