#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "projection.cuh"
#include "rasterization.cuh"
#include "sh.cuh"
#include "misc.cuh"

#include "splat_tile_intersector.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <tuple>
#include <optional>

#define TORCH_INDUCTOR_CPP_WRAPPER
#include <torch/extension.h>
#include <torch/types.h>


#define BLOCK_DIM3 dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1)


inline __host__ float4 tuple2float4(std::tuple<float, float, float, float> v) {
    return {std::get<0>(v), std::get<1>(v), std::get<2>(v), std::get<3>(v)};
}

inline __host__ dim3 tuple2dim3(std::tuple<unsigned, unsigned, unsigned> v) {
    return {std::get<0>(v), std::get<1>(v), std::get<2>(v)};
}

inline __host__ dim3 whb2tb(unsigned width, unsigned height, unsigned block_width=BLOCK_WIDTH) {
    return {
        (width + block_width - 1) / block_width,
        (height + block_width - 1) / block_width,
        1
    };
}


template<typename T, int ndim>
TensorView<T, ndim> tensor2view(torch::Tensor& tensor) {
    TensorView<T, ndim> view;
    view.data = tensor.data_ptr<T>();
    for (int i = 0; i < ndim; i++) {
        view.shape[i] = tensor.size(i);
        view.strides[i] = *(tensor.strides().begin() + i);
    }
    return view;
}



torch::Tensor compute_sh_forward_tensor(
    const std::string &method,
    const unsigned num_points,
    const unsigned degree,
    const unsigned degrees_to_use,
    torch::Tensor &viewdirs,  // [..., 3]
    torch::Tensor &coeffs0,   // [..., 3]
    torch::Tensor &coeffs   // [..., K, 3]
) {
    DEVICE_GUARD(viewdirs);
    unsigned num_bases = num_sh_bases(degree);
    if (coeffs0.ndimension() != 2 || coeffs0.size(0) != num_points ||
        coeffs0.size(1) != 3) {
        AT_ERROR("coeffs0 must have dimensions (N, 3)");
    }
    if (coeffs.ndimension() != 3 || coeffs.size(0) != num_points ||
        coeffs.size(1) != num_bases-1 || coeffs.size(2) != 3) {
        AT_ERROR("coeffs must have dimensions (N, D, 3)");
    }
    torch::Tensor colors = torch::empty({num_points, 3}, coeffs.options());

    #define _TEMP_ARGS  \
        num_points, degree, degrees_to_use, \
        (float3 *)viewdirs.contiguous().data_ptr<float>(), \
        coeffs0.contiguous().data_ptr<float>(), \
        coeffs.contiguous().data_ptr<float>(), \
        colors.contiguous().data_ptr<float>()

    if (method == "poly") {
        compute_sh_forward_kernel<SHType::Poly>
        <<<_LAUNGH_ARGS_1D(num_points)>>>(_TEMP_ARGS);
    } else if (method == "fast") {
        compute_sh_forward_kernel<SHType::Fast>
        <<<_LAUNGH_ARGS_1D(num_points)>>>(_TEMP_ARGS);
    } else {
        AT_ERROR("Invalid method: ", method);
    }

    #undef _TEMP_ARGS

    return colors;
}


std::tuple<torch::Tensor, torch::Tensor>
compute_sh_backward_tensor(
    const std::string &method,
    const unsigned num_points,
    const unsigned degree,
    const unsigned degrees_to_use,
    torch::Tensor &viewdirs,  // [..., 3]
    torch::Tensor &v_colors  // [..., 3]
) {
    DEVICE_GUARD(viewdirs);
    if (viewdirs.ndimension() != 2 || viewdirs.size(0) != num_points ||
        viewdirs.size(1) != 3) {
        AT_ERROR("viewdirs must have dimensions (N, 3)");
    }
    if (v_colors.ndimension() != 2 || v_colors.size(0) != num_points ||
        v_colors.size(1) != 3) {
        AT_ERROR("v_colors must have dimensions (N, 3)");
    }
    unsigned num_bases = num_sh_bases(degree);
    torch::Tensor v_coeffs0 = torch::zeros({num_points, 3}, v_colors.options());
    torch::Tensor v_coeffs = torch::zeros({num_points, num_bases-1, 3}, v_colors.options());

    #define _TEMP_ARGS  \
        num_points, degree, degrees_to_use, \
        (float3 *)viewdirs.contiguous().data_ptr<float>(), \
        v_colors.contiguous().data_ptr<float>(), \
        v_coeffs0.contiguous().data_ptr<float>(), \
        v_coeffs.contiguous().data_ptr<float>()

    if (method == "poly") {
        compute_sh_backward_kernel<SHType::Poly>
        <<<_LAUNGH_ARGS_1D(num_points)>>>(_TEMP_ARGS);
    } else if (method == "fast") {
        compute_sh_backward_kernel<SHType::Fast>
        <<<_LAUNGH_ARGS_1D(num_points)>>>(_TEMP_ARGS);
    } else {
        AT_ERROR("Invalid method: ", method);
    }

    #undef _TEMP_ARGS

    return std::make_tuple(v_coeffs0, v_coeffs);
}



torch::Tensor render_undistortion_map_tensor(
    const unsigned w,
    const unsigned h,
    const std::string camera_model,
    const std::tuple<float, float, float, float> intrins,
    const std::tuple<float, float, float, float> dist_coeffs
) {
    const dim3 tile_bounds = whb2tb(w, h);
    const dim3 img_size = {w, h, 1};

    auto options = torch::dtype(torch::kFloat32).device(torch::kCUDA, -1);
    torch::Tensor out_img = torch::empty({h, w, 2}, options);

    if (camera_model == "OPENCV_FISHEYE")
        render_undistortion_map_kernel<CameraType::OPENCV_FISHEYE>
        <<<tile_bounds, BLOCK_DIM3>>>(
            tile_bounds, img_size,
            tuple2float4(intrins), tuple2float4(dist_coeffs),
            (float2 *)out_img.contiguous().data_ptr<float>()
        );

    else AT_ERROR("Invalid camera model: ", camera_model);

    return out_img;
}


std::tuple<
    torch::Tensor,
    torch::Tensor
> map_gaussian_to_intersects_tensor(
    const int num_points,
    const int num_intersects,
    const torch::Tensor &positions,
    const torch::Tensor &bounds,
    const torch::Tensor &cum_tiles_hit,
    const unsigned img_height,
    const unsigned img_width
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(positions);
    CHECK_INPUT(bounds);
    CHECK_INPUT(cum_tiles_hit);

    const dim3 tile_bounds = whb2tb(img_width, img_height);

    auto int32 = positions.options().dtype(torch::kInt32);
    auto int64 = positions.options().dtype(torch::kInt64);
    torch::Tensor gaussian_ids_unsorted =
        torch::zeros({num_intersects}, int32);
    torch::Tensor isect_ids_unsorted =
        torch::zeros({num_intersects}, int64);

    map_gaussian_to_intersects<<<_LAUNGH_ARGS_1D(num_points)>>>(
        num_points,
        (float3 *)positions.contiguous().data_ptr<float>(),
        (int4 *)bounds.contiguous().data_ptr<int32_t>(),
        cum_tiles_hit.contiguous().data_ptr<int32_t>(),
        tile_bounds,
        // Outputs.
        isect_ids_unsorted.contiguous().data_ptr<int64_t>(),
        gaussian_ids_unsorted.contiguous().data_ptr<int32_t>()
    );

    return std::make_tuple(isect_ids_unsorted, gaussian_ids_unsorted);
}


torch::Tensor get_tile_bin_edges_tensor(
    int num_intersects,
    const torch::Tensor &isect_ids_sorted, 
    const unsigned img_height,
    const unsigned img_width
) {
    DEVICE_GUARD(isect_ids_sorted);
    CHECK_INPUT(isect_ids_sorted);

    const dim3 tile_bounds = whb2tb(img_width, img_height);
    int num_tiles = tile_bounds.x * tile_bounds.y;
    torch::Tensor tile_bins = torch::zeros(
        {num_tiles, 2}, isect_ids_sorted.options().dtype(torch::kInt32)
    );
    get_tile_bin_edges<<<_LAUNGH_ARGS_1D(num_intersects)>>>(
        num_intersects,
        isect_ids_sorted.contiguous().data_ptr<int64_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>()
    );
    return tile_bins;
}







torch::Tensor render_background_sh_forward_tensor(
    const unsigned w,
    const unsigned h,
    std::string camera_model,
    const std::tuple<float, float, float, float> intrins,
    const std::optional<torch::Tensor> &undistortion_map_,
    const torch::Tensor &rotation,
    const unsigned sh_degree,
    const torch::Tensor &sh_coeffs
) {
    DEVICE_GUARD(sh_coeffs);
    CHECK_INPUT(sh_coeffs);
    CHECK_INPUT(rotation);

    if (rotation.numel() != 9) {
        AT_ERROR("rotation must be 3x3");
    }
    if (sh_coeffs.ndimension() != 2 ||
        sh_coeffs.size(0) != sh_degree*sh_degree ||
        sh_coeffs.size(1) != 3) {
        AT_ERROR("sh_coeffs must be (sh_regree**2, 3)");
    }

    const dim3 tile_bounds = whb2tb(w, h);
    const dim3 img_size = {w, h, 1};

    auto options = sh_coeffs.options();
    torch::Tensor out_color = torch::empty({h, w, 3}, options);

    if (camera_model == "") {
        render_background_sh_forward_kernel<CameraType::Undistorted>
        <<<tile_bounds, BLOCK_DIM3>>>(
            tile_bounds, img_size,
            tuple2float4(intrins), nullptr,
            rotation.contiguous().data_ptr<float>(),
            sh_degree,
            (float3 *)sh_coeffs.contiguous().data_ptr<float>(),
            (float3 *)out_color.contiguous().data_ptr<float>()
        );
    }

    else {
        const torch::Tensor& undistortion_map = undistortion_map_.value();
        CHECK_INPUT(undistortion_map);

        render_background_sh_forward_kernel<CameraType::GenericDistorted>
        <<<tile_bounds, BLOCK_DIM3>>>(
            tile_bounds, img_size,
            tuple2float4(intrins),
            (float2 *)undistortion_map.contiguous().data_ptr<float>(),
            rotation.contiguous().data_ptr<float>(),
            sh_degree,
            (float3 *)sh_coeffs.contiguous().data_ptr<float>(),
            (float3 *)out_color.contiguous().data_ptr<float>()
        );
    }

    return out_color;
}


std::tuple<
    torch::Tensor,  // v_rotation
    torch::Tensor  // v_sh_coeffs
> render_background_sh_backward_tensor(
    const unsigned w,
    const unsigned h,
    const std::string camera_model,
    const std::tuple<float, float, float, float> intrins,
    const std::optional<torch::Tensor> &undistortion_map_,
    const torch::Tensor &rotation,
    const unsigned sh_degree,
    const torch::Tensor &sh_coeffs,
    const torch::Tensor &out_color,
    const torch::Tensor &v_out_color
) {
    DEVICE_GUARD(sh_coeffs);
    CHECK_INPUT(sh_coeffs);
    CHECK_INPUT(rotation);
    CHECK_INPUT(v_out_color);

    if (rotation.numel() != 9) {
        AT_ERROR("rotation must be 3x3");
    }
    if (sh_coeffs.ndimension() != 2 ||
        sh_coeffs.size(0) != sh_degree*sh_degree ||
        sh_coeffs.size(1) != 3) {
        AT_ERROR("sh_coeffs shape must be (sh_regree**2, 3)");
    }
    if (out_color.ndimension() != 3 ||
        out_color.size(0) != h ||
        out_color.size(1) != w ||
        out_color.size(2) != 3) {
        AT_ERROR("out_color shape must be (h, w, 3)");
    }
    if (v_out_color.ndimension() != 3 ||
        v_out_color.size(0) != h ||
        v_out_color.size(1) != w ||
        v_out_color.size(2) != 3) {
        AT_ERROR("v_out_color shape must be (h, w, 3)");
    }

    // unsigned block_width = BLOCK_WIDTH;
    unsigned block_width = 32;  // 1024 threads
    const dim3 tile_bounds = whb2tb(w, h, block_width);
    const dim3 img_size = {w, h, 1};

    auto options = sh_coeffs.options();
    torch::Tensor v_rotation = torch::zeros({3, 3}, options);
    torch::Tensor v_sh_coeffs = torch::zeros({sh_degree*sh_degree, 3}, options);

    #define _TEMP_ARGS \
        rotation.contiguous().data_ptr<float>(), \
        sh_degree, \
        (float3 *)sh_coeffs.contiguous().data_ptr<float>(), \
        (float3 *)out_color.contiguous().data_ptr<float>(), \
        (float3 *)v_out_color.contiguous().data_ptr<float>(), \
        (float3 *)v_rotation.contiguous().data_ptr<float>(), \
        (float3 *)v_sh_coeffs.contiguous().data_ptr<float>()

    if (camera_model == "") {
        render_background_sh_backward_kernel<CameraType::Undistorted>
        <<<tile_bounds, dim3(block_width, block_width, 1)>>>(
            tile_bounds, img_size,
            tuple2float4(intrins), nullptr,
            _TEMP_ARGS
        );
    }
    else {
        const torch::Tensor& undistortion_map = undistortion_map_.value();
        CHECK_INPUT(undistortion_map);
        render_background_sh_backward_kernel<CameraType::GenericDistorted>
        <<<tile_bounds, dim3(block_width, block_width, 1)>>>(
            tile_bounds, img_size,
            tuple2float4(intrins),
            (float2 *)undistortion_map.contiguous().data_ptr<float>(),
            _TEMP_ARGS
        );
    }

    #undef _TEMP_ARGS

    return std::make_tuple(v_rotation, v_sh_coeffs);
}


torch::Tensor compute_per_splat_losses_forward_tensor(
    torch::Tensor &scales,  // [N, 3] or [N, 2]
    torch::Tensor &opacities,  // [N, 1]
    torch::Tensor &quats,  // [N, 4]
    float mcmc_opacity_reg_weight,
    float mcmc_scale_reg_weight,
    float max_gauss_ratio,
    float scale_regularization_weight,
    float erank_reg_weight,
    float erank_reg_weight_s3,
    float quat_norm_reg_weight
) {
    DEVICE_GUARD(scales);
    CHECK_INPUT(scales);
    CHECK_INPUT(opacities);
    CHECK_INPUT(quats);

    const size_t num_points = opacities.size(0);
    const bool is_3dgs = (scales.size(-1) == 3);

    if (scales.ndimension() != 2 || scales.size(0) != num_points ||
        (scales.size(1) != 3 && scales.size(1) != 2))
        AT_ERROR("scales shape must be (n, 2) or (n, 3)");
    if (opacities.ndimension() != 2 || opacities.size(0) != num_points || opacities.size(1) != 1)
        AT_ERROR("opacities shape must be (n, 1)");
    if (quats.ndimension() != 2 || quats.size(0) != num_points || quats.size(1) != 4)
        AT_ERROR("quats shape must be (n, 4)");

    torch::Tensor loss = torch::zeros({kNumPerSplatLosses}, opacities.options());

    per_splat_losses_forward_kernel<<<_LAUNGH_ARGS_1D(num_points)>>>(
        is_3dgs,
        num_points,
        scales.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        quats.contiguous().data_ptr<float>(),
        loss.data_ptr<float>(),
        mcmc_opacity_reg_weight,
        mcmc_scale_reg_weight,
        max_gauss_ratio,
        scale_regularization_weight,
        erank_reg_weight,
        erank_reg_weight_s3,
        quat_norm_reg_weight
    );

    return loss;
}


std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
compute_per_splat_losses_backward_tensor(
    torch::Tensor &scales,  // [N, 3] or [N, 2]
    torch::Tensor &opacities,  // [N, 1]
    torch::Tensor &quats,  // [N, 4]
    torch::Tensor &v_losses,  // [kNumPerSplatLosses]
    float mcmc_opacity_reg_weight,
    float mcmc_scale_reg_weight,
    float max_gauss_ratio,
    float scale_regularization_weight,
    float erank_reg_weight,
    float erank_reg_weight_s3,
    float quat_norm_reg_weight
) {
    DEVICE_GUARD(scales);
    CHECK_INPUT(scales);
    CHECK_INPUT(opacities);
    CHECK_INPUT(quats);
    CHECK_INPUT(v_losses);

    const size_t num_points = opacities.size(0);
    const bool is_3dgs = (scales.size(-1) == 3);

    if (scales.ndimension() != 2 || scales.size(0) != num_points ||
        (scales.size(1) != 3 && scales.size(1) != 2))
        AT_ERROR("scales shape must be (n, 2) or (n, 3)");
    if (opacities.ndimension() != 2 || opacities.size(0) != num_points || opacities.size(1) != 1)
        AT_ERROR("opacities shape must be (n, 1)");
    if (quats.ndimension() != 2 || quats.size(0) != num_points || quats.size(1) != 4)
        AT_ERROR("quats shape must be (n, 4)");
    if (v_losses.ndimension() != 1 || v_losses.size(0) != kNumPerSplatLosses)
        AT_ERROR("v_losses shape must be (kNumPerSplatLosses,)");

    torch::Tensor v_scales = torch::empty_like(scales);
    torch::Tensor v_opacities = torch::empty_like(opacities);
    torch::Tensor v_quats = torch::empty_like(quats);

    per_splat_losses_backward_kernel<<<_LAUNGH_ARGS_1D(num_points)>>>(
        is_3dgs,
        num_points,
        scales.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        quats.contiguous().data_ptr<float>(),
        v_losses.data_ptr<float>(),
        v_scales.contiguous().data_ptr<float>(),
        v_opacities.contiguous().data_ptr<float>(),
        v_quats.contiguous().data_ptr<float>(),
        mcmc_opacity_reg_weight,
        mcmc_scale_reg_weight,
        max_gauss_ratio,
        scale_regularization_weight,
        erank_reg_weight,
        erank_reg_weight_s3,
        quat_norm_reg_weight
    );

    return std::make_tuple(v_scales, v_opacities, v_quats);
}



torch::Tensor blend_background_forward_tensor(
    torch::Tensor &rgb,  // [H, W, 3]
    torch::Tensor &alpha,  // [H, W, 1]
    torch::Tensor &background  // [H, W, 3]
) {
    DEVICE_GUARD(rgb);
    CHECK_CUDA(rgb);
    CHECK_CUDA(alpha);
    CHECK_CUDA(background);

    if (rgb.ndimension() != 3 || rgb.size(2) != 3)
        AT_ERROR("rgb shape must be (h, w, 3)");
    long h = rgb.size(0), w = rgb.size(1);
    if (alpha.ndimension() != 3 || alpha.size(0) != h || alpha.size(1) != w || alpha.size(2) != 1)
        AT_ERROR("alpha shape must be (h, w, 1)");
    if (background.ndimension() != 3 || background.size(0) != h || background.size(1) != w || background.size(2) != 3)
        AT_ERROR("background shape must be (h, w, 3)");

    torch::Tensor out_rgb = torch::empty({h, w, 3}, rgb.options());

    blend_background_forward_kernel<<<_LAUNGH_ARGS_1D(h*w)>>>(
        tensor2view<float, 3>(rgb), tensor2view<float, 3>(alpha), tensor2view<float, 3>(background),
        tensor2view<float, 3>(out_rgb)
    );

    return out_rgb;
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
blend_background_backward_tensor(
    torch::Tensor &rgb,  // [H, W, 3]
    torch::Tensor &alpha,  // [H, W, 1]
    torch::Tensor &background,  // [H, W, 3]
    torch::Tensor &v_out_rgb  // [H, W, 3]
) {
    DEVICE_GUARD(rgb);
    CHECK_CUDA(rgb);
    CHECK_CUDA(alpha);
    CHECK_CUDA(background);
    CHECK_CUDA(v_out_rgb);

    if (rgb.ndimension() != 3 || rgb.size(2) != 3)
        AT_ERROR("rgb shape must be (h, w, 3)");
    long h = rgb.size(0), w = rgb.size(1);
    if (alpha.ndimension() != 3 || alpha.size(0) != h || alpha.size(1) != w || alpha.size(2) != 1)
        AT_ERROR("alpha shape must be (h, w, 1)");
    if (background.ndimension() != 3 || background.size(0) != h || background.size(1) != w || background.size(2) != 3)
        AT_ERROR("background shape must be (h, w, 3)");
    if (v_out_rgb.ndimension() != 3 || v_out_rgb.size(0) != h || v_out_rgb.size(1) != w || v_out_rgb.size(2) != 3)
        AT_ERROR("v_out_rgb shape must be (h, w, 3)");

    torch::Tensor v_rgb = torch::empty({h, w, 3}, rgb.options());
    torch::Tensor v_alpha = torch::empty({h, w, 1}, alpha.options());
    torch::Tensor v_background = torch::empty({h, w, 3}, background.options());

    blend_background_backward_kernel<<<_LAUNGH_ARGS_1D(h*w)>>>(
        tensor2view<float, 3>(rgb), tensor2view<float, 3>(alpha), tensor2view<float, 3>(background),
        tensor2view<float, 3>(v_out_rgb),
        tensor2view<float, 3>(v_rgb), tensor2view<float, 3>(v_alpha), tensor2view<float, 3>(v_background)
    );

    return std::make_tuple(v_rgb, v_alpha, v_background);
}


std::tuple<torch::Tensor, torch::Tensor>
intersect_splat_tile(
    torch::Tensor& means,
    torch::Tensor& scales,
    torch::Tensor& opacs,
    torch::Tensor& quats,
    torch::Tensor& viewmats,
    torch::Tensor& Ks
) {
    SplatBuffers splat_buffers = {means, scales, opacs, quats};
    TileBuffers tile_buffers = {viewmats, Ks};

    return SplatTileIntersector(
        means.options(),
        splat_buffers,
        tile_buffers
    ).getIntersections_lbvh();
}
