#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "projection.cuh"
#include "rasterization.cuh"
#include "rasterization_sorted.cuh"
#include "sh.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <tuple>


inline __host__ float4 tuple2float4(std::tuple<float, float, float, float> v) {
    return {std::get<0>(v), std::get<1>(v), std::get<2>(v), std::get<3>(v)};
}

inline __host__ dim3 tuple2dim3(std::tuple<int, int, int> v) {
    return {std::get<0>(v), std::get<1>(v), std::get<2>(v)};
}


torch::Tensor compute_sh_forward_tensor(
    const std::string &method,
    const unsigned num_points,
    const unsigned degree,
    const unsigned degrees_to_use,
    torch::Tensor &viewdirs,  // [..., 3]
    torch::Tensor &coeffs0,   // [..., 3]
    torch::Tensor &coeffs   // [..., K, 3]
) {
    DEVICE_GUARD(viewdirs);
    unsigned num_bases = num_sh_bases(degree);
    if (coeffs0.ndimension() != 2 || coeffs0.size(0) != num_points ||
        coeffs0.size(1) != 3) {
        AT_ERROR("coeffs0 must have dimensions (N, 3)");
    }
    if (coeffs.ndimension() != 3 || coeffs.size(0) != num_points ||
        coeffs.size(1) != num_bases-1 || coeffs.size(2) != 3) {
        AT_ERROR("coeffs must have dimensions (N, D, 3)");
    }
    torch::Tensor colors = torch::empty({num_points, 3}, coeffs.options());    
    if (method == "poly") {
        compute_sh_forward_kernel<SHType::Poly><<<
            (num_points + N_THREADS - 1) / N_THREADS,
            N_THREADS>>>(
            num_points,
            degree,
            degrees_to_use,
            (float3 *)viewdirs.contiguous().data_ptr<float>(),
            coeffs0.contiguous().data_ptr<float>(),
            coeffs.contiguous().data_ptr<float>(),
            colors.contiguous().data_ptr<float>()
        );
    } else if (method == "fast") {
        compute_sh_forward_kernel<SHType::Fast><<<
            (num_points + N_THREADS - 1) / N_THREADS,
            N_THREADS>>>(
            num_points,
            degree,
            degrees_to_use,
            (float3 *)viewdirs.contiguous().data_ptr<float>(),
            coeffs0.contiguous().data_ptr<float>(),
            coeffs.contiguous().data_ptr<float>(),
            colors.contiguous().data_ptr<float>()
        );
    } else {
        AT_ERROR("Invalid method: ", method);
    }
    return colors;
}


std::tuple<torch::Tensor, torch::Tensor>
compute_sh_backward_tensor(
    const std::string &method,
    const unsigned num_points,
    const unsigned degree,
    const unsigned degrees_to_use,
    torch::Tensor &viewdirs,  // [..., 3]
    torch::Tensor &v_colors  // [..., 3]
) {
    DEVICE_GUARD(viewdirs);
    if (viewdirs.ndimension() != 2 || viewdirs.size(0) != num_points ||
        viewdirs.size(1) != 3) {
        AT_ERROR("viewdirs must have dimensions (N, 3)");
    }
    if (v_colors.ndimension() != 2 || v_colors.size(0) != num_points ||
        v_colors.size(1) != 3) {
        AT_ERROR("v_colors must have dimensions (N, 3)");
    }
    unsigned num_bases = num_sh_bases(degree);
    torch::Tensor v_coeffs0 =
        torch::zeros({num_points, 3}, v_colors.options());
    torch::Tensor v_coeffs =
        torch::zeros({num_points, num_bases-1, 3}, v_colors.options());
    if (method == "poly") {
        compute_sh_backward_kernel<SHType::Poly><<<
            (num_points + N_THREADS - 1) / N_THREADS,
            N_THREADS>>>(
            num_points,
            degree,
            degrees_to_use,
            (float3 *)viewdirs.contiguous().data_ptr<float>(),
            v_colors.contiguous().data_ptr<float>(),
            v_coeffs0.contiguous().data_ptr<float>(),
            v_coeffs.contiguous().data_ptr<float>()
        );
    } else if (method == "fast") {
        compute_sh_backward_kernel<SHType::Fast><<<
            (num_points + N_THREADS - 1) / N_THREADS,
            N_THREADS>>>(
            num_points,
            degree,
            degrees_to_use,
            (float3 *)viewdirs.contiguous().data_ptr<float>(),
            v_colors.contiguous().data_ptr<float>(),
            v_coeffs0.contiguous().data_ptr<float>(),
            v_coeffs.contiguous().data_ptr<float>()
        );
    } else {
        AT_ERROR("Invalid method: ", method);
    }
    return std::make_tuple(v_coeffs0, v_coeffs);
}


std::tuple<
    torch::Tensor,  // bounds, [N, 4], int
    torch::Tensor,  // num_tiles_hit, [N]
    torch::Tensor,  // positions, [N, 3]
    torch::Tensor,  // axes_u, [N, 3]
    torch::Tensor  // axes_v, [N, 4]
    // torch::Tensor  // depth_grads, [N, 2]
> project_gaussians_forward_tensor(
    const int num_points,
    torch::Tensor &means3d,  // [N, 3]
    torch::Tensor &scales,  // [N, 2]
    torch::Tensor &quats,  // [N, 4]
    torch::Tensor &viewmat,  // [3|4, 4]
    std::tuple<float, float, float, float> intrins,
    const unsigned img_height,
    const unsigned img_width,
    const unsigned block_width,
    const float clip_thresh
) {
    DEVICE_GUARD(means3d);

    dim3 tile_bounds_dim3;
    tile_bounds_dim3.x = int((img_width + block_width - 1) / block_width);
    tile_bounds_dim3.y = int((img_height + block_width - 1) / block_width);
    tile_bounds_dim3.z = 1;

    auto int32 = means3d.options().dtype(torch::kInt32);
    auto float32 = means3d.options().dtype(torch::kFloat32);
    torch::Tensor bounds_d = torch::zeros({num_points, 4}, int32);
    torch::Tensor num_tiles_hit_d = torch::zeros({num_points}, int32);
    torch::Tensor positions_d = torch::zeros({num_points, 3}, float32);
    torch::Tensor axes_u_d = torch::zeros({num_points, 3}, float32);
    torch::Tensor axes_v_d = torch::zeros({num_points, 3}, float32);
    // torch::Tensor depth_grads_d = torch::zeros({num_points, 2}, float32);

    project_gaussians_forward_kernel<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        (float3 *)means3d.contiguous().data_ptr<float>(),
        (float2 *)scales.contiguous().data_ptr<float>(),
        (float4 *)quats.contiguous().data_ptr<float>(),
        viewmat.contiguous().data_ptr<float>(),
        tuple2float4(intrins),
        tile_bounds_dim3,
        block_width,
        clip_thresh,
        // Outputs.
        (int4 *)bounds_d.contiguous().data_ptr<int32_t>(),
        num_tiles_hit_d.contiguous().data_ptr<int32_t>(),
        (float3 *)positions_d.contiguous().data_ptr<float>(),
        (float3 *)axes_u_d.contiguous().data_ptr<float>(),
        (float3 *)axes_v_d.contiguous().data_ptr<float>()
        // (float2 *)depth_grads_d.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        bounds_d, num_tiles_hit_d,
        positions_d, axes_u_d, axes_v_d
        // depth_grads_d
    );
}

std::tuple<
    torch::Tensor,  // v_means3d
    torch::Tensor,  // v_scales
    torch::Tensor,  // v_quats
    torch::Tensor  // v_viewmat, [4, 4]
> project_gaussians_backward_tensor(
    const int num_points,
    torch::Tensor &means3d,  // [N, 3]
    torch::Tensor &scales,  // [N, 2]
    torch::Tensor &quats,  // [N, 4]
    torch::Tensor &viewmat,  // [3|4, 4]
    std::tuple<float, float, float, float> intrins,
    torch::Tensor &num_tiles_hit,  // [N], int
    torch::Tensor &v_positions,  // [N, 3]
    torch::Tensor &v_axes_u,  // [N, 3]
    torch::Tensor &v_axes_v  // [N, 3]
    // torch::Tensor &v_depth_grads  // [N, 2]
) {
    DEVICE_GUARD(means3d);

    auto int32 = means3d.options().dtype(torch::kInt32);
    auto float32 = means3d.options().dtype(torch::kFloat32);
    torch::Tensor v_means3d = torch::zeros({num_points, 3}, float32);
    torch::Tensor v_scales = torch::zeros({num_points, 2}, float32);
    torch::Tensor v_quats = torch::zeros({num_points, 4}, float32);
    torch::Tensor v_viewmat = torch::zeros({4, 4}, float32);

    project_gaussians_backward_kernel<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        (float3 *)means3d.contiguous().data_ptr<float>(),
        (float2 *)scales.contiguous().data_ptr<float>(),
        (float4 *)quats.contiguous().data_ptr<float>(),
        viewmat.contiguous().data_ptr<float>(),
        tuple2float4(intrins),
        num_tiles_hit.contiguous().data_ptr<int32_t>(),
        (float3 *)v_positions.contiguous().data_ptr<float>(),
        (float3 *)v_axes_u.contiguous().data_ptr<float>(),
        (float3 *)v_axes_v.contiguous().data_ptr<float>(),
        // (float2 *)v_depth_grads.contiguous().data_ptr<float>(),
        // Outputs.
        (float3 *)v_means3d.contiguous().data_ptr<float>(),
        (float2 *)v_scales.contiguous().data_ptr<float>(),
        (float4 *)v_quats.contiguous().data_ptr<float>(),
        (float *)v_viewmat.contiguous().data_ptr<float>()
    );

    return std::make_tuple(v_means3d, v_scales, v_quats, v_viewmat);
}


std::tuple<
    torch::Tensor,
    torch::Tensor
> map_gaussian_to_intersects_tensor(
    const int num_points,
    const int num_intersects,
    const torch::Tensor &positions,
    const torch::Tensor &bounds,
    const torch::Tensor &cum_tiles_hit,
    const std::tuple<int, int, int> tile_bounds,
    const unsigned block_width
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(positions);
    CHECK_INPUT(bounds);
    CHECK_INPUT(cum_tiles_hit);

    dim3 tile_bounds_dim3 = tuple2dim3(tile_bounds);

    auto int32 = positions.options().dtype(torch::kInt32);
    auto int64 = positions.options().dtype(torch::kInt64);
    torch::Tensor gaussian_ids_unsorted =
        torch::zeros({num_intersects}, int32);
    torch::Tensor isect_ids_unsorted =
        torch::zeros({num_intersects}, int64);

    map_gaussian_to_intersects<<<
        (num_points + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_points,
        (float3 *)positions.contiguous().data_ptr<float>(),
        (int4 *)bounds.contiguous().data_ptr<int32_t>(),
        cum_tiles_hit.contiguous().data_ptr<int32_t>(),
        tile_bounds_dim3,
        block_width,
        // Outputs.
        isect_ids_unsorted.contiguous().data_ptr<int64_t>(),
        gaussian_ids_unsorted.contiguous().data_ptr<int32_t>()
    );

    return std::make_tuple(isect_ids_unsorted, gaussian_ids_unsorted);
}


torch::Tensor get_tile_bin_edges_tensor(
    int num_intersects, const torch::Tensor &isect_ids_sorted, 
    const std::tuple<int, int, int> tile_bounds
) {
    DEVICE_GUARD(isect_ids_sorted);
    CHECK_INPUT(isect_ids_sorted);
    int num_tiles = std::get<0>(tile_bounds) * std::get<1>(tile_bounds);
    torch::Tensor tile_bins = torch::zeros(
        {num_tiles, 2}, isect_ids_sorted.options().dtype(torch::kInt32)
    );
    get_tile_bin_edges<<<
        (num_intersects + N_THREADS - 1) / N_THREADS,
        N_THREADS>>>(
        num_intersects,
        isect_ids_sorted.contiguous().data_ptr<int64_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>()
    );
    return tile_bins;
}



std::tuple<
    torch::Tensor,  // new_opacities
    torch::Tensor  // new_scales
> compute_relocation_tensor(
    torch::Tensor &opacities,
    torch::Tensor &scales,
    torch::Tensor &ratios,
    torch::Tensor &binoms,
    const int n_max
) {
    DEVICE_GUARD(opacities);
    CHECK_INPUT(opacities);
    CHECK_INPUT(scales);
    CHECK_INPUT(ratios);
    CHECK_INPUT(binoms);
    torch::Tensor new_opacities = torch::empty_like(opacities);
    torch::Tensor new_scales = torch::empty_like(scales);

    uint32_t N = opacities.size(0);
    uint32_t num_scales = scales.size(1);
    if (N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        compute_relocation_kernel<<<(N + N_THREADS - 1) / N_THREADS, N_THREADS, 0, stream>>>(
            (int)N, (int)num_scales,
            opacities.data_ptr<float>(),
            scales.data_ptr<float>(),
            ratios.data_ptr<int>(),
            binoms.data_ptr<float>(),
            n_max,
            new_opacities.data_ptr<float>(),
            new_scales.data_ptr<float>()
        );
    }
    return std::make_tuple(new_opacities, new_scales);
}

std::tuple<
    torch::Tensor,  // new_position_offsets
    torch::Tensor,  // new_opacities
    torch::Tensor  // new_scales
> compute_relocation_split_tensor(
    torch::Tensor &positions,
    torch::Tensor &quats,
    torch::Tensor &opacities,
    torch::Tensor &scales
) {
    DEVICE_GUARD(opacities);
    CHECK_INPUT(opacities);
    CHECK_INPUT(scales);
    CHECK_INPUT(positions);
    CHECK_INPUT(quats);

    torch::Tensor new_position_offsets = torch::empty_like(positions);
    torch::Tensor new_opacities = torch::empty_like(opacities);
    torch::Tensor new_scales = torch::empty_like(scales);

    uint32_t N = opacities.size(0);
    uint32_t num_scales = scales.size(1);
    if (N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        compute_relocation_split_kernel<<<(N + N_THREADS - 1) / N_THREADS, N_THREADS, 0, stream>>>(
            (int)N,
            (float3*)positions.data_ptr<float>(),
            (float4*)quats.data_ptr<float>(),
            opacities.data_ptr<float>(),
            (float2*)scales.data_ptr<float>(),
            (float3*)new_position_offsets.data_ptr<float>(),
            new_opacities.data_ptr<float>(),
            (float2*)new_scales.data_ptr<float>()
        );
    }
    return std::make_tuple(
        new_position_offsets,
        new_opacities, new_scales
    );
}





std::tuple<
    torch::Tensor,  // final_index
    torch::Tensor,  // out_img
    torch::Tensor  // out_alpha
> rasterize_simple_forward_tensor(
    const std::tuple<int, int, int> tile_bounds,
    const std::tuple<int, int, int> block,
    const std::tuple<int, int, int> img_size,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &gaussian_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const torch::Tensor &opacities,
    const torch::Tensor &background
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(gaussian_ids_sorted);
    CHECK_INPUT(tile_bins);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(background);

    dim3 tile_bounds_dim3 = tuple2dim3(tile_bounds);
    dim3 block_dim3 = tuple2dim3(block);
    dim3 img_size_dim3 = tuple2dim3(img_size);

    const int channels = colors.size(1);
    const int img_width = img_size_dim3.x;
    const int img_height = img_size_dim3.y;

    auto int32 = positions.options().dtype(torch::kInt32);
    auto float32 = positions.options().dtype(torch::kFloat32);
    torch::Tensor final_idx = torch::zeros(
        {img_height, img_width}, int32
    );
    torch::Tensor out_img = torch::zeros(
        {img_height, img_width, channels}, float32
    );
    torch::Tensor out_alpha = torch::zeros(
        {img_height, img_width, 1}, float32
    );

    rasterize_simple_forward_kernel<<<tile_bounds_dim3, block_dim3>>>(
        tile_bounds_dim3,
        img_size_dim3,
        tuple2float4(intrins),
        gaussian_ids_sorted.contiguous().data_ptr<int32_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        *(float3 *)background.contiguous().data_ptr<float>(),
        // outputs
        final_idx.contiguous().data_ptr<int>(),
        (float3 *)out_img.contiguous().data_ptr<float>(),
        out_alpha.contiguous().data_ptr<float>()
    );

    return std::make_tuple(final_idx, out_img, out_alpha);
}

std::tuple<
    torch::Tensor, // v_positions
    torch::Tensor, // v_positions_xy_abs
    torch::Tensor, // v_axes_u
    torch::Tensor, // v_axes_v
    torch::Tensor, // v_colors
    torch::Tensor // v_opacities
> rasterize_simple_backward_tensor(
    const unsigned img_height,
    const unsigned img_width,
    const unsigned block_width,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &gaussians_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const torch::Tensor &opacities,
    const torch::Tensor &background,
    const torch::Tensor &final_idx,
    const torch::Tensor &output_alpha,
    const torch::Tensor &v_output,
    const torch::Tensor &v_output_alpha
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(final_idx);
    CHECK_INPUT(output_alpha);
    CHECK_INPUT(v_output);
    CHECK_INPUT(v_output_alpha);

    if (positions.ndimension() != 2 || positions.size(1) != 3) {
        AT_ERROR("positions must have dimensions (num_points, 3)");
    }

    if (colors.ndimension() != 2 || colors.size(1) != 3) {
        AT_ERROR("colors must have 2 dimensions");
    }

    const int num_points = positions.size(0);
    const dim3 tile_bounds = {
        (img_width + block_width - 1) / block_width,
        (img_height + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {img_width, img_height, 1};
    const int channels = colors.size(1);

    auto options = positions.options();
    torch::Tensor v_positions = torch::zeros({num_points, 3}, options);
    torch::Tensor v_positions_xy_abs = torch::zeros({num_points, 2}, options);
    torch::Tensor v_axes_u = torch::zeros({num_points, 3}, options);
    torch::Tensor v_axes_v = torch::zeros({num_points, 3}, options);
    torch::Tensor v_colors = torch::zeros({num_points, channels}, options);
    torch::Tensor v_opacities = torch::zeros({num_points, 1}, options);

    rasterize_simple_backward_kernel<<<tile_bounds, block>>>(
        tile_bounds,
        img_size,
        tuple2float4(intrins),
        gaussians_ids_sorted.contiguous().data_ptr<int>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        *(float3 *)background.contiguous().data_ptr<float>(),
        final_idx.contiguous().data_ptr<int>(),
        output_alpha.contiguous().data_ptr<float>(),
        (float3 *)v_output.contiguous().data_ptr<float>(),
        v_output_alpha.contiguous().data_ptr<float>(),
        // outputs
        (float3 *)v_positions.contiguous().data_ptr<float>(),
        (float2 *)v_positions_xy_abs.contiguous().data_ptr<float>(),
        (float3 *)v_axes_u.contiguous().data_ptr<float>(),
        (float3 *)v_axes_v.contiguous().data_ptr<float>(),
        (float3 *)v_colors.contiguous().data_ptr<float>(),
        v_opacities.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        v_positions, v_positions_xy_abs,
        v_axes_u, v_axes_v,
        v_colors, v_opacities
    );
}



std::tuple<
    torch::Tensor,  // final_index
    torch::Tensor,  // out_img
    torch::Tensor  // out_visibility
> rasterize_depth_forward_tensor(
    const int depth_mode,
    const std::tuple<int, int, int> tile_bounds,
    const std::tuple<int, int, int> block,
    const std::tuple<int, int, int> img_size,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &gaussian_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &opacities
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(gaussian_ids_sorted);
    CHECK_INPUT(tile_bins);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(opacities);

    dim3 tile_bounds_dim3 = tuple2dim3(tile_bounds);
    dim3 block_dim3 = tuple2dim3(block);
    dim3 img_size_dim3 = tuple2dim3(img_size);

    const int img_width = img_size_dim3.x;
    const int img_height = img_size_dim3.y;

    auto int32 = positions.options().dtype(torch::kInt32);
    auto float32 = positions.options().dtype(torch::kFloat32);
    torch::Tensor final_idx = torch::zeros(
        {img_height, img_width}, int32
    );
    torch::Tensor out_depth = torch::zeros(
        {img_height, img_width, 1}, float32
    );
    torch::Tensor out_visibility = torch::zeros(
        {img_height, img_width, 2}, float32
    );

    rasterize_depth_forward_kernel<<<tile_bounds_dim3, block_dim3>>>(
        depth_mode,
        tile_bounds_dim3,
        img_size_dim3,
        tuple2float4(intrins),
        gaussian_ids_sorted.contiguous().data_ptr<int32_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        // outputs
        final_idx.contiguous().data_ptr<int>(),
        out_depth.contiguous().data_ptr<float>(),
        (float2 *)out_visibility.contiguous().data_ptr<float>()
    );

    return std::make_tuple(final_idx, out_depth, out_visibility);
}

std::tuple<
    torch::Tensor, // v_positions
    torch::Tensor, // v_positions_xy_abs
    torch::Tensor, // v_axes_u
    torch::Tensor, // v_axes_v
    torch::Tensor // v_opacities
> rasterize_depth_backward_tensor(
    const int depth_mode,
    const unsigned img_height,
    const unsigned img_width,
    const unsigned block_width,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &gaussians_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &opacities,
    const torch::Tensor &final_idx,
    const torch::Tensor &output_depth,
    const torch::Tensor &output_visibility,
    const torch::Tensor &v_output_depth
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(opacities);
    CHECK_INPUT(final_idx);
    CHECK_INPUT(output_depth);
    CHECK_INPUT(output_visibility);
    CHECK_INPUT(v_output_depth);

    if (positions.ndimension() != 2 || positions.size(1) != 3) {
        AT_ERROR("positions must have dimensions (num_points, 3)");
    }
    if (output_visibility.ndimension() != 3 || output_visibility.size(2) != 2) {
        AT_ERROR("output_visibility must have dimensions (h, w, 2)");
    }

    const int num_points = positions.size(0);
    const dim3 tile_bounds = {
        (img_width + block_width - 1) / block_width,
        (img_height + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {img_width, img_height, 1};

    auto options = positions.options();
    torch::Tensor v_positions = torch::zeros({num_points, 3}, options);
    torch::Tensor v_positions_xy_abs = torch::zeros({num_points, 2}, options);
    torch::Tensor v_axes_u = torch::zeros({num_points, 3}, options);
    torch::Tensor v_axes_v = torch::zeros({num_points, 3}, options);
    torch::Tensor v_opacities = torch::zeros({num_points, 1}, options);

    rasterize_depth_backward_kernel<<<tile_bounds, block>>>(
        depth_mode,
        tile_bounds,
        img_size,
        tuple2float4(intrins),
        gaussians_ids_sorted.contiguous().data_ptr<int>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        final_idx.contiguous().data_ptr<int>(),
        output_depth.contiguous().data_ptr<float>(),
        (float2 *)output_visibility.contiguous().data_ptr<float>(),
        v_output_depth.contiguous().data_ptr<float>(),
        // outputs
        (float3 *)v_positions.contiguous().data_ptr<float>(),
        (float2 *)v_positions_xy_abs.contiguous().data_ptr<float>(),
        (float3 *)v_axes_u.contiguous().data_ptr<float>(),
        (float3 *)v_axes_v.contiguous().data_ptr<float>(),
        v_opacities.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        v_positions, v_positions_xy_abs,
        v_axes_u, v_axes_v,
        v_opacities
    );
}



std::tuple<
    torch::Tensor, // final_idx
    torch::Tensor,  // out_alpha
    torch::Tensor,  // out_img
    torch::Tensor,  // out_depth
    torch::Tensor,  // out_normal
    torch::Tensor  // out_reg_depth
> rasterize_forward_tensor(
    const std::tuple<int, int, int> tile_bounds,
    const std::tuple<int, int, int> block,
    const std::tuple<int, int, int> img_size,
    const std::tuple<float, float, float, float> intrins,
    const float depth_reg_pairwise_factor,
    const torch::Tensor &gaussian_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const torch::Tensor &ch_coeffs,
    const torch::Tensor &opacities,
    // const torch::Tensor &background,
    const torch::Tensor &depth_ref_im
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(gaussian_ids_sorted);
    CHECK_INPUT(tile_bins);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(ch_coeffs);
    CHECK_INPUT(opacities);
    // CHECK_INPUT(background);
    CHECK_INPUT(depth_ref_im);

    dim3 tile_bounds_dim3 = tuple2dim3(tile_bounds);
    dim3 block_dim3 = tuple2dim3(block);
    dim3 img_size_dim3 = tuple2dim3(img_size);

    const int channels = colors.size(1);
    const int img_width = img_size_dim3.x;
    const int img_height = img_size_dim3.y;

    auto int32 = positions.options().dtype(torch::kInt32);
    auto float32 = positions.options().dtype(torch::kFloat32);
    torch::Tensor final_idx = torch::zeros(
        {img_height, img_width}, int32
    );
    torch::Tensor out_alpha = torch::zeros(
        {img_height, img_width, 1}, float32
    );
    torch::Tensor out_img = torch::zeros(
        {img_height, img_width, channels}, float32
    );
    torch::Tensor out_depth = torch::zeros(
        {img_height, img_width, 2}, float32
    );
    torch::Tensor out_normal = torch::zeros(
        {img_height, img_width, 3}, float32
    );
    torch::Tensor out_reg_depth = torch::zeros(
        {img_height, img_width, 1}, float32
    );

    rasterize_forward_kernel<<<tile_bounds_dim3, block_dim3>>>(
        tile_bounds_dim3,
        img_size_dim3,
        tuple2float4(intrins),
        depth_reg_pairwise_factor,
        gaussian_ids_sorted.contiguous().data_ptr<int32_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        (unsigned)ch_degree_r, (unsigned)ch_degree_r_to_use,
        (unsigned)ch_degree_phi, (unsigned)ch_degree_phi_to_use,
        (float3 *)ch_coeffs.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        // *(float3 *)background.contiguous().data_ptr<float>(),
        depth_ref_im.contiguous().data_ptr<float>(),
        // outputs
        final_idx.contiguous().data_ptr<int>(),
        out_alpha.contiguous().data_ptr<float>(),
        (float3 *)out_img.contiguous().data_ptr<float>(),
        (float2 *)out_depth.contiguous().data_ptr<float>(),
        (float3 *)out_normal.contiguous().data_ptr<float>(),
        out_reg_depth.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        final_idx, out_alpha,
        out_img, out_depth,
        out_normal, out_reg_depth
    );
}


std::tuple<
    torch::Tensor, // v_positions
    torch::Tensor, // v_positions_xy_abs
    torch::Tensor, // v_axes_u
    torch::Tensor, // v_axes_v
    torch::Tensor, // v_colors
    torch::Tensor, // v_ch_coeffs
    // torch::Tensor, // v_ch_coeffs_abs
    torch::Tensor, // v_opacities
    // torch::Tensor, // v_background
    torch::Tensor  // v_depth_ref_im
> rasterize_backward_tensor(
    const unsigned img_height,
    const unsigned img_width,
    const unsigned block_width,
    const std::tuple<float, float, float, float> intrins,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float depth_reg_pairwise_factor,
    const torch::Tensor &gaussians_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const torch::Tensor &ch_coeffs,
    const torch::Tensor &opacities,
    // const torch::Tensor &background,
    const torch::Tensor &depth_ref_im,
    const torch::Tensor &final_idx,
    const torch::Tensor &output_alpha,
    const torch::Tensor &output_depth,
    const torch::Tensor &v_output_alpha,
    const torch::Tensor &v_output,
    const torch::Tensor &v_output_depth,
    const torch::Tensor &v_output_normal,
    const torch::Tensor &v_output_reg_depth
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(ch_coeffs);
    CHECK_INPUT(opacities);
    CHECK_INPUT(final_idx);
    CHECK_INPUT(output_alpha);
    CHECK_INPUT(output_depth);
    CHECK_INPUT(v_output_alpha);
    CHECK_INPUT(v_output);
    CHECK_INPUT(v_output_depth);
    CHECK_INPUT(v_output_normal);
    CHECK_INPUT(v_output_reg_depth);

    if (positions.ndimension() != 2 || positions.size(1) != 3) {
        AT_ERROR("xys must have dimensions (num_points, 2)");
    }
    if (colors.ndimension() != 2 || colors.size(1) != 3) {
        AT_ERROR("colors must have 2 dimensions");
    }
    if (output_depth.ndimension() != 3 || output_depth.size(2) != 2) {
        AT_ERROR("output_depth_grad must have 3 dimensions");
    }
    if (v_output_depth.ndimension() != 3 || v_output_depth.size(2) != 2) {
        AT_ERROR("v_output_depth must have 3 dimensions");
    }
    if (depth_ref_im.ndimension() != 3 || depth_ref_im.size(2) != 1) {
        AT_ERROR("depth_ref_im must have 3 dimensions");
    }

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);

    const int num_points = positions.size(0);
    const dim3 tile_bounds = {
        (img_width + block_width - 1) / block_width,
        (img_height + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {img_width, img_height, 1};
    const int channels = colors.size(1);

    auto options = positions.options();
    torch::Tensor v_positions = torch::zeros({num_points, 3}, options);
    torch::Tensor v_positions_xy_abs = torch::zeros({num_points, 2}, options);
    torch::Tensor v_axes_u = torch::zeros({num_points, 3}, options);
    torch::Tensor v_axes_v = torch::zeros({num_points, 3}, options);
    torch::Tensor v_colors = torch::zeros({num_points, channels}, options);
    torch::Tensor v_ch_coeffs = torch::zeros({num_points, dim_ch, channels}, options);
    // torch::Tensor v_ch_coeffs_abs = torch::zeros({num_points, 1}, options);
    torch::Tensor v_opacities = torch::zeros({num_points, 1}, options);
    // torch::Tensor v_background = torch::zeros({3}, options);
    torch::Tensor v_depth_ref_im = torch::zeros({img_height, img_width, 1}, options);

    rasterize_backward_kernel<<<tile_bounds, block>>>(
        tile_bounds, img_size, tuple2float4(intrins),
        (unsigned)ch_degree_r, (unsigned)ch_degree_r_to_use,
        (unsigned)ch_degree_phi, (unsigned)ch_degree_phi_to_use,
        depth_reg_pairwise_factor,
        gaussians_ids_sorted.contiguous().data_ptr<int>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        (float3 *)ch_coeffs.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        // *(float3 *)background.contiguous().data_ptr<float>(),
        depth_ref_im.contiguous().data_ptr<float>(),
        final_idx.contiguous().data_ptr<int>(),
        output_alpha.contiguous().data_ptr<float>(),
        (float2 *)output_depth.contiguous().data_ptr<float>(),
        v_output_alpha.contiguous().data_ptr<float>(),
        (float3 *)v_output.contiguous().data_ptr<float>(),
        (float2 *)v_output_depth.contiguous().data_ptr<float>(),
        (float3 *)v_output_normal.contiguous().data_ptr<float>(),
        v_output_reg_depth.contiguous().data_ptr<float>(),
        // outputs
        (float3 *)v_positions.contiguous().data_ptr<float>(),
        (float2 *)v_positions_xy_abs.contiguous().data_ptr<float>(),
        (float3 *)v_axes_u.contiguous().data_ptr<float>(),
        (float3 *)v_axes_v.contiguous().data_ptr<float>(),
        (float3 *)v_colors.contiguous().data_ptr<float>(),
        (float3 *)v_ch_coeffs.contiguous().data_ptr<float>(),
        // v_ch_coeffs_abs.contiguous().data_ptr<float>(),
        v_opacities.contiguous().data_ptr<float>(),
        // (float3 *)v_background.contiguous().data_ptr<float>(),
        v_depth_ref_im.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        v_positions, v_positions_xy_abs,
        v_axes_u, v_axes_v,
        v_colors, v_ch_coeffs, //v_ch_coeffs_abs,
        v_opacities,
        // v_background,
        v_depth_ref_im
    );
}



std::tuple<
    torch::Tensor, // final_idx
    torch::Tensor,  // out_alpha
    torch::Tensor,  // out_img
    torch::Tensor,  // out_depth
    torch::Tensor,  // out_normal
    torch::Tensor  // out_depth_reg
> rasterize_simplified_forward_tensor(
    const std::tuple<int, int, int> tile_bounds,
    const std::tuple<int, int, int> block,
    const std::tuple<int, int, int> img_size,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &gaussian_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const torch::Tensor &opacities
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(gaussian_ids_sorted);
    CHECK_INPUT(tile_bins);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);

    dim3 tile_bounds_dim3 = tuple2dim3(tile_bounds);
    dim3 block_dim3 = tuple2dim3(block);
    dim3 img_size_dim3 = tuple2dim3(img_size);

    const int channels = colors.size(1);
    const int img_width = img_size_dim3.x;
    const int img_height = img_size_dim3.y;

    auto int32 = positions.options().dtype(torch::kInt32);
    auto float32 = positions.options().dtype(torch::kFloat32);
    torch::Tensor final_idx = torch::zeros(
        {img_height, img_width}, int32
    );
    torch::Tensor out_alpha = torch::zeros(
        {img_height, img_width, 1}, float32
    );
    torch::Tensor out_img = torch::zeros(
        {img_height, img_width, channels}, float32
    );
    torch::Tensor out_depth = torch::zeros(
        {img_height, img_width, 2}, float32
    );
    torch::Tensor out_normal = torch::zeros(
        {img_height, img_width, 3}, float32
    );
    torch::Tensor out_depth_reg = torch::zeros(
        {img_height, img_width, 1}, float32
    );

    rasterize_simplified_forward_kernel<<<tile_bounds_dim3, block_dim3>>>(
        tile_bounds_dim3,
        img_size_dim3,
        tuple2float4(intrins),
        gaussian_ids_sorted.contiguous().data_ptr<int32_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        // outputs
        final_idx.contiguous().data_ptr<int>(),
        out_alpha.contiguous().data_ptr<float>(),
        (float3 *)out_img.contiguous().data_ptr<float>(),
        (float2 *)out_depth.contiguous().data_ptr<float>(),
        (float3 *)out_normal.contiguous().data_ptr<float>(),
        out_depth_reg.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        final_idx, out_alpha,
        out_img,
        out_depth, out_normal, out_depth_reg
    );
}


std::tuple<
    torch::Tensor, // v_positions
    torch::Tensor, // v_positions_xy_abs
    torch::Tensor, // v_axes_u
    torch::Tensor, // v_axes_v
    torch::Tensor, // v_colors
    torch::Tensor // v_opacities
> rasterize_simplified_backward_tensor(
    const unsigned img_height,
    const unsigned img_width,
    const unsigned block_width,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &gaussians_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const torch::Tensor &opacities,
    const torch::Tensor &final_idx,
    const torch::Tensor &output_alpha,
    const torch::Tensor &output_depth,
    const torch::Tensor &v_output_alpha,
    const torch::Tensor &v_output_img,
    const torch::Tensor &v_output_depth,
    const torch::Tensor &v_output_normal,
    const torch::Tensor &v_output_depth_reg
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(final_idx);
    CHECK_INPUT(output_alpha);
    CHECK_INPUT(output_depth);
    CHECK_INPUT(v_output_alpha);
    CHECK_INPUT(v_output_img);
    CHECK_INPUT(v_output_depth);
    CHECK_INPUT(v_output_normal);
    CHECK_INPUT(v_output_depth_reg);

    if (positions.ndimension() != 2 || positions.size(1) != 3) {
        AT_ERROR("xys must have dimensions (num_points, 2)");
    }
    if (colors.ndimension() != 2 || colors.size(1) != 3) {
        AT_ERROR("colors must have 2 dimensions");
    }
    if (v_output_depth.ndimension() != 3 || v_output_depth.size(2) != 2) {
        AT_ERROR("v_output_depth must have 3 dimensions");
    }
    if (v_output_normal.ndimension() != 3 || v_output_normal.size(2) != 3) {
        AT_ERROR("v_output_normal must have 3 dimensions");
    }
    if (v_output_depth_reg.ndimension() != 3 || v_output_depth_reg.size(2) != 1) {
        AT_ERROR("v_output_depth_reg must have 3 dimensions");
    }

    const int num_points = positions.size(0);
    const dim3 tile_bounds = {
        (img_width + block_width - 1) / block_width,
        (img_height + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {img_width, img_height, 1};
    const int channels = colors.size(1);

    auto options = positions.options();
    torch::Tensor v_positions = torch::zeros({num_points, 3}, options);
    torch::Tensor v_positions_xy_abs = torch::zeros({num_points, 2}, options);
    torch::Tensor v_axes_u = torch::zeros({num_points, 3}, options);
    torch::Tensor v_axes_v = torch::zeros({num_points, 3}, options);
    torch::Tensor v_colors = torch::zeros({num_points, channels}, options);
    torch::Tensor v_opacities = torch::zeros({num_points, 1}, options);

    rasterize_simplified_backward_kernel<<<tile_bounds, block>>>(
        tile_bounds, img_size, tuple2float4(intrins),
        gaussians_ids_sorted.contiguous().data_ptr<int>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        final_idx.contiguous().data_ptr<int>(),
        output_alpha.contiguous().data_ptr<float>(),
        (float2 *)output_depth.contiguous().data_ptr<float>(),
        v_output_alpha.contiguous().data_ptr<float>(),
        (float3 *)v_output_img.contiguous().data_ptr<float>(),
        (float2 *)v_output_depth.contiguous().data_ptr<float>(),
        (float3 *)v_output_normal.contiguous().data_ptr<float>(),
        v_output_depth_reg.contiguous().data_ptr<float>(),
        // outputs
        (float3 *)v_positions.contiguous().data_ptr<float>(),
        (float2 *)v_positions_xy_abs.contiguous().data_ptr<float>(),
        (float3 *)v_axes_u.contiguous().data_ptr<float>(),
        (float3 *)v_axes_v.contiguous().data_ptr<float>(),
        (float3 *)v_colors.contiguous().data_ptr<float>(),
        v_opacities.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        v_positions, v_positions_xy_abs,
        v_axes_u, v_axes_v,
        v_colors, v_opacities
    );
}



std::tuple<
    torch::Tensor,  // num_intersects
    torch::Tensor,  // sorted_indices
    torch::Tensor  // sorted_depths
> rasterize_indices_tensor(
    const std::tuple<int, int, int> tile_bounds,
    const std::tuple<int, int, int> block,
    const std::tuple<int, int, int> img_size,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &gaussian_ids_sorted,
    const torch::Tensor &tile_bins,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &opacities
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(gaussian_ids_sorted);
    CHECK_INPUT(tile_bins);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(opacities);

    dim3 tile_bounds_dim3 = tuple2dim3(tile_bounds);
    dim3 block_dim3 = tuple2dim3(block);
    dim3 img_size_dim3 = tuple2dim3(img_size);

    const int img_width = img_size_dim3.x;
    const int img_height = img_size_dim3.y;

    auto int32 = positions.options().dtype(torch::kInt32);
    auto float32 = positions.options().dtype(torch::kFloat32);
    torch::Tensor num_intersects = torch::empty(
        {img_height, img_width}, int32
    );
    torch::Tensor sorted_indices = torch::empty(
        {img_height, img_width, MAX_SORTED_SPLATS}, int32
    );
    torch::Tensor sorted_depths = torch::empty(
        {img_height, img_width, MAX_SORTED_SPLATS}, float32
    );

    rasterize_indices_kernel<<<tile_bounds_dim3, block_dim3>>>(
        tile_bounds_dim3,
        img_size_dim3,
        tuple2float4(intrins),
        gaussian_ids_sorted.contiguous().data_ptr<int32_t>(),
        (int2 *)tile_bins.contiguous().data_ptr<int>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        // outputs
        num_intersects.contiguous().data_ptr<int>(),
        sorted_indices.contiguous().data_ptr<int32_t>(),
        sorted_depths.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        num_intersects, sorted_indices, sorted_depths
    );
}


void sort_per_pixel_tensor(
    const std::string &method,
    const std::tuple<int, int, int> tile_bounds,
    const std::tuple<int, int, int> block,
    const std::tuple<int, int, int> img_size,
    torch::Tensor &num_intersects,  // [h, w]
    torch::Tensor &indices,  // [h, w, MAX_SORTED_SPLATS]
    torch::Tensor &depths  // [h, w, MAX_SORTED_SPLATS]
) {
    DEVICE_GUARD(num_intersects);
    if (indices.ndimension() != 3 || indices.size(2) != MAX_SORTED_SPLATS) {
        AT_ERROR("indices must have dimensions (h, w, MAX_SORTED_SPLATS)");
    }
    if (depths.ndimension() != 3 || depths.size(2) != MAX_SORTED_SPLATS) {
        AT_ERROR("depths must have dimensions (h, w, MAX_SORTED_SPLATS)");
    }

    dim3 tile_bounds_dim3 = tuple2dim3(tile_bounds);
    dim3 block_dim3 = tuple2dim3(block);
    dim3 img_size_dim3 = tuple2dim3(img_size);

    if (method == "insertion") {
        sort_per_pixel_kernel<PerPixelSortType::InsertionSort>
        <<<tile_bounds_dim3, block_dim3>>>(
            tile_bounds_dim3, img_size_dim3,
            num_intersects.contiguous().data_ptr<int>(),
            indices.contiguous().data_ptr<int32_t>(),
            depths.contiguous().data_ptr<float>()
        );
    } else if (method == "quick") {
        sort_per_pixel_kernel<PerPixelSortType::QuickSort>
        <<<tile_bounds_dim3, block_dim3>>>(
            tile_bounds_dim3, img_size_dim3,
            num_intersects.contiguous().data_ptr<int>(),
            indices.contiguous().data_ptr<int32_t>(),
            depths.contiguous().data_ptr<float>()
        );
    } else if (method == "heap") {
        sort_per_pixel_kernel<PerPixelSortType::HeapSort>
        <<<tile_bounds_dim3, block_dim3>>>(
            tile_bounds_dim3, img_size_dim3,
            num_intersects.contiguous().data_ptr<int>(),
            indices.contiguous().data_ptr<int32_t>(),
            depths.contiguous().data_ptr<float>()
        );
    } else if (method == "random_quick") {
        sort_per_pixel_kernel<PerPixelSortType::RandomizedQuickSort>
        <<<tile_bounds_dim3, block_dim3>>>(
            tile_bounds_dim3, img_size_dim3,
            num_intersects.contiguous().data_ptr<int>(),
            indices.contiguous().data_ptr<int32_t>(),
            depths.contiguous().data_ptr<float>()
        );
    } else {
        AT_ERROR("Invalid sorting method: ", method);
    }
}



std::tuple<
    torch::Tensor,  // out_img
    torch::Tensor  // out_alpha
> rasterize_simple_sorted_forward_tensor(
    const std::tuple<int, int, int> tile_bounds,
    const std::tuple<int, int, int> block,
    const std::tuple<int, int, int> img_size,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &sorted_indices,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const torch::Tensor &opacities,
    const torch::Tensor &background
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(sorted_indices);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(background);

    dim3 tile_bounds_dim3 = tuple2dim3(tile_bounds);
    dim3 block_dim3 = tuple2dim3(block);
    dim3 img_size_dim3 = tuple2dim3(img_size);

    const int channels = colors.size(1);
    const int img_width = img_size_dim3.x;
    const int img_height = img_size_dim3.y;

    auto int32 = positions.options().dtype(torch::kInt32);
    auto float32 = positions.options().dtype(torch::kFloat32);
    torch::Tensor final_idx = torch::zeros(
        {img_height, img_width}, int32
    );
    torch::Tensor out_img = torch::zeros(
        {img_height, img_width, channels}, float32
    );
    torch::Tensor out_alpha = torch::zeros(
        {img_height, img_width, 1}, float32
    );

    rasterize_simple_sorted_forward_kernel<<<tile_bounds_dim3, block_dim3>>>(
        img_size_dim3,
        tuple2float4(intrins),
        sorted_indices.contiguous().data_ptr<int32_t>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        *(float3 *)background.contiguous().data_ptr<float>(),
        // outputs
        (float3 *)out_img.contiguous().data_ptr<float>(),
        out_alpha.contiguous().data_ptr<float>()
    );

    return std::make_tuple(out_img, out_alpha);
}


std::tuple<
    torch::Tensor, // v_positions
    torch::Tensor, // v_positions_xy_abs
    torch::Tensor, // v_axes_u
    torch::Tensor, // v_axes_v
    torch::Tensor, // v_colors
    torch::Tensor // v_opacities
> rasterize_simple_sorted_backward_tensor(
    const unsigned img_height,
    const unsigned img_width,
    const unsigned block_width,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &num_intersects,
    const torch::Tensor &sorted_indices,
    const torch::Tensor &positions,
    const torch::Tensor &axes_u,
    const torch::Tensor &axes_v,
    const torch::Tensor &colors,
    const torch::Tensor &opacities,
    const torch::Tensor &background,
    const torch::Tensor &output_alpha,
    const torch::Tensor &v_output,
    const torch::Tensor &v_output_alpha
) {
    DEVICE_GUARD(positions);
    CHECK_INPUT(num_intersects);
    CHECK_INPUT(sorted_indices);
    CHECK_INPUT(positions);
    CHECK_INPUT(axes_u);
    CHECK_INPUT(axes_v);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(output_alpha);
    CHECK_INPUT(v_output);
    CHECK_INPUT(v_output_alpha);

    if (positions.ndimension() != 2 || positions.size(1) != 3) {
        AT_ERROR("positions must have dimensions (num_points, 3)");
    }

    if (colors.ndimension() != 2 || colors.size(1) != 3) {
        AT_ERROR("colors must have 2 dimensions");
    }

    const int num_points = positions.size(0);
    const dim3 tile_bounds = {
        (img_width + block_width - 1) / block_width,
        (img_height + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {img_width, img_height, 1};
    const int channels = colors.size(1);

    auto options = positions.options();
    torch::Tensor v_positions = torch::zeros({num_points, 3}, options);
    torch::Tensor v_positions_xy_abs = torch::zeros({num_points, 2}, options);
    torch::Tensor v_axes_u = torch::zeros({num_points, 3}, options);
    torch::Tensor v_axes_v = torch::zeros({num_points, 3}, options);
    torch::Tensor v_colors = torch::zeros({num_points, channels}, options);
    torch::Tensor v_opacities = torch::zeros({num_points, 1}, options);

    rasterize_simple_sorted_backward_kernel<<<tile_bounds, block>>>(
        img_size,
        tuple2float4(intrins),
        num_intersects.contiguous().data_ptr<int>(),
        sorted_indices.contiguous().data_ptr<int32_t>(),
        (float3 *)positions.contiguous().data_ptr<float>(),
        (float3 *)axes_u.contiguous().data_ptr<float>(),
        (float3 *)axes_v.contiguous().data_ptr<float>(),
        (float3 *)colors.contiguous().data_ptr<float>(),
        opacities.contiguous().data_ptr<float>(),
        *(float3 *)background.contiguous().data_ptr<float>(),
        output_alpha.contiguous().data_ptr<float>(),
        (float3 *)v_output.contiguous().data_ptr<float>(),
        v_output_alpha.contiguous().data_ptr<float>(),
        // outputs
        (float3 *)v_positions.contiguous().data_ptr<float>(),
        (float2 *)v_positions_xy_abs.contiguous().data_ptr<float>(),
        (float3 *)v_axes_u.contiguous().data_ptr<float>(),
        (float3 *)v_axes_v.contiguous().data_ptr<float>(),
        (float3 *)v_colors.contiguous().data_ptr<float>(),
        v_opacities.contiguous().data_ptr<float>()
    );

    return std::make_tuple(
        v_positions, v_positions_xy_abs,
        v_axes_u, v_axes_v,
        v_colors, v_opacities
    );
}





torch::Tensor render_background_sh_forward_tensor(
    const unsigned w,
    const unsigned h,
    const unsigned block_width,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &rotation,
    const unsigned sh_degree,
    const torch::Tensor &sh_coeffs
) {
    DEVICE_GUARD(sh_coeffs);
    CHECK_INPUT(sh_coeffs);
    CHECK_INPUT(rotation);

    if (rotation.numel() != 9) {
        AT_ERROR("rotation must be 3x3");
    }
    if (sh_coeffs.ndimension() != 2 ||
        sh_coeffs.size(0) != sh_degree*sh_degree ||
        sh_coeffs.size(1) != 3) {
        AT_ERROR("sh_coeffs must be (sh_regree**2, 3)");
    }

    const dim3 tile_bounds = {
        (w + block_width - 1) / block_width,
        (h + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {w, h, 1};

    auto options = sh_coeffs.options();
    torch::Tensor out_color = torch::empty({h, w, 3}, options);

    render_background_sh_forward_kernel<<<tile_bounds, block>>>(
        tile_bounds, img_size,
        tuple2float4(intrins),
        rotation.contiguous().data_ptr<float>(),
        sh_degree,
        (float3 *)sh_coeffs.contiguous().data_ptr<float>(),
        (float3 *)out_color.contiguous().data_ptr<float>()
    );

    return out_color;
}


std::tuple<
    torch::Tensor,  // v_rotation
    torch::Tensor  // v_sh_coeffs
> render_background_sh_backward_tensor(
    const unsigned w,
    const unsigned h,
    const unsigned block_width,
    const std::tuple<float, float, float, float> intrins,
    const torch::Tensor &rotation,
    const unsigned sh_degree,
    const torch::Tensor &sh_coeffs,
    const torch::Tensor &out_color,
    const torch::Tensor &v_out_color
) {
    DEVICE_GUARD(sh_coeffs);
    CHECK_INPUT(sh_coeffs);
    CHECK_INPUT(rotation);
    CHECK_INPUT(v_out_color);

    if (rotation.numel() != 9) {
        AT_ERROR("rotation must be 3x3");
    }
    if (sh_coeffs.ndimension() != 2 ||
        sh_coeffs.size(0) != sh_degree*sh_degree ||
        sh_coeffs.size(1) != 3) {
        AT_ERROR("sh_coeffs shape must be (sh_regree**2, 3)");
    }
    if (out_color.ndimension() != 3 ||
        out_color.size(0) != h ||
        out_color.size(1) != w ||
        out_color.size(2) != 3) {
        AT_ERROR("out_color shape must be (h, w, 3)");
    }
    if (v_out_color.ndimension() != 3 ||
        v_out_color.size(0) != h ||
        v_out_color.size(1) != w ||
        v_out_color.size(2) != 3) {
        AT_ERROR("v_out_color shape must be (h, w, 3)");
    }

    const dim3 tile_bounds = {
        (w + block_width - 1) / block_width,
        (h + block_width - 1) / block_width,
        1
    };
    const dim3 block(block_width, block_width, 1);
    const dim3 img_size = {w, h, 1};

    auto options = sh_coeffs.options();
    torch::Tensor v_rotation = torch::zeros({3, 3}, options);
    torch::Tensor v_sh_coeffs = torch::zeros({sh_degree*sh_degree, 3}, options);

    render_background_sh_backward_kernel<<<tile_bounds, block>>>(
        tile_bounds, img_size,
        tuple2float4(intrins),
        rotation.contiguous().data_ptr<float>(),
        sh_degree,
        (float3 *)sh_coeffs.contiguous().data_ptr<float>(),
        (float3 *)out_color.contiguous().data_ptr<float>(),
        (float3 *)v_out_color.contiguous().data_ptr<float>(),
        v_rotation.contiguous().data_ptr<float>(),
        (float3 *)v_sh_coeffs.contiguous().data_ptr<float>()
    );

    return std::make_tuple(v_rotation, v_sh_coeffs);
}
