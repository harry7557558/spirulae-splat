#include "hip/hip_runtime.h"
#include "SphericalHarmonics.cuh"

#include <hip/hip_cooperative_groups.h>

#include "common.cuh"

// TODO: gradient to viewdir; masks;

namespace cg = cooperative_groups;

__device__ __constant__ float SH_C0 = 0.28209479177387814f;
__device__ __constant__ float SH_C1 = 0.4886025119029199f;
__device__ __constant__ float SH_C2[] = {
    1.0925484305920792f,
    -1.0925484305920792f,
    0.31539156525252005f,
    -1.0925484305920792f,
    0.5462742152960396f};
__device__ __constant__ float SH_C3[] = {
    -0.5900435899266435f,
    2.890611442640554f,
    -0.4570457994644658f,
    0.3731763325901154f,
    -0.4570457994644658f,
    1.445305721320277f,
    -0.5900435899266435f};
__device__ __constant__ float SH_C4[] = {
    2.5033429417967046f,
    -1.7701307697799304,
    0.9461746957575601f,
    -0.6690465435572892f,
    0.10578554691520431f,
    -0.6690465435572892f,
    0.47308734787878004f,
    -1.7701307697799304f,
    0.6258357354491761f};


// This function is used in both host and device code
__host__ __device__ unsigned num_sh_bases(const unsigned degree) {
    if (degree == 0)
        return 1;
    if (degree == 1)
        return 4;
    if (degree == 2)
        return 9;
    if (degree == 3)
        return 16;
    return 25;
}



// Evaluate spherical harmonics bases at unit direction for high orders using approach described by
// Efficient Spherical Harmonic Evaluation, Peter-Pike Sloan, JCGT 2013
// See https://jcgt.org/published/0002/02/06/ for reference implementation
__device__ void sh_coeffs_to_color_fast(
    const unsigned degree,
    const float3 &viewdir,
    const float3 &coeffs0,
    const float3 *coeffs,
    float3 &colors
) {
    colors = 0.2820947917738781f * coeffs0;
    if (degree < 1) {
        colors = fmax(colors+0.5f, 0.0f);
        return;
    }

    float norm = sqrt(
        viewdir.x * viewdir.x + viewdir.y * viewdir.y + viewdir.z * viewdir.z
    );
    float x = viewdir.x / norm;
    float y = viewdir.y / norm;
    float z = viewdir.z / norm;

    float fTmp0A = 0.48860251190292f;
    colors += fTmp0A *
            (-y * coeffs[0] +
            z * coeffs[1] -
            x * coeffs[2]);
    if (degree < 2) {
        colors = fmax(colors+0.5f, 0.0f);
        return;
    }
    float z2 = z * z;

    float fTmp0B = -1.092548430592079f * z;
    float fTmp1A = 0.5462742152960395f;
    float fC1 = x * x - y * y;
    float fS1 = 2.f * x * y;
    float pSH6 = (0.9461746957575601f * z2 - 0.3153915652525201f);
    float pSH7 = fTmp0B * x;
    float pSH5 = fTmp0B * y;
    float pSH8 = fTmp1A * fC1;
    float pSH4 = fTmp1A * fS1;
    colors +=
        pSH4 * coeffs[3] + pSH5 * coeffs[4] +
        pSH6 * coeffs[5] + pSH7 * coeffs[6] +
        pSH8 * coeffs[7];
    if (degree < 3) {
        colors = fmax(colors+0.5f, 0.0f);
        return;
    }

    float fTmp0C = -2.285228997322329f * z2 + 0.4570457994644658f;
    float fTmp1B = 1.445305721320277f * z;
    float fTmp2A = -0.5900435899266435f;
    float fC2 = x * fC1 - y * fS1;
    float fS2 = x * fS1 + y * fC1;
    float pSH12 = z * (1.865881662950577f * z2 - 1.119528997770346f);
    float pSH13 = fTmp0C * x;
    float pSH11 = fTmp0C * y;
    float pSH14 = fTmp1B * fC1;
    float pSH10 = fTmp1B * fS1;
    float pSH15 = fTmp2A * fC2;
    float pSH9  = fTmp2A * fS2;
    colors += pSH9  * coeffs[8] +
            pSH10 * coeffs[9] +
            pSH11 * coeffs[10] +
            pSH12 * coeffs[11] +
            pSH13 * coeffs[12] +
            pSH14 * coeffs[13] +
            pSH15 * coeffs[14];
    if (degree < 4) {
        colors = fmax(colors+0.5f, 0.0f);
        return;
    }

    float fTmp0D = z * (-4.683325804901025f * z2 + 2.007139630671868f);
    float fTmp1C = 3.31161143515146f * z2 - 0.47308734787878f;
    float fTmp2B = -1.770130769779931f * z;
    float fTmp3A = 0.6258357354491763f;
    float fC3 = x * fC2 - y * fS2;
    float fS3 = x * fS2 + y * fC2;
    float pSH20 = (1.984313483298443f * z * pSH12 - 1.006230589874905f * pSH6);
    float pSH21 = fTmp0D * x;
    float pSH19 = fTmp0D * y;
    float pSH22 = fTmp1C * fC1;
    float pSH18 = fTmp1C * fS1;
    float pSH23 = fTmp2B * fC2;
    float pSH17 = fTmp2B * fS2;
    float pSH24 = fTmp3A * fC3;
    float pSH16 = fTmp3A * fS3;
    colors += pSH16 * coeffs[15] +
            pSH17 * coeffs[16] +
            pSH18 * coeffs[17] +
            pSH19 * coeffs[18] +
            pSH20 * coeffs[19] +
            pSH21 * coeffs[20] +
            pSH22 * coeffs[21] +
            pSH23 * coeffs[22] +
            pSH24 * coeffs[23];
    colors = fmax(colors+0.5f, 0.0f);
}

__device__ void sh_coeffs_to_color_fast_vjp(
    const unsigned degree,
    const float3 &viewdir,
    const float3 *coeffs,
    const float3 &colors,
    float3 v_colors,
    float3 &v_coeffs0,
    float3 *v_coeffs,
    float3 *v_viewdir
) {
    v_colors.x = (colors.x == 0.0f ? 0.0f : v_colors.x);
    v_colors.y = (colors.y == 0.0f ? 0.0f : v_colors.y);
    v_colors.z = (colors.z == 0.0f ? 0.0f : v_colors.z);

    v_coeffs0 = 0.2820947917738781f * v_colors;
    if (degree < 1) {
        return;
    }
    float inorm = rsqrtf(
        viewdir.x * viewdir.x + viewdir.y * viewdir.y + viewdir.z * viewdir.z
    );
    float x = viewdir.x * inorm;
    float y = viewdir.y * inorm;
    float z = viewdir.z * inorm;
    float v_x = 0.f, v_y = 0.f, v_z = 0.f;


    float fTmp0A = 0.48860251190292f;
    v_coeffs[0] = -fTmp0A * y * v_colors;
    v_coeffs[1] = fTmp0A * z * v_colors;
    v_coeffs[2] = -fTmp0A * x * v_colors;
    if (v_viewdir != nullptr) {
        v_x += -fTmp0A * dot(coeffs[2], v_colors);
        v_y += -fTmp0A * dot(coeffs[0], v_colors);
        v_z += fTmp0A * dot(coeffs[1], v_colors);
    }
    if (degree < 2) {
        if (v_viewdir != nullptr) {
            float3 dir_n = make_float3(x, y, z);
            float3 v_dir_n = make_float3(v_x, v_y, v_z);
            float3 v_d = (v_dir_n - dot(v_dir_n, dir_n) * dir_n) * inorm;
            *v_viewdir = v_d;
        }
        return;
    }

    float z2 = z * z;
    float fTmp0B = -1.092548430592079f * z;
    float fTmp1A = 0.5462742152960395f;
    float fC1 = x * x - y * y;
    float fS1 = 2.f * x * y;
    float pSH6 = (0.9461746957575601f * z2 - 0.3153915652525201f);
    float pSH7 = fTmp0B * x;
    float pSH5 = fTmp0B * y;
    float pSH8 = fTmp1A * fC1;
    float pSH4 = fTmp1A * fS1;
    v_coeffs[3] = pSH4 * v_colors;
    v_coeffs[4] = pSH5 * v_colors;
    v_coeffs[5] = pSH6 * v_colors;
    v_coeffs[6] = pSH7 * v_colors;
    v_coeffs[7] = pSH8 * v_colors;

    float fTmp0B_z, fC1_x, fC1_y, fS1_x, fS1_y, pSH6_z, pSH7_x, pSH7_z, pSH5_y,
        pSH5_z, pSH8_x, pSH8_y, pSH4_x, pSH4_y;
    if (v_viewdir != nullptr) {
        fTmp0B_z = -1.092548430592079f;
        fC1_x = 2.f * x;
        fC1_y = -2.f * y;
        fS1_x = 2.f * y;
        fS1_y = 2.f * x;
        pSH6_z = 2.f * 0.9461746957575601f * z;
        pSH7_x = fTmp0B;
        pSH7_z = fTmp0B_z * x;
        pSH5_y = fTmp0B;
        pSH5_z = fTmp0B_z * y;
        pSH8_x = 0.5462742152960395f * fC1_x;
        pSH8_y = 0.5462742152960395f * fC1_y;
        pSH4_x = 0.5462742152960395f * fS1_x;
        pSH4_y = 0.5462742152960395f * fS1_y;

        v_x += dot(v_colors,
            pSH4_x * coeffs[3] + pSH8_x * coeffs[7] +
                pSH7_x * coeffs[6]);
        v_y += dot(v_colors,
            pSH4_y * coeffs[3] + pSH8_y * coeffs[7] +
                pSH5_y * coeffs[4]);
        v_z += dot(v_colors,
            pSH6_z * coeffs[5] + pSH7_z * coeffs[6] +
                pSH5_z * coeffs[4]);
    }

    if (degree < 3) {
        if (v_viewdir != nullptr) {
            float3 dir_n = make_float3(x, y, z);
            float3 v_dir_n = make_float3(v_x, v_y, v_z);
            float3 v_d = (v_dir_n - dot(v_dir_n, dir_n) * dir_n) * inorm;
            *v_viewdir = v_d;
        }
        return;
    }

    float fTmp0C = -2.285228997322329f * z2 + 0.4570457994644658f;
    float fTmp1B = 1.445305721320277f * z;
    float fTmp2A = -0.5900435899266435f;
    float fC2 = x * fC1 - y * fS1;
    float fS2 = x * fS1 + y * fC1;
    float pSH12 = z * (1.865881662950577f * z2 - 1.119528997770346f);
    float pSH13 = fTmp0C * x;
    float pSH11 = fTmp0C * y;
    float pSH14 = fTmp1B * fC1;
    float pSH10 = fTmp1B * fS1;
    float pSH15 = fTmp2A * fC2;
    float pSH9  = fTmp2A * fS2;
    v_coeffs[8] = pSH9 * v_colors;
    v_coeffs[9] = pSH10 * v_colors;
    v_coeffs[10] = pSH11 * v_colors;
    v_coeffs[11] = pSH12 * v_colors;
    v_coeffs[12] = pSH13 * v_colors;
    v_coeffs[13] = pSH14 * v_colors;
    v_coeffs[14] = pSH15 * v_colors;

    float fTmp0C_z, fTmp1B_z, fC2_x, fC2_y, fS2_x, fS2_y, pSH12_z, pSH13_x,
        pSH13_z, pSH11_y, pSH11_z, pSH14_x, pSH14_y, pSH14_z, pSH10_x, pSH10_y,
        pSH10_z, pSH15_x, pSH15_y, pSH9_x, pSH9_y;
    if (v_viewdir != nullptr) {
        fTmp0C_z = -2.285228997322329f * 2.f * z;
        fTmp1B_z = 1.445305721320277f;
        fC2_x = fC1 + x * fC1_x - y * fS1_x;
        fC2_y = x * fC1_y - fS1 - y * fS1_y;
        fS2_x = fS1 + x * fS1_x + y * fC1_x;
        fS2_y = x * fS1_y + fC1 + y * fC1_y;
        pSH12_z = 3.f * 1.865881662950577f * z2 - 1.119528997770346f;
        pSH13_x = fTmp0C;
        pSH13_z = fTmp0C_z * x;
        pSH11_y = fTmp0C;
        pSH11_z = fTmp0C_z * y;
        pSH14_x = fTmp1B * fC1_x;
        pSH14_y = fTmp1B * fC1_y;
        pSH14_z = fTmp1B_z * fC1;
        pSH10_x = fTmp1B * fS1_x;
        pSH10_y = fTmp1B * fS1_y;
        pSH10_z = fTmp1B_z * fS1;
        pSH15_x = -0.5900435899266435f * fC2_x;
        pSH15_y = -0.5900435899266435f * fC2_y;
        pSH9_x = -0.5900435899266435f * fS2_x;
        pSH9_y = -0.5900435899266435f * fS2_y;

        v_x += dot(v_colors,
            pSH9_x * coeffs[8] + pSH15_x * coeffs[14] +
                pSH10_x * coeffs[9] + pSH14_x * coeffs[13] +
                pSH13_x * coeffs[12]);
        v_y += dot(v_colors,
            pSH9_y * coeffs[8] + pSH15_y * coeffs[14] +
                pSH10_y * coeffs[9] + pSH14_y * coeffs[13] +
                pSH11_y * coeffs[10]);
        v_z += dot(v_colors,
            pSH12_z * coeffs[11] + pSH13_z * coeffs[12] +
                pSH11_z * coeffs[10] + pSH14_z * coeffs[13] +
                pSH10_z * coeffs[9]);
    }

    if (degree < 4) {
        if (v_viewdir != nullptr) {
            float3 dir_n = make_float3(x, y, z);
            float3 v_dir_n = make_float3(v_x, v_y, v_z);
            float3 v_d = (v_dir_n - dot(v_dir_n, dir_n) * dir_n) * inorm;
            *v_viewdir = v_d;
        }
        return;
    }

    float fTmp0D = z * (-4.683325804901025f * z2 + 2.007139630671868f);
    float fTmp1C = 3.31161143515146f * z2 - 0.47308734787878f;
    float fTmp2B = -1.770130769779931f * z;
    float fTmp3A = 0.6258357354491763f;
    float fC3 = x * fC2 - y * fS2;
    float fS3 = x * fS2 + y * fC2;
    float pSH20 = (1.984313483298443f * z * pSH12 + -1.006230589874905f * pSH6);
    float pSH21 = fTmp0D * x;
    float pSH19 = fTmp0D * y;
    float pSH22 = fTmp1C * fC1;
    float pSH18 = fTmp1C * fS1;
    float pSH23 = fTmp2B * fC2;
    float pSH17 = fTmp2B * fS2;
    float pSH24 = fTmp3A * fC3;
    float pSH16 = fTmp3A * fS3;
    v_coeffs[15] = pSH16 * v_colors;
    v_coeffs[16] = pSH17 * v_colors;
    v_coeffs[17] = pSH18 * v_colors;
    v_coeffs[18] = pSH19 * v_colors;
    v_coeffs[19] = pSH20 * v_colors;
    v_coeffs[20] = pSH21 * v_colors;
    v_coeffs[21] = pSH22 * v_colors;
    v_coeffs[22] = pSH23 * v_colors;
    v_coeffs[23] = pSH24 * v_colors;

    float fTmp0D_z, fTmp1C_z, fTmp2B_z, fC3_x, fC3_y, fS3_x, fS3_y, pSH20_z,
        pSH21_x, pSH21_z, pSH19_y, pSH19_z, pSH22_x, pSH22_y, pSH22_z, pSH18_x,
        pSH18_y, pSH18_z, pSH23_x, pSH23_y, pSH23_z, pSH17_x, pSH17_y, pSH17_z,
        pSH24_x, pSH24_y, pSH16_x, pSH16_y;
    if (v_viewdir != nullptr) {
        fTmp0D_z = 3.f * -4.683325804901025f * z2 + 2.007139630671868f;
        fTmp1C_z = 2.f * 3.31161143515146f * z;
        fTmp2B_z = -1.770130769779931f;
        fC3_x = fC2 + x * fC2_x - y * fS2_x;
        fC3_y = x * fC2_y - fS2 - y * fS2_y;
        fS3_x = fS2 + y * fC2_x + x * fS2_x;
        fS3_y = x * fS2_y + fC2 + y * fC2_y;
        pSH20_z = 1.984313483298443f * (pSH12 + z * pSH12_z) +
                  -1.006230589874905f * pSH6_z;
        pSH21_x = fTmp0D;
        pSH21_z = fTmp0D_z * x;
        pSH19_y = fTmp0D;
        pSH19_z = fTmp0D_z * y;
        pSH22_x = fTmp1C * fC1_x;
        pSH22_y = fTmp1C * fC1_y;
        pSH22_z = fTmp1C_z * fC1;
        pSH18_x = fTmp1C * fS1_x;
        pSH18_y = fTmp1C * fS1_y;
        pSH18_z = fTmp1C_z * fS1;
        pSH23_x = fTmp2B * fC2_x;
        pSH23_y = fTmp2B * fC2_y;
        pSH23_z = fTmp2B_z * fC2;
        pSH17_x = fTmp2B * fS2_x;
        pSH17_y = fTmp2B * fS2_y;
        pSH17_z = fTmp2B_z * fS2;
        pSH24_x = 0.6258357354491763f * fC3_x;
        pSH24_y = 0.6258357354491763f * fC3_y;
        pSH16_x = 0.6258357354491763f * fS3_x;
        pSH16_y = 0.6258357354491763f * fS3_y;

        v_x += dot(v_colors,
            pSH16_x * coeffs[15] + pSH24_x * coeffs[23] +
                pSH17_x * coeffs[16] + pSH23_x * coeffs[22] +
                pSH18_x * coeffs[17] + pSH22_x * coeffs[21] +
                pSH21_x * coeffs[20]);
        v_y += dot(v_colors,
            pSH16_y * coeffs[15] + pSH24_y * coeffs[23] +
                pSH17_y * coeffs[16] + pSH23_y * coeffs[22] +
                pSH18_y * coeffs[17] + pSH22_y * coeffs[21] +
                pSH19_y * coeffs[18]);
        v_z += dot(v_colors,
            pSH20_z * coeffs[19] + pSH21_z * coeffs[20] +
                pSH19_z * coeffs[18] + pSH22_z * coeffs[21] +
                pSH18_z * coeffs[17] + pSH23_z * coeffs[22] +
                pSH17_z * coeffs[16]);

        float3 dir_n = make_float3(x, y, z);
        float3 v_dir_n = make_float3(v_x, v_y, v_z);
        float3 v_d = (v_dir_n - dot(v_dir_n, dir_n) * dir_n) * inorm;
        *v_viewdir = v_d;
    }
}

__global__ void compute_sh_forward_kernel(
    const unsigned num_points,
    const unsigned degree,
    const unsigned degrees_to_use,
    const float3* __restrict__ viewdirs,
    const float3* __restrict__ coeffs0,
    const float3* __restrict__ coeffs,
    float3* __restrict__ colors
) {
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= num_points) {
        return;
    }
    unsigned num_bases = num_sh_bases(degree);
    unsigned idx_sh0 = idx;
    unsigned idx_sh = (num_bases-1) * idx;
    unsigned idx_col = idx;

    sh_coeffs_to_color_fast(
        degrees_to_use, viewdirs[idx], coeffs0[idx_sh0], &coeffs[idx_sh], colors[idx_col]
    );
}

__global__ void compute_sh_backward_kernel(
    const unsigned num_points,
    const unsigned degree,
    const unsigned degrees_to_use,
    const float3* __restrict__ viewdirs,
    const float3* __restrict__ coeffs,
    const float3* __restrict__ colors,
    const float3* __restrict__ v_colors,
    float3* __restrict__ v_coeffs0,
    float3* __restrict__ v_coeffs,
    float3* __restrict__ v_viewdirs
) {
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= num_points) {
        return;
    }
    unsigned num_bases = num_sh_bases(degree);
    unsigned idx_sh0 = idx;
    unsigned idx_sh = (num_bases-1) * idx;
    unsigned idx_col = idx;
    
    sh_coeffs_to_color_fast_vjp(
        degrees_to_use,
        viewdirs[idx], &coeffs[idx_sh],
        colors[idx_col], v_colors[idx_col],
        v_coeffs0[idx_sh0], &v_coeffs[idx_sh],
        v_viewdirs ? &v_viewdirs[idx] : nullptr
    );
}


torch::Tensor compute_sh_forward_tensor(
    const unsigned degree,
    const unsigned degrees_to_use,
    torch::Tensor &viewdirs,  // [..., 3]
    torch::Tensor &coeffs0,   // [..., 3]
    torch::Tensor &coeffs   // [..., K, 3]
) {
    DEVICE_GUARD(viewdirs);
    CHECK_INPUT(viewdirs);
    CHECK_INPUT(coeffs0);
    CHECK_INPUT(coeffs);
    unsigned num_bases = num_sh_bases(degree);
    long num_points = viewdirs.numel() / 3;

    if (coeffs0.ndimension() < 2 || coeffs0.size(-1) != 3) {
        AT_ERROR("coeffs0 must have dimensions (..., 3)");
    }
    if (coeffs.ndimension() < 3 || coeffs.size(-1) != 3 || coeffs.size(-2) != num_bases-1) {
        AT_ERROR("coeffs must have dimensions (..., D, 3)");
    }
    torch::Tensor colors = torch::empty_like(coeffs0);

    compute_sh_forward_kernel <<<_LAUNCH_ARGS_1D(num_points, 256)>>>(
        num_points, degree, degrees_to_use,
        (float3*)viewdirs.data_ptr<float>(),
        (float3*)coeffs0.data_ptr<float>(),
        (float3*)coeffs.data_ptr<float>(),
        (float3*)colors.data_ptr<float>()
    );

    return colors;
}



std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
compute_sh_backward_tensor(
    const unsigned degree,
    const unsigned degrees_to_use,
    torch::Tensor &viewdirs,  // [..., 3]
    torch::Tensor &coeffs,  // [..., 3]
    torch::Tensor &colors,  // [..., 3]
    torch::Tensor &v_colors  // [..., 3]
) {
    DEVICE_GUARD(viewdirs);
    CHECK_INPUT(viewdirs);
    CHECK_INPUT(coeffs);
    CHECK_INPUT(colors);
    CHECK_INPUT(v_colors);
    unsigned num_bases = num_sh_bases(degree);
    long num_points = viewdirs.numel() / 3;

    if (viewdirs.ndimension() < 2 || viewdirs.size(-1) != 3) {
        AT_ERROR("viewdirs must have dimensions (..., 3)");
    }
    if (colors.ndimension() < 2 || colors.size(-1) != 3) {
        AT_ERROR("colors must have dimensions (..., 3)");
    }
    if (v_colors.ndimension() < 2 || v_colors.size(-1) != 3) {
        AT_ERROR("v_colors must have dimensions (..., 3)");
    }
    torch::Tensor v_coeffs0 = torch::empty_like(colors);
    torch::Tensor v_coeffs = torch::empty_like(coeffs);
    torch::Tensor v_viewdirs = torch::empty_like(viewdirs);

    compute_sh_backward_kernel<<<_LAUNCH_ARGS_1D(num_points, 256)>>>(
        num_points, degree, degrees_to_use,
        (float3 *)viewdirs.data_ptr<float>(),
        (float3*)coeffs.data_ptr<float>(),
        (float3*)colors.data_ptr<float>(),
        (float3*)v_colors.data_ptr<float>(),
        (float3*)v_coeffs0.data_ptr<float>(),
        (float3*)v_coeffs.data_ptr<float>(),
        (float3*)v_viewdirs.data_ptr<float>()
    );

    return std::make_tuple(v_coeffs0, v_coeffs, v_viewdirs);
}

