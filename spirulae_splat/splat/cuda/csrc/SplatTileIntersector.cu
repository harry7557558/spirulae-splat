#include "hip/hip_runtime.h"
#include "SplatTileIntersector.cuh"

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <vector>

#include <c10/cuda/CUDAGuard.h>

#include "helpers.cuh"

// #define DEBUG


struct Splat {
    float3 mean;
    glm::mat3 cov;

    __device__ __forceinline__ void getAABB(float3 &aabb_min, float3 &aabb_max) const {
        float3 extend = make_float3(sqrtf(cov[0][0]), sqrtf(cov[1][1]), sqrtf(cov[2][2]));
        if (isfinite(dot(mean, extend))) {
            aabb_min = mean - extend;
            aabb_max = mean + extend;
        } else {
            aabb_min = aabb_max = make_float3(0.0f);
        }
    }
};

__device__ __forceinline__ Splat loadSplat(unsigned splatIdx, const SplatBuffers& buffers) {

    glm::vec3 mean = buffers.means[splatIdx];
    glm::vec3 scales = buffers.scales[splatIdx];
    float opac = buffers.opacs[splatIdx];
    float4 quat = buffers.quats[splatIdx];

    // opac = 1.0f / (1.0f+__expf(-opac));
    // scales = { __expf(scales.x), __expf(scales.y), __expf(scales.z) };
    quat = normalize(quat);

    float extend = fmin(3.33f, sqrt(2.0f * __logf(opac / ALPHA_THRESHOLD)));

    glm::mat3 S = {
        scales.x, 0.0f, 0.0f,
        0.0f, scales.y, 0.0f,
        0.0f, 0.0f, scales.z
    };
    glm::mat3 R = quat_to_rotmat(quat);
    glm::mat3 M = extend * R * S;
    return {
        {mean.x, mean.y, mean.z},
        M * glm::transpose(M)
    };
}


struct Ray {
    float3 ro;
    float3 rd;

    __device__ __forceinline__ bool isOverlap(float3 aabb_min, float3 aabb_max) const {
        float3 aabb_center = 0.5f*(aabb_min+aabb_max);
        float3 aabb_size = 0.5f*(aabb_max-aabb_min);
        float3 m = 1.0f / rd;
        float3 n = m * (ro - aabb_center);
        float3 k = fabs(m) * aabb_size;
        float3 t1 = -n - k;
        float3 t2 = -n + k;
        float tn = fmax(fmax(t1.x, t1.y), t1.z);
        float tf = fmin(fmin(t2.x, t2.y), t2.z);
        return (tn < tf && tf > 0.0f);
    }

    // return negative if no overlap, strictly positive for sorting ID
    __device__ __forceinline__ float isOverlap(const Splat &splat) const {
        float3 aabb_min, aabb_max;
        splat.getAABB(aabb_min, aabb_max);
        if (!isOverlap(aabb_min, aabb_max))
            return -1.0f;
        return dot(splat.mean-ro, rd);  // negative if center is behind
    }

};

struct Tile {
    float x0, x1, y0, y1;
    glm::mat4x3 view;
    glm::vec3 ro, rd;
    glm::vec3 n0, n1, n2, n3;

    __device__ __forceinline__ void precompute() {
        glm::mat3 R = glm::transpose(glm::mat3(view));
        ro = -R * glm::vec3(view[3]);
        rd = R[2];
        glm::vec3 e0 = R * glm::vec3(x0, y0, 1.0f);
        glm::vec3 e1 = R * glm::vec3(x0, y1, 1.0f);
        glm::vec3 e2 = R * glm::vec3(x1, y1, 1.0f);
        glm::vec3 e3 = R * glm::vec3(x1, y0, 1.0f);
        n0 = glm::cross(e0, e1);
        n1 = glm::cross(e1, e2);
        n2 = glm::cross(e2, e3);
        n3 = glm::cross(e3, e0);
    }

    __device__ __forceinline__ bool isOverlap(float3 aabb_min, float3 aabb_max) const {

        float3 c_ = 0.5f*(aabb_min+aabb_max);
        float3 r_ = 0.5f*(aabb_max-aabb_min);
        glm::vec3 c = {c_.x, c_.y, c_.z};
        glm::vec3 r = {r_.x, r_.y, r_.z};

        // intersection test using separating axis theorem
        // has false positive, may be good enough in practice
        glm::vec3 roc = c - ro;
        float s0 = glm::dot(n0, roc) - glm::dot(r, glm::abs(n0));
        float s1 = glm::dot(n1, roc) - glm::dot(r, glm::abs(n1));
        float s2 = glm::dot(n2, roc) - glm::dot(r, glm::abs(n2));
        float s3 = glm::dot(n3, roc) - glm::dot(r, glm::abs(n3));
        float s = fmax(fmax(s0, s1), fmax(s2, s3));
        float sz = -glm::dot(rd, roc) - glm::dot(r, glm::abs(rd));
        return fmax(s, sz) < 0.0f;
    }

    // return negative if no overlap, strictly positive for sorting ID
    __device__ __forceinline__ float isOverlap(const Splat &splat) const {
        // TODO
        float3 aabb_min, aabb_max;
        splat.getAABB(aabb_min, aabb_max);
        if (!isOverlap(aabb_min, aabb_max))
            return -1.0f;
        glm::vec3 mean = {splat.mean.x, splat.mean.y, splat.mean.z};
        return glm::dot(mean-ro, rd);  // negative if center is behind
    }

};

__device__ __forceinline__ Tile loadTile(unsigned tileIdx, const TileBuffers buffers) {
    static_assert(sizeof(glm::mat4) == 16*sizeof(float));
    static_assert(sizeof(glm::mat3) == 9*sizeof(float));

    glm::mat4 view = glm::transpose(buffers.viewmats[tileIdx]);
    glm::mat3 intrins = buffers.Ks[tileIdx];  // take it as row major

    float fx = intrins[0][0];
    float fy = intrins[1][1];
    float cx = intrins[0][2];
    float cy = intrins[1][2];
    // printf("%f %f %f %f\n", fx, fy, cx, cy);

    Tile res = {
        -cx / fx, (buffers.width - cx) / fx,
        -cy / fy, (buffers.height - cy) / fy,
        glm::mat4x3(glm::vec3(view[0]), glm::vec3(view[1]), glm::vec3(view[2]), glm::vec3(view[3]))
    };
    res.precompute();
    return res;
}


__global__ void computeSplatAABB(
    const SplatBuffers& splats,
    float3* __restrict__ aabb,
    float3* __restrict__ aabb_reduced
) {
    unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= splats.size)
        return;

    Splat g = loadSplat(gid, splats);
    float3 aabb_min, aabb_max;
    g.getAABB(aabb_min, aabb_max);
    if (aabb != nullptr) {
        aabb[2*gid+0] = aabb_min;
        aabb[2*gid+1] = aabb_max;
    }

    auto block = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> warp = cg::tiled_partition<WARP_SIZE>(block);

    aabb_min.x = cg::reduce(warp, aabb_min.x, cg::less<float>());
    if (warp.thread_rank() == 0) atomicMin((float*)aabb_reduced+0, aabb_min.x);
    aabb_min.y = cg::reduce(warp, aabb_min.y, cg::less<float>());
    if (warp.thread_rank() == 0) atomicMin((float*)aabb_reduced+1, aabb_min.y);
    aabb_min.z = cg::reduce(warp, aabb_min.z, cg::less<float>());
    if (warp.thread_rank() == 0) atomicMin((float*)aabb_reduced+2, aabb_min.z);

    aabb_max.x = cg::reduce(warp, aabb_max.x, cg::greater<float>());
    if (warp.thread_rank() == 0) atomicMax((float*)aabb_reduced+3, aabb_max.x);
    aabb_max.y = cg::reduce(warp, aabb_max.y, cg::greater<float>());
    if (warp.thread_rank() == 0) atomicMax((float*)aabb_reduced+4, aabb_max.y);
    aabb_max.z = cg::reduce(warp, aabb_max.z, cg::greater<float>());
    if (warp.thread_rank() == 0) atomicMax((float*)aabb_reduced+5, aabb_max.z);

}


template<uint BRANCH_FACTOR>
__device__ unsigned getLevel(
    float3 aabb_min, float3 aabb_max,
    float3 root_min, float3 root_max,
    unsigned num_levels
) {
    float3 size = aabb_max - aabb_min;
    float max_size = fmaxf(size.x, fmaxf(size.y, size.z));
    float3 root_size = root_max - root_min;
    float root_max_size = fmaxf(root_size.x, fmaxf(root_size.y, root_size.z));

    // will overlap with max 8 cells if root is cube
    float ratio = fmaxf(root_max_size / max_size, 1.0f);
    float level = __logf(ratio) / __logf(BRANCH_FACTOR);
    return min(max((unsigned)level, (unsigned)0), num_levels-1);
}

template<uint BRANCH_FACTOR>
__device__ __forceinline__ uint getSubcellOffset(uint3 subcell) {
    uint3 i = subcell % BRANCH_FACTOR;
    return (i.z * BRANCH_FACTOR + i.y) * BRANCH_FACTOR + i.x;
}

template<uint BRANCH_FACTOR>
__global__ void countCellOverlaps(
    const SplatBuffers& splats,
    float3 root_min, float3 root_max,
    unsigned num_levels,
    unsigned* __restrict__ overlap_counts
) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= splats.size) return;
    
    Splat g = loadSplat(tid, splats);
    float3 aabb_min, aabb_max;
    g.getAABB(aabb_min, aabb_max);
    
    unsigned level = getLevel<BRANCH_FACTOR>(aabb_min, aabb_max, root_min, root_max, num_levels);
    
    unsigned count_splat = 1;
    unsigned count_subcell = level;
    overlap_counts[tid] = count_splat + count_subcell;
}

__device__ __forceinline__ uint64_t insert_2_zeros_between_bits(uint64_t x) {
    x = (x | (x << 32)) & (uint64_t)0xFFFF00000000FFFFULL;
    x = (x | (x << 16)) & (uint64_t)0x00FF0000FF0000FFULL;
    x = (x | (x << 8))  & (uint64_t)0x100F00F00F00F00FULL;
    x = (x | (x << 4))  & (uint64_t)0x10C30C30C30C30C3ULL;
    x = (x | (x << 2))  & (uint64_t)0x1249249249249249ULL;
    return x;
}

template<uint BRANCH_FACTOR>
__device__ __forceinline__ uint64_t getCellKey(
    unsigned level,
    uint3 pos, bool isSplat
) {
    static_assert(BRANCH_FACTOR == 2);
    // 6 bit level, 1 bit splat vs cell, (64-6-1)/3=19 Morton bits in each dimension
    constexpr unsigned kMortonBitsPerDim = 19;
    uint64_t x = (uint64_t)(pos.x & ((1<<level)-1)) << (kMortonBitsPerDim - level);
    uint64_t y = (uint64_t)(pos.y & ((1<<level)-1)) << (kMortonBitsPerDim - level);
    uint64_t z = (uint64_t)(pos.z & ((1<<level)-1)) << (kMortonBitsPerDim - level);
  #if 0
    x = insert_2_zeros_between_bits(x) & (((uint64_t)1<<(3*kMortonBitsPerDim))-1);
    y = insert_2_zeros_between_bits(y) & (((uint64_t)1<<(3*kMortonBitsPerDim))-1);
    z = insert_2_zeros_between_bits(z) & (((uint64_t)1<<(3*kMortonBitsPerDim))-1);
    uint64_t morton = (x * 2 + y) * 2 + z;
  #else
    uint64_t morton = (((x << kMortonBitsPerDim) | y) << kMortonBitsPerDim) | z;
  #endif
    // printf("%u  %d %d %d  %llx\n", level, pos.x, pos.y, pos.z, morton);
    return ((
        (((uint64_t)level << (3*kMortonBitsPerDim)) | morton) << 1
    ) | (uint64_t)isSplat);
}

template<uint BRANCH_FACTOR>
__global__ void fillCellOverlaps(
    const SplatBuffers& splats,
    float3 root_min, float3 root_max,
    unsigned num_levels,
    unsigned* __restrict__ overlap_offsets,
    uint64_t* __restrict__ cell_keys,
    int32_t* __restrict__ splat_ids,
    uint8_t* __restrict__ subcell_masks,
    int32_t* __restrict__ subcell_ids,
    float3* __restrict__ subcell_aabb
) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= splats.size) return;
    
    Splat g = loadSplat(tid, splats);
    float3 aabb_min, aabb_max;
    g.getAABB(aabb_min, aabb_max);
    float3 aabb_center = (aabb_min + aabb_max) * 0.5f;

    unsigned level = getLevel<BRANCH_FACTOR>(aabb_min, aabb_max, root_min, root_max, num_levels);
    unsigned offset = overlap_offsets[tid];
    unsigned num_cells = overlap_offsets[tid+1] - offset;
    if (num_cells == 0)
        return;
    unsigned idx = 0;

    // Fill splat cells
    float scale = powf(BRANCH_FACTOR, level);
    float3 cell_size = (root_max - root_min) / scale;
    
  #if 0
    uint3 min_cell = make_uint3((aabb_min - root_min) / cell_size);
    uint3 max_cell = make_uint3((aabb_max - root_min) / cell_size);
    min_cell = clamp(min_cell, 0, (int)(scale-0.99f));
    max_cell = clamp(max_cell, 0, (int)(scale-0.99f));
    max_cell.x = min(max_cell.x, min_cell.x + 1);
    max_cell.y = min(max_cell.y, min_cell.y + 1);
    max_cell.z = min(max_cell.z, min_cell.z + 1);
    uint3 cells[8];  // max this number of cells guaranteed by getLevel
    int cellsRef[8];
  #else
    uint3 min_cell = make_uint3((aabb_center - root_min) / cell_size + 0.5f);
    uint3 max_cell = min_cell;
    uint3 cells[1];
    int cellsRef[1];
  #endif
    
    uint cellCount = 0;
    for (uint z = min_cell.z; z <= max_cell.z; z++) {
        for (uint y = min_cell.y; y <= max_cell.y; y++) {
            for (uint x = min_cell.x; x <= max_cell.x; x++) {
                cell_keys[offset+idx] = getCellKey<BRANCH_FACTOR>(level, {x, y, z}, true);
                splat_ids[offset+idx] = tid;
                subcell_masks[offset+idx] = (uint8_t)0;
                subcell_ids[offset+idx] = -1;
                subcell_aabb[2*(offset+idx)+0] = aabb_min;
                subcell_aabb[2*(offset+idx)+1] = aabb_max;
                cells[cellCount] = {x, y, z};
                cellsRef[cellCount] = offset+idx;
                if (++idx >= num_cells)
                    return;
                ++cellCount;
            }
        }
    }
    
    // Fill parent cells
    while (level--) {
        // reduce cell list while writing grid
        // notice cells are sorted by z, then y, then x
        uint i0 = 0;
        for (uint i = 0; i < cellCount; i++) {
            // uint8_t mask = (uint8_t)1 << getSubcellOffset<BRANCH_FACTOR>(cells[i]);
            uint8_t mask = (uint8_t)(1 + getSubcellOffset<BRANCH_FACTOR>(cells[i]));
            cells[i0] = (cells[i] >> 1);
            cell_keys[offset+idx] = getCellKey<BRANCH_FACTOR>(level, cells[i0], false);;
            splat_ids[offset+idx] = -1;
            subcell_masks[offset+idx] = mask;
            subcell_ids[offset+idx] = cellsRef[i];
            subcell_aabb[2*(offset+idx)+0] = aabb_min;
            subcell_aabb[2*(offset+idx)+1] = aabb_max;
            if (i0 == 0 || cells[i0-1] != cells[i0]) {
                cellsRef[i0] = offset+idx;
                ++i0;
            }
            if (++idx >= num_cells)
                return;
        }
        cellCount = i0;
        if (level == 0) break;
    }

    // set the rest empty
    while (idx < num_cells) {
        cell_keys[offset+idx] = (~((uint64_t)0)) >> 1;
        splat_ids[offset+idx] = -1;
        subcell_masks[offset+idx] = (uint8_t)0;
        subcell_ids[offset+idx] = -1;
        idx++;
    }
}



template<typename T>
__global__ void invertPermutation(
    size_t size,
    const T* __restrict__ perm,
    T* __restrict__ inverse
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
    T p = perm[idx];
    inverse[p] = (T)idx;
}

__global__ void gatherAndRemap(
    size_t size,
    const int32_t* __restrict__ subcell,
    const int32_t* __restrict__ perm,
    const int32_t* __restrict__ perm_inverse,
    int32_t* __restrict__ subcell_out
) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    int32_t temp = subcell[perm[i]];
    subcell_out[i] = (temp == -1) ? -1 : perm_inverse[temp];
}

__global__ void getCellDifferential(
    unsigned num_elem,
    const uint64_t* __restrict__ keys,
    int32_t* __restrict__ cell_id_differentials,
    int32_t* __restrict__ splat_differentials
) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid > num_elem) return;

    uint64_t key = keys[tid];

    cell_id_differentials[tid] = (tid == num_elem) ? 0 :
        (int)((key >> 1) != (keys[tid+1] >> 1));

    splat_differentials[tid] = (tid == num_elem) ? 0 :
        (int)(key & 1);
}

__global__ void fillTreeSplats(
    unsigned num_overlaps,
    const uint64_t* __restrict__ cell_keys,
    const int32_t* __restrict__ splat_ids,
    const unsigned* __restrict__ splat_idx_map,
    const unsigned* __restrict__ cell_id_map,
    unsigned* __restrict__ splatRanges,
    unsigned* __restrict__ splatIndices
) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_overlaps) return;

    // filter only splats (not cells)
    if ((int)cell_keys[tid] & 1 == 1) {

        // update splat idx
        unsigned splat_idx = splat_idx_map[tid];
        splatIndices[splat_idx] = splat_ids[tid];

        // update range
        if (tid == 0 || cell_keys[tid] != cell_keys[tid-1])
            splatRanges[cell_id_map[tid]*2] = splat_idx;
        if (tid == num_overlaps-1 || cell_keys[tid] != cell_keys[tid+1])
            splatRanges[cell_id_map[tid]*2+1] = splat_idx + 1;
    }
}


template<typename T>
__device__ __forceinline__ void lower_upper_bounds(
    const T *arr, unsigned n, T value, unsigned &lo, unsigned &hi
) {
    unsigned left = 0, right = n;

    // Find lower bound (first index >= value)
    while (left < right) {
        int mid = left + (right - left) / 2;
        if (arr[mid] < value)
            left = mid + 1;
        else
            right = mid;
    }
    lo = left;

    // Find upper bound (first index > value)
    right = n;
    while (left < right) {
        int mid = left + (right - left) / 2;
        if (arr[mid] <= value)
            left = mid + 1;
        else
            right = mid;
    }
    hi = left;
}



template<uint BRANCH_FACTOR>
__global__ void fillTreeSubcells_perCell(
    unsigned num_overlaps,
    unsigned num_cells,
    const unsigned* __restrict__ cell_id_map,
    const int32_t* __restrict__ subcell_ids,
    const uint8_t* __restrict__ subcell_masks,
    const float3* __restrict__ subcell_aabb,
    int32_t* __restrict__ children,
    float3* __restrict__ treeAABB
) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_cells) return;

    // binary search bounds [i0, i1)
    unsigned i0, i1;
    lower_upper_bounds(cell_id_map, num_overlaps, tid, i0, i1);
    // printf("%u [%u, %u)\n", tid, i0, i1);

    float3 aabbMin, aabbMax;

    // fill
    constexpr unsigned B3 = BRANCH_FACTOR * BRANCH_FACTOR * BRANCH_FACTOR;
    for (unsigned i = i0; i < i1; i++) {

        // update children
        int mask = (int)subcell_masks[i];
        if (mask != 0) {
            // unsigned cid = tid * B3 + (31 - __clz(mask));
            unsigned cid = tid * B3 + (mask - 1);
            unsigned sid = subcell_ids[i];
            children[cid] = cell_id_map[sid];
        }

        // update AABB
        if (i == i0) {
            aabbMin = subcell_aabb[2*i+0];
            aabbMax = subcell_aabb[2*i+1];
        } else {
            aabbMin = fmin(aabbMin, subcell_aabb[2*i+0]);
            aabbMax = fmax(aabbMax, subcell_aabb[2*i+1]);
        }
    }

    treeAABB[2*tid+0] = aabbMin;
    treeAABB[2*tid+1] = aabbMax;
}


__global__ void fillTreeSubcells_initAABB(
    unsigned num_cells,
    float3* __restrict__ treeAABB
) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_cells) return;
    treeAABB[2*tid+0] = make_float3(1e10);
    treeAABB[2*tid+1] = -make_float3(1e10);
}

template<uint BRANCH_FACTOR>
__global__ void fillTreeSubcells_perOverlap(
    unsigned num_overlaps,
    unsigned num_cells,
    const unsigned* __restrict__ cell_id_map,
    const int32_t* __restrict__ subcell_ids,
    const uint8_t* __restrict__ subcell_masks,
    const float3* __restrict__ subcell_aabb,
    int32_t* __restrict__ children,
    float3* __restrict__ treeAABB
) {
    unsigned oid = blockIdx.x * blockDim.x + threadIdx.x;
    if (oid >= num_overlaps) return;
    unsigned tid = cell_id_map[oid];

    constexpr unsigned B3 = BRANCH_FACTOR * BRANCH_FACTOR * BRANCH_FACTOR;

    // update children
    int mask = (int)subcell_masks[oid];
    if (mask != 0) {
        unsigned cid = tid * B3 + (mask - 1);
        unsigned sid = subcell_ids[oid];
        atomicMax(&children[cid], cell_id_map[sid]);
    }

    // update AABB
    float3 aabbMin = subcell_aabb[2*oid+0];
    atomicMin(&treeAABB[2*tid+0].x, aabbMin.x);
    atomicMin(&treeAABB[2*tid+0].y, aabbMin.y);
    atomicMin(&treeAABB[2*tid+0].z, aabbMin.z);
    float3 aabbMax = subcell_aabb[2*oid+1];
    atomicMax(&treeAABB[2*tid+1].x, aabbMax.x);
    atomicMax(&treeAABB[2*tid+1].y, aabbMax.y);
    atomicMax(&treeAABB[2*tid+1].z, aabbMax.z);
}


template<uint MAX_NUM_LEVELS, uint BRANCH_FACTOR>
__global__ void getTileSplatIntersections_octree(
    const TileBuffers tiles, const SplatBuffers& splats,
    const float3 rootAABBMin, const float3 rootAABBMax,
    const int32_t* __restrict__ children,
    const float3* __restrict__ treeAABB,
    const uint32_t* __restrict__ splatRanges,
    const uint32_t* __restrict__ splatIndices,
    uint32_t* __restrict__ intersect_counts,  // to be filled or exclusive scan
    uint32_t* __restrict__ intersectionSplatID  // nullptr or to be filled
) {
    static_assert(BRANCH_FACTOR == 2);
    constexpr uint kNumSubtree = BRANCH_FACTOR*BRANCH_FACTOR*BRANCH_FACTOR;  // 8
    constexpr uint kThreadsPerTile = BRANCH_FACTOR*BRANCH_FACTOR*BRANCH_FACTOR;  // 8
    constexpr uint kTilesPerWarp = 32 / kThreadsPerTile;  // 4

    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned globalTileIdx = tid / kThreadsPerTile;

    unsigned warpTileIdx = threadIdx.x / kThreadsPerTile;
    unsigned tileThreadIdx = threadIdx.x % kThreadsPerTile;
    bool isActive = (globalTileIdx < tiles.size);

    bool isCountingPass = (intersectionSplatID == nullptr);
    uint32_t intersectGlobalOffset = 0, intersectGlobalOffsetMax = 0;
    if (!isCountingPass) {
        intersectGlobalOffset = intersect_counts[globalTileIdx];
        intersectGlobalOffsetMax = intersect_counts[globalTileIdx+1];
    }

    Tile tile;
    if (isActive)
        tile = loadTile(globalTileIdx, tiles);

    struct StackElem {
        uint32_t cellIdx;
        uint32_t level;
        uint3 offset;
    };
    __shared__ StackElem stack[kTilesPerWarp][(MAX_NUM_LEVELS+1)*kThreadsPerTile];
    uint stackSize = 0;
    if (isActive && tile.isOverlap(treeAABB[0], treeAABB[1])) {
        if (tileThreadIdx == 0)
            stack[warpTileIdx][stackSize] = {0, 0, make_uint3(0)};
        stackSize++;
    }
    __shared__ uint numSplatIntersects[kTilesPerWarp];
    if (tileThreadIdx == 0)
        numSplatIntersects[warpTileIdx] = 0;
    __syncwarp();

    for (uint _num_steps = 0; _num_steps < 65536; _num_steps++) {

        if (stackSize == 0)
            isActive = false;
        if (__ballot_sync(~0u, isActive) == 0)
            break;

        --stackSize;
        StackElem elem;
        if (isActive)
            elem = stack[warpTileIdx][stackSize];

        // Process splats
        if (isActive) {
            uint gi0 = splatRanges[2*elem.cellIdx+0], gi1 = splatRanges[2*elem.cellIdx+1];

            for (uint gi_ = gi0; gi_ < gi1; gi_ += kThreadsPerTile) {
                uint gi = gi_ + tileThreadIdx;
                if (gi < gi1) {
                    uint splatIdx = splatIndices[gi];
                    Splat splat = loadSplat(splatIdx, splats);
                    float overlap = tile.isOverlap(splat);
                    if (overlap > 0.0) {
                        uint idx = atomicAdd(&numSplatIntersects[warpTileIdx], 1) + intersectGlobalOffset;
                        if (idx < intersectGlobalOffsetMax) {
                            intersectionSplatID[idx] = splatIdx;
                        }
                    }
                }
            }
        }
        __syncwarp();
        if (!isCountingPass && numSplatIntersects[warpTileIdx] >= intersectGlobalOffsetMax-intersectGlobalOffset)
            isActive = false;
        if (__ballot_sync(~0u, isActive) == 0)
            break;

        // Process subcells
        #pragma unroll
        for (uint si_ = 0; si_ < kNumSubtree; si_ += kThreadsPerTile) {
            uint si = si_ + tileThreadIdx;
            int childIdx = isActive && si < kNumSubtree ?
                children[kNumSubtree*elem.cellIdx + si] : -1;
            int isActiveChild = int(childIdx >= 0);
            if (isActiveChild)
                isActiveChild &= tile.isOverlap(treeAABB[2*childIdx+0], treeAABB[2*childIdx+1]);

            int inclusiveActiveSum = isActiveChild;
            #pragma unroll
            for (unsigned offset = 1; offset < kThreadsPerTile; offset <<= 1) {
                int temp = __shfl_up_sync(~0u, inclusiveActiveSum, offset, kThreadsPerTile);
                if (tileThreadIdx >= offset)
                    inclusiveActiveSum += temp;
            }
            int exclusiveActiveSum = inclusiveActiveSum - isActiveChild;

            int last_lane = warpTileIdx * kThreadsPerTile + (kThreadsPerTile - 1);
            int activeSum = __shfl_sync(~0u, inclusiveActiveSum, last_lane, kThreadsPerTile);

            if (isActiveChild != 0) {
                uint3 delta = {
                    si / (MAX_NUM_LEVELS * MAX_NUM_LEVELS),
                    (si / MAX_NUM_LEVELS) % MAX_NUM_LEVELS,
                    si % MAX_NUM_LEVELS
                };
                stack[warpTileIdx][stackSize+exclusiveActiveSum] = {
                    (uint)childIdx,
                    elem.level + 1,
                    elem.offset * make_uint3(MAX_NUM_LEVELS) + delta
                };
            }
            stackSize += activeSum;
        }
        __syncwarp();
    }

    if (tileThreadIdx == 0) {
        if (isCountingPass)
            intersect_counts[globalTileIdx] = numSplatIntersects[warpTileIdx];
        else {
            uint32_t idx = numSplatIntersects[warpTileIdx] + intersectGlobalOffset;
            while (idx < intersectGlobalOffsetMax) {
                intersectionSplatID[idx] = 0;
                ++idx;
            }
        }
    }
}


__global__ void getTileSplatIntersections_brute(
    const TileBuffers tiles, const SplatBuffers& splats,
    uint32_t* __restrict__ intersect_counts,  // to be filled or exclusive scan
    uint32_t* __restrict__ intersectionSplatID  // nullptr or to be filled
) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= tiles.size)
        return;

    bool isCountingPass = (intersectionSplatID == nullptr);
    uint32_t intersectGlobalOffset = 0, intersectGlobalOffsetMax = 0;
    if (!isCountingPass) {
        intersectGlobalOffset = intersect_counts[tid];
        intersectGlobalOffsetMax = intersect_counts[tid+1];
    }
    uint32_t intersectCount = 0;

    Tile tile = loadTile(tid, tiles);

    for (uint32_t sid = 0; sid < splats.size; sid++) {
        Splat splat = loadSplat(sid, splats);
        float overlap = tile.isOverlap(splat);
        if (overlap > 0.0) {
            uint32_t idx = intersectGlobalOffset + intersectCount;
            intersectCount += 1;
            if (idx < intersectGlobalOffsetMax) {
                intersectionSplatID[idx] = sid;
            }
        }
    }

    if (isCountingPass)
        intersect_counts[tid] = intersectCount;
    else {
        uint32_t idx = intersectCount + intersectGlobalOffset;
        while (idx < intersectGlobalOffsetMax) {
            intersectionSplatID[idx] = 0;
            ++idx;
        }
    }
}


__device__ __forceinline__ uint64_t getSplatSortingKey(
    uint level, uint3 pos
) {
    // 7 bit level, (64-7-1)/3=19 Morton bits in each dimension
    constexpr uint kMortonBitsPerDim = 19;
    uint64_t x = (uint64_t)(pos.x & ((1<<level)-1)) << (kMortonBitsPerDim - level);
    uint64_t y = (uint64_t)(pos.y & ((1<<level)-1)) << (kMortonBitsPerDim - level);
    uint64_t z = (uint64_t)(pos.z & ((1<<level)-1)) << (kMortonBitsPerDim - level);
    x = insert_2_zeros_between_bits(x) & (((uint64_t)1<<(3*kMortonBitsPerDim))-1);
    y = insert_2_zeros_between_bits(y) & (((uint64_t)1<<(3*kMortonBitsPerDim))-1);
    z = insert_2_zeros_between_bits(z) & (((uint64_t)1<<(3*kMortonBitsPerDim))-1);
    uint64_t morton = (x * 2 + y) * 2 + z;
    return ((uint64_t)level << (3*kMortonBitsPerDim)) | morton;
}

__global__ void fillSplatSortingKeys(
    const SplatBuffers& splats,
    float3 root_min, float3 root_max,
    unsigned num_levels,
    uint64_t* __restrict__ splat_keys
) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= splats.size) return;
    
    Splat g = loadSplat(tid, splats);
    float3 aabb_min, aabb_max;
    g.getAABB(aabb_min, aabb_max);
    float3 aabb_center = (aabb_min + aabb_max) * 0.5f;

    // unsigned level = getLevel<2>(aabb_min, aabb_max, root_min, root_max, num_levels);
    unsigned level = num_levels-1;

    float scale = exp2f(level);
    float3 cell_size = (root_max - root_min) / scale;
    uint3 cell = make_uint3((aabb_center - root_min) / cell_size + 0.5f);

    uint64_t key = getSplatSortingKey(level, cell);
    splat_keys[tid] = key;
}

__global__ void fillLbvhInternalNodes(
    unsigned num_splats,
    const uint64_t* __restrict__ morton,
    const int32_t* __restrict__ splat_idx,
    int2* __restrict__ internal_nodes,
    int32_t* __restrict__ parent_nodes
) {
    // https://developer.nvidia.com/blog/parallelforall/wp-content/uploads/2012/11/karras2012hpg_paper.pdf
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_splats-1) return;

    #define delta(i, j) \
        (((j)<0 || (j)>=num_splats) ? -1 : morton[i] == morton[j] ? 64 + __clz(i ^ j) : __clzll(morton[i] ^ morton[j]))

    // Determine direction of the range (+1 or -1)
    int d = delta(i,i+1) - delta(i,i-1);
    d = d > 0 ? 1 : d < 0 ? -1 : 0;

    // Compute upper bound for the length of the range
    int delta_min = delta(i, i-d);
    int lmax = 2;
    while (delta(i, i+lmax*d) > delta_min)
        lmax <<= 1;

    // Find the other end using binary search
    int l = 0;
    for (int t = lmax>>1; t >= 1; t >>= 1)
        if (delta(i, i+(l+t)*d) > delta_min)
            l += t;
    int j = i + l * d;

    // Find the split position using binary search
    int delta_node = delta(i, j);
    int s = 0;
    for (int tf = 2, t; (t = (l+tf-1)/tf) >= 1; tf <<= 1)
        if (delta(i, i+(s+t)*d) > delta_node)
            s += t;
    int gamma = i + s*d + min(d, 0);

    // Output child pointers
    // regular for internal node, bit flip for leaf
    int left = min(i,j) == gamma ? ~splat_idx[gamma] : gamma;
    int right = max(i,j) == gamma+1 ? ~splat_idx[gamma+1] : gamma+1;
    internal_nodes[i] = make_int2(left, right);

    // Output parent pointers
    if (left >= 0)
        atomicMax(&parent_nodes[left], i);
    if (right >= 0)
        atomicMax(&parent_nodes[right], i);

    #undef delta
}

__global__ void computeLbvhAABB(
    const SplatBuffers& splats,
    const int2* __restrict__ internal_nodes,
    const int32_t* __restrict__ parent_nodes,
    float3* __restrict__ treeAABB
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= splats.size-1) return;

    int2 children = internal_nodes[i];
    if (children.x >= 0 && children.y >= 0)
        return;

    // find splat AABB
    float3 aabb_min, aabb_max;
    if (children.x < 0)
        loadSplat(~children.x, splats).getAABB(aabb_min, aabb_max);
    if (children.y < 0) {
        float3 aabb_min1, aabb_max1;
        loadSplat(~children.y, splats).getAABB(aabb_min1, aabb_max1);
        if (children.x < 0)
            aabb_min = fmin(aabb_min, aabb_min1),
            aabb_max = fmax(aabb_max, aabb_max1);
        else
            aabb_min = aabb_min1, aabb_max = aabb_max1;
    }

    // fill parent AABB
    do {
        #if 0
        atomicMin(&treeAABB[2*i].x, aabb_min.x);
        atomicMin(&treeAABB[2*i].y, aabb_min.y);
        atomicMin(&treeAABB[2*i].z, aabb_min.z);
        atomicMax(&treeAABB[2*i+1].x, aabb_max.x);
        atomicMax(&treeAABB[2*i+1].y, aabb_max.y);
        atomicMax(&treeAABB[2*i+1].z, aabb_max.z);
        #else
        if (atomicMin(&treeAABB[2*i].x, aabb_min.x) < aabb_min.x &
            atomicMin(&treeAABB[2*i].y, aabb_min.y) < aabb_min.y &
            atomicMin(&treeAABB[2*i].z, aabb_min.z) < aabb_min.z &
            atomicMax(&treeAABB[2*i+1].x, aabb_max.x) > aabb_max.x &
            atomicMax(&treeAABB[2*i+1].y, aabb_max.y) > aabb_max.y &
            atomicMax(&treeAABB[2*i+1].z, aabb_max.z) > aabb_max.z
        ) break;
        #endif
    } while ((i = parent_nodes[i]) >= 0);

}


__global__ void getTileSplatIntersections_lbvh(
    const TileBuffers tiles, const SplatBuffers& splats,
    const int2* __restrict__ internal_nodes,
    float3* __restrict__ treeAABB,
    uint32_t* __restrict__ intersect_counts,  // to be filled or exclusive scan
    uint32_t* __restrict__ intersectionSplatID  // nullptr or to be filled
) {
    // one thread per tile
    unsigned tileIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tileIdx >= tiles.size)
        return;
    uint laneIdx = tileIdx % WARP_SIZE;

    bool isCountingPass = (intersectionSplatID == nullptr);
    uint32_t intersectGlobalOffset = 0, intersectGlobalOffsetMax = 0;
    if (!isCountingPass) {
        intersectGlobalOffset = intersect_counts[tileIdx];
        intersectGlobalOffsetMax = intersect_counts[tileIdx+1];
    }

    Tile tile = loadTile(tileIdx, tiles);

    struct StackElem {
        uint32_t nodeIdx;
    };
    constexpr uint MAX_STACK_SIZE = 8*sizeof(int32_t)+1;
    __shared__ StackElem stack[WARP_SIZE][MAX_STACK_SIZE];
    uint stackSize = 0;
    if (tile.isOverlap(treeAABB[0], treeAABB[1])) {
        stack[laneIdx][stackSize] = { 0 };
        stackSize++;
    }
    uint numSplatIntersects = 0;

    for (uint _num_steps = 0; _num_steps < 65536; _num_steps++) {
        if (stackSize == 0)
            break;

        --stackSize;
        StackElem elem = stack[laneIdx][stackSize];
        int2 node = internal_nodes[elem.nodeIdx];
        // printf("[%u] stack %u - node %d %d\n", _num_steps, stackSize, node.x, node.y);

        for (uint ci = 0; ci < 2; ci++) {
            int childIdx = ci == 0 ? node.x : node.y;
            // splat
            if (childIdx < 0) {
                int splatIdx = ~childIdx;
                Splat splat = loadSplat(splatIdx, splats);
                float overlap = tile.isOverlap(splat);
                if (overlap > 0.0) {
                    uint idx = numSplatIntersects + intersectGlobalOffset;
                    if (idx < intersectGlobalOffsetMax) {
                        intersectionSplatID[idx] = splatIdx;
                    }
                    numSplatIntersects += 1;
                }
            }
            // node
            else if (tile.isOverlap(treeAABB[2*childIdx+0], treeAABB[2*childIdx+1])
                    && stackSize < MAX_STACK_SIZE) {
                stack[laneIdx][stackSize] = { (uint)childIdx };
                stackSize += 1;
            }
        }

    }

    if (isCountingPass)
        intersect_counts[tileIdx] = numSplatIntersects;
    else {
        uint32_t idx = numSplatIntersects + intersectGlobalOffset;
        while (idx < intersectGlobalOffsetMax) {
            intersectionSplatID[idx] = 0;
            ++idx;
        }
    }
}


__global__ void getTileSplatIntersections_lbvh_warp(
    const TileBuffers tiles, const SplatBuffers& splats,
    const int2* __restrict__ internal_nodes,
    float3* __restrict__ treeAABB,
    uint32_t* __restrict__ intersect_counts,  // to be filled or exclusive scan
    uint32_t* __restrict__ intersectionSplatID  // nullptr or to be filled
) {
    // one warp per tile, blockDim.x must be warp size
    unsigned tileIdx = blockIdx.x;
    if (tileIdx >= tiles.size)
        return;
    unsigned laneIdx = threadIdx.x % WARP_SIZE;

    bool isCountingPass = (intersectionSplatID == nullptr);
    uint32_t intersectGlobalOffset = 0, intersectGlobalOffsetMax = 0;
    if (!isCountingPass) {
        intersectGlobalOffset = intersect_counts[tileIdx];
        intersectGlobalOffsetMax = intersect_counts[tileIdx+1];
    }

    Tile tile = loadTile(tileIdx, tiles);

    struct StackElem {
        uint32_t nodeIdx;
    };
    constexpr uint MAX_STACK_SIZE = (8*sizeof(int32_t)+1)*WARP_SIZE;
    __shared__ StackElem stack[MAX_STACK_SIZE];
    uint stackSize = 0;
    if (tile.isOverlap(treeAABB[0], treeAABB[1])) {
        if (laneIdx == 0)
            stack[stackSize] = { 0 };
        stackSize++;
    }
    __syncwarp();
    uint numSplatIntersects = 0;

    auto block = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> warp = cg::tiled_partition<WARP_SIZE>(block);

    for (uint _num_steps = 0; _num_steps < 65536; _num_steps++) {
        if (stackSize == 0)
            break;

        int elemIdx = (int)stackSize - (int)laneIdx - 1;
        bool active = elemIdx >= 0;

        StackElem elem;
        if (active) elem = stack[elemIdx];

        stackSize -= __syncthreads_count(active);

        #pragma unroll
        for (uint ci = 0; ci < 2; ci++) {

            bool hasSplat = false;
            bool hasNode = false;
            int splatIdx = 0;
            int nodeIdx = 0;

            if (active) {
                int2 node = internal_nodes[elem.nodeIdx];

                int childIdx = ci == 0 ? node.x : node.y;
                // splat
                if (childIdx < 0) {
                    splatIdx = ~childIdx;
                    Splat splat = loadSplat(splatIdx, splats);
                    float overlap = tile.isOverlap(splat);
                    if (overlap > 0.0)
                        hasSplat = true;
                }
                // node
                else if (tile.isOverlap(treeAABB[2*childIdx+0], treeAABB[2*childIdx+1])) {
                    nodeIdx = childIdx;
                    hasNode = true;
                }
            }

            // fill splats
            if (!isCountingPass) {
                int inclusiveScan = (int)hasSplat;
                #pragma unroll
                for (unsigned offset = 1; offset < WARP_SIZE; offset <<= 1) {
                    int temp = __shfl_up_sync(~0u, inclusiveScan, offset, WARP_SIZE);
                    if (laneIdx >= offset)
                        inclusiveScan += temp;
                }
                int exclusiveScan = inclusiveScan - (int)hasSplat;

                uint idx = (numSplatIntersects + exclusiveScan) + intersectGlobalOffset;
                if (hasSplat && idx < intersectGlobalOffsetMax)
                    intersectionSplatID[idx] = splatIdx;
            }
            numSplatIntersects += __syncthreads_count(hasSplat);

            // fill nodes
            {
                int inclusiveScan = (int)hasNode;
                #pragma unroll
                for (unsigned offset = 1; offset < WARP_SIZE; offset <<= 1) {
                    int temp = __shfl_up_sync(~0u, inclusiveScan, offset, WARP_SIZE);
                    if (laneIdx >= offset)
                        inclusiveScan += temp;
                }
                int exclusiveScan = inclusiveScan - (int)hasNode;

                uint idx = stackSize + exclusiveScan;
                if (hasNode && idx < MAX_STACK_SIZE)
                    stack[idx] = { (uint)nodeIdx };
            }
            stackSize += __syncthreads_count(hasNode);
            stackSize = min(stackSize, MAX_STACK_SIZE);
        }

    }

    if (isCountingPass)
        intersect_counts[tileIdx] = numSplatIntersects;
    else {
        uint32_t idx = numSplatIntersects + intersectGlobalOffset;
        while (idx < intersectGlobalOffsetMax) {
            if (idx + laneIdx < intersectGlobalOffsetMax)
                intersectionSplatID[idx + laneIdx] = 0;
            idx += WARP_SIZE;
        }
    }
}


__forceinline__ torch::Tensor exclusiveScan(torch::Tensor &tensor) {
    torch::Tensor result = torch::empty_like(tensor);
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(nullptr, temp_storage_bytes,
        (unsigned*)tensor.data_ptr<int32_t>(),
        (unsigned*)result.data_ptr<int32_t>(),
        tensor.size(0));
    torch::Tensor temp_storage = torch::empty({(long)temp_storage_bytes}, tensor.options().dtype(torch::kUInt8));
    hipcub::DeviceScan::ExclusiveSum(temp_storage.data_ptr<uint8_t>(), temp_storage_bytes,
        (unsigned*)tensor.data_ptr<int32_t>(),
        (unsigned*)result.data_ptr<int32_t>(),
        tensor.size(0));
    return result;
}

__forceinline__ torch::Tensor invertPermutation(torch::Tensor &tensor) {
    constexpr uint block = 256;
    torch::Tensor result = torch::empty_like(tensor);
    invertPermutation<int32_t><<<(tensor.size(0)+block-1)/block, block>>>(
        tensor.size(0),
        tensor.data_ptr<int32_t>(),
        result.data_ptr<int32_t>()
    );
    return result;
}

template<typename T>
void _print_tensor(std::string name, torch::Tensor tensor) {
    hipDeviceSynchronize();
    printf("%s ", name.c_str());
    // printf("\n"); return;
    tensor = tensor.cpu();
    if (tensor.ndimension() == 1) {
        for (unsigned i = 0; i < tensor.size(0); i++)
            printf("%lg ", (double)tensor.data_ptr<T>()[i]);
    }
    else if (tensor.ndimension() == 2) {
        for (unsigned i = 0; i < tensor.size(0); i++) {
            for (unsigned j = 0; j < tensor.size(1); j++)
                printf("%lg ", (double)tensor.data_ptr<T>()[i*tensor.size(1)+j]);
            printf(" ");
        }
    }
    printf("\n");
}

// #define print_tensor(dtype, tensor) _print_tensor<dtype>(#tensor, tensor)
#define print_tensor(dtype, tensor) ;


#ifdef DEBUG
void printAABB(float3 p0, float3 p1) {
    float3 c = -0.5f*(p0+p1), r = -0.5f*(p1-p0);
    printf("\\max\\left(\\left|x%+f\\right|%+f,\\left|y%+f\\right|%+f,\\left|z%+f\\right|%+f\\right)=0\n", c.x, r.x, c.y, r.y, c.z, r.z);
}

void printAABB_wireframe(float3 p0, float3 p1) {
    // B\left(x_{0},y_{0},z_{0},x_{1},y_{1},z_{1},t\right)=\left[\left(x_{0},y_{0},z_{0}\right),\left(x_{1},y_{0},z_{0}\right),\left(x_{1},y_{1},z_{0}\right),\left(x_{0},y_{1},z_{0}\right),\left(x_{0},y_{0},z_{0}\right),\left(x_{0},y_{1},z_{0}\right),\left(x_{0},y_{1},z_{1}\right),\left(x_{0},y_{0},z_{1}\right),\left(x_{0},y_{0},z_{0}\right),\left(x_{1},y_{0},z_{0}\right),\left(x_{1},y_{0},z_{1}\right),\left(x_{0},y_{0},z_{1}\right)\right]\left(1-t\right)+\left[\left(x_{0},y_{0},z_{1}\right),\left(x_{1},y_{0},z_{1}\right),\left(x_{1},y_{1},z_{1}\right),\left(x_{0},y_{1},z_{1}\right),\left(x_{1},y_{0},z_{0}\right),\left(x_{1},y_{1},z_{0}\right),\left(x_{1},y_{1},z_{1}\right),\left(x_{1},y_{0},z_{1}\right),\left(x_{0},y_{1},z_{0}\right),\left(x_{1},y_{1},z_{0}\right),\left(x_{1},y_{1},z_{1}\right),\left(x_{0},y_{1},z_{1}\right)\right]t
    printf("B\\left(%f,%f,%f,%f,%f,%f,t\\right)\n", p0.x, p0.y, p0.z, p1.x, p1.y, p1.z);
}

void printTile(glm::vec3 ro, glm::vec3 rd) {
    rd = glm::normalize(rd);
    printf("\\left(%f,%f,%f\\right)+\\left(%f,%f,%f\\right)100t\n",
        ro.x, ro.y, ro.z, rd.x, rd.y, rd.z);
}

void printCells(torch::Tensor cell_keys, torch::Tensor splat_ids, torch::Tensor subcell_masks, torch::Tensor subcell_ids) {
    int num_cells = cell_keys.size(0);
    printf("%d overlaps\n", num_cells);
    for (int i = 0; i < num_cells; i++) {
        uint64_t cell_key = cell_keys[i].item().toUInt64();
        int32_t splat_id = splat_ids[i].item().toInt();
        uint8_t subcell_mask = subcell_masks[i].item().toByte();
        int32_t subcell_id = subcell_ids[i].item().toInt();
        printf("[%d] layer %d, offset %llx, alias %b, splat %d, subcell %03b, %d\n", i, (int)(cell_key>>58), cell_key>>1, (unsigned)(cell_key&1), splat_id, (unsigned)(subcell_mask==0?0:31-__builtin_clz((unsigned)subcell_mask)), subcell_id);
    }
}

void checkMatch(std::string name, const torch::Tensor &a, const torch::Tensor &b) {
    if (a.size(0) != b.size(0)) {
        printf("%s: shape mismatch (%d != %d)\n", name.c_str(), (int)a.size(0), (int)b.size(0));
        return;
    }
    int numdiff = torch::abs(a - b).clip(0, 1).sum().item<int>();
    printf("%s: %d / %d mismatch\n", name.c_str(), numdiff, (int)a.size(0));
}
#endif


SplatTileIntersector::SplatTileIntersector(
    c10::TensorOptions tensorOptions,
    const SplatBuffers &splats,
    TileBuffers tiles
) : splats(splats), tiles(tiles)
{
    tensorF32 = tensorOptions.dtype(torch::kFloat32);
    tensorI32 = tensorOptions.dtype(torch::kInt32);
    tensorI16 = tensorOptions.dtype(torch::kInt16);
    tensorI64 = tensorOptions.dtype(torch::kInt64);
    tensorU8 = tensorOptions.dtype(torch::kUInt8);

    #ifdef DEBUG
    std::chrono::system_clock::time_point t0, t1;

    for (int i = 0; i < 1000; i++) {
        t0 = std::chrono::high_resolution_clock::now();
        // auto [icm1, sid1] = getIntersections_octree<12, 2>();
        auto [icm1, sid1] = getIntersections_lbvh();
        t1 = std::chrono::high_resolution_clock::now();
        printf("tree: %.2f ms\n", std::chrono::duration<float>(t1-t0).count()*1e3f);

        // continue;

        t0 = std::chrono::high_resolution_clock::now();
        auto [icm0, sid0] = getIntersections_brute();
        t1 = std::chrono::high_resolution_clock::now();
        printf("brute: %.2f ms\n", std::chrono::duration<float>(t1-t0).count()*1e3f);

        icm0 = icm0.cpu();
        icm1 = icm1.cpu();
        sid0 = sid0.cpu();
        sid1 = sid1.cpu();
        for (int k = 0; k+1 < icm0.size(0); k++) {
            int i0 = icm0[k].item<int>();
            int i1 = icm0[k+1].item<int>();
            std::sort(sid0.data_ptr<int>()+i0, sid0.data_ptr<int>()+i1);
            i0 = icm1[k].item<int>();
            i1 = icm1[k+1].item<int>();
            std::sort(sid1.data_ptr<int>()+i0, sid1.data_ptr<int>()+i1);
        }
        checkMatch("icm", icm1, icm0);
        checkMatch("sid", sid1, sid0);
    }
    exit(0);
    #endif
}

std::tuple<torch::Tensor, torch::Tensor> SplatTileIntersector::getIntersections_brute() {
    constexpr unsigned warp = 32;

    torch::Tensor intersection_count = torch::zeros({tiles.size+1}, tensorI32);
    getTileSplatIntersections_brute<<<(tiles.size+warp-1)/warp, warp>>>(
        tiles, splats,
        (uint32_t*)intersection_count.data_ptr<int32_t>(),
        nullptr
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    hipDeviceSynchronize();
    print_tensor(int, intersection_count);

    torch::Tensor intersection_count_map = exclusiveScan(intersection_count);
    print_tensor(int, intersection_count_map);
    unsigned total_intersections = (unsigned)intersection_count_map[tiles.size].item<int32_t>();

    torch::Tensor intersectionSplatID = torch::empty({total_intersections}, tensorI32);
    getTileSplatIntersections_brute<<<(tiles.size+warp-1)/warp, warp>>>(
        tiles, splats,
        (uint32_t*)intersection_count_map.data_ptr<int32_t>(),
        (uint32_t*)intersectionSplatID.data_ptr<int32_t>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    print_tensor(int32_t, intersectionSplatID);

    return std::make_tuple(intersection_count_map, intersectionSplatID);
}


std::tuple<torch::Tensor, torch::Tensor> SplatTileIntersector::getIntersections_octree() {
    constexpr uint MAX_NUM_LEVELS = 12;
    constexpr uint BRANCH_FACTOR = 2;

    static_assert(MAX_NUM_LEVELS < 16);
    static_assert(BRANCH_FACTOR == 2 || BRANCH_FACTOR == 3 || BRANCH_FACTOR == 4);

    constexpr unsigned block = 256;
    constexpr unsigned warp = 32;
    constexpr int kFloatPInfByte = 0x7f;  // 0x7f7f7f7f -> 3.39615e+38
    constexpr int kFloatNInfByte = 0xfe;  // 0xfefefefe -> -1.69474e+38

    // find splat AABB
    // torch::Tensor splat_aabb = torch::empty({splats.size, 2, 3}, tensorF32);
    torch::Tensor root_aabb_tensor = torch::empty({2, 3}, tensorF32);
    hipMemset(root_aabb_tensor.data_ptr<float>()+0, kFloatPInfByte, 3*sizeof(float));
    hipMemset(root_aabb_tensor.data_ptr<float>()+3, kFloatNInfByte, 3*sizeof(float));
    computeSplatAABB<<<(splats.size+block-1)/block, block>>>(
        splats,
        // (float3*)splat_aabb.data_ptr<float>(),
        nullptr,
        (float3*)root_aabb_tensor.data_ptr<float>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    #if 0
    {
        torch::Tensor splat_aabb_cpu = splat_aabb.cpu();
        for (int i = 0; i < splats.size; i++) {
            float3* ps = (float3*)splat_aabb_cpu.data_ptr<float>() + 2*i;
            printAABB(ps[0], ps[1]);
        }
    }
    #endif

    // find root AABB, pad them to cubes
    {
        root_aabb_tensor = root_aabb_tensor.cpu();
        float3* root_aabb = (float3*)root_aabb_tensor.data_ptr<float>();
        rootAABBMin = root_aabb[0];
        rootAABBMax = root_aabb[1];
        float3 center = 0.5f * (rootAABBMax + rootAABBMin);
        float3 extend = 0.5f * (rootAABBMax - rootAABBMin);
        float max_size = 1.01f * fmax(extend.x, fmax(extend.y, extend.z));
        rootAABBMin = center - make_float3(max_size);
        rootAABBMax = center + make_float3(max_size);        
    }
    // printAABB(rootAABBMin, rootAABBMax);
    // printf("%f %f %f  %f %f %f\n", rootAABBMin.x, rootAABBMin.y, rootAABBMin.z, rootAABBMax.x, rootAABBMax.y, rootAABBMax.z);

    // determine number of levels
    unsigned numLevels = MAX_NUM_LEVELS;
    
    // count number of cell overlaps for each splat
    torch::Tensor splat_cell_overlap_counts = torch::zeros({splats.size+1}, tensorI32);
    countCellOverlaps<BRANCH_FACTOR><<<(splats.size+block-1)/block, block>>>(
        splats, rootAABBMin, rootAABBMax, numLevels,
        (unsigned*)splat_cell_overlap_counts.data_ptr<int32_t>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    print_tensor(int32_t, splat_cell_overlap_counts);

    // prefix sum to get offsets
    torch::Tensor splat_cell_overlap_offsets = exclusiveScan(splat_cell_overlap_counts);
    print_tensor(int32_t, splat_cell_overlap_offsets);
    unsigned total_overlaps = (unsigned)splat_cell_overlap_offsets[splats.size].item<int32_t>();
    // printf("%d\n", (int)total_overlaps);

    // fill overlap data
    torch::Tensor cell_keys = torch::zeros({total_overlaps}, tensorI64);
    torch::Tensor splat_ids = torch::zeros({total_overlaps}, tensorI32);
    torch::Tensor subcell_masks = torch::zeros({total_overlaps}, tensorU8);
    torch::Tensor subcell_ids = torch::zeros({total_overlaps}, tensorI32);
    torch::Tensor subcell_aabb = torch::zeros({total_overlaps, 2, 3}, tensorF32);
    fillCellOverlaps<BRANCH_FACTOR><<<(splats.size+block-1)/block, block>>>(
        splats, rootAABBMin, rootAABBMax, numLevels,
        (unsigned*)splat_cell_overlap_offsets.data_ptr<int32_t>(),
        (uint64_t*)cell_keys.data_ptr<int64_t>(),
        splat_ids.data_ptr<int32_t>(),
        subcell_masks.data_ptr<uint8_t>(),
        subcell_ids.data_ptr<int32_t>(),
        (float3*)subcell_aabb.data_ptr<float>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    // printCells(cell_keys, splat_ids, subcell_masks, subcell_ids);

    // sort cells by keys
    // thrust::device_ptr<uint64_t> keys_ptr((uint64_t*)cell_keys.data_ptr<int64_t>());
    // thrust::device_ptr<int32_t> vals1_ptr(splat_ids.data_ptr<int32_t>());
    // thrust::device_ptr<uint64_t> vals2_ptr((uint64_t*)subcell_masks.data_ptr<int64_t>());
    // thrust::device_ptr<int32_t> vals3_ptr(subcell_ids.data_ptr<int32_t>());

    // auto first = thrust::make_zip_iterator(thrust::make_tuple(vals1_ptr, vals2_ptr, vals3_ptr));
    // thrust::sort_by_key(keys_ptr, keys_ptr + total_overlaps, first);

    auto [cell_keys_sorted, cell_keys_argsort] = torch::sort(cell_keys);
    cell_keys_argsort = cell_keys_argsort.to(torch::kInt32);
    cell_keys = cell_keys_sorted;
    splat_ids = torch::index(splat_ids, {cell_keys_argsort});
    subcell_masks = torch::index(subcell_masks, {cell_keys_argsort});
    subcell_aabb = torch::index(subcell_aabb, {cell_keys_argsort});
    #if 0
    subcell_ids = torch::index(subcell_ids, {cell_keys_argsort});
    subcell_ids = torch::where(subcell_ids == -1, subcell_ids, torch::index(cell_keys_argsort.argsort(), {subcell_ids})).contiguous().to(torch::kInt32);
    #else
    torch::Tensor cell_keys_argsort_argsort = invertPermutation(cell_keys_argsort);
    CHECK_DEVICE_ERROR(hipGetLastError());
    torch::Tensor new_subcell_ids = torch::empty_like(subcell_ids);
    gatherAndRemap<<<(total_overlaps+block-1)/block, block>>>(
        total_overlaps,
        subcell_ids.data_ptr<int32_t>(),
        cell_keys_argsort.data_ptr<int32_t>(),
        cell_keys_argsort_argsort.data_ptr<int32_t>(),
        new_subcell_ids.data_ptr<int32_t>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    subcell_ids = new_subcell_ids;
    #endif
    
    print_tensor(long, cell_keys_argsort);
    print_tensor(long, cell_keys_argsort.argsort());
    // printCells(cell_keys, splat_ids, subcell_masks, subcell_ids);

    // Get number of cells and splats as well as index map
    torch::Tensor cell_id_differential_map = torch::empty({total_overlaps+1}, tensorI32);
    torch::Tensor is_splat_map = torch::empty({total_overlaps+1}, tensorI32);
    getCellDifferential<<<(total_overlaps+block-1)/block, block>>>(
        total_overlaps,
        (uint64_t*)cell_keys.data_ptr<int64_t>(),
        cell_id_differential_map.data_ptr<int32_t>(),
        is_splat_map.data_ptr<int32_t>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());

    torch::Tensor cell_id_map = exclusiveScan(cell_id_differential_map);
    torch::Tensor splat_idx_map = exclusiveScan(is_splat_map);
    unsigned total_cells = (unsigned)cell_id_map[total_overlaps].item<int32_t>();
    unsigned total_splat_overlaps = (unsigned)splat_idx_map[total_overlaps].item<int32_t>();
    // print_tensor(int, cell_id_differential_map);
    print_tensor(int, cell_id_map);
    // print_tensor(int, is_splat_map);
    print_tensor(int, splat_idx_map);
    // printf("%d cells, %d splat overlaps\n", (int)total_cells, (int)total_splat_overlaps);

    // Fill splats
    torch::Tensor splatRanges = torch::zeros({total_cells, 2}, tensorI32);
    torch::Tensor splatIndices = torch::empty({total_splat_overlaps}, tensorI32);

    torch::Tensor treeAABB = torch::empty({total_cells, 2, 3}, tensorF32);
    fillTreeSplats<<<(total_overlaps+block-1)/block, block>>>(
        total_overlaps,
        (uint64_t*)cell_keys.data_ptr<int64_t>(),
        splat_ids.data_ptr<int32_t>(),
        (unsigned*)splat_idx_map.data_ptr<int32_t>(),
        (unsigned*)cell_id_map.data_ptr<int32_t>(),
        (unsigned*)splatRanges.data_ptr<int32_t>(),
        (unsigned*)splatIndices.data_ptr<int32_t>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    print_tensor(int, splatRanges);
    print_tensor(int, splatIndices);
    print_tensor(uint8_t, subcell_masks);

    // Fill subcell map
    constexpr unsigned B3 = BRANCH_FACTOR * BRANCH_FACTOR * BRANCH_FACTOR;
    torch::Tensor children = torch::empty({total_cells, B3}, tensorI32);
    hipMemset(children.data_ptr<int32_t>(), 0xff, total_cells*B3*sizeof(int32_t));
    
    print_tensor(int, cell_id_map);
    // print_tensor(int, torch::arange(cell_id_map.size(0)).cuda().to(torch::kInt32) - cell_id_map);
    print_tensor(int, subcell_ids);
    #if 0
    fillTreeSubcells_perCell<BRANCH_FACTOR><<<(total_cells+block-1)/block, block>>>(
        total_overlaps, total_cells,
        (unsigned*)cell_id_map.data_ptr<int32_t>(),
        subcell_ids.data_ptr<int32_t>(),
        subcell_masks.data_ptr<uint8_t>(),
        (float3*)subcell_aabb.data_ptr<float>(),
        children.data_ptr<int32_t>(),
        (float3*)treeAABB.data_ptr<float>()
    );
    #else
    fillTreeSubcells_initAABB<<<(total_cells+block-1)/block, block>>>(
        total_cells,
        (float3*)treeAABB.data_ptr<float>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    fillTreeSubcells_perOverlap<BRANCH_FACTOR><<<(total_overlaps+block-1)/block, block>>>(
        total_overlaps, total_cells,
        (unsigned*)cell_id_map.data_ptr<int32_t>(),
        subcell_ids.data_ptr<int32_t>(),
        subcell_masks.data_ptr<uint8_t>(),
        (float3*)subcell_aabb.data_ptr<float>(),
        children.data_ptr<int32_t>(),
        (float3*)treeAABB.data_ptr<float>()
    );
    #endif
    CHECK_DEVICE_ERROR(hipGetLastError());
    print_tensor(int, children);
    hipDeviceSynchronize();
    // std::cout << treeAABB << std::endl;
    #ifdef DEBUG
    if (0) {
        torch::Tensor treeAABB_cpu = treeAABB.cpu();
        for (int i = 0; i < total_cells; i++) {
            float3* p = (float3*)treeAABB_cpu.data_ptr<float>() + 2*i;
            printAABB_wireframe(p[0], p[1]);
        }
    }
    #endif
    // return;

    // Traverse tree - get counts
    torch::Tensor intersection_count = torch::zeros({tiles.size+1}, tensorI32);
    print_tensor(int, intersection_count);
    getTileSplatIntersections_octree<MAX_NUM_LEVELS, BRANCH_FACTOR>
    <<<(tiles.size*B3+warp-1)/warp, warp>>>(
        tiles, splats, rootAABBMin, rootAABBMax,
        children.data_ptr<int32_t>(),
        (float3*)treeAABB.data_ptr<float>(),
        (unsigned*)splatRanges.data_ptr<int32_t>(),
        (unsigned*)splatIndices.data_ptr<int32_t>(),
        (uint32_t*)intersection_count.data_ptr<int32_t>(),
        nullptr
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    hipDeviceSynchronize();
    print_tensor(int, intersection_count);

    // Traverse tree - get offsets
    torch::Tensor intersection_count_map = exclusiveScan(intersection_count);
    print_tensor(int, intersection_count_map);
    unsigned total_intersections = (unsigned)intersection_count_map[tiles.size].item<int32_t>();

    // Traverse tree - write data
    torch::Tensor intersectionSplatID = torch::empty({total_intersections}, tensorI32);
    getTileSplatIntersections_octree<MAX_NUM_LEVELS, BRANCH_FACTOR>
    <<<(tiles.size*B3+warp-1)/warp, warp>>>(
        tiles, splats, rootAABBMin, rootAABBMax,
        children.data_ptr<int32_t>(),
        (float3*)treeAABB.data_ptr<float>(),
        (unsigned*)splatRanges.data_ptr<int32_t>(),
        (unsigned*)splatIndices.data_ptr<int32_t>(),
        (uint32_t*)intersection_count_map.data_ptr<int32_t>(),
        (uint32_t*)intersectionSplatID.data_ptr<int32_t>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    print_tensor(int32_t, intersectionSplatID);

    return std::make_tuple(intersection_count_map, intersectionSplatID);
}


std::tuple<torch::Tensor, torch::Tensor> SplatTileIntersector::getIntersections_lbvh() {
    constexpr uint MAX_NUM_LEVELS = 12;

    static_assert(MAX_NUM_LEVELS < 32);

    constexpr unsigned block = 256;
    constexpr unsigned warp = 32;
    constexpr int kFloatPInfByte = 0x7f;  // 0x7f7f7f7f -> 3.39615e+38
    constexpr int kFloatNInfByte = 0xfe;  // 0xfefefefe -> -1.69474e+38

    // find splat AABB
    torch::Tensor splat_aabb = torch::empty({splats.size, 2, 3}, tensorF32);
    torch::Tensor root_aabb_tensor = torch::empty({2, 3}, tensorF32);
    hipMemset(root_aabb_tensor.data_ptr<float>()+0, kFloatPInfByte, 3*sizeof(float));
    hipMemset(root_aabb_tensor.data_ptr<float>()+3, kFloatNInfByte, 3*sizeof(float));
    computeSplatAABB<<<(splats.size+block-1)/block, block>>>(
        splats,
        (float3*)splat_aabb.data_ptr<float>(),
        (float3*)root_aabb_tensor.data_ptr<float>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    #ifdef DEBUG
    if (0) {
        torch::Tensor splat_aabb_cpu = splat_aabb.cpu();
        for (int i = 0; i < splats.size; i++) {
            float3* ps = (float3*)splat_aabb_cpu.data_ptr<float>() + 2*i;
            printAABB(ps[0], ps[1]);
        }
    }
    #endif

    // find root AABB, pad them to cubes
    {
        root_aabb_tensor = root_aabb_tensor.cpu();
        float3* root_aabb = (float3*)root_aabb_tensor.data_ptr<float>();
        rootAABBMin = root_aabb[0];
        rootAABBMax = root_aabb[1];
        float3 center = 0.5f * (rootAABBMax + rootAABBMin);
        float3 extend = 0.5f * (rootAABBMax - rootAABBMin);
        float max_size = 1.01f * fmax(extend.x, fmax(extend.y, extend.z));
        rootAABBMin = center - make_float3(max_size);
        rootAABBMax = center + make_float3(max_size);
    }
    // printAABB_wireframe(rootAABBMin, rootAABBMax);

    // compute sorting keys (level and Morton code)
    torch::Tensor morton = torch::empty({splats.size}, tensorI64);
    fillSplatSortingKeys<<<(splats.size+block-1)/block, block>>>(
        splats, rootAABBMin, rootAABBMax, MAX_NUM_LEVELS,
        (uint64_t*)morton.data_ptr<int64_t>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    hipDeviceSynchronize();

    auto [sorted_morton, splat_argsort] = torch::sort(morton);
    splat_argsort = splat_argsort.to(torch::kInt32);

    #ifdef DEBUG
    if (0) {
        splat_aabb = torch::index(splat_aabb, {splat_argsort});
        // torch::Tensor splat_idx = invertPermutation(splat_argsort);
        torch::Tensor splat_aabb_cpu = splat_aabb.cpu();
        float3* aabb = (float3*)splat_aabb_cpu.data_ptr<float>();
        printf("\\left[");
        for (int i = 0; i < splats.size; i++) {
            float3 p = 0.5f*(aabb[2*i]+aabb[2*i+1]);
            printf("\\left(%f,%f,%f\\right),", p.x, p.y, p.z);
        }
        printf("\b\\right]\n");
    }
    #endif

    // Build tree
    torch::Tensor internal_nodes = torch::empty({splats.size-1, 2}, tensorI32);
    torch::Tensor parent_nodes = torch::empty({splats.size-1}, tensorI32);
    hipMemset(parent_nodes.data_ptr<int32_t>(), 0xff, (splats.size-1)*sizeof(int32_t));
    fillLbvhInternalNodes<<<((splats.size-1)+block-1)/block, block>>>(
        splats.size,
        (uint64_t*)sorted_morton.data_ptr<int64_t>(),
        splat_argsort.data_ptr<int32_t>(),
        (int2*)internal_nodes.data_ptr<int32_t>(),
        parent_nodes.data_ptr<int32_t>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    hipDeviceSynchronize();
    print_tensor(int, internal_nodes);
    print_tensor(int, parent_nodes);

    // Compute AABB
    torch::Tensor treeAABB = torch::empty({splats.size, 2, 3}, tensorF32);
    fillTreeSubcells_initAABB<<<((splats.size-1)+block-1)/block, block>>>(
        splats.size-1,
        (float3*)treeAABB.data_ptr<float>()
    );
    computeLbvhAABB<<<((splats.size-1)+block-1)/block, block>>>(
        splats,
        (int2*)internal_nodes.data_ptr<int32_t>(),
        parent_nodes.data_ptr<int32_t>(),
        (float3*)treeAABB.data_ptr<float>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    hipDeviceSynchronize();
    #ifdef DEBUG
    if (0) {
        torch::Tensor treeAABB_cpu = treeAABB.cpu();
        for (int i = 0; i < splats.size-1; i++) {
            float3* p = (float3*)treeAABB_cpu.data_ptr<float>() + 2*i;
            printAABB_wireframe(p[0], p[1]);
        }
    }
    #endif

    // Traverse to find intersections
    torch::Tensor intersection_count = torch::zeros({tiles.size+1}, tensorI32);
    // getTileSplatIntersections_lbvh<<<(tiles.size+warp-1)/warp, warp>>>(
    getTileSplatIntersections_lbvh_warp<<<tiles.size, warp>>>(
        tiles, splats,
        (int2*)internal_nodes.data_ptr<int32_t>(),
        (float3*)treeAABB.data_ptr<float>(),
        (uint32_t*)intersection_count.data_ptr<int32_t>(),
        nullptr
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    hipDeviceSynchronize();
    print_tensor(int, intersection_count);

    torch::Tensor intersection_count_map = exclusiveScan(intersection_count);
    hipDeviceSynchronize();
    print_tensor(int, intersection_count_map);
    unsigned total_intersections = (unsigned)intersection_count_map[tiles.size].item<int32_t>();

    torch::Tensor intersectionSplatID = torch::empty({total_intersections}, tensorI32);
    // getTileSplatIntersections_lbvh<<<(tiles.size+warp-1)/warp, warp>>>(
    getTileSplatIntersections_lbvh_warp<<<tiles.size, warp>>>(
        tiles, splats,
        (int2*)internal_nodes.data_ptr<int32_t>(),
        (float3*)treeAABB.data_ptr<float>(),
        (uint32_t*)intersection_count_map.data_ptr<int32_t>(),
        (uint32_t*)intersectionSplatID.data_ptr<int32_t>()
    );
    CHECK_DEVICE_ERROR(hipGetLastError());
    hipDeviceSynchronize();
    print_tensor(int32_t, intersectionSplatID);

    return std::make_tuple(intersection_count_map, intersectionSplatID);
}


#ifdef DEBUG

#include <stdio.h>

#include <regex>
#include <filesystem>
#include <fstream>

torch::Tensor loadBinaryToTensor(const std::string& filepath) {
    namespace fs = std::filesystem;
    
    // Extract filename from path
    fs::path p(filepath);
    std::string filename = p.filename().string();
    
    // Remove extension if present
    size_t dot_pos = filename.rfind('.');
    if (dot_pos != std::string::npos) {
        filename = filename.substr(0, dot_pos);
    }
    
    // Parse dimensions from filename using regex
    // Pattern: alphanumeric_name followed by numbers separated by underscores
    std::regex pattern(R"(^[a-zA-Z0-9]+_(.+)$)");
    std::smatch match;
    
    if (!std::regex_match(filename, match, pattern)) {
        throw std::runtime_error("Filename does not match expected pattern");
    }
    
    std::string dims_str = match[1].str();
    
    // Parse dimension values
    std::vector<int64_t> shape;
    std::stringstream ss(dims_str);
    std::string token;
    
    while (std::getline(ss, token, '_')) {
        if (!token.empty()) {
            try {
                shape.push_back(std::stoll(token));
            } catch (const std::exception& e) {
                throw std::runtime_error("Failed to parse dimension: " + token);
            }
        }
    }
    
    if (shape.empty()) {
        throw std::runtime_error("No dimensions found in filename");
    }
    
    // Calculate total number of elements
    int64_t total_elements = 1;
    for (int64_t dim : shape) {
        total_elements *= dim;
    }
    
    // Read binary file
    std::ifstream file(filepath, std::ios::binary | std::ios::ate);
    if (!file.is_open()) {
        throw std::runtime_error("Failed to open file: " + filepath);
    }
    
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    
    int64_t expected_bytes = total_elements * sizeof(float);
    if (file_size != expected_bytes) {
        throw std::runtime_error(
            "File size mismatch. Expected " + std::to_string(expected_bytes) +
            " bytes, but file contains " + std::to_string(file_size) + " bytes"
        );
    }
    
    // Allocate buffer and read data
    auto buffer = std::make_unique<float[]>(total_elements);
    file.read(reinterpret_cast<char*>(buffer.get()), file_size);
    file.close();
    
    if (!file) {
        throw std::runtime_error("Failed to read file completely");
    }
    
    // Create torch tensor on CUDA device
    torch::Tensor tensor = torch::from_blob(
        buffer.get(),
        shape,
        torch::kFloat32
    ).clone();
    
    return tensor.to(torch::kCUDA);
}

int main(int argc, char** argv) {
    auto seed = std::stoll(argv[3]);
    torch::manual_seed(seed);

    // unsigned num_splat = std::stod(argv[1]);
    // torch::Tensor means = torch::randn({num_splat, 3}).cuda();
    // torch::Tensor scales = 0.2*torch::randn({num_splat, 3}).cuda() - 0.4f*logf(num_splat) - 0.0f;
    // torch::Tensor opacs = torch::randn({num_splat, 1}).cuda();
    // torch::Tensor quats = torch::randn({num_splat, 4}).cuda();
    // unsigned num_tiles = std::stod(argv[2]);
    // auto [viewmats, Ks] = generate_random_camera_poses(num_tiles, seed);

    // if (0) {
    //     torch::Tensor tile_ro_cpu = tile_apex.cpu();
    //     torch::Tensor tile_rd_cpu = tile_dirs.cpu();
    //     for (unsigned i = 0; i < num_tiles; i++)
    //         printTile(((glm::vec3*)tile_ro_cpu.data_ptr<float>())[i],
    //         ((glm::vec3*)tile_rd_cpu.data_ptr<float>())[i]);
    // }

    // scales = torch::exp(scales);
    // opacs = torch::sigmoid(opacs);

    torch::Tensor means = loadBinaryToTensor("means_765390_3.bin");
    torch::Tensor scales = loadBinaryToTensor("scales_765390_3.bin");
    torch::Tensor opacs = loadBinaryToTensor("opacities_765390.bin");
    torch::Tensor quats = loadBinaryToTensor("quats_765390_4.bin");
    torch::Tensor viewmats = loadBinaryToTensor("viewmats_672_4_4.bin");
    torch::Tensor Ks = loadBinaryToTensor("Ks_672_3_3.bin");

    SplatTileIntersector::intersect_splat_tile(
        means, scales, opacs, quats,
        TILE_SIZE, TILE_SIZE,
        viewmats, Ks
    );
    return 0;
}

#endif


// /usr/local/cuda-12.8/bin/nvcc -I/media/harry/d/gs/spirulae-splat/spirulae_splat/splat/cuda/csrc/glm -I/home/harry/.venv/base/lib/python3.12/site-packages/torch/include -I/home/harry/.venv/base/lib/python3.12/site-packages/torch/include/torch/csrc/api/include -I/usr/local/cuda-12.8/include /media/harry/d/gs/spirulae-splat/spirulae_splat/splat/cuda/csrc/SplatTileIntersector.cu -o ./temp -D__CUDA_NO_HALF_OPERATORS__ -D__CUDA_NO_HALF_CONVERSIONS__ -D__CUDA_NO_BFLOAT16_CONVERSIONS__ -D__CUDA_NO_HALF2_OPERATORS__ -DDEBUG --expt-relaxed-constexpr --compiler-options ''"'"'-fPIC'"'"'' -O3 --use_fast_math --expt-relaxed-constexpr -Xcudafe=--diag_suppress=20012 -Xcudafe=--diag_suppress=550 -DTORCH_API_INCLUDE_EXTENSION_H -gencode=arch=compute_120,code=compute_120 -gencode=arch=compute_120,code=sm_120 -std=c++17 -L/home/harry/.venv/base/lib/python3.12/site-packages/torch/lib -L/usr/local/cuda-12.8/lib64 -L/usr/lib/x86_64-linux-gnu -lc10 -ltorch -ltorch_cpu -lcudart -lc10_cuda -ltorch_cuda

