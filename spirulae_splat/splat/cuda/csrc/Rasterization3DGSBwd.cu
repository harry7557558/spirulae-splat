#include "hip/hip_runtime.h"
// Modified from https://github.com/nerfstudio-project/gsplat/blob/main/gsplat/cuda/csrc/RasterizeToPixels3DGSBwd.cu

#include "common.cuh"

#include <gsplat/Common.h>
#include <gsplat/Utils.cuh>

#include <hipcub/hipcub.hpp>

#include "Primitive.cuh"


constexpr uint BLOCK_SIZE = TILE_SIZE * TILE_SIZE;
constexpr uint SPLAT_BATCH_SIZE = 128;


template <uint32_t CDIM>
__global__ void rasterize_to_pixels_3dgs_bwd_kernel(
    const uint32_t I,
    const uint32_t N,
    const uint32_t n_isects,
    const bool packed,
    // fwd inputs
    Vanilla3DGS::Screen::Buffer splat_buffer,
    const float *__restrict__ colors,      // [..., N, CDIM] or [nnz, CDIM]
    const float *__restrict__ backgrounds, // [..., CDIM] or [nnz, CDIM]
    const bool *__restrict__ masks,           // [..., tile_height, tile_width]
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_width,
    const uint32_t tile_height,
    const int32_t *__restrict__ tile_offsets, // [..., tile_height, tile_width]
    const int32_t *__restrict__ flatten_ids,  // [n_isects]
    // fwd outputs
    const float
        *__restrict__ render_Ts,      // [..., image_height, image_width, 1]
    const int32_t *__restrict__ last_ids, // [..., image_height, image_width]
    // grad outputs
    const float *__restrict__ v_render_colors, // [..., image_height,
                                                  // image_width, CDIM]
    const float
        *__restrict__ v_render_alphas, // [..., image_height, image_width, 1]
    // grad inputs
    Vanilla3DGS::Screen::Buffer v_splat_buffer,
    float *__restrict__ v_colors   // [..., N, CDIM] or [nnz, CDIM]
) {
    auto block = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> warp = cg::tiled_partition<WARP_SIZE>(block);
    uint32_t image_id = block.group_index().x;
    uint32_t tile_id = block.group_index().y * tile_width + block.group_index().z;
    uint32_t thread_id = block.thread_rank();

    tile_offsets += image_id * tile_height * tile_width;
    render_Ts += image_id * image_height * image_width;
    last_ids += image_id * image_height * image_width;
    v_render_colors += image_id * image_height * image_width * CDIM;
    v_render_alphas += image_id * image_height * image_width;
    if (backgrounds != nullptr) {
        backgrounds += image_id * CDIM;
    }
    if (masks != nullptr) {
        masks += image_id * tile_height * tile_width;
    }

    // when the mask is provided, do nothing and return if
    // this tile is labeled as False
    if (masks != nullptr && !masks[tile_id]) {
        return;
    }

    // load pixels
    __shared__ int32_t pix_bin_final[BLOCK_SIZE];
    __shared__ float2 pix_Ts_with_grad[BLOCK_SIZE];
    __shared__ float v_pix_colors[BLOCK_SIZE*CDIM];
    #pragma unroll
    for (uint pix_id0 = 0; pix_id0 < BLOCK_SIZE; pix_id0 += SPLAT_BATCH_SIZE) {
        static_assert(BLOCK_SIZE % SPLAT_BATCH_SIZE == 0);
        uint pix_id_local = pix_id0 + thread_id;
        int pix_x = block.group_index().z * TILE_SIZE + pix_id_local % TILE_SIZE;
        int pix_y = block.group_index().y * TILE_SIZE + pix_id_local / TILE_SIZE;
        uint pix_id_global = pix_y * image_width + pix_x;
        bool inside = (pix_x < image_width && pix_y < image_height);
        
        int32_t bin_final = (inside ? last_ids[pix_id_global] : 0);
        pix_bin_final[pix_id_local] = bin_final;
        pix_Ts_with_grad[pix_id_local] = {
            (inside ? render_Ts[pix_id_global] : 0.0f),
            (inside ? -v_render_alphas[pix_id_global] : 0.0f)
        };
        #pragma unroll
        for (uint k = 0; k < CDIM; k++) {
            v_pix_colors[pix_id_local * CDIM + k] =
                (inside ? v_render_colors[pix_id_global * CDIM + k] : 0.0f);
        }
    }
    static_assert(CDIM <= SPLAT_BATCH_SIZE);
    __shared__ float pix_background[CDIM];
    if (thread_id < CDIM && backgrounds != nullptr)
        pix_background[thread_id] = backgrounds[thread_id];
    block.sync();

    // threads fist load splats, then swept through pixels
    // do this in batches

    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (image_id == I - 1) && (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const uint32_t num_splat_batches =
        _CEIL_DIV(range_end - range_start, SPLAT_BATCH_SIZE);

    // if (warp.thread_rank() == 0)
    //     printf("range_start=%d range_end=%d num_splat_batches=%u\n", range_start, range_end, num_splat_batches);
    for (uint32_t splat_b = 0; splat_b < num_splat_batches; ++splat_b) {
        const int32_t splat_batch_end = range_end - 1 - SPLAT_BATCH_SIZE * splat_b;
        const int32_t splat_batch_size = min(SPLAT_BATCH_SIZE, splat_batch_end + 1 - range_start);
        const int32_t splat_idx = splat_batch_end - thread_id;

        // load splats
        Vanilla3DGS::Screen splat;
        uint32_t splat_gid;
        float splat_color[CDIM];
        if (splat_idx >= range_start) {
            splat_gid = flatten_ids[splat_idx]; // flatten index in [I * N] or [nnz]
            splat = Vanilla3DGS::Screen::load(splat_buffer, splat_gid);
            #pragma unroll
            for (uint32_t k = 0; k < CDIM; ++k)
                splat_color[k] = colors[splat_gid * CDIM + k];
        }

        // accumulate gradient
        float v_rgb_local[CDIM] = {0.f};
        #pragma unroll
        for (uint32_t k = 0; k < CDIM; ++k)
            v_rgb_local[k] = 0.f;
        Vanilla3DGS::Screen v_splat = Vanilla3DGS::Screen::zero();

        // thread 0 takes last splat, 1 takes second last, etc.
        // at t=0, thread 0 (splat -1) undo pixel 0
        // at t=1, thread 0 (splat -1) undo pixel 1, thread 1 (splat -2) undo pixel 0
        // ......

        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = 0; t < splat_batch_size + BLOCK_SIZE - 1; ++t) {
            int pix_id = t - thread_id;

            int pix_local_x = pix_id % TILE_SIZE;
            int pix_local_y = pix_id / TILE_SIZE;
            int pix_global_x = block.group_index().z * TILE_SIZE + pix_local_x;
            int pix_global_y = block.group_index().y * TILE_SIZE + pix_local_y;
            const float px = (float)pix_global_x + 0.5f;
            const float py = (float)pix_global_y + 0.5f;
            if ((pix_id >= 0 && pix_id < BLOCK_SIZE &&
                pix_global_x < image_width && pix_global_y < image_height &&
                splat_idx >= range_start) &&
                splat_idx <= pix_bin_final[pix_id]
            ) {

            // evaluate alpha and early skip
            float alpha = splat.evaluate_alpha(px, py);
            if (alpha >= ALPHA_THRESHOLD) {

            // printf("t=%d, thread %u, splat %d (%u), pix_id %d, pix %d %d\n", t, thread_id, splat_idx-range_start, splat_gid, pix_id, pix_global_x, pix_global_y);

            // forward:
            // \left(c_{1},T_{1}\right)=\left(c_{0}+\alpha_{i}T_{0}c_{i},\ T_{0}\left(1-\alpha_{i}\right)\right)
            float T1 = pix_Ts_with_grad[pix_id].x;
            float v_T1 = pix_Ts_with_grad[pix_id].y;

            // undo pixel:
            // T_{0}=\frac{T_{1}}{1-\alpha_{i}}
            float ra = 1.0f / (1.0f - alpha);
            float T0 = T1 * ra;

            // gradient to alpha:
            // \frac{dL}{d\alpha_{i}}
            // = \frac{dL}{dc_{1}}\frac{dc_{1}}{d\alpha_{i}}+\frac{dL}{dT_{1}}\frac{dT_{1}}{d\alpha_{i}}
            // = T_{0}\frac{dL}{dc_{1}}c_{i}-\frac{dL}{dT_{1}}T_{0}

            // gradient to color:
            // \frac{dL}{dc_{i}}
            // = \frac{dL}{dc_{1}}\frac{dc_{1}}{dc_{i}}
            // = \alpha_{i}T_{0}\frac{dL}{dc_{1}}

            // update pixel gradient:
            // \frac{dL}{dT_{0}}
            // = \frac{dL}{dc_{1}}\frac{dc_{1}}{dT_{0}}+\frac{dL}{dT_{1}}\frac{dT_{1}}{dT_{0}}
            // = \alpha_{i}\frac{dL}{dc_{1}}c_{i}+\frac{dL}{dT_{1}}\left(1-\alpha_{i}\right)

            float v_alpha = -v_T1 * T0;  // gradient to alpha
            float v_T0 = v_T1 * (1.0f - alpha);  // update pixel gradient
            #pragma unroll
            for (uint32_t k = 0; k < CDIM; ++k) {
                float c = splat_color[k];
                float v_c = v_pix_colors[pix_id * CDIM + k];
                v_alpha += c * v_c * T0;  // gradient to alpha
                v_rgb_local[k] += alpha * T0 * v_c;  // gradient to color
                v_T0 += c * v_c * alpha; // update pixel gradient
            }

            // backward diff splat
            v_splat += splat.evaluate_alpha_vjp(px, py, v_alpha);

            // update pixel states
            pix_Ts_with_grad[pix_id] = { T0, v_T0 };
            // v_pix_colors remains the same

            }}
            block.sync();
        }

        // accumulate gradient
        {
            if (splat_idx >= range_start) {

                float *v_rgb_ptr = (float *)(v_colors) + CDIM * splat_gid;
                #pragma unroll
                for (uint32_t k = 0; k < CDIM; ++k) {
                    if (v_rgb_local[k] != 0.0f)
                        atomicAdd(v_rgb_ptr + k, v_rgb_local[k]);
                }

                v_splat.atomicAddBuffer(v_splat_buffer, splat_gid);
            }
        }
    }
}


template <uint32_t CDIM>
void launch_rasterize_to_pixels_3dgs_bwd_kernel(
    // Gaussian parameters
    Vanilla3DGS::Screen::Tensor splats,
    const at::Tensor colors,                    // [..., N, 3] or [nnz, 3]
    const std::optional<at::Tensor> backgrounds, // [..., 3]
    const std::optional<at::Tensor> masks,       // [..., tile_height, tile_width]
    // image size
    const uint32_t image_width,
    const uint32_t image_height,
    // intersections
    const at::Tensor tile_offsets, // [..., tile_height, tile_width]
    const at::Tensor flatten_ids,  // [n_isects]
    // forward outputs
    const at::Tensor render_Ts, // [..., image_height, image_width, 1]
    const at::Tensor last_ids,      // [..., image_height, image_width]
    // gradients of outputs
    const at::Tensor v_render_colors, // [..., image_height, image_width, 3]
    const at::Tensor v_render_alphas, // [..., image_height, image_width, 1]
    // outputs
    Vanilla3DGS::Screen::Tensor v_splats,
    at::Tensor v_colors                    // [..., N, 3] or [nnz, 3]
) {
    bool packed = splats.isPacked();
    uint32_t N = packed ? 0 : splats.size(); // number of gaussians
    uint32_t I = render_Ts.numel() / (image_height * image_width); // number of images
    uint32_t tile_height = tile_offsets.size(-2);
    uint32_t tile_width = tile_offsets.size(-1);
    uint32_t n_isects = flatten_ids.size(0);

    // Each block covers a tile on the image. In total there are
    // I * tile_height * tile_width blocks.
    dim3 threads = {SPLAT_BATCH_SIZE, 1, 1};
    dim3 grid = {I, tile_height, tile_width};

    // int64_t shmem_size =
    //     TILE_SIZE * TILE_SIZE *
    //     (sizeof(int32_t) + sizeof(glm::vec3) + sizeof(glm::vec3) + sizeof(float) * CDIM);

    if (n_isects == 0) {
        // skip the kernel launch if there are no elements
        return;
    }

    // TODO: an optimization can be done by passing the actual number of
    // channels into the kernel functions and avoid necessary global memory
    // writes. This requires moving the channel padding from python to C side.
    // if (hipFuncSetAttribute(reinterpret_cast<const void*>(
    //         rasterize_to_pixels_3dgs_bwd_kernel<CDIM>),
    //         hipFuncAttributeMaxDynamicSharedMemorySize,
    //         shmem_size
    //     ) != hipSuccess) {
    //     AT_ERROR(
    //         "Failed to set maximum shared memory size (requested ",
    //         shmem_size,
    //         " bytes), try lowering tile_size."
    //     );
    // }

    rasterize_to_pixels_3dgs_bwd_kernel<CDIM>
        // <<<grid, threads, shmem_size, at::cuda::getCurrentCUDAStream()>>>(
        <<<grid, threads>>>(
            I,
            N,
            n_isects,
            packed,
            splats,
            colors.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                    : nullptr,
            masks.has_value() ? masks.value().data_ptr<bool>() : nullptr,
            image_width,
            image_height,
            tile_width,
            tile_height,
            tile_offsets.data_ptr<int32_t>(),
            flatten_ids.data_ptr<int32_t>(),
            render_Ts.data_ptr<float>(),
            last_ids.data_ptr<int32_t>(),
            v_render_colors.data_ptr<float>(),
            v_render_alphas.data_ptr<float>(),
            v_splats,
            v_colors.data_ptr<float>()
        );
    CHECK_DEVICE_ERROR(hipGetLastError());
}


std::tuple<
    Vanilla3DGS::Screen::TensorTuple,
    at::Tensor,  // v_colors
    std::optional<at::Tensor>  // absgrad
> rasterize_to_pixels_3dgs_bwd(
    // Gaussian parameters
    Vanilla3DGS::Screen::TensorTuple splats_tuple,
    const at::Tensor colors,                    // [..., N, channels] or [nnz, channels]
    const std::optional<at::Tensor> backgrounds, // [..., channels]
    const std::optional<at::Tensor> masks,       // [..., tile_height, tile_width]
    // image size
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    // intersections
    const at::Tensor tile_offsets, // [..., tile_height, tile_width]
    const at::Tensor flatten_ids,  // [n_isects]
    // forward outputs
    const at::Tensor render_Ts, // [..., image_height, image_width, 1]
    const at::Tensor last_ids,      // [..., image_height, image_width]
    // gradients of outputs
    const at::Tensor v_render_colors, // [..., image_height, image_width, channels]
    const at::Tensor v_render_alphas, // [..., image_height, image_width, 1]
    // options
    bool absgrad
) {
    DEVICE_GUARD(colors);
    CHECK_INPUT(colors);
    CHECK_INPUT(tile_offsets);
    CHECK_INPUT(flatten_ids);
    CHECK_INPUT(render_Ts);
    CHECK_INPUT(last_ids);
    CHECK_INPUT(v_render_colors);
    CHECK_INPUT(v_render_alphas);
    if (backgrounds.has_value())
        CHECK_INPUT(backgrounds.value());
    if (masks.has_value())
        CHECK_INPUT(masks.value());

    if (tile_size != TILE_SIZE)
        AT_ERROR("Unsupported tile size");

    uint32_t channels = colors.size(-1);

    Vanilla3DGS::Screen::Tensor splats(splats_tuple);
    Vanilla3DGS::Screen::Tensor v_splats = splats.zeros_like(absgrad);

    at::Tensor v_colors = torch::zeros_like(colors, splats.options());

#define __LAUNCH_KERNEL__(N)                                                   \
    case N:                                                                    \
        launch_rasterize_to_pixels_3dgs_bwd_kernel<N>(                         \
            splats,                                                            \
            colors,                                                            \
            backgrounds,                                                       \
            masks,                                                             \
            image_width,                                                       \
            image_height,                                                      \
            tile_offsets,                                                      \
            flatten_ids,                                                       \
            render_Ts,                                                         \
            last_ids,                                                          \
            v_render_colors,                                                   \
            v_render_alphas,                                                   \
            v_splats,                                                          \
            v_colors                                                           \
        );                                                                     \
        break;

    // TODO: an optimization can be done by passing the actual number of
    // channels into the kernel functions and avoid necessary global memory
    // writes. This requires moving the channel padding from python to C side.
    switch (channels) {
        __LAUNCH_KERNEL__(1)
        __LAUNCH_KERNEL__(2)
        __LAUNCH_KERNEL__(3)
        __LAUNCH_KERNEL__(4)
        __LAUNCH_KERNEL__(5)
        // __LAUNCH_KERNEL__(8)
        // __LAUNCH_KERNEL__(9)
        // __LAUNCH_KERNEL__(16)
        // __LAUNCH_KERNEL__(17)
        // __LAUNCH_KERNEL__(32)
        // __LAUNCH_KERNEL__(33)
        // __LAUNCH_KERNEL__(64)
        // __LAUNCH_KERNEL__(65)
        // __LAUNCH_KERNEL__(128)
        // __LAUNCH_KERNEL__(129)
        // __LAUNCH_KERNEL__(256)
        // __LAUNCH_KERNEL__(257)
        // __LAUNCH_KERNEL__(512)
        // __LAUNCH_KERNEL__(513)
    default:
        AT_ERROR("Unsupported number of channels: ", channels);
    }
#undef __LAUNCH_KERNEL__

    return std::make_tuple(
        v_splats.tuple(), v_colors, v_splats.absgrad
    );
}


// Explicit Instantiation: this should match how it is being called in .cpp
// file.
// TODO: this is slow to compile, can we do something about it?
#define __INS__(CDIM)                                                          \
    template void launch_rasterize_to_pixels_3dgs_bwd_kernel<CDIM>(            \
        Vanilla3DGS::Screen::Tensor splats,                                                \
        const at::Tensor colors,                                               \
        const std::optional<at::Tensor> backgrounds,                            \
        const std::optional<at::Tensor> masks,                                  \
        uint32_t image_width,                                                  \
        uint32_t image_height,                                                 \
        const at::Tensor tile_offsets,                                         \
        const at::Tensor flatten_ids,                                          \
        const at::Tensor render_Ts,                                            \
        const at::Tensor last_ids,                                             \
        const at::Tensor v_render_colors,                                      \
        const at::Tensor v_render_alphas,                                      \
        Vanilla3DGS::Screen::Tensor v_splats,                                              \
        at::Tensor v_opacities                                                 \
    );

__INS__(1)
__INS__(2)
__INS__(3)
__INS__(4)
__INS__(5)
// __INS__(8)
// __INS__(9)
// __INS__(16)
// __INS__(17)
// __INS__(32)
// __INS__(33)
// __INS__(64)
// __INS__(65)
// __INS__(128)
// __INS__(129)
// __INS__(256)
// __INS__(257)
// __INS__(512)
// __INS__(513)
#undef __INS__
