#include "hip/hip_runtime.h"
#include "BackgroundSphericalHarmonics.cuh"

#include "common.cuh"
#include "camera.cuh"
#include <algorithm>


enum class CameraType {
    // undistorted vs generic distorted
	Undistorted,
    GenericDistorted,
    // (near-)exact distortion
    OPENCV,
    OPENCV_FISHEYE,
    // approximate distortion
    // same rasterization, distort using Jacobian in projection
    OPENCV_approx,
    OPENCV_FISHEYE_approx,
};



template<CameraType CAMERA_TYPE>
__global__ void render_background_sh_forward_kernel(
    _ARGS_render_background_sh_forward_kernel
) {
    unsigned i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned j = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t pix_id = i * img_size.x + j;

    if (i >= img_size.y || j >= img_size.x) return;

    float fx = intrins.x, fy = intrins.y;
    float cx = intrins.z, cy = intrins.w;

    glm::vec2 pos_2d = { (j + 0.5f - cx) / fx, (i + 0.5f - cy) / fy };
    if (CAMERA_TYPE == CameraType::GenericDistorted) {
        float2 pos_2d_u = undistortion_map[pix_id];
        if (isnan(pos_2d.x+pos_2d.y)) {
            out_img[pix_id] = {0.0f, 0.0f, 0.0f};
            return;
        }
        else
            pos_2d = { pos_2d_u.x, pos_2d_u.y };
    }

    float xi = pos_2d.x;
    float yi = -pos_2d.y;
    float zi = -1.0f;
    float xr = rotation[0] * xi + rotation[1] * yi + rotation[2] * zi;
    float yr = rotation[3] * xi + rotation[4] * yi + rotation[5] * zi;
    float zr = rotation[6] * xi + rotation[7] * yi + rotation[8] * zi;
    float norm = sqrtf(fmaxf(xr * xr + yr * yr + zr * zr, 1e-12f));
    float x = isfinite(xr) ? xr / norm : 0.0f;
    float y = isfinite(yr) ? yr / norm : 0.0f;
    float z = isfinite(zr) ? zr / norm : 0.0f;

    float xx = x*x, yy = y*y, zz = z*z;

    glm::vec3 color = glm::vec3(0.0f);
    glm::vec3 *sh_coeffs = (glm::vec3*)sh_coeffs_float3;

    // l0
    color += 0.28209479177387814f * sh_coeffs[0];

    // l1
    if (sh_degree > 1) {
        color += 0.4886025119029199f * y * sh_coeffs[1];
        color += 0.4886025119029199f * z * sh_coeffs[2];
        color += 0.4886025119029199f * x * sh_coeffs[3];
    }

    // l2
    if (sh_degree > 2) {
        color += 1.0925484305920792f * x * y * sh_coeffs[4];
        color += 1.0925484305920792f * y * z * sh_coeffs[5];
        color += (0.9461746957575601f * zz - 0.31539156525251999f) * sh_coeffs[6];
        color += 1.0925484305920792f * x * z * sh_coeffs[7];
        color += 0.5462742152960396f * (xx - yy) * sh_coeffs[8];
    }

    // l3
    if (sh_degree > 3) {
        color += 0.5900435899266435f * y * (3.0f * xx - yy) * sh_coeffs[9];
        color += 2.890611442640554f * x * y * z * sh_coeffs[10];
        color += 0.4570457994644658f * y * (5.0f * zz - 1.0f) * sh_coeffs[11];
        color += 0.3731763325901154f * z * (5.0f * zz - 3.0f) * sh_coeffs[12];
        color += 0.4570457994644658f * x * (5.0f * zz - 1.0f) * sh_coeffs[13];
        color += 1.445305721320277f * z * (xx - yy) * sh_coeffs[14];
        color += 0.5900435899266435f * x * (xx - 3.0f * yy) * sh_coeffs[15];
    }

    // l4
    if (sh_degree > 4) {
        color += 2.5033429417967046f * x * y * (xx - yy) * sh_coeffs[16];
        color += 1.7701307697799304f * y * z * (3.0f * xx - yy) * sh_coeffs[17];
        color += 0.9461746957575601f * x * y * (7.0f * zz - 1.0f) * sh_coeffs[18];
        color += 0.6690465435572892f * y * z * (7.0f * zz - 3.0f) * sh_coeffs[19];
        color += 0.10578554691520431f * (35.0f * zz * zz - 30.0f * zz + 3.0f) * sh_coeffs[20];
        color += 0.6690465435572892f * x * z * (7.0f * zz - 3.0f) * sh_coeffs[21];
        color += 0.47308734787878004f * (xx - yy) * (7.0f * zz - 1.0f) * sh_coeffs[22];
        color += 1.7701307697799304f * x * z * (xx - 3.0f * yy) * sh_coeffs[23];
        color += 0.6258357354491761f * (xx * (xx - 3.0f * yy) - yy * (3.0f * xx - yy)) * sh_coeffs[24];
    }

    color.x = fmaxf(color.x + 0.5f, 0.0f);
    color.y = fmaxf(color.y + 0.5f, 0.0f);
    color.z = fmaxf(color.z + 0.5f, 0.0f);

    out_img[pix_id] = *(float3*)&color;
}


template<CameraType CAMERA_TYPE>
__global__ void render_background_sh_backward_kernel(
    _ARGS_render_background_sh_backward_kernel
) {
    unsigned i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned j = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t pix_id = i * img_size.x + j;

    bool inside = (i < img_size.y && j < img_size.x);

    unsigned idx = i * img_size.x + j;
    glm::vec3 v_color = glm::vec3(0.0);
    if (inside) {
        glm::vec3 color = ((glm::vec3*)out_color)[idx];
        v_color = ((glm::vec3*)v_out_color)[idx];
        if (color.x == 0.0f || !isfinite(v_color.x)) v_color.x = 0.0f;
        if (color.y == 0.0f || !isfinite(v_color.y)) v_color.y = 0.0f;
        if (color.z == 0.0f || !isfinite(v_color.z)) v_color.z = 0.0f;
        // v_color = glm::clamp(v_color, -glm::vec3(1e4f), glm::vec3(1e4f));
    }

    float fx = intrins.x, fy = intrins.y;
    float cx = intrins.z, cy = intrins.w;

    glm::vec2 pos_2d = { (j + 0.5f - cx) / fx, (i + 0.5f - cy) / fy };
    if (CAMERA_TYPE == CameraType::GenericDistorted && inside) {
        float2 pos_2d_u = undistortion_map[pix_id];
        if (isnan(pos_2d.x+pos_2d.y))
            inside = false;
        else
            pos_2d = { pos_2d_u.x, pos_2d_u.y };
    }
    if (__syncthreads_count(inside) == 0)
        return;

    auto block = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> warp = cg::tiled_partition<WARP_SIZE>(block);

    float xi = pos_2d.x;
    float yi = -pos_2d.y;
    float zi = -1.0f;
    float xr = rotation[0] * xi + rotation[1] * yi + rotation[2] * zi;
    float yr = rotation[3] * xi + rotation[4] * yi + rotation[5] * zi;
    float zr = rotation[6] * xi + rotation[7] * yi + rotation[8] * zi;
    float norm2 = xr * xr + yr * yr + zr * zr;
    float norm = sqrtf(fmaxf(norm2, 1e-12f));
    float x = inside && isfinite(xr) ? xr / norm : 0.0f;
    float y = inside && isfinite(yr) ? yr / norm : 0.0f;
    float z = inside && isfinite(zr) ? zr / norm : 0.0f;

    float xx = x*x, yy = y*y, zz = z*z;

    float v_x = 0.0f, v_y = 0.0f, v_z = 0.0f;
    float v_xx = 0.0f, v_yy = 0.0f, v_zz = 0.0f;

    glm::vec3 *sh_coeffs = (glm::vec3*)sh_coeffs_float3;

    __shared__ glm::vec3 atomic_reduce[WARP_SIZE];  // assume WARP_SIZE^2 >= block_size

    unsigned thread_idx = block.thread_rank();
    unsigned warp_idx = thread_idx/WARP_SIZE;
    unsigned lane_idx = thread_idx%WARP_SIZE;

    glm::vec3 temp3;
    float temp;
    #define _BLOCK_REDUCE_VEC3() \
        warpSum3(temp3, warp); \
        if (warp.thread_rank() == 0) \
            atomic_reduce[warp_idx] = temp3; \
        __syncthreads(); \
        temp = 0.0; \
        if (warp_idx < 3 && lane_idx < (blockDim.x*blockDim.y/WARP_SIZE)) \
            temp = atomic_reduce[lane_idx][warp_idx]; \
        warpSum(temp, warp);

    #define _ATOMIC_ADD(address, idx) \
        _BLOCK_REDUCE_VEC3(); \
        if (warp_idx < 3 && lane_idx == 0) \
            atomicAdd((float*)address + (3*idx+warp_idx), temp);

    // l0
    float v_color_dot_sh_coeff = 0.0f;
    temp3 = 0.28209479177387814f * v_color;
    _ATOMIC_ADD(v_sh_coeffs, 0);

    // l1 - manually calculated
    if (sh_degree > 1) {

        // color += 0.4886025119029199f * y * sh_coeffs[1];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[1]);
        v_y += 0.4886025119029199f * v_color_dot_sh_coeff;
        temp3 = 0.4886025119029199f * y * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 1);

        // color += 0.4886025119029199f * z * sh_coeffs[2];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[2]);
        v_z += 0.4886025119029199f * v_color_dot_sh_coeff;
        temp3 = 0.4886025119029199f * z * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 2);

        // color += 0.4886025119029199f * x * sh_coeffs[3];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[3]);
        v_x += 0.4886025119029199f * v_color_dot_sh_coeff;
        temp3 = 0.4886025119029199f * x * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 3);
    }

    // l2 - manually calculated
    if (sh_degree > 2) {

        // color += 1.0925484305920792f * x * y * sh_coeffs[4];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[4]);
        v_x += 1.0925484305920792f * y * v_color_dot_sh_coeff;
        v_y += 1.0925484305920792f * x * v_color_dot_sh_coeff;
        temp3 = 1.0925484305920792f * x * y * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 4);

        // color += 1.0925484305920792f * y * z * sh_coeffs[5];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[5]);
        v_z += 1.0925484305920792f * y * v_color_dot_sh_coeff;
        v_y += 1.0925484305920792f * z * v_color_dot_sh_coeff;
        temp3 = 1.0925484305920792f * y * z * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 5);

        // color += (0.9461746957575601f * zz - 0.31539156525251999f) * sh_coeffs[6];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[6]);
        v_zz += 0.9461746957575601f * v_color_dot_sh_coeff;
        temp3 = (0.9461746957575601f * zz - 0.31539156525251999f) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 6);

        // color += 1.0925484305920792f * x * z * sh_coeffs[7];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[7]);
        v_x += 1.0925484305920792f * z * v_color_dot_sh_coeff;
        v_z += 1.0925484305920792f * x * v_color_dot_sh_coeff;
        temp3 = 1.0925484305920792f * x * z * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 7);

        // color += 0.5462742152960396f * (xx - yy) * sh_coeffs[8];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[8]);
        v_xx += 0.5462742152960396f * v_color_dot_sh_coeff;
        v_yy -= 0.5462742152960396f * v_color_dot_sh_coeff;
        temp3 = 0.5462742152960396f * (xx - yy) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 8);
    }

    // l3 - AI generated, one incorrect line commented
    if (sh_degree > 3) {
        // color += 0.5900435899266435f * y * (3.0f * xx - yy) * sh_coeffs[9];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[9]);
        v_xx += 1.7701307697799305f * y * v_color_dot_sh_coeff;
        v_yy -= 0.5900435899266435f * y * v_color_dot_sh_coeff;
        v_y += 0.5900435899266435f * (3.0f * xx - yy) * v_color_dot_sh_coeff;
        temp3 = 0.5900435899266435f * y * (3.0f * xx - yy) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 9);

        // color += 2.890611442640554f * x * y * z * sh_coeffs[10];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[10]);
        v_x += 2.890611442640554f * y * z * v_color_dot_sh_coeff;
        v_y += 2.890611442640554f * x * z * v_color_dot_sh_coeff;
        v_z += 2.890611442640554f * x * y * v_color_dot_sh_coeff;
        temp3 = 2.890611442640554f * x * y * z * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 10);

        // color += 0.4570457994644658f * y * (5.0f * zz - 1.0f) * sh_coeffs[11];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[11]);
        v_zz += 2.285228997322329f * y * v_color_dot_sh_coeff;
        v_y += 0.4570457994644658f * (5.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        temp3 = 0.4570457994644658f * y * (5.0f * zz - 1.0f) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 11);

        // color += 0.3731763325901154f * z * (5.0f * zz - 3.0f) * sh_coeffs[12];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[12]);
        v_z += 0.3731763325901154f * (5.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_zz += 1.865881662950577f * z * v_color_dot_sh_coeff;
        temp3 = 0.3731763325901154f * z * (5.0f * zz - 3.0f) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 12);

        // color += 0.4570457994644658f * x * (5.0f * zz - 1.0f) * sh_coeffs[13];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[13]);
        v_x += 0.4570457994644658f * (5.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_zz += 2.285228997322329f * x * v_color_dot_sh_coeff;
        temp3 = 0.4570457994644658f * x * (5.0f * zz - 1.0f) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 13);

        // color += 1.445305721320277f * z * (xx - yy) * sh_coeffs[14];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[14]);
        v_xx += 1.445305721320277f * z * v_color_dot_sh_coeff;
        v_yy -= 1.445305721320277f * z * v_color_dot_sh_coeff;
        v_z += 1.445305721320277f * (xx - yy) * v_color_dot_sh_coeff;
        temp3 = 1.445305721320277f * z * (xx - yy) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 14);

        // color += 0.5900435899266435f * x * (xx - 3.0f * yy) * sh_coeffs[15];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[15]);
        // v_xx += 1.1800871798532870f * x * v_color_dot_sh_coeff;
        v_xx += 0.5900435899266435f * x * v_color_dot_sh_coeff;
        v_yy -= 1.7701307697799305f * x * v_color_dot_sh_coeff;
        v_x += 0.5900435899266435f * (xx - 3.0f * yy) * v_color_dot_sh_coeff;
        temp3 = 0.5900435899266435f * x * (xx - 3.0f * yy) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 15);
    }

    // l4 - AI generated, two incorrect lines commented
    if (sh_degree > 4) {
        // color += 2.5033429417967046f * x * y * (xx - yy) * sh_coeffs[16];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[16]);
        v_x += 2.5033429417967046f * y * (xx - yy) * v_color_dot_sh_coeff;
        v_y += 2.5033429417967046f * x * (xx - yy) * v_color_dot_sh_coeff;
        v_xx += 2.5033429417967046f * x * y * v_color_dot_sh_coeff;
        v_yy -= 2.5033429417967046f * x * y * v_color_dot_sh_coeff;
        temp3 = 2.5033429417967046f * x * y * (xx - yy) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 16);

        // color += 1.7701307697799304f * y * z * (3.0f * xx - yy) * sh_coeffs[17];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[17]);
        v_xx += 5.3103923093397912f * y * z * v_color_dot_sh_coeff;
        v_yy -= 1.7701307697799304f * y * z * v_color_dot_sh_coeff;
        v_y += 1.7701307697799304f * z * (3.0f * xx - yy) * v_color_dot_sh_coeff;
        v_z += 1.7701307697799304f * y * (3.0f * xx - yy) * v_color_dot_sh_coeff;
        temp3 = 1.7701307697799304f * y * z * (3.0f * xx - yy) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 17);

        // color += 0.9461746957575601f * x * y * (7.0f * zz - 1.0f) * sh_coeffs[18];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[18]);
        v_x += 0.9461746957575601f * y * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_y += 0.9461746957575601f * x * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_zz += 6.6232228703029207f * x * y * v_color_dot_sh_coeff;
        temp3 = 0.9461746957575601f * x * y * (7.0f * zz - 1.0f) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 18);

        // color += 0.6690465435572892f * y * z * (7.0f * zz - 3.0f) * sh_coeffs[19];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[19]);
        v_y += 0.6690465435572892f * z * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_z += 0.6690465435572892f * y * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_zz += 4.6833258049010244f * y * z * v_color_dot_sh_coeff;
        temp3 = 0.6690465435572892f * y * z * (7.0f * zz - 3.0f) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 19);

        // color += 0.10578554691520431f * (35.0f * zz * zz - 30.0f * zz + 3.0f) * sh_coeffs[20];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[20]);
        v_zz += 0.10578554691520431f * (70.0f * zz - 30.0f) * v_color_dot_sh_coeff;
        temp3 = 0.10578554691520431f * (35.0f * zz * zz - 30.0f * zz + 3.0f) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 20);

        // color += 0.6690465435572892f * x * z * (7.0f * zz - 3.0f) * sh_coeffs[21];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[21]);
        v_x += 0.6690465435572892f * z * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_z += 0.6690465435572892f * x * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_zz += 4.6833258049010244f * x * z * v_color_dot_sh_coeff;
        temp3 = 0.6690465435572892f * x * z * (7.0f * zz - 3.0f) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 21);

        // color += 0.47308734787878004f * (xx - yy) * (7.0f * zz - 1.0f) * sh_coeffs[22];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[22]);
        v_xx += 0.47308734787878004f * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_yy -= 0.47308734787878004f * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_zz += 3.3116114351514603f * (xx - yy) * v_color_dot_sh_coeff;
        temp3 = 0.47308734787878004f * (xx - yy) * (7.0f * zz - 1.0f) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 22);

        // color += 1.7701307697799304f * x * z * (xx - 3.0f * yy) * sh_coeffs[23];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[23]);
        v_x += 1.7701307697799304f * z * (xx - 3.0f * yy) * v_color_dot_sh_coeff;
        v_z += 1.7701307697799304f * x * (xx - 3.0f * yy) * v_color_dot_sh_coeff;
        v_xx += 1.7701307697799304f * x * z * v_color_dot_sh_coeff;
        v_yy -= 5.3103923093397912f * x * z * v_color_dot_sh_coeff;
        temp3 = 1.7701307697799304f * x * z * (xx - 3.0f * yy) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 23);

        // color += 0.6258357354491761f * (xx * (xx - 3.0f * yy) - yy * (3.0f * xx - yy)) * sh_coeffs[24];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[24]);
        // v_xx += 0.6258357354491761f * (4.0f * xx - 6.0f * yy) * v_color_dot_sh_coeff;
        // v_yy += 0.6258357354491761f * (6.0f * yy - 12.0f * xx) * v_color_dot_sh_coeff;
        v_xx += 0.6258357354491761f * (2.0f * xx - 6.0f * yy) * v_color_dot_sh_coeff;
        v_yy += 0.6258357354491761f * (2.0f * yy - 6.0f * xx) * v_color_dot_sh_coeff;
        temp3 = 0.6258357354491761f * (xx * (xx - 3.0f * yy) - yy * (3.0f * xx - yy)) * v_color;
        _ATOMIC_ADD(v_sh_coeffs, 24);
    }

    v_x += v_xx * 2.0f*x;
    v_y += v_yy * 2.0f*y;
    v_z += v_zz * 2.0f*z;

    glm::vec3 xyz = glm::vec3(x, y, z);
    glm::mat3 dp_dpr = (glm::mat3(1.0f) - glm::outerProduct(xyz, xyz)) / norm;
    glm::vec3 v_p = dp_dpr * glm::vec3(v_x, v_y, v_z);
    v_p *= (inside ? 1.0f : 0.0f);

  #if 0
    float tmp[9] = {
        v_p.x * xi, v_p.x * yi, v_p.x * zi,
        v_p.y * xi, v_p.y * yi, v_p.y * zi,
        v_p.z * xi, v_p.z * yi, v_p.z * zi
    };
    #pragma unroll
    for (int i = 0; i < 9; i++) {
        warpSum(tmp[i], warp);
        if (warp.thread_rank() == i)
            atomicAdd((float*)v_rotation + i, tmp[i]);
    }
  #else
    temp3 = glm::vec3(v_p.x * xi, v_p.x * yi, v_p.x * zi);
    _ATOMIC_ADD(v_rotation, 0);
    temp3 = glm::vec3(v_p.y * xi, v_p.y * yi, v_p.y * zi);
    _ATOMIC_ADD(v_rotation, 1);
    temp3 = glm::vec3(v_p.z * xi, v_p.z * yi, v_p.z * zi);
    _ATOMIC_ADD(v_rotation, 2);
  #endif
    #undef _BLOCK_REDUCE_VEC3
    #undef _ATOMIC_ADD
}



torch::Tensor render_background_sh_forward_tensor(
    const unsigned w,
    const unsigned h,
    std::string camera_model,
    const std::tuple<float, float, float, float> intrins,
    const std::optional<torch::Tensor> &undistortion_map_,
    const torch::Tensor &rotation,
    const unsigned sh_degree,
    const torch::Tensor &sh_coeffs
) {
    DEVICE_GUARD(sh_coeffs);
    CHECK_INPUT(sh_coeffs);
    CHECK_INPUT(rotation);

    if (rotation.numel() != 9) {
        AT_ERROR("rotation must be 3x3");
    }
    if (sh_coeffs.ndimension() != 2 ||
        sh_coeffs.size(0) != sh_degree*sh_degree ||
        sh_coeffs.size(1) != 3) {
        AT_ERROR("sh_coeffs must be (sh_regree**2, 3)");
    }

    const dim3 img_size = {w, h, 1};

    auto options = sh_coeffs.options();
    torch::Tensor out_color = torch::empty({h, w, 3}, options);

    if (camera_model == "") {
        render_background_sh_forward_kernel<CameraType::Undistorted>
        <<<_LAUNCH_ARGS_2D(w, h, TILE_SIZE, TILE_SIZE)>>>(
            img_size,
            tuple2float4(intrins), nullptr,
            rotation.contiguous().data_ptr<float>(),
            sh_degree,
            (float3 *)sh_coeffs.contiguous().data_ptr<float>(),
            (float3 *)out_color.contiguous().data_ptr<float>()
        );
    }

    else {
        const torch::Tensor& undistortion_map = undistortion_map_.value();
        CHECK_INPUT(undistortion_map);

        render_background_sh_forward_kernel<CameraType::GenericDistorted>
        <<<_LAUNCH_ARGS_2D(w, h, TILE_SIZE, TILE_SIZE)>>>(
            img_size,
            tuple2float4(intrins),
            (float2 *)undistortion_map.contiguous().data_ptr<float>(),
            rotation.contiguous().data_ptr<float>(),
            sh_degree,
            (float3 *)sh_coeffs.contiguous().data_ptr<float>(),
            (float3 *)out_color.contiguous().data_ptr<float>()
        );
    }

    return out_color;
}


std::tuple<
    torch::Tensor,  // v_rotation
    torch::Tensor  // v_sh_coeffs
> render_background_sh_backward_tensor(
    const unsigned w,
    const unsigned h,
    const std::string camera_model,
    const std::tuple<float, float, float, float> intrins,
    const std::optional<torch::Tensor> &undistortion_map_,
    const torch::Tensor &rotation,
    const unsigned sh_degree,
    const torch::Tensor &sh_coeffs,
    const torch::Tensor &out_color,
    const torch::Tensor &v_out_color
) {
    DEVICE_GUARD(sh_coeffs);
    CHECK_INPUT(sh_coeffs);
    CHECK_INPUT(rotation);
    CHECK_INPUT(v_out_color);

    if (rotation.numel() != 9) {
        AT_ERROR("rotation must be 3x3");
    }
    if (sh_coeffs.ndimension() != 2 ||
        sh_coeffs.size(0) != sh_degree*sh_degree ||
        sh_coeffs.size(1) != 3) {
        AT_ERROR("sh_coeffs shape must be (sh_regree**2, 3)");
    }
    if (out_color.ndimension() != 3 ||
        out_color.size(0) != h ||
        out_color.size(1) != w ||
        out_color.size(2) != 3) {
        AT_ERROR("out_color shape must be (h, w, 3)");
    }
    if (v_out_color.ndimension() != 3 ||
        v_out_color.size(0) != h ||
        v_out_color.size(1) != w ||
        v_out_color.size(2) != 3) {
        AT_ERROR("v_out_color shape must be (h, w, 3)");
    }

    // unsigned block_width = TILE_SIZE;
    unsigned block_width = 32;  // 1024 threads
    const dim3 img_size = {w, h, 1};

    auto options = sh_coeffs.options();
    torch::Tensor v_rotation = torch::zeros({3, 3}, options);
    torch::Tensor v_sh_coeffs = torch::zeros({sh_degree*sh_degree, 3}, options);

    #define _TEMP_ARGS \
        rotation.contiguous().data_ptr<float>(), \
        sh_degree, \
        (float3 *)sh_coeffs.contiguous().data_ptr<float>(), \
        (float3 *)out_color.contiguous().data_ptr<float>(), \
        (float3 *)v_out_color.contiguous().data_ptr<float>(), \
        (float3 *)v_rotation.contiguous().data_ptr<float>(), \
        (float3 *)v_sh_coeffs.contiguous().data_ptr<float>()

    if (camera_model == "") {
        render_background_sh_backward_kernel<CameraType::Undistorted>
        <<<_LAUNCH_ARGS_2D(w, h, block_width, block_width)>>>(
            img_size,
            tuple2float4(intrins), nullptr,
            _TEMP_ARGS
        );
    }
    else {
        const torch::Tensor& undistortion_map = undistortion_map_.value();
        CHECK_INPUT(undistortion_map);
        render_background_sh_backward_kernel<CameraType::GenericDistorted>
        <<<_LAUNCH_ARGS_2D(w, h, block_width, block_width)>>>(
            img_size,
            tuple2float4(intrins),
            (float2 *)undistortion_map.contiguous().data_ptr<float>(),
            _TEMP_ARGS
        );
    }

    #undef _TEMP_ARGS

    return std::make_tuple(v_rotation, v_sh_coeffs);
}



template __global__ void render_background_sh_forward_kernel<CameraType::Undistorted>(
    _ARGS_render_background_sh_forward_kernel
);
template __global__ void render_background_sh_forward_kernel<CameraType::GenericDistorted>(
    _ARGS_render_background_sh_forward_kernel
);
template __global__ void render_background_sh_backward_kernel<CameraType::Undistorted>(
    _ARGS_render_background_sh_backward_kernel
);
template __global__ void render_background_sh_backward_kernel<CameraType::GenericDistorted>(
    _ARGS_render_background_sh_backward_kernel
);
