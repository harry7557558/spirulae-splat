#include "hip/hip_runtime.h"
#include "PixelWise.cuh"

#define TensorView _Slang_TensorView
#include "generated/slang_all.cu"
#undef TensorView

#include "common.cuh"

__global__ void blend_background_forward_kernel(
    const TensorView<float, 3> in_rgb,
    const TensorView<float, 3> in_alpha,
    const TensorView<float, 3> in_background,
    TensorView<float, 3> out_rgb
) {
    unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= in_rgb.shape[0]*in_rgb.shape[1])
        return;
    unsigned y = gid / in_rgb.shape[1];
    unsigned x = gid % in_rgb.shape[1];

    float3 rgb = in_rgb.load3f(y, x);
    float alpha = in_alpha.load1f(y, x);
    float3 background = in_background.load3f(y, x);

    rgb = blend_background(rgb, alpha, background);

    out_rgb.store3f(y, x, rgb);
}


__global__ void blend_background_backward_kernel(
    const TensorView<float, 3> in_rgb,
    const TensorView<float, 3> in_alpha,
    const TensorView<float, 3> in_background,
    const TensorView<float, 3> v_out_rgb,
    TensorView<float, 3> v_in_rgb,
    TensorView<float, 3> v_in_alpha,
    TensorView<float, 3> v_in_background
) {
    unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= in_rgb.shape[0]*in_rgb.shape[1])
        return;
    unsigned y = gid / in_rgb.shape[1];
    unsigned x = gid % in_rgb.shape[1];

    float3 rgb = in_rgb.load3f(y, x);
    float alpha = in_alpha.load1f(y, x);
    float3 background = in_background.load3f(y, x);

    float3 v_out = v_out_rgb.load3f(y, x);

    float3 v_rgb; float v_alpha; float3 v_background;
    blend_background_bwd(
        rgb, alpha, background,
        v_out,
        &v_rgb, &v_alpha, &v_background
    );

    v_in_rgb.store3f(y, x, v_rgb);
    v_in_alpha.store1f(y, x, v_alpha);
    v_in_background.store3f(y, x, v_background);

}



torch::Tensor blend_background_forward_tensor(
    torch::Tensor &rgb,  // [H, W, 3]
    torch::Tensor &alpha,  // [H, W, 1]
    torch::Tensor &background  // [H, W, 3]
) {
    DEVICE_GUARD(rgb);
    CHECK_CUDA(rgb);
    CHECK_CUDA(alpha);
    CHECK_CUDA(background);

    if (rgb.ndimension() != 3 || rgb.size(2) != 3)
        AT_ERROR("rgb shape must be (h, w, 3)");
    long h = rgb.size(0), w = rgb.size(1);
    if (alpha.ndimension() != 3 || alpha.size(0) != h || alpha.size(1) != w || alpha.size(2) != 1)
        AT_ERROR("alpha shape must be (h, w, 1)");
    if (background.ndimension() != 3 || background.size(0) != h || background.size(1) != w || background.size(2) != 3)
        AT_ERROR("background shape must be (h, w, 3)");

    torch::Tensor out_rgb = torch::empty({h, w, 3}, rgb.options());

    blend_background_forward_kernel<<<_LAUNGH_ARGS_1D(h*w)>>>(
        tensor2view<float, 3>(rgb), tensor2view<float, 3>(alpha), tensor2view<float, 3>(background),
        tensor2view<float, 3>(out_rgb)
    );

    return out_rgb;
}


std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
blend_background_backward_tensor(
    torch::Tensor &rgb,  // [H, W, 3]
    torch::Tensor &alpha,  // [H, W, 1]
    torch::Tensor &background,  // [H, W, 3]
    torch::Tensor &v_out_rgb  // [H, W, 3]
) {
    DEVICE_GUARD(rgb);
    CHECK_CUDA(rgb);
    CHECK_CUDA(alpha);
    CHECK_CUDA(background);
    CHECK_CUDA(v_out_rgb);

    if (rgb.ndimension() != 3 || rgb.size(2) != 3)
        AT_ERROR("rgb shape must be (h, w, 3)");
    long h = rgb.size(0), w = rgb.size(1);
    if (alpha.ndimension() != 3 || alpha.size(0) != h || alpha.size(1) != w || alpha.size(2) != 1)
        AT_ERROR("alpha shape must be (h, w, 1)");
    if (background.ndimension() != 3 || background.size(0) != h || background.size(1) != w || background.size(2) != 3)
        AT_ERROR("background shape must be (h, w, 3)");
    if (v_out_rgb.ndimension() != 3 || v_out_rgb.size(0) != h || v_out_rgb.size(1) != w || v_out_rgb.size(2) != 3)
        AT_ERROR("v_out_rgb shape must be (h, w, 3)");

    torch::Tensor v_rgb = torch::empty({h, w, 3}, rgb.options());
    torch::Tensor v_alpha = torch::empty({h, w, 1}, alpha.options());
    torch::Tensor v_background = torch::empty({h, w, 3}, background.options());

    blend_background_backward_kernel<<<_LAUNGH_ARGS_1D(h*w)>>>(
        tensor2view<float, 3>(rgb), tensor2view<float, 3>(alpha), tensor2view<float, 3>(background),
        tensor2view<float, 3>(v_out_rgb),
        tensor2view<float, 3>(v_rgb), tensor2view<float, 3>(v_alpha), tensor2view<float, 3>(v_background)
    );

    return std::make_tuple(v_rgb, v_alpha, v_background);
}

