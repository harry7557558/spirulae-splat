#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include "ch.cuh"
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <iostream>
#include <hip/hip_fp16.h>
namespace cg = cooperative_groups;


template<typename vec3>
inline __device__ void warpSum3(vec3& val, cg::thread_block_tile<32>& tile){
    val.x = cg::reduce(tile, val.x, cg::plus<float>());
    val.y = cg::reduce(tile, val.y, cg::plus<float>());
    val.z = cg::reduce(tile, val.z, cg::plus<float>());
}

template<typename vec2>
inline __device__ void warpSum2(vec2& val, cg::thread_block_tile<32>& tile){
    val.x = cg::reduce(tile, val.x, cg::plus<float>());
    val.y = cg::reduce(tile, val.y, cg::plus<float>());
}

inline __device__ void warpSum(float& val, cg::thread_block_tile<32>& tile){
    val = cg::reduce(tile, val, cg::plus<float>());
}



#if 0
__global__ void rasterize_sorted_indices_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    int* __restrict__ out_indices
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int tr = block.thread_rank();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // number of elements in the buffer
    int buffer_size = 0;
    // sorted global index
    __shared__ int32_t sorted_indices_[MAX_SORTED_SPLATS*MAX_BLOCK_SIZE];
    int32_t *sorted_indices = &sorted_indices_[tr*MAX_SORTED_SPLATS];
    // 24 bit depth, 8 bit weight
    __shared__ uint32_t sorted_buffer_[MAX_SORTED_SPLATS*MAX_BLOCK_SIZE];
    uint32_t *sorted_buffer = &sorted_buffer_[tr*MAX_SORTED_SPLATS];
    // index of element with minimum weight contribution
    uint8_t min_index;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            id_batch[tr] = g_id;
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            printf("%d", buffer_size);

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            // 24 bit depth
            uint32_t cur_depth = (uint32_t)(pos.z/(pos.z+1.0f) * 16777215.0f);

            // add buffer
            if (buffer_size == 0) {
                uint8_t weight = (uint8_t)(255.0f*alpha+0.5f);
                if (weight > 0) {
                    sorted_indices[0] = id_batch[t];
                    sorted_buffer[0] = (cur_depth << 8) + (uint32_t)weight;
                    min_index = 0;
                    buffer_size = 1;
                }
                continue;
            }

            // find insertion index
            int ins_index = buffer_size;
            while (--ins_index >= 0) {
                uint32_t depth_i = sorted_buffer[ins_index] >> 8;
                if (depth_i > cur_depth)
                    break;
            }
            ins_index++;

            // calculate weight
            uint8_t cur_weight = ins_index == 0 ? (uint8_t)255 :
                (uint8_t)(alpha * (uint8_t)sorted_buffer[ins_index-1] + 0.5f);
            uint8_t min_weight = (uint8_t)sorted_buffer[min_index];
            if (cur_weight == 0 || (
                buffer_size >= MAX_SORTED_SPLATS && cur_weight <= min_weight))
                continue;

            // insert vs replace
            float mult = 1.0f - alpha;
            uint8_t new_min_weight = ins_index <= min_index ?
                (uint8_t)(min_weight * mult + 0.5f) : min_weight;
            bool replace_before = min_index < ins_index && (
                new_min_weight == 0 || (buffer_size >= MAX_SORTED_SPLATS && new_min_weight < cur_weight));
            min_weight = min(new_min_weight, cur_weight);
            uint8_t new_min_index = min_index;

            // replace an element before the insert index
            if (replace_before) {
                // update min index for before
                min_weight = (uint8_t)(-1);
                for (int i = 0; i < min_index; i++) {
                    uint8_t weight = (uint8_t)sorted_buffer[i];
                    if (weight <= min_weight)
                        min_weight = weight, new_min_index = i;
                }
                // shift elements
                for (int i = min_index; i < ins_index; i++) {
                    sorted_indices[i] = sorted_indices[i+1];
                    uint32_t info = sorted_buffer[i+1];
                    if ((uint8_t)info <= min_weight)
                        min_weight = (uint8_t)info, new_min_index = i;
                    sorted_buffer[i] = info;
                    // not updating weight here; guess it shouldn't matter much?
                }
                // insert
                sorted_indices[ins_index] = id_batch[t];
                sorted_buffer[ins_index] = (cur_depth << 8) + (uint32_t)cur_weight;
                if (cur_weight < min_weight)
                    min_weight = cur_weight, new_min_index = ins_index;
                // update weights for after, squeeze zero weights
                int offset = 1;
                for (int i = ins_index+1; i+offset <= buffer_size; i++) {
                    uint32_t info = sorted_buffer[i+offset];
                    uint8_t new_weight = (uint8_t)(mult * (uint8_t)info + 0.5f);
                    if (new_weight == 0) {
                        offset++, i--;
                        continue;
                    }
                    if (new_weight <= min_weight)
                        min_weight = new_weight, new_min_index = i;
                    sorted_indices[i] = sorted_indices[i+offset];
                    sorted_buffer[i] = ((info >> 8) << 8) | (uint32_t)new_weight;
                }
                buffer_size -= offset-1;
                min_index = new_min_index;
                continue;
            }

            // replace an element after the insert index
            bool replace_after = min_index >= ins_index && (
                min_weight == 0 || (buffer_size >= MAX_SORTED_SPLATS && min_weight < cur_weight));
            if (replace_after) {
                // update min index for before
                min_weight = (uint8_t)(-1);
                for (int i = 0; i < ins_index; i++) {
                    uint8_t weight = (uint8_t)sorted_buffer[i];
                    if (weight <= min_weight)
                        min_weight = weight, new_min_index = i;
                }
                // shift elements
                for (int i = min_index; i > ins_index; i--) {
                    sorted_indices[i] = sorted_indices[i-1];
                    sorted_buffer[i] = sorted_buffer[i-1];
                }
                // insert
                sorted_indices[ins_index] = id_batch[t];
                sorted_buffer[ins_index] = (cur_depth << 8) + (uint32_t)cur_weight;
                if (cur_weight < min_weight)
                    min_weight = cur_weight, new_min_index = ins_index;
                // update weights for after, squeeze zero weights
                int offset = 0;
                for (int i = ins_index+1; i+offset < buffer_size; i++) {
                    uint32_t info = sorted_buffer[i+offset];
                    uint8_t new_weight = (uint8_t)(mult * (uint8_t)info + 0.5f);
                    if (new_weight == 0) {
                        offset++, i--;
                        continue;
                    }
                    if (new_weight <= min_weight)
                        min_weight = new_weight, new_min_index = i;
                    if (offset > 0)
                        sorted_indices[i] = sorted_indices[i+offset];
                    sorted_buffer[i] = ((info >> 8) << 8) | (uint32_t)new_weight;
                }
                buffer_size -= offset;
                min_index = new_min_index;
                continue;
            }

            // insert an element
            {
                // shift elements
                bool has_zero = false;
                for (int i = buffer_size; i > ins_index; i++) {
                    sorted_indices[i] = sorted_indices[i-1];
                    uint32_t info = sorted_buffer[i-1];
                    uint8_t new_weight = (uint8_t)(mult * (uint8_t)info + 0.5f);
                    if (new_weight == 0) {
                        has_zero = true;
                        continue;
                    }
                    if (new_weight <= min_weight)
                        min_weight = new_weight, new_min_index = i;
                    sorted_buffer[i] = ((info >> 8) << 8) | (uint32_t)new_weight;
                }
                buffer_size += 1;
                // insert
                sorted_indices[ins_index] = id_batch[t];
                sorted_buffer[ins_index] = (cur_depth << 8) + (uint32_t)cur_weight;
                if (cur_weight < min_weight)
                    min_weight = cur_weight, new_min_index = ins_index;
                // squeeze zero weights
                if (has_zero) {
                    int offset = 0;
                    for (int i = ins_index+1; i+offset < buffer_size; i++) {
                        uint32_t info = sorted_buffer[i+offset];
                        if ((uint8_t)info == 0) {
                            offset++, i--;
                            continue;
                        }
                        if ((uint8_t)info <= min_weight)
                            min_weight = (uint8_t)info, new_min_index = i;
                        if (offset > 0) {
                            sorted_indices[i] = sorted_indices[i+offset];
                            sorted_buffer[i] = info;
                        }
                    }
                    buffer_size -= offset;
                }
                min_index = new_min_index;
                continue;
            }

        }
    }

    if (inside) {
        int* out = &out_indices[pix_id*MAX_SORTED_SPLATS];
        for (int i = 0; i < buffer_size; i++)
            out[i] = sorted_indices[i];
        // assume the rest are filled with -1
    }
}
#endif


__global__ void rasterize_simple_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    int* __restrict__ final_index,
    float3* __restrict__ out_img,
    float* __restrict__ out_alpha
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float3 pix_out = {0.f, 0.f, 0.f};
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            glm::vec3 color = color_batch[t];

            const float vis = alpha * T;
            pix_out.x = pix_out.x + color.x * vis;
            pix_out.y = pix_out.y + color.y * vis;
            pix_out.z = pix_out.z + color.z * vis;
            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        float3 final_color;
        final_color.x = pix_out.x + T * background.x;
        final_color.y = pix_out.y + T * background.y;
        final_color.z = pix_out.z + T * background.z;
        out_img[pix_id] = final_color;
        out_alpha[pix_id] = 1.0f - T;
    }
}


__global__ void rasterize_simple_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    const int* __restrict__ final_index,
    const float* __restrict__ output_alpha,
    const float3* __restrict__ v_output,
    const float* __restrict__ v_output_alpha,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float* __restrict__ v_opacities,
    float2* __restrict__ v_anisotropies
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // the contribution from gaussians behind the current one
    float3 buffer = {0.f, 0.f, 0.f};
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    const float3 v_out = v_output[pix_id];
    const float v_out_alpha = v_output_alpha[pix_id];

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, aniso, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            glm::vec3 v_color_local = {0.f, 0.f, 0.f};
            float v_opacity_local = 0.f;
            glm::vec2 v_anisotropy_local = {0.f, 0.f};
            //initialize everything to 0, only set if the lane is valid
            if(valid){
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                // update v_rgb for this gaussian
                float v_alpha = 0.f;
                v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};

                const glm::vec3 color = color_batch[t];
                const glm::vec3 opacity = opacity_batch[t];
                // contribution from this pixel
                v_alpha += (color.x * T - buffer.x) * ra * v_out.x;
                v_alpha += (color.y * T - buffer.y) * ra * v_out.y;
                v_alpha += (color.z * T - buffer.z) * ra * v_out.z;

                v_alpha += T_final * ra * v_out_alpha;
                // contribution from background pixel
                v_alpha += -T_final * ra * background.x * v_out.x;
                v_alpha += -T_final * ra * background.y * v_out.y;
                v_alpha += -T_final * ra * background.z * v_out.z;
                // update the running sum
                buffer.x += color.x * vis;
                buffer.y += color.y * vis;
                buffer.z += color.z * vis;

                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha,
                    v_uv, v_opacity_local, v_anisotropy_local
                );
                glm::mat2x3 v_axis_uv;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    glm::vec3(0), v_uv,
                    v_position_local, v_axis_uv
                );
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                // v_position_xy_abs_local /= pos.z;
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];

                T = next_T;
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum3(v_color_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_anisotropy_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                float* v_color_ptr = (float*)(v_colors);
                atomicAdd(v_color_ptr + 3*g + 0, v_color_local.x);
                atomicAdd(v_color_ptr + 3*g + 1, v_color_local.y);
                atomicAdd(v_color_ptr + 3*g + 2, v_color_local.z);
                
                atomicAdd(v_opacities + g, v_opacity_local);
                float* v_anisotropy_ptr = (float*)(v_anisotropies);
                atomicAdd(v_anisotropy_ptr + 2*g + 0, v_anisotropy_local.x);
                atomicAdd(v_anisotropy_ptr + 2*g + 1, v_anisotropy_local.y);
            }
        }
    }
}





__global__ void rasterize_depth_forward_kernel(
    const int depth_mode,
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    int* __restrict__ final_index,
    float* __restrict__ out_depth,
    float2* __restrict__ out_visibility
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    float interp = 1.0f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float output_depth = 0.0f;
    float output_visibility = 0.0f;
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            // mean depth
            if (depth_mode == DEPTH_MODE_MEAN) {

                // const float depth = pos.z;
                const float depth = poi.z;
                float vis = alpha * T;
                output_depth += vis * depth;

            }  // depth_mode

            // median depth
            else if (depth_mode == DEPTH_MODE_MEDIAN) {

                const float next_depth = poi.z;
                if (next_T < DEPTH_REG_MEDIAN_TH) {
                    if (T < 0.99999f) {
                        // https://www.desmos.com/3d/fttajoozww
                        interp = (1.0f-alpha)/alpha * (T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH;
                        interp = glm::clamp(interp, 0.0f, 1.0f);
                        output_depth = output_depth + (next_depth-output_depth)*interp;
                    }
                    else {
                        output_depth = next_depth;
                    }
                    T = next_T;
                    cur_idx = batch_start + t;
                    done = true;
                    break;
                }
                output_depth = next_depth;

            }  // depth_mode

            T = next_T;
            cur_idx = batch_start + t;
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        if (depth_mode == DEPTH_MODE_MEAN) {
            out_depth[pix_id] = T == 1.0f ? output_depth : output_depth / (1.0f-T);
            // out_depth[pix_id] = output_depth;
            out_visibility[pix_id] = {T, 1.0f-T};
        }
        else if (depth_mode == DEPTH_MODE_MEDIAN) {
            out_depth[pix_id] = output_depth;
            out_visibility[pix_id] = {T, interp};
        }
    }
}


__global__ void rasterize_depth_backward_kernel(
    const int depth_mode,
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const int* __restrict__ final_index,
    const float* __restrict__ out_depth,
    const float2* __restrict__ out_visibility,
    const float* __restrict__ v_out_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float* __restrict__ v_opacities,
    float2* __restrict__ v_anisotropies
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // this is the T AFTER the last gaussian in this pixel
    float2 meta_out = out_visibility[pix_id];
    float T_final = meta_out.x;
    float T = T_final;
    float v_T = 0.0f;
    const float interp = meta_out.y;
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    float output_depth = out_depth[pix_id];
    float v_output_depth = v_out_depth[pix_id];
    float v_out_alpha = 0.0f;
    float v_depth = 0.f;
    float v_depth_next = 0.f;
    float v_alpha = 0.f;
    float v_interp = 0.f;
    if (depth_mode == DEPTH_MODE_MEAN) {
        if (T != 1.0f) {
            float alpha = 1.0f-T;
            v_out_alpha = -output_depth / fmax(alpha, 1e-4f) * v_output_depth;
            output_depth *= alpha;
            v_output_depth /= fmax(alpha, 1e-4);
        }
    }

    float depth_buffer = 0.0f;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, aniso, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            float v_opacity_local = 0.f;
            glm::vec2 v_anisotropy_local = {0.f, 0.f};
            //initialize everything to 0, only set if the lane is valid
            if(valid) {
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                float depth = poi.z;

                // mean depth
                if (depth_mode == DEPTH_MODE_MEAN) {

                    v_depth = vis * v_output_depth;
                    v_alpha = (depth * T - depth_buffer) * ra * v_output_depth;
                    v_alpha += T_final * ra * v_out_alpha;
                    depth_buffer += depth * vis;

                }  // depth_mode

                // median depth
                else if (depth_mode == DEPTH_MODE_MEDIAN) {

                    // depth gradient
                    if (T == T_final) {
                        v_depth = v_output_depth * interp;
                        v_depth_next = v_output_depth * (1.0f-interp);
                    }
                    else {
                        v_depth = v_depth_next;
                        v_depth_next = 0.0f;
                    }

                    // alpha gradient
                    if (T == T_final && interp < 1.0f && interp > 0.0f) {
                        float depth_0 = (output_depth-depth*interp) / (1.0f-interp);
                        v_interp = (depth-depth_0) * v_output_depth;
                        v_alpha = (next_T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH * \
                            v_interp / safe_denom(-alpha*alpha, 1e-3);
                        v_T = (1.0f-alpha)/alpha * v_interp / DEPTH_REG_MEDIAN_TH;
                    }
                    else {
                        v_alpha = v_T * (-next_T);
                        v_T = v_T * (1.0f-alpha);
                    }

                }  // depth_mode

                T = next_T;

                // backward
                const glm::vec3 opacity = opacity_batch[t];
                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha,
                    v_uv, v_opacity_local, v_anisotropy_local
                );
                glm::mat2x3 v_axis_uv;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    {0.f, 0.f, v_depth}, v_uv,
                    v_position_local, v_axis_uv
                );
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_anisotropy_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                atomicAdd(v_opacities + g, v_opacity_local);
                float* v_anisotropy_ptr = (float*)(v_anisotropies);
                atomicAdd(v_anisotropy_ptr + 2*g + 0, v_anisotropy_local.x);
                atomicAdd(v_anisotropy_ptr + 2*g + 1, v_anisotropy_local.y);
            }
        }
    }
}






__global__ void rasterize_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const float depth_reg_pairwise_factor,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float3* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    // const float3& __restrict__ background,
    const float2* __restrict__ depth_grads,
    const float3* __restrict__ depth_ref_im,
    int* __restrict__ final_index,
    float* __restrict__ out_alpha,
    float3* __restrict__ out_img,
    float4* __restrict__ out_depth_grad,
    float* __restrict__ out_reg_depth,
    float* __restrict__ out_reg_normal
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];
    // __shared__ glm::vec2 depth_grad_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float T = 1.f;  // current/total visibility
    float2 g_sum = {0.f, 0.f};  // sum of "normals"
    float3 pix_out = {0.f, 0.f, 0.f};  // output radiance
    float vis_sum = 0.f;  // output alpha
    float depth_sum = 0.f;  // output depth
    float depth_squared_sum = 0.f;  // for L2 depth regularizer
    const float3 depth_ref_raw = inside ?
        depth_ref_im[pix_id] : make_float3(0.f, 0.f, 0.f);
    const float2 depth_normal_ref = {depth_ref_raw.x, depth_ref_raw.y};
    const float depth_ref = depth_ref_raw.z;
    float reg_depth_p = 0.f, reg_depth_i = 0.f;  // output depth regularizer
    float reg_normal = 0.f;  // output normal regularizer
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }
        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            // const float2 depth_grad = depth_grads[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
            // depth_grad_batch[tr] = {depth_grad.x, depth_grad.y};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            glm::vec3 color_0 = color_batch[t];
            glm::vec3 color;
            if (dim_ch > 0) {
                int32_t g_id = id_batch[t];
                const glm::vec3* coeffs = (glm::vec3*)&ch_coeffs[dim_ch*g_id];
                glm::vec3 ch_color = ch_coeffs_to_color(
                    ch_degree_r, ch_degree_r_to_use,
                    ch_degree_phi, ch_degree_phi_to_use,
                    coeffs, {uv.x, uv.y}
                );
                color = color_0 / (1.0f+glm::exp(-ch_color));
            }
            else color = color_0;

            const float vis = alpha * T;
            #if DEPTH_REG_L == 01 && false
            const float depth = pos.z;
            #else
            const float depth = poi.z;
            #endif
            const glm::vec2 g_i = *(glm::vec2*)&depth_grads[id_batch[t]];
            const float g_i_norm = glm::length(g_i) + 1e-6f;
            const glm::vec2 n_i = g_i / g_i_norm;

            pix_out.x = pix_out.x + color.x * vis;
            pix_out.y = pix_out.y + color.y * vis;
            pix_out.z = pix_out.z + color.z * vis;
            {  // depth regularization
                float pairwise_l1 = vis*depth * vis_sum - vis * depth_sum;  // requires pos.z for depth
                float pairwise_l2 = vis * (vis_sum*depth*depth + depth_squared_sum - 2.0f*depth*depth_sum);
                float intersect_l1 = vis * abs(depth - depth_ref);
                float intersect_l2 = vis * (depth-depth_ref) * (depth-depth_ref);
                reg_depth_p += pairwise_l2;
                reg_depth_i += intersect_l1;
            }
            reg_normal += vis * (1.0f - (n_i.x*depth_normal_ref.x+n_i.y*depth_normal_ref.y));
            vis_sum += vis;
            depth_sum += vis*depth;
            depth_squared_sum += vis*depth*depth;
            g_sum.x = g_sum.x + vis * g_i.x;
            g_sum.y = g_sum.y + vis * g_i.y;

            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        out_alpha[pix_id] = 1.0f - T;
        float3 final_color;
        // final_color.x = pix_out.x + T * background.x;
        // final_color.y = pix_out.y + T * background.y;
        // final_color.z = pix_out.z + T * background.z;
        final_color.x = pix_out.x;
        final_color.y = pix_out.y;
        final_color.z = pix_out.z;
        out_img[pix_id] = final_color;
        out_depth_grad[pix_id] = {g_sum.x, g_sum.y, depth_sum, depth_squared_sum};
        out_reg_normal[pix_id] = reg_normal;
        out_reg_depth[pix_id] = reg_depth_i + (reg_depth_p-reg_depth_i) * depth_reg_pairwise_factor;
    }
}


__global__ void rasterize_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float depth_reg_pairwise_factor,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float3* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    // const float3& __restrict__ background,
    const float2* __restrict__ depth_grads,
    const float3* __restrict__ depth_ref_im,
    const int* __restrict__ final_index,
    const float* __restrict__ output_alpha,
    const float4* __restrict__ output_depth_grad,
    const float* __restrict__ v_output_alpha,
    const float3* __restrict__ v_output,
    const float4* __restrict__ v_output_depth_grad,
    const float* __restrict__ v_output_reg_depth,
    const float* __restrict__ v_output_reg_normal,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float3* __restrict__ v_ch_coeffs,
    // float* __restrict__ v_ch_coeffs_abs,
    float* __restrict__ v_opacities,
    float2* __restrict__ v_anisotropies,
    // float3* __restrict__ v_background,
    float2* __restrict__ v_depth_grad,
    float3* __restrict__ v_depth_ref_im
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);
    assert(dim_ch <= MAX_CH_FLOAT3);

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];
    // __shared__ glm::vec2 depth_grad_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    const float4 out_depth_grad = output_depth_grad[pix_id];
    const float3 v_out = v_output[pix_id];
    const float4 v_out_depth_grad = v_output_depth_grad[pix_id];
    const float v_out_alpha = v_output_alpha[pix_id];
    const float v_out_reg_depth = v_output_reg_depth[pix_id];
    const float v_reg_depth_p = v_out_reg_depth * depth_reg_pairwise_factor;
    const float v_reg_depth_i = v_out_reg_depth * (1.0f-depth_reg_pairwise_factor);
    const float v_out_reg_normal = v_output_reg_normal[pix_id];
    const glm::vec2 v_g_sum = {v_out_depth_grad.x, v_out_depth_grad.y};
    const float v_depth_sum = v_out_depth_grad.z;
    const float v_depth_squared_sum = v_out_depth_grad.w;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();

    // regularization
    const float3 depth_ref_raw = inside ?
        depth_ref_im[pix_id] : make_float3(0.f, 0.f, 0.f);
    const float2 depth_normal_ref = {depth_ref_raw.x, depth_ref_raw.y};
    const float depth_ref = depth_ref_raw.z;
    glm::vec2 n_bar = {depth_normal_ref.x, depth_normal_ref.y};
    glm::vec2 v_n_bar = {0.f, 0.f};
    float v_depth_ref = 0.f;

    const float vis_sum_final = 1.0f - T_final;
    const float depth_sum_final = out_depth_grad.z;
    const float depth_squared_sum_final = out_depth_grad.w;
    float vis_sum = vis_sum_final;
    float depth_sum = depth_sum_final;
    float depth_squared_sum = depth_squared_sum_final;
    glm::vec2 g_sum = {out_depth_grad.x, out_depth_grad.y};

    float3 buffer = {0.f, 0.f, 0.f};
    float4 buffer_depth = {0.f, 0.f, 0.f, 0.f};
    float buffer_depth_reg = 0.f;
    float buffer_normal_reg = 0.f;
    
    float v_sum_vis = v_out_alpha;

    // gradient
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    T = T_final;
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            // const float2 depth_grad = depth_grads[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
            // depth_grad_batch[tr] = {depth_grad.x, depth_grad.y};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, aniso, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            glm::vec3 v_color_local = {0.f, 0.f, 0.f};
            glm::vec2 v_depth_grad_local = {0.f, 0.f};
            float v_opacity_local = 0.f;
            glm::vec2 v_anisotropy_local = {0.f, 0.f};
            glm::vec3 v_ch_coeff_local[MAX_CH_FLOAT3];
            for (int i = 0; i < dim_ch; i++)
                v_ch_coeff_local[i] = {0.f, 0.f, 0.f};
            float v_ch_coeff_abs_local = 0.f;
            //initialize everything to 0, only set if the lane is valid
            if(valid){
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                // update accumulation
                v_depth_grad_local.x += vis * v_g_sum.x;
                v_depth_grad_local.y += vis * v_g_sum.y;
                glm::vec3 v_poi = {0.f, 0.f, 0.f};
                #if DEPTH_REG_L == 01 && false
                const float depth = pos.z;
                v_position_local.z += vis * v_depth_sum;
                v_position_local.z += vis * 2.0f*depth * v_depth_squared_sum;
                #else
                const float depth = poi.z;
                v_poi.z += vis * v_depth_sum;
                v_poi.z += vis * 2.0f*depth * v_depth_squared_sum;
                #endif

                // update depth regularizer
                const glm::vec2 depth_grad = *(glm::vec2*)&depth_grads[id_batch[t]];
                float vis_sum_next = vis_sum - vis;
                float depth_sum_next = depth_sum - vis*depth;
                float depth_squared_sum_next = depth_squared_sum - vis*depth*depth;
                #if 0  // pairwise L1, requires pos.z for depth
                v_position_local.z += v_reg_depth_p * vis * (vis_sum_next - (vis_sum_final-vis_sum));
                float reg_depth_i_p = (
                    depth * vis_sum_next - depth_sum_next +
                    (depth_sum_final-depth_sum) - depth * (vis_sum_final-vis_sum)
                );
                #else  // pairwise L2
                v_poi.z += v_reg_depth_p * vis * 2.0f * (
                    vis_sum_final * depth - depth_sum_final);
                float reg_depth_i_p =
                    vis_sum_final*depth*depth + depth_squared_sum_final
                    - 2.0f*depth*depth_sum_final;
                #endif
                #if 1  // L1 with intersected depth
                float v_z = v_reg_depth_i * vis * glm::sign(depth-depth_ref);
                v_poi.z += v_z;
                v_depth_ref += (-v_z);
                float reg_depth_i_i = abs(depth-depth_ref);
                #else  // L2 with intersected depth
                float v_z = v_reg_depth_i * vis * 2.0f*(depth-depth_ref);
                v_poi.z += v_z;
                v_depth_ref += (-v_z);
                float reg_depth_i_i = (depth-depth_ref) * (depth-depth_ref);
                #endif
                float reg_depth_i = reg_depth_i_i + (reg_depth_i_p-reg_depth_i_i) * depth_reg_pairwise_factor;

                // update normal regularizer
                glm::vec2 g_i = {depth_grad.x, depth_grad.y};
                float g_i_norm = glm::length(g_i) + 1e-6f;
                glm::vec2 n_i = g_i / g_i_norm;
                glm::mat2 J_i = (glm::mat2(1.0f) - glm::outerProduct(n_i, n_i)) / g_i_norm;
                float reg_normal_i = 1.0f - dot(n_i, n_bar);
                glm::vec2 v_normal_glm = v_out_reg_normal * (-vis) * J_i * n_bar;
                v_depth_grad_local.x += v_normal_glm.x;
                v_depth_grad_local.y += v_normal_glm.y;
                v_n_bar += vis * (-n_i) * v_out_reg_normal;

                // update color
                glm::vec3 v_color_1 = {vis * v_out.x, vis * v_out.y, vis * v_out.z};
                const glm::vec3 opacity = opacity_batch[t];
                const glm::vec3 color_0 = color_batch[t];
                glm::vec3 color_1;
                glm::vec2 v_uv_ch = {0.f, 0.f};
                if (dim_ch > 0) {
                    glm::vec3 v_ch_color_sigmoid = v_color_1 * color_0;
                    #if 0
                    int32_t g_id = id_batch[t];
                    glm::vec3 ch_color = ch_coeffs_to_color(
                        ch_degree_r, ch_degree_r_to_use,
                        ch_degree_phi, ch_degree_phi_to_use,
                        (glm::vec3*)&ch_coeffs[dim_ch*g_id], {uv.x, uv.y}
                    );
                    glm::vec3 ch_color_sigmoid = 1.0f / (1.0f+glm::exp(-ch_color));
                    glm::vec3 v_ch_color = v_ch_color_sigmoid * ch_color_sigmoid*(1.0f-ch_color_sigmoid);
                    ch_coeffs_to_color_vjp(
                        ch_degree_r, ch_degree_r_to_use,
                        ch_degree_phi, ch_degree_phi_to_use,
                        (glm::vec3*)&ch_coeffs[dim_ch*g_id],
                        {uv.x, uv.y},
                        v_ch_color,
                        ch_color,
                        v_ch_coeff_local, v_ch_coeff_abs_local,
                        v_uv_ch
                    );
                    #else
                    // makes overall training 0.1x faster
                    int32_t g_id = id_batch[t];
                    glm::vec3 ch_color_sigmoid;
                    ch_coeffs_to_color_sigmoid_vjp(
                        ch_degree_r, ch_degree_r_to_use,
                        ch_degree_phi, ch_degree_phi_to_use,
                        (glm::vec3*)&ch_coeffs[dim_ch*g_id],
                        {uv.x, uv.y},
                        v_ch_color_sigmoid,
                        ch_color_sigmoid,
                        v_ch_coeff_local, v_ch_coeff_abs_local,
                        v_uv_ch
                    );
                    #endif
                    color_1 = color_0 * ch_color_sigmoid;
                    v_color_local = v_color_1 * ch_color_sigmoid;
                }
                else {
                    color_1 = color_0;
                    v_color_local = v_color_1;
                }

                float v_alpha = 0.0f;
                // contribution from this pixel
                v_alpha += (color_1.x * T - buffer.x) * ra * v_out.x;
                v_alpha += (color_1.y * T - buffer.y) * ra * v_out.y;
                v_alpha += (color_1.z * T - buffer.z) * ra * v_out.z;
                v_alpha += T_final * ra * v_out_alpha;
                // v_alpha += -T_final * ra * background.x * v_out.x;
                // v_alpha += -T_final * ra * background.y * v_out.y;
                // v_alpha += -T_final * ra * background.z * v_out.z;
                float v_alpha_color_only = v_alpha;
                v_alpha += (depth_grad.x * T - buffer_depth.x) * ra * v_g_sum.x;
                v_alpha += (depth_grad.y * T - buffer_depth.y) * ra * v_g_sum.y;
                v_alpha += (depth * T - buffer_depth.z) * ra * v_depth_sum;
                v_alpha += (depth*depth * T - buffer_depth.w) * ra * v_depth_squared_sum;
                v_alpha += (reg_depth_i * T - buffer_depth_reg) * ra * v_out_reg_depth;
                v_alpha += (reg_normal_i * T - buffer_normal_reg) * ra * v_out_reg_normal;

                // update the running sum
                buffer.x += color_1.x * vis;
                buffer.y += color_1.y * vis;
                buffer.z += color_1.z * vis;
                buffer_depth.x += depth_grad.x * vis;
                buffer_depth.y += depth_grad.y * vis;
                buffer_depth.z += depth * vis;
                buffer_depth.w += depth*depth * vis;
                buffer_depth_reg += reg_depth_i * vis;
                buffer_normal_reg += reg_normal_i * vis;

                // grad
                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha,
                    v_uv, v_opacity_local, v_anisotropy_local
                );
                v_uv += v_uv_ch;
                glm::mat2x3 v_axis_uv;
                glm::vec3 v_position_local_temp;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    v_poi, v_uv,
                    v_position_local_temp, v_axis_uv
                );
                v_position_local += v_position_local_temp;
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];

                // absgrad (color only)
                #if 0
                float v_opacity_local_1;
                glm::vec2 v_anisotropy_local_1;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha_color_only,
                    v_uv, v_opacity_local_1, v_anisotropy_local_1
                );
                v_uv += v_uv_ch;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    glm::vec3(0), v_uv,
                    v_position_local_temp, v_axis_uv
                );
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local_temp));
                #endif

                // next loop
                T = next_T;
                vis_sum = vis_sum_next;
                depth_sum = depth_sum_next;
                depth_squared_sum = depth_squared_sum_next;
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum3(v_color_local, warp);
            for (int i = 0; i < dim_ch; i++)
                warpSum3(v_ch_coeff_local[i], warp);
            warpSum(v_ch_coeff_abs_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_anisotropy_local, warp);
            warpSum2(v_depth_grad_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                float* v_color_ptr = (float*)(v_colors);
                atomicAdd(v_color_ptr + 3*g + 0, v_color_local.x);
                atomicAdd(v_color_ptr + 3*g + 1, v_color_local.y);
                atomicAdd(v_color_ptr + 3*g + 2, v_color_local.z);
                float* v_ch_coeffs_ptr = (float*)(v_ch_coeffs);
                for (int i = 0; i < dim_ch; i++) {
                    atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g + 3*i + 0, v_ch_coeff_local[i].x);
                    atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g + 3*i + 1, v_ch_coeff_local[i].y);
                    atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g + 3*i + 2, v_ch_coeff_local[i].z);
                }
                // atomicAdd(v_ch_coeffs_abs + g, v_ch_coeff_abs_local);

                atomicAdd(v_opacities + g, v_opacity_local);
                float* v_anisotropy_ptr = (float*)(v_anisotropies);
                atomicAdd(v_anisotropy_ptr + 2*g + 0, v_anisotropy_local.x);
                atomicAdd(v_anisotropy_ptr + 2*g + 1, v_anisotropy_local.y);

                float* v_depth_grad_ptr = (float*)(v_depth_grad);
                atomicAdd(v_depth_grad_ptr + 2*g + 0, v_depth_grad_local.x);
                atomicAdd(v_depth_grad_ptr + 2*g + 1, v_depth_grad_local.y);
            }
        }
    }

    if (inside) {
        v_depth_ref_im[pix_id] = {v_n_bar.x, v_n_bar.y, v_depth_ref};

        // background gradient
        #if 0
        float3 v_bkg = {
            v_out.x * T_final,
            v_out.y * T_final,
            v_out.z * T_final
        };
        atomicAdd((float*)v_background+0, v_bkg.x);
        atomicAdd((float*)v_background+1, v_bkg.y);
        atomicAdd((float*)v_background+2, v_bkg.z);
        #endif
    }

}




__global__ void render_background_sh_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const float* rotation,  // row major 3x3
    const unsigned sh_degree,
    const float3* __restrict__ sh_coeffs_float3,
    float3* __restrict__ out_img
) {
    unsigned i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= img_size.y || j >= img_size.x) return;

    float xi = (j + 0.5f - cx) / fx;
    float yi = -(i + 0.5f - cy) / fy;
    float zi = -1.0f;
    float xr = rotation[0] * xi + rotation[1] * yi + rotation[2] * zi;
    float yr = rotation[3] * xi + rotation[4] * yi + rotation[5] * zi;
    float zr = rotation[6] * xi + rotation[7] * yi + rotation[8] * zi;
    float norm = sqrtf(xr * xr + yr * yr + zr * zr);
    float x = xr / norm;
    float y = yr / norm;
    float z = zr / norm;

    float xx = x*x, yy = y*y, zz = z*z;

    glm::vec3 color = glm::vec3(0.0f);
    glm::vec3 *sh_coeffs = (glm::vec3*)sh_coeffs_float3;

    // l0
    color += 0.28209479177387814f * sh_coeffs[0];

    // l1
    if (sh_degree > 1) {
        color += 0.4886025119029199f * y * sh_coeffs[1];
        color += 0.4886025119029199f * z * sh_coeffs[2];
        color += 0.4886025119029199f * x * sh_coeffs[3];
    }

    // l2
    if (sh_degree > 2) {
        color += 1.0925484305920792f * x * y * sh_coeffs[4];
        color += 1.0925484305920792f * y * z * sh_coeffs[5];
        color += (0.9461746957575601f * zz - 0.31539156525251999f) * sh_coeffs[6];
        color += 1.0925484305920792f * x * z * sh_coeffs[7];
        color += 0.5462742152960396f * (xx - yy) * sh_coeffs[8];
    }

    // l3
    if (sh_degree > 3) {
        color += 0.5900435899266435f * y * (3.0f * xx - yy) * sh_coeffs[9];
        color += 2.890611442640554f * x * y * z * sh_coeffs[10];
        color += 0.4570457994644658f * y * (5.0f * zz - 1.0f) * sh_coeffs[11];
        color += 0.3731763325901154f * z * (5.0f * zz - 3.0f) * sh_coeffs[12];
        color += 0.4570457994644658f * x * (5.0f * zz - 1.0f) * sh_coeffs[13];
        color += 1.445305721320277f * z * (xx - yy) * sh_coeffs[14];
        color += 0.5900435899266435f * x * (xx - 3.0f * yy) * sh_coeffs[15];
    }

    // l4
    if (sh_degree > 4) {
        color += 2.5033429417967046f * x * y * (xx - yy) * sh_coeffs[16];
        color += 1.7701307697799304f * y * z * (3.0f * xx - yy) * sh_coeffs[17];
        color += 0.9461746957575601f * x * y * (7.0f * zz - 1.0f) * sh_coeffs[18];
        color += 0.6690465435572892f * y * z * (7.0f * zz - 3.0f) * sh_coeffs[19];
        color += 0.10578554691520431f * (35.0f * zz * zz - 30.0f * zz + 3.0f) * sh_coeffs[20];
        color += 0.6690465435572892f * x * z * (7.0f * zz - 3.0f) * sh_coeffs[21];
        color += 0.47308734787878004f * (xx - yy) * (7.0f * zz - 1.0f) * sh_coeffs[22];
        color += 1.7701307697799304f * x * z * (xx - 3.0f * yy) * sh_coeffs[23];
        color += 0.6258357354491761f * (xx * (xx - 3.0f * yy) - yy * (3.0f * xx - yy)) * sh_coeffs[24];
    }

    color.x = fmaxf(color.x + 0.5f, 0.0f);
    color.y = fmaxf(color.y + 0.5f, 0.0f);
    color.z = fmaxf(color.z + 0.5f, 0.0f);

    out_img[i * img_size.x + j] = *(float3*)&color;
}


__global__ void render_background_sh_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const float* rotation,  // row major 3x3
    const unsigned sh_degree,
    const float3* __restrict__ sh_coeffs_float3,
    const float3* __restrict__ out_color,
    const float3* __restrict__ v_out_color,
    float* __restrict__ v_rotation,
    float3* __restrict__ v_sh_coeffs
) {
    unsigned i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= img_size.y || j >= img_size.x) return;

    unsigned idx = i * img_size.x + j;
    glm::vec3 color = ((glm::vec3*)out_color)[idx];
    glm::vec3 v_color = ((glm::vec3*)v_out_color)[idx];
    if (color.x <= 1e-6f) v_color.x = 0.0f;
    if (color.y <= 1e-6f) v_color.y = 0.0f;
    if (color.z <= 1e-6f) v_color.z = 0.0f;

    float xi = (j + 0.5f - cx) / fx;
    float yi = -(i + 0.5f - cy) / fy;
    float zi = -1.0f;
    float xr = rotation[0] * xi + rotation[1] * yi + rotation[2] * zi;
    float yr = rotation[3] * xi + rotation[4] * yi + rotation[5] * zi;
    float zr = rotation[6] * xi + rotation[7] * yi + rotation[8] * zi;
    float norm2 = xr * xr + yr * yr + zr * zr;
    float norm = sqrtf(norm2);
    float x = xr / norm;
    float y = yr / norm;
    float z = zr / norm;

    float xx = x*x, yy = y*y, zz = z*z;

    float v_x = 0.0f, v_y = 0.0f, v_z = 0.0f;
    float v_xx = 0.0f, v_yy = 0.0f, v_zz = 0.0f;

    glm::vec3 *sh_coeffs = (glm::vec3*)sh_coeffs_float3;

    // l0
    float v_color_dot_sh_coeff = 0.0f;
    glm::vec3 v_sh = 0.28209479177387814f * v_color;
    atomicAdd(&v_sh_coeffs[0].x, v_sh.x);
    atomicAdd(&v_sh_coeffs[0].y, v_sh.y);
    atomicAdd(&v_sh_coeffs[0].z, v_sh.z);

    // l1 - manually calculated
    if (sh_degree > 1) {

        // color += 0.4886025119029199f * y * sh_coeffs[1];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[1]);
        v_y += 0.4886025119029199f * v_color_dot_sh_coeff;
        v_sh = 0.4886025119029199f * y * v_color;
        atomicAdd(&v_sh_coeffs[1].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[1].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[1].z, v_sh.z);

        // color += 0.4886025119029199f * z * sh_coeffs[2];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[2]);
        v_z += 0.4886025119029199f * v_color_dot_sh_coeff;
        v_sh = 0.4886025119029199f * z * v_color;
        atomicAdd(&v_sh_coeffs[2].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[2].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[2].z, v_sh.z);

        // color += 0.4886025119029199f * x * sh_coeffs[3];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[3]);
        v_x += 0.4886025119029199f * v_color_dot_sh_coeff;
        v_sh = 0.4886025119029199f * x * v_color;
        atomicAdd(&v_sh_coeffs[3].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[3].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[3].z, v_sh.z);
    }

    // l2 - manually calculated
    if (sh_degree > 2) {

        // color += 1.0925484305920792f * x * y * sh_coeffs[4];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[4]);
        v_x += 1.0925484305920792f * y * v_color_dot_sh_coeff;
        v_y += 1.0925484305920792f * x * v_color_dot_sh_coeff;
        v_sh = 1.0925484305920792f * x * y * v_color;
        atomicAdd(&v_sh_coeffs[4].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[4].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[4].z, v_sh.z);

        // color += 1.0925484305920792f * y * z * sh_coeffs[5];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[5]);
        v_z += 1.0925484305920792f * y * v_color_dot_sh_coeff;
        v_y += 1.0925484305920792f * z * v_color_dot_sh_coeff;
        v_sh = 1.0925484305920792f * y * z * v_color;
        atomicAdd(&v_sh_coeffs[5].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[5].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[5].z, v_sh.z);

        // color += (0.9461746957575601f * zz - 0.31539156525251999f) * sh_coeffs[6];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[6]);
        v_zz += 0.9461746957575601f * v_color_dot_sh_coeff;
        v_sh = (0.9461746957575601f * zz - 0.31539156525251999f) * v_color;
        atomicAdd(&v_sh_coeffs[6].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[6].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[6].z, v_sh.z);

        // color += 1.0925484305920792f * x * z * sh_coeffs[7];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[7]);
        v_x += 1.0925484305920792f * z * v_color_dot_sh_coeff;
        v_z += 1.0925484305920792f * x * v_color_dot_sh_coeff;
        v_sh = 1.0925484305920792f * x * z * v_color;
        atomicAdd(&v_sh_coeffs[7].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[7].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[7].z, v_sh.z);

        // color += 0.5462742152960396f * (xx - yy) * sh_coeffs[8];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[8]);
        v_xx += 0.5462742152960396f * v_color_dot_sh_coeff;
        v_yy -= 0.5462742152960396f * v_color_dot_sh_coeff;
        v_sh = 0.5462742152960396f * (xx - yy) * v_color;
        atomicAdd(&v_sh_coeffs[8].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[8].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[8].z, v_sh.z);
    }

    // l3 - AI generated, one incorrect line commented
    if (sh_degree > 3) {
        // color += 0.5900435899266435f * y * (3.0f * xx - yy) * sh_coeffs[9];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[9]);
        v_xx += 1.7701307697799305f * y * v_color_dot_sh_coeff;
        v_yy -= 0.5900435899266435f * y * v_color_dot_sh_coeff;
        v_y += 0.5900435899266435f * (3.0f * xx - yy) * v_color_dot_sh_coeff;
        v_sh = 0.5900435899266435f * y * (3.0f * xx - yy) * v_color;
        atomicAdd(&v_sh_coeffs[9].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[9].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[9].z, v_sh.z);

        // color += 2.890611442640554f * x * y * z * sh_coeffs[10];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[10]);
        v_x += 2.890611442640554f * y * z * v_color_dot_sh_coeff;
        v_y += 2.890611442640554f * x * z * v_color_dot_sh_coeff;
        v_z += 2.890611442640554f * x * y * v_color_dot_sh_coeff;
        v_sh = 2.890611442640554f * x * y * z * v_color;
        atomicAdd(&v_sh_coeffs[10].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[10].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[10].z, v_sh.z);

        // color += 0.4570457994644658f * y * (5.0f * zz - 1.0f) * sh_coeffs[11];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[11]);
        v_zz += 2.285228997322329f * y * v_color_dot_sh_coeff;
        v_y += 0.4570457994644658f * (5.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_sh = 0.4570457994644658f * y * (5.0f * zz - 1.0f) * v_color;
        atomicAdd(&v_sh_coeffs[11].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[11].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[11].z, v_sh.z);

        // color += 0.3731763325901154f * z * (5.0f * zz - 3.0f) * sh_coeffs[12];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[12]);
        v_z += 0.3731763325901154f * (5.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_zz += 1.865881662950577f * z * v_color_dot_sh_coeff;
        v_sh = 0.3731763325901154f * z * (5.0f * zz - 3.0f) * v_color;
        atomicAdd(&v_sh_coeffs[12].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[12].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[12].z, v_sh.z);

        // color += 0.4570457994644658f * x * (5.0f * zz - 1.0f) * sh_coeffs[13];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[13]);
        v_x += 0.4570457994644658f * (5.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_zz += 2.285228997322329f * x * v_color_dot_sh_coeff;
        v_sh = 0.4570457994644658f * x * (5.0f * zz - 1.0f) * v_color;
        atomicAdd(&v_sh_coeffs[13].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[13].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[13].z, v_sh.z);

        // color += 1.445305721320277f * z * (xx - yy) * sh_coeffs[14];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[14]);
        v_xx += 1.445305721320277f * z * v_color_dot_sh_coeff;
        v_yy -= 1.445305721320277f * z * v_color_dot_sh_coeff;
        v_z += 1.445305721320277f * (xx - yy) * v_color_dot_sh_coeff;
        v_sh = 1.445305721320277f * z * (xx - yy) * v_color;
        atomicAdd(&v_sh_coeffs[14].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[14].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[14].z, v_sh.z);

        // color += 0.5900435899266435f * x * (xx - 3.0f * yy) * sh_coeffs[15];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[15]);
        // v_xx += 1.1800871798532870f * x * v_color_dot_sh_coeff;
        v_xx += 0.5900435899266435f * x * v_color_dot_sh_coeff;
        v_yy -= 1.7701307697799305f * x * v_color_dot_sh_coeff;
        v_x += 0.5900435899266435f * (xx - 3.0f * yy) * v_color_dot_sh_coeff;
        v_sh = 0.5900435899266435f * x * (xx - 3.0f * yy) * v_color;
        atomicAdd(&v_sh_coeffs[15].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[15].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[15].z, v_sh.z);
    }

    // l4 - AI generated, two incorrect lines commented
    if (sh_degree > 4) {
        // color += 2.5033429417967046f * x * y * (xx - yy) * sh_coeffs[16];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[16]);
        v_x += 2.5033429417967046f * y * (xx - yy) * v_color_dot_sh_coeff;
        v_y += 2.5033429417967046f * x * (xx - yy) * v_color_dot_sh_coeff;
        v_xx += 2.5033429417967046f * x * y * v_color_dot_sh_coeff;
        v_yy -= 2.5033429417967046f * x * y * v_color_dot_sh_coeff;
        v_sh = 2.5033429417967046f * x * y * (xx - yy) * v_color;
        atomicAdd(&v_sh_coeffs[16].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[16].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[16].z, v_sh.z);

        // color += 1.7701307697799304f * y * z * (3.0f * xx - yy) * sh_coeffs[17];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[17]);
        v_xx += 5.3103923093397912f * y * z * v_color_dot_sh_coeff;
        v_yy -= 1.7701307697799304f * y * z * v_color_dot_sh_coeff;
        v_y += 1.7701307697799304f * z * (3.0f * xx - yy) * v_color_dot_sh_coeff;
        v_z += 1.7701307697799304f * y * (3.0f * xx - yy) * v_color_dot_sh_coeff;
        v_sh = 1.7701307697799304f * y * z * (3.0f * xx - yy) * v_color;
        atomicAdd(&v_sh_coeffs[17].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[17].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[17].z, v_sh.z);

        // color += 0.9461746957575601f * x * y * (7.0f * zz - 1.0f) * sh_coeffs[18];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[18]);
        v_x += 0.9461746957575601f * y * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_y += 0.9461746957575601f * x * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_zz += 6.6232228703029207f * x * y * v_color_dot_sh_coeff;
        v_sh = 0.9461746957575601f * x * y * (7.0f * zz - 1.0f) * v_color;
        atomicAdd(&v_sh_coeffs[18].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[18].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[18].z, v_sh.z);

        // color += 0.6690465435572892f * y * z * (7.0f * zz - 3.0f) * sh_coeffs[19];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[19]);
        v_y += 0.6690465435572892f * z * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_z += 0.6690465435572892f * y * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_zz += 4.6833258049010244f * y * z * v_color_dot_sh_coeff;
        v_sh = 0.6690465435572892f * y * z * (7.0f * zz - 3.0f) * v_color;
        atomicAdd(&v_sh_coeffs[19].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[19].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[19].z, v_sh.z);

        // color += 0.10578554691520431f * (35.0f * zz * zz - 30.0f * zz + 3.0f) * sh_coeffs[20];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[20]);
        v_zz += 0.10578554691520431f * (70.0f * zz - 30.0f) * v_color_dot_sh_coeff;
        v_sh = 0.10578554691520431f * (35.0f * zz * zz - 30.0f * zz + 3.0f) * v_color;
        atomicAdd(&v_sh_coeffs[20].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[20].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[20].z, v_sh.z);

        // color += 0.6690465435572892f * x * z * (7.0f * zz - 3.0f) * sh_coeffs[21];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[21]);
        v_x += 0.6690465435572892f * z * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_z += 0.6690465435572892f * x * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_zz += 4.6833258049010244f * x * z * v_color_dot_sh_coeff;
        v_sh = 0.6690465435572892f * x * z * (7.0f * zz - 3.0f) * v_color;
        atomicAdd(&v_sh_coeffs[21].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[21].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[21].z, v_sh.z);

        // color += 0.47308734787878004f * (xx - yy) * (7.0f * zz - 1.0f) * sh_coeffs[22];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[22]);
        v_xx += 0.47308734787878004f * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_yy -= 0.47308734787878004f * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_zz += 3.3116114351514603f * (xx - yy) * v_color_dot_sh_coeff;
        v_sh = 0.47308734787878004f * (xx - yy) * (7.0f * zz - 1.0f) * v_color;
        atomicAdd(&v_sh_coeffs[22].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[22].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[22].z, v_sh.z);

        // color += 1.7701307697799304f * x * z * (xx - 3.0f * yy) * sh_coeffs[23];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[23]);
        v_x += 1.7701307697799304f * z * (xx - 3.0f * yy) * v_color_dot_sh_coeff;
        v_z += 1.7701307697799304f * x * (xx - 3.0f * yy) * v_color_dot_sh_coeff;
        v_xx += 1.7701307697799304f * x * z * v_color_dot_sh_coeff;
        v_yy -= 5.3103923093397912f * x * z * v_color_dot_sh_coeff;
        v_sh = 1.7701307697799304f * x * z * (xx - 3.0f * yy) * v_color;
        atomicAdd(&v_sh_coeffs[23].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[23].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[23].z, v_sh.z);

        // color += 0.6258357354491761f * (xx * (xx - 3.0f * yy) - yy * (3.0f * xx - yy)) * sh_coeffs[24];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[24]);
        // v_xx += 0.6258357354491761f * (4.0f * xx - 6.0f * yy) * v_color_dot_sh_coeff;
        // v_yy += 0.6258357354491761f * (6.0f * yy - 12.0f * xx) * v_color_dot_sh_coeff;
        v_xx += 0.6258357354491761f * (2.0f * xx - 6.0f * yy) * v_color_dot_sh_coeff;
        v_yy += 0.6258357354491761f * (2.0f * yy - 6.0f * xx) * v_color_dot_sh_coeff;
        v_sh = 0.6258357354491761f * (xx * (xx - 3.0f * yy) - yy * (3.0f * xx - yy)) * v_color;
        atomicAdd(&v_sh_coeffs[24].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[24].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[24].z, v_sh.z);
    }

    v_x += v_xx * 2.0f*x;
    v_y += v_yy * 2.0f*y;
    v_z += v_zz * 2.0f*z;

    glm::vec3 xyz = glm::vec3(x, y, z);
    glm::mat3 dp_dpr = (glm::mat3(1.0f) - glm::outerProduct(xyz, xyz)) / norm;
    glm::vec3 v_p = dp_dpr * glm::vec3(v_x, v_y, v_z);
    float v_xi = rotation[0] * v_p.x + rotation[3] * v_p.y + rotation[6] * v_p.z;
    float v_yi = rotation[1] * v_p.x + rotation[4] * v_p.y + rotation[7] * v_p.z;
    float v_zi = rotation[2] * v_p.x + rotation[5] * v_p.y + rotation[8] * v_p.z;

    atomicAdd(&v_rotation[0], v_p.x * xi);
    atomicAdd(&v_rotation[1], v_p.x * yi);
    atomicAdd(&v_rotation[2], v_p.x * zi);
    atomicAdd(&v_rotation[3], v_p.y * xi);
    atomicAdd(&v_rotation[4], v_p.y * yi);
    atomicAdd(&v_rotation[5], v_p.y * zi);
    atomicAdd(&v_rotation[6], v_p.z * xi);
    atomicAdd(&v_rotation[7], v_p.z * yi);
    atomicAdd(&v_rotation[8], v_p.z * zi);
}
