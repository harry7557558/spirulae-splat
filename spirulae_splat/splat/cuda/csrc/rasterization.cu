#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include "ch.cuh"
#include <algorithm>



__global__ void rasterize_simple_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    int* __restrict__ final_index,
    float3* __restrict__ out_img,
    float* __restrict__ out_alpha
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float3 pix_out = {0.f, 0.f, 0.f};
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            glm::vec3 color = color_batch[t];

            const float vis = alpha * T;
            pix_out.x = pix_out.x + color.x * vis;
            pix_out.y = pix_out.y + color.y * vis;
            pix_out.z = pix_out.z + color.z * vis;
            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        float3 final_color;
        final_color.x = pix_out.x + T * background.x;
        final_color.y = pix_out.y + T * background.y;
        final_color.z = pix_out.z + T * background.z;
        out_img[pix_id] = final_color;
        out_alpha[pix_id] = 1.0f - T;
    }
}


__global__ void rasterize_simple_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    const int* __restrict__ final_index,
    const float* __restrict__ output_alpha,
    const float3* __restrict__ v_output,
    const float* __restrict__ v_output_alpha,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float* __restrict__ v_opacities,
    float2* __restrict__ v_anisotropies
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // the contribution from gaussians behind the current one
    float3 buffer = {0.f, 0.f, 0.f};
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    const float3 v_out = nan_to_num(v_output[pix_id]);
    const float v_out_alpha = nan_to_num(v_output_alpha[pix_id]);

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, aniso, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            glm::vec3 v_color_local = {0.f, 0.f, 0.f};
            float v_opacity_local = 0.f;
            glm::vec2 v_anisotropy_local = {0.f, 0.f};
            //initialize everything to 0, only set if the lane is valid
            if(valid){
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                // update v_rgb for this gaussian
                float v_alpha = 0.f;
                v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};

                const glm::vec3 color = color_batch[t];
                const glm::vec3 opacity = opacity_batch[t];
                // contribution from this pixel
                v_alpha += (color.x * T - buffer.x) * ra * v_out.x;
                v_alpha += (color.y * T - buffer.y) * ra * v_out.y;
                v_alpha += (color.z * T - buffer.z) * ra * v_out.z;

                v_alpha += T_final * ra * v_out_alpha;
                // contribution from background pixel
                v_alpha += -T_final * ra * background.x * v_out.x;
                v_alpha += -T_final * ra * background.y * v_out.y;
                v_alpha += -T_final * ra * background.z * v_out.z;
                // update the running sum
                buffer.x += color.x * vis;
                buffer.y += color.y * vis;
                buffer.z += color.z * vis;

                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha,
                    v_uv, v_opacity_local, v_anisotropy_local
                );
                glm::mat2x3 v_axis_uv = glm::mat2x3(0.0f);
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    glm::vec3(0), v_uv,
                    v_position_local, v_axis_uv
                );
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                // v_position_xy_abs_local /= pos.z;
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];

                T = next_T;
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum3(v_color_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_anisotropy_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                float* v_color_ptr = (float*)(v_colors);
                atomicAdd(v_color_ptr + 3*g + 0, v_color_local.x);
                atomicAdd(v_color_ptr + 3*g + 1, v_color_local.y);
                atomicAdd(v_color_ptr + 3*g + 2, v_color_local.z);
                
                atomicAdd(v_opacities + g, v_opacity_local);
                float* v_anisotropy_ptr = (float*)(v_anisotropies);
                atomicAdd(v_anisotropy_ptr + 2*g + 0, v_anisotropy_local.x);
                atomicAdd(v_anisotropy_ptr + 2*g + 1, v_anisotropy_local.y);
            }
        }
    }
}





// rewritten to test if FP16 is faster (answer: no)
template<typename floatt>
__global__ void rasterize_depth_forward_kernel(
    const int depth_mode,
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const vec3<floatt>* __restrict__ positions,
    const vec3<floatt>* __restrict__ axes_u,
    const vec3<floatt>* __restrict__ axes_v,
    const floatt* __restrict__ opacities,
    const vec2<floatt>* __restrict__ anisotropies,
    int* __restrict__ final_index,
    floatt* __restrict__ out_depth,
    vec2<floatt>* __restrict__ out_visibility
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ floatt position_batch_[3*MAX_BLOCK_SIZE];
    __shared__ floatt axes_uv_batch_[6*MAX_BLOCK_SIZE];
    __shared__ floatt opacity_batch_[3*MAX_BLOCK_SIZE];
    vec3<floatt>* const position_batch = (vec3<floatt>*)position_batch_;
    mat2x3<floatt>* const axes_uv_batch = (mat2x3<floatt>*)axes_uv_batch_;
    vec3<floatt>* const opacity_batch = (vec3<floatt>*)opacity_batch_;

    // current visibility left to render
    float T = 1.f;
    float interp = 1.0f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float output_depth = 0.0f;
    float output_visibility = 0.0f;
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const vec3<floatt> pos = positions[g_id];
            const floatt opac = opacities[g_id];
            const vec2<floatt> aniso = anisotropies[g_id];
            const vec3<floatt> v0 = axes_u[g_id];
            const vec3<floatt> v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = (glm::vec3)position_batch[t];
            glm::vec2 aniso = {(float)opacity_batch[t].x, (float)opacity_batch[t].y};
            float opac = (float)opacity_batch[t].z;
            glm::mat2x3 axis_uv = (glm::mat2x3)axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            // mean depth
            if (depth_mode == DEPTH_MODE_MEAN) {

                // const float depth_raw = pos.z;
                const float depth_raw = poi.z;
                const float depth = depth_map(depth_raw);
                float vis = alpha * T;
                output_depth += vis * depth;

            }  // depth_mode

            // median depth
            else if (depth_mode == DEPTH_MODE_MEDIAN) {

                const float next_depth_raw = poi.z;
                const float next_depth = depth_map(next_depth_raw);
                if (next_T < DEPTH_REG_MEDIAN_TH) {
                    if (T < 0.99999f) {
                        // https://www.desmos.com/3d/fttajoozww
                        interp = (1.0f-alpha)/alpha * (T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH;
                        interp = glm::clamp(interp, 0.0f, 1.0f);
                        output_depth = output_depth + (next_depth-output_depth)*interp;
                    }
                    else {
                        output_depth = next_depth;
                    }
                    T = next_T;
                    cur_idx = batch_start + t;
                    done = true;
                    break;
                }
                output_depth = next_depth;

            }  // depth_mode

            T = next_T;
            cur_idx = batch_start + t;
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        if (depth_mode == DEPTH_MODE_MEAN) {
            float depth = T == 1.0f ? output_depth : output_depth / (1.0f-T);
            // out_depth[pix_id] = (floatt)depth_inv_map(depth);
            out_depth[pix_id] = (floatt)depth;
            out_visibility[pix_id] = {(floatt)T, (floatt)(1.0f-T)};
        }
        else if (depth_mode == DEPTH_MODE_MEDIAN) {
            // out_depth[pix_id] = (floatt)depth_inv_map(output_depth);
            out_depth[pix_id] = (floatt)output_depth;
            out_visibility[pix_id] = {(floatt)T, (floatt)interp};
        }
    }
}


// rewritten to test if FP16 is faster (answer: no)
template<typename floatt>
__global__ void rasterize_depth_backward_kernel(
    const int depth_mode,
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const vec3<floatt>* __restrict__ positions,
    const vec3<floatt>* __restrict__ axes_u,
    const vec3<floatt>* __restrict__ axes_v,
    const floatt* __restrict__ opacities,
    const vec2<floatt>* __restrict__ anisotropies,
    const int* __restrict__ final_index,
    const floatt* __restrict__ out_depth,
    const vec2<floatt>* __restrict__ out_visibility,
    const floatt* __restrict__ v_out_depth,
    vec3<floatt>* __restrict__ v_positions,
    vec2<floatt>* __restrict__ v_positions_xy_abs,
    vec3<floatt>* __restrict__ v_axes_u,
    vec3<floatt>* __restrict__ v_axes_v,
    floatt* __restrict__ v_opacities,
    vec2<floatt>* __restrict__ v_anisotropies
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // this is the T AFTER the last gaussian in this pixel
    glm::vec2 meta_out = (glm::vec2)out_visibility[pix_id];
    float T_final = meta_out.x;
    float T = T_final;
    float v_T = 0.0f;
    const float interp = meta_out.y;
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ floatt position_batch_[3*MAX_BLOCK_SIZE];
    __shared__ floatt axes_uv_batch_[6*MAX_BLOCK_SIZE];
    __shared__ floatt opacity_batch_[3*MAX_BLOCK_SIZE];
    vec3<floatt>* const position_batch = (vec3<floatt>*)position_batch_;
    mat2x3<floatt>* const axes_uv_batch = (mat2x3<floatt>*)axes_uv_batch_;
    vec3<floatt>* const opacity_batch = (vec3<floatt>*)opacity_batch_;

    // df/d_out for this pixel
    float output_depth = (float)out_depth[pix_id];
    float v_output_depth = nan_to_num((float)v_out_depth[pix_id]);
    float v_out_alpha = 0.0f;
    float v_depth = 0.f;
    float v_depth_next = 0.f;
    float v_alpha = 0.f;
    float v_interp = 0.f;
    if (depth_mode == DEPTH_MODE_MEAN) {
        if (T != 1.0f) {
            float alpha = 1.0f-T;
            v_out_alpha = -output_depth / fmax(alpha, 1e-4f) * v_output_depth;
            output_depth *= alpha;
            v_output_depth /= fmax(alpha, 1e-4);
        }
    }

    float depth_buffer = 0.0f;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const vec3<floatt> pos = positions[g_id];
            const floatt opac = opacities[g_id];
            const vec2<floatt> aniso = anisotropies[g_id];
            const vec3<floatt> v0 = axes_u[g_id];
            const vec3<floatt> v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = (glm::vec3)position_batch[t];
            glm::vec2 aniso = {(float)opacity_batch[t].x, (float)opacity_batch[t].y};
            float opac = (float)opacity_batch[t].z;
            glm::mat2x3 axis_uv = (glm::mat2x3)axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, aniso, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            float v_opacity_local = 0.f;
            glm::vec2 v_anisotropy_local = {0.f, 0.f};
            //initialize everything to 0, only set if the lane is valid
            if(valid) {
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                float depth_raw = poi.z;
                float depth = depth_map(depth_raw);

                // mean depth
                if (depth_mode == DEPTH_MODE_MEAN) {

                    v_depth = vis * v_output_depth;
                    v_alpha = (depth * T - depth_buffer) * ra * v_output_depth;
                    v_alpha += T_final * ra * v_out_alpha;
                    depth_buffer += depth * vis;

                }  // depth_mode

                // median depth
                else if (depth_mode == DEPTH_MODE_MEDIAN) {

                    // depth gradient
                    if (T == T_final) {
                        v_depth = v_output_depth * interp;
                        v_depth_next = v_output_depth * (1.0f-interp);
                    }
                    else {
                        v_depth = v_depth_next;
                        v_depth_next = 0.0f;
                    }

                    // alpha gradient
                    if (T == T_final && interp < 1.0f && interp > 0.0f) {
                        float depth_0 = (output_depth-depth*interp) / (1.0f-interp);
                        v_interp = (depth-depth_0) * v_output_depth;
                        v_alpha = (next_T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH * \
                            v_interp / safe_denom(-alpha*alpha, 1e-3);
                        v_T = (1.0f-alpha)/alpha * v_interp / DEPTH_REG_MEDIAN_TH;
                    }
                    else {
                        v_alpha = v_T * (-next_T);
                        v_T = v_T * (1.0f-alpha);
                    }

                }  // depth_mode

                T = next_T;

                // backward
                const glm::vec3 opacity = opacity_batch[t];
                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha,
                    v_uv, v_opacity_local, v_anisotropy_local
                );
                glm::mat2x3 v_axis_uv = glm::mat2x3(0.0f);
                float v_depth_raw = depth_map_vjp(depth_raw, v_depth);
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    {0.f, 0.f, v_depth_raw}, v_uv,
                    v_position_local, v_axis_uv
                );
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_anisotropy_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                floatt* v_position_ptr = (floatt*)(v_positions);
                vec3<floatt> v_position_local_ = (vec3<floatt>)v_position_local;
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local_.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local_.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local_.z);
                floatt* v_positions_xy_abs_ptr = (floatt*)(v_positions_xy_abs);
                vec2<floatt> v_position_xy_abs_local_ = (vec2<floatt>)v_position_xy_abs_local;
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local_.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local_.y);

                floatt* v_axis_u_ptr = (floatt*)(v_axes_u);
                vec3<floatt> v_axis_u_local_ = (vec3<floatt>)v_axis_u_local;
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local_.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local_.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local_.z);
                floatt* v_axis_v_ptr = (floatt*)(v_axes_v);
                vec3<floatt> v_axis_v_local_ = (vec3<floatt>)v_axis_v_local;
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local_.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local_.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local_.z);
                
                floatt v_opacity_local_ = (floatt)v_opacity_local;
                atomicAdd(v_opacities + g, v_opacity_local_);
                floatt* v_anisotropy_ptr = (floatt*)(v_anisotropies);
                vec2<floatt> v_anisotropy_local_ = (vec2<floatt>)v_anisotropy_local;
                atomicAdd(v_anisotropy_ptr + 2*g + 0, v_anisotropy_local_.x);
                atomicAdd(v_anisotropy_ptr + 2*g + 1, v_anisotropy_local_.y);
            }
        }
    }
}






__global__ void rasterize_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const float depth_reg_pairwise_factor,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float3* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    // const float3& __restrict__ background,
    const float* __restrict__ depth_ref_im,
    int* __restrict__ final_index,
    float* __restrict__ out_alpha,
    float3* __restrict__ out_img,
    float2* __restrict__ out_depth,
    float3* __restrict__ out_normal,
    float* __restrict__ out_reg_depth
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float T = 1.f;  // current/total visibility
    float3 normal_out = {0.f, 0.f, 0.f};  // sum of normals
    float3 pix_out = {0.f, 0.f, 0.f};  // output radiance
    float vis_sum = 0.f;  // output alpha
    float depth_sum = 0.f;  // output depth
    float depth_squared_sum = 0.f;  // for L2 depth regularizer
    const float depth_ref = inside ? depth_ref_im[pix_id] : 0.f;
    float reg_depth_p = 0.f, reg_depth_i = 0.f;  // output depth regularizer
    float reg_normal = 0.f;  // output normal regularizer
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }
        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            // const float2 depth_grad = depth_grads[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
            // depth_grad_batch[tr] = {depth_grad.x, depth_grad.y};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            glm::vec3 color_0 = color_batch[t];
            glm::vec3 color;
            if (dim_ch > 0) {
                int32_t g_id = id_batch[t];
                const glm::vec3* coeffs = (glm::vec3*)&ch_coeffs[dim_ch*g_id];
                glm::vec3 ch_color = ch_coeffs_to_color(
                    ch_degree_r, ch_degree_r_to_use,
                    ch_degree_phi, ch_degree_phi_to_use,
                    coeffs, {uv.x, uv.y}
                );
                color = color_0 / (1.0f+glm::exp(-ch_color));
            }
            else color = color_0;

            const float vis = alpha * T;
            #if DEPTH_REG_L == 01 && false
            const float depth_raw = pos.z;
            #else
            const float depth_raw = poi.z;
            #endif
            const float depth = depth_map(depth_raw);

            pix_out.x = pix_out.x + color.x * vis;
            pix_out.y = pix_out.y + color.y * vis;
            pix_out.z = pix_out.z + color.z * vis;

            // depth regularization
            {
                float pairwise_l1 = vis*depth * vis_sum - vis * depth_sum;  // requires pos.z for depth
                float pairwise_l2 = vis * (vis_sum*depth*depth + depth_squared_sum - 2.0f*depth*depth_sum);
                float intersect_l1 = vis * abs(depth - depth_ref);
                float intersect_l2 = vis * (depth-depth_ref) * (depth-depth_ref);
                reg_depth_p += pairwise_l2;
                reg_depth_i += intersect_l1;
            }
            vis_sum += vis;
            depth_sum += vis*depth;
            depth_squared_sum += vis*depth*depth;

            // normal regularization
            glm::vec3 normal = get_normal_from_axisuv(axis_uv, poi);
            normal_out.x = normal_out.x + normal.x * vis;
            normal_out.y = normal_out.y + normal.y * vis;
            normal_out.z = normal_out.z + normal.z * vis;

            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        out_alpha[pix_id] = 1.0f - T;
        float3 final_color;
        // final_color.x = pix_out.x + T * background.x;
        // final_color.y = pix_out.y + T * background.y;
        // final_color.z = pix_out.z + T * background.z;
        final_color.x = pix_out.x;
        final_color.y = pix_out.y;
        final_color.z = pix_out.z;
        out_img[pix_id] = final_color;
        out_depth[pix_id] = {depth_sum, depth_squared_sum};
        out_normal[pix_id] = normal_out;
        out_reg_depth[pix_id] = reg_depth_i + (reg_depth_p-reg_depth_i) * depth_reg_pairwise_factor;
    }
}


__global__ void rasterize_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float depth_reg_pairwise_factor,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float3* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    // const float3& __restrict__ background,
    const float* __restrict__ depth_ref_im,
    const int* __restrict__ final_index,
    const float* __restrict__ output_alpha,
    const float2* __restrict__ output_depth,
    const float* __restrict__ v_output_alpha,
    const float3* __restrict__ v_output,
    const float2* __restrict__ v_output_depth,
    const float3* __restrict__ v_output_normal,
    const float* __restrict__ v_output_reg_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float3* __restrict__ v_ch_coeffs,
    // float* __restrict__ v_ch_coeffs_abs,
    float* __restrict__ v_opacities,
    float2* __restrict__ v_anisotropies,
    // float3* __restrict__ v_background,
    float* __restrict__ v_depth_ref_im
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);
    assert(dim_ch <= MAX_CH_FLOAT3);

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];
    // __shared__ glm::vec2 depth_grad_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    const float2 out_depth = output_depth[pix_id];
    const float3 v_out = nan_to_num(v_output[pix_id]);
    const float2 v_out_depth = nan_to_num(v_output_depth[pix_id]);
    const float3 v_out_normal = nan_to_num(v_output_normal[pix_id]);
    const float v_out_alpha = nan_to_num(v_output_alpha[pix_id]);
    const float v_out_reg_depth = nan_to_num(v_output_reg_depth[pix_id]);
    const float v_reg_depth_p = v_out_reg_depth * depth_reg_pairwise_factor;
    const float v_reg_depth_i = v_out_reg_depth * (1.0f-depth_reg_pairwise_factor);
    const float v_depth_sum = v_out_depth.x;
    const float v_depth_squared_sum = v_out_depth.y;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();

    // regularization
    const float depth_ref = inside ? depth_ref_im[pix_id] : 0.f;
    float v_depth_ref = 0.f;

    const float vis_sum_final = 1.0f - T_final;
    const float depth_sum_final = out_depth.x;
    const float depth_squared_sum_final = out_depth.y;
    float vis_sum = vis_sum_final;
    float depth_sum = depth_sum_final;
    float depth_squared_sum = depth_squared_sum_final;

    float3 buffer = {0.f, 0.f, 0.f};
    float2 buffer_depth = {0.f, 0.f};
    float3 buffer_normal = {0.f, 0.f, 0.f};
    float buffer_depth_reg = 0.f;
    
    float v_sum_vis = v_out_alpha;

    // gradient
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    T = T_final;
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, aniso, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            glm::vec3 v_color_local = {0.f, 0.f, 0.f};
            float v_opacity_local = 0.f;
            glm::vec2 v_anisotropy_local = {0.f, 0.f};
            glm::vec3 v_ch_coeff_local[MAX_CH_FLOAT3];
            for (int i = 0; i < dim_ch; i++)
                v_ch_coeff_local[i] = {0.f, 0.f, 0.f};
            float v_ch_coeff_abs_local = 0.f;
            //initialize everything to 0, only set if the lane is valid
            if(valid){
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                // update accumulation
                float v_depth = 0.0f;
                #if DEPTH_REG_L == 01 && false
                const float depth_raw = pos.z;
                const float depth = depth_map(depth_raw);
                v_depth += vis * v_depth_sum;
                v_depth += vis * 2.0f*depth * v_depth_squared_sum;
                #else
                const float depth_raw = poi.z;
                const float depth = depth_map(depth_raw);
                v_depth += vis * v_depth_sum;
                v_depth += vis * 2.0f*depth * v_depth_squared_sum;
                #endif

                // update depth regularizer
                float vis_sum_next = vis_sum - vis;
                float depth_sum_next = depth_sum - vis*depth;
                float depth_squared_sum_next = depth_squared_sum - vis*depth*depth;
                #if 0  // pairwise L1, requires pos.z for depth
                v_depth += v_reg_depth_p * vis * (vis_sum_next - (vis_sum_final-vis_sum));
                float reg_depth_i_p = (
                    depth * vis_sum_next - depth_sum_next +
                    (depth_sum_final-depth_sum) - depth * (vis_sum_final-vis_sum)
                );
                v_position_local.z = depth_map_vjp(depth_raw, v_depth);
                #else  // pairwise L2
                v_depth += v_reg_depth_p * vis * 2.0f * (
                    vis_sum_final * depth - depth_sum_final);
                float reg_depth_i_p =
                    vis_sum_final*depth*depth + depth_squared_sum_final
                    - 2.0f*depth*depth_sum_final;
                #endif
                #if 1  // L1 with intersected depth
                float v_z = v_reg_depth_i * vis * glm::sign(depth-depth_ref);
                v_depth += v_z;
                v_depth_ref += (-v_z);
                float reg_depth_i_i = abs(depth-depth_ref);
                #else  // L2 with intersected depth
                float v_z = v_reg_depth_i * vis * 2.0f*(depth-depth_ref);
                v_depth += v_z;
                v_depth_ref += (-v_z);
                float reg_depth_i_i = (depth-depth_ref) * (depth-depth_ref);
                #endif
                float reg_depth_i = reg_depth_i_i + (reg_depth_i_p-reg_depth_i_i) * depth_reg_pairwise_factor;

                float v_depth_raw = depth_map_vjp(depth_raw, v_depth);
                glm::vec3 v_poi = {0.f, 0.f, v_depth_raw};

                // normal regularization
                glm::vec3 v_normal = {vis * v_out_normal.x, vis * v_out_normal.y, vis * v_out_normal.z};
                glm::mat2x3 v_axis_uv; glm::vec3 normal;
                get_normal_from_axisuv_vjp(axis_uv, poi, v_normal, normal, v_axis_uv);

                // update color
                glm::vec3 v_color_1 = {vis * v_out.x, vis * v_out.y, vis * v_out.z};
                const glm::vec3 opacity = opacity_batch[t];
                const glm::vec3 color_0 = color_batch[t];
                glm::vec3 color_1;
                glm::vec2 v_uv_ch = {0.f, 0.f};
                if (dim_ch > 0) {
                    glm::vec3 v_ch_color_sigmoid = v_color_1 * color_0;
                    #if 0
                    int32_t g_id = id_batch[t];
                    glm::vec3 ch_color = ch_coeffs_to_color(
                        ch_degree_r, ch_degree_r_to_use,
                        ch_degree_phi, ch_degree_phi_to_use,
                        (glm::vec3*)&ch_coeffs[dim_ch*g_id], {uv.x, uv.y}
                    );
                    glm::vec3 ch_color_sigmoid = 1.0f / (1.0f+glm::exp(-ch_color));
                    glm::vec3 v_ch_color = v_ch_color_sigmoid * ch_color_sigmoid*(1.0f-ch_color_sigmoid);
                    ch_coeffs_to_color_vjp(
                        ch_degree_r, ch_degree_r_to_use,
                        ch_degree_phi, ch_degree_phi_to_use,
                        (glm::vec3*)&ch_coeffs[dim_ch*g_id],
                        {uv.x, uv.y},
                        v_ch_color,
                        ch_color,
                        v_ch_coeff_local, v_ch_coeff_abs_local,
                        v_uv_ch
                    );
                    #else
                    // makes overall training 0.1x faster
                    int32_t g_id = id_batch[t];
                    glm::vec3 ch_color_sigmoid;
                    ch_coeffs_to_color_sigmoid_vjp(
                        ch_degree_r, ch_degree_r_to_use,
                        ch_degree_phi, ch_degree_phi_to_use,
                        (glm::vec3*)&ch_coeffs[dim_ch*g_id],
                        {uv.x, uv.y},
                        v_ch_color_sigmoid,
                        ch_color_sigmoid,
                        v_ch_coeff_local, v_ch_coeff_abs_local,
                        v_uv_ch
                    );
                    #endif
                    color_1 = color_0 * ch_color_sigmoid;
                    v_color_local = v_color_1 * ch_color_sigmoid;
                }
                else {
                    color_1 = color_0;
                    v_color_local = v_color_1;
                }

                float v_alpha = 0.0f;
                // contribution from this pixel
                v_alpha += (color_1.x * T - buffer.x) * ra * v_out.x;
                v_alpha += (color_1.y * T - buffer.y) * ra * v_out.y;
                v_alpha += (color_1.z * T - buffer.z) * ra * v_out.z;
                v_alpha += T_final * ra * v_out_alpha;
                // v_alpha += -T_final * ra * background.x * v_out.x;
                // v_alpha += -T_final * ra * background.y * v_out.y;
                // v_alpha += -T_final * ra * background.z * v_out.z;
                float v_alpha_color_only = v_alpha;
                v_alpha += (depth * T - buffer_depth.x) * ra * v_depth_sum;
                v_alpha += (depth*depth * T - buffer_depth.y) * ra * v_depth_squared_sum;
                v_alpha += (reg_depth_i * T - buffer_depth_reg) * ra * v_out_reg_depth;
                v_alpha += (normal.x * T - buffer_normal.x) * ra * v_out_normal.x;
                v_alpha += (normal.y * T - buffer_normal.y) * ra * v_out_normal.y;
                v_alpha += (normal.z * T - buffer_normal.z) * ra * v_out_normal.z;

                // update the running sum
                buffer.x += color_1.x * vis;
                buffer.y += color_1.y * vis;
                buffer.z += color_1.z * vis;
                buffer_depth.x += depth * vis;
                buffer_depth.y += depth*depth * vis;
                buffer_depth_reg += reg_depth_i * vis;
                buffer_normal.x += normal.x * vis;
                buffer_normal.y += normal.y * vis;
                buffer_normal.z += normal.z * vis;

                // grad
                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha,
                    v_uv, v_opacity_local, v_anisotropy_local
                );
                v_uv += v_uv_ch;
                glm::vec3 v_position_local_temp;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    v_poi, v_uv,
                    v_position_local_temp, v_axis_uv
                );
                v_position_local += v_position_local_temp;
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];

                // absgrad (color only)
                #if 0
                float v_opacity_local_1;
                glm::vec2 v_anisotropy_local_1;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha_color_only,
                    v_uv, v_opacity_local_1, v_anisotropy_local_1
                );
                v_uv += v_uv_ch;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    glm::vec3(0), v_uv,
                    v_position_local_temp, v_axis_uv
                );
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local_temp));
                #endif

                // next loop
                T = next_T;
                vis_sum = vis_sum_next;
                depth_sum = depth_sum_next;
                depth_squared_sum = depth_squared_sum_next;
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum3(v_color_local, warp);
            for (int i = 0; i < dim_ch; i++)
                warpSum3(v_ch_coeff_local[i], warp);
            warpSum(v_ch_coeff_abs_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_anisotropy_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                float* v_color_ptr = (float*)(v_colors);
                atomicAdd(v_color_ptr + 3*g + 0, v_color_local.x);
                atomicAdd(v_color_ptr + 3*g + 1, v_color_local.y);
                atomicAdd(v_color_ptr + 3*g + 2, v_color_local.z);
                float* v_ch_coeffs_ptr = (float*)(v_ch_coeffs);
                for (int i = 0; i < dim_ch; i++) {
                    atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g + 3*i + 0, v_ch_coeff_local[i].x);
                    atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g + 3*i + 1, v_ch_coeff_local[i].y);
                    atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g + 3*i + 2, v_ch_coeff_local[i].z);
                }
                // atomicAdd(v_ch_coeffs_abs + g, v_ch_coeff_abs_local);

                atomicAdd(v_opacities + g, v_opacity_local);
                float* v_anisotropy_ptr = (float*)(v_anisotropies);
                atomicAdd(v_anisotropy_ptr + 2*g + 0, v_anisotropy_local.x);
                atomicAdd(v_anisotropy_ptr + 2*g + 1, v_anisotropy_local.y);
            }
        }
    }

    if (inside) {
        v_depth_ref_im[pix_id] = v_depth_ref;

        // background gradient
        #if 0
        float3 v_bkg = {
            v_out.x * T_final,
            v_out.y * T_final,
            v_out.z * T_final
        };
        atomicAdd((float*)v_background+0, v_bkg.x);
        atomicAdd((float*)v_background+1, v_bkg.y);
        atomicAdd((float*)v_background+2, v_bkg.z);
        #endif
    }

}



__global__ void rasterize_simplified_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    int* __restrict__ final_index,
    float* __restrict__ out_alpha,
    float3* __restrict__ out_img,
    float2* __restrict__ out_depth,  // { depth, depth^2 }
    float3* __restrict__ out_normal,
    float* __restrict__ out_depth_reg
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float T = 1.f;  // current/total visibility
    float3 normal_out = {0.f, 0.f, 0.f};  // sum of normals
    float3 pix_out = {0.f, 0.f, 0.f};  // output radiance
    float vis_sum = 0.f;  // output alpha
    float depth_sum = 0.f;  // output depth
    float depth_squared_sum = 0.f;  // for L2 depth regularizer
    float reg_depth_p = 0.f;  // output depth regularizer
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }
        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);
            const float vis = alpha * T;

            // color
            glm::vec3 color = color_batch[t];
            pix_out.x = pix_out.x + color.x * vis;
            pix_out.y = pix_out.y + color.y * vis;
            pix_out.z = pix_out.z + color.z * vis;

            // depth regularization
            const float depth_raw = poi.z;
            const float depth = depth_map(depth_raw);
            {
                float pairwise_l1 = vis*depth * vis_sum - vis * depth_sum;  // requires pos.z for depth
                float pairwise_l2 = vis * (vis_sum*depth*depth + depth_squared_sum - 2.0f*depth*depth_sum);
                reg_depth_p += pairwise_l2;
            }
            vis_sum += vis;
            depth_sum += vis*depth;
            depth_squared_sum += vis*depth*depth;

            // normal regularization
            glm::vec3 normal = get_normal_from_axisuv(axis_uv, poi);
            normal_out.x = normal_out.x + normal.x * vis;
            normal_out.y = normal_out.y + normal.y * vis;
            normal_out.z = normal_out.z + normal.z * vis;

            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        out_alpha[pix_id] = 1.0f - T;
        out_img[pix_id] = pix_out;
        out_depth[pix_id] = { depth_sum, depth_squared_sum };
        out_normal[pix_id] = normal_out;
        out_depth_reg[pix_id] = reg_depth_p;
    }
}


__global__ void rasterize_simplified_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const int* __restrict__ final_index,
    const float* __restrict__ output_alpha,
    const float2* __restrict__ output_depth,
    const float* __restrict__ v_output_alpha,
    const float3* __restrict__ v_output_img,
    const float2* __restrict__ v_output_depth,
    const float3* __restrict__ v_output_normal,
    const float* __restrict__ v_output_depth_reg,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float* __restrict__ v_opacities,
    float2* __restrict__ v_anisotropies
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    const float3 v_out = nan_to_num(v_output_img[pix_id]);
    const float2 v_out_depth = nan_to_num(v_output_depth[pix_id]);
    const float3 v_out_normal = nan_to_num(v_output_normal[pix_id]);
    const float v_out_alpha = nan_to_num(v_output_alpha[pix_id]);
    const float v_reg_depth_p = nan_to_num(v_output_depth_reg[pix_id]);
    const float v_depth_sum = v_out_depth.x;
    const float v_depth_squared_sum = v_out_depth.y;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();

    const float vis_sum_final = 1.0f - T_final;
    const float depth_sum_final = output_depth[pix_id].x;
    const float depth_squared_sum_final = output_depth[pix_id].y;
    float vis_sum = vis_sum_final;

    float3 buffer = {0.f, 0.f, 0.f};
    float2 buffer_depth = {0.f, 0.f};  // depth, depth^2
    float3 buffer_normal = {0.f, 0.f, 0.f};
    float buffer_depth_reg = 0.f;

    float v_sum_vis = v_out_alpha;

    // gradient
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    T = T_final;
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, aniso, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            glm::vec3 v_color_local = {0.f, 0.f, 0.f};
            float v_opacity_local = 0.f;
            glm::vec2 v_anisotropy_local = {0.f, 0.f};
            //initialize everything to 0, only set if the lane is valid
            if(valid){
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                // update accumulation
                const float depth_raw = poi.z;
                const float depth = depth_map(depth_raw);
                float v_depth = 0.0f;
                v_depth += vis * v_depth_sum;
                v_depth += vis * 2.0f*depth * v_depth_squared_sum;

                // update depth regularizer
                float vis_sum_next = vis_sum - vis;
                // pairwise L2
                v_depth += v_reg_depth_p * vis * 2.0f * (
                    vis_sum_final * depth - depth_sum_final);
                float reg_depth_i =
                    vis_sum_final*depth*depth + depth_squared_sum_final
                    - 2.0f*depth*depth_sum_final;

                float v_depth_raw = depth_map_vjp(depth_raw, v_depth);
                glm::vec3 v_poi = {0.f, 0.f, v_depth_raw};

                // update color
                const glm::vec3 opacity = opacity_batch[t];
                glm::vec3 color = color_batch[t];
                v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};

                // normal regularization
                glm::vec3 v_normal = {vis * v_out_normal.x, vis * v_out_normal.y, vis * v_out_normal.z};
                glm::mat2x3 v_axis_uv; glm::vec3 normal;
                get_normal_from_axisuv_vjp(axis_uv, poi, v_normal, normal, v_axis_uv);

                float v_alpha = 0.0f;
                // contribution from this pixel
                v_alpha += (color.x * T - buffer.x) * ra * v_out.x;
                v_alpha += (color.y * T - buffer.y) * ra * v_out.y;
                v_alpha += (color.z * T - buffer.z) * ra * v_out.z;
                v_alpha += T_final * ra * v_out_alpha;
                // v_alpha += -T_final * ra * background.x * v_out.x;
                // v_alpha += -T_final * ra * background.y * v_out.y;
                // v_alpha += -T_final * ra * background.z * v_out.z;
                float v_alpha_color_only = v_alpha;
                v_alpha += (depth * T - buffer_depth.x) * ra * v_depth_sum;
                v_alpha += (depth*depth * T - buffer_depth.y) * ra * v_depth_squared_sum;
                v_alpha += (reg_depth_i * T - buffer_depth_reg) * ra * v_reg_depth_p;
                v_alpha += (normal.x * T - buffer_normal.x) * ra * v_out_normal.x;
                v_alpha += (normal.y * T - buffer_normal.y) * ra * v_out_normal.y;
                v_alpha += (normal.z * T - buffer_normal.z) * ra * v_out_normal.z;

                // update the running sum
                buffer.x += color.x * vis;
                buffer.y += color.y * vis;
                buffer.z += color.z * vis;
                buffer_depth.x += depth * vis;
                buffer_depth.y += depth*depth * vis;
                buffer_depth_reg += reg_depth_i * vis;
                buffer_normal.x += normal.x * vis;
                buffer_normal.y += normal.y * vis;
                buffer_normal.z += normal.z * vis;

                // grad
                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha,
                    v_uv, v_opacity_local, v_anisotropy_local
                );
                glm::vec3 v_position_local_temp;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    v_poi, v_uv,
                    v_position_local_temp, v_axis_uv
                );
                v_position_local += v_position_local_temp;
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];

                // next loop
                T = next_T;
                vis_sum = vis_sum_next;
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum3(v_color_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_anisotropy_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                float* v_color_ptr = (float*)(v_colors);
                atomicAdd(v_color_ptr + 3*g + 0, v_color_local.x);
                atomicAdd(v_color_ptr + 3*g + 1, v_color_local.y);
                atomicAdd(v_color_ptr + 3*g + 2, v_color_local.z);

                atomicAdd(v_opacities + g, v_opacity_local);
                float* v_anisotropy_ptr = (float*)(v_anisotropies);
                atomicAdd(v_anisotropy_ptr + 2*g + 0, v_anisotropy_local.x);
                atomicAdd(v_anisotropy_ptr + 2*g + 1, v_anisotropy_local.y);
            }
        }
    }

}




__global__ void render_background_sh_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const float* rotation,  // row major 3x3
    const unsigned sh_degree,
    const float3* __restrict__ sh_coeffs_float3,
    float3* __restrict__ out_img
) {
    unsigned i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= img_size.y || j >= img_size.x) return;

    float xi = (j + 0.5f - cx) / fx;
    float yi = -(i + 0.5f - cy) / fy;
    float zi = -1.0f;
    float xr = rotation[0] * xi + rotation[1] * yi + rotation[2] * zi;
    float yr = rotation[3] * xi + rotation[4] * yi + rotation[5] * zi;
    float zr = rotation[6] * xi + rotation[7] * yi + rotation[8] * zi;
    float norm = sqrtf(xr * xr + yr * yr + zr * zr);
    float x = xr / norm;
    float y = yr / norm;
    float z = zr / norm;

    float xx = x*x, yy = y*y, zz = z*z;

    glm::vec3 color = glm::vec3(0.0f);
    glm::vec3 *sh_coeffs = (glm::vec3*)sh_coeffs_float3;

    // l0
    color += 0.28209479177387814f * sh_coeffs[0];

    // l1
    if (sh_degree > 1) {
        color += 0.4886025119029199f * y * sh_coeffs[1];
        color += 0.4886025119029199f * z * sh_coeffs[2];
        color += 0.4886025119029199f * x * sh_coeffs[3];
    }

    // l2
    if (sh_degree > 2) {
        color += 1.0925484305920792f * x * y * sh_coeffs[4];
        color += 1.0925484305920792f * y * z * sh_coeffs[5];
        color += (0.9461746957575601f * zz - 0.31539156525251999f) * sh_coeffs[6];
        color += 1.0925484305920792f * x * z * sh_coeffs[7];
        color += 0.5462742152960396f * (xx - yy) * sh_coeffs[8];
    }

    // l3
    if (sh_degree > 3) {
        color += 0.5900435899266435f * y * (3.0f * xx - yy) * sh_coeffs[9];
        color += 2.890611442640554f * x * y * z * sh_coeffs[10];
        color += 0.4570457994644658f * y * (5.0f * zz - 1.0f) * sh_coeffs[11];
        color += 0.3731763325901154f * z * (5.0f * zz - 3.0f) * sh_coeffs[12];
        color += 0.4570457994644658f * x * (5.0f * zz - 1.0f) * sh_coeffs[13];
        color += 1.445305721320277f * z * (xx - yy) * sh_coeffs[14];
        color += 0.5900435899266435f * x * (xx - 3.0f * yy) * sh_coeffs[15];
    }

    // l4
    if (sh_degree > 4) {
        color += 2.5033429417967046f * x * y * (xx - yy) * sh_coeffs[16];
        color += 1.7701307697799304f * y * z * (3.0f * xx - yy) * sh_coeffs[17];
        color += 0.9461746957575601f * x * y * (7.0f * zz - 1.0f) * sh_coeffs[18];
        color += 0.6690465435572892f * y * z * (7.0f * zz - 3.0f) * sh_coeffs[19];
        color += 0.10578554691520431f * (35.0f * zz * zz - 30.0f * zz + 3.0f) * sh_coeffs[20];
        color += 0.6690465435572892f * x * z * (7.0f * zz - 3.0f) * sh_coeffs[21];
        color += 0.47308734787878004f * (xx - yy) * (7.0f * zz - 1.0f) * sh_coeffs[22];
        color += 1.7701307697799304f * x * z * (xx - 3.0f * yy) * sh_coeffs[23];
        color += 0.6258357354491761f * (xx * (xx - 3.0f * yy) - yy * (3.0f * xx - yy)) * sh_coeffs[24];
    }

    color.x = fmaxf(color.x + 0.5f, 0.0f);
    color.y = fmaxf(color.y + 0.5f, 0.0f);
    color.z = fmaxf(color.z + 0.5f, 0.0f);

    out_img[i * img_size.x + j] = *(float3*)&color;
}


__global__ void render_background_sh_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float fx,
    const float fy,
    const float cx,
    const float cy,
    const float* rotation,  // row major 3x3
    const unsigned sh_degree,
    const float3* __restrict__ sh_coeffs_float3,
    const float3* __restrict__ out_color,
    const float3* __restrict__ v_out_color,
    float* __restrict__ v_rotation,
    float3* __restrict__ v_sh_coeffs
) {
    unsigned i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= img_size.y || j >= img_size.x) return;

    unsigned idx = i * img_size.x + j;
    glm::vec3 color = ((glm::vec3*)out_color)[idx];
    glm::vec3 v_color = ((glm::vec3*)v_out_color)[idx];
    if (color.x <= 1e-6f) v_color.x = 0.0f;
    if (color.y <= 1e-6f) v_color.y = 0.0f;
    if (color.z <= 1e-6f) v_color.z = 0.0f;

    float xi = (j + 0.5f - cx) / fx;
    float yi = -(i + 0.5f - cy) / fy;
    float zi = -1.0f;
    float xr = rotation[0] * xi + rotation[1] * yi + rotation[2] * zi;
    float yr = rotation[3] * xi + rotation[4] * yi + rotation[5] * zi;
    float zr = rotation[6] * xi + rotation[7] * yi + rotation[8] * zi;
    float norm2 = xr * xr + yr * yr + zr * zr;
    float norm = sqrtf(norm2);
    float x = xr / norm;
    float y = yr / norm;
    float z = zr / norm;

    float xx = x*x, yy = y*y, zz = z*z;

    float v_x = 0.0f, v_y = 0.0f, v_z = 0.0f;
    float v_xx = 0.0f, v_yy = 0.0f, v_zz = 0.0f;

    glm::vec3 *sh_coeffs = (glm::vec3*)sh_coeffs_float3;

    // l0
    float v_color_dot_sh_coeff = 0.0f;
    glm::vec3 v_sh = 0.28209479177387814f * v_color;
    atomicAdd(&v_sh_coeffs[0].x, v_sh.x);
    atomicAdd(&v_sh_coeffs[0].y, v_sh.y);
    atomicAdd(&v_sh_coeffs[0].z, v_sh.z);

    // l1 - manually calculated
    if (sh_degree > 1) {

        // color += 0.4886025119029199f * y * sh_coeffs[1];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[1]);
        v_y += 0.4886025119029199f * v_color_dot_sh_coeff;
        v_sh = 0.4886025119029199f * y * v_color;
        atomicAdd(&v_sh_coeffs[1].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[1].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[1].z, v_sh.z);

        // color += 0.4886025119029199f * z * sh_coeffs[2];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[2]);
        v_z += 0.4886025119029199f * v_color_dot_sh_coeff;
        v_sh = 0.4886025119029199f * z * v_color;
        atomicAdd(&v_sh_coeffs[2].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[2].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[2].z, v_sh.z);

        // color += 0.4886025119029199f * x * sh_coeffs[3];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[3]);
        v_x += 0.4886025119029199f * v_color_dot_sh_coeff;
        v_sh = 0.4886025119029199f * x * v_color;
        atomicAdd(&v_sh_coeffs[3].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[3].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[3].z, v_sh.z);
    }

    // l2 - manually calculated
    if (sh_degree > 2) {

        // color += 1.0925484305920792f * x * y * sh_coeffs[4];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[4]);
        v_x += 1.0925484305920792f * y * v_color_dot_sh_coeff;
        v_y += 1.0925484305920792f * x * v_color_dot_sh_coeff;
        v_sh = 1.0925484305920792f * x * y * v_color;
        atomicAdd(&v_sh_coeffs[4].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[4].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[4].z, v_sh.z);

        // color += 1.0925484305920792f * y * z * sh_coeffs[5];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[5]);
        v_z += 1.0925484305920792f * y * v_color_dot_sh_coeff;
        v_y += 1.0925484305920792f * z * v_color_dot_sh_coeff;
        v_sh = 1.0925484305920792f * y * z * v_color;
        atomicAdd(&v_sh_coeffs[5].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[5].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[5].z, v_sh.z);

        // color += (0.9461746957575601f * zz - 0.31539156525251999f) * sh_coeffs[6];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[6]);
        v_zz += 0.9461746957575601f * v_color_dot_sh_coeff;
        v_sh = (0.9461746957575601f * zz - 0.31539156525251999f) * v_color;
        atomicAdd(&v_sh_coeffs[6].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[6].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[6].z, v_sh.z);

        // color += 1.0925484305920792f * x * z * sh_coeffs[7];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[7]);
        v_x += 1.0925484305920792f * z * v_color_dot_sh_coeff;
        v_z += 1.0925484305920792f * x * v_color_dot_sh_coeff;
        v_sh = 1.0925484305920792f * x * z * v_color;
        atomicAdd(&v_sh_coeffs[7].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[7].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[7].z, v_sh.z);

        // color += 0.5462742152960396f * (xx - yy) * sh_coeffs[8];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[8]);
        v_xx += 0.5462742152960396f * v_color_dot_sh_coeff;
        v_yy -= 0.5462742152960396f * v_color_dot_sh_coeff;
        v_sh = 0.5462742152960396f * (xx - yy) * v_color;
        atomicAdd(&v_sh_coeffs[8].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[8].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[8].z, v_sh.z);
    }

    // l3 - AI generated, one incorrect line commented
    if (sh_degree > 3) {
        // color += 0.5900435899266435f * y * (3.0f * xx - yy) * sh_coeffs[9];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[9]);
        v_xx += 1.7701307697799305f * y * v_color_dot_sh_coeff;
        v_yy -= 0.5900435899266435f * y * v_color_dot_sh_coeff;
        v_y += 0.5900435899266435f * (3.0f * xx - yy) * v_color_dot_sh_coeff;
        v_sh = 0.5900435899266435f * y * (3.0f * xx - yy) * v_color;
        atomicAdd(&v_sh_coeffs[9].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[9].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[9].z, v_sh.z);

        // color += 2.890611442640554f * x * y * z * sh_coeffs[10];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[10]);
        v_x += 2.890611442640554f * y * z * v_color_dot_sh_coeff;
        v_y += 2.890611442640554f * x * z * v_color_dot_sh_coeff;
        v_z += 2.890611442640554f * x * y * v_color_dot_sh_coeff;
        v_sh = 2.890611442640554f * x * y * z * v_color;
        atomicAdd(&v_sh_coeffs[10].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[10].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[10].z, v_sh.z);

        // color += 0.4570457994644658f * y * (5.0f * zz - 1.0f) * sh_coeffs[11];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[11]);
        v_zz += 2.285228997322329f * y * v_color_dot_sh_coeff;
        v_y += 0.4570457994644658f * (5.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_sh = 0.4570457994644658f * y * (5.0f * zz - 1.0f) * v_color;
        atomicAdd(&v_sh_coeffs[11].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[11].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[11].z, v_sh.z);

        // color += 0.3731763325901154f * z * (5.0f * zz - 3.0f) * sh_coeffs[12];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[12]);
        v_z += 0.3731763325901154f * (5.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_zz += 1.865881662950577f * z * v_color_dot_sh_coeff;
        v_sh = 0.3731763325901154f * z * (5.0f * zz - 3.0f) * v_color;
        atomicAdd(&v_sh_coeffs[12].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[12].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[12].z, v_sh.z);

        // color += 0.4570457994644658f * x * (5.0f * zz - 1.0f) * sh_coeffs[13];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[13]);
        v_x += 0.4570457994644658f * (5.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_zz += 2.285228997322329f * x * v_color_dot_sh_coeff;
        v_sh = 0.4570457994644658f * x * (5.0f * zz - 1.0f) * v_color;
        atomicAdd(&v_sh_coeffs[13].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[13].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[13].z, v_sh.z);

        // color += 1.445305721320277f * z * (xx - yy) * sh_coeffs[14];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[14]);
        v_xx += 1.445305721320277f * z * v_color_dot_sh_coeff;
        v_yy -= 1.445305721320277f * z * v_color_dot_sh_coeff;
        v_z += 1.445305721320277f * (xx - yy) * v_color_dot_sh_coeff;
        v_sh = 1.445305721320277f * z * (xx - yy) * v_color;
        atomicAdd(&v_sh_coeffs[14].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[14].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[14].z, v_sh.z);

        // color += 0.5900435899266435f * x * (xx - 3.0f * yy) * sh_coeffs[15];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[15]);
        // v_xx += 1.1800871798532870f * x * v_color_dot_sh_coeff;
        v_xx += 0.5900435899266435f * x * v_color_dot_sh_coeff;
        v_yy -= 1.7701307697799305f * x * v_color_dot_sh_coeff;
        v_x += 0.5900435899266435f * (xx - 3.0f * yy) * v_color_dot_sh_coeff;
        v_sh = 0.5900435899266435f * x * (xx - 3.0f * yy) * v_color;
        atomicAdd(&v_sh_coeffs[15].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[15].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[15].z, v_sh.z);
    }

    // l4 - AI generated, two incorrect lines commented
    if (sh_degree > 4) {
        // color += 2.5033429417967046f * x * y * (xx - yy) * sh_coeffs[16];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[16]);
        v_x += 2.5033429417967046f * y * (xx - yy) * v_color_dot_sh_coeff;
        v_y += 2.5033429417967046f * x * (xx - yy) * v_color_dot_sh_coeff;
        v_xx += 2.5033429417967046f * x * y * v_color_dot_sh_coeff;
        v_yy -= 2.5033429417967046f * x * y * v_color_dot_sh_coeff;
        v_sh = 2.5033429417967046f * x * y * (xx - yy) * v_color;
        atomicAdd(&v_sh_coeffs[16].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[16].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[16].z, v_sh.z);

        // color += 1.7701307697799304f * y * z * (3.0f * xx - yy) * sh_coeffs[17];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[17]);
        v_xx += 5.3103923093397912f * y * z * v_color_dot_sh_coeff;
        v_yy -= 1.7701307697799304f * y * z * v_color_dot_sh_coeff;
        v_y += 1.7701307697799304f * z * (3.0f * xx - yy) * v_color_dot_sh_coeff;
        v_z += 1.7701307697799304f * y * (3.0f * xx - yy) * v_color_dot_sh_coeff;
        v_sh = 1.7701307697799304f * y * z * (3.0f * xx - yy) * v_color;
        atomicAdd(&v_sh_coeffs[17].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[17].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[17].z, v_sh.z);

        // color += 0.9461746957575601f * x * y * (7.0f * zz - 1.0f) * sh_coeffs[18];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[18]);
        v_x += 0.9461746957575601f * y * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_y += 0.9461746957575601f * x * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_zz += 6.6232228703029207f * x * y * v_color_dot_sh_coeff;
        v_sh = 0.9461746957575601f * x * y * (7.0f * zz - 1.0f) * v_color;
        atomicAdd(&v_sh_coeffs[18].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[18].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[18].z, v_sh.z);

        // color += 0.6690465435572892f * y * z * (7.0f * zz - 3.0f) * sh_coeffs[19];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[19]);
        v_y += 0.6690465435572892f * z * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_z += 0.6690465435572892f * y * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_zz += 4.6833258049010244f * y * z * v_color_dot_sh_coeff;
        v_sh = 0.6690465435572892f * y * z * (7.0f * zz - 3.0f) * v_color;
        atomicAdd(&v_sh_coeffs[19].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[19].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[19].z, v_sh.z);

        // color += 0.10578554691520431f * (35.0f * zz * zz - 30.0f * zz + 3.0f) * sh_coeffs[20];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[20]);
        v_zz += 0.10578554691520431f * (70.0f * zz - 30.0f) * v_color_dot_sh_coeff;
        v_sh = 0.10578554691520431f * (35.0f * zz * zz - 30.0f * zz + 3.0f) * v_color;
        atomicAdd(&v_sh_coeffs[20].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[20].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[20].z, v_sh.z);

        // color += 0.6690465435572892f * x * z * (7.0f * zz - 3.0f) * sh_coeffs[21];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[21]);
        v_x += 0.6690465435572892f * z * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_z += 0.6690465435572892f * x * (7.0f * zz - 3.0f) * v_color_dot_sh_coeff;
        v_zz += 4.6833258049010244f * x * z * v_color_dot_sh_coeff;
        v_sh = 0.6690465435572892f * x * z * (7.0f * zz - 3.0f) * v_color;
        atomicAdd(&v_sh_coeffs[21].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[21].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[21].z, v_sh.z);

        // color += 0.47308734787878004f * (xx - yy) * (7.0f * zz - 1.0f) * sh_coeffs[22];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[22]);
        v_xx += 0.47308734787878004f * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_yy -= 0.47308734787878004f * (7.0f * zz - 1.0f) * v_color_dot_sh_coeff;
        v_zz += 3.3116114351514603f * (xx - yy) * v_color_dot_sh_coeff;
        v_sh = 0.47308734787878004f * (xx - yy) * (7.0f * zz - 1.0f) * v_color;
        atomicAdd(&v_sh_coeffs[22].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[22].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[22].z, v_sh.z);

        // color += 1.7701307697799304f * x * z * (xx - 3.0f * yy) * sh_coeffs[23];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[23]);
        v_x += 1.7701307697799304f * z * (xx - 3.0f * yy) * v_color_dot_sh_coeff;
        v_z += 1.7701307697799304f * x * (xx - 3.0f * yy) * v_color_dot_sh_coeff;
        v_xx += 1.7701307697799304f * x * z * v_color_dot_sh_coeff;
        v_yy -= 5.3103923093397912f * x * z * v_color_dot_sh_coeff;
        v_sh = 1.7701307697799304f * x * z * (xx - 3.0f * yy) * v_color;
        atomicAdd(&v_sh_coeffs[23].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[23].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[23].z, v_sh.z);

        // color += 0.6258357354491761f * (xx * (xx - 3.0f * yy) - yy * (3.0f * xx - yy)) * sh_coeffs[24];
        v_color_dot_sh_coeff = glm::dot(v_color, sh_coeffs[24]);
        // v_xx += 0.6258357354491761f * (4.0f * xx - 6.0f * yy) * v_color_dot_sh_coeff;
        // v_yy += 0.6258357354491761f * (6.0f * yy - 12.0f * xx) * v_color_dot_sh_coeff;
        v_xx += 0.6258357354491761f * (2.0f * xx - 6.0f * yy) * v_color_dot_sh_coeff;
        v_yy += 0.6258357354491761f * (2.0f * yy - 6.0f * xx) * v_color_dot_sh_coeff;
        v_sh = 0.6258357354491761f * (xx * (xx - 3.0f * yy) - yy * (3.0f * xx - yy)) * v_color;
        atomicAdd(&v_sh_coeffs[24].x, v_sh.x);
        atomicAdd(&v_sh_coeffs[24].y, v_sh.y);
        atomicAdd(&v_sh_coeffs[24].z, v_sh.z);
    }

    v_x += v_xx * 2.0f*x;
    v_y += v_yy * 2.0f*y;
    v_z += v_zz * 2.0f*z;

    glm::vec3 xyz = glm::vec3(x, y, z);
    glm::mat3 dp_dpr = (glm::mat3(1.0f) - glm::outerProduct(xyz, xyz)) / norm;
    glm::vec3 v_p = dp_dpr * glm::vec3(v_x, v_y, v_z);
    float v_xi = rotation[0] * v_p.x + rotation[3] * v_p.y + rotation[6] * v_p.z;
    float v_yi = rotation[1] * v_p.x + rotation[4] * v_p.y + rotation[7] * v_p.z;
    float v_zi = rotation[2] * v_p.x + rotation[5] * v_p.y + rotation[8] * v_p.z;

    atomicAdd(&v_rotation[0], v_p.x * xi);
    atomicAdd(&v_rotation[1], v_p.x * yi);
    atomicAdd(&v_rotation[2], v_p.x * zi);
    atomicAdd(&v_rotation[3], v_p.y * xi);
    atomicAdd(&v_rotation[4], v_p.y * yi);
    atomicAdd(&v_rotation[5], v_p.y * zi);
    atomicAdd(&v_rotation[6], v_p.z * xi);
    atomicAdd(&v_rotation[7], v_p.z * yi);
    atomicAdd(&v_rotation[8], v_p.z * zi);
}




template __global__ void rasterize_depth_forward_kernel<float>(
    const int depth_mode,
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const vec3<float>* __restrict__ positions,
    const vec3<float>* __restrict__ axes_u,
    const vec3<float>* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const vec2<float>* __restrict__ anisotropies,
    int* __restrict__ final_index,
    float* __restrict__ out_depth,
    vec2<float>* __restrict__ out_visibility
);

template __global__ void rasterize_depth_forward_kernel<halfc>(
    const int depth_mode,
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const vec3<halfc>* __restrict__ positions,
    const vec3<halfc>* __restrict__ axes_u,
    const vec3<halfc>* __restrict__ axes_v,
    const halfc* __restrict__ opacities,
    const vec2<halfc>* __restrict__ anisotropies,
    int* __restrict__ final_index,
    halfc* __restrict__ out_depth,
    vec2<halfc>* __restrict__ out_visibility
);

template __global__ void rasterize_depth_backward_kernel<float>(
    const int depth_mode,
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const vec3<float>* __restrict__ positions,
    const vec3<float>* __restrict__ axes_u,
    const vec3<float>* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const vec2<float>* __restrict__ anisotropies,
    const int* __restrict__ final_index,
    const float* __restrict__ out_depth,
    const vec2<float>* __restrict__ out_visibility,
    const float* __restrict__ v_out_depth,
    vec3<float>* __restrict__ v_positions,
    vec2<float>* __restrict__ v_positions_xy_abs,
    vec3<float>* __restrict__ v_axes_u,
    vec3<float>* __restrict__ v_axes_v,
    float* __restrict__ v_opacities,
    vec2<float>* __restrict__ v_anisotropies
);

template __global__ void rasterize_depth_backward_kernel<halfc>(
    const int depth_mode,
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const vec3<halfc>* __restrict__ positions,
    const vec3<halfc>* __restrict__ axes_u,
    const vec3<halfc>* __restrict__ axes_v,
    const halfc* __restrict__ opacities,
    const vec2<halfc>* __restrict__ anisotropies,
    const int* __restrict__ final_index,
    const halfc* __restrict__ out_depth,
    const vec2<halfc>* __restrict__ out_visibility,
    const halfc* __restrict__ v_out_depth,
    vec3<halfc>* __restrict__ v_positions,
    vec2<halfc>* __restrict__ v_positions_xy_abs,
    vec3<halfc>* __restrict__ v_axes_u,
    vec3<halfc>* __restrict__ v_axes_v,
    halfc* __restrict__ v_opacities,
    vec2<halfc>* __restrict__ v_anisotropies
);
