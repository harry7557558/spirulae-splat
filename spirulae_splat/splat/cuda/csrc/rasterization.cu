#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include "ch.cuh"
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <iostream>
#include <hip/hip_fp16.h>
namespace cg = cooperative_groups;


template<typename vec3>
inline __device__ void warpSum3(vec3& val, cg::thread_block_tile<32>& tile){
    val.x = cg::reduce(tile, val.x, cg::plus<float>());
    val.y = cg::reduce(tile, val.y, cg::plus<float>());
    val.z = cg::reduce(tile, val.z, cg::plus<float>());
}

template<typename vec2>
inline __device__ void warpSum2(vec2& val, cg::thread_block_tile<32>& tile){
    val.x = cg::reduce(tile, val.x, cg::plus<float>());
    val.y = cg::reduce(tile, val.y, cg::plus<float>());
}

inline __device__ void warpSum(float& val, cg::thread_block_tile<32>& tile){
    val = cg::reduce(tile, val, cg::plus<float>());
}



#if 0
__global__ void rasterize_sorted_indices_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    int* __restrict__ out_indices
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int tr = block.thread_rank();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // number of elements in the buffer
    int buffer_size = 0;
    // sorted global index
    __shared__ int32_t sorted_indices_[MAX_SORTED_SPLATS*MAX_BLOCK_SIZE];
    int32_t *sorted_indices = &sorted_indices_[tr*MAX_SORTED_SPLATS];
    // 24 bit depth, 8 bit weight
    __shared__ uint32_t sorted_buffer_[MAX_SORTED_SPLATS*MAX_BLOCK_SIZE];
    uint32_t *sorted_buffer = &sorted_buffer_[tr*MAX_SORTED_SPLATS];
    // index of element with minimum weight contribution
    uint8_t min_index;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            id_batch[tr] = g_id;
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            printf("%d", buffer_size);

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            // 24 bit depth
            uint32_t cur_depth = (uint32_t)(pos.z/(pos.z+1.0f) * 16777215.0f);

            // add buffer
            if (buffer_size == 0) {
                uint8_t weight = (uint8_t)(255.0f*alpha+0.5f);
                if (weight > 0) {
                    sorted_indices[0] = id_batch[t];
                    sorted_buffer[0] = (cur_depth << 8) + (uint32_t)weight;
                    min_index = 0;
                    buffer_size = 1;
                }
                continue;
            }

            // find insertion index
            int ins_index = buffer_size;
            while (--ins_index >= 0) {
                uint32_t depth_i = sorted_buffer[ins_index] >> 8;
                if (depth_i > cur_depth)
                    break;
            }
            ins_index++;

            // calculate weight
            uint8_t cur_weight = ins_index == 0 ? (uint8_t)255 :
                (uint8_t)(alpha * (uint8_t)sorted_buffer[ins_index-1] + 0.5f);
            uint8_t min_weight = (uint8_t)sorted_buffer[min_index];
            if (cur_weight == 0 || (
                buffer_size >= MAX_SORTED_SPLATS && cur_weight <= min_weight))
                continue;

            // insert vs replace
            float mult = 1.0f - alpha;
            uint8_t new_min_weight = ins_index <= min_index ?
                (uint8_t)(min_weight * mult + 0.5f) : min_weight;
            bool replace_before = min_index < ins_index && (
                new_min_weight == 0 || (buffer_size >= MAX_SORTED_SPLATS && new_min_weight < cur_weight));
            min_weight = min(new_min_weight, cur_weight);
            uint8_t new_min_index = min_index;

            // replace an element before the insert index
            if (replace_before) {
                // update min index for before
                min_weight = (uint8_t)(-1);
                for (int i = 0; i < min_index; i++) {
                    uint8_t weight = (uint8_t)sorted_buffer[i];
                    if (weight <= min_weight)
                        min_weight = weight, new_min_index = i;
                }
                // shift elements
                for (int i = min_index; i < ins_index; i++) {
                    sorted_indices[i] = sorted_indices[i+1];
                    uint32_t info = sorted_buffer[i+1];
                    if ((uint8_t)info <= min_weight)
                        min_weight = (uint8_t)info, new_min_index = i;
                    sorted_buffer[i] = info;
                    // not updating weight here; guess it shouldn't matter much?
                }
                // insert
                sorted_indices[ins_index] = id_batch[t];
                sorted_buffer[ins_index] = (cur_depth << 8) + (uint32_t)cur_weight;
                if (cur_weight < min_weight)
                    min_weight = cur_weight, new_min_index = ins_index;
                // update weights for after, squeeze zero weights
                int offset = 1;
                for (int i = ins_index+1; i+offset <= buffer_size; i++) {
                    uint32_t info = sorted_buffer[i+offset];
                    uint8_t new_weight = (uint8_t)(mult * (uint8_t)info + 0.5f);
                    if (new_weight == 0) {
                        offset++, i--;
                        continue;
                    }
                    if (new_weight <= min_weight)
                        min_weight = new_weight, new_min_index = i;
                    sorted_indices[i] = sorted_indices[i+offset];
                    sorted_buffer[i] = ((info >> 8) << 8) | (uint32_t)new_weight;
                }
                buffer_size -= offset-1;
                min_index = new_min_index;
                continue;
            }

            // replace an element after the insert index
            bool replace_after = min_index >= ins_index && (
                min_weight == 0 || (buffer_size >= MAX_SORTED_SPLATS && min_weight < cur_weight));
            if (replace_after) {
                // update min index for before
                min_weight = (uint8_t)(-1);
                for (int i = 0; i < ins_index; i++) {
                    uint8_t weight = (uint8_t)sorted_buffer[i];
                    if (weight <= min_weight)
                        min_weight = weight, new_min_index = i;
                }
                // shift elements
                for (int i = min_index; i > ins_index; i--) {
                    sorted_indices[i] = sorted_indices[i-1];
                    sorted_buffer[i] = sorted_buffer[i-1];
                }
                // insert
                sorted_indices[ins_index] = id_batch[t];
                sorted_buffer[ins_index] = (cur_depth << 8) + (uint32_t)cur_weight;
                if (cur_weight < min_weight)
                    min_weight = cur_weight, new_min_index = ins_index;
                // update weights for after, squeeze zero weights
                int offset = 0;
                for (int i = ins_index+1; i+offset < buffer_size; i++) {
                    uint32_t info = sorted_buffer[i+offset];
                    uint8_t new_weight = (uint8_t)(mult * (uint8_t)info + 0.5f);
                    if (new_weight == 0) {
                        offset++, i--;
                        continue;
                    }
                    if (new_weight <= min_weight)
                        min_weight = new_weight, new_min_index = i;
                    if (offset > 0)
                        sorted_indices[i] = sorted_indices[i+offset];
                    sorted_buffer[i] = ((info >> 8) << 8) | (uint32_t)new_weight;
                }
                buffer_size -= offset;
                min_index = new_min_index;
                continue;
            }

            // insert an element
            {
                // shift elements
                bool has_zero = false;
                for (int i = buffer_size; i > ins_index; i++) {
                    sorted_indices[i] = sorted_indices[i-1];
                    uint32_t info = sorted_buffer[i-1];
                    uint8_t new_weight = (uint8_t)(mult * (uint8_t)info + 0.5f);
                    if (new_weight == 0) {
                        has_zero = true;
                        continue;
                    }
                    if (new_weight <= min_weight)
                        min_weight = new_weight, new_min_index = i;
                    sorted_buffer[i] = ((info >> 8) << 8) | (uint32_t)new_weight;
                }
                buffer_size += 1;
                // insert
                sorted_indices[ins_index] = id_batch[t];
                sorted_buffer[ins_index] = (cur_depth << 8) + (uint32_t)cur_weight;
                if (cur_weight < min_weight)
                    min_weight = cur_weight, new_min_index = ins_index;
                // squeeze zero weights
                if (has_zero) {
                    int offset = 0;
                    for (int i = ins_index+1; i+offset < buffer_size; i++) {
                        uint32_t info = sorted_buffer[i+offset];
                        if ((uint8_t)info == 0) {
                            offset++, i--;
                            continue;
                        }
                        if ((uint8_t)info <= min_weight)
                            min_weight = (uint8_t)info, new_min_index = i;
                        if (offset > 0) {
                            sorted_indices[i] = sorted_indices[i+offset];
                            sorted_buffer[i] = info;
                        }
                    }
                    buffer_size -= offset;
                }
                min_index = new_min_index;
                continue;
            }

        }
    }

    if (inside) {
        int* out = &out_indices[pix_id*MAX_SORTED_SPLATS];
        for (int i = 0; i < buffer_size; i++)
            out[i] = sorted_indices[i];
        // assume the rest are filled with -1
    }
}
#endif


__global__ void rasterize_simple_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    int* __restrict__ final_index,
    float3* __restrict__ out_img,
    float* __restrict__ out_alpha
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float3 pix_out = {0.f, 0.f, 0.f};
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            glm::vec3 color = color_batch[t];

            const float vis = alpha * T;
            pix_out.x = pix_out.x + color.x * vis;
            pix_out.y = pix_out.y + color.y * vis;
            pix_out.z = pix_out.z + color.z * vis;
            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        float3 final_color;
        final_color.x = pix_out.x + T * background.x;
        final_color.y = pix_out.y + T * background.y;
        final_color.z = pix_out.z + T * background.z;
        out_img[pix_id] = final_color;
        out_alpha[pix_id] = 1.0f - T;
    }
}


__global__ void rasterize_simple_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    const int* __restrict__ final_index,
    const float* __restrict__ output_alpha,
    const float3* __restrict__ v_output,
    const float* __restrict__ v_output_alpha,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float* __restrict__ v_opacities,
    float2* __restrict__ v_anisotropies
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // the contribution from gaussians behind the current one
    float3 buffer = {0.f, 0.f, 0.f};
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    const float3 v_out = v_output[pix_id];
    const float v_out_alpha = v_output_alpha[pix_id];

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, aniso, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            glm::vec3 v_color_local = {0.f, 0.f, 0.f};
            float v_opacity_local = 0.f;
            glm::vec2 v_anisotropy_local = {0.f, 0.f};
            //initialize everything to 0, only set if the lane is valid
            if(valid){
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                // update v_rgb for this gaussian
                float v_alpha = 0.f;
                v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};

                const glm::vec3 color = color_batch[t];
                const glm::vec3 opacity = opacity_batch[t];
                // contribution from this pixel
                v_alpha += (color.x * T - buffer.x) * ra * v_out.x;
                v_alpha += (color.y * T - buffer.y) * ra * v_out.y;
                v_alpha += (color.z * T - buffer.z) * ra * v_out.z;

                v_alpha += T_final * ra * v_out_alpha;
                // contribution from background pixel
                v_alpha += -T_final * ra * background.x * v_out.x;
                v_alpha += -T_final * ra * background.y * v_out.y;
                v_alpha += -T_final * ra * background.z * v_out.z;
                // update the running sum
                buffer.x += color.x * vis;
                buffer.y += color.y * vis;
                buffer.z += color.z * vis;

                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha,
                    v_uv, v_opacity_local, v_anisotropy_local
                );
                glm::mat2x3 v_axis_uv;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    glm::vec3(0), v_uv,
                    v_position_local, v_axis_uv
                );
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                // v_position_xy_abs_local /= pos.z;
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];

                T = next_T;
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum3(v_color_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_anisotropy_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                float* v_color_ptr = (float*)(v_colors);
                atomicAdd(v_color_ptr + 3*g + 0, v_color_local.x);
                atomicAdd(v_color_ptr + 3*g + 1, v_color_local.y);
                atomicAdd(v_color_ptr + 3*g + 2, v_color_local.z);
                
                atomicAdd(v_opacities + g, v_opacity_local);
                float* v_anisotropy_ptr = (float*)(v_anisotropies);
                atomicAdd(v_anisotropy_ptr + 2*g + 0, v_anisotropy_local.x);
                atomicAdd(v_anisotropy_ptr + 2*g + 1, v_anisotropy_local.y);
            }
        }
    }
}





__global__ void rasterize_depth_forward_kernel(
    const int depth_mode,
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    int* __restrict__ final_index,
    float* __restrict__ out_depth,
    float2* __restrict__ out_visibility
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    float interp = 1.0f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float output_depth = 0.0f;
    float output_visibility = 0.0f;
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            // mean depth
            if (depth_mode == DEPTH_MODE_MEAN) {

                // const float depth = pos.z;
                const float depth = poi.z;
                float vis = alpha * T;
                output_depth += vis * depth;

            }  // depth_mode

            // median depth
            else if (depth_mode == DEPTH_MODE_MEDIAN) {

                const float next_depth = poi.z;
                if (next_T < DEPTH_REG_MEDIAN_TH) {
                    if (T < 0.99999f) {
                        // https://www.desmos.com/3d/fttajoozww
                        interp = (1.0f-alpha)/alpha * (T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH;
                        interp = glm::clamp(interp, 0.0f, 1.0f);
                        output_depth = output_depth + (next_depth-output_depth)*interp;
                    }
                    else {
                        output_depth = next_depth;
                    }
                    T = next_T;
                    cur_idx = batch_start + t;
                    done = true;
                    break;
                }
                output_depth = next_depth;

            }  // depth_mode

            T = next_T;
            cur_idx = batch_start + t;
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        if (depth_mode == DEPTH_MODE_MEAN) {
            out_depth[pix_id] = T == 1.0f ? output_depth : output_depth / (1.0f-T);
            // out_depth[pix_id] = output_depth;
            out_visibility[pix_id] = {T, 1.0f-T};
        }
        else if (depth_mode == DEPTH_MODE_MEDIAN) {
            out_depth[pix_id] = output_depth;
            out_visibility[pix_id] = {T, interp};
        }
    }
}


__global__ void rasterize_depth_backward_kernel(
    const int depth_mode,
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const int* __restrict__ final_index,
    const float* __restrict__ out_depth,
    const float2* __restrict__ out_visibility,
    const float* __restrict__ v_out_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float* __restrict__ v_opacities,
    float2* __restrict__ v_anisotropies
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // this is the T AFTER the last gaussian in this pixel
    float2 meta_out = out_visibility[pix_id];
    float T_final = meta_out.x;
    float T = T_final;
    float v_T = 0.0f;
    const float interp = meta_out.y;
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    float output_depth = out_depth[pix_id];
    float v_output_depth = v_out_depth[pix_id];
    float v_out_alpha = 0.0f;
    float v_depth = 0.f;
    float v_depth_next = 0.f;
    float v_alpha = 0.f;
    float v_interp = 0.f;
    if (depth_mode == DEPTH_MODE_MEAN) {
        if (T != 1.0f) {
            float alpha = 1.0f-T;
            v_out_alpha = -output_depth / fmax(alpha, 1e-4f) * v_output_depth;
            output_depth *= alpha;
            v_output_depth /= fmax(alpha, 1e-4);
        }
    }

    float depth_buffer = 0.0f;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, aniso, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            float v_opacity_local = 0.f;
            glm::vec2 v_anisotropy_local = {0.f, 0.f};
            //initialize everything to 0, only set if the lane is valid
            if(valid) {
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                float depth = poi.z;

                // mean depth
                if (depth_mode == DEPTH_MODE_MEAN) {

                    v_depth = vis * v_output_depth;
                    v_alpha = (depth * T - depth_buffer) * ra * v_output_depth;
                    v_alpha += T_final * ra * v_out_alpha;
                    depth_buffer += depth * vis;

                }  // depth_mode

                // median depth
                else if (depth_mode == DEPTH_MODE_MEDIAN) {

                    // depth gradient
                    if (T == T_final) {
                        v_depth = v_output_depth * interp;
                        v_depth_next = v_output_depth * (1.0f-interp);
                    }
                    else {
                        v_depth = v_depth_next;
                        v_depth_next = 0.0f;
                    }

                    // alpha gradient
                    if (T == T_final && interp < 1.0f && interp > 0.0f) {
                        float depth_0 = (output_depth-depth*interp) / (1.0f-interp);
                        v_interp = (depth-depth_0) * v_output_depth;
                        v_alpha = (next_T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH * \
                            v_interp / safe_denom(-alpha*alpha, 1e-3);
                        v_T = (1.0f-alpha)/alpha * v_interp / DEPTH_REG_MEDIAN_TH;
                    }
                    else {
                        v_alpha = v_T * (-next_T);
                        v_T = v_T * (1.0f-alpha);
                    }

                }  // depth_mode

                T = next_T;

                // backward
                const glm::vec3 opacity = opacity_batch[t];
                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha,
                    v_uv, v_opacity_local, v_anisotropy_local
                );
                glm::mat2x3 v_axis_uv;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    {0.f, 0.f, v_depth}, v_uv,
                    v_position_local, v_axis_uv
                );
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_anisotropy_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                atomicAdd(v_opacities + g, v_opacity_local);
                float* v_anisotropy_ptr = (float*)(v_anisotropies);
                atomicAdd(v_anisotropy_ptr + 2*g + 0, v_anisotropy_local.x);
                atomicAdd(v_anisotropy_ptr + 2*g + 1, v_anisotropy_local.y);
            }
        }
    }
}






__global__ void rasterize_forward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const float depth_reg_pairwise_factor,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float3* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    const float2* __restrict__ depth_grads,
    const float3* __restrict__ depth_ref_im,
    int* __restrict__ final_index,
    float* __restrict__ out_alpha,
    float3* __restrict__ out_img,
    float4* __restrict__ out_depth_grad,
    float* __restrict__ out_reg_depth,
    float* __restrict__ out_reg_normal
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];
    // __shared__ glm::vec2 depth_grad_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float T = 1.f;  // current/total visibility
    float2 g_sum = {0.f, 0.f};  // sum of "normals"
    float3 pix_out = {0.f, 0.f, 0.f};  // output radiance
    float vis_sum = 0.f;  // output alpha
    float depth_sum = 0.f;  // output depth
    float depth_squared_sum = 0.f;  // for L2 depth regularizer
    const float3 depth_ref_raw = inside ?
        depth_ref_im[pix_id] : make_float3(0.f, 0.f, 0.f);
    const float2 depth_normal_ref = {depth_ref_raw.x, depth_ref_raw.y};
    const float depth_ref = depth_ref_raw.z;
    float reg_depth_p = 0.f, reg_depth_i = 0.f;  // output depth regularizer
    float reg_normal = 0.f;  // output normal regularizer
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }
        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            // const float2 depth_grad = depth_grads[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
            // depth_grad_batch[tr] = {depth_grad.x, depth_grad.y};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            glm::vec3 color_0 = color_batch[t];
            glm::vec3 color;
            if (dim_ch > 0) {
                int32_t g_id = id_batch[t];
                const glm::vec3* coeffs = (glm::vec3*)&ch_coeffs[dim_ch*g_id];
                glm::vec3 ch_color = ch_coeffs_to_color(
                    ch_degree_r, ch_degree_r_to_use,
                    ch_degree_phi, ch_degree_phi_to_use,
                    coeffs, {uv.x, uv.y}
                );
                color = color_0 / (1.0f+glm::exp(-ch_color));
            }
            else color = color_0;

            const float vis = alpha * T;
            #if DEPTH_REG_L == 01 && false
            const float depth = pos.z;
            #else
            const float depth = poi.z;
            #endif
            const glm::vec2 g_i = *(glm::vec2*)&depth_grads[id_batch[t]];
            const float g_i_norm = glm::length(g_i) + 1e-6f;
            const glm::vec2 n_i = g_i / g_i_norm;

            pix_out.x = pix_out.x + color.x * vis;
            pix_out.y = pix_out.y + color.y * vis;
            pix_out.z = pix_out.z + color.z * vis;
            {  // depth regularization
                float pairwise_l1 = vis*depth * vis_sum - vis * depth_sum;  // requires pos.z for depth
                float pairwise_l2 = vis * (vis_sum*depth*depth + depth_squared_sum - 2.0f*depth*depth_sum);
                float intersect_l1 = vis * abs(depth - depth_ref);
                float intersect_l2 = vis * (depth-depth_ref) * (depth-depth_ref);
                reg_depth_p += pairwise_l2;
                reg_depth_i += intersect_l1;
            }
            reg_normal += vis * (1.0f - (n_i.x*depth_normal_ref.x+n_i.y*depth_normal_ref.y));
            vis_sum += vis;
            depth_sum += vis*depth;
            depth_squared_sum += vis*depth*depth;
            g_sum.x = g_sum.x + vis * g_i.x;
            g_sum.y = g_sum.y + vis * g_i.y;

            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        out_alpha[pix_id] = 1.0f - T;
        float3 final_color;
        final_color.x = pix_out.x + T * background.x;
        final_color.y = pix_out.y + T * background.y;
        final_color.z = pix_out.z + T * background.z;
        out_img[pix_id] = final_color;
        out_depth_grad[pix_id] = {g_sum.x, g_sum.y, depth_sum, depth_squared_sum};
        out_reg_normal[pix_id] = reg_normal;
        out_reg_depth[pix_id] = reg_depth_i + (reg_depth_p-reg_depth_i) * depth_reg_pairwise_factor;
    }
}


__global__ void rasterize_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float depth_reg_pairwise_factor,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float3* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    const float2* __restrict__ depth_grads,
    const float3* __restrict__ depth_ref_im,
    const int* __restrict__ final_index,
    const float* __restrict__ output_alpha,
    const float4* __restrict__ output_depth_grad,
    const float* __restrict__ v_output_alpha,
    const float3* __restrict__ v_output,
    const float4* __restrict__ v_output_depth_grad,
    const float* __restrict__ v_output_reg_depth,
    const float* __restrict__ v_output_reg_normal,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float3* __restrict__ v_ch_coeffs,
    float* __restrict__ v_ch_coeffs_abs,
    float* __restrict__ v_opacities,
    float2* __restrict__ v_anisotropies,
    float2* __restrict__ v_depth_grad,
    float3* __restrict__ v_depth_ref_im
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);
    assert(dim_ch <= MAX_CH_FLOAT3);

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];
    // __shared__ glm::vec2 depth_grad_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    const float4 out_depth_grad = output_depth_grad[pix_id];
    const float3 v_out = v_output[pix_id];
    const float4 v_out_depth_grad = v_output_depth_grad[pix_id];
    const float v_out_alpha = v_output_alpha[pix_id];
    const float v_out_reg_depth = v_output_reg_depth[pix_id];
    const float v_reg_depth_p = v_out_reg_depth * depth_reg_pairwise_factor;
    const float v_reg_depth_i = v_out_reg_depth * (1.0f-depth_reg_pairwise_factor);
    const float v_out_reg_normal = v_output_reg_normal[pix_id];
    const glm::vec2 v_g_sum = {v_out_depth_grad.x, v_out_depth_grad.y};
    const float v_depth_sum = v_out_depth_grad.z;
    const float v_depth_squared_sum = v_out_depth_grad.w;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();

    // regularization
    const float3 depth_ref_raw = inside ?
        depth_ref_im[pix_id] : make_float3(0.f, 0.f, 0.f);
    const float2 depth_normal_ref = {depth_ref_raw.x, depth_ref_raw.y};
    const float depth_ref = depth_ref_raw.z;
    glm::vec2 n_bar = {depth_normal_ref.x, depth_normal_ref.y};
    glm::vec2 v_n_bar = {0.f, 0.f};
    float v_depth_ref = 0.f;

    const float vis_sum_final = 1.0f - T_final;
    const float depth_sum_final = out_depth_grad.z;
    const float depth_squared_sum_final = out_depth_grad.w;
    float vis_sum = vis_sum_final;
    float depth_sum = depth_sum_final;
    float depth_squared_sum = depth_squared_sum_final;
    glm::vec2 g_sum = {out_depth_grad.x, out_depth_grad.y};

    float3 buffer = {0.f, 0.f, 0.f};
    float4 buffer_depth = {0.f, 0.f, 0.f, 0.f};
    float buffer_depth_reg = 0.f;
    float buffer_normal_reg = 0.f;
    
    float v_sum_vis = v_out_alpha;

    // second run through, full gradient calculation
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    T = T_final;
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            // const float2 depth_grad = depth_grads[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
            // depth_grad_batch[tr] = {depth_grad.x, depth_grad.y};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, aniso, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            glm::vec3 v_color_local = {0.f, 0.f, 0.f};
            glm::vec2 v_depth_grad_local = {0.f, 0.f};
            float v_opacity_local = 0.f;
            glm::vec2 v_anisotropy_local = {0.f, 0.f};
            glm::vec3 v_ch_coeff_local[MAX_CH_FLOAT3];
            for (int i = 0; i < dim_ch; i++)
                v_ch_coeff_local[i] = {0.f, 0.f, 0.f};
            float v_ch_coeff_abs_local = 0.f;
            //initialize everything to 0, only set if the lane is valid
            if(valid){
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                // update accumulation
                v_depth_grad_local.x += vis * v_g_sum.x;
                v_depth_grad_local.y += vis * v_g_sum.y;
                glm::vec3 v_poi = {0.f, 0.f, 0.f};
                #if DEPTH_REG_L == 01 && false
                const float depth = pos.z;
                v_position_local.z += vis * v_depth_sum;
                v_position_local.z += vis * 2.0f*depth * v_depth_squared_sum;
                #else
                const float depth = poi.z;
                v_poi.z += vis * v_depth_sum;
                v_poi.z += vis * 2.0f*depth * v_depth_squared_sum;
                #endif

                // update depth regularizer
                const glm::vec2 depth_grad = *(glm::vec2*)&depth_grads[id_batch[t]];
                float vis_sum_next = vis_sum - vis;
                float depth_sum_next = depth_sum - vis*depth;
                float depth_squared_sum_next = depth_squared_sum - vis*depth*depth;
                #if 0  // pairwise L1, requires pos.z for depth
                v_position_local.z += v_reg_depth_p * vis * (vis_sum_next - (vis_sum_final-vis_sum));
                float reg_depth_i_p = (
                    depth * vis_sum_next - depth_sum_next +
                    (depth_sum_final-depth_sum) - depth * (vis_sum_final-vis_sum)
                );
                #else  // pairwise L2
                v_poi.z += v_reg_depth_p * vis * 2.0f * (
                    vis_sum_final * depth - depth_sum_final);
                float reg_depth_i_p =
                    vis_sum_final*depth*depth + depth_squared_sum_final
                    - 2.0f*depth*depth_sum_final;
                #endif
                #if 1  // L1 with intersected depth
                float v_z = v_reg_depth_i * vis * glm::sign(depth-depth_ref);
                v_poi.z += v_z;
                v_depth_ref += (-v_z);
                float reg_depth_i_i = abs(depth-depth_ref);
                #else  // L2 with intersected depth
                float v_z = v_reg_depth_i * vis * 2.0f*(depth-depth_ref);
                v_poi.z += v_z;
                v_depth_ref += (-v_z);
                float reg_depth_i_i = (depth-depth_ref) * (depth-depth_ref);
                #endif
                float reg_depth_i = reg_depth_i_i + (reg_depth_i_p-reg_depth_i_i) * depth_reg_pairwise_factor;

                // update normal regularizer
                glm::vec2 g_i = {depth_grad.x, depth_grad.y};
                float g_i_norm = glm::length(g_i) + 1e-6f;
                glm::vec2 n_i = g_i / g_i_norm;
                glm::mat2 J_i = (glm::mat2(1.0f) - glm::outerProduct(n_i, n_i)) / g_i_norm;
                float reg_normal_i = 1.0f - dot(n_i, n_bar);
                glm::vec2 v_normal_glm = v_out_reg_normal * (-vis) * J_i * n_bar;
                v_depth_grad_local.x += v_normal_glm.x;
                v_depth_grad_local.y += v_normal_glm.y;
                v_n_bar += vis * (-n_i) * v_out_reg_normal;

                // update color
                glm::vec3 v_color_1 = {vis * v_out.x, vis * v_out.y, vis * v_out.z};
                const glm::vec3 opacity = opacity_batch[t];
                const glm::vec3 color_0 = color_batch[t];
                glm::vec3 color_1;
                glm::vec2 v_uv_ch = {0.f, 0.f};
                if (dim_ch > 0) {
                    glm::vec3 v_ch_color_sigmoid = v_color_1 * color_0;
                    #if 0
                    int32_t g_id = id_batch[t];
                    glm::vec3 ch_color = ch_coeffs_to_color(
                        ch_degree_r, ch_degree_r_to_use,
                        ch_degree_phi, ch_degree_phi_to_use,
                        (glm::vec3*)&ch_coeffs[dim_ch*g_id], {uv.x, uv.y}
                    );
                    glm::vec3 ch_color_sigmoid = 1.0f / (1.0f+glm::exp(-ch_color));
                    glm::vec3 v_ch_color = v_ch_color_sigmoid * ch_color_sigmoid*(1.0f-ch_color_sigmoid);
                    ch_coeffs_to_color_vjp(
                        ch_degree_r, ch_degree_r_to_use,
                        ch_degree_phi, ch_degree_phi_to_use,
                        (glm::vec3*)&ch_coeffs[dim_ch*g_id],
                        {uv.x, uv.y},
                        v_ch_color,
                        ch_color,
                        v_ch_coeff_local, v_ch_coeff_abs_local,
                        v_uv_ch
                    );
                    #else
                    // makes overall training 0.1x faster
                    int32_t g_id = id_batch[t];
                    glm::vec3 ch_color_sigmoid;
                    ch_coeffs_to_color_sigmoid_vjp(
                        ch_degree_r, ch_degree_r_to_use,
                        ch_degree_phi, ch_degree_phi_to_use,
                        (glm::vec3*)&ch_coeffs[dim_ch*g_id],
                        {uv.x, uv.y},
                        v_ch_color_sigmoid,
                        ch_color_sigmoid,
                        v_ch_coeff_local, v_ch_coeff_abs_local,
                        v_uv_ch
                    );
                    #endif
                    color_1 = color_0 * ch_color_sigmoid;
                    v_color_local = v_color_1 * ch_color_sigmoid;
                }
                else {
                    color_1 = color_0;
                    v_color_local = v_color_1;
                }

                float v_alpha = 0.0f;
                // contribution from this pixel
                v_alpha += (color_1.x * T - buffer.x) * ra * v_out.x;
                v_alpha += (color_1.y * T - buffer.y) * ra * v_out.y;
                v_alpha += (color_1.z * T - buffer.z) * ra * v_out.z;
                v_alpha += T_final * ra * v_out_alpha;
                v_alpha += -T_final * ra * background.x * v_out.x;
                v_alpha += -T_final * ra * background.y * v_out.y;
                v_alpha += -T_final * ra * background.z * v_out.z;
                float v_alpha_color_only = v_alpha;
                v_alpha += (depth_grad.x * T - buffer_depth.x) * ra * v_g_sum.x;
                v_alpha += (depth_grad.y * T - buffer_depth.y) * ra * v_g_sum.y;
                v_alpha += (depth * T - buffer_depth.z) * ra * v_depth_sum;
                v_alpha += (depth*depth * T - buffer_depth.w) * ra * v_depth_squared_sum;
                v_alpha += (reg_depth_i * T - buffer_depth_reg) * ra * v_out_reg_depth;
                v_alpha += (reg_normal_i * T - buffer_normal_reg) * ra * v_out_reg_normal;

                // update the running sum
                buffer.x += color_1.x * vis;
                buffer.y += color_1.y * vis;
                buffer.z += color_1.z * vis;
                buffer_depth.x += depth_grad.x * vis;
                buffer_depth.y += depth_grad.y * vis;
                buffer_depth.z += depth * vis;
                buffer_depth.w += depth*depth * vis;
                buffer_depth_reg += reg_depth_i * vis;
                buffer_normal_reg += reg_normal_i * vis;

                // grad
                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha,
                    v_uv, v_opacity_local, v_anisotropy_local
                );
                v_uv += v_uv_ch;
                glm::mat2x3 v_axis_uv;
                glm::vec3 v_position_local_temp;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    v_poi, v_uv,
                    v_position_local_temp, v_axis_uv
                );
                v_position_local += v_position_local_temp;
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];

                // absgrad (color only)
                #if 0
                float v_opacity_local_1;
                glm::vec2 v_anisotropy_local_1;
                get_alpha_vjp(
                    uv, opacity.z, glm::vec2(opacity),
                    v_alpha_color_only,
                    v_uv, v_opacity_local_1, v_anisotropy_local_1
                );
                v_uv += v_uv_ch;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    glm::vec3(0), v_uv,
                    v_position_local_temp, v_axis_uv
                );
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local_temp));
                #endif

                // next loop
                T = next_T;
                vis_sum = vis_sum_next;
                depth_sum = depth_sum_next;
                depth_squared_sum = depth_squared_sum_next;
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum3(v_color_local, warp);
            for (int i = 0; i < dim_ch; i++)
                warpSum3(v_ch_coeff_local[i], warp);
            warpSum(v_ch_coeff_abs_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_anisotropy_local, warp);
            warpSum2(v_depth_grad_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                float* v_color_ptr = (float*)(v_colors);
                atomicAdd(v_color_ptr + 3*g + 0, v_color_local.x);
                atomicAdd(v_color_ptr + 3*g + 1, v_color_local.y);
                atomicAdd(v_color_ptr + 3*g + 2, v_color_local.z);
                float* v_ch_coeffs_ptr = (float*)(v_ch_coeffs);
                for (int i = 0; i < dim_ch; i++) {
                    atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g + 3*i + 0, v_ch_coeff_local[i].x);
                    atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g + 3*i + 1, v_ch_coeff_local[i].y);
                    atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g + 3*i + 2, v_ch_coeff_local[i].z);
                }
                atomicAdd(v_ch_coeffs_abs + g, v_ch_coeff_abs_local);

                atomicAdd(v_opacities + g, v_opacity_local);
                float* v_anisotropy_ptr = (float*)(v_anisotropies);
                atomicAdd(v_anisotropy_ptr + 2*g + 0, v_anisotropy_local.x);
                atomicAdd(v_anisotropy_ptr + 2*g + 1, v_anisotropy_local.y);

                float* v_depth_grad_ptr = (float*)(v_depth_grad);
                atomicAdd(v_depth_grad_ptr + 2*g + 0, v_depth_grad_local.x);
                atomicAdd(v_depth_grad_ptr + 2*g + 1, v_depth_grad_local.y);
            }
        }
    }

    if (inside) {
        v_depth_ref_im[pix_id] = {v_n_bar.x, v_n_bar.y, v_depth_ref};
    }

}


