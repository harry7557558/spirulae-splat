#include "hip/hip_runtime.h"
#include "ProjectionEWA3DGSHetero.cuh"

#include "helpers.cuh"
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include <gsplat/Common.h>
#include <gsplat/Utils.cuh>


template<typename T>
__device__ __forceinline__ unsigned upper_bound(
    const T *arr, unsigned n, T value
) {
    unsigned left = 0, right = n;
    while (left < right) {
        int mid = left + (right - left) / 2;
        if (arr[mid] <= value)
            left = mid + 1;
        else
            right = mid;
    }
    return left;
}

__global__ void projection_ewa_3dgs_hetero_forward_kernel(
    const uint32_t C,
    const uint32_t nnz,
    const float *__restrict__ means,    // [N, 3]
    const float *__restrict__ quats,    // [N, 4]
    const float *__restrict__ scales,   // [N, 3]
    const float *__restrict__ opacities, // [N]
    const float *__restrict__ viewmats, // [C, 4, 4]
    const float *__restrict__ Ks,       // [C, 3, 3]
    const uint32_t image_width,  // TILE_SIZE
    const uint32_t image_height,  // TILE_SIZE
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    const gsplat::CameraModelType camera_model,
    const int32_t* __restrict__ intersection_count_map,  // [C+1]
    const int32_t* __restrict__ intersection_splat_id,  // [nnz]
    // outputs
    int64_t *__restrict__ camera_ids,    // [nnz]
    int64_t *__restrict__ gaussian_ids,  // [nnz]
    int32_t *__restrict__ radii,         // [nnz, 2]
    float *__restrict__ means2d,      // [nnz, 2]
    float *__restrict__ depths,       // [nnz]
    float *__restrict__ conics,       // [nnz, 3]
    float *__restrict__ compensations // [nnz] optional
) {
    int32_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t camera_idx = upper_bound(intersection_count_map, C+1, thread_idx) - 1;
    int32_t gauss_idx = intersection_splat_id[thread_idx];

    bool valid = (thread_idx < nnz);

    // check if points are with camera near and far plane
    glm::vec3 mean_c;
    glm::mat3 R;
    if (valid) {
        // shift pointers to the current camera and gaussian
        means += gauss_idx * 3;
        viewmats += camera_idx * 16;

        // glm is column-major but input is row-major
        R = glm::mat3(
            viewmats[0],
            viewmats[4],
            viewmats[8], // 1st column
            viewmats[1],
            viewmats[5],
            viewmats[9], // 2nd column
            viewmats[2],
            viewmats[6],
            viewmats[10] // 3rd column
        );
        glm::vec3 t = { viewmats[3], viewmats[7], viewmats[11] };

        // transform Gaussian center to camera space
        gsplat::posW2C(R, t, glm::make_vec3(means), mean_c);
        if (mean_c.z < near_plane || mean_c.z > far_plane) {
            valid = false;
        }
    }

    // check if the perspective projection is valid.
    glm::mat2 covar2d;
    glm::vec2 mean2d = glm::vec2(0);
    glm::mat2 covar2d_inv = glm::mat2(0);
    float compensation = 0.0f;
    float det;
    if (valid) {
        // transform Gaussian covariance to camera space
        glm::mat3 covar;
        
        quats += gauss_idx * 4;
        scales += gauss_idx * 3;
        gsplat::quat_scale_to_covar_preci(
            glm::make_vec4(quats), glm::make_vec3(scales), &covar, nullptr
        );

        glm::mat3 covar_c;
        gsplat::covarW2C(R, covar, covar_c);

        Ks += camera_idx * 9;
        switch (camera_model) {
        case gsplat::CameraModelType::PINHOLE: // perspective projection
            gsplat::persp_proj(
                mean_c,
                covar_c,
                Ks[0],
                Ks[4],
                Ks[2],
                Ks[5],
                covar2d,
                mean2d
            );
            break;
        case gsplat::CameraModelType::ORTHO: // orthographic projection
            gsplat::ortho_proj(
                mean_c,
                covar_c,
                Ks[0],
                Ks[4],
                Ks[2],
                Ks[5],
                covar2d,
                mean2d
            );
            break;
        case gsplat::CameraModelType::FISHEYE: // fisheye projection
            gsplat::fisheye_proj(
                mean_c,
                covar_c,
                Ks[0],
                Ks[4],
                Ks[2],
                Ks[5],
                covar2d,
                mean2d
            );
            break;
        }

        det = gsplat::add_blur(eps2d, covar2d, compensation);
        if (det <= 0.f) {
            valid = false;
        } else {
            // compute the inverse of the 2d covariance
            covar2d_inv = glm::inverse(covar2d);
        }
    }

    // check if the points are in the image region
    float radius_x, radius_y;
    if (valid) {
        float extend = 3.33f;
        if (opacities != nullptr) {
            float opacity = opacities[gauss_idx];
            if (compensations != nullptr) {
                // we assume compensation term will be applied later on.
                opacity *= compensation;
            }
            if (opacity < ALPHA_THRESHOLD) {
                valid = false;
            }
            // Compute opacity-aware bounding box.
            // https://arxiv.org/pdf/2402.00525 Section B.2
            extend = min(extend, sqrt(2.0f * __logf(opacity / ALPHA_THRESHOLD)));
        }
        
        // compute tight rectangular bounding box (non differentiable)
        // https://arxiv.org/pdf/2402.00525
        radius_x = ceilf(extend * sqrtf(covar2d[0][0]));
        radius_y = ceilf(extend * sqrtf(covar2d[1][1]));
        
        if (radius_x <= radius_clip && radius_y <= radius_clip) {
            valid = false;
        }

        // mask out gaussians outside the image region
        if (mean2d.x + radius_x <= 0 || mean2d.x - radius_x >= image_width ||
            mean2d.y + radius_y <= 0 || mean2d.y - radius_y >= image_height) {
            valid = false;
        }
    }

    if (thread_idx < nnz) {
        // write to outputs
        camera_ids[thread_idx] = camera_idx;
        gaussian_ids[thread_idx] = gauss_idx;
        radii[thread_idx * 2] = (int32_t)radius_x * int(valid);
        radii[thread_idx * 2 + 1] = (int32_t)radius_y * int(valid);
        means2d[thread_idx * 2] = mean2d.x;
        means2d[thread_idx * 2 + 1] = mean2d.y;
        depths[thread_idx] = valid ? mean_c.z : -0.0f;
        conics[thread_idx * 3] = covar2d_inv[0][0];
        conics[thread_idx * 3 + 1] = covar2d_inv[0][1];
        conics[thread_idx * 3 + 2] = covar2d_inv[1][1];
        if (compensations != nullptr) {
            compensations[thread_idx] = compensation;
        }
    }
}




std::tuple<
    at::Tensor,  // camera_ids
    at::Tensor,  // gaussian_ids
    at::Tensor,  // radii
    at::Tensor,  // means2d
    at::Tensor,  // depths
    at::Tensor,  // conics
    at::Tensor  // compensations
> projection_ewa_3dgs_hetero_forward_tensor(
    // inputs
    const at::Tensor means,                // [..., N, 3]
    const at::Tensor quats,  // [..., N, 4]
    const at::Tensor scales, // [..., N, 3]
    const at::optional<at::Tensor> opacities, // [..., N]
    const at::Tensor viewmats,             // [..., C, 4, 4]
    const at::Tensor Ks,                   // [..., C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    const bool calc_compensations,
    const gsplat::CameraModelType camera_model,
    const at::Tensor intersection_count_map,  // [C+1]
    const at::Tensor intersection_splat_id  // [nnz]
) {
    uint32_t N = means.size(-2);          // number of gaussians
    uint32_t C = viewmats.size(-3);       // number of cameras
    uint32_t nnz = intersection_splat_id.size(-1);  // number of intersections

    auto opt = means.options();
    at::Tensor camera_ids = at::empty({nnz}, opt.dtype(at::kLong));
    at::Tensor gaussian_ids = at::empty({nnz}, opt.dtype(at::kLong));
    at::Tensor radii = at::empty({nnz, 2}, opt.dtype(at::kInt));
    at::Tensor means2d = at::empty({nnz, 2}, opt);
    at::Tensor depths = at::empty({nnz}, opt);
    at::Tensor conics = at::empty({nnz, 3}, opt);
    at::Tensor compensations;
    if (calc_compensations) {
        compensations = at::empty({nnz}, opt);
    }

    constexpr uint block = 256;
    projection_ewa_3dgs_hetero_forward_kernel<<<_CEIL_DIV(nnz, block), block>>>(
        C,
        nnz,
        means.data_ptr<float>(),
        quats.data_ptr<float>(),
        scales.data_ptr<float>(),
        opacities.has_value() ? opacities.value().data_ptr<float>() : nullptr,
        viewmats.data_ptr<float>(),
        Ks.data_ptr<float>(),
        image_width,
        image_height,
        eps2d,
        near_plane,
        far_plane,
        radius_clip,
        camera_model,
        intersection_count_map.data_ptr<int32_t>(),
        intersection_splat_id.data_ptr<int32_t>(),
        camera_ids.data_ptr<int64_t>(),
        gaussian_ids.data_ptr<int64_t>(),
        radii.data_ptr<int32_t>(),
        means2d.data_ptr<float>(),
        depths.data_ptr<float>(),
        conics.data_ptr<float>(),
        calc_compensations ? compensations.data_ptr<float>() : nullptr
    );

    return std::make_tuple(
        camera_ids,
        gaussian_ids,
        radii,
        means2d,
        depths,
        conics,
        compensations
    );
}
