#include "hip/hip_runtime.h"
#include "ProjectionEWA3DGSHetero.cuh"

#include "helpers.cuh"
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include <gsplat/Common.h>
#include <gsplat/Utils.cuh>


template<typename T>
__device__ __forceinline__ unsigned upper_bound(
    const T *arr, unsigned n, T value
) {
    unsigned left = 0, right = n;
    while (left < right) {
        int mid = left + (right - left) / 2;
        if (arr[mid] <= value)
            left = mid + 1;
        else
            right = mid;
    }
    return left;
}

__global__ void projection_ewa_3dgs_hetero_forward_kernel(
    const uint32_t C,
    const uint32_t nnz,
    const float *__restrict__ means,    // [N, 3]
    const float *__restrict__ quats,    // [N, 4]
    const float *__restrict__ scales,   // [N, 3]
    const float *__restrict__ opacities, // [N]
    const float *__restrict__ viewmats, // [C, 4, 4]
    const float *__restrict__ Ks,       // [C, 3, 3]
    const uint32_t image_width,  // TILE_SIZE
    const uint32_t image_height,  // TILE_SIZE
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    const gsplat::CameraModelType camera_model,
    const int32_t* __restrict__ intersection_count_map,  // [C+1]
    const int32_t* __restrict__ intersection_splat_id,  // [nnz]
    // outputs
    int64_t *__restrict__ camera_ids,    // [nnz]
    int64_t *__restrict__ gaussian_ids,  // [nnz]
    int32_t *__restrict__ radii,         // [nnz, 2]
    float *__restrict__ means2d,      // [nnz, 2]
    float *__restrict__ depths,       // [nnz]
    float *__restrict__ conics,       // [nnz, 3]
    float *__restrict__ compensations // [nnz] optional
) {
    int32_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_idx >= nnz)
        return;
    int32_t camera_idx = upper_bound(intersection_count_map, C+1, thread_idx) - 1;
    int32_t splat_idx = intersection_splat_id[thread_idx];

    bool valid = true;

    // check if points are with camera near and far plane
    glm::vec3 mean_c;
    glm::mat3 R;
    if (valid) {
        // shift pointers to the current camera and gaussian
        means += splat_idx * 3;
        viewmats += camera_idx * 16;

        // glm is column-major but input is row-major
        R = glm::mat3(
            viewmats[0],
            viewmats[4],
            viewmats[8], // 1st column
            viewmats[1],
            viewmats[5],
            viewmats[9], // 2nd column
            viewmats[2],
            viewmats[6],
            viewmats[10] // 3rd column
        );
        glm::vec3 t = { viewmats[3], viewmats[7], viewmats[11] };

        // transform Gaussian center to camera space
        gsplat::posW2C(R, t, glm::make_vec3(means), mean_c);
        if (mean_c.z < near_plane || mean_c.z > far_plane) {
            valid = false;
        }
    }

    // check if the perspective projection is valid.
    glm::mat2 covar2d;
    glm::vec2 mean2d = glm::vec2(0);
    glm::mat2 covar2d_inv = glm::mat2(0);
    float compensation = 0.0f;
    float det;
    if (valid) {
        // transform Gaussian covariance to camera space
        glm::mat3 covar;
        
        quats += splat_idx * 4;
        scales += splat_idx * 3;
        gsplat::quat_scale_to_covar_preci(
            glm::make_vec4(quats), glm::make_vec3(scales), &covar, nullptr
        );

        glm::mat3 covar_c;
        gsplat::covarW2C(R, covar, covar_c);

        Ks += camera_idx * 9;
        switch (camera_model) {
        case gsplat::CameraModelType::PINHOLE: // perspective projection
            gsplat::persp_proj(
                mean_c,
                covar_c,
                Ks[0],
                Ks[4],
                Ks[2],
                Ks[5],
                covar2d,
                mean2d
            );
            break;
        case gsplat::CameraModelType::ORTHO: // orthographic projection
            gsplat::ortho_proj(
                mean_c,
                covar_c,
                Ks[0],
                Ks[4],
                Ks[2],
                Ks[5],
                covar2d,
                mean2d
            );
            break;
        case gsplat::CameraModelType::FISHEYE: // fisheye projection
            gsplat::fisheye_proj(
                mean_c,
                covar_c,
                Ks[0],
                Ks[4],
                Ks[2],
                Ks[5],
                covar2d,
                mean2d
            );
            break;
        }

        det = gsplat::add_blur(eps2d, covar2d, compensation);
        if (det <= 0.f) {
            valid = false;
        } else {
            // compute the inverse of the 2d covariance
            covar2d_inv = glm::inverse(covar2d);
        }
    }

    // check if the points are in the image region
    float radius_x, radius_y;
    if (valid) {
        float extend = 3.33f;
        if (opacities != nullptr) {
            float opacity = opacities[splat_idx];
            if (compensations != nullptr) {
                // we assume compensation term will be applied later on.
                opacity *= compensation;
            }
            if (opacity < ALPHA_THRESHOLD) {
                valid = false;
            }
            // Compute opacity-aware bounding box.
            // https://arxiv.org/pdf/2402.00525 Section B.2
            extend = min(extend, sqrt(2.0f * __logf(opacity / ALPHA_THRESHOLD)));
        }
        
        // compute tight rectangular bounding box (non differentiable)
        // https://arxiv.org/pdf/2402.00525
        radius_x = ceilf(extend * sqrtf(covar2d[0][0]));
        radius_y = ceilf(extend * sqrtf(covar2d[1][1]));
        
        if (radius_x <= radius_clip && radius_y <= radius_clip) {
            valid = false;
        }

        // mask out gaussians outside the image region
        if (mean2d.x + radius_x <= 0 || mean2d.x - radius_x >= image_width ||
            mean2d.y + radius_y <= 0 || mean2d.y - radius_y >= image_height) {
            valid = false;
        }
    }

    {
        // write to outputs
        camera_ids[thread_idx] = camera_idx;
        gaussian_ids[thread_idx] = splat_idx;
        radii[thread_idx * 2] = (int32_t)radius_x * int(valid);
        radii[thread_idx * 2 + 1] = (int32_t)radius_y * int(valid);
        means2d[thread_idx * 2] = mean2d.x;
        means2d[thread_idx * 2 + 1] = mean2d.y;
        depths[thread_idx] = valid ? mean_c.z : -0.0f;
        conics[thread_idx * 3] = covar2d_inv[0][0];
        conics[thread_idx * 3 + 1] = covar2d_inv[0][1];
        conics[thread_idx * 3 + 2] = covar2d_inv[1][1];
        if (compensations != nullptr) {
            compensations[thread_idx] = compensation;
        }
    }
}


std::tuple<
    at::Tensor,  // camera_ids
    at::Tensor,  // gaussian_ids
    at::Tensor,  // radii
    at::Tensor,  // means2d
    at::Tensor,  // depths
    at::Tensor,  // conics
    at::Tensor  // compensations
> projection_ewa_3dgs_hetero_forward_tensor(
    // inputs
    const at::Tensor means,                // [..., N, 3]
    const at::Tensor quats,  // [..., N, 4]
    const at::Tensor scales, // [..., N, 3]
    const std::optional<at::Tensor> opacities, // [..., N]
    const at::Tensor viewmats,             // [..., C, 4, 4]
    const at::Tensor Ks,                   // [..., C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    const bool calc_compensations,
    const gsplat::CameraModelType camera_model,
    const at::Tensor intersection_count_map,  // [C+1]
    const at::Tensor intersection_splat_id  // [nnz]
) {
    uint32_t N = means.size(-2);          // number of gaussians
    uint32_t C = viewmats.size(-3);       // number of cameras
    uint32_t nnz = intersection_splat_id.size(-1);  // number of intersections

    auto opt = means.options();
    at::Tensor camera_ids = at::empty({nnz}, opt.dtype(at::kLong));
    at::Tensor gaussian_ids = at::empty({nnz}, opt.dtype(at::kLong));
    at::Tensor radii = at::empty({nnz, 2}, opt.dtype(at::kInt));
    at::Tensor means2d = at::empty({nnz, 2}, opt);
    at::Tensor depths = at::empty({nnz}, opt);
    at::Tensor conics = at::empty({nnz, 3}, opt);
    at::Tensor compensations;
    if (calc_compensations) {
        compensations = at::empty({nnz}, opt);
    }

    constexpr uint block = 256;
    projection_ewa_3dgs_hetero_forward_kernel<<<_CEIL_DIV(nnz, block), block>>>(
        C,
        nnz,
        means.data_ptr<float>(),
        quats.data_ptr<float>(),
        scales.data_ptr<float>(),
        opacities.has_value() ? opacities.value().data_ptr<float>() : nullptr,
        viewmats.data_ptr<float>(),
        Ks.data_ptr<float>(),
        image_width,
        image_height,
        eps2d,
        near_plane,
        far_plane,
        radius_clip,
        camera_model,
        intersection_count_map.data_ptr<int32_t>(),
        intersection_splat_id.data_ptr<int32_t>(),
        camera_ids.data_ptr<int64_t>(),
        gaussian_ids.data_ptr<int64_t>(),
        radii.data_ptr<int32_t>(),
        means2d.data_ptr<float>(),
        depths.data_ptr<float>(),
        conics.data_ptr<float>(),
        calc_compensations ? compensations.data_ptr<float>() : nullptr
    );

    return std::make_tuple(
        camera_ids,
        gaussian_ids,
        radii,
        means2d,
        depths,
        conics,
        compensations
    );
}




__global__ void projection_ewa_3dgs_hetero_backward_kernel(
    // fwd inputs
    const uint32_t C,
    const uint32_t nnz,
    const float *__restrict__ means,    // [N, 3]
    const float *__restrict__ quats,    // [N, 4]
    const float *__restrict__ scales,   // [N, 3]
    const float *__restrict__ viewmats, // [C, 4, 4]
    const float *__restrict__ Ks,       // [C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const gsplat::CameraModelType camera_model,
    // fwd outputs
    const int64_t *__restrict__ camera_ids,     // [nnz]
    const int64_t *__restrict__ gaussian_ids,   // [nnz]
    const float *__restrict__ conics,        // [nnz, 3]
    const float *__restrict__ compensations, // [nnz] optional
    // grad outputs
    const float *__restrict__ v_means2d,       // [nnz, 2]
    const float *__restrict__ v_depths,        // [nnz]
    const float *__restrict__ v_conics,        // [nnz, 3]
    const float *__restrict__ v_compensations, // [nnz] optional
    const bool sparse_grad, // whether the outputs are in COO format [nnz, ...]
    // grad inputs
    float *__restrict__ v_means,   // [N, 3] or [nnz, 3]
    float *__restrict__ v_quats,   // [N, 4] or [nnz, 4]
    float *__restrict__ v_scales,  // [N, 3] or [nnz, 3]
    float *__restrict__ v_viewmats // [C, 4, 4]
) {
    // parallelize over nnz.
    int32_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_idx >= nnz)
        return;
    int32_t camera_idx = camera_ids[thread_idx];
    int32_t splat_idx = gaussian_ids[thread_idx];

    // shift pointers to the current camera and gaussian
    means += splat_idx * 3;
    viewmats += camera_idx * 16;
    Ks += camera_idx * 9;

    conics += thread_idx * 3;

    v_means2d += thread_idx * 2;
    v_depths += thread_idx;
    v_conics += thread_idx * 3;

    // vjp: compute the inverse of the 2d covariance
    glm::mat2 covar2d_inv(conics[0], conics[1], conics[1], conics[2]);
    glm::mat2 v_covar2d_inv(v_conics[0], v_conics[1] * .5f, v_conics[1] * .5f, v_conics[2]);
    glm::mat2 v_covar2d(0.f);
    gsplat::inverse_vjp(covar2d_inv, v_covar2d_inv, v_covar2d);

    if (v_compensations != nullptr) {
        // vjp: compensation term
        const float compensation = compensations[thread_idx];
        const float v_compensation = v_compensations[thread_idx];
        gsplat::add_blur_vjp(
            eps2d, covar2d_inv, compensation, v_compensation, v_covar2d
        );
    }

    // transform Gaussian to camera space
   glm:: mat3 R(
        viewmats[0],
        viewmats[4],
        viewmats[8], // 1st column
        viewmats[1],
        viewmats[5],
        viewmats[9], // 2nd column
        viewmats[2],
        viewmats[6],
        viewmats[10] // 3rd column
    );
    glm::vec3 t(viewmats[3], viewmats[7], viewmats[11]);
    glm::mat3 covar;
    glm::vec4 quat;
    glm::vec3 scale;
    {
        // compute it from quaternions and scales
        quat = glm::make_vec4(quats + splat_idx * 4);
        scale = glm::make_vec3(scales + splat_idx * 3);
        gsplat::quat_scale_to_covar_preci(quat, scale, &covar, nullptr);
    }
    glm::vec3 mean_c;
    gsplat::posW2C(R, t, glm::make_vec3(means), mean_c);
    glm::mat3 covar_c;
    gsplat::covarW2C(R, covar, covar_c);

    float fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    glm::mat3 v_covar_c(0.f);
    glm::vec3 v_mean_c(0.f);
    switch (camera_model) {
    case gsplat::CameraModelType::PINHOLE: // perspective projection
        gsplat::persp_proj_vjp(
            mean_c,
            covar_c,
            fx,
            fy,
            cx,
            cy,
            v_covar2d,
            glm::make_vec2(v_means2d),
            v_mean_c,
            v_covar_c
        );
        break;
    case gsplat::CameraModelType::ORTHO: // orthographic projection
        gsplat::ortho_proj_vjp(
            mean_c,
            covar_c,
            fx,
            fy,
            cx,
            cy,
            v_covar2d,
            glm::make_vec2(v_means2d),
            v_mean_c,
            v_covar_c
        );
        break;
    case gsplat::CameraModelType::FISHEYE: // fisheye projection
        gsplat::fisheye_proj_vjp(
            mean_c,
            covar_c,
            fx,
            fy,
            cx,
            cy,
            v_covar2d,
            glm::make_vec2(v_means2d),
            v_mean_c,
            v_covar_c
        );
        break;
    }

    // add contribution from v_depths
    v_mean_c.z += v_depths[0];

    // vjp: transform Gaussian covariance to camera space
    glm::vec3 v_mean(0.f);
    glm::mat3 v_covar(0.f);
    glm::mat3 v_R(0.f);
    glm::vec3 v_t(0.f);
    gsplat::posW2C_VJP(R, t, glm::make_vec3(means), v_mean_c, v_R, v_t, v_mean);
    gsplat::covarW2C_VJP(R, covar, v_covar_c, v_R, v_covar);

    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    if (sparse_grad) {
        // write out results with sparse layout
        if (v_means != nullptr) {
            v_means += thread_idx * 3;
            #pragma unroll
            for (uint32_t i = 0; i < 3; i++) {
                v_means[i] = v_mean[i];
            }
        }
        {
            glm::mat3 rotmat = gsplat::quat_to_rotmat(quat);
            glm::vec4 v_quat(0.f);
            glm::vec3 v_scale(0.f);
            gsplat::quat_scale_to_covar_vjp(
                quat, scale, rotmat, v_covar, v_quat, v_scale
            );
            v_quats += thread_idx * 4;
            v_scales += thread_idx * 3;
            v_quats[0] = v_quat[0];
            v_quats[1] = v_quat[1];
            v_quats[2] = v_quat[2];
            v_quats[3] = v_quat[3];
            v_scales[0] = v_scale[0];
            v_scales[1] = v_scale[1];
            v_scales[2] = v_scale[2];
        }
    } else {
        // write out results with dense layout
        // #if __CUDA_ARCH__ >= 700
        // write out results with warp-level reduction
        auto warp_group_g = cg::labeled_partition(warp, splat_idx);
        if (v_means != nullptr) {
            warpSum(v_mean, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_means += splat_idx * 3;
                #pragma unroll
                for (uint32_t i = 0; i < 3; i++) {
                    atomicAdd(v_means + i, v_mean[i]);
                }
            }
        }
        {
            // Directly output gradients w.r.t. the quaternion and scale
            glm::mat3 rotmat = gsplat::quat_to_rotmat(quat);
            glm::vec4 v_quat(0.f);
            glm::vec3 v_scale(0.f);
            gsplat::quat_scale_to_covar_vjp(
                quat, scale, rotmat, v_covar, v_quat, v_scale
            );
            warpSum(v_quat, warp_group_g);
            warpSum(v_scale, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_quats += splat_idx * 4;
                v_scales += splat_idx * 3;
                atomicAdd(v_quats, v_quat[0]);
                atomicAdd(v_quats + 1, v_quat[1]);
                atomicAdd(v_quats + 2, v_quat[2]);
                atomicAdd(v_quats + 3, v_quat[3]);
                atomicAdd(v_scales, v_scale[0]);
                atomicAdd(v_scales + 1, v_scale[1]);
                atomicAdd(v_scales + 2, v_scale[2]);
            }
        }
    }
    // v_viewmats is always in dense layout
    if (v_viewmats != nullptr) {
        auto warp_group_c = cg::labeled_partition(warp, camera_idx);
        warpSum(v_R, warp_group_c);
        warpSum(v_t, warp_group_c);
        if (warp_group_c.thread_rank() == 0) {
            v_viewmats += camera_idx * 16;
            #pragma unroll
            for (uint32_t i = 0; i < 3; i++) { // rows
                #pragma unroll
                for (uint32_t j = 0; j < 3; j++) { // cols
                    atomicAdd(v_viewmats + i * 4 + j, v_R[j][i]);
                }
                atomicAdd(v_viewmats + i * 4 + 3, v_t[i]);
            }
        }
    }
}



std::tuple<
    at::Tensor,  // v_means
    at::Tensor,  // v_quats
    at::Tensor,  // v_scales
    at::Tensor  // v_viewmats
> projection_ewa_3dgs_hetero_backward_tensor(
    // fwd inputs
    const at::Tensor means, // [..., N, 3]
    const at::Tensor quats, // [..., N, 4]
    const at::Tensor scales, // [..., N, 3]
    const at::Tensor viewmats, // [..., C, 4, 4]
    const at::Tensor Ks, // [..., C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const gsplat::CameraModelType camera_model,
    // fwd outputs
    const at::Tensor camera_ids, // [nnz]
    const at::Tensor gaussian_ids, // [nnz]
    const at::Tensor conics, // [nnz, 3]
    const std::optional<at::Tensor> compensations, // [nnz] optional
    // grad outputs
    const at::Tensor v_means2d, // [nnz, 2]
    const at::Tensor v_depths, // [nnz]
    const at::Tensor v_conics, // [nnz, 3]
    const std::optional<at::Tensor> v_compensations, // [nnz] optional
    const bool viewmats_requires_grad,
    const bool sparse_grad
) {
    uint32_t N = means.size(-2);          // number of gaussians
    uint32_t C = viewmats.size(-3);       // number of cameras
    uint32_t nnz = camera_ids.size(0);

    auto opt = means.options();
    at::Tensor v_means, v_quats, v_scales, v_viewmats;
    if (sparse_grad) {
        v_means = at::zeros({nnz, 3}, opt);
        v_quats = at::zeros({nnz, 4}, opt);
        v_scales = at::zeros({nnz, 3}, opt);
    } else {
        v_means = at::zeros_like(means);
        v_quats = at::zeros_like(quats, opt);
        v_scales = at::zeros_like(scales, opt);
    }
    if (viewmats_requires_grad) {
        v_viewmats = at::zeros_like(viewmats, opt);
    }

    constexpr uint block = 256;
    projection_ewa_3dgs_hetero_backward_kernel<<<_CEIL_DIV(nnz, block), block>>>(
        C,
        nnz,
        means.data_ptr<float>(),
        quats.data_ptr<float>(),
        scales.data_ptr<float>(),
        viewmats.data_ptr<float>(),
        Ks.data_ptr<float>(),
        image_width,
        image_height,
        eps2d,
        camera_model,
        camera_ids.data_ptr<int64_t>(),
        gaussian_ids.data_ptr<int64_t>(),
        conics.data_ptr<float>(),
        compensations.has_value()
            ? compensations.value().data_ptr<float>()
            : nullptr,
        v_means2d.data_ptr<float>(),
        v_depths.data_ptr<float>(),
        v_conics.data_ptr<float>(),
        v_compensations.has_value()
            ? v_compensations.value().data_ptr<float>()
            : nullptr,
        sparse_grad,
        v_means.data_ptr<float>(),
        v_quats.data_ptr<float>(),
        v_scales.data_ptr<float>(),
        viewmats_requires_grad
            ? v_viewmats.data_ptr<float>()
            : nullptr
    );

    return std::make_tuple(v_means, v_quats, v_scales, v_viewmats);
}
