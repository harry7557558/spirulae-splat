#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#define TensorView _Slang_TensorView
#include "generated/slang_all.cu"
#undef TensorView

#include "misc.cuh"


__global__ void per_splat_losses_forward_kernel(
    bool is_3dgs,
    const size_t num_points,
    const float* __restrict__ scales_buffer,
    const float* __restrict__ opacities_buffer,
    const float* __restrict__ quats_buffer,
    float* __restrict__ out_losses,
    float max_gauss_ratio,
    float scale_regularization_weight,
    float mcmc_opacity_reg_weight,
    float mcmc_scale_reg_weight,
    float erank_reg_weight,
    float erank_reg_weight_s3,
    float quat_norm_reg_weight
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    FixedArray<float, kNumPerSplatLosses> losses;

    bool inside = idx < num_points;
    if (inside) {
        float3 scale;
        if (is_3dgs) scale = { scales_buffer[3*idx+0], scales_buffer[3*idx+1], scales_buffer[3*idx+2] };
        else scale = { scales_buffer[2*idx+0], scales_buffer[2*idx+1], 0.0f };
        float opacity = opacities_buffer[idx];
        float4 quat = { quats_buffer[4*idx+0], quats_buffer[4*idx+1], quats_buffer[4*idx+2], quats_buffer[4*idx+3] };
        per_splat_losses(
            is_3dgs,
            scale, opacity, quat,
            max_gauss_ratio,
            scale_regularization_weight,
            mcmc_opacity_reg_weight,
            mcmc_scale_reg_weight,
            erank_reg_weight,
            erank_reg_weight_s3,
            quat_norm_reg_weight,
            &losses
        );
    }

    auto block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    for (int i = 0; i < kNumPerSplatLosses; i++) {
        float loss = inside ? losses[i] : 0.0f;
        float loss_reduced = cg::reduce(warp, loss, cg::plus<float>());
        loss_reduced = loss_reduced / (float)num_points;
        if (warp.thread_rank() == 0) {
            atomicAdd(&out_losses[i], loss_reduced);
        }
    }
}


__global__ void per_splat_losses_backward_kernel(
    bool is_3dgs,
    const size_t num_points,
    const float* __restrict__ scales_buffer,
    const float* __restrict__ opacities_buffer,
    const float* __restrict__ quats_buffer,
    const float* __restrict__ v_out_losses,
    float* __restrict__ v_scales_buffer,
    float* __restrict__ v_opacities_buffer,
    float* __restrict__ v_quats_buffer,
    float max_gauss_ratio,
    float scale_regularization_weight,
    float mcmc_opacity_reg_weight,
    float mcmc_scale_reg_weight,
    float erank_reg_weight,
    float erank_reg_weight_s3,
    float quat_norm_reg_weight
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    bool inside = idx < num_points;
    if (!inside) return;

    FixedArray<float, kNumPerSplatLosses> v_losses;
    for (int i = 0; i < kNumPerSplatLosses; i++)
        v_losses[i] = v_out_losses[i] / (float)num_points;

    float3 scale;
    if (is_3dgs) scale = { scales_buffer[3*idx+0], scales_buffer[3*idx+1], scales_buffer[3*idx+2] };
    else scale = { scales_buffer[2*idx+0], scales_buffer[2*idx+1], 0.0f };
    float opacity = opacities_buffer[idx];
    float4 quat = { quats_buffer[4*idx+0], quats_buffer[4*idx+1], quats_buffer[4*idx+2], quats_buffer[4*idx+3] };

    float3 v_scale;
    float v_opacity;
    float4 v_quat;

    per_splat_losses_bwd(
        is_3dgs,
        scale, opacity, quat,
        v_losses,
        &v_scale, &v_opacity, &v_quat,
        max_gauss_ratio,
        scale_regularization_weight,
        mcmc_opacity_reg_weight,
        mcmc_scale_reg_weight,
        erank_reg_weight,
        erank_reg_weight_s3,
        quat_norm_reg_weight
    );

    if (is_3dgs) v_scales_buffer[3*idx+0] = v_scale.x, v_scales_buffer[3*idx+1] = v_scale.y, v_scales_buffer[3*idx+2] = v_scale.z;
    else v_scales_buffer[2*idx+0] = v_scale.x, v_scales_buffer[2*idx+1] = v_scale.y;
    v_opacities_buffer[idx] = v_opacity;
    v_quats_buffer[4*idx+0] = v_quat.x, v_quats_buffer[4*idx+1] = v_quat.y, v_quats_buffer[4*idx+2] = v_quat.z, v_quats_buffer[4*idx+3] = v_quat.w;
}


__global__ void blend_background_forward_kernel(
    const TensorView<float, 3> in_rgb,
    const TensorView<float, 3> in_alpha,
    const TensorView<float, 3> in_background,
    TensorView<float, 3> out_rgb
) {
    unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= in_rgb.shape[0]*in_rgb.shape[1])
        return;
    unsigned y = gid / in_rgb.shape[1];
    unsigned x = gid % in_rgb.shape[1];

    float3 rgb = in_rgb.load3f(y, x);
    float alpha = in_alpha.load1f(y, x);
    float3 background = in_background.load3f(y, x);

    rgb = blend_background(rgb, alpha, background);

    out_rgb.store3f(y, x, rgb);
}


__global__ void blend_background_backward_kernel(
    const TensorView<float, 3> in_rgb,
    const TensorView<float, 3> in_alpha,
    const TensorView<float, 3> in_background,
    const TensorView<float, 3> v_out_rgb,
    TensorView<float, 3> v_in_rgb,
    TensorView<float, 3> v_in_alpha,
    TensorView<float, 3> v_in_background
) {
    unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= in_rgb.shape[0]*in_rgb.shape[1])
        return;
    unsigned y = gid / in_rgb.shape[1];
    unsigned x = gid % in_rgb.shape[1];

    float3 rgb = in_rgb.load3f(y, x);
    float alpha = in_alpha.load1f(y, x);
    float3 background = in_background.load3f(y, x);

    float3 v_out = v_out_rgb.load3f(y, x);

    float3 v_rgb; float v_alpha; float3 v_background;
    blend_background_bwd(
        rgb, alpha, background,
        v_out,
        &v_rgb, &v_alpha, &v_background
    );

    v_in_rgb.store3f(y, x, v_rgb);
    v_in_alpha.store1f(y, x, v_alpha);
    v_in_background.store3f(y, x, v_background);

}
