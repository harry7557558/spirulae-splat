#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

#define TensorView _Slang_TensorView
#include "generated/projection.cu"
#undef TensorView

#include "common.cuh"

#include <gsplat/Common.h>
#include <gsplat/Utils.cuh>

typedef Matrix<float, 2, 2> float2x2;
typedef Matrix<float, 3, 3> float3x3;


#include "Primitive.cuh"


template<gsplat::CameraModelType camera_model>
__global__ void projection_ewa_3dgs_fused_fwd_kernel(
    const uint32_t B,
    const uint32_t C,
    const uint32_t N,
    const Vanilla3DGS::World::Buffer splats_world,
    const float *__restrict__ viewmats, // [B, C, 4, 4]
    const float *__restrict__ Ks,       // [B, C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    // outputs
    int2 *__restrict__ radii,         // [B, C, N, 2]
    float *__restrict__ depths,       // [B, C, N]
    Vanilla3DGS::Screen::Buffer splats_screen
) {
    // parallelize over B * C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= B * C * N) {
        return;
    }
    const uint32_t bid = idx / (C * N); // batch id
    const uint32_t cid = (idx / N) % C; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    viewmats += bid * C * 16 + cid * 16;
    Ks += bid * C * 9 + cid * 9;

    // glm is column-major but input is row-major
    float3x3 R = {
        viewmats[0], viewmats[1], viewmats[2],  // 1st row
        viewmats[4], viewmats[5], viewmats[6],  // 2nd row
        viewmats[8], viewmats[9], viewmats[10],  // 3rd row
    };
    float3 t = { viewmats[3], viewmats[7], viewmats[11] };

    float3 mean = splats_world.means[bid * N + gid];
    float4 quat = splats_world.quats[bid * N + gid];
    float3 scale = splats_world.scales[bid * N + gid];
    float in_opacity = splats_world.opacities[bid * N + gid];
    float fx = Ks[0], fy = Ks[4], cx = Ks[2], cy = Ks[5];

    int2 radius;
    float2 mean2d;
    float depth;
    float3 conic;
    float opacity;
    switch (camera_model) {
    case gsplat::CameraModelType::PINHOLE: // perspective projection
        projection_3dgs_persp(
            mean, quat, scale, in_opacity, R, t, fx, fy, cx, cy,
            image_width, image_height, eps2d, near_plane, far_plane, radius_clip,
            &radius, &mean2d, &depth, &conic, &opacity
        );
        break;
    case gsplat::CameraModelType::ORTHO: // orthographic projection
        projection_3dgs_ortho(
            mean, quat, scale, in_opacity, R, t, fx, fy, cx, cy,
            image_width, image_height, eps2d, near_plane, far_plane, radius_clip,
            &radius, &mean2d, &depth, &conic, &opacity
        );
        break;
    case gsplat::CameraModelType::FISHEYE: // fisheye projection
        projection_3dgs_fisheye(
            mean, quat, scale, in_opacity, R, t, fx, fy, cx, cy,
            image_width, image_height, eps2d, near_plane, far_plane, radius_clip,
            &radius, &mean2d, &depth, &conic, &opacity
        );
        break;
    }
    radii[idx] = radius;
    if (radius.x * radius.y > 0) {
        depths[idx] = depth;
        splats_screen.means2d[idx] = mean2d;
        splats_screen.conics[idx] = conic;
        splats_screen.opacities[idx] = opacity;
    }

}

std::tuple<
    at::Tensor,  // radii
    at::Tensor,  // depths
    Vanilla3DGS::Screen::TensorTuple  // out splats
> projection_ewa_3dgs_forward_tensor(
    // inputs
    const Vanilla3DGS::World::TensorTuple &in_splats,
    const at::Tensor viewmats,             // [..., C, 4, 4]
    const at::Tensor Ks,                   // [..., C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    bool is_antialiased,
    const gsplat::CameraModelType camera_model
) {
    Vanilla3DGS::World::Tensor splats_world(in_splats);
    uint32_t N = splats_world.size();    // number of gaussians
    uint32_t C = viewmats.size(-3); // number of cameras
    uint32_t B = splats_world.batchSize();    // number of batches

    auto opt = splats_world.options();
    at::Tensor radii = at::empty({C, N, 2}, opt.dtype(at::kInt));
    at::Tensor means2d = at::empty({C, N, 2}, opt);
    at::Tensor depths = at::empty({C, N}, opt);
    at::Tensor conics = at::empty({C, N, 3}, opt);
    at::Tensor out_opacities = at::empty({C, N}, opt);

    Vanilla3DGS::Screen::Tensor splats_screen = std::make_tuple(
        means2d, conics, out_opacities
    );

    #define _LAUNCH_ARGS \
        <<<_CEIL_DIV(B*C*N, block), block>>>( \
            B, C, N, \
            splats_world.buffer(), \
            viewmats.data_ptr<float>(), \
            Ks.data_ptr<float>(), \
            image_width, \
            image_height, \
            eps2d, \
            near_plane, \
            far_plane, \
            radius_clip, \
            (int2*)radii.data_ptr<int32_t>(), \
            depths.data_ptr<float>(), \
            splats_screen.buffer() \
        )

    constexpr uint block = 256;
    if (camera_model == gsplat::CameraModelType::PINHOLE)
        projection_ewa_3dgs_fused_fwd_kernel<gsplat::CameraModelType::PINHOLE> _LAUNCH_ARGS;
    else if (camera_model == gsplat::CameraModelType::ORTHO)
        projection_ewa_3dgs_fused_fwd_kernel<gsplat::CameraModelType::ORTHO> _LAUNCH_ARGS;
    else if (camera_model == gsplat::CameraModelType::FISHEYE)
        projection_ewa_3dgs_fused_fwd_kernel<gsplat::CameraModelType::FISHEYE> _LAUNCH_ARGS;

    #undef _LAUNCH_ARGS

    return std::make_tuple(radii, depths, std::make_tuple(means2d, conics, out_opacities));
}

template<gsplat::CameraModelType camera_model>
__global__ void projection_ewa_3dgs_fused_bwd_kernel(
    // fwd inputs
    const uint32_t B,
    const uint32_t C,
    const uint32_t N,
    const Vanilla3DGS::World::Buffer splats_world,
    const float *__restrict__ viewmats, // [B, C, 4, 4]
    const float *__restrict__ Ks,       // [B, C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    // fwd outputs
    const int2 *__restrict__ radii,          // [B, C, N, 2]
    // grad outputs
    const float *__restrict__ v_depths,        // [B, C, N]
    Vanilla3DGS::Screen::Buffer v_splats_screen,
    // grad inputs
    Vanilla3DGS::World::Buffer v_splats_world,
    float *__restrict__ v_viewmats // [B, C, 4, 4] optional
) {
    // parallelize over B * C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= B * C * N || radii[idx].x * radii[idx].y <= 0) {
        return;
    }
    const uint32_t bid = idx / (C * N); // batch id
    const uint32_t cid = (idx / N) % C; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    viewmats += bid * C * 16 + cid * 16;
    Ks += bid * C * 9 + cid * 9;

    // glm is column-major but input is row-major
    float3x3 R = {
        viewmats[0], viewmats[1], viewmats[2],  // 1st row
        viewmats[4], viewmats[5], viewmats[6],  // 2nd row
        viewmats[8], viewmats[9], viewmats[10],  // 3rd row
    };
    float3 t = { viewmats[3], viewmats[7], viewmats[11] };

    float3 mean = splats_world.means[bid * N + gid];
    float4 quat = splats_world.quats[bid * N + gid];
    float3 scale = splats_world.scales[bid * N + gid];
    float opacity = splats_world.opacities[bid * N + gid];
    float fx = Ks[0], fy = Ks[4], cx = Ks[2], cy = Ks[5];

    float3 v_mean = {0.f, 0.f, 0.f};
    float4 v_quat = {0.f, 0.f, 0.f, 0.f};
    float3 v_scale = {0.f, 0.f, 0.f};
    float v_opacity = 0.f;
    float3x3 v_R = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0.f};
    float3 v_t = {0.f, 0.f, 0.f};
    switch (camera_model) {
    case gsplat::CameraModelType::PINHOLE: // perspective projection
        projection_3dgs_persp_vjp(
            mean, quat, scale, opacity, R, t, fx, fy, cx, cy,
            image_width, image_height, eps2d,
            v_splats_screen.means2d[idx], 0.f, v_splats_screen.conics[idx], v_splats_screen.opacities[idx],
            &v_mean, &v_quat, &v_scale, &v_opacity, &v_R, &v_t
        );
        break;
    case gsplat::CameraModelType::ORTHO: // orthographic projection
        projection_3dgs_ortho_vjp(
            mean, quat, scale, opacity, R, t, fx, fy, cx, cy,
            image_width, image_height, eps2d,
            v_splats_screen.means2d[idx], 0.f, v_splats_screen.conics[idx], v_splats_screen.opacities[idx],
            &v_mean, &v_quat, &v_scale, &v_opacity, &v_R, &v_t
        );
        break;
    case gsplat::CameraModelType::FISHEYE: // fisheye projection
        projection_3dgs_fisheye_vjp(
            mean, quat, scale, opacity, R, t, fx, fy, cx, cy,
            image_width, image_height, eps2d,
            v_splats_screen.means2d[idx], 0.f, v_splats_screen.conics[idx], v_splats_screen.opacities[idx],
            &v_mean, &v_quat, &v_scale, &v_opacity, &v_R, &v_t
        );
        break;
    }

    auto warp = cg::tiled_partition<WARP_SIZE>(cg::this_thread_block());
    auto warp_group_g = cg::labeled_partition(warp, gid);
    Vanilla3DGS::World v_splat = { v_mean, v_quat, v_scale, v_opacity };
    v_splat.atomicAddBuffer(v_splats_world, bid * N + gid);

    if (v_viewmats != nullptr) {
        auto warp_group_c = cg::labeled_partition(warp, cid);
        warpSum(v_R[0], warp_group_c);
        warpSum(v_R[1], warp_group_c);
        warpSum(v_R[2], warp_group_c);
        warpSum(v_t, warp_group_c);
        if (warp_group_c.thread_rank() == 0) {
            v_viewmats += bid * C * 16 + cid * 16;
            #pragma unroll
            for (uint32_t i = 0; i < 3; i++) { // rows
                atomicAdd(v_viewmats + i * 4 + 0, v_R[i].x);
                atomicAdd(v_viewmats + i * 4 + 1, v_R[i].y);
                atomicAdd(v_viewmats + i * 4 + 2, v_R[i].z);
            }
            atomicAdd(v_viewmats + 0 * 4 + 3, v_t.x);
            atomicAdd(v_viewmats + 1 * 4 + 3, v_t.y);
            atomicAdd(v_viewmats + 2 * 4 + 3, v_t.z);
        }
    }

}

std::tuple<
    Vanilla3DGS::World::TensorTuple,  // v_splats
    at::Tensor  // v_viewmats
> projection_ewa_3dgs_backward_tensor(
    // inputs
    // fwd inputs
    const Vanilla3DGS::World::TensorTuple &splats_world_tuple,
    const at::Tensor viewmats,             // [..., C, 4, 4]
    const at::Tensor Ks,                   // [..., C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const gsplat::CameraModelType camera_model,
    // fwd outputs
    const at::Tensor radii,                       // [..., C, N, 2]
    // grad outputs
    const at::Tensor v_depths,                      // [..., C, N]
    const Vanilla3DGS::Screen::TensorTuple &v_splats_screen_tuple,
    const bool viewmats_requires_grad
) {
    Vanilla3DGS::World::Tensor splats_world(splats_world_tuple);
    uint32_t N = splats_world.size();    // number of gaussians
    uint32_t C = viewmats.size(-3); // number of cameras
    uint32_t B = splats_world.batchSize();    // number of batches

    Vanilla3DGS::Screen::Tensor v_splats_screen(v_splats_screen_tuple);

    Vanilla3DGS::World::Tensor v_splats_world = splats_world.zeros_like();

    auto opt = splats_world.options();
    at::Tensor v_viewmats;
    if (viewmats_requires_grad)
        v_viewmats = at::zeros_like(viewmats, opt);

    #define _LAUNCH_ARGS \
        <<<_CEIL_DIV(B*C*N, block), block>>>( \
            B, C, N, \
            splats_world.buffer(), \
            viewmats.data_ptr<float>(), \
            Ks.data_ptr<float>(), \
            image_width, \
            image_height, \
            eps2d, \
            (int2*)radii.data_ptr<int32_t>(), \
            v_depths.data_ptr<float>(), \
            v_splats_screen.buffer(), \
            v_splats_world.buffer(), \
            viewmats_requires_grad ? v_viewmats.data_ptr<float>() : nullptr \
        )

    constexpr uint block = 256;
    if (camera_model == gsplat::CameraModelType::PINHOLE)
        projection_ewa_3dgs_fused_bwd_kernel<gsplat::CameraModelType::PINHOLE> _LAUNCH_ARGS;
    else if (camera_model == gsplat::CameraModelType::ORTHO)
        projection_ewa_3dgs_fused_bwd_kernel<gsplat::CameraModelType::ORTHO> _LAUNCH_ARGS;
    else if (camera_model == gsplat::CameraModelType::FISHEYE)
        projection_ewa_3dgs_fused_bwd_kernel<gsplat::CameraModelType::FISHEYE> _LAUNCH_ARGS;

    #undef _LAUNCH_ARGS

    return std::make_tuple(v_splats_world.tuple(), v_viewmats);
}

