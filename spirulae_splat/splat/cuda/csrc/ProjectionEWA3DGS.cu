#include "hip/hip_runtime.h"
#include "ProjectionEWA3DGS.cuh"

#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

#include "common.cuh"

#include <gsplat/Common.h>
#include <gsplat/Utils.cuh>


template<typename SplatPrimitive, gsplat::CameraModelType camera_model>
__global__ void projection_fused_fwd_kernel(
    const uint32_t B,
    const uint32_t C,
    const uint32_t N,
    const bool antialiased,
    const typename SplatPrimitive::World::Buffer splats_world,
    const float *__restrict__ viewmats, // [B, C, 4, 4]
    const float *__restrict__ Ks,       // [B, C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    // outputs
    int2 *__restrict__ radii,         // [B, C, N, 2]
    float *__restrict__ depths,       // [B, C, N]
    typename SplatPrimitive::Screen::Buffer splats_screen
) {
    // parallelize over B * C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= B * C * N) {
        return;
    }
    const uint32_t bid = idx / (C * N); // batch id
    const uint32_t cid = (idx / N) % C; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // Load camera
    viewmats += bid * C * 16 + cid * 16;
    Ks += bid * C * 9 + cid * 9;
    float3x3 R = {
        viewmats[0], viewmats[1], viewmats[2],  // 1st row
        viewmats[4], viewmats[5], viewmats[6],  // 2nd row
        viewmats[8], viewmats[9], viewmats[10],  // 3rd row
    };
    float3 t = { viewmats[3], viewmats[7], viewmats[11] };
    float fx = Ks[0], fy = Ks[4], cx = Ks[2], cy = Ks[5];
    typename SplatPrimitive::FwdProjCamera cam = {
        R, t, fx, fy, cx, cy,
        image_width, image_height, antialiased,
        near_plane, far_plane, radius_clip
    };

    // Load splat
    typename SplatPrimitive::World splat_world =
        SplatPrimitive::World::load(splats_world, bid * N + gid);

    // Projection
    int2 radius;
    float depth;
    typename SplatPrimitive::Screen splat_screen;
    switch (camera_model) {
    case gsplat::CameraModelType::PINHOLE: // perspective projection
        SplatPrimitive::project_persp(splat_world, cam, splat_screen, radius, depth);
        break;
    // case gsplat::CameraModelType::ORTHO: // orthographic projection
    //     SplatPrimitive::project_ortho(splat_world, cam, splat_screen, radius, depth);
    //     break;
    // case gsplat::CameraModelType::FISHEYE: // fisheye projection
    //     SplatPrimitive::project_fisheye(splat_world, cam, splat_screen, radius, depth);
    //     break;
    }

    // Save results
    radii[idx] = radius;
    if (radius.x * radius.y > 0) {
        depths[idx] = depth;
        splat_screen.saveBuffer(splats_screen, idx);
    }
}

std::tuple<
    at::Tensor,  // radii
    at::Tensor,  // depths
    Vanilla3DGS::Screen::TensorTuple  // out splats
> projection_ewa_3dgs_forward_tensor(
    const bool antialiased,
    // inputs
    const Vanilla3DGS::World::TensorTuple &in_splats,
    const at::Tensor viewmats,             // [..., C, 4, 4]
    const at::Tensor Ks,                   // [..., C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const float near_plane,
    const float far_plane,
    const float radius_clip,
    const gsplat::CameraModelType camera_model
) {
    Vanilla3DGS::World::Tensor splats_world(in_splats);
    uint32_t N = splats_world.size();    // number of gaussians
    uint32_t C = viewmats.size(-3); // number of cameras
    uint32_t B = splats_world.batchSize();    // number of batches

    auto opt = splats_world.options();
    at::Tensor radii = at::empty({C, N, 2}, opt.dtype(at::kInt));
    at::Tensor means2d = at::empty({C, N, 2}, opt);
    at::Tensor depths = at::empty({C, N}, opt);
    at::Tensor conics = at::empty({C, N, 3}, opt);
    at::Tensor out_opacities = at::empty({C, N}, opt);

    Vanilla3DGS::Screen::Tensor splats_screen = std::make_tuple(
        means2d, conics, out_opacities
    );

    #define _LAUNCH_ARGS \
        <<<_CEIL_DIV(B*C*N, block), block>>>( \
            B, C, N, \
            antialiased, \
            splats_world.buffer(), \
            viewmats.data_ptr<float>(), \
            Ks.data_ptr<float>(), \
            image_width, \
            image_height, \
            eps2d, \
            near_plane, \
            far_plane, \
            radius_clip, \
            (int2*)radii.data_ptr<int32_t>(), \
            depths.data_ptr<float>(), \
            splats_screen.buffer() \
        )

    constexpr uint block = 256;
    if (camera_model == gsplat::CameraModelType::PINHOLE)
        projection_fused_fwd_kernel<Vanilla3DGS, gsplat::CameraModelType::PINHOLE> _LAUNCH_ARGS;
    // else if (camera_model == gsplat::CameraModelType::ORTHO)
    //     projection_fused_fwd_kernel<Vanilla3DGS, gsplat::CameraModelType::ORTHO> _LAUNCH_ARGS;
    // else if (camera_model == gsplat::CameraModelType::FISHEYE)
    //     projection_fused_fwd_kernel<Vanilla3DGS, gsplat::CameraModelType::FISHEYE> _LAUNCH_ARGS;
    else
        throw std::runtime_error("Unsupported camera model");

    #undef _LAUNCH_ARGS

    return std::make_tuple(radii, depths, std::make_tuple(means2d, conics, out_opacities));
}

template<typename SplatPrimitive, gsplat::CameraModelType camera_model>
__global__ void projection_fused_bwd_kernel(
    // fwd inputs
    const uint32_t B,
    const uint32_t C,
    const uint32_t N,
    const bool antialiased,
    const typename SplatPrimitive::World::Buffer splats_world,
    const float *__restrict__ viewmats, // [B, C, 4, 4]
    const float *__restrict__ Ks,       // [B, C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    // fwd outputs
    const int2 *__restrict__ radii,          // [B, C, N, 2]
    // grad outputs
    const float *__restrict__ v_depths,        // [B, C, N]
    typename SplatPrimitive::Screen::Buffer v_splats_screen,
    // grad inputs
    typename SplatPrimitive::World::Buffer v_splats_world,
    float *__restrict__ v_viewmats // [B, C, 4, 4] optional
) {
    // parallelize over B * C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= B * C * N || radii[idx].x * radii[idx].y <= 0) {
        return;
    }
    const uint32_t bid = idx / (C * N); // batch id
    const uint32_t cid = (idx / N) % C; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // Load camera
    viewmats += bid * C * 16 + cid * 16;
    Ks += bid * C * 9 + cid * 9;
    float3x3 R = {
        viewmats[0], viewmats[1], viewmats[2],  // 1st row
        viewmats[4], viewmats[5], viewmats[6],  // 2nd row
        viewmats[8], viewmats[9], viewmats[10],  // 3rd row
    };
    float3 t = { viewmats[3], viewmats[7], viewmats[11] };
    float fx = Ks[0], fy = Ks[4], cx = Ks[2], cy = Ks[5];
    typename SplatPrimitive::BwdProjCamera cam = {
        R, t, fx, fy, cx, cy,
        image_width, image_height, antialiased,
    };

    // Load splat
    typename SplatPrimitive::World splat_world =
        SplatPrimitive::World::load(splats_world, bid * N + gid);
    typename SplatPrimitive::Screen v_splat_screen =
        SplatPrimitive::Screen::load(v_splats_screen, idx);

    // Projection
    typename SplatPrimitive::World v_splat_world = SplatPrimitive::World::zero();
    float3x3 v_R = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0.f};
    float3 v_t = {0.f, 0.f, 0.f};
    switch (camera_model) {
    case gsplat::CameraModelType::PINHOLE: // perspective projection
        SplatPrimitive::project_persp_vjp(splat_world, cam, v_splat_screen, 0.f, v_splat_world, v_R, v_t);
        break;
    // case gsplat::CameraModelType::ORTHO: // orthographic projection
    //     SplatPrimitive::project_ortho_vjp(splat_world, cam, v_splat_screen, 0.f, v_splat_world, v_R, v_t);
    //     break;
    // case gsplat::CameraModelType::FISHEYE: // fisheye projection
    //     SplatPrimitive::project_fisheye_vjp(splat_world, cam, v_splat_screen, 0.f, v_splat_world, v_R, v_t);
    //     break;
    }

    // Save results
    auto warp = cg::tiled_partition<WARP_SIZE>(cg::this_thread_block());
    auto warp_group_g = cg::labeled_partition(warp, gid);
    v_splat_world.atomicAddBuffer(v_splats_world, bid * N + gid);

    if (v_viewmats != nullptr) {
        auto warp_group_c = cg::labeled_partition(warp, cid);
        warpSum(v_R[0], warp_group_c);
        warpSum(v_R[1], warp_group_c);
        warpSum(v_R[2], warp_group_c);
        warpSum(v_t, warp_group_c);
        if (warp_group_c.thread_rank() == 0) {
            v_viewmats += bid * C * 16 + cid * 16;
            #pragma unroll
            for (uint32_t i = 0; i < 3; i++) { // rows
                atomicAdd(v_viewmats + i * 4 + 0, v_R[i].x);
                atomicAdd(v_viewmats + i * 4 + 1, v_R[i].y);
                atomicAdd(v_viewmats + i * 4 + 2, v_R[i].z);
            }
            atomicAdd(v_viewmats + 0 * 4 + 3, v_t.x);
            atomicAdd(v_viewmats + 1 * 4 + 3, v_t.y);
            atomicAdd(v_viewmats + 2 * 4 + 3, v_t.z);
        }
    }

}

std::tuple<
    Vanilla3DGS::World::TensorTuple,  // v_splats
    at::Tensor  // v_viewmats
> projection_ewa_3dgs_backward_tensor(
    const bool antialiased,
    // fwd inputs
    const Vanilla3DGS::World::TensorTuple &splats_world_tuple,
    const at::Tensor viewmats,             // [..., C, 4, 4]
    const at::Tensor Ks,                   // [..., C, 3, 3]
    const uint32_t image_width,
    const uint32_t image_height,
    const float eps2d,
    const gsplat::CameraModelType camera_model,
    // fwd outputs
    const at::Tensor radii,                       // [..., C, N, 2]
    // grad outputs
    const at::Tensor v_depths,                      // [..., C, N]
    const Vanilla3DGS::Screen::TensorTuple &v_splats_screen_tuple,
    const bool viewmats_requires_grad
) {
    Vanilla3DGS::World::Tensor splats_world(splats_world_tuple);
    uint32_t N = splats_world.size();    // number of gaussians
    uint32_t C = viewmats.size(-3); // number of cameras
    uint32_t B = splats_world.batchSize();    // number of batches

    Vanilla3DGS::Screen::Tensor v_splats_screen(v_splats_screen_tuple);

    Vanilla3DGS::World::Tensor v_splats_world = splats_world.zeros_like();

    auto opt = splats_world.options();
    at::Tensor v_viewmats;
    if (viewmats_requires_grad)
        v_viewmats = at::zeros_like(viewmats, opt);

    #define _LAUNCH_ARGS \
        <<<_CEIL_DIV(B*C*N, block), block>>>( \
            B, C, N, \
            antialiased, \
            splats_world.buffer(), \
            viewmats.data_ptr<float>(), \
            Ks.data_ptr<float>(), \
            image_width, \
            image_height, \
            eps2d, \
            (int2*)radii.data_ptr<int32_t>(), \
            v_depths.data_ptr<float>(), \
            v_splats_screen.buffer(), \
            v_splats_world.buffer(), \
            viewmats_requires_grad ? v_viewmats.data_ptr<float>() : nullptr \
        )

    constexpr uint block = 256;
    if (camera_model == gsplat::CameraModelType::PINHOLE)
        projection_fused_bwd_kernel<Vanilla3DGS, gsplat::CameraModelType::PINHOLE> _LAUNCH_ARGS;
    // else if (camera_model == gsplat::CameraModelType::ORTHO)
    //     projection_fused_bwd_kernel<Vanilla3DGS, gsplat::CameraModelType::ORTHO> _LAUNCH_ARGS;
    // else if (camera_model == gsplat::CameraModelType::FISHEYE)
    //     projection_fused_bwd_kernel<Vanilla3DGS, gsplat::CameraModelType::FISHEYE> _LAUNCH_ARGS;
    else
        throw std::runtime_error("Unsupported camera model");

    #undef _LAUNCH_ARGS

    return std::make_tuple(v_splats_world.tuple(), v_viewmats);
}

