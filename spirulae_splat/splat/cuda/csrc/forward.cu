#include "hip/hip_runtime.h"
#include "forward.cuh"
#include "helpers.cuh"
#include "ch.cuh"
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <iostream>
#include <hip/hip_fp16.h>

namespace cg = cooperative_groups;

// kernel function for projecting each gaussian on device
// each thread processes one gaussian
__global__ void project_gaussians_forward_kernel(
    const int num_points,
    const float3* __restrict__ means3d,
    const float2* __restrict__ scales,
    const float4* __restrict__ quats,
    const float* __restrict__ viewmat,
    const float4 intrins,
    const dim3 tile_bounds,
    const unsigned block_width,
    const float clip_thresh,
    int4* __restrict__ bounds,
    int32_t* __restrict__ num_tiles_hit,
    float3* __restrict__ positions,
    float3* __restrict__ axes_u,
    float3* __restrict__ axes_v,
    float2* __restrict__ depth_grads
) {
    unsigned idx = cg::this_grid().thread_rank(); // idx of thread within grid
    if (idx >= num_points) {
        return;
    }
    bounds[idx] = {0, 0, 0, 0};
    num_tiles_hit[idx] = 0;

    glm::mat3 R0 = glm::mat3(
        viewmat[0], viewmat[4], viewmat[8],
        viewmat[1], viewmat[5], viewmat[9],
        viewmat[2], viewmat[6], viewmat[10]
    );
    glm::vec3 T0 = { viewmat[3], viewmat[7], viewmat[11] };

    // world to view
    glm::vec3 p_world = *(glm::vec3*)&means3d[idx];
    glm::vec3 p_view = R0 * p_world + T0;
    if (!(p_view.z >= clip_thresh))
        return;

    // patch orientation
    float2 scale = scales[idx];
    float4 quat = quats[idx];
    glm::mat3 Rq = quat_to_rotmat(quat);
    glm::mat3 R = R0 * Rq;
    glm::vec3 V0 = scale.x * R[0];
    glm::vec3 V1 = scale.y * R[1];

    // project to 2d
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    float2 center;
    float3 bound;
    const float kr = visibility_kernel_radius();
    project_ellipse_bound(p_view, kr*V0, kr*V1, fx, fy, cx, cy, center, bound);

    // compute the projected area
    int2 tile_min, tile_max;
    get_tile_bbox(center, {bound.x,bound.y}, tile_bounds, tile_min, tile_max, block_width);
    int32_t tile_area = (tile_max.x - tile_min.x) * (tile_max.y - tile_min.y);
    if (tile_area <= 0)
        return;

    // compute the depth gradient
    float2 depth_grad = projected_depth_grad(p_view, R, fx, fy);

    // output
    bounds[idx] = {tile_min.x, tile_min.y, tile_max.x, tile_max.y};
    num_tiles_hit[idx] = tile_area;
    positions[idx] = {p_view.x, p_view.y, p_view.z};
    axes_u[idx] = {V0.x, V0.y, V0.z};
    axes_v[idx] = {V1.x, V1.y, V1.z};
    depth_grads[idx] = {depth_grad.x, depth_grad.y};
}

// kernel to map each intersection from tile ID and depth to a gaussian
// writes output to isect_ids and gaussian_ids
__global__ void map_gaussian_to_intersects(
    const int num_points,
    const float3* __restrict__ positions,
    int4* __restrict__ bounds,
    const int32_t* __restrict__ cum_tiles_hit,
    const dim3 tile_bounds,
    const unsigned block_width,
    int64_t* __restrict__ isect_ids,
    int32_t* __restrict__ gaussian_ids
) {
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= num_points)
        return;
    int4 bound = bounds[idx];
    if (min(bound.z-bound.x, bound.w-bound.y) <= 0)
        return;

    // update the intersection info for all tiles this gaussian hits
    int32_t cur_idx = (idx == 0) ? 0 : cum_tiles_hit[idx - 1];
    // printf("point %d starting at %d\n", idx, cur_idx);
    int64_t depth_id = (int64_t) * (int32_t *)&(positions[idx].z);
    for (int i = bound.y; i < bound.w; ++i) {
        for (int j = bound.x; j < bound.z; ++j) {
            // isect_id is tile ID and depth as int32
            int64_t tile_id = i * tile_bounds.x + j; // tile within image
            isect_ids[cur_idx] = (tile_id << 32) | depth_id; // tile | depth id
            gaussian_ids[cur_idx] = idx;                     // 3D gaussian id
            ++cur_idx; // handles gaussians that hit more than one tile
        }
    }
    // printf("point %d ending at %d\n", idx, cur_idx);
}

// kernel to map sorted intersection IDs to tile bins
// expect that intersection IDs are sorted by increasing tile ID
// i.e. intersections of a tile are in contiguous chunks
__global__ void get_tile_bin_edges(
    const int num_intersects, const int64_t* __restrict__ isect_ids_sorted, int2* __restrict__ tile_bins
) {
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= num_intersects)
        return;
    // save the indices where the tile_id changes
    int32_t cur_tile_idx = (int32_t)(isect_ids_sorted[idx] >> 32);
    if (idx == 0 || idx == num_intersects - 1) {
        if (idx == 0)
            tile_bins[cur_tile_idx].x = 0;
        if (idx == num_intersects - 1)
            tile_bins[cur_tile_idx].y = num_intersects;
    }
    if (idx == 0)
        return;
    int32_t prev_tile_idx = (int32_t)(isect_ids_sorted[idx - 1] >> 32);
    if (prev_tile_idx != cur_tile_idx) {
        tile_bins[prev_tile_idx].y = idx;
        tile_bins[cur_tile_idx].x = idx;
        return;
    }
}


__global__ void rasterize_simple_forward(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    int* __restrict__ final_index,
    float3* __restrict__ out_img,
    float* __restrict__ out_alpha
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float3 pix_out = {0.f, 0.f, 0.f};
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            glm::vec3 color = color_batch[t];

            const float vis = alpha * T;
            pix_out.x = pix_out.x + color.x * vis;
            pix_out.y = pix_out.y + color.y * vis;
            pix_out.z = pix_out.z + color.z * vis;
            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        float3 final_color;
        final_color.x = pix_out.x + T * background.x;
        final_color.y = pix_out.y + T * background.y;
        final_color.z = pix_out.z + T * background.z;
        out_img[pix_id] = final_color;
        out_alpha[pix_id] = 1.0f - T;
    }
}

__global__ void rasterize_depth_forward(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    int* __restrict__ final_index,
    float* __restrict__ out_depth,
    float2* __restrict__ out_visibility
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    float interp = 1.0f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float median_depth = 0.0f;
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);
            // const float next_depth = pos.z;
            const float next_depth = poi.z;
            if (next_T < 0.5f) {
                if (T < 0.99999f) {
                    // https://www.desmos.com/3d/4kuwygxuio
                    interp = (1.0f-alpha)/alpha * (2.0f*T-1.0f);
                    interp = glm::clamp(interp, 0.0f, 1.0f);
                    median_depth = median_depth + (next_depth-median_depth)*interp;
                }
                else {
                    median_depth = next_depth;
                }
                T = next_T;
                cur_idx = batch_start + t;
                done = true;
                break;
            }
            median_depth = next_depth;
            T = next_T;
            cur_idx = batch_start + t;
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        out_depth[pix_id] = median_depth;
        out_visibility[pix_id] = {T, interp};
    }
}

__global__ void rasterize_forward(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const unsigned ch_degree_r,
    const unsigned ch_degree_phi,
    const float* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    const float2* __restrict__ depth_grads,
    const float3* __restrict__ depth_ref_im,
    int* __restrict__ final_index,
    float* __restrict__ out_alpha,
    float3* __restrict__ out_img,
    float4* __restrict__ out_depth_grad,
    float* __restrict__ out_reg_depth,
    float* __restrict__ out_reg_normal
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 color_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];
    // __shared__ glm::vec2 depth_grad_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float T = 1.f;  // current/total visibility
    float2 g_sum = {0.f, 0.f};  // sum of "normals"
    float3 pix_out = {0.f, 0.f, 0.f};  // output radiance
    float vis_sum = 0.f;  // output alpha
    float depth_sum = 0.f;  // output depth
    float depth_squared_sum = 0.f;  // for L2 depth regularizer
    const float3 depth_ref_raw = inside ?
        depth_ref_im[pix_id] : make_float3(0.f, 0.f, 0.f);
    const float2 depth_normal_ref = {depth_ref_raw.x, depth_ref_raw.y};
    const float depth_ref = depth_ref_raw.z;
    float reg_depth = 0.f;  // output depth regularizer
    float reg_normal = 0.f;  // output normal regularizer
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }
        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            // const float2 depth_grad = depth_grads[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_batch[tr] = {color.x, color.y, color.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
            // depth_grad_batch[tr] = {depth_grad.x, depth_grad.y};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            const float next_T = T * (1.f - alpha);

            glm::vec3 color_0 = color_batch[t];
            glm::vec3 color;
            if (dim_ch > 0) {
                int32_t g_id = id_batch[t];
                const float* coeffs = &ch_coeffs[3*dim_ch*g_id];
                glm::vec3 ch_color;
                ch_coeffs_to_color(
                    ch_degree_r, ch_degree_phi,
                    coeffs, {uv.x, uv.y}, &ch_color.x
                );
                color = color_0 / (1.0f+glm::exp(-ch_color));
            }
            else color = color_0;

            const float vis = alpha * T;
            #if DEPTH_REG_L == 01
            const float depth = pos.z;
            #else
            const float depth = poi.z;
            #endif
            const glm::vec2 g_i = *(glm::vec2*)&depth_grads[id_batch[t]];
            const float g_i_norm = glm::length(g_i) + 1e-6f;
            const glm::vec2 n_i = g_i / g_i_norm;

            pix_out.x = pix_out.x + color.x * vis;
            pix_out.y = pix_out.y + color.y * vis;
            pix_out.z = pix_out.z + color.z * vis;
            #if DEPTH_REG_L == 01
            reg_depth += vis*depth * vis_sum - vis * depth_sum;
            #elif DEPTH_REG_L == 02
            reg_depth += vis * (vis_sum*depth*depth + depth_squared_sum - 2.0f*depth*depth_sum);
            #elif DEPTH_REG_L == 11
            reg_depth += vis * abs(depth - depth_ref);
            #elif DEPTH_REG_L == 12
            reg_depth += vis * (depth-depth_ref) * (depth-depth_ref);
            #endif
            reg_normal += vis * (1.0f - (n_i.x*depth_normal_ref.x+n_i.y*depth_normal_ref.y));
            vis_sum += vis;
            depth_sum += vis*depth;
            depth_squared_sum += vis*depth*depth;
            g_sum.x = g_sum.x + vis * g_i.x;
            g_sum.y = g_sum.y + vis * g_i.y;

            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        out_alpha[pix_id] = 1.0f - T;
        float3 final_color;
        final_color.x = pix_out.x + T * background.x;
        final_color.y = pix_out.y + T * background.y;
        final_color.z = pix_out.z + T * background.z;
        out_img[pix_id] = final_color;
        out_depth_grad[pix_id] = {g_sum.x, g_sum.y, depth_sum, depth_squared_sum};
        out_reg_normal[pix_id] = reg_normal;
        out_reg_depth[pix_id] = reg_depth;
    }
}

// device helper to get screen space depth gradient
__device__ float2 projected_depth_grad(
    const glm::vec3 p, const glm::mat3 R,
    const float fx, const float fy
) {
    glm::vec3 n1 = R[2];
    glm::mat3 invJ = glm::mat3(
        p.z/fx, 0.0f, 0.0f,
        0.0f, p.z/fy, 0.0f,
        p.x/p.z, p.y/p.z, 1.0f
    );
    glm::vec3 n = glm::transpose(invJ) * n1;
    // n.z = safe_denom(n.z, 1e-3f);
    // return { -n.x/n.z, -n.y/n.z };
    return {-n.x*n.z, -n.y*n.z};
}
