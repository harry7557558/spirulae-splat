#include "hip/hip_runtime.h"
#include "slang.cuh"

struct DiffPair_float_0
{
    float primal_0;
    float differential_0;
};

inline __device__ void _d_max_0(DiffPair_float_0 * dpx_0, DiffPair_float_0 * dpy_0, float dOut_0)
{
    DiffPair_float_0 _S1 = *dpx_0;
    float _S2;
    if(((*dpx_0).primal_0) > ((*dpy_0).primal_0))
    {
        _S2 = dOut_0;
    }
    else
    {
        if(((*dpx_0).primal_0) < ((*dpy_0).primal_0))
        {
            _S2 = 0.0f;
        }
        else
        {
            _S2 = 0.5f * dOut_0;
        }
    }
    dpx_0->primal_0 = _S1.primal_0;
    dpx_0->differential_0 = _S2;
    DiffPair_float_0 _S3 = *dpy_0;
    if(((*dpy_0).primal_0) > (_S1.primal_0))
    {
        _S2 = dOut_0;
    }
    else
    {
        if(((*dpy_0).primal_0) < ((*dpx_0).primal_0))
        {
            _S2 = 0.0f;
        }
        else
        {
            _S2 = 0.5f * dOut_0;
        }
    }
    dpy_0->primal_0 = _S3.primal_0;
    dpy_0->differential_0 = _S2;
    return;
}

inline __device__ void _d_sqrt_0(DiffPair_float_0 * dpx_1, float dOut_1)
{
    float _S4 = 0.5f / (F32_sqrt(((F32_max((1.00000001168609742e-07f), ((*dpx_1).primal_0)))))) * dOut_1;
    dpx_1->primal_0 = (*dpx_1).primal_0;
    dpx_1->differential_0 = _S4;
    return;
}

inline __device__ void _d_rsqrt_0(DiffPair_float_0 * dpx_2, float dOut_2)
{
    float _S5 = -0.5f / ((*dpx_2).primal_0 * (F32_sqrt(((*dpx_2).primal_0)))) * dOut_2;
    dpx_2->primal_0 = (*dpx_2).primal_0;
    dpx_2->differential_0 = _S5;
    return;
}

inline __device__ Matrix<float, 3, 3>  transpose_0(Matrix<float, 3, 3>  x_0)
{
    Matrix<float, 3, 3>  result_0;
    int r_0 = int(0);
    for(;;)
    {
        if(r_0 < int(3))
        {
        }
        else
        {
            break;
        }
        int c_0 = int(0);
        for(;;)
        {
            if(c_0 < int(3))
            {
            }
            else
            {
                break;
            }
            *_slang_vector_get_element_ptr(((&result_0)->rows + (r_0)), c_0) = _slang_vector_get_element(x_0.rows[c_0], r_0);
            c_0 = c_0 + int(1);
        }
        r_0 = r_0 + int(1);
    }
    return result_0;
}

inline __device__ Matrix<float, 3, 2>  transpose_1(Matrix<float, 2, 3>  x_1)
{
    Matrix<float, 3, 2>  result_1;
    int r_1 = int(0);
    for(;;)
    {
        if(r_1 < int(3))
        {
        }
        else
        {
            break;
        }
        int c_1 = int(0);
        for(;;)
        {
            if(c_1 < int(2))
            {
            }
            else
            {
                break;
            }
            *_slang_vector_get_element_ptr(((&result_1)->rows + (r_1)), c_1) = _slang_vector_get_element(x_1.rows[c_1], r_1);
            c_1 = c_1 + int(1);
        }
        r_1 = r_1 + int(1);
    }
    return result_1;
}

inline __device__ Matrix<float, 2, 3>  transpose_2(Matrix<float, 3, 2>  x_2)
{
    Matrix<float, 2, 3>  result_2;
    int r_2 = int(0);
    for(;;)
    {
        if(r_2 < int(2))
        {
        }
        else
        {
            break;
        }
        int c_2 = int(0);
        for(;;)
        {
            if(c_2 < int(3))
            {
            }
            else
            {
                break;
            }
            *_slang_vector_get_element_ptr(((&result_2)->rows + (r_2)), c_2) = _slang_vector_get_element(x_2.rows[c_2], r_2);
            c_2 = c_2 + int(1);
        }
        r_2 = r_2 + int(1);
    }
    return result_2;
}

inline __device__ Matrix<float, 3, 3>  quat_to_rotmat(float4  quat_0)
{
    float x_3 = quat_0.y;
    float inv_norm_0 = (F32_rsqrt((x_3 * x_3 + quat_0.z * quat_0.z + quat_0.w * quat_0.w + quat_0.x * quat_0.x)));
    float x_4 = quat_0.y * inv_norm_0;
    float y_0 = quat_0.z * inv_norm_0;
    float z_0 = quat_0.w * inv_norm_0;
    float w_0 = quat_0.x * inv_norm_0;
    float x2_0 = x_4 * x_4;
    float y2_0 = y_0 * y_0;
    float z2_0 = z_0 * z_0;
    float xy_0 = x_4 * y_0;
    float xz_0 = x_4 * z_0;
    float yz_0 = y_0 * z_0;
    float wx_0 = w_0 * x_4;
    float wy_0 = w_0 * y_0;
    float wz_0 = w_0 * z_0;
    return transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_0 + z2_0), 2.0f * (xy_0 + wz_0), 2.0f * (xz_0 - wy_0), 2.0f * (xy_0 - wz_0), 1.0f - 2.0f * (x2_0 + z2_0), 2.0f * (yz_0 + wx_0), 2.0f * (xz_0 + wy_0), 2.0f * (yz_0 - wx_0), 1.0f - 2.0f * (x2_0 + y2_0)));
}

struct DiffPair_matrixx3Cfloatx2C3x2C3x3E_0
{
    Matrix<float, 3, 3>  primal_0;
    Matrix<float, 3, 3>  differential_0;
};

struct DiffPair_vectorx3Cfloatx2C3x3E_0
{
    float3  primal_0;
    float3  differential_0;
};

inline __device__ void _d_mul_0(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * left_0, DiffPair_vectorx3Cfloatx2C3x3E_0 * right_0, float3  dOut_3)
{
    float _S6 = (*left_0).primal_0.rows[int(0)].x * dOut_3.x;
    Matrix<float, 3, 3>  left_d_result_0;
    *&(((&left_d_result_0)->rows + (int(0)))->x) = (*right_0).primal_0.x * dOut_3.x;
    float sum_0 = _S6 + (*left_0).primal_0.rows[int(1)].x * dOut_3.y;
    *&(((&left_d_result_0)->rows + (int(1)))->x) = (*right_0).primal_0.x * dOut_3.y;
    float sum_1 = sum_0 + (*left_0).primal_0.rows[int(2)].x * dOut_3.z;
    *&(((&left_d_result_0)->rows + (int(2)))->x) = (*right_0).primal_0.x * dOut_3.z;
    float3  right_d_result_0;
    *&((&right_d_result_0)->x) = sum_1;
    float _S7 = (*left_0).primal_0.rows[int(0)].y * dOut_3.x;
    *&(((&left_d_result_0)->rows + (int(0)))->y) = (*right_0).primal_0.y * dOut_3.x;
    float sum_2 = _S7 + (*left_0).primal_0.rows[int(1)].y * dOut_3.y;
    *&(((&left_d_result_0)->rows + (int(1)))->y) = (*right_0).primal_0.y * dOut_3.y;
    float sum_3 = sum_2 + (*left_0).primal_0.rows[int(2)].y * dOut_3.z;
    *&(((&left_d_result_0)->rows + (int(2)))->y) = (*right_0).primal_0.y * dOut_3.z;
    *&((&right_d_result_0)->y) = sum_3;
    float _S8 = (*left_0).primal_0.rows[int(0)].z * dOut_3.x;
    *&(((&left_d_result_0)->rows + (int(0)))->z) = (*right_0).primal_0.z * dOut_3.x;
    float sum_4 = _S8 + (*left_0).primal_0.rows[int(1)].z * dOut_3.y;
    *&(((&left_d_result_0)->rows + (int(1)))->z) = (*right_0).primal_0.z * dOut_3.y;
    float sum_5 = sum_4 + (*left_0).primal_0.rows[int(2)].z * dOut_3.z;
    *&(((&left_d_result_0)->rows + (int(2)))->z) = (*right_0).primal_0.z * dOut_3.z;
    *&((&right_d_result_0)->z) = sum_5;
    left_0->primal_0 = (*left_0).primal_0;
    left_0->differential_0 = left_d_result_0;
    right_0->primal_0 = (*right_0).primal_0;
    right_0->differential_0 = right_d_result_0;
    return;
}

inline __device__ float3  mul_0(Matrix<float, 3, 3>  left_1, float3  right_1)
{
    float3  result_3;
    int i_0 = int(0);
    for(;;)
    {
        if(i_0 < int(3))
        {
        }
        else
        {
            break;
        }
        int j_0 = int(0);
        float sum_6 = 0.0f;
        for(;;)
        {
            if(j_0 < int(3))
            {
            }
            else
            {
                break;
            }
            float sum_7 = sum_6 + _slang_vector_get_element(left_1.rows[i_0], j_0) * _slang_vector_get_element(right_1, j_0);
            j_0 = j_0 + int(1);
            sum_6 = sum_7;
        }
        *_slang_vector_get_element_ptr(&result_3, i_0) = sum_6;
        i_0 = i_0 + int(1);
    }
    return result_3;
}

inline __device__ void posW2C(Matrix<float, 3, 3>  R_0, float3  t_0, float3  pW_0, float3  * pC_0)
{
    *pC_0 = mul_0(R_0, pW_0) + t_0;
    return;
}

inline __device__ void mul_1(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * left_2, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * right_2, Matrix<float, 3, 3>  dOut_4)
{
    Matrix<float, 3, 3>  left_d_result_1;
    *&(((&left_d_result_1)->rows + (int(0)))->x) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(0)))->y) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(0)))->z) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(1)))->x) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(1)))->y) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(1)))->z) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(2)))->x) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(2)))->y) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(2)))->z) = 0.0f;
    Matrix<float, 3, 3>  right_d_result_1;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(0)))->x) = *&(((&left_d_result_1)->rows + (int(0)))->x) + (*right_2).primal_0.rows[int(0)].x * dOut_4.rows[int(0)].x;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = *&(((&right_d_result_1)->rows + (int(0)))->x) + (*left_2).primal_0.rows[int(0)].x * dOut_4.rows[int(0)].x;
    *&(((&left_d_result_1)->rows + (int(0)))->y) = *&(((&left_d_result_1)->rows + (int(0)))->y) + (*right_2).primal_0.rows[int(1)].x * dOut_4.rows[int(0)].x;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = *&(((&right_d_result_1)->rows + (int(1)))->x) + (*left_2).primal_0.rows[int(0)].y * dOut_4.rows[int(0)].x;
    *&(((&left_d_result_1)->rows + (int(0)))->z) = *&(((&left_d_result_1)->rows + (int(0)))->z) + (*right_2).primal_0.rows[int(2)].x * dOut_4.rows[int(0)].x;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = *&(((&right_d_result_1)->rows + (int(2)))->x) + (*left_2).primal_0.rows[int(0)].z * dOut_4.rows[int(0)].x;
    *&(((&left_d_result_1)->rows + (int(0)))->x) = *&(((&left_d_result_1)->rows + (int(0)))->x) + (*right_2).primal_0.rows[int(0)].y * dOut_4.rows[int(0)].y;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = *&(((&right_d_result_1)->rows + (int(0)))->y) + (*left_2).primal_0.rows[int(0)].x * dOut_4.rows[int(0)].y;
    *&(((&left_d_result_1)->rows + (int(0)))->y) = *&(((&left_d_result_1)->rows + (int(0)))->y) + (*right_2).primal_0.rows[int(1)].y * dOut_4.rows[int(0)].y;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = *&(((&right_d_result_1)->rows + (int(1)))->y) + (*left_2).primal_0.rows[int(0)].y * dOut_4.rows[int(0)].y;
    *&(((&left_d_result_1)->rows + (int(0)))->z) = *&(((&left_d_result_1)->rows + (int(0)))->z) + (*right_2).primal_0.rows[int(2)].y * dOut_4.rows[int(0)].y;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = *&(((&right_d_result_1)->rows + (int(2)))->y) + (*left_2).primal_0.rows[int(0)].z * dOut_4.rows[int(0)].y;
    *&(((&left_d_result_1)->rows + (int(0)))->x) = *&(((&left_d_result_1)->rows + (int(0)))->x) + (*right_2).primal_0.rows[int(0)].z * dOut_4.rows[int(0)].z;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = *&(((&right_d_result_1)->rows + (int(0)))->z) + (*left_2).primal_0.rows[int(0)].x * dOut_4.rows[int(0)].z;
    *&(((&left_d_result_1)->rows + (int(0)))->y) = *&(((&left_d_result_1)->rows + (int(0)))->y) + (*right_2).primal_0.rows[int(1)].z * dOut_4.rows[int(0)].z;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = *&(((&right_d_result_1)->rows + (int(1)))->z) + (*left_2).primal_0.rows[int(0)].y * dOut_4.rows[int(0)].z;
    *&(((&left_d_result_1)->rows + (int(0)))->z) = *&(((&left_d_result_1)->rows + (int(0)))->z) + (*right_2).primal_0.rows[int(2)].z * dOut_4.rows[int(0)].z;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = *&(((&right_d_result_1)->rows + (int(2)))->z) + (*left_2).primal_0.rows[int(0)].z * dOut_4.rows[int(0)].z;
    *&(((&left_d_result_1)->rows + (int(1)))->x) = *&(((&left_d_result_1)->rows + (int(1)))->x) + (*right_2).primal_0.rows[int(0)].x * dOut_4.rows[int(1)].x;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = *&(((&right_d_result_1)->rows + (int(0)))->x) + (*left_2).primal_0.rows[int(1)].x * dOut_4.rows[int(1)].x;
    *&(((&left_d_result_1)->rows + (int(1)))->y) = *&(((&left_d_result_1)->rows + (int(1)))->y) + (*right_2).primal_0.rows[int(1)].x * dOut_4.rows[int(1)].x;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = *&(((&right_d_result_1)->rows + (int(1)))->x) + (*left_2).primal_0.rows[int(1)].y * dOut_4.rows[int(1)].x;
    *&(((&left_d_result_1)->rows + (int(1)))->z) = *&(((&left_d_result_1)->rows + (int(1)))->z) + (*right_2).primal_0.rows[int(2)].x * dOut_4.rows[int(1)].x;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = *&(((&right_d_result_1)->rows + (int(2)))->x) + (*left_2).primal_0.rows[int(1)].z * dOut_4.rows[int(1)].x;
    *&(((&left_d_result_1)->rows + (int(1)))->x) = *&(((&left_d_result_1)->rows + (int(1)))->x) + (*right_2).primal_0.rows[int(0)].y * dOut_4.rows[int(1)].y;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = *&(((&right_d_result_1)->rows + (int(0)))->y) + (*left_2).primal_0.rows[int(1)].x * dOut_4.rows[int(1)].y;
    *&(((&left_d_result_1)->rows + (int(1)))->y) = *&(((&left_d_result_1)->rows + (int(1)))->y) + (*right_2).primal_0.rows[int(1)].y * dOut_4.rows[int(1)].y;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = *&(((&right_d_result_1)->rows + (int(1)))->y) + (*left_2).primal_0.rows[int(1)].y * dOut_4.rows[int(1)].y;
    *&(((&left_d_result_1)->rows + (int(1)))->z) = *&(((&left_d_result_1)->rows + (int(1)))->z) + (*right_2).primal_0.rows[int(2)].y * dOut_4.rows[int(1)].y;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = *&(((&right_d_result_1)->rows + (int(2)))->y) + (*left_2).primal_0.rows[int(1)].z * dOut_4.rows[int(1)].y;
    *&(((&left_d_result_1)->rows + (int(1)))->x) = *&(((&left_d_result_1)->rows + (int(1)))->x) + (*right_2).primal_0.rows[int(0)].z * dOut_4.rows[int(1)].z;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = *&(((&right_d_result_1)->rows + (int(0)))->z) + (*left_2).primal_0.rows[int(1)].x * dOut_4.rows[int(1)].z;
    *&(((&left_d_result_1)->rows + (int(1)))->y) = *&(((&left_d_result_1)->rows + (int(1)))->y) + (*right_2).primal_0.rows[int(1)].z * dOut_4.rows[int(1)].z;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = *&(((&right_d_result_1)->rows + (int(1)))->z) + (*left_2).primal_0.rows[int(1)].y * dOut_4.rows[int(1)].z;
    *&(((&left_d_result_1)->rows + (int(1)))->z) = *&(((&left_d_result_1)->rows + (int(1)))->z) + (*right_2).primal_0.rows[int(2)].z * dOut_4.rows[int(1)].z;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = *&(((&right_d_result_1)->rows + (int(2)))->z) + (*left_2).primal_0.rows[int(1)].z * dOut_4.rows[int(1)].z;
    *&(((&left_d_result_1)->rows + (int(2)))->x) = *&(((&left_d_result_1)->rows + (int(2)))->x) + (*right_2).primal_0.rows[int(0)].x * dOut_4.rows[int(2)].x;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = *&(((&right_d_result_1)->rows + (int(0)))->x) + (*left_2).primal_0.rows[int(2)].x * dOut_4.rows[int(2)].x;
    *&(((&left_d_result_1)->rows + (int(2)))->y) = *&(((&left_d_result_1)->rows + (int(2)))->y) + (*right_2).primal_0.rows[int(1)].x * dOut_4.rows[int(2)].x;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = *&(((&right_d_result_1)->rows + (int(1)))->x) + (*left_2).primal_0.rows[int(2)].y * dOut_4.rows[int(2)].x;
    *&(((&left_d_result_1)->rows + (int(2)))->z) = *&(((&left_d_result_1)->rows + (int(2)))->z) + (*right_2).primal_0.rows[int(2)].x * dOut_4.rows[int(2)].x;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = *&(((&right_d_result_1)->rows + (int(2)))->x) + (*left_2).primal_0.rows[int(2)].z * dOut_4.rows[int(2)].x;
    *&(((&left_d_result_1)->rows + (int(2)))->x) = *&(((&left_d_result_1)->rows + (int(2)))->x) + (*right_2).primal_0.rows[int(0)].y * dOut_4.rows[int(2)].y;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = *&(((&right_d_result_1)->rows + (int(0)))->y) + (*left_2).primal_0.rows[int(2)].x * dOut_4.rows[int(2)].y;
    *&(((&left_d_result_1)->rows + (int(2)))->y) = *&(((&left_d_result_1)->rows + (int(2)))->y) + (*right_2).primal_0.rows[int(1)].y * dOut_4.rows[int(2)].y;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = *&(((&right_d_result_1)->rows + (int(1)))->y) + (*left_2).primal_0.rows[int(2)].y * dOut_4.rows[int(2)].y;
    *&(((&left_d_result_1)->rows + (int(2)))->z) = *&(((&left_d_result_1)->rows + (int(2)))->z) + (*right_2).primal_0.rows[int(2)].y * dOut_4.rows[int(2)].y;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = *&(((&right_d_result_1)->rows + (int(2)))->y) + (*left_2).primal_0.rows[int(2)].z * dOut_4.rows[int(2)].y;
    *&(((&left_d_result_1)->rows + (int(2)))->x) = *&(((&left_d_result_1)->rows + (int(2)))->x) + (*right_2).primal_0.rows[int(0)].z * dOut_4.rows[int(2)].z;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = *&(((&right_d_result_1)->rows + (int(0)))->z) + (*left_2).primal_0.rows[int(2)].x * dOut_4.rows[int(2)].z;
    *&(((&left_d_result_1)->rows + (int(2)))->y) = *&(((&left_d_result_1)->rows + (int(2)))->y) + (*right_2).primal_0.rows[int(1)].z * dOut_4.rows[int(2)].z;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = *&(((&right_d_result_1)->rows + (int(1)))->z) + (*left_2).primal_0.rows[int(2)].y * dOut_4.rows[int(2)].z;
    *&(((&left_d_result_1)->rows + (int(2)))->z) = *&(((&left_d_result_1)->rows + (int(2)))->z) + (*right_2).primal_0.rows[int(2)].z * dOut_4.rows[int(2)].z;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = *&(((&right_d_result_1)->rows + (int(2)))->z) + (*left_2).primal_0.rows[int(2)].z * dOut_4.rows[int(2)].z;
    left_2->primal_0 = (*left_2).primal_0;
    left_2->differential_0 = left_d_result_1;
    right_2->primal_0 = (*right_2).primal_0;
    right_2->differential_0 = right_d_result_1;
    return;
}

struct DiffPair_matrixx3Cfloatx2C2x2C3x3E_0
{
    Matrix<float, 2, 3>  primal_0;
    Matrix<float, 2, 3>  differential_0;
};

inline __device__ void mul_2(DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 * left_3, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * right_3, Matrix<float, 2, 3>  dOut_5)
{
    Matrix<float, 2, 3>  left_d_result_2;
    *&(((&left_d_result_2)->rows + (int(0)))->x) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(0)))->y) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(0)))->z) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(1)))->x) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(1)))->y) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(1)))->z) = 0.0f;
    Matrix<float, 3, 3>  right_d_result_2;
    *&(((&right_d_result_2)->rows + (int(0)))->x) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(0)))->y) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(0)))->z) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(1)))->x) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(1)))->y) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(1)))->z) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(2)))->x) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(2)))->y) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(2)))->z) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(0)))->x) = *&(((&left_d_result_2)->rows + (int(0)))->x) + (*right_3).primal_0.rows[int(0)].x * dOut_5.rows[int(0)].x;
    *&(((&right_d_result_2)->rows + (int(0)))->x) = *&(((&right_d_result_2)->rows + (int(0)))->x) + (*left_3).primal_0.rows[int(0)].x * dOut_5.rows[int(0)].x;
    *&(((&left_d_result_2)->rows + (int(0)))->y) = *&(((&left_d_result_2)->rows + (int(0)))->y) + (*right_3).primal_0.rows[int(1)].x * dOut_5.rows[int(0)].x;
    *&(((&right_d_result_2)->rows + (int(1)))->x) = *&(((&right_d_result_2)->rows + (int(1)))->x) + (*left_3).primal_0.rows[int(0)].y * dOut_5.rows[int(0)].x;
    *&(((&left_d_result_2)->rows + (int(0)))->z) = *&(((&left_d_result_2)->rows + (int(0)))->z) + (*right_3).primal_0.rows[int(2)].x * dOut_5.rows[int(0)].x;
    *&(((&right_d_result_2)->rows + (int(2)))->x) = *&(((&right_d_result_2)->rows + (int(2)))->x) + (*left_3).primal_0.rows[int(0)].z * dOut_5.rows[int(0)].x;
    *&(((&left_d_result_2)->rows + (int(0)))->x) = *&(((&left_d_result_2)->rows + (int(0)))->x) + (*right_3).primal_0.rows[int(0)].y * dOut_5.rows[int(0)].y;
    *&(((&right_d_result_2)->rows + (int(0)))->y) = *&(((&right_d_result_2)->rows + (int(0)))->y) + (*left_3).primal_0.rows[int(0)].x * dOut_5.rows[int(0)].y;
    *&(((&left_d_result_2)->rows + (int(0)))->y) = *&(((&left_d_result_2)->rows + (int(0)))->y) + (*right_3).primal_0.rows[int(1)].y * dOut_5.rows[int(0)].y;
    *&(((&right_d_result_2)->rows + (int(1)))->y) = *&(((&right_d_result_2)->rows + (int(1)))->y) + (*left_3).primal_0.rows[int(0)].y * dOut_5.rows[int(0)].y;
    *&(((&left_d_result_2)->rows + (int(0)))->z) = *&(((&left_d_result_2)->rows + (int(0)))->z) + (*right_3).primal_0.rows[int(2)].y * dOut_5.rows[int(0)].y;
    *&(((&right_d_result_2)->rows + (int(2)))->y) = *&(((&right_d_result_2)->rows + (int(2)))->y) + (*left_3).primal_0.rows[int(0)].z * dOut_5.rows[int(0)].y;
    *&(((&left_d_result_2)->rows + (int(0)))->x) = *&(((&left_d_result_2)->rows + (int(0)))->x) + (*right_3).primal_0.rows[int(0)].z * dOut_5.rows[int(0)].z;
    *&(((&right_d_result_2)->rows + (int(0)))->z) = *&(((&right_d_result_2)->rows + (int(0)))->z) + (*left_3).primal_0.rows[int(0)].x * dOut_5.rows[int(0)].z;
    *&(((&left_d_result_2)->rows + (int(0)))->y) = *&(((&left_d_result_2)->rows + (int(0)))->y) + (*right_3).primal_0.rows[int(1)].z * dOut_5.rows[int(0)].z;
    *&(((&right_d_result_2)->rows + (int(1)))->z) = *&(((&right_d_result_2)->rows + (int(1)))->z) + (*left_3).primal_0.rows[int(0)].y * dOut_5.rows[int(0)].z;
    *&(((&left_d_result_2)->rows + (int(0)))->z) = *&(((&left_d_result_2)->rows + (int(0)))->z) + (*right_3).primal_0.rows[int(2)].z * dOut_5.rows[int(0)].z;
    *&(((&right_d_result_2)->rows + (int(2)))->z) = *&(((&right_d_result_2)->rows + (int(2)))->z) + (*left_3).primal_0.rows[int(0)].z * dOut_5.rows[int(0)].z;
    *&(((&left_d_result_2)->rows + (int(1)))->x) = *&(((&left_d_result_2)->rows + (int(1)))->x) + (*right_3).primal_0.rows[int(0)].x * dOut_5.rows[int(1)].x;
    *&(((&right_d_result_2)->rows + (int(0)))->x) = *&(((&right_d_result_2)->rows + (int(0)))->x) + (*left_3).primal_0.rows[int(1)].x * dOut_5.rows[int(1)].x;
    *&(((&left_d_result_2)->rows + (int(1)))->y) = *&(((&left_d_result_2)->rows + (int(1)))->y) + (*right_3).primal_0.rows[int(1)].x * dOut_5.rows[int(1)].x;
    *&(((&right_d_result_2)->rows + (int(1)))->x) = *&(((&right_d_result_2)->rows + (int(1)))->x) + (*left_3).primal_0.rows[int(1)].y * dOut_5.rows[int(1)].x;
    *&(((&left_d_result_2)->rows + (int(1)))->z) = *&(((&left_d_result_2)->rows + (int(1)))->z) + (*right_3).primal_0.rows[int(2)].x * dOut_5.rows[int(1)].x;
    *&(((&right_d_result_2)->rows + (int(2)))->x) = *&(((&right_d_result_2)->rows + (int(2)))->x) + (*left_3).primal_0.rows[int(1)].z * dOut_5.rows[int(1)].x;
    *&(((&left_d_result_2)->rows + (int(1)))->x) = *&(((&left_d_result_2)->rows + (int(1)))->x) + (*right_3).primal_0.rows[int(0)].y * dOut_5.rows[int(1)].y;
    *&(((&right_d_result_2)->rows + (int(0)))->y) = *&(((&right_d_result_2)->rows + (int(0)))->y) + (*left_3).primal_0.rows[int(1)].x * dOut_5.rows[int(1)].y;
    *&(((&left_d_result_2)->rows + (int(1)))->y) = *&(((&left_d_result_2)->rows + (int(1)))->y) + (*right_3).primal_0.rows[int(1)].y * dOut_5.rows[int(1)].y;
    *&(((&right_d_result_2)->rows + (int(1)))->y) = *&(((&right_d_result_2)->rows + (int(1)))->y) + (*left_3).primal_0.rows[int(1)].y * dOut_5.rows[int(1)].y;
    *&(((&left_d_result_2)->rows + (int(1)))->z) = *&(((&left_d_result_2)->rows + (int(1)))->z) + (*right_3).primal_0.rows[int(2)].y * dOut_5.rows[int(1)].y;
    *&(((&right_d_result_2)->rows + (int(2)))->y) = *&(((&right_d_result_2)->rows + (int(2)))->y) + (*left_3).primal_0.rows[int(1)].z * dOut_5.rows[int(1)].y;
    *&(((&left_d_result_2)->rows + (int(1)))->x) = *&(((&left_d_result_2)->rows + (int(1)))->x) + (*right_3).primal_0.rows[int(0)].z * dOut_5.rows[int(1)].z;
    *&(((&right_d_result_2)->rows + (int(0)))->z) = *&(((&right_d_result_2)->rows + (int(0)))->z) + (*left_3).primal_0.rows[int(1)].x * dOut_5.rows[int(1)].z;
    *&(((&left_d_result_2)->rows + (int(1)))->y) = *&(((&left_d_result_2)->rows + (int(1)))->y) + (*right_3).primal_0.rows[int(1)].z * dOut_5.rows[int(1)].z;
    *&(((&right_d_result_2)->rows + (int(1)))->z) = *&(((&right_d_result_2)->rows + (int(1)))->z) + (*left_3).primal_0.rows[int(1)].y * dOut_5.rows[int(1)].z;
    *&(((&left_d_result_2)->rows + (int(1)))->z) = *&(((&left_d_result_2)->rows + (int(1)))->z) + (*right_3).primal_0.rows[int(2)].z * dOut_5.rows[int(1)].z;
    *&(((&right_d_result_2)->rows + (int(2)))->z) = *&(((&right_d_result_2)->rows + (int(2)))->z) + (*left_3).primal_0.rows[int(1)].z * dOut_5.rows[int(1)].z;
    left_3->primal_0 = (*left_3).primal_0;
    left_3->differential_0 = left_d_result_2;
    right_3->primal_0 = (*right_3).primal_0;
    right_3->differential_0 = right_d_result_2;
    return;
}

struct DiffPair_matrixx3Cfloatx2C3x2C2x3E_0
{
    Matrix<float, 3, 2>  primal_0;
    Matrix<float, 3, 2>  differential_0;
};

inline __device__ void mul_3(DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 * left_4, DiffPair_matrixx3Cfloatx2C3x2C2x3E_0 * right_4, Matrix<float, 2, 2>  dOut_6)
{
    Matrix<float, 2, 3>  left_d_result_3;
    *&(((&left_d_result_3)->rows + (int(0)))->x) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(0)))->y) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(0)))->z) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(1)))->x) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(1)))->y) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(1)))->z) = 0.0f;
    Matrix<float, 3, 2>  right_d_result_3;
    *&(((&right_d_result_3)->rows + (int(0)))->x) = 0.0f;
    *&(((&right_d_result_3)->rows + (int(0)))->y) = 0.0f;
    *&(((&right_d_result_3)->rows + (int(1)))->x) = 0.0f;
    *&(((&right_d_result_3)->rows + (int(1)))->y) = 0.0f;
    *&(((&right_d_result_3)->rows + (int(2)))->x) = 0.0f;
    *&(((&right_d_result_3)->rows + (int(2)))->y) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(0)))->x) = *&(((&left_d_result_3)->rows + (int(0)))->x) + (*right_4).primal_0.rows[int(0)].x * dOut_6.rows[int(0)].x;
    *&(((&right_d_result_3)->rows + (int(0)))->x) = *&(((&right_d_result_3)->rows + (int(0)))->x) + (*left_4).primal_0.rows[int(0)].x * dOut_6.rows[int(0)].x;
    *&(((&left_d_result_3)->rows + (int(0)))->y) = *&(((&left_d_result_3)->rows + (int(0)))->y) + (*right_4).primal_0.rows[int(1)].x * dOut_6.rows[int(0)].x;
    *&(((&right_d_result_3)->rows + (int(1)))->x) = *&(((&right_d_result_3)->rows + (int(1)))->x) + (*left_4).primal_0.rows[int(0)].y * dOut_6.rows[int(0)].x;
    *&(((&left_d_result_3)->rows + (int(0)))->z) = *&(((&left_d_result_3)->rows + (int(0)))->z) + (*right_4).primal_0.rows[int(2)].x * dOut_6.rows[int(0)].x;
    *&(((&right_d_result_3)->rows + (int(2)))->x) = *&(((&right_d_result_3)->rows + (int(2)))->x) + (*left_4).primal_0.rows[int(0)].z * dOut_6.rows[int(0)].x;
    *&(((&left_d_result_3)->rows + (int(0)))->x) = *&(((&left_d_result_3)->rows + (int(0)))->x) + (*right_4).primal_0.rows[int(0)].y * dOut_6.rows[int(0)].y;
    *&(((&right_d_result_3)->rows + (int(0)))->y) = *&(((&right_d_result_3)->rows + (int(0)))->y) + (*left_4).primal_0.rows[int(0)].x * dOut_6.rows[int(0)].y;
    *&(((&left_d_result_3)->rows + (int(0)))->y) = *&(((&left_d_result_3)->rows + (int(0)))->y) + (*right_4).primal_0.rows[int(1)].y * dOut_6.rows[int(0)].y;
    *&(((&right_d_result_3)->rows + (int(1)))->y) = *&(((&right_d_result_3)->rows + (int(1)))->y) + (*left_4).primal_0.rows[int(0)].y * dOut_6.rows[int(0)].y;
    *&(((&left_d_result_3)->rows + (int(0)))->z) = *&(((&left_d_result_3)->rows + (int(0)))->z) + (*right_4).primal_0.rows[int(2)].y * dOut_6.rows[int(0)].y;
    *&(((&right_d_result_3)->rows + (int(2)))->y) = *&(((&right_d_result_3)->rows + (int(2)))->y) + (*left_4).primal_0.rows[int(0)].z * dOut_6.rows[int(0)].y;
    *&(((&left_d_result_3)->rows + (int(1)))->x) = *&(((&left_d_result_3)->rows + (int(1)))->x) + (*right_4).primal_0.rows[int(0)].x * dOut_6.rows[int(1)].x;
    *&(((&right_d_result_3)->rows + (int(0)))->x) = *&(((&right_d_result_3)->rows + (int(0)))->x) + (*left_4).primal_0.rows[int(1)].x * dOut_6.rows[int(1)].x;
    *&(((&left_d_result_3)->rows + (int(1)))->y) = *&(((&left_d_result_3)->rows + (int(1)))->y) + (*right_4).primal_0.rows[int(1)].x * dOut_6.rows[int(1)].x;
    *&(((&right_d_result_3)->rows + (int(1)))->x) = *&(((&right_d_result_3)->rows + (int(1)))->x) + (*left_4).primal_0.rows[int(1)].y * dOut_6.rows[int(1)].x;
    *&(((&left_d_result_3)->rows + (int(1)))->z) = *&(((&left_d_result_3)->rows + (int(1)))->z) + (*right_4).primal_0.rows[int(2)].x * dOut_6.rows[int(1)].x;
    *&(((&right_d_result_3)->rows + (int(2)))->x) = *&(((&right_d_result_3)->rows + (int(2)))->x) + (*left_4).primal_0.rows[int(1)].z * dOut_6.rows[int(1)].x;
    *&(((&left_d_result_3)->rows + (int(1)))->x) = *&(((&left_d_result_3)->rows + (int(1)))->x) + (*right_4).primal_0.rows[int(0)].y * dOut_6.rows[int(1)].y;
    *&(((&right_d_result_3)->rows + (int(0)))->y) = *&(((&right_d_result_3)->rows + (int(0)))->y) + (*left_4).primal_0.rows[int(1)].x * dOut_6.rows[int(1)].y;
    *&(((&left_d_result_3)->rows + (int(1)))->y) = *&(((&left_d_result_3)->rows + (int(1)))->y) + (*right_4).primal_0.rows[int(1)].y * dOut_6.rows[int(1)].y;
    *&(((&right_d_result_3)->rows + (int(1)))->y) = *&(((&right_d_result_3)->rows + (int(1)))->y) + (*left_4).primal_0.rows[int(1)].y * dOut_6.rows[int(1)].y;
    *&(((&left_d_result_3)->rows + (int(1)))->z) = *&(((&left_d_result_3)->rows + (int(1)))->z) + (*right_4).primal_0.rows[int(2)].y * dOut_6.rows[int(1)].y;
    *&(((&right_d_result_3)->rows + (int(2)))->y) = *&(((&right_d_result_3)->rows + (int(2)))->y) + (*left_4).primal_0.rows[int(1)].z * dOut_6.rows[int(1)].y;
    left_4->primal_0 = (*left_4).primal_0;
    left_4->differential_0 = left_d_result_3;
    right_4->primal_0 = (*right_4).primal_0;
    right_4->differential_0 = right_d_result_3;
    return;
}

inline __device__ Matrix<float, 3, 3>  mul_4(Matrix<float, 3, 3>  left_5, Matrix<float, 3, 3>  right_5)
{
    Matrix<float, 3, 3>  result_4;
    int r_3 = int(0);
    for(;;)
    {
        if(r_3 < int(3))
        {
        }
        else
        {
            break;
        }
        int c_3 = int(0);
        for(;;)
        {
            if(c_3 < int(3))
            {
            }
            else
            {
                break;
            }
            int i_1 = int(0);
            float sum_8 = 0.0f;
            for(;;)
            {
                if(i_1 < int(3))
                {
                }
                else
                {
                    break;
                }
                float sum_9 = sum_8 + _slang_vector_get_element(left_5.rows[r_3], i_1) * _slang_vector_get_element(right_5.rows[i_1], c_3);
                i_1 = i_1 + int(1);
                sum_8 = sum_9;
            }
            *_slang_vector_get_element_ptr(((&result_4)->rows + (r_3)), c_3) = sum_8;
            c_3 = c_3 + int(1);
        }
        r_3 = r_3 + int(1);
    }
    return result_4;
}

inline __device__ Matrix<float, 2, 3>  mul_5(Matrix<float, 2, 3>  left_6, Matrix<float, 3, 3>  right_6)
{
    Matrix<float, 2, 3>  result_5;
    int r_4 = int(0);
    for(;;)
    {
        if(r_4 < int(2))
        {
        }
        else
        {
            break;
        }
        int c_4 = int(0);
        for(;;)
        {
            if(c_4 < int(3))
            {
            }
            else
            {
                break;
            }
            int i_2 = int(0);
            float sum_10 = 0.0f;
            for(;;)
            {
                if(i_2 < int(3))
                {
                }
                else
                {
                    break;
                }
                float sum_11 = sum_10 + _slang_vector_get_element(left_6.rows[r_4], i_2) * _slang_vector_get_element(right_6.rows[i_2], c_4);
                i_2 = i_2 + int(1);
                sum_10 = sum_11;
            }
            *_slang_vector_get_element_ptr(((&result_5)->rows + (r_4)), c_4) = sum_10;
            c_4 = c_4 + int(1);
        }
        r_4 = r_4 + int(1);
    }
    return result_5;
}

inline __device__ Matrix<float, 2, 2>  mul_6(Matrix<float, 2, 3>  left_7, Matrix<float, 3, 2>  right_7)
{
    Matrix<float, 2, 2>  result_6;
    int r_5 = int(0);
    for(;;)
    {
        if(r_5 < int(2))
        {
        }
        else
        {
            break;
        }
        int c_5 = int(0);
        for(;;)
        {
            if(c_5 < int(2))
            {
            }
            else
            {
                break;
            }
            int i_3 = int(0);
            float sum_12 = 0.0f;
            for(;;)
            {
                if(i_3 < int(3))
                {
                }
                else
                {
                    break;
                }
                float sum_13 = sum_12 + _slang_vector_get_element(left_7.rows[r_5], i_3) * _slang_vector_get_element(right_7.rows[i_3], c_5);
                i_3 = i_3 + int(1);
                sum_12 = sum_13;
            }
            *_slang_vector_get_element_ptr(((&result_6)->rows + (r_5)), c_5) = sum_12;
            c_5 = c_5 + int(1);
        }
        r_5 = r_5 + int(1);
    }
    return result_6;
}

inline __device__ void covarW2C(Matrix<float, 3, 3>  R_1, Matrix<float, 3, 3>  covarW_0, Matrix<float, 3, 3>  * covarC_0)
{
    *covarC_0 = mul_4(mul_4(R_1, covarW_0), transpose_0(R_1));
    return;
}

inline __device__ void quat_scale_to_covar(float4  quat_1, float3  scale_0, Matrix<float, 3, 3>  * covar_0)
{
    float x_5 = quat_1.y;
    float inv_norm_1 = (F32_rsqrt((x_5 * x_5 + quat_1.z * quat_1.z + quat_1.w * quat_1.w + quat_1.x * quat_1.x)));
    float x_6 = quat_1.y * inv_norm_1;
    float y_1 = quat_1.z * inv_norm_1;
    float z_1 = quat_1.w * inv_norm_1;
    float w_1 = quat_1.x * inv_norm_1;
    float x2_1 = x_6 * x_6;
    float y2_1 = y_1 * y_1;
    float z2_1 = z_1 * z_1;
    float xy_1 = x_6 * y_1;
    float xz_1 = x_6 * z_1;
    float yz_1 = y_1 * z_1;
    float wx_1 = w_1 * x_6;
    float wy_1 = w_1 * y_1;
    float wz_1 = w_1 * z_1;
    Matrix<float, 3, 3>  M_0 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_1 + z2_1), 2.0f * (xy_1 + wz_1), 2.0f * (xz_1 - wy_1), 2.0f * (xy_1 - wz_1), 1.0f - 2.0f * (x2_1 + z2_1), 2.0f * (yz_1 + wx_1), 2.0f * (xz_1 + wy_1), 2.0f * (yz_1 - wx_1), 1.0f - 2.0f * (x2_1 + y2_1))), makeMatrix<float, 3, 3> (scale_0.x, 0.0f, 0.0f, 0.0f, scale_0.y, 0.0f, 0.0f, 0.0f, scale_0.z));
    *covar_0 = mul_4(M_0, transpose_0(M_0));
    return;
}

inline __device__ void _d_min_0(DiffPair_float_0 * dpx_3, DiffPair_float_0 * dpy_1, float dOut_7)
{
    DiffPair_float_0 _S9 = *dpx_3;
    float _S10;
    if(((*dpx_3).primal_0) < ((*dpy_1).primal_0))
    {
        _S10 = dOut_7;
    }
    else
    {
        if(((*dpx_3).primal_0) > ((*dpy_1).primal_0))
        {
            _S10 = 0.0f;
        }
        else
        {
            _S10 = 0.5f * dOut_7;
        }
    }
    dpx_3->primal_0 = _S9.primal_0;
    dpx_3->differential_0 = _S10;
    DiffPair_float_0 _S11 = *dpy_1;
    if(((*dpy_1).primal_0) < (_S9.primal_0))
    {
        _S10 = dOut_7;
    }
    else
    {
        if(((*dpy_1).primal_0) > ((*dpx_3).primal_0))
        {
            _S10 = 0.0f;
        }
        else
        {
            _S10 = 0.5f * dOut_7;
        }
    }
    dpy_1->primal_0 = _S11.primal_0;
    dpy_1->differential_0 = _S10;
    return;
}

inline __device__ void persp_proj_3dgs(float3  mean3d_0, Matrix<float, 3, 3>  cov3d_0, float fx_0, float fy_0, float cx_0, float cy_0, uint width_0, uint height_0, Matrix<float, 2, 2>  * cov2d_0, float2  * mean2d_0)
{
    float _S12 = float(width_0);
    float _S13 = float(height_0);
    float _S14 = 0.30000001192092896f * (0.5f * _S12 / fx_0);
    float _S15 = 0.30000001192092896f * (0.5f * _S13 / fy_0);
    float rz_0 = 1.0f / mean3d_0.z;
    float rz2_0 = rz_0 * rz_0;
    Matrix<float, 2, 3>  J_0 = makeMatrix<float, 2, 3> (fx_0 * rz_0, 0.0f, - fx_0 * (mean3d_0.z * (F32_min(((_S12 - cx_0) / fx_0 + _S14), ((F32_max((- (cx_0 / fx_0 + _S14)), (mean3d_0.x * rz_0))))))) * rz2_0, 0.0f, fy_0 * rz_0, - fy_0 * (mean3d_0.z * (F32_min(((_S13 - cy_0) / fy_0 + _S15), ((F32_max((- (cy_0 / fy_0 + _S15)), (mean3d_0.y * rz_0))))))) * rz2_0);
    *cov2d_0 = mul_6(mul_5(J_0, cov3d_0), transpose_1(J_0));
    *mean2d_0 = make_float2 (fx_0 * mean3d_0.x * rz_0 + cx_0, fy_0 * mean3d_0.y * rz_0 + cy_0);
    return;
}

inline __device__ void persp_proj_3dgs(float3  mean3d_1, Matrix<float, 3, 3>  cov3d_1, float fx_1, float fy_1, float cx_1, float cy_1, Matrix<float, 2, 2>  * cov2d_1, float2  * mean2d_1)
{
    float rz_1 = 1.0f / mean3d_1.z;
    float rz2_1 = rz_1 * rz_1;
    Matrix<float, 2, 3>  J_1 = makeMatrix<float, 2, 3> (fx_1 * rz_1, 0.0f, - fx_1 * mean3d_1.x * rz2_1, 0.0f, fy_1 * rz_1, - fy_1 * mean3d_1.y * rz2_1);
    *cov2d_1 = mul_6(mul_5(J_1, cov3d_1), transpose_1(J_1));
    *mean2d_1 = make_float2 (fx_1 * mean3d_1.x * rz_1 + cx_1, fy_1 * mean3d_1.y * rz_1 + cy_1);
    return;
}

inline __device__ float dot_0(float2  x_7, float2  y_2)
{
    int i_4 = int(0);
    float result_7 = 0.0f;
    for(;;)
    {
        if(i_4 < int(2))
        {
        }
        else
        {
            break;
        }
        float result_8 = result_7 + _slang_vector_get_element(x_7, i_4) * _slang_vector_get_element(y_2, i_4);
        i_4 = i_4 + int(1);
        result_7 = result_8;
    }
    return result_7;
}

inline __device__ float length_0(float2  x_8)
{
    return (F32_sqrt((dot_0(x_8, x_8))));
}

inline __device__ void _d_atan2_0(DiffPair_float_0 * dpy_2, DiffPair_float_0 * dpx_4, float dOut_8)
{
    DiffPair_float_0 _S16 = *dpx_4;
    float _S17 = - (*dpy_2).primal_0 / ((*dpx_4).primal_0 * (*dpx_4).primal_0 + (*dpy_2).primal_0 * (*dpy_2).primal_0) * dOut_8;
    dpx_4->primal_0 = (*dpx_4).primal_0;
    dpx_4->differential_0 = _S17;
    float _S18 = _S16.primal_0 / (_S16.primal_0 * _S16.primal_0 + (*dpy_2).primal_0 * (*dpy_2).primal_0) * dOut_8;
    dpy_2->primal_0 = (*dpy_2).primal_0;
    dpy_2->differential_0 = _S18;
    return;
}

inline __device__ void fisheye_proj_3dgs(float3  mean3d_2, Matrix<float, 3, 3>  cov3d_2, float fx_2, float fy_2, float cx_2, float cy_2, Matrix<float, 2, 2>  * cov2d_2, float2  * mean2d_2)
{
    float xy_len_0 = length_0(make_float2 (mean3d_2.x, mean3d_2.y)) + 1.00000001168609742e-07f;
    float theta_0 = (F32_atan2((xy_len_0), (mean3d_2.z + 1.00000001168609742e-07f)));
    *mean2d_2 = make_float2 (mean3d_2.x * fx_2 * theta_0 / xy_len_0 + cx_2, mean3d_2.y * fy_2 * theta_0 / xy_len_0 + cy_2);
    float x2_2 = mean3d_2.x * mean3d_2.x + 1.00000001168609742e-07f;
    float y2_2 = mean3d_2.y * mean3d_2.y;
    float xy_2 = mean3d_2.x * mean3d_2.y;
    float x2y2_0 = x2_2 + y2_2;
    float x2y2z2_inv_0 = 1.0f / (x2y2_0 + mean3d_2.z * mean3d_2.z);
    float b_0 = (F32_atan2((xy_len_0), (mean3d_2.z))) / xy_len_0 / x2y2_0;
    float a_0 = mean3d_2.z * x2y2z2_inv_0 / x2y2_0;
    float _S19 = a_0 - b_0;
    Matrix<float, 2, 3>  J_2 = makeMatrix<float, 2, 3> (fx_2 * (x2_2 * a_0 + y2_2 * b_0), fx_2 * xy_2 * _S19, - fx_2 * mean3d_2.x * x2y2z2_inv_0, fy_2 * xy_2 * _S19, fy_2 * (y2_2 * a_0 + x2_2 * b_0), - fy_2 * mean3d_2.y * x2y2z2_inv_0);
    *cov2d_2 = mul_6(mul_5(J_2, cov3d_2), transpose_1(J_2));
    return;
}

inline __device__ void ortho_proj_3dgs(float3  mean3d_3, Matrix<float, 3, 3>  cov3d_3, float fx_3, float fy_3, float cx_3, float cy_3, Matrix<float, 2, 2>  * cov2d_3, float2  * mean2d_3)
{
    Matrix<float, 2, 3>  J_3 = makeMatrix<float, 2, 3> (fx_3, 0.0f, 0.0f, 0.0f, fy_3, 0.0f);
    *cov2d_3 = mul_6(mul_5(J_3, cov3d_3), transpose_1(J_3));
    *mean2d_3 = make_float2 (fx_3 * mean3d_3.x + cx_3, fy_3 * mean3d_3.y + cy_3);
    return;
}

inline __device__ float add_blur(float eps2d_0, Matrix<float, 2, 2>  * covar_1, float * compensation_0)
{
    float det_orig_0 = *&((covar_1->rows + (int(0)))->x) * *&((covar_1->rows + (int(1)))->y) - *&((covar_1->rows + (int(0)))->y) * *&((covar_1->rows + (int(1)))->x);
    *&((covar_1->rows + (int(0)))->x) = *&((covar_1->rows + (int(0)))->x) + eps2d_0;
    float _S20 = *&((covar_1->rows + (int(1)))->y) + eps2d_0;
    *&((covar_1->rows + (int(1)))->y) = _S20;
    float det_blur_0 = *&((covar_1->rows + (int(0)))->x) * _S20 - *&((covar_1->rows + (int(0)))->y) * *&((covar_1->rows + (int(1)))->x);
    *compensation_0 = (F32_sqrt(((F32_max((0.0f), (det_orig_0 / det_blur_0))))));
    return det_blur_0;
}

inline __device__ Matrix<float, 2, 2>  inverse(Matrix<float, 2, 2>  m_0)
{
    float invdet_0 = 1.0f / (m_0.rows[int(0)].x * m_0.rows[int(1)].y - m_0.rows[int(0)].y * m_0.rows[int(1)].x);
    return makeMatrix<float, 2, 2> (m_0.rows[int(1)].y * invdet_0, - m_0.rows[int(0)].y * invdet_0, - m_0.rows[int(1)].x * invdet_0, m_0.rows[int(0)].x * invdet_0);
}

inline __device__ void _d_exp_0(DiffPair_float_0 * dpx_5, float dOut_9)
{
    float _S21 = (F32_exp(((*dpx_5).primal_0))) * dOut_9;
    dpx_5->primal_0 = (*dpx_5).primal_0;
    dpx_5->differential_0 = _S21;
    return;
}

inline __device__ void _d_log_0(DiffPair_float_0 * dpx_6, float dOut_10)
{
    float _S22 = 1.0f / (*dpx_6).primal_0 * dOut_10;
    dpx_6->primal_0 = (*dpx_6).primal_0;
    dpx_6->differential_0 = _S22;
    return;
}

inline __device__ void projection_3dgs_persp(float3  mean_0, float4  quat_2, float3  scale_1, float in_opacity_0, Matrix<float, 3, 3>  R_2, float3  t_1, float fx_4, float fy_4, float cx_4, float cy_4, uint image_width_0, uint image_height_0, float eps2d_1, float near_plane_0, float far_plane_0, float radius_clip_0, int2  * radii_0, float2  * mean2d_4, float * depth_0, float3  * conic_0, float * opacity_0)
{
    for(;;)
    {
        float3  mean_c_0 = mul_0(R_2, mean_0) + t_1;
        float _S23 = mean_c_0.z;
        bool _S24;
        if(_S23 < near_plane_0)
        {
            _S24 = true;
        }
        else
        {
            _S24 = _S23 > far_plane_0;
        }
        if(_S24)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        float x_9 = quat_2.y;
        float inv_norm_2 = (F32_rsqrt((x_9 * x_9 + quat_2.z * quat_2.z + quat_2.w * quat_2.w + quat_2.x * quat_2.x)));
        float x_10 = quat_2.y * inv_norm_2;
        float y_3 = quat_2.z * inv_norm_2;
        float z_2 = quat_2.w * inv_norm_2;
        float w_2 = quat_2.x * inv_norm_2;
        float x2_3 = x_10 * x_10;
        float y2_3 = y_3 * y_3;
        float z2_2 = z_2 * z_2;
        float xy_3 = x_10 * y_3;
        float xz_2 = x_10 * z_2;
        float yz_2 = y_3 * z_2;
        float wx_2 = w_2 * x_10;
        float wy_2 = w_2 * y_3;
        float wz_2 = w_2 * z_2;
        Matrix<float, 3, 3>  M_1 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_3 + z2_2), 2.0f * (xy_3 + wz_2), 2.0f * (xz_2 - wy_2), 2.0f * (xy_3 - wz_2), 1.0f - 2.0f * (x2_3 + z2_2), 2.0f * (yz_2 + wx_2), 2.0f * (xz_2 + wy_2), 2.0f * (yz_2 - wx_2), 1.0f - 2.0f * (x2_3 + y2_3))), makeMatrix<float, 3, 3> (scale_1.x, 0.0f, 0.0f, 0.0f, scale_1.y, 0.0f, 0.0f, 0.0f, scale_1.z));
        Matrix<float, 3, 3>  covar_c_0 = mul_4(mul_4(R_2, mul_4(M_1, transpose_0(M_1))), transpose_0(R_2));
        Matrix<float, 2, 2>  covar2d_0;
        float _S25 = float(image_width_0);
        float _S26 = float(image_height_0);
        float _S27 = 0.30000001192092896f * (0.5f * _S25 / fx_4);
        float _S28 = 0.30000001192092896f * (0.5f * _S26 / fy_4);
        float rz_2 = 1.0f / mean_c_0.z;
        float rz2_2 = rz_2 * rz_2;
        Matrix<float, 2, 3>  J_4 = makeMatrix<float, 2, 3> (fx_4 * rz_2, 0.0f, - fx_4 * (mean_c_0.z * (F32_min(((_S25 - cx_4) / fx_4 + _S27), ((F32_max((- (cx_4 / fx_4 + _S27)), (mean_c_0.x * rz_2))))))) * rz2_2, 0.0f, fy_4 * rz_2, - fy_4 * (mean_c_0.z * (F32_min(((_S26 - cy_4) / fy_4 + _S28), ((F32_max((- (cy_4 / fy_4 + _S28)), (mean_c_0.y * rz_2))))))) * rz2_2);
        covar2d_0 = mul_6(mul_5(J_4, covar_c_0), transpose_1(J_4));
        *mean2d_4 = make_float2 (fx_4 * mean_c_0.x * rz_2 + cx_4, fy_4 * mean_c_0.y * rz_2 + cy_4);
        *&(((&covar2d_0)->rows + (int(0)))->x) = *&(((&covar2d_0)->rows + (int(0)))->x) + eps2d_1;
        float _S29 = *&(((&covar2d_0)->rows + (int(1)))->y) + eps2d_1;
        *&(((&covar2d_0)->rows + (int(1)))->y) = _S29;
        if((*&(((&covar2d_0)->rows + (int(0)))->x) * _S29 - *&(((&covar2d_0)->rows + (int(0)))->y) * *&(((&covar2d_0)->rows + (int(1)))->x)) <= 0.0f)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        float invdet_1 = 1.0f / (covar2d_0.rows[int(0)].x * covar2d_0.rows[int(1)].y - covar2d_0.rows[int(0)].y * covar2d_0.rows[int(1)].x);
        Matrix<float, 2, 2>  _S30 = makeMatrix<float, 2, 2> (covar2d_0.rows[int(1)].y * invdet_1, - covar2d_0.rows[int(0)].y * invdet_1, - covar2d_0.rows[int(1)].x * invdet_1, covar2d_0.rows[int(0)].x * invdet_1);
        *opacity_0 = 1.0f / (1.0f + (F32_exp((- in_opacity_0))));
        if((*opacity_0) < 0.00392156885936856f)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        float extend_0 = (F32_min((3.32999992370605469f), ((F32_sqrt((2.0f * (F32_log((*opacity_0 / 0.00392156885936856f)))))))));
        float radius_x_0 = (F32_ceil((extend_0 * (F32_sqrt((covar2d_0[int(0)].x))))));
        float radius_y_0 = (F32_ceil((extend_0 * (F32_sqrt((covar2d_0[int(1)].y))))));
        if(radius_x_0 <= radius_clip_0)
        {
            _S24 = radius_y_0 <= radius_clip_0;
        }
        else
        {
            _S24 = false;
        }
        if(_S24)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        if(((*mean2d_4).x + radius_x_0) <= 0.0f)
        {
            _S24 = true;
        }
        else
        {
            _S24 = ((*mean2d_4).x - radius_x_0) >= _S25;
        }
        if(_S24)
        {
            _S24 = true;
        }
        else
        {
            _S24 = ((*mean2d_4).y + radius_y_0) <= 0.0f;
        }
        if(_S24)
        {
            _S24 = true;
        }
        else
        {
            _S24 = ((*mean2d_4).y - radius_y_0) >= _S26;
        }
        if(_S24)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        *radii_0 = make_int2 (int(radius_x_0), int(radius_y_0));
        *depth_0 = _S23;
        *conic_0 = make_float3 (_S30.rows[int(0)].x, _S30.rows[int(0)].y, _S30.rows[int(1)].y);
        break;
    }
    return;
}

inline __device__ void projection_3dgs_fisheye(float3  mean_1, float4  quat_3, float3  scale_2, float in_opacity_1, Matrix<float, 3, 3>  R_3, float3  t_2, float fx_5, float fy_5, float cx_5, float cy_5, uint image_width_1, uint image_height_1, float eps2d_2, float near_plane_1, float far_plane_1, float radius_clip_1, int2  * radii_1, float2  * mean2d_5, float * depth_1, float3  * conic_1, float * opacity_1)
{
    for(;;)
    {
        float3  mean_c_1 = mul_0(R_3, mean_1) + t_2;
        float _S31 = mean_c_1.z;
        bool _S32;
        if(_S31 < near_plane_1)
        {
            _S32 = true;
        }
        else
        {
            _S32 = _S31 > far_plane_1;
        }
        if(_S32)
        {
            *radii_1 = make_int2 (int(0), int(0));
            break;
        }
        float x_11 = quat_3.y;
        float inv_norm_3 = (F32_rsqrt((x_11 * x_11 + quat_3.z * quat_3.z + quat_3.w * quat_3.w + quat_3.x * quat_3.x)));
        float x_12 = quat_3.y * inv_norm_3;
        float y_4 = quat_3.z * inv_norm_3;
        float z_3 = quat_3.w * inv_norm_3;
        float w_3 = quat_3.x * inv_norm_3;
        float x2_4 = x_12 * x_12;
        float y2_4 = y_4 * y_4;
        float z2_3 = z_3 * z_3;
        float xy_4 = x_12 * y_4;
        float xz_3 = x_12 * z_3;
        float yz_3 = y_4 * z_3;
        float wx_3 = w_3 * x_12;
        float wy_3 = w_3 * y_4;
        float wz_3 = w_3 * z_3;
        Matrix<float, 3, 3>  M_2 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_4 + z2_3), 2.0f * (xy_4 + wz_3), 2.0f * (xz_3 - wy_3), 2.0f * (xy_4 - wz_3), 1.0f - 2.0f * (x2_4 + z2_3), 2.0f * (yz_3 + wx_3), 2.0f * (xz_3 + wy_3), 2.0f * (yz_3 - wx_3), 1.0f - 2.0f * (x2_4 + y2_4))), makeMatrix<float, 3, 3> (scale_2.x, 0.0f, 0.0f, 0.0f, scale_2.y, 0.0f, 0.0f, 0.0f, scale_2.z));
        Matrix<float, 3, 3>  covar_c_1 = mul_4(mul_4(R_3, mul_4(M_2, transpose_0(M_2))), transpose_0(R_3));
        Matrix<float, 2, 2>  covar2d_1;
        float xy_len_1 = length_0(make_float2 (mean_c_1.x, mean_c_1.y)) + 1.00000001168609742e-07f;
        float theta_1 = (F32_atan2((xy_len_1), (mean_c_1.z + 1.00000001168609742e-07f)));
        *mean2d_5 = make_float2 (mean_c_1.x * fx_5 * theta_1 / xy_len_1 + cx_5, mean_c_1.y * fy_5 * theta_1 / xy_len_1 + cy_5);
        float x2_5 = mean_c_1.x * mean_c_1.x + 1.00000001168609742e-07f;
        float y2_5 = mean_c_1.y * mean_c_1.y;
        float xy_5 = mean_c_1.x * mean_c_1.y;
        float x2y2_1 = x2_5 + y2_5;
        float x2y2z2_inv_1 = 1.0f / (x2y2_1 + mean_c_1.z * mean_c_1.z);
        float b_1 = (F32_atan2((xy_len_1), (mean_c_1.z))) / xy_len_1 / x2y2_1;
        float a_1 = mean_c_1.z * x2y2z2_inv_1 / x2y2_1;
        float _S33 = a_1 - b_1;
        Matrix<float, 2, 3>  J_5 = makeMatrix<float, 2, 3> (fx_5 * (x2_5 * a_1 + y2_5 * b_1), fx_5 * xy_5 * _S33, - fx_5 * mean_c_1.x * x2y2z2_inv_1, fy_5 * xy_5 * _S33, fy_5 * (y2_5 * a_1 + x2_5 * b_1), - fy_5 * mean_c_1.y * x2y2z2_inv_1);
        covar2d_1 = mul_6(mul_5(J_5, covar_c_1), transpose_1(J_5));
        *&(((&covar2d_1)->rows + (int(0)))->x) = *&(((&covar2d_1)->rows + (int(0)))->x) + eps2d_2;
        float _S34 = *&(((&covar2d_1)->rows + (int(1)))->y) + eps2d_2;
        *&(((&covar2d_1)->rows + (int(1)))->y) = _S34;
        if((*&(((&covar2d_1)->rows + (int(0)))->x) * _S34 - *&(((&covar2d_1)->rows + (int(0)))->y) * *&(((&covar2d_1)->rows + (int(1)))->x)) <= 0.0f)
        {
            *radii_1 = make_int2 (int(0), int(0));
            break;
        }
        float invdet_2 = 1.0f / (covar2d_1.rows[int(0)].x * covar2d_1.rows[int(1)].y - covar2d_1.rows[int(0)].y * covar2d_1.rows[int(1)].x);
        Matrix<float, 2, 2>  _S35 = makeMatrix<float, 2, 2> (covar2d_1.rows[int(1)].y * invdet_2, - covar2d_1.rows[int(0)].y * invdet_2, - covar2d_1.rows[int(1)].x * invdet_2, covar2d_1.rows[int(0)].x * invdet_2);
        *opacity_1 = 1.0f / (1.0f + (F32_exp((- in_opacity_1))));
        if((*opacity_1) < 0.00392156885936856f)
        {
            *radii_1 = make_int2 (int(0), int(0));
            break;
        }
        float extend_1 = (F32_min((3.32999992370605469f), ((F32_sqrt((2.0f * (F32_log((*opacity_1 / 0.00392156885936856f)))))))));
        float radius_x_1 = (F32_ceil((extend_1 * (F32_sqrt((covar2d_1[int(0)].x))))));
        float radius_y_1 = (F32_ceil((extend_1 * (F32_sqrt((covar2d_1[int(1)].y))))));
        if(radius_x_1 <= radius_clip_1)
        {
            _S32 = radius_y_1 <= radius_clip_1;
        }
        else
        {
            _S32 = false;
        }
        if(_S32)
        {
            *radii_1 = make_int2 (int(0), int(0));
            break;
        }
        if(((*mean2d_5).x + radius_x_1) <= 0.0f)
        {
            _S32 = true;
        }
        else
        {
            _S32 = ((*mean2d_5).x - radius_x_1) >= float(image_width_1);
        }
        if(_S32)
        {
            _S32 = true;
        }
        else
        {
            _S32 = ((*mean2d_5).y + radius_y_1) <= 0.0f;
        }
        if(_S32)
        {
            _S32 = true;
        }
        else
        {
            _S32 = ((*mean2d_5).y - radius_y_1) >= float(image_height_1);
        }
        if(_S32)
        {
            *radii_1 = make_int2 (int(0), int(0));
            break;
        }
        *radii_1 = make_int2 (int(radius_x_1), int(radius_y_1));
        *depth_1 = _S31;
        *conic_1 = make_float3 (_S35.rows[int(0)].x, _S35.rows[int(0)].y, _S35.rows[int(1)].y);
        break;
    }
    return;
}

inline __device__ void projection_3dgs_ortho(float3  mean_2, float4  quat_4, float3  scale_3, float in_opacity_2, Matrix<float, 3, 3>  R_4, float3  t_3, float fx_6, float fy_6, float cx_6, float cy_6, uint image_width_2, uint image_height_2, float eps2d_3, float near_plane_2, float far_plane_2, float radius_clip_2, int2  * radii_2, float2  * mean2d_6, float * depth_2, float3  * conic_2, float * opacity_2)
{
    for(;;)
    {
        float3  mean_c_2 = mul_0(R_4, mean_2) + t_3;
        float _S36 = mean_c_2.z;
        bool _S37;
        if(_S36 < near_plane_2)
        {
            _S37 = true;
        }
        else
        {
            _S37 = _S36 > far_plane_2;
        }
        if(_S37)
        {
            *radii_2 = make_int2 (int(0), int(0));
            break;
        }
        float x_13 = quat_4.y;
        float inv_norm_4 = (F32_rsqrt((x_13 * x_13 + quat_4.z * quat_4.z + quat_4.w * quat_4.w + quat_4.x * quat_4.x)));
        float x_14 = quat_4.y * inv_norm_4;
        float y_5 = quat_4.z * inv_norm_4;
        float z_4 = quat_4.w * inv_norm_4;
        float w_4 = quat_4.x * inv_norm_4;
        float x2_6 = x_14 * x_14;
        float y2_6 = y_5 * y_5;
        float z2_4 = z_4 * z_4;
        float xy_6 = x_14 * y_5;
        float xz_4 = x_14 * z_4;
        float yz_4 = y_5 * z_4;
        float wx_4 = w_4 * x_14;
        float wy_4 = w_4 * y_5;
        float wz_4 = w_4 * z_4;
        Matrix<float, 3, 3>  M_3 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_6 + z2_4), 2.0f * (xy_6 + wz_4), 2.0f * (xz_4 - wy_4), 2.0f * (xy_6 - wz_4), 1.0f - 2.0f * (x2_6 + z2_4), 2.0f * (yz_4 + wx_4), 2.0f * (xz_4 + wy_4), 2.0f * (yz_4 - wx_4), 1.0f - 2.0f * (x2_6 + y2_6))), makeMatrix<float, 3, 3> (scale_3.x, 0.0f, 0.0f, 0.0f, scale_3.y, 0.0f, 0.0f, 0.0f, scale_3.z));
        Matrix<float, 3, 3>  covar_c_2 = mul_4(mul_4(R_4, mul_4(M_3, transpose_0(M_3))), transpose_0(R_4));
        Matrix<float, 2, 2>  covar2d_2;
        Matrix<float, 2, 3>  J_6 = makeMatrix<float, 2, 3> (fx_6, 0.0f, 0.0f, 0.0f, fy_6, 0.0f);
        covar2d_2 = mul_6(mul_5(J_6, covar_c_2), transpose_1(J_6));
        *mean2d_6 = make_float2 (fx_6 * mean_c_2.x + cx_6, fy_6 * mean_c_2.y + cy_6);
        *&(((&covar2d_2)->rows + (int(0)))->x) = *&(((&covar2d_2)->rows + (int(0)))->x) + eps2d_3;
        float _S38 = *&(((&covar2d_2)->rows + (int(1)))->y) + eps2d_3;
        *&(((&covar2d_2)->rows + (int(1)))->y) = _S38;
        if((*&(((&covar2d_2)->rows + (int(0)))->x) * _S38 - *&(((&covar2d_2)->rows + (int(0)))->y) * *&(((&covar2d_2)->rows + (int(1)))->x)) <= 0.0f)
        {
            *radii_2 = make_int2 (int(0), int(0));
            break;
        }
        float invdet_3 = 1.0f / (covar2d_2.rows[int(0)].x * covar2d_2.rows[int(1)].y - covar2d_2.rows[int(0)].y * covar2d_2.rows[int(1)].x);
        Matrix<float, 2, 2>  _S39 = makeMatrix<float, 2, 2> (covar2d_2.rows[int(1)].y * invdet_3, - covar2d_2.rows[int(0)].y * invdet_3, - covar2d_2.rows[int(1)].x * invdet_3, covar2d_2.rows[int(0)].x * invdet_3);
        *opacity_2 = 1.0f / (1.0f + (F32_exp((- in_opacity_2))));
        if((*opacity_2) < 0.00392156885936856f)
        {
            *radii_2 = make_int2 (int(0), int(0));
            break;
        }
        float extend_2 = (F32_min((3.32999992370605469f), ((F32_sqrt((2.0f * (F32_log((*opacity_2 / 0.00392156885936856f)))))))));
        float radius_x_2 = (F32_ceil((extend_2 * (F32_sqrt((covar2d_2[int(0)].x))))));
        float radius_y_2 = (F32_ceil((extend_2 * (F32_sqrt((covar2d_2[int(1)].y))))));
        if(radius_x_2 <= radius_clip_2)
        {
            _S37 = radius_y_2 <= radius_clip_2;
        }
        else
        {
            _S37 = false;
        }
        if(_S37)
        {
            *radii_2 = make_int2 (int(0), int(0));
            break;
        }
        if(((*mean2d_6).x + radius_x_2) <= 0.0f)
        {
            _S37 = true;
        }
        else
        {
            _S37 = ((*mean2d_6).x - radius_x_2) >= float(image_width_2);
        }
        if(_S37)
        {
            _S37 = true;
        }
        else
        {
            _S37 = ((*mean2d_6).y + radius_y_2) <= 0.0f;
        }
        if(_S37)
        {
            _S37 = true;
        }
        else
        {
            _S37 = ((*mean2d_6).y - radius_y_2) >= float(image_height_2);
        }
        if(_S37)
        {
            *radii_2 = make_int2 (int(0), int(0));
            break;
        }
        *radii_2 = make_int2 (int(radius_x_2), int(radius_y_2));
        *depth_2 = _S36;
        *conic_2 = make_float3 (_S39.rows[int(0)].x, _S39.rows[int(0)].y, _S39.rows[int(1)].y);
        break;
    }
    return;
}

inline __device__ float3  s_primal_ctx_mul_0(Matrix<float, 3, 3>  _S40, float3  _S41)
{
    return mul_0(_S40, _S41);
}

inline __device__ float s_primal_ctx_rsqrt_0(float _S42)
{
    return (F32_rsqrt((_S42)));
}

inline __device__ Matrix<float, 3, 3>  s_primal_ctx_mul_1(Matrix<float, 3, 3>  _S43, Matrix<float, 3, 3>  _S44)
{
    return mul_4(_S43, _S44);
}

inline __device__ float s_primal_ctx_max_0(float _S45, float _S46)
{
    return (F32_max((_S45), (_S46)));
}

inline __device__ float s_primal_ctx_min_0(float _S47, float _S48)
{
    return (F32_min((_S47), (_S48)));
}

inline __device__ Matrix<float, 2, 3>  s_primal_ctx_mul_2(Matrix<float, 2, 3>  _S49, Matrix<float, 3, 3>  _S50)
{
    return mul_5(_S49, _S50);
}

inline __device__ Matrix<float, 2, 2>  s_primal_ctx_mul_3(Matrix<float, 2, 3>  _S51, Matrix<float, 3, 2>  _S52)
{
    return mul_6(_S51, _S52);
}

inline __device__ float s_primal_ctx_exp_0(float _S53)
{
    return (F32_exp((_S53)));
}

inline __device__ float s_primal_ctx_log_0(float _S54)
{
    return (F32_log((_S54)));
}

inline __device__ float s_primal_ctx_sqrt_0(float _S55)
{
    return (F32_sqrt((_S55)));
}

inline __device__ void s_bwd_prop_sqrt_0(DiffPair_float_0 * _S56, float _S57)
{
    _d_sqrt_0(_S56, _S57);
    return;
}

inline __device__ void s_bwd_prop_min_0(DiffPair_float_0 * _S58, DiffPair_float_0 * _S59, float _S60)
{
    _d_min_0(_S58, _S59, _S60);
    return;
}

inline __device__ void s_bwd_prop_log_0(DiffPair_float_0 * _S61, float _S62)
{
    _d_log_0(_S61, _S62);
    return;
}

inline __device__ void s_bwd_prop_exp_0(DiffPair_float_0 * _S63, float _S64)
{
    _d_exp_0(_S63, _S64);
    return;
}

inline __device__ void s_bwd_prop_max_0(DiffPair_float_0 * _S65, DiffPair_float_0 * _S66, float _S67)
{
    _d_max_0(_S65, _S66, _S67);
    return;
}

inline __device__ void s_bwd_prop_mul_0(DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 * _S68, DiffPair_matrixx3Cfloatx2C3x2C2x3E_0 * _S69, Matrix<float, 2, 2>  _S70)
{
    mul_3(_S68, _S69, _S70);
    return;
}

inline __device__ void s_bwd_prop_mul_1(DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 * _S71, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S72, Matrix<float, 2, 3>  _S73)
{
    mul_2(_S71, _S72, _S73);
    return;
}

inline __device__ void s_bwd_prop_mul_2(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S74, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S75, Matrix<float, 3, 3>  _S76)
{
    mul_1(_S74, _S75, _S76);
    return;
}

inline __device__ void s_bwd_prop_rsqrt_0(DiffPair_float_0 * _S77, float _S78)
{
    _d_rsqrt_0(_S77, _S78);
    return;
}

inline __device__ void s_bwd_prop_mul_3(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S79, DiffPair_vectorx3Cfloatx2C3x3E_0 * _S80, float3  _S81)
{
    _d_mul_0(_S79, _S80, _S81);
    return;
}

inline __device__ void projection_3dgs_persp_vjp(float3  mean_3, float4  quat_5, float3  scale_4, float in_opacity_3, Matrix<float, 3, 3>  R_5, float3  t_4, float fx_7, float fy_7, float cx_7, float cy_7, uint image_width_3, uint image_height_3, float eps2d_4, float2  v_mean2d_0, float v_depth_0, float3  v_conic_0, float v_opacity_0, float3  * v_mean_0, float4  * v_quat_0, float3  * v_scale_0, float * v_in_opacity_0, Matrix<float, 3, 3>  * v_R_0, float3  * v_t_0)
{
    float3  mean_c_3 = s_primal_ctx_mul_0(R_5, mean_3) + t_4;
    float _S82 = quat_5.y;
    float _S83 = _S82 * _S82 + quat_5.z * quat_5.z + quat_5.w * quat_5.w + quat_5.x * quat_5.x;
    float _S84 = s_primal_ctx_rsqrt_0(_S83);
    float x_15 = quat_5.y * _S84;
    float y_6 = quat_5.z * _S84;
    float z_5 = quat_5.w * _S84;
    float w_5 = quat_5.x * _S84;
    float x2_7 = x_15 * x_15;
    float y2_7 = y_6 * y_6;
    float z2_5 = z_5 * z_5;
    float xy_7 = x_15 * y_6;
    float xz_5 = x_15 * z_5;
    float yz_5 = y_6 * z_5;
    float wx_5 = w_5 * x_15;
    float wy_5 = w_5 * y_6;
    float wz_5 = w_5 * z_5;
    Matrix<float, 3, 3>  _S85 = transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_7 + z2_5), 2.0f * (xy_7 + wz_5), 2.0f * (xz_5 - wy_5), 2.0f * (xy_7 - wz_5), 1.0f - 2.0f * (x2_7 + z2_5), 2.0f * (yz_5 + wx_5), 2.0f * (xz_5 + wy_5), 2.0f * (yz_5 - wx_5), 1.0f - 2.0f * (x2_7 + y2_7)));
    Matrix<float, 3, 3>  S_0 = makeMatrix<float, 3, 3> (scale_4.x, 0.0f, 0.0f, 0.0f, scale_4.y, 0.0f, 0.0f, 0.0f, scale_4.z);
    Matrix<float, 3, 3>  _S86 = s_primal_ctx_mul_1(_S85, S_0);
    Matrix<float, 3, 3>  _S87 = transpose_0(_S86);
    Matrix<float, 3, 3>  _S88 = s_primal_ctx_mul_1(_S86, _S87);
    Matrix<float, 3, 3>  _S89 = s_primal_ctx_mul_1(R_5, _S88);
    Matrix<float, 3, 3>  _S90 = transpose_0(R_5);
    Matrix<float, 3, 3>  _S91 = s_primal_ctx_mul_1(_S89, _S90);
    float _S92 = float(image_width_3);
    float _S93 = float(image_height_3);
    float _S94 = 0.30000001192092896f * (0.5f * _S92 / fx_7);
    float lim_x_pos_0 = (_S92 - cx_7) / fx_7 + _S94;
    float _S95 = 0.30000001192092896f * (0.5f * _S93 / fy_7);
    float lim_y_pos_0 = (_S93 - cy_7) / fy_7 + _S95;
    float rz_3 = 1.0f / mean_c_3.z;
    float _S96 = mean_c_3.z * mean_c_3.z;
    float rz2_3 = rz_3 * rz_3;
    float _S97 = - (cx_7 / fx_7 + _S94);
    float _S98 = mean_c_3.x * rz_3;
    float _S99 = s_primal_ctx_max_0(_S97, _S98);
    float _S100 = s_primal_ctx_min_0(lim_x_pos_0, _S99);
    float _S101 = - (cy_7 / fy_7 + _S95);
    float _S102 = mean_c_3.y * rz_3;
    float _S103 = s_primal_ctx_max_0(_S101, _S102);
    float _S104 = s_primal_ctx_min_0(lim_y_pos_0, _S103);
    float _S105 = - fx_7;
    float _S106 = _S105 * (mean_c_3.z * _S100);
    float _S107 = - fy_7;
    float _S108 = _S107 * (mean_c_3.z * _S104);
    Matrix<float, 2, 3>  J_7 = makeMatrix<float, 2, 3> (fx_7 * rz_3, 0.0f, _S106 * rz2_3, 0.0f, fy_7 * rz_3, _S108 * rz2_3);
    Matrix<float, 2, 3>  _S109 = s_primal_ctx_mul_2(J_7, _S91);
    Matrix<float, 3, 2>  _S110 = transpose_1(J_7);
    Matrix<float, 2, 2>  _S111 = s_primal_ctx_mul_3(_S109, _S110);
    float _S112 = fx_7 * mean_c_3.x;
    float _S113 = fy_7 * mean_c_3.y;
    float _S114 = _S111.rows[int(0)].y * _S111.rows[int(1)].x;
    float det_orig_1 = _S111.rows[int(0)].x * _S111.rows[int(1)].y - _S114;
    float _S115 = _S111.rows[int(0)].x + eps2d_4;
    Matrix<float, 2, 2>  _S116 = _S111;
    *&(((&_S116)->rows + (int(0)))->x) = _S115;
    float _S117 = _S111.rows[int(1)].y + eps2d_4;
    *&(((&_S116)->rows + (int(1)))->y) = _S117;
    float det_blur_1 = _S115 * _S117 - _S114;
    float _S118 = det_orig_1 / det_blur_1;
    float _S119 = det_blur_1 * det_blur_1;
    float _S120 = s_primal_ctx_max_0(0.0f, _S118);
    float invdet_4 = 1.0f / det_blur_1;
    float _S121 = - _S111.rows[int(0)].y;
    float _S122 = - _S111.rows[int(1)].x;
    float _S123 = - in_opacity_3;
    float _S124 = 1.0f + s_primal_ctx_exp_0(_S123);
    float _S125 = _S124 * _S124;
    float _S126 = 1.0f / _S124 / 0.00392156885936856f;
    float _S127 = 2.0f * s_primal_ctx_log_0(_S126);
    float _S128 = s_primal_ctx_sqrt_0(_S127);
    float _S129 = _S116.rows[int(0)].x;
    float _S130 = _S116.rows[int(1)].y;
    float2  _S131 = make_float2 (0.0f);
    float2  _S132 = _S131;
    *&((&_S132)->y) = v_conic_0.z;
    float2  _S133 = _S131;
    *&((&_S133)->y) = v_conic_0.y;
    *&((&_S133)->x) = v_conic_0.x;
    DiffPair_float_0 _S134;
    (&_S134)->primal_0 = _S130;
    (&_S134)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S134, 0.0f);
    DiffPair_float_0 _S135;
    (&_S135)->primal_0 = _S129;
    (&_S135)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S135, 0.0f);
    DiffPair_float_0 _S136;
    (&_S136)->primal_0 = 3.32999992370605469f;
    (&_S136)->differential_0 = 0.0f;
    DiffPair_float_0 _S137;
    (&_S137)->primal_0 = _S128;
    (&_S137)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S136, &_S137, 0.0f);
    DiffPair_float_0 _S138;
    (&_S138)->primal_0 = _S127;
    (&_S138)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S138, _S137.differential_0);
    float _S139 = 2.0f * _S138.differential_0;
    DiffPair_float_0 _S140;
    (&_S140)->primal_0 = _S126;
    (&_S140)->differential_0 = 0.0f;
    s_bwd_prop_log_0(&_S140, _S139);
    float _S141 = - ((v_opacity_0 + 254.9999847412109375f * _S140.differential_0) / _S125);
    DiffPair_float_0 _S142;
    (&_S142)->primal_0 = _S123;
    (&_S142)->differential_0 = 0.0f;
    s_bwd_prop_exp_0(&_S142, _S141);
    float _S143 = - _S142.differential_0;
    Matrix<float, 2, 2>  _S144 = makeMatrix<float, 2, 2> (0.0f);
    Matrix<float, 2, 2>  _S145 = _S144;
    _S145[int(1)] = _S132;
    _S145[int(0)] = _S133;
    float _S146 = invdet_4 * _S145.rows[int(1)].y;
    float _S147 = - (invdet_4 * _S145.rows[int(1)].x);
    float _S148 = - (invdet_4 * _S145.rows[int(0)].y);
    float _S149 = invdet_4 * _S145.rows[int(0)].x;
    float _S150 = - ((_S115 * _S145.rows[int(1)].y + _S122 * _S145.rows[int(1)].x + _S121 * _S145.rows[int(0)].y + _S117 * _S145.rows[int(0)].x) / _S119);
    DiffPair_float_0 _S151;
    (&_S151)->primal_0 = _S120;
    (&_S151)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S151, 0.0f);
    DiffPair_float_0 _S152;
    (&_S152)->primal_0 = 0.0f;
    (&_S152)->differential_0 = 0.0f;
    DiffPair_float_0 _S153;
    (&_S153)->primal_0 = _S118;
    (&_S153)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S152, &_S153, _S151.differential_0);
    float _S154 = _S153.differential_0 / _S119;
    float s_diff_det_orig_T_0 = det_blur_1 * _S154;
    float _S155 = _S150 + det_orig_1 * - _S154;
    float _S156 = - _S155;
    float _S157 = _S115 * _S155;
    float _S158 = _S117 * _S155;
    float2  _S159 = make_float2 (0.0f, _S134.differential_0);
    float2  _S160 = make_float2 (_S135.differential_0, 0.0f);
    Matrix<float, 2, 2>  _S161 = _S144;
    _S161[int(1)] = _S159;
    _S161[int(0)] = _S160;
    _S116 = _S161;
    *&(((&_S116)->rows + (int(1)))->y) = 0.0f;
    float _S162 = _S149 + _S157 + _S161.rows[int(1)].y;
    *&(((&_S116)->rows + (int(0)))->x) = 0.0f;
    float _S163 = _S146 + _S158 + _S161.rows[int(0)].x;
    float _S164 = _S156 + - s_diff_det_orig_T_0;
    float _S165 = _S147 + _S111.rows[int(0)].y * _S164;
    float _S166 = _S148 + _S111.rows[int(1)].x * _S164;
    float _S167 = _S111.rows[int(1)].y * s_diff_det_orig_T_0;
    float _S168 = _S162 + _S111.rows[int(0)].x * s_diff_det_orig_T_0;
    float2  _S169 = _S131;
    *&((&_S169)->x) = _S165;
    *&((&_S169)->y) = _S168;
    float _S170 = _S163 + _S167;
    float2  _S171 = _S131;
    *&((&_S171)->y) = _S166;
    *&((&_S171)->x) = _S170;
    float _S172 = _S113 * v_mean2d_0.y;
    float _S173 = fy_7 * (rz_3 * v_mean2d_0.y);
    float _S174 = _S112 * v_mean2d_0.x;
    float _S175 = fx_7 * (rz_3 * v_mean2d_0.x);
    Matrix<float, 2, 2>  _S176 = _S144;
    _S176[int(1)] = _S169;
    _S176[int(0)] = _S171;
    Matrix<float, 2, 2>  _S177 = _S116 + _S176;
    Matrix<float, 2, 3>  _S178 = makeMatrix<float, 2, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S179;
    (&_S179)->primal_0 = _S109;
    (&_S179)->differential_0 = _S178;
    Matrix<float, 3, 2>  _S180 = makeMatrix<float, 3, 2> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C2x3E_0 _S181;
    (&_S181)->primal_0 = _S110;
    (&_S181)->differential_0 = _S180;
    s_bwd_prop_mul_0(&_S179, &_S181, _S177);
    Matrix<float, 2, 3>  _S182 = transpose_2(_S181.differential_0);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S183;
    (&_S183)->primal_0 = J_7;
    (&_S183)->differential_0 = _S178;
    Matrix<float, 3, 3>  _S184 = makeMatrix<float, 3, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S185;
    (&_S185)->primal_0 = _S91;
    (&_S185)->differential_0 = _S184;
    s_bwd_prop_mul_1(&_S183, &_S185, _S179.differential_0);
    Matrix<float, 2, 3>  _S186 = _S182 + _S183.differential_0;
    float _S187 = _S108 * _S186.rows[int(1)].z;
    float s_diff_ty_T_0 = _S107 * (rz2_3 * _S186.rows[int(1)].z);
    float _S188 = fy_7 * _S186.rows[int(1)].y;
    float _S189 = _S106 * _S186.rows[int(0)].z;
    float s_diff_tx_T_0 = _S105 * (rz2_3 * _S186.rows[int(0)].z);
    float _S190 = fx_7 * _S186.rows[int(0)].x;
    float _S191 = mean_c_3.z * s_diff_ty_T_0;
    float _S192 = _S104 * s_diff_ty_T_0;
    DiffPair_float_0 _S193;
    (&_S193)->primal_0 = lim_y_pos_0;
    (&_S193)->differential_0 = 0.0f;
    DiffPair_float_0 _S194;
    (&_S194)->primal_0 = _S103;
    (&_S194)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S193, &_S194, _S191);
    DiffPair_float_0 _S195;
    (&_S195)->primal_0 = _S101;
    (&_S195)->differential_0 = 0.0f;
    DiffPair_float_0 _S196;
    (&_S196)->primal_0 = _S102;
    (&_S196)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S195, &_S196, _S194.differential_0);
    float _S197 = mean_c_3.y * _S196.differential_0;
    float _S198 = rz_3 * _S196.differential_0;
    float _S199 = mean_c_3.z * s_diff_tx_T_0;
    float _S200 = _S100 * s_diff_tx_T_0;
    DiffPair_float_0 _S201;
    (&_S201)->primal_0 = lim_x_pos_0;
    (&_S201)->differential_0 = 0.0f;
    DiffPair_float_0 _S202;
    (&_S202)->primal_0 = _S99;
    (&_S202)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S201, &_S202, _S199);
    DiffPair_float_0 _S203;
    (&_S203)->primal_0 = _S97;
    (&_S203)->differential_0 = 0.0f;
    DiffPair_float_0 _S204;
    (&_S204)->primal_0 = _S98;
    (&_S204)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S203, &_S204, _S202.differential_0);
    float _S205 = rz_3 * (_S187 + _S189);
    float _S206 = _S192 + _S200 + - ((_S172 + _S174 + _S188 + _S190 + _S197 + mean_c_3.x * _S204.differential_0 + _S205 + _S205) / _S96);
    float _S207 = _S173 + _S198;
    float _S208 = _S175 + rz_3 * _S204.differential_0;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S209;
    (&_S209)->primal_0 = _S89;
    (&_S209)->differential_0 = _S184;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S210;
    (&_S210)->primal_0 = _S90;
    (&_S210)->differential_0 = _S184;
    s_bwd_prop_mul_2(&_S209, &_S210, _S185.differential_0);
    Matrix<float, 3, 3>  _S211 = transpose_0(_S210.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S212;
    (&_S212)->primal_0 = R_5;
    (&_S212)->differential_0 = _S184;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S213;
    (&_S213)->primal_0 = _S88;
    (&_S213)->differential_0 = _S184;
    s_bwd_prop_mul_2(&_S212, &_S213, _S209.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S214;
    (&_S214)->primal_0 = _S86;
    (&_S214)->differential_0 = _S184;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S215;
    (&_S215)->primal_0 = _S87;
    (&_S215)->differential_0 = _S184;
    s_bwd_prop_mul_2(&_S214, &_S215, _S213.differential_0);
    Matrix<float, 3, 3>  _S216 = _S214.differential_0 + transpose_0(_S215.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S217;
    (&_S217)->primal_0 = _S85;
    (&_S217)->differential_0 = _S184;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S218;
    (&_S218)->primal_0 = S_0;
    (&_S218)->differential_0 = _S184;
    s_bwd_prop_mul_2(&_S217, &_S218, _S216);
    Matrix<float, 3, 3>  _S219 = transpose_0(_S217.differential_0);
    float _S220 = 2.0f * - _S219.rows[int(2)].z;
    float _S221 = 2.0f * _S219.rows[int(2)].y;
    float _S222 = 2.0f * _S219.rows[int(2)].x;
    float _S223 = 2.0f * _S219.rows[int(1)].z;
    float _S224 = 2.0f * - _S219.rows[int(1)].y;
    float _S225 = 2.0f * _S219.rows[int(1)].x;
    float _S226 = 2.0f * _S219.rows[int(0)].z;
    float _S227 = 2.0f * _S219.rows[int(0)].y;
    float _S228 = 2.0f * - _S219.rows[int(0)].x;
    float _S229 = - _S225 + _S227;
    float _S230 = _S222 + - _S226;
    float _S231 = - _S221 + _S223;
    float _S232 = _S221 + _S223;
    float _S233 = _S222 + _S226;
    float _S234 = _S225 + _S227;
    float _S235 = z_5 * (_S224 + _S228);
    float _S236 = y_6 * (_S220 + _S228);
    float _S237 = x_15 * (_S220 + _S224);
    float _S238 = z_5 * _S229 + y_6 * _S230 + x_15 * _S231;
    float _S239 = _S84 * _S238;
    float _S240 = w_5 * _S229 + y_6 * _S232 + x_15 * _S233 + _S235 + _S235;
    float _S241 = _S84 * _S240;
    float _S242 = w_5 * _S230 + z_5 * _S232 + x_15 * _S234 + _S236 + _S236;
    float _S243 = _S84 * _S242;
    float _S244 = w_5 * _S231 + z_5 * _S233 + y_6 * _S234 + _S237 + _S237;
    float _S245 = _S84 * _S244;
    float _S246 = quat_5.x * _S238 + quat_5.w * _S240 + quat_5.z * _S242 + quat_5.y * _S244;
    DiffPair_float_0 _S247;
    (&_S247)->primal_0 = _S83;
    (&_S247)->differential_0 = 0.0f;
    s_bwd_prop_rsqrt_0(&_S247, _S246);
    float _S248 = quat_5.x * _S247.differential_0;
    float _S249 = quat_5.w * _S247.differential_0;
    float _S250 = quat_5.z * _S247.differential_0;
    float _S251 = quat_5.y * _S247.differential_0;
    float _S252 = _S241 + _S249 + _S249;
    float _S253 = _S243 + _S250 + _S250;
    float _S254 = _S245 + _S251 + _S251;
    float _S255 = _S239 + _S248 + _S248;
    float3  _S256 = make_float3 (0.0f, 0.0f, v_depth_0);
    float3  _S257 = make_float3 (0.0f);
    float3  _S258 = _S257;
    *&((&_S258)->z) = _S206;
    *&((&_S258)->y) = _S207;
    *&((&_S258)->x) = _S208;
    float3  _S259 = _S256 + _S258;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S260;
    (&_S260)->primal_0 = R_5;
    (&_S260)->differential_0 = _S184;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S261;
    (&_S261)->primal_0 = mean_3;
    (&_S261)->differential_0 = _S257;
    s_bwd_prop_mul_3(&_S260, &_S261, _S259);
    Matrix<float, 3, 3>  _S262 = _S211 + _S212.differential_0 + _S260.differential_0;
    float3  _S263 = _S257;
    *&((&_S263)->z) = _S218.differential_0.rows[int(2)].z;
    *&((&_S263)->y) = _S218.differential_0.rows[int(1)].y;
    *&((&_S263)->x) = _S218.differential_0.rows[int(0)].x;
    float4  _S264 = make_float4 (0.0f);
    *&((&_S264)->w) = _S252;
    *&((&_S264)->z) = _S253;
    *&((&_S264)->y) = _S254;
    *&((&_S264)->x) = _S255;
    *v_mean_0 = _S261.differential_0;
    *v_quat_0 = _S264;
    *v_scale_0 = _S263;
    *v_in_opacity_0 = _S143;
    *v_R_0 = _S262;
    *v_t_0 = _S259;
    return;
}

inline __device__ float s_primal_ctx_atan2_0(float _S265, float _S266)
{
    return (F32_atan2((_S265), (_S266)));
}

inline __device__ void s_bwd_prop_atan2_0(DiffPair_float_0 * _S267, DiffPair_float_0 * _S268, float _S269)
{
    _d_atan2_0(_S267, _S268, _S269);
    return;
}

struct DiffPair_vectorx3Cfloatx2C2x3E_0
{
    float2  primal_0;
    float2  differential_0;
};

inline __device__ void s_bwd_prop_length_impl_0(DiffPair_vectorx3Cfloatx2C2x3E_0 * dpx_7, float _s_dOut_0)
{
    float _S270 = (*dpx_7).primal_0.x;
    float _S271 = (*dpx_7).primal_0.y;
    DiffPair_float_0 _S272;
    (&_S272)->primal_0 = _S270 * _S270 + _S271 * _S271;
    (&_S272)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S272, _s_dOut_0);
    float _S273 = (*dpx_7).primal_0.y * _S272.differential_0;
    float _S274 = _S273 + _S273;
    float _S275 = (*dpx_7).primal_0.x * _S272.differential_0;
    float _S276 = _S275 + _S275;
    float2  _S277 = make_float2 (0.0f);
    *&((&_S277)->y) = _S274;
    *&((&_S277)->x) = _S276;
    dpx_7->primal_0 = (*dpx_7).primal_0;
    dpx_7->differential_0 = _S277;
    return;
}

inline __device__ void s_bwd_length_impl_0(DiffPair_vectorx3Cfloatx2C2x3E_0 * _S278, float _S279)
{
    s_bwd_prop_length_impl_0(_S278, _S279);
    return;
}

inline __device__ void projection_3dgs_fisheye_vjp(float3  mean_4, float4  quat_6, float3  scale_5, float in_opacity_4, Matrix<float, 3, 3>  R_6, float3  t_5, float fx_8, float fy_8, float cx_8, float cy_8, uint image_width_4, uint image_height_4, float eps2d_5, float2  v_mean2d_1, float v_depth_1, float3  v_conic_1, float v_opacity_1, float3  * v_mean_1, float4  * v_quat_1, float3  * v_scale_1, float * v_in_opacity_1, Matrix<float, 3, 3>  * v_R_1, float3  * v_t_1)
{
    float3  mean_c_4 = s_primal_ctx_mul_0(R_6, mean_4) + t_5;
    float _S280 = quat_6.y;
    float _S281 = _S280 * _S280 + quat_6.z * quat_6.z + quat_6.w * quat_6.w + quat_6.x * quat_6.x;
    float _S282 = s_primal_ctx_rsqrt_0(_S281);
    float x_16 = quat_6.y * _S282;
    float y_7 = quat_6.z * _S282;
    float z_6 = quat_6.w * _S282;
    float w_6 = quat_6.x * _S282;
    float x2_8 = x_16 * x_16;
    float y2_8 = y_7 * y_7;
    float z2_6 = z_6 * z_6;
    float xy_8 = x_16 * y_7;
    float xz_6 = x_16 * z_6;
    float yz_6 = y_7 * z_6;
    float wx_6 = w_6 * x_16;
    float wy_6 = w_6 * y_7;
    float wz_6 = w_6 * z_6;
    Matrix<float, 3, 3>  _S283 = transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_8 + z2_6), 2.0f * (xy_8 + wz_6), 2.0f * (xz_6 - wy_6), 2.0f * (xy_8 - wz_6), 1.0f - 2.0f * (x2_8 + z2_6), 2.0f * (yz_6 + wx_6), 2.0f * (xz_6 + wy_6), 2.0f * (yz_6 - wx_6), 1.0f - 2.0f * (x2_8 + y2_8)));
    Matrix<float, 3, 3>  S_1 = makeMatrix<float, 3, 3> (scale_5.x, 0.0f, 0.0f, 0.0f, scale_5.y, 0.0f, 0.0f, 0.0f, scale_5.z);
    Matrix<float, 3, 3>  _S284 = s_primal_ctx_mul_1(_S283, S_1);
    Matrix<float, 3, 3>  _S285 = transpose_0(_S284);
    Matrix<float, 3, 3>  _S286 = s_primal_ctx_mul_1(_S284, _S285);
    Matrix<float, 3, 3>  _S287 = s_primal_ctx_mul_1(R_6, _S286);
    Matrix<float, 3, 3>  _S288 = transpose_0(R_6);
    Matrix<float, 3, 3>  _S289 = s_primal_ctx_mul_1(_S287, _S288);
    float2  _S290 = make_float2 (mean_c_4.x, mean_c_4.y);
    float xy_len_2 = length_0(_S290) + 1.00000001168609742e-07f;
    float _S291 = mean_c_4.z + 1.00000001168609742e-07f;
    float _S292 = s_primal_ctx_atan2_0(xy_len_2, _S291);
    float _S293 = mean_c_4.x * fx_8;
    float _S294 = _S293 * _S292;
    float _S295 = xy_len_2 * xy_len_2;
    float _S296 = mean_c_4.y * fy_8;
    float _S297 = _S296 * _S292;
    float x2_9 = mean_c_4.x * mean_c_4.x + 1.00000001168609742e-07f;
    float y2_9 = mean_c_4.y * mean_c_4.y;
    float xy_9 = mean_c_4.x * mean_c_4.y;
    float x2y2_2 = x2_9 + y2_9;
    float _S298 = x2y2_2 + mean_c_4.z * mean_c_4.z;
    float x2y2z2_inv_2 = 1.0f / _S298;
    float _S299 = _S298 * _S298;
    float _S300 = s_primal_ctx_atan2_0(xy_len_2, mean_c_4.z);
    float _S301 = _S300 / xy_len_2;
    float b_2 = _S301 / x2y2_2;
    float _S302 = x2y2_2 * x2y2_2;
    float _S303 = mean_c_4.z * x2y2z2_inv_2;
    float a_2 = _S303 / x2y2_2;
    float _S304 = fx_8 * xy_9;
    float _S305 = a_2 - b_2;
    float _S306 = - fx_8;
    float _S307 = _S306 * mean_c_4.x;
    float _S308 = fy_8 * xy_9;
    float _S309 = - fy_8;
    float _S310 = _S309 * mean_c_4.y;
    Matrix<float, 2, 3>  J_8 = makeMatrix<float, 2, 3> (fx_8 * (x2_9 * a_2 + y2_9 * b_2), _S304 * _S305, _S307 * x2y2z2_inv_2, _S308 * _S305, fy_8 * (y2_9 * a_2 + x2_9 * b_2), _S310 * x2y2z2_inv_2);
    Matrix<float, 2, 3>  _S311 = s_primal_ctx_mul_2(J_8, _S289);
    Matrix<float, 3, 2>  _S312 = transpose_1(J_8);
    Matrix<float, 2, 2>  _S313 = s_primal_ctx_mul_3(_S311, _S312);
    float _S314 = _S313.rows[int(0)].y * _S313.rows[int(1)].x;
    float det_orig_2 = _S313.rows[int(0)].x * _S313.rows[int(1)].y - _S314;
    float _S315 = _S313.rows[int(0)].x + eps2d_5;
    Matrix<float, 2, 2>  _S316 = _S313;
    *&(((&_S316)->rows + (int(0)))->x) = _S315;
    float _S317 = _S313.rows[int(1)].y + eps2d_5;
    *&(((&_S316)->rows + (int(1)))->y) = _S317;
    float det_blur_2 = _S315 * _S317 - _S314;
    float _S318 = det_orig_2 / det_blur_2;
    float _S319 = det_blur_2 * det_blur_2;
    float _S320 = s_primal_ctx_max_0(0.0f, _S318);
    float invdet_5 = 1.0f / det_blur_2;
    float _S321 = - _S313.rows[int(0)].y;
    float _S322 = - _S313.rows[int(1)].x;
    float _S323 = - in_opacity_4;
    float _S324 = 1.0f + s_primal_ctx_exp_0(_S323);
    float _S325 = _S324 * _S324;
    float _S326 = 1.0f / _S324 / 0.00392156885936856f;
    float _S327 = 2.0f * s_primal_ctx_log_0(_S326);
    float _S328 = s_primal_ctx_sqrt_0(_S327);
    float _S329 = _S316.rows[int(0)].x;
    float _S330 = _S316.rows[int(1)].y;
    float2  _S331 = make_float2 (0.0f);
    float2  _S332 = _S331;
    *&((&_S332)->y) = v_conic_1.z;
    float2  _S333 = _S331;
    *&((&_S333)->y) = v_conic_1.y;
    *&((&_S333)->x) = v_conic_1.x;
    DiffPair_float_0 _S334;
    (&_S334)->primal_0 = _S330;
    (&_S334)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S334, 0.0f);
    DiffPair_float_0 _S335;
    (&_S335)->primal_0 = _S329;
    (&_S335)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S335, 0.0f);
    DiffPair_float_0 _S336;
    (&_S336)->primal_0 = 3.32999992370605469f;
    (&_S336)->differential_0 = 0.0f;
    DiffPair_float_0 _S337;
    (&_S337)->primal_0 = _S328;
    (&_S337)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S336, &_S337, 0.0f);
    DiffPair_float_0 _S338;
    (&_S338)->primal_0 = _S327;
    (&_S338)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S338, _S337.differential_0);
    float _S339 = 2.0f * _S338.differential_0;
    DiffPair_float_0 _S340;
    (&_S340)->primal_0 = _S326;
    (&_S340)->differential_0 = 0.0f;
    s_bwd_prop_log_0(&_S340, _S339);
    float _S341 = - ((v_opacity_1 + 254.9999847412109375f * _S340.differential_0) / _S325);
    DiffPair_float_0 _S342;
    (&_S342)->primal_0 = _S323;
    (&_S342)->differential_0 = 0.0f;
    s_bwd_prop_exp_0(&_S342, _S341);
    float _S343 = - _S342.differential_0;
    Matrix<float, 2, 2>  _S344 = makeMatrix<float, 2, 2> (0.0f);
    Matrix<float, 2, 2>  _S345 = _S344;
    _S345[int(1)] = _S332;
    _S345[int(0)] = _S333;
    float _S346 = invdet_5 * _S345.rows[int(1)].y;
    float _S347 = - (invdet_5 * _S345.rows[int(1)].x);
    float _S348 = - (invdet_5 * _S345.rows[int(0)].y);
    float _S349 = invdet_5 * _S345.rows[int(0)].x;
    float _S350 = - ((_S315 * _S345.rows[int(1)].y + _S322 * _S345.rows[int(1)].x + _S321 * _S345.rows[int(0)].y + _S317 * _S345.rows[int(0)].x) / _S319);
    DiffPair_float_0 _S351;
    (&_S351)->primal_0 = _S320;
    (&_S351)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S351, 0.0f);
    DiffPair_float_0 _S352;
    (&_S352)->primal_0 = 0.0f;
    (&_S352)->differential_0 = 0.0f;
    DiffPair_float_0 _S353;
    (&_S353)->primal_0 = _S318;
    (&_S353)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S352, &_S353, _S351.differential_0);
    float _S354 = _S353.differential_0 / _S319;
    float s_diff_det_orig_T_1 = det_blur_2 * _S354;
    float _S355 = _S350 + det_orig_2 * - _S354;
    float _S356 = - _S355;
    float _S357 = _S315 * _S355;
    float _S358 = _S317 * _S355;
    float2  _S359 = make_float2 (0.0f, _S334.differential_0);
    float2  _S360 = make_float2 (_S335.differential_0, 0.0f);
    Matrix<float, 2, 2>  _S361 = _S344;
    _S361[int(1)] = _S359;
    _S361[int(0)] = _S360;
    _S316 = _S361;
    *&(((&_S316)->rows + (int(1)))->y) = 0.0f;
    float _S362 = _S349 + _S357 + _S361.rows[int(1)].y;
    *&(((&_S316)->rows + (int(0)))->x) = 0.0f;
    float _S363 = _S346 + _S358 + _S361.rows[int(0)].x;
    float _S364 = _S356 + - s_diff_det_orig_T_1;
    float _S365 = _S347 + _S313.rows[int(0)].y * _S364;
    float _S366 = _S348 + _S313.rows[int(1)].x * _S364;
    float _S367 = _S313.rows[int(1)].y * s_diff_det_orig_T_1;
    float _S368 = _S362 + _S313.rows[int(0)].x * s_diff_det_orig_T_1;
    float2  _S369 = _S331;
    *&((&_S369)->x) = _S365;
    *&((&_S369)->y) = _S368;
    float _S370 = _S363 + _S367;
    float2  _S371 = _S331;
    *&((&_S371)->y) = _S366;
    *&((&_S371)->x) = _S370;
    Matrix<float, 2, 2>  _S372 = _S344;
    _S372[int(1)] = _S369;
    _S372[int(0)] = _S371;
    Matrix<float, 2, 2>  _S373 = _S316 + _S372;
    Matrix<float, 2, 3>  _S374 = makeMatrix<float, 2, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S375;
    (&_S375)->primal_0 = _S311;
    (&_S375)->differential_0 = _S374;
    Matrix<float, 3, 2>  _S376 = makeMatrix<float, 3, 2> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C2x3E_0 _S377;
    (&_S377)->primal_0 = _S312;
    (&_S377)->differential_0 = _S376;
    s_bwd_prop_mul_0(&_S375, &_S377, _S373);
    Matrix<float, 2, 3>  _S378 = transpose_2(_S377.differential_0);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S379;
    (&_S379)->primal_0 = J_8;
    (&_S379)->differential_0 = _S374;
    Matrix<float, 3, 3>  _S380 = makeMatrix<float, 3, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S381;
    (&_S381)->primal_0 = _S289;
    (&_S381)->differential_0 = _S380;
    s_bwd_prop_mul_1(&_S379, &_S381, _S375.differential_0);
    Matrix<float, 2, 3>  _S382 = _S378 + _S379.differential_0;
    float _S383 = _S310 * _S382.rows[int(1)].z;
    float _S384 = _S309 * (x2y2z2_inv_2 * _S382.rows[int(1)].z);
    float _S385 = fy_8 * _S382.rows[int(1)].y;
    float _S386 = b_2 * _S385;
    float _S387 = a_2 * _S385;
    float _S388 = fy_8 * (_S305 * _S382.rows[int(1)].x);
    float _S389 = _S307 * _S382.rows[int(0)].z;
    float _S390 = _S306 * (x2y2z2_inv_2 * _S382.rows[int(0)].z);
    float _S391 = _S308 * _S382.rows[int(1)].x + _S304 * _S382.rows[int(0)].y;
    float _S392 = fx_8 * (_S305 * _S382.rows[int(0)].y);
    float _S393 = fx_8 * _S382.rows[int(0)].x;
    float _S394 = b_2 * _S393;
    float _S395 = a_2 * _S393;
    float _S396 = (y2_9 * _S385 + _S391 + x2_9 * _S393) / _S302;
    float _S397 = _S303 * - _S396;
    float _S398 = x2y2_2 * _S396;
    float _S399 = mean_c_4.z * _S398;
    float _S400 = x2y2z2_inv_2 * _S398;
    float _S401 = (x2_9 * _S385 + - _S391 + y2_9 * _S393) / _S302;
    float _S402 = _S301 * - _S401;
    float _S403 = x2y2_2 * _S401 / _S295;
    float _S404 = _S300 * - _S403;
    float _S405 = xy_len_2 * _S403;
    DiffPair_float_0 _S406;
    (&_S406)->primal_0 = xy_len_2;
    (&_S406)->differential_0 = 0.0f;
    DiffPair_float_0 _S407;
    (&_S407)->primal_0 = mean_c_4.z;
    (&_S407)->differential_0 = 0.0f;
    s_bwd_prop_atan2_0(&_S406, &_S407, _S405);
    float _S408 = - ((_S383 + _S389 + _S399) / _S299);
    float _S409 = mean_c_4.z * _S408;
    float _S410 = _S397 + _S402 + _S408;
    float _S411 = _S388 + _S392;
    float _S412 = mean_c_4.x * _S411;
    float _S413 = mean_c_4.y * _S411;
    float _S414 = mean_c_4.y * (_S387 + _S394 + _S410);
    float _S415 = mean_c_4.x * (_S386 + _S395 + _S410);
    float _S416 = v_mean2d_1.y / _S295;
    float _S417 = _S297 * - _S416;
    float _S418 = xy_len_2 * _S416;
    float _S419 = fy_8 * (_S292 * _S418);
    float _S420 = v_mean2d_1.x / _S295;
    float _S421 = _S294 * - _S420;
    float _S422 = xy_len_2 * _S420;
    float _S423 = fx_8 * (_S292 * _S422);
    float _S424 = _S296 * _S418 + _S293 * _S422;
    DiffPair_float_0 _S425;
    (&_S425)->primal_0 = xy_len_2;
    (&_S425)->differential_0 = 0.0f;
    DiffPair_float_0 _S426;
    (&_S426)->primal_0 = _S291;
    (&_S426)->differential_0 = 0.0f;
    s_bwd_prop_atan2_0(&_S425, &_S426, _S424);
    float _S427 = _S404 + _S406.differential_0 + _S417 + _S421 + _S425.differential_0;
    DiffPair_vectorx3Cfloatx2C2x3E_0 _S428;
    (&_S428)->primal_0 = _S290;
    (&_S428)->differential_0 = _S331;
    s_bwd_length_impl_0(&_S428, _S427);
    float _S429 = _S400 + _S407.differential_0 + _S409 + _S409 + _S426.differential_0;
    float _S430 = _S384 + _S412 + _S414 + _S414 + _S419 + _S428.differential_0.y;
    float _S431 = _S390 + _S413 + _S415 + _S415 + _S423 + _S428.differential_0.x;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S432;
    (&_S432)->primal_0 = _S287;
    (&_S432)->differential_0 = _S380;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S433;
    (&_S433)->primal_0 = _S288;
    (&_S433)->differential_0 = _S380;
    s_bwd_prop_mul_2(&_S432, &_S433, _S381.differential_0);
    Matrix<float, 3, 3>  _S434 = transpose_0(_S433.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S435;
    (&_S435)->primal_0 = R_6;
    (&_S435)->differential_0 = _S380;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S436;
    (&_S436)->primal_0 = _S286;
    (&_S436)->differential_0 = _S380;
    s_bwd_prop_mul_2(&_S435, &_S436, _S432.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S437;
    (&_S437)->primal_0 = _S284;
    (&_S437)->differential_0 = _S380;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S438;
    (&_S438)->primal_0 = _S285;
    (&_S438)->differential_0 = _S380;
    s_bwd_prop_mul_2(&_S437, &_S438, _S436.differential_0);
    Matrix<float, 3, 3>  _S439 = _S437.differential_0 + transpose_0(_S438.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S440;
    (&_S440)->primal_0 = _S283;
    (&_S440)->differential_0 = _S380;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S441;
    (&_S441)->primal_0 = S_1;
    (&_S441)->differential_0 = _S380;
    s_bwd_prop_mul_2(&_S440, &_S441, _S439);
    Matrix<float, 3, 3>  _S442 = transpose_0(_S440.differential_0);
    float _S443 = 2.0f * - _S442.rows[int(2)].z;
    float _S444 = 2.0f * _S442.rows[int(2)].y;
    float _S445 = 2.0f * _S442.rows[int(2)].x;
    float _S446 = 2.0f * _S442.rows[int(1)].z;
    float _S447 = 2.0f * - _S442.rows[int(1)].y;
    float _S448 = 2.0f * _S442.rows[int(1)].x;
    float _S449 = 2.0f * _S442.rows[int(0)].z;
    float _S450 = 2.0f * _S442.rows[int(0)].y;
    float _S451 = 2.0f * - _S442.rows[int(0)].x;
    float _S452 = - _S448 + _S450;
    float _S453 = _S445 + - _S449;
    float _S454 = - _S444 + _S446;
    float _S455 = _S444 + _S446;
    float _S456 = _S445 + _S449;
    float _S457 = _S448 + _S450;
    float _S458 = z_6 * (_S447 + _S451);
    float _S459 = y_7 * (_S443 + _S451);
    float _S460 = x_16 * (_S443 + _S447);
    float _S461 = z_6 * _S452 + y_7 * _S453 + x_16 * _S454;
    float _S462 = _S282 * _S461;
    float _S463 = w_6 * _S452 + y_7 * _S455 + x_16 * _S456 + _S458 + _S458;
    float _S464 = _S282 * _S463;
    float _S465 = w_6 * _S453 + z_6 * _S455 + x_16 * _S457 + _S459 + _S459;
    float _S466 = _S282 * _S465;
    float _S467 = w_6 * _S454 + z_6 * _S456 + y_7 * _S457 + _S460 + _S460;
    float _S468 = _S282 * _S467;
    float _S469 = quat_6.x * _S461 + quat_6.w * _S463 + quat_6.z * _S465 + quat_6.y * _S467;
    DiffPair_float_0 _S470;
    (&_S470)->primal_0 = _S281;
    (&_S470)->differential_0 = 0.0f;
    s_bwd_prop_rsqrt_0(&_S470, _S469);
    float _S471 = quat_6.x * _S470.differential_0;
    float _S472 = quat_6.w * _S470.differential_0;
    float _S473 = quat_6.z * _S470.differential_0;
    float _S474 = quat_6.y * _S470.differential_0;
    float _S475 = _S464 + _S472 + _S472;
    float _S476 = _S466 + _S473 + _S473;
    float _S477 = _S468 + _S474 + _S474;
    float _S478 = _S462 + _S471 + _S471;
    float3  _S479 = make_float3 (0.0f, 0.0f, v_depth_1);
    float3  _S480 = make_float3 (0.0f);
    float3  _S481 = _S480;
    *&((&_S481)->z) = _S429;
    *&((&_S481)->y) = _S430;
    *&((&_S481)->x) = _S431;
    float3  _S482 = _S479 + _S481;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S483;
    (&_S483)->primal_0 = R_6;
    (&_S483)->differential_0 = _S380;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S484;
    (&_S484)->primal_0 = mean_4;
    (&_S484)->differential_0 = _S480;
    s_bwd_prop_mul_3(&_S483, &_S484, _S482);
    Matrix<float, 3, 3>  _S485 = _S434 + _S435.differential_0 + _S483.differential_0;
    float3  _S486 = _S480;
    *&((&_S486)->z) = _S441.differential_0.rows[int(2)].z;
    *&((&_S486)->y) = _S441.differential_0.rows[int(1)].y;
    *&((&_S486)->x) = _S441.differential_0.rows[int(0)].x;
    float4  _S487 = make_float4 (0.0f);
    *&((&_S487)->w) = _S475;
    *&((&_S487)->z) = _S476;
    *&((&_S487)->y) = _S477;
    *&((&_S487)->x) = _S478;
    *v_mean_1 = _S484.differential_0;
    *v_quat_1 = _S487;
    *v_scale_1 = _S486;
    *v_in_opacity_1 = _S343;
    *v_R_1 = _S485;
    *v_t_1 = _S482;
    return;
}

inline __device__ void projection_3dgs_ortho_vjp(float3  mean_5, float4  quat_7, float3  scale_6, float in_opacity_5, Matrix<float, 3, 3>  R_7, float3  t_6, float fx_9, float fy_9, float cx_9, float cy_9, uint image_width_5, uint image_height_5, float eps2d_6, float2  v_mean2d_2, float v_depth_2, float3  v_conic_2, float v_opacity_2, float3  * v_mean_2, float4  * v_quat_2, float3  * v_scale_2, float * v_in_opacity_2, Matrix<float, 3, 3>  * v_R_2, float3  * v_t_2)
{
    float _S488 = quat_7.y;
    float _S489 = _S488 * _S488 + quat_7.z * quat_7.z + quat_7.w * quat_7.w + quat_7.x * quat_7.x;
    float _S490 = s_primal_ctx_rsqrt_0(_S489);
    float x_17 = quat_7.y * _S490;
    float y_8 = quat_7.z * _S490;
    float z_7 = quat_7.w * _S490;
    float w_7 = quat_7.x * _S490;
    float x2_10 = x_17 * x_17;
    float y2_10 = y_8 * y_8;
    float z2_7 = z_7 * z_7;
    float xy_10 = x_17 * y_8;
    float xz_7 = x_17 * z_7;
    float yz_7 = y_8 * z_7;
    float wx_7 = w_7 * x_17;
    float wy_7 = w_7 * y_8;
    float wz_7 = w_7 * z_7;
    Matrix<float, 3, 3>  _S491 = transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_10 + z2_7), 2.0f * (xy_10 + wz_7), 2.0f * (xz_7 - wy_7), 2.0f * (xy_10 - wz_7), 1.0f - 2.0f * (x2_10 + z2_7), 2.0f * (yz_7 + wx_7), 2.0f * (xz_7 + wy_7), 2.0f * (yz_7 - wx_7), 1.0f - 2.0f * (x2_10 + y2_10)));
    Matrix<float, 3, 3>  S_2 = makeMatrix<float, 3, 3> (scale_6.x, 0.0f, 0.0f, 0.0f, scale_6.y, 0.0f, 0.0f, 0.0f, scale_6.z);
    Matrix<float, 3, 3>  _S492 = s_primal_ctx_mul_1(_S491, S_2);
    Matrix<float, 3, 3>  _S493 = transpose_0(_S492);
    Matrix<float, 3, 3>  _S494 = s_primal_ctx_mul_1(_S492, _S493);
    Matrix<float, 3, 3>  _S495 = s_primal_ctx_mul_1(R_7, _S494);
    Matrix<float, 3, 3>  _S496 = transpose_0(R_7);
    Matrix<float, 3, 3>  _S497 = s_primal_ctx_mul_1(_S495, _S496);
    Matrix<float, 2, 3>  J_9 = makeMatrix<float, 2, 3> (fx_9, 0.0f, 0.0f, 0.0f, fy_9, 0.0f);
    Matrix<float, 2, 3>  _S498 = s_primal_ctx_mul_2(J_9, _S497);
    Matrix<float, 3, 2>  _S499 = transpose_1(J_9);
    Matrix<float, 2, 2>  _S500 = s_primal_ctx_mul_3(_S498, _S499);
    float _S501 = _S500.rows[int(0)].y * _S500.rows[int(1)].x;
    float det_orig_3 = _S500.rows[int(0)].x * _S500.rows[int(1)].y - _S501;
    float _S502 = _S500.rows[int(0)].x + eps2d_6;
    Matrix<float, 2, 2>  _S503 = _S500;
    *&(((&_S503)->rows + (int(0)))->x) = _S502;
    float _S504 = _S500.rows[int(1)].y + eps2d_6;
    *&(((&_S503)->rows + (int(1)))->y) = _S504;
    float det_blur_3 = _S502 * _S504 - _S501;
    float _S505 = det_orig_3 / det_blur_3;
    float _S506 = det_blur_3 * det_blur_3;
    float _S507 = s_primal_ctx_max_0(0.0f, _S505);
    float invdet_6 = 1.0f / det_blur_3;
    float _S508 = - _S500.rows[int(0)].y;
    float _S509 = - _S500.rows[int(1)].x;
    float _S510 = - in_opacity_5;
    float _S511 = 1.0f + s_primal_ctx_exp_0(_S510);
    float _S512 = _S511 * _S511;
    float _S513 = 1.0f / _S511 / 0.00392156885936856f;
    float _S514 = 2.0f * s_primal_ctx_log_0(_S513);
    float _S515 = s_primal_ctx_sqrt_0(_S514);
    float _S516 = _S503.rows[int(0)].x;
    float _S517 = _S503.rows[int(1)].y;
    float2  _S518 = make_float2 (0.0f);
    float2  _S519 = _S518;
    *&((&_S519)->y) = v_conic_2.z;
    float2  _S520 = _S518;
    *&((&_S520)->y) = v_conic_2.y;
    *&((&_S520)->x) = v_conic_2.x;
    DiffPair_float_0 _S521;
    (&_S521)->primal_0 = _S517;
    (&_S521)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S521, 0.0f);
    DiffPair_float_0 _S522;
    (&_S522)->primal_0 = _S516;
    (&_S522)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S522, 0.0f);
    DiffPair_float_0 _S523;
    (&_S523)->primal_0 = 3.32999992370605469f;
    (&_S523)->differential_0 = 0.0f;
    DiffPair_float_0 _S524;
    (&_S524)->primal_0 = _S515;
    (&_S524)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S523, &_S524, 0.0f);
    DiffPair_float_0 _S525;
    (&_S525)->primal_0 = _S514;
    (&_S525)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S525, _S524.differential_0);
    float _S526 = 2.0f * _S525.differential_0;
    DiffPair_float_0 _S527;
    (&_S527)->primal_0 = _S513;
    (&_S527)->differential_0 = 0.0f;
    s_bwd_prop_log_0(&_S527, _S526);
    float _S528 = - ((v_opacity_2 + 254.9999847412109375f * _S527.differential_0) / _S512);
    DiffPair_float_0 _S529;
    (&_S529)->primal_0 = _S510;
    (&_S529)->differential_0 = 0.0f;
    s_bwd_prop_exp_0(&_S529, _S528);
    float _S530 = - _S529.differential_0;
    Matrix<float, 2, 2>  _S531 = makeMatrix<float, 2, 2> (0.0f);
    Matrix<float, 2, 2>  _S532 = _S531;
    _S532[int(1)] = _S519;
    _S532[int(0)] = _S520;
    float _S533 = invdet_6 * _S532.rows[int(1)].y;
    float _S534 = - (invdet_6 * _S532.rows[int(1)].x);
    float _S535 = - (invdet_6 * _S532.rows[int(0)].y);
    float _S536 = invdet_6 * _S532.rows[int(0)].x;
    float _S537 = - ((_S502 * _S532.rows[int(1)].y + _S509 * _S532.rows[int(1)].x + _S508 * _S532.rows[int(0)].y + _S504 * _S532.rows[int(0)].x) / _S506);
    DiffPair_float_0 _S538;
    (&_S538)->primal_0 = _S507;
    (&_S538)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S538, 0.0f);
    DiffPair_float_0 _S539;
    (&_S539)->primal_0 = 0.0f;
    (&_S539)->differential_0 = 0.0f;
    DiffPair_float_0 _S540;
    (&_S540)->primal_0 = _S505;
    (&_S540)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S539, &_S540, _S538.differential_0);
    float _S541 = _S540.differential_0 / _S506;
    float s_diff_det_orig_T_2 = det_blur_3 * _S541;
    float _S542 = _S537 + det_orig_3 * - _S541;
    float _S543 = - _S542;
    float _S544 = _S502 * _S542;
    float _S545 = _S504 * _S542;
    float2  _S546 = make_float2 (0.0f, _S521.differential_0);
    float2  _S547 = make_float2 (_S522.differential_0, 0.0f);
    Matrix<float, 2, 2>  _S548 = _S531;
    _S548[int(1)] = _S546;
    _S548[int(0)] = _S547;
    _S503 = _S548;
    *&(((&_S503)->rows + (int(1)))->y) = 0.0f;
    float _S549 = _S536 + _S544 + _S548.rows[int(1)].y;
    *&(((&_S503)->rows + (int(0)))->x) = 0.0f;
    float _S550 = _S533 + _S545 + _S548.rows[int(0)].x;
    float _S551 = _S543 + - s_diff_det_orig_T_2;
    float _S552 = _S534 + _S500.rows[int(0)].y * _S551;
    float _S553 = _S535 + _S500.rows[int(1)].x * _S551;
    float _S554 = _S500.rows[int(1)].y * s_diff_det_orig_T_2;
    float _S555 = _S549 + _S500.rows[int(0)].x * s_diff_det_orig_T_2;
    float2  _S556 = _S518;
    *&((&_S556)->x) = _S552;
    *&((&_S556)->y) = _S555;
    float _S557 = _S550 + _S554;
    float2  _S558 = _S518;
    *&((&_S558)->y) = _S553;
    *&((&_S558)->x) = _S557;
    float _S559 = fy_9 * v_mean2d_2.y;
    float _S560 = fx_9 * v_mean2d_2.x;
    Matrix<float, 2, 2>  _S561 = _S531;
    _S561[int(1)] = _S556;
    _S561[int(0)] = _S558;
    Matrix<float, 2, 2>  _S562 = _S503 + _S561;
    Matrix<float, 2, 3>  _S563 = makeMatrix<float, 2, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S564;
    (&_S564)->primal_0 = _S498;
    (&_S564)->differential_0 = _S563;
    Matrix<float, 3, 2>  _S565 = makeMatrix<float, 3, 2> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C2x3E_0 _S566;
    (&_S566)->primal_0 = _S499;
    (&_S566)->differential_0 = _S565;
    s_bwd_prop_mul_0(&_S564, &_S566, _S562);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S567;
    (&_S567)->primal_0 = J_9;
    (&_S567)->differential_0 = _S563;
    Matrix<float, 3, 3>  _S568 = makeMatrix<float, 3, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S569;
    (&_S569)->primal_0 = _S497;
    (&_S569)->differential_0 = _S568;
    s_bwd_prop_mul_1(&_S567, &_S569, _S564.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S570;
    (&_S570)->primal_0 = _S495;
    (&_S570)->differential_0 = _S568;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S571;
    (&_S571)->primal_0 = _S496;
    (&_S571)->differential_0 = _S568;
    s_bwd_prop_mul_2(&_S570, &_S571, _S569.differential_0);
    Matrix<float, 3, 3>  _S572 = transpose_0(_S571.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S573;
    (&_S573)->primal_0 = R_7;
    (&_S573)->differential_0 = _S568;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S574;
    (&_S574)->primal_0 = _S494;
    (&_S574)->differential_0 = _S568;
    s_bwd_prop_mul_2(&_S573, &_S574, _S570.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S575;
    (&_S575)->primal_0 = _S492;
    (&_S575)->differential_0 = _S568;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S576;
    (&_S576)->primal_0 = _S493;
    (&_S576)->differential_0 = _S568;
    s_bwd_prop_mul_2(&_S575, &_S576, _S574.differential_0);
    Matrix<float, 3, 3>  _S577 = _S575.differential_0 + transpose_0(_S576.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S578;
    (&_S578)->primal_0 = _S491;
    (&_S578)->differential_0 = _S568;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S579;
    (&_S579)->primal_0 = S_2;
    (&_S579)->differential_0 = _S568;
    s_bwd_prop_mul_2(&_S578, &_S579, _S577);
    Matrix<float, 3, 3>  _S580 = transpose_0(_S578.differential_0);
    float _S581 = 2.0f * - _S580.rows[int(2)].z;
    float _S582 = 2.0f * _S580.rows[int(2)].y;
    float _S583 = 2.0f * _S580.rows[int(2)].x;
    float _S584 = 2.0f * _S580.rows[int(1)].z;
    float _S585 = 2.0f * - _S580.rows[int(1)].y;
    float _S586 = 2.0f * _S580.rows[int(1)].x;
    float _S587 = 2.0f * _S580.rows[int(0)].z;
    float _S588 = 2.0f * _S580.rows[int(0)].y;
    float _S589 = 2.0f * - _S580.rows[int(0)].x;
    float _S590 = - _S586 + _S588;
    float _S591 = _S583 + - _S587;
    float _S592 = - _S582 + _S584;
    float _S593 = _S582 + _S584;
    float _S594 = _S583 + _S587;
    float _S595 = _S586 + _S588;
    float _S596 = z_7 * (_S585 + _S589);
    float _S597 = y_8 * (_S581 + _S589);
    float _S598 = x_17 * (_S581 + _S585);
    float _S599 = z_7 * _S590 + y_8 * _S591 + x_17 * _S592;
    float _S600 = _S490 * _S599;
    float _S601 = w_7 * _S590 + y_8 * _S593 + x_17 * _S594 + _S596 + _S596;
    float _S602 = _S490 * _S601;
    float _S603 = w_7 * _S591 + z_7 * _S593 + x_17 * _S595 + _S597 + _S597;
    float _S604 = _S490 * _S603;
    float _S605 = w_7 * _S592 + z_7 * _S594 + y_8 * _S595 + _S598 + _S598;
    float _S606 = _S490 * _S605;
    float _S607 = quat_7.x * _S599 + quat_7.w * _S601 + quat_7.z * _S603 + quat_7.y * _S605;
    DiffPair_float_0 _S608;
    (&_S608)->primal_0 = _S489;
    (&_S608)->differential_0 = 0.0f;
    s_bwd_prop_rsqrt_0(&_S608, _S607);
    float _S609 = quat_7.x * _S608.differential_0;
    float _S610 = quat_7.w * _S608.differential_0;
    float _S611 = quat_7.z * _S608.differential_0;
    float _S612 = quat_7.y * _S608.differential_0;
    float _S613 = _S602 + _S610 + _S610;
    float _S614 = _S604 + _S611 + _S611;
    float _S615 = _S606 + _S612 + _S612;
    float _S616 = _S600 + _S609 + _S609;
    float3  _S617 = make_float3 (0.0f, 0.0f, v_depth_2);
    float3  _S618 = make_float3 (0.0f);
    float3  _S619 = _S618;
    *&((&_S619)->y) = _S559;
    *&((&_S619)->x) = _S560;
    float3  _S620 = _S617 + _S619;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S621;
    (&_S621)->primal_0 = R_7;
    (&_S621)->differential_0 = _S568;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S622;
    (&_S622)->primal_0 = mean_5;
    (&_S622)->differential_0 = _S618;
    s_bwd_prop_mul_3(&_S621, &_S622, _S620);
    Matrix<float, 3, 3>  _S623 = _S572 + _S573.differential_0 + _S621.differential_0;
    float3  _S624 = _S618;
    *&((&_S624)->z) = _S579.differential_0.rows[int(2)].z;
    *&((&_S624)->y) = _S579.differential_0.rows[int(1)].y;
    *&((&_S624)->x) = _S579.differential_0.rows[int(0)].x;
    float4  _S625 = make_float4 (0.0f);
    *&((&_S625)->w) = _S613;
    *&((&_S625)->z) = _S614;
    *&((&_S625)->y) = _S615;
    *&((&_S625)->x) = _S616;
    *v_mean_2 = _S622.differential_0;
    *v_quat_2 = _S625;
    *v_scale_2 = _S624;
    *v_in_opacity_2 = _S530;
    *v_R_2 = _S623;
    *v_t_2 = _S620;
    return;
}

