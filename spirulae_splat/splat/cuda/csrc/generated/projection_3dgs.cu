#include "hip/hip_runtime.h"
#include "slang.cuh"

struct DiffPair_float_0
{
    float primal_0;
    float differential_0;
};

inline __device__ void _d_max_0(DiffPair_float_0 * dpx_0, DiffPair_float_0 * dpy_0, float dOut_0)
{
    DiffPair_float_0 _S1 = *dpx_0;
    float _S2;
    if(((*dpx_0).primal_0) > ((*dpy_0).primal_0))
    {
        _S2 = dOut_0;
    }
    else
    {
        if(((*dpx_0).primal_0) < ((*dpy_0).primal_0))
        {
            _S2 = 0.0f;
        }
        else
        {
            _S2 = 0.5f * dOut_0;
        }
    }
    dpx_0->primal_0 = _S1.primal_0;
    dpx_0->differential_0 = _S2;
    DiffPair_float_0 _S3 = *dpy_0;
    if(((*dpy_0).primal_0) > (_S1.primal_0))
    {
        _S2 = dOut_0;
    }
    else
    {
        if(((*dpy_0).primal_0) < ((*dpx_0).primal_0))
        {
            _S2 = 0.0f;
        }
        else
        {
            _S2 = 0.5f * dOut_0;
        }
    }
    dpy_0->primal_0 = _S3.primal_0;
    dpy_0->differential_0 = _S2;
    return;
}

inline __device__ void _d_sqrt_0(DiffPair_float_0 * dpx_1, float dOut_1)
{
    float _S4 = 0.5f / (F32_sqrt(((F32_max((1.00000001168609742e-07f), ((*dpx_1).primal_0)))))) * dOut_1;
    dpx_1->primal_0 = (*dpx_1).primal_0;
    dpx_1->differential_0 = _S4;
    return;
}

inline __device__ void _d_rsqrt_0(DiffPair_float_0 * dpx_2, float dOut_2)
{
    float _S5 = -0.5f / ((*dpx_2).primal_0 * (F32_sqrt(((*dpx_2).primal_0)))) * dOut_2;
    dpx_2->primal_0 = (*dpx_2).primal_0;
    dpx_2->differential_0 = _S5;
    return;
}

inline __device__ Matrix<float, 3, 3>  transpose_0(Matrix<float, 3, 3>  x_0)
{
    Matrix<float, 3, 3>  result_0;
    int r_0 = int(0);
    for(;;)
    {
        if(r_0 < int(3))
        {
        }
        else
        {
            break;
        }
        int c_0 = int(0);
        for(;;)
        {
            if(c_0 < int(3))
            {
            }
            else
            {
                break;
            }
            *_slang_vector_get_element_ptr(((&result_0)->rows + (r_0)), c_0) = _slang_vector_get_element(x_0.rows[c_0], r_0);
            c_0 = c_0 + int(1);
        }
        r_0 = r_0 + int(1);
    }
    return result_0;
}

inline __device__ Matrix<float, 3, 2>  transpose_1(Matrix<float, 2, 3>  x_1)
{
    Matrix<float, 3, 2>  result_1;
    int r_1 = int(0);
    for(;;)
    {
        if(r_1 < int(3))
        {
        }
        else
        {
            break;
        }
        int c_1 = int(0);
        for(;;)
        {
            if(c_1 < int(2))
            {
            }
            else
            {
                break;
            }
            *_slang_vector_get_element_ptr(((&result_1)->rows + (r_1)), c_1) = _slang_vector_get_element(x_1.rows[c_1], r_1);
            c_1 = c_1 + int(1);
        }
        r_1 = r_1 + int(1);
    }
    return result_1;
}

inline __device__ Matrix<float, 2, 3>  transpose_2(Matrix<float, 3, 2>  x_2)
{
    Matrix<float, 2, 3>  result_2;
    int r_2 = int(0);
    for(;;)
    {
        if(r_2 < int(2))
        {
        }
        else
        {
            break;
        }
        int c_2 = int(0);
        for(;;)
        {
            if(c_2 < int(3))
            {
            }
            else
            {
                break;
            }
            *_slang_vector_get_element_ptr(((&result_2)->rows + (r_2)), c_2) = _slang_vector_get_element(x_2.rows[c_2], r_2);
            c_2 = c_2 + int(1);
        }
        r_2 = r_2 + int(1);
    }
    return result_2;
}

inline __device__ Matrix<float, 3, 3>  quat_to_rotmat(float4  quat_0)
{
    float x_3 = quat_0.y;
    float inv_norm_0 = (F32_rsqrt((x_3 * x_3 + quat_0.z * quat_0.z + quat_0.w * quat_0.w + quat_0.x * quat_0.x)));
    float x_4 = quat_0.y * inv_norm_0;
    float y_0 = quat_0.z * inv_norm_0;
    float z_0 = quat_0.w * inv_norm_0;
    float w_0 = quat_0.x * inv_norm_0;
    float x2_0 = x_4 * x_4;
    float y2_0 = y_0 * y_0;
    float z2_0 = z_0 * z_0;
    float xy_0 = x_4 * y_0;
    float xz_0 = x_4 * z_0;
    float yz_0 = y_0 * z_0;
    float wx_0 = w_0 * x_4;
    float wy_0 = w_0 * y_0;
    float wz_0 = w_0 * z_0;
    return transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_0 + z2_0), 2.0f * (xy_0 + wz_0), 2.0f * (xz_0 - wy_0), 2.0f * (xy_0 - wz_0), 1.0f - 2.0f * (x2_0 + z2_0), 2.0f * (yz_0 + wx_0), 2.0f * (xz_0 + wy_0), 2.0f * (yz_0 - wx_0), 1.0f - 2.0f * (x2_0 + y2_0)));
}

struct DiffPair_matrixx3Cfloatx2C3x2C3x3E_0
{
    Matrix<float, 3, 3>  primal_0;
    Matrix<float, 3, 3>  differential_0;
};

struct DiffPair_vectorx3Cfloatx2C3x3E_0
{
    float3  primal_0;
    float3  differential_0;
};

inline __device__ void _d_mul_0(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * left_0, DiffPair_vectorx3Cfloatx2C3x3E_0 * right_0, float3  dOut_3)
{
    float _S6 = (*left_0).primal_0.rows[int(0)].x * dOut_3.x;
    Matrix<float, 3, 3>  left_d_result_0;
    *&(((&left_d_result_0)->rows + (int(0)))->x) = (*right_0).primal_0.x * dOut_3.x;
    float sum_0 = _S6 + (*left_0).primal_0.rows[int(1)].x * dOut_3.y;
    *&(((&left_d_result_0)->rows + (int(1)))->x) = (*right_0).primal_0.x * dOut_3.y;
    float sum_1 = sum_0 + (*left_0).primal_0.rows[int(2)].x * dOut_3.z;
    *&(((&left_d_result_0)->rows + (int(2)))->x) = (*right_0).primal_0.x * dOut_3.z;
    float3  right_d_result_0;
    *&((&right_d_result_0)->x) = sum_1;
    float _S7 = (*left_0).primal_0.rows[int(0)].y * dOut_3.x;
    *&(((&left_d_result_0)->rows + (int(0)))->y) = (*right_0).primal_0.y * dOut_3.x;
    float sum_2 = _S7 + (*left_0).primal_0.rows[int(1)].y * dOut_3.y;
    *&(((&left_d_result_0)->rows + (int(1)))->y) = (*right_0).primal_0.y * dOut_3.y;
    float sum_3 = sum_2 + (*left_0).primal_0.rows[int(2)].y * dOut_3.z;
    *&(((&left_d_result_0)->rows + (int(2)))->y) = (*right_0).primal_0.y * dOut_3.z;
    *&((&right_d_result_0)->y) = sum_3;
    float _S8 = (*left_0).primal_0.rows[int(0)].z * dOut_3.x;
    *&(((&left_d_result_0)->rows + (int(0)))->z) = (*right_0).primal_0.z * dOut_3.x;
    float sum_4 = _S8 + (*left_0).primal_0.rows[int(1)].z * dOut_3.y;
    *&(((&left_d_result_0)->rows + (int(1)))->z) = (*right_0).primal_0.z * dOut_3.y;
    float sum_5 = sum_4 + (*left_0).primal_0.rows[int(2)].z * dOut_3.z;
    *&(((&left_d_result_0)->rows + (int(2)))->z) = (*right_0).primal_0.z * dOut_3.z;
    *&((&right_d_result_0)->z) = sum_5;
    left_0->primal_0 = (*left_0).primal_0;
    left_0->differential_0 = left_d_result_0;
    right_0->primal_0 = (*right_0).primal_0;
    right_0->differential_0 = right_d_result_0;
    return;
}

inline __device__ float3  mul_0(Matrix<float, 3, 3>  left_1, float3  right_1)
{
    float3  result_3;
    int i_0 = int(0);
    for(;;)
    {
        if(i_0 < int(3))
        {
        }
        else
        {
            break;
        }
        int j_0 = int(0);
        float sum_6 = 0.0f;
        for(;;)
        {
            if(j_0 < int(3))
            {
            }
            else
            {
                break;
            }
            float sum_7 = sum_6 + _slang_vector_get_element(left_1.rows[i_0], j_0) * _slang_vector_get_element(right_1, j_0);
            j_0 = j_0 + int(1);
            sum_6 = sum_7;
        }
        *_slang_vector_get_element_ptr(&result_3, i_0) = sum_6;
        i_0 = i_0 + int(1);
    }
    return result_3;
}

inline __device__ void posW2C(Matrix<float, 3, 3>  R_0, float3  t_0, float3  pW_0, float3  * pC_0)
{
    *pC_0 = mul_0(R_0, pW_0) + t_0;
    return;
}

inline __device__ void mul_1(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * left_2, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * right_2, Matrix<float, 3, 3>  dOut_4)
{
    Matrix<float, 3, 3>  left_d_result_1;
    *&(((&left_d_result_1)->rows + (int(0)))->x) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(0)))->y) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(0)))->z) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(1)))->x) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(1)))->y) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(1)))->z) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(2)))->x) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(2)))->y) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(2)))->z) = 0.0f;
    Matrix<float, 3, 3>  right_d_result_1;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = 0.0f;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = 0.0f;
    *&(((&left_d_result_1)->rows + (int(0)))->x) = *&(((&left_d_result_1)->rows + (int(0)))->x) + (*right_2).primal_0.rows[int(0)].x * dOut_4.rows[int(0)].x;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = *&(((&right_d_result_1)->rows + (int(0)))->x) + (*left_2).primal_0.rows[int(0)].x * dOut_4.rows[int(0)].x;
    *&(((&left_d_result_1)->rows + (int(0)))->y) = *&(((&left_d_result_1)->rows + (int(0)))->y) + (*right_2).primal_0.rows[int(1)].x * dOut_4.rows[int(0)].x;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = *&(((&right_d_result_1)->rows + (int(1)))->x) + (*left_2).primal_0.rows[int(0)].y * dOut_4.rows[int(0)].x;
    *&(((&left_d_result_1)->rows + (int(0)))->z) = *&(((&left_d_result_1)->rows + (int(0)))->z) + (*right_2).primal_0.rows[int(2)].x * dOut_4.rows[int(0)].x;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = *&(((&right_d_result_1)->rows + (int(2)))->x) + (*left_2).primal_0.rows[int(0)].z * dOut_4.rows[int(0)].x;
    *&(((&left_d_result_1)->rows + (int(0)))->x) = *&(((&left_d_result_1)->rows + (int(0)))->x) + (*right_2).primal_0.rows[int(0)].y * dOut_4.rows[int(0)].y;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = *&(((&right_d_result_1)->rows + (int(0)))->y) + (*left_2).primal_0.rows[int(0)].x * dOut_4.rows[int(0)].y;
    *&(((&left_d_result_1)->rows + (int(0)))->y) = *&(((&left_d_result_1)->rows + (int(0)))->y) + (*right_2).primal_0.rows[int(1)].y * dOut_4.rows[int(0)].y;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = *&(((&right_d_result_1)->rows + (int(1)))->y) + (*left_2).primal_0.rows[int(0)].y * dOut_4.rows[int(0)].y;
    *&(((&left_d_result_1)->rows + (int(0)))->z) = *&(((&left_d_result_1)->rows + (int(0)))->z) + (*right_2).primal_0.rows[int(2)].y * dOut_4.rows[int(0)].y;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = *&(((&right_d_result_1)->rows + (int(2)))->y) + (*left_2).primal_0.rows[int(0)].z * dOut_4.rows[int(0)].y;
    *&(((&left_d_result_1)->rows + (int(0)))->x) = *&(((&left_d_result_1)->rows + (int(0)))->x) + (*right_2).primal_0.rows[int(0)].z * dOut_4.rows[int(0)].z;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = *&(((&right_d_result_1)->rows + (int(0)))->z) + (*left_2).primal_0.rows[int(0)].x * dOut_4.rows[int(0)].z;
    *&(((&left_d_result_1)->rows + (int(0)))->y) = *&(((&left_d_result_1)->rows + (int(0)))->y) + (*right_2).primal_0.rows[int(1)].z * dOut_4.rows[int(0)].z;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = *&(((&right_d_result_1)->rows + (int(1)))->z) + (*left_2).primal_0.rows[int(0)].y * dOut_4.rows[int(0)].z;
    *&(((&left_d_result_1)->rows + (int(0)))->z) = *&(((&left_d_result_1)->rows + (int(0)))->z) + (*right_2).primal_0.rows[int(2)].z * dOut_4.rows[int(0)].z;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = *&(((&right_d_result_1)->rows + (int(2)))->z) + (*left_2).primal_0.rows[int(0)].z * dOut_4.rows[int(0)].z;
    *&(((&left_d_result_1)->rows + (int(1)))->x) = *&(((&left_d_result_1)->rows + (int(1)))->x) + (*right_2).primal_0.rows[int(0)].x * dOut_4.rows[int(1)].x;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = *&(((&right_d_result_1)->rows + (int(0)))->x) + (*left_2).primal_0.rows[int(1)].x * dOut_4.rows[int(1)].x;
    *&(((&left_d_result_1)->rows + (int(1)))->y) = *&(((&left_d_result_1)->rows + (int(1)))->y) + (*right_2).primal_0.rows[int(1)].x * dOut_4.rows[int(1)].x;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = *&(((&right_d_result_1)->rows + (int(1)))->x) + (*left_2).primal_0.rows[int(1)].y * dOut_4.rows[int(1)].x;
    *&(((&left_d_result_1)->rows + (int(1)))->z) = *&(((&left_d_result_1)->rows + (int(1)))->z) + (*right_2).primal_0.rows[int(2)].x * dOut_4.rows[int(1)].x;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = *&(((&right_d_result_1)->rows + (int(2)))->x) + (*left_2).primal_0.rows[int(1)].z * dOut_4.rows[int(1)].x;
    *&(((&left_d_result_1)->rows + (int(1)))->x) = *&(((&left_d_result_1)->rows + (int(1)))->x) + (*right_2).primal_0.rows[int(0)].y * dOut_4.rows[int(1)].y;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = *&(((&right_d_result_1)->rows + (int(0)))->y) + (*left_2).primal_0.rows[int(1)].x * dOut_4.rows[int(1)].y;
    *&(((&left_d_result_1)->rows + (int(1)))->y) = *&(((&left_d_result_1)->rows + (int(1)))->y) + (*right_2).primal_0.rows[int(1)].y * dOut_4.rows[int(1)].y;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = *&(((&right_d_result_1)->rows + (int(1)))->y) + (*left_2).primal_0.rows[int(1)].y * dOut_4.rows[int(1)].y;
    *&(((&left_d_result_1)->rows + (int(1)))->z) = *&(((&left_d_result_1)->rows + (int(1)))->z) + (*right_2).primal_0.rows[int(2)].y * dOut_4.rows[int(1)].y;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = *&(((&right_d_result_1)->rows + (int(2)))->y) + (*left_2).primal_0.rows[int(1)].z * dOut_4.rows[int(1)].y;
    *&(((&left_d_result_1)->rows + (int(1)))->x) = *&(((&left_d_result_1)->rows + (int(1)))->x) + (*right_2).primal_0.rows[int(0)].z * dOut_4.rows[int(1)].z;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = *&(((&right_d_result_1)->rows + (int(0)))->z) + (*left_2).primal_0.rows[int(1)].x * dOut_4.rows[int(1)].z;
    *&(((&left_d_result_1)->rows + (int(1)))->y) = *&(((&left_d_result_1)->rows + (int(1)))->y) + (*right_2).primal_0.rows[int(1)].z * dOut_4.rows[int(1)].z;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = *&(((&right_d_result_1)->rows + (int(1)))->z) + (*left_2).primal_0.rows[int(1)].y * dOut_4.rows[int(1)].z;
    *&(((&left_d_result_1)->rows + (int(1)))->z) = *&(((&left_d_result_1)->rows + (int(1)))->z) + (*right_2).primal_0.rows[int(2)].z * dOut_4.rows[int(1)].z;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = *&(((&right_d_result_1)->rows + (int(2)))->z) + (*left_2).primal_0.rows[int(1)].z * dOut_4.rows[int(1)].z;
    *&(((&left_d_result_1)->rows + (int(2)))->x) = *&(((&left_d_result_1)->rows + (int(2)))->x) + (*right_2).primal_0.rows[int(0)].x * dOut_4.rows[int(2)].x;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = *&(((&right_d_result_1)->rows + (int(0)))->x) + (*left_2).primal_0.rows[int(2)].x * dOut_4.rows[int(2)].x;
    *&(((&left_d_result_1)->rows + (int(2)))->y) = *&(((&left_d_result_1)->rows + (int(2)))->y) + (*right_2).primal_0.rows[int(1)].x * dOut_4.rows[int(2)].x;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = *&(((&right_d_result_1)->rows + (int(1)))->x) + (*left_2).primal_0.rows[int(2)].y * dOut_4.rows[int(2)].x;
    *&(((&left_d_result_1)->rows + (int(2)))->z) = *&(((&left_d_result_1)->rows + (int(2)))->z) + (*right_2).primal_0.rows[int(2)].x * dOut_4.rows[int(2)].x;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = *&(((&right_d_result_1)->rows + (int(2)))->x) + (*left_2).primal_0.rows[int(2)].z * dOut_4.rows[int(2)].x;
    *&(((&left_d_result_1)->rows + (int(2)))->x) = *&(((&left_d_result_1)->rows + (int(2)))->x) + (*right_2).primal_0.rows[int(0)].y * dOut_4.rows[int(2)].y;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = *&(((&right_d_result_1)->rows + (int(0)))->y) + (*left_2).primal_0.rows[int(2)].x * dOut_4.rows[int(2)].y;
    *&(((&left_d_result_1)->rows + (int(2)))->y) = *&(((&left_d_result_1)->rows + (int(2)))->y) + (*right_2).primal_0.rows[int(1)].y * dOut_4.rows[int(2)].y;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = *&(((&right_d_result_1)->rows + (int(1)))->y) + (*left_2).primal_0.rows[int(2)].y * dOut_4.rows[int(2)].y;
    *&(((&left_d_result_1)->rows + (int(2)))->z) = *&(((&left_d_result_1)->rows + (int(2)))->z) + (*right_2).primal_0.rows[int(2)].y * dOut_4.rows[int(2)].y;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = *&(((&right_d_result_1)->rows + (int(2)))->y) + (*left_2).primal_0.rows[int(2)].z * dOut_4.rows[int(2)].y;
    *&(((&left_d_result_1)->rows + (int(2)))->x) = *&(((&left_d_result_1)->rows + (int(2)))->x) + (*right_2).primal_0.rows[int(0)].z * dOut_4.rows[int(2)].z;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = *&(((&right_d_result_1)->rows + (int(0)))->z) + (*left_2).primal_0.rows[int(2)].x * dOut_4.rows[int(2)].z;
    *&(((&left_d_result_1)->rows + (int(2)))->y) = *&(((&left_d_result_1)->rows + (int(2)))->y) + (*right_2).primal_0.rows[int(1)].z * dOut_4.rows[int(2)].z;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = *&(((&right_d_result_1)->rows + (int(1)))->z) + (*left_2).primal_0.rows[int(2)].y * dOut_4.rows[int(2)].z;
    *&(((&left_d_result_1)->rows + (int(2)))->z) = *&(((&left_d_result_1)->rows + (int(2)))->z) + (*right_2).primal_0.rows[int(2)].z * dOut_4.rows[int(2)].z;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = *&(((&right_d_result_1)->rows + (int(2)))->z) + (*left_2).primal_0.rows[int(2)].z * dOut_4.rows[int(2)].z;
    left_2->primal_0 = (*left_2).primal_0;
    left_2->differential_0 = left_d_result_1;
    right_2->primal_0 = (*right_2).primal_0;
    right_2->differential_0 = right_d_result_1;
    return;
}

struct DiffPair_matrixx3Cfloatx2C2x2C3x3E_0
{
    Matrix<float, 2, 3>  primal_0;
    Matrix<float, 2, 3>  differential_0;
};

inline __device__ void mul_2(DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 * left_3, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * right_3, Matrix<float, 2, 3>  dOut_5)
{
    Matrix<float, 2, 3>  left_d_result_2;
    *&(((&left_d_result_2)->rows + (int(0)))->x) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(0)))->y) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(0)))->z) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(1)))->x) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(1)))->y) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(1)))->z) = 0.0f;
    Matrix<float, 3, 3>  right_d_result_2;
    *&(((&right_d_result_2)->rows + (int(0)))->x) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(0)))->y) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(0)))->z) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(1)))->x) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(1)))->y) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(1)))->z) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(2)))->x) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(2)))->y) = 0.0f;
    *&(((&right_d_result_2)->rows + (int(2)))->z) = 0.0f;
    *&(((&left_d_result_2)->rows + (int(0)))->x) = *&(((&left_d_result_2)->rows + (int(0)))->x) + (*right_3).primal_0.rows[int(0)].x * dOut_5.rows[int(0)].x;
    *&(((&right_d_result_2)->rows + (int(0)))->x) = *&(((&right_d_result_2)->rows + (int(0)))->x) + (*left_3).primal_0.rows[int(0)].x * dOut_5.rows[int(0)].x;
    *&(((&left_d_result_2)->rows + (int(0)))->y) = *&(((&left_d_result_2)->rows + (int(0)))->y) + (*right_3).primal_0.rows[int(1)].x * dOut_5.rows[int(0)].x;
    *&(((&right_d_result_2)->rows + (int(1)))->x) = *&(((&right_d_result_2)->rows + (int(1)))->x) + (*left_3).primal_0.rows[int(0)].y * dOut_5.rows[int(0)].x;
    *&(((&left_d_result_2)->rows + (int(0)))->z) = *&(((&left_d_result_2)->rows + (int(0)))->z) + (*right_3).primal_0.rows[int(2)].x * dOut_5.rows[int(0)].x;
    *&(((&right_d_result_2)->rows + (int(2)))->x) = *&(((&right_d_result_2)->rows + (int(2)))->x) + (*left_3).primal_0.rows[int(0)].z * dOut_5.rows[int(0)].x;
    *&(((&left_d_result_2)->rows + (int(0)))->x) = *&(((&left_d_result_2)->rows + (int(0)))->x) + (*right_3).primal_0.rows[int(0)].y * dOut_5.rows[int(0)].y;
    *&(((&right_d_result_2)->rows + (int(0)))->y) = *&(((&right_d_result_2)->rows + (int(0)))->y) + (*left_3).primal_0.rows[int(0)].x * dOut_5.rows[int(0)].y;
    *&(((&left_d_result_2)->rows + (int(0)))->y) = *&(((&left_d_result_2)->rows + (int(0)))->y) + (*right_3).primal_0.rows[int(1)].y * dOut_5.rows[int(0)].y;
    *&(((&right_d_result_2)->rows + (int(1)))->y) = *&(((&right_d_result_2)->rows + (int(1)))->y) + (*left_3).primal_0.rows[int(0)].y * dOut_5.rows[int(0)].y;
    *&(((&left_d_result_2)->rows + (int(0)))->z) = *&(((&left_d_result_2)->rows + (int(0)))->z) + (*right_3).primal_0.rows[int(2)].y * dOut_5.rows[int(0)].y;
    *&(((&right_d_result_2)->rows + (int(2)))->y) = *&(((&right_d_result_2)->rows + (int(2)))->y) + (*left_3).primal_0.rows[int(0)].z * dOut_5.rows[int(0)].y;
    *&(((&left_d_result_2)->rows + (int(0)))->x) = *&(((&left_d_result_2)->rows + (int(0)))->x) + (*right_3).primal_0.rows[int(0)].z * dOut_5.rows[int(0)].z;
    *&(((&right_d_result_2)->rows + (int(0)))->z) = *&(((&right_d_result_2)->rows + (int(0)))->z) + (*left_3).primal_0.rows[int(0)].x * dOut_5.rows[int(0)].z;
    *&(((&left_d_result_2)->rows + (int(0)))->y) = *&(((&left_d_result_2)->rows + (int(0)))->y) + (*right_3).primal_0.rows[int(1)].z * dOut_5.rows[int(0)].z;
    *&(((&right_d_result_2)->rows + (int(1)))->z) = *&(((&right_d_result_2)->rows + (int(1)))->z) + (*left_3).primal_0.rows[int(0)].y * dOut_5.rows[int(0)].z;
    *&(((&left_d_result_2)->rows + (int(0)))->z) = *&(((&left_d_result_2)->rows + (int(0)))->z) + (*right_3).primal_0.rows[int(2)].z * dOut_5.rows[int(0)].z;
    *&(((&right_d_result_2)->rows + (int(2)))->z) = *&(((&right_d_result_2)->rows + (int(2)))->z) + (*left_3).primal_0.rows[int(0)].z * dOut_5.rows[int(0)].z;
    *&(((&left_d_result_2)->rows + (int(1)))->x) = *&(((&left_d_result_2)->rows + (int(1)))->x) + (*right_3).primal_0.rows[int(0)].x * dOut_5.rows[int(1)].x;
    *&(((&right_d_result_2)->rows + (int(0)))->x) = *&(((&right_d_result_2)->rows + (int(0)))->x) + (*left_3).primal_0.rows[int(1)].x * dOut_5.rows[int(1)].x;
    *&(((&left_d_result_2)->rows + (int(1)))->y) = *&(((&left_d_result_2)->rows + (int(1)))->y) + (*right_3).primal_0.rows[int(1)].x * dOut_5.rows[int(1)].x;
    *&(((&right_d_result_2)->rows + (int(1)))->x) = *&(((&right_d_result_2)->rows + (int(1)))->x) + (*left_3).primal_0.rows[int(1)].y * dOut_5.rows[int(1)].x;
    *&(((&left_d_result_2)->rows + (int(1)))->z) = *&(((&left_d_result_2)->rows + (int(1)))->z) + (*right_3).primal_0.rows[int(2)].x * dOut_5.rows[int(1)].x;
    *&(((&right_d_result_2)->rows + (int(2)))->x) = *&(((&right_d_result_2)->rows + (int(2)))->x) + (*left_3).primal_0.rows[int(1)].z * dOut_5.rows[int(1)].x;
    *&(((&left_d_result_2)->rows + (int(1)))->x) = *&(((&left_d_result_2)->rows + (int(1)))->x) + (*right_3).primal_0.rows[int(0)].y * dOut_5.rows[int(1)].y;
    *&(((&right_d_result_2)->rows + (int(0)))->y) = *&(((&right_d_result_2)->rows + (int(0)))->y) + (*left_3).primal_0.rows[int(1)].x * dOut_5.rows[int(1)].y;
    *&(((&left_d_result_2)->rows + (int(1)))->y) = *&(((&left_d_result_2)->rows + (int(1)))->y) + (*right_3).primal_0.rows[int(1)].y * dOut_5.rows[int(1)].y;
    *&(((&right_d_result_2)->rows + (int(1)))->y) = *&(((&right_d_result_2)->rows + (int(1)))->y) + (*left_3).primal_0.rows[int(1)].y * dOut_5.rows[int(1)].y;
    *&(((&left_d_result_2)->rows + (int(1)))->z) = *&(((&left_d_result_2)->rows + (int(1)))->z) + (*right_3).primal_0.rows[int(2)].y * dOut_5.rows[int(1)].y;
    *&(((&right_d_result_2)->rows + (int(2)))->y) = *&(((&right_d_result_2)->rows + (int(2)))->y) + (*left_3).primal_0.rows[int(1)].z * dOut_5.rows[int(1)].y;
    *&(((&left_d_result_2)->rows + (int(1)))->x) = *&(((&left_d_result_2)->rows + (int(1)))->x) + (*right_3).primal_0.rows[int(0)].z * dOut_5.rows[int(1)].z;
    *&(((&right_d_result_2)->rows + (int(0)))->z) = *&(((&right_d_result_2)->rows + (int(0)))->z) + (*left_3).primal_0.rows[int(1)].x * dOut_5.rows[int(1)].z;
    *&(((&left_d_result_2)->rows + (int(1)))->y) = *&(((&left_d_result_2)->rows + (int(1)))->y) + (*right_3).primal_0.rows[int(1)].z * dOut_5.rows[int(1)].z;
    *&(((&right_d_result_2)->rows + (int(1)))->z) = *&(((&right_d_result_2)->rows + (int(1)))->z) + (*left_3).primal_0.rows[int(1)].y * dOut_5.rows[int(1)].z;
    *&(((&left_d_result_2)->rows + (int(1)))->z) = *&(((&left_d_result_2)->rows + (int(1)))->z) + (*right_3).primal_0.rows[int(2)].z * dOut_5.rows[int(1)].z;
    *&(((&right_d_result_2)->rows + (int(2)))->z) = *&(((&right_d_result_2)->rows + (int(2)))->z) + (*left_3).primal_0.rows[int(1)].z * dOut_5.rows[int(1)].z;
    left_3->primal_0 = (*left_3).primal_0;
    left_3->differential_0 = left_d_result_2;
    right_3->primal_0 = (*right_3).primal_0;
    right_3->differential_0 = right_d_result_2;
    return;
}

struct DiffPair_matrixx3Cfloatx2C3x2C2x3E_0
{
    Matrix<float, 3, 2>  primal_0;
    Matrix<float, 3, 2>  differential_0;
};

inline __device__ void mul_3(DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 * left_4, DiffPair_matrixx3Cfloatx2C3x2C2x3E_0 * right_4, Matrix<float, 2, 2>  dOut_6)
{
    Matrix<float, 2, 3>  left_d_result_3;
    *&(((&left_d_result_3)->rows + (int(0)))->x) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(0)))->y) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(0)))->z) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(1)))->x) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(1)))->y) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(1)))->z) = 0.0f;
    Matrix<float, 3, 2>  right_d_result_3;
    *&(((&right_d_result_3)->rows + (int(0)))->x) = 0.0f;
    *&(((&right_d_result_3)->rows + (int(0)))->y) = 0.0f;
    *&(((&right_d_result_3)->rows + (int(1)))->x) = 0.0f;
    *&(((&right_d_result_3)->rows + (int(1)))->y) = 0.0f;
    *&(((&right_d_result_3)->rows + (int(2)))->x) = 0.0f;
    *&(((&right_d_result_3)->rows + (int(2)))->y) = 0.0f;
    *&(((&left_d_result_3)->rows + (int(0)))->x) = *&(((&left_d_result_3)->rows + (int(0)))->x) + (*right_4).primal_0.rows[int(0)].x * dOut_6.rows[int(0)].x;
    *&(((&right_d_result_3)->rows + (int(0)))->x) = *&(((&right_d_result_3)->rows + (int(0)))->x) + (*left_4).primal_0.rows[int(0)].x * dOut_6.rows[int(0)].x;
    *&(((&left_d_result_3)->rows + (int(0)))->y) = *&(((&left_d_result_3)->rows + (int(0)))->y) + (*right_4).primal_0.rows[int(1)].x * dOut_6.rows[int(0)].x;
    *&(((&right_d_result_3)->rows + (int(1)))->x) = *&(((&right_d_result_3)->rows + (int(1)))->x) + (*left_4).primal_0.rows[int(0)].y * dOut_6.rows[int(0)].x;
    *&(((&left_d_result_3)->rows + (int(0)))->z) = *&(((&left_d_result_3)->rows + (int(0)))->z) + (*right_4).primal_0.rows[int(2)].x * dOut_6.rows[int(0)].x;
    *&(((&right_d_result_3)->rows + (int(2)))->x) = *&(((&right_d_result_3)->rows + (int(2)))->x) + (*left_4).primal_0.rows[int(0)].z * dOut_6.rows[int(0)].x;
    *&(((&left_d_result_3)->rows + (int(0)))->x) = *&(((&left_d_result_3)->rows + (int(0)))->x) + (*right_4).primal_0.rows[int(0)].y * dOut_6.rows[int(0)].y;
    *&(((&right_d_result_3)->rows + (int(0)))->y) = *&(((&right_d_result_3)->rows + (int(0)))->y) + (*left_4).primal_0.rows[int(0)].x * dOut_6.rows[int(0)].y;
    *&(((&left_d_result_3)->rows + (int(0)))->y) = *&(((&left_d_result_3)->rows + (int(0)))->y) + (*right_4).primal_0.rows[int(1)].y * dOut_6.rows[int(0)].y;
    *&(((&right_d_result_3)->rows + (int(1)))->y) = *&(((&right_d_result_3)->rows + (int(1)))->y) + (*left_4).primal_0.rows[int(0)].y * dOut_6.rows[int(0)].y;
    *&(((&left_d_result_3)->rows + (int(0)))->z) = *&(((&left_d_result_3)->rows + (int(0)))->z) + (*right_4).primal_0.rows[int(2)].y * dOut_6.rows[int(0)].y;
    *&(((&right_d_result_3)->rows + (int(2)))->y) = *&(((&right_d_result_3)->rows + (int(2)))->y) + (*left_4).primal_0.rows[int(0)].z * dOut_6.rows[int(0)].y;
    *&(((&left_d_result_3)->rows + (int(1)))->x) = *&(((&left_d_result_3)->rows + (int(1)))->x) + (*right_4).primal_0.rows[int(0)].x * dOut_6.rows[int(1)].x;
    *&(((&right_d_result_3)->rows + (int(0)))->x) = *&(((&right_d_result_3)->rows + (int(0)))->x) + (*left_4).primal_0.rows[int(1)].x * dOut_6.rows[int(1)].x;
    *&(((&left_d_result_3)->rows + (int(1)))->y) = *&(((&left_d_result_3)->rows + (int(1)))->y) + (*right_4).primal_0.rows[int(1)].x * dOut_6.rows[int(1)].x;
    *&(((&right_d_result_3)->rows + (int(1)))->x) = *&(((&right_d_result_3)->rows + (int(1)))->x) + (*left_4).primal_0.rows[int(1)].y * dOut_6.rows[int(1)].x;
    *&(((&left_d_result_3)->rows + (int(1)))->z) = *&(((&left_d_result_3)->rows + (int(1)))->z) + (*right_4).primal_0.rows[int(2)].x * dOut_6.rows[int(1)].x;
    *&(((&right_d_result_3)->rows + (int(2)))->x) = *&(((&right_d_result_3)->rows + (int(2)))->x) + (*left_4).primal_0.rows[int(1)].z * dOut_6.rows[int(1)].x;
    *&(((&left_d_result_3)->rows + (int(1)))->x) = *&(((&left_d_result_3)->rows + (int(1)))->x) + (*right_4).primal_0.rows[int(0)].y * dOut_6.rows[int(1)].y;
    *&(((&right_d_result_3)->rows + (int(0)))->y) = *&(((&right_d_result_3)->rows + (int(0)))->y) + (*left_4).primal_0.rows[int(1)].x * dOut_6.rows[int(1)].y;
    *&(((&left_d_result_3)->rows + (int(1)))->y) = *&(((&left_d_result_3)->rows + (int(1)))->y) + (*right_4).primal_0.rows[int(1)].y * dOut_6.rows[int(1)].y;
    *&(((&right_d_result_3)->rows + (int(1)))->y) = *&(((&right_d_result_3)->rows + (int(1)))->y) + (*left_4).primal_0.rows[int(1)].y * dOut_6.rows[int(1)].y;
    *&(((&left_d_result_3)->rows + (int(1)))->z) = *&(((&left_d_result_3)->rows + (int(1)))->z) + (*right_4).primal_0.rows[int(2)].y * dOut_6.rows[int(1)].y;
    *&(((&right_d_result_3)->rows + (int(2)))->y) = *&(((&right_d_result_3)->rows + (int(2)))->y) + (*left_4).primal_0.rows[int(1)].z * dOut_6.rows[int(1)].y;
    left_4->primal_0 = (*left_4).primal_0;
    left_4->differential_0 = left_d_result_3;
    right_4->primal_0 = (*right_4).primal_0;
    right_4->differential_0 = right_d_result_3;
    return;
}

inline __device__ Matrix<float, 3, 3>  mul_4(Matrix<float, 3, 3>  left_5, Matrix<float, 3, 3>  right_5)
{
    Matrix<float, 3, 3>  result_4;
    int r_3 = int(0);
    for(;;)
    {
        if(r_3 < int(3))
        {
        }
        else
        {
            break;
        }
        int c_3 = int(0);
        for(;;)
        {
            if(c_3 < int(3))
            {
            }
            else
            {
                break;
            }
            int i_1 = int(0);
            float sum_8 = 0.0f;
            for(;;)
            {
                if(i_1 < int(3))
                {
                }
                else
                {
                    break;
                }
                float sum_9 = sum_8 + _slang_vector_get_element(left_5.rows[r_3], i_1) * _slang_vector_get_element(right_5.rows[i_1], c_3);
                i_1 = i_1 + int(1);
                sum_8 = sum_9;
            }
            *_slang_vector_get_element_ptr(((&result_4)->rows + (r_3)), c_3) = sum_8;
            c_3 = c_3 + int(1);
        }
        r_3 = r_3 + int(1);
    }
    return result_4;
}

inline __device__ Matrix<float, 2, 3>  mul_5(Matrix<float, 2, 3>  left_6, Matrix<float, 3, 3>  right_6)
{
    Matrix<float, 2, 3>  result_5;
    int r_4 = int(0);
    for(;;)
    {
        if(r_4 < int(2))
        {
        }
        else
        {
            break;
        }
        int c_4 = int(0);
        for(;;)
        {
            if(c_4 < int(3))
            {
            }
            else
            {
                break;
            }
            int i_2 = int(0);
            float sum_10 = 0.0f;
            for(;;)
            {
                if(i_2 < int(3))
                {
                }
                else
                {
                    break;
                }
                float sum_11 = sum_10 + _slang_vector_get_element(left_6.rows[r_4], i_2) * _slang_vector_get_element(right_6.rows[i_2], c_4);
                i_2 = i_2 + int(1);
                sum_10 = sum_11;
            }
            *_slang_vector_get_element_ptr(((&result_5)->rows + (r_4)), c_4) = sum_10;
            c_4 = c_4 + int(1);
        }
        r_4 = r_4 + int(1);
    }
    return result_5;
}

inline __device__ Matrix<float, 2, 2>  mul_6(Matrix<float, 2, 3>  left_7, Matrix<float, 3, 2>  right_7)
{
    Matrix<float, 2, 2>  result_6;
    int r_5 = int(0);
    for(;;)
    {
        if(r_5 < int(2))
        {
        }
        else
        {
            break;
        }
        int c_5 = int(0);
        for(;;)
        {
            if(c_5 < int(2))
            {
            }
            else
            {
                break;
            }
            int i_3 = int(0);
            float sum_12 = 0.0f;
            for(;;)
            {
                if(i_3 < int(3))
                {
                }
                else
                {
                    break;
                }
                float sum_13 = sum_12 + _slang_vector_get_element(left_7.rows[r_5], i_3) * _slang_vector_get_element(right_7.rows[i_3], c_5);
                i_3 = i_3 + int(1);
                sum_12 = sum_13;
            }
            *_slang_vector_get_element_ptr(((&result_6)->rows + (r_5)), c_5) = sum_12;
            c_5 = c_5 + int(1);
        }
        r_5 = r_5 + int(1);
    }
    return result_6;
}

inline __device__ void covarW2C(Matrix<float, 3, 3>  R_1, Matrix<float, 3, 3>  covarW_0, Matrix<float, 3, 3>  * covarC_0)
{
    *covarC_0 = mul_4(mul_4(R_1, covarW_0), transpose_0(R_1));
    return;
}

inline __device__ void quat_scale_to_covar(float4  quat_1, float3  scale_0, Matrix<float, 3, 3>  * covar_0)
{
    float x_5 = quat_1.y;
    float inv_norm_1 = (F32_rsqrt((x_5 * x_5 + quat_1.z * quat_1.z + quat_1.w * quat_1.w + quat_1.x * quat_1.x)));
    float x_6 = quat_1.y * inv_norm_1;
    float y_1 = quat_1.z * inv_norm_1;
    float z_1 = quat_1.w * inv_norm_1;
    float w_1 = quat_1.x * inv_norm_1;
    float x2_1 = x_6 * x_6;
    float y2_1 = y_1 * y_1;
    float z2_1 = z_1 * z_1;
    float xy_1 = x_6 * y_1;
    float xz_1 = x_6 * z_1;
    float yz_1 = y_1 * z_1;
    float wx_1 = w_1 * x_6;
    float wy_1 = w_1 * y_1;
    float wz_1 = w_1 * z_1;
    Matrix<float, 3, 3>  M_0 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_1 + z2_1), 2.0f * (xy_1 + wz_1), 2.0f * (xz_1 - wy_1), 2.0f * (xy_1 - wz_1), 1.0f - 2.0f * (x2_1 + z2_1), 2.0f * (yz_1 + wx_1), 2.0f * (xz_1 + wy_1), 2.0f * (yz_1 - wx_1), 1.0f - 2.0f * (x2_1 + y2_1))), makeMatrix<float, 3, 3> (scale_0.x, 0.0f, 0.0f, 0.0f, scale_0.y, 0.0f, 0.0f, 0.0f, scale_0.z));
    *covar_0 = mul_4(M_0, transpose_0(M_0));
    return;
}

inline __device__ Matrix<float, 2, 2>  inverse(Matrix<float, 2, 2>  m_0)
{
    float invdet_0 = 1.0f / (m_0.rows[int(0)].x * m_0.rows[int(1)].y - m_0.rows[int(0)].y * m_0.rows[int(1)].x);
    return makeMatrix<float, 2, 2> (m_0.rows[int(1)].y * invdet_0, - m_0.rows[int(0)].y * invdet_0, - m_0.rows[int(1)].x * invdet_0, m_0.rows[int(0)].x * invdet_0);
}

inline __device__ void _d_min_0(DiffPair_float_0 * dpx_3, DiffPair_float_0 * dpy_1, float dOut_7)
{
    DiffPair_float_0 _S9 = *dpx_3;
    float _S10;
    if(((*dpx_3).primal_0) < ((*dpy_1).primal_0))
    {
        _S10 = dOut_7;
    }
    else
    {
        if(((*dpx_3).primal_0) > ((*dpy_1).primal_0))
        {
            _S10 = 0.0f;
        }
        else
        {
            _S10 = 0.5f * dOut_7;
        }
    }
    dpx_3->primal_0 = _S9.primal_0;
    dpx_3->differential_0 = _S10;
    DiffPair_float_0 _S11 = *dpy_1;
    if(((*dpy_1).primal_0) < (_S9.primal_0))
    {
        _S10 = dOut_7;
    }
    else
    {
        if(((*dpy_1).primal_0) > ((*dpx_3).primal_0))
        {
            _S10 = 0.0f;
        }
        else
        {
            _S10 = 0.5f * dOut_7;
        }
    }
    dpy_1->primal_0 = _S11.primal_0;
    dpy_1->differential_0 = _S10;
    return;
}

inline __device__ void persp_proj_3dgs(float3  mean3d_0, Matrix<float, 3, 3>  cov3d_0, float fx_0, float fy_0, float cx_0, float cy_0, uint width_0, uint height_0, Matrix<float, 2, 2>  * cov2d_0, float2  * mean2d_0)
{
    float _S12 = float(width_0);
    float _S13 = float(height_0);
    float _S14 = 0.30000001192092896f * (0.5f * _S12 / fx_0);
    float _S15 = 0.30000001192092896f * (0.5f * _S13 / fy_0);
    float rz_0 = 1.0f / mean3d_0.z;
    float rz2_0 = rz_0 * rz_0;
    Matrix<float, 2, 3>  J_0 = makeMatrix<float, 2, 3> (fx_0 * rz_0, 0.0f, - fx_0 * (mean3d_0.z * (F32_min(((_S12 - cx_0) / fx_0 + _S14), ((F32_max((- (cx_0 / fx_0 + _S14)), (mean3d_0.x * rz_0))))))) * rz2_0, 0.0f, fy_0 * rz_0, - fy_0 * (mean3d_0.z * (F32_min(((_S13 - cy_0) / fy_0 + _S15), ((F32_max((- (cy_0 / fy_0 + _S15)), (mean3d_0.y * rz_0))))))) * rz2_0);
    *cov2d_0 = mul_6(mul_5(J_0, cov3d_0), transpose_1(J_0));
    *mean2d_0 = make_float2 (fx_0 * mean3d_0.x * rz_0 + cx_0, fy_0 * mean3d_0.y * rz_0 + cy_0);
    return;
}

inline __device__ void persp_proj_3dgs(float3  mean3d_1, Matrix<float, 3, 3>  cov3d_1, float fx_1, float fy_1, float cx_1, float cy_1, Matrix<float, 2, 2>  * cov2d_1, float2  * mean2d_1)
{
    float rz_1 = 1.0f / mean3d_1.z;
    float rz2_1 = rz_1 * rz_1;
    Matrix<float, 2, 3>  J_1 = makeMatrix<float, 2, 3> (fx_1 * rz_1, 0.0f, - fx_1 * mean3d_1.x * rz2_1, 0.0f, fy_1 * rz_1, - fy_1 * mean3d_1.y * rz2_1);
    *cov2d_1 = mul_6(mul_5(J_1, cov3d_1), transpose_1(J_1));
    *mean2d_1 = make_float2 (fx_1 * mean3d_1.x * rz_1 + cx_1, fy_1 * mean3d_1.y * rz_1 + cy_1);
    return;
}

inline __device__ float dot_0(float2  x_7, float2  y_2)
{
    int i_4 = int(0);
    float result_7 = 0.0f;
    for(;;)
    {
        if(i_4 < int(2))
        {
        }
        else
        {
            break;
        }
        float result_8 = result_7 + _slang_vector_get_element(x_7, i_4) * _slang_vector_get_element(y_2, i_4);
        i_4 = i_4 + int(1);
        result_7 = result_8;
    }
    return result_7;
}

inline __device__ float length_0(float2  x_8)
{
    return (F32_sqrt((dot_0(x_8, x_8))));
}

inline __device__ void _d_atan2_0(DiffPair_float_0 * dpy_2, DiffPair_float_0 * dpx_4, float dOut_8)
{
    DiffPair_float_0 _S16 = *dpx_4;
    float _S17 = - (*dpy_2).primal_0 / ((*dpx_4).primal_0 * (*dpx_4).primal_0 + (*dpy_2).primal_0 * (*dpy_2).primal_0) * dOut_8;
    dpx_4->primal_0 = (*dpx_4).primal_0;
    dpx_4->differential_0 = _S17;
    float _S18 = _S16.primal_0 / (_S16.primal_0 * _S16.primal_0 + (*dpy_2).primal_0 * (*dpy_2).primal_0) * dOut_8;
    dpy_2->primal_0 = (*dpy_2).primal_0;
    dpy_2->differential_0 = _S18;
    return;
}

inline __device__ void fisheye_proj_3dgs(float3  mean3d_2, Matrix<float, 3, 3>  cov3d_2, float fx_2, float fy_2, float cx_2, float cy_2, Matrix<float, 2, 2>  * cov2d_2, float2  * mean2d_2)
{
    float xy_len_0 = length_0(make_float2 (mean3d_2.x, mean3d_2.y)) + 1.00000001168609742e-07f;
    float theta_0 = (F32_atan2((xy_len_0), (mean3d_2.z + 1.00000001168609742e-07f)));
    *mean2d_2 = make_float2 (mean3d_2.x * fx_2 * theta_0 / xy_len_0 + cx_2, mean3d_2.y * fy_2 * theta_0 / xy_len_0 + cy_2);
    float x2_2 = mean3d_2.x * mean3d_2.x + 1.00000001168609742e-07f;
    float y2_2 = mean3d_2.y * mean3d_2.y;
    float xy_2 = mean3d_2.x * mean3d_2.y;
    float x2y2_0 = x2_2 + y2_2;
    float x2y2z2_inv_0 = 1.0f / (x2y2_0 + mean3d_2.z * mean3d_2.z);
    float b_0 = (F32_atan2((xy_len_0), (mean3d_2.z))) / xy_len_0 / x2y2_0;
    float a_0 = mean3d_2.z * x2y2z2_inv_0 / x2y2_0;
    float _S19 = a_0 - b_0;
    Matrix<float, 2, 3>  J_2 = makeMatrix<float, 2, 3> (fx_2 * (x2_2 * a_0 + y2_2 * b_0), fx_2 * xy_2 * _S19, - fx_2 * mean3d_2.x * x2y2z2_inv_0, fy_2 * xy_2 * _S19, fy_2 * (y2_2 * a_0 + x2_2 * b_0), - fy_2 * mean3d_2.y * x2y2z2_inv_0);
    *cov2d_2 = mul_6(mul_5(J_2, cov3d_2), transpose_1(J_2));
    return;
}

inline __device__ void ortho_proj_3dgs(float3  mean3d_3, Matrix<float, 3, 3>  cov3d_3, float fx_3, float fy_3, float cx_3, float cy_3, Matrix<float, 2, 2>  * cov2d_3, float2  * mean2d_3)
{
    Matrix<float, 2, 3>  J_3 = makeMatrix<float, 2, 3> (fx_3, 0.0f, 0.0f, 0.0f, fy_3, 0.0f);
    *cov2d_3 = mul_6(mul_5(J_3, cov3d_3), transpose_1(J_3));
    *mean2d_3 = make_float2 (fx_3 * mean3d_3.x + cx_3, fy_3 * mean3d_3.y + cy_3);
    return;
}

inline __device__ float add_blur(float eps2d_0, Matrix<float, 2, 2>  * covar_1, float * compensation_0)
{
    float det_orig_0 = *&((covar_1->rows + (int(0)))->x) * *&((covar_1->rows + (int(1)))->y) - *&((covar_1->rows + (int(0)))->y) * *&((covar_1->rows + (int(1)))->x);
    *&((covar_1->rows + (int(0)))->x) = *&((covar_1->rows + (int(0)))->x) + eps2d_0;
    float _S20 = *&((covar_1->rows + (int(1)))->y) + eps2d_0;
    *&((covar_1->rows + (int(1)))->y) = _S20;
    float det_blur_0 = *&((covar_1->rows + (int(0)))->x) * _S20 - *&((covar_1->rows + (int(0)))->y) * *&((covar_1->rows + (int(1)))->x);
    *compensation_0 = (F32_sqrt(((F32_max((0.0f), (det_orig_0 / det_blur_0))))));
    return det_blur_0;
}

inline __device__ void _d_exp_0(DiffPair_float_0 * dpx_5, float dOut_9)
{
    float _S21 = (F32_exp(((*dpx_5).primal_0))) * dOut_9;
    dpx_5->primal_0 = (*dpx_5).primal_0;
    dpx_5->differential_0 = _S21;
    return;
}

inline __device__ float3  exp_0(float3  x_9)
{
    float3  result_9;
    int i_5 = int(0);
    for(;;)
    {
        if(i_5 < int(3))
        {
        }
        else
        {
            break;
        }
        *_slang_vector_get_element_ptr(&result_9, i_5) = (F32_exp((_slang_vector_get_element(x_9, i_5))));
        i_5 = i_5 + int(1);
    }
    return result_9;
}

inline __device__ void _d_exp_vector_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dpx_6, float3  dOut_10)
{
    float3  _S22 = exp_0((*dpx_6).primal_0) * dOut_10;
    dpx_6->primal_0 = (*dpx_6).primal_0;
    dpx_6->differential_0 = _S22;
    return;
}

inline __device__ void _d_log_0(DiffPair_float_0 * dpx_7, float dOut_11)
{
    float _S23 = 1.0f / (*dpx_7).primal_0 * dOut_11;
    dpx_7->primal_0 = (*dpx_7).primal_0;
    dpx_7->differential_0 = _S23;
    return;
}

inline __device__ void projection_3dgs_persp(bool antialiased_0, float3  mean_0, float4  quat_2, float3  scale_1, float in_opacity_0, Matrix<float, 3, 3>  R_2, float3  t_1, float fx_4, float fy_4, float cx_4, float cy_4, uint image_width_0, uint image_height_0, float eps2d_1, float near_plane_0, float far_plane_0, float radius_clip_0, int2  * radii_0, float * depth_0, float2  * mean2d_4, float3  * conic_0, float * opacity_0)
{
    for(;;)
    {
        float3  mean_c_0 = mul_0(R_2, mean_0) + t_1;
        float _S24 = mean_c_0.z;
        bool _S25;
        if(_S24 < near_plane_0)
        {
            _S25 = true;
        }
        else
        {
            _S25 = _S24 > far_plane_0;
        }
        if(_S25)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        float3  _S26 = exp_0(scale_1);
        float x_10 = quat_2.y;
        float inv_norm_2 = (F32_rsqrt((x_10 * x_10 + quat_2.z * quat_2.z + quat_2.w * quat_2.w + quat_2.x * quat_2.x)));
        float x_11 = quat_2.y * inv_norm_2;
        float y_3 = quat_2.z * inv_norm_2;
        float z_2 = quat_2.w * inv_norm_2;
        float w_2 = quat_2.x * inv_norm_2;
        float x2_3 = x_11 * x_11;
        float y2_3 = y_3 * y_3;
        float z2_2 = z_2 * z_2;
        float xy_3 = x_11 * y_3;
        float xz_2 = x_11 * z_2;
        float yz_2 = y_3 * z_2;
        float wx_2 = w_2 * x_11;
        float wy_2 = w_2 * y_3;
        float wz_2 = w_2 * z_2;
        Matrix<float, 3, 3>  M_1 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_3 + z2_2), 2.0f * (xy_3 + wz_2), 2.0f * (xz_2 - wy_2), 2.0f * (xy_3 - wz_2), 1.0f - 2.0f * (x2_3 + z2_2), 2.0f * (yz_2 + wx_2), 2.0f * (xz_2 + wy_2), 2.0f * (yz_2 - wx_2), 1.0f - 2.0f * (x2_3 + y2_3))), makeMatrix<float, 3, 3> (_S26.x, 0.0f, 0.0f, 0.0f, _S26.y, 0.0f, 0.0f, 0.0f, _S26.z));
        Matrix<float, 3, 3>  covar_c_0 = mul_4(mul_4(R_2, mul_4(M_1, transpose_0(M_1))), transpose_0(R_2));
        Matrix<float, 2, 2>  covar2d_0;
        float _S27 = float(image_width_0);
        float _S28 = float(image_height_0);
        float _S29 = 0.30000001192092896f * (0.5f * _S27 / fx_4);
        float _S30 = 0.30000001192092896f * (0.5f * _S28 / fy_4);
        float rz_2 = 1.0f / mean_c_0.z;
        float rz2_2 = rz_2 * rz_2;
        Matrix<float, 2, 3>  J_4 = makeMatrix<float, 2, 3> (fx_4 * rz_2, 0.0f, - fx_4 * (mean_c_0.z * (F32_min(((_S27 - cx_4) / fx_4 + _S29), ((F32_max((- (cx_4 / fx_4 + _S29)), (mean_c_0.x * rz_2))))))) * rz2_2, 0.0f, fy_4 * rz_2, - fy_4 * (mean_c_0.z * (F32_min(((_S28 - cy_4) / fy_4 + _S30), ((F32_max((- (cy_4 / fy_4 + _S30)), (mean_c_0.y * rz_2))))))) * rz2_2);
        covar2d_0 = mul_6(mul_5(J_4, covar_c_0), transpose_1(J_4));
        *mean2d_4 = make_float2 (fx_4 * mean_c_0.x * rz_2 + cx_4, fy_4 * mean_c_0.y * rz_2 + cy_4);
        float det_orig_1 = *&(((&covar2d_0)->rows + (int(0)))->x) * *&(((&covar2d_0)->rows + (int(1)))->y) - *&(((&covar2d_0)->rows + (int(0)))->y) * *&(((&covar2d_0)->rows + (int(1)))->x);
        *&(((&covar2d_0)->rows + (int(0)))->x) = *&(((&covar2d_0)->rows + (int(0)))->x) + eps2d_1;
        float _S31 = *&(((&covar2d_0)->rows + (int(1)))->y) + eps2d_1;
        *&(((&covar2d_0)->rows + (int(1)))->y) = _S31;
        float det_blur_1 = *&(((&covar2d_0)->rows + (int(0)))->x) * _S31 - *&(((&covar2d_0)->rows + (int(0)))->y) * *&(((&covar2d_0)->rows + (int(1)))->x);
        float compensation_1 = (F32_sqrt(((F32_max((0.0f), (det_orig_1 / det_blur_1))))));
        if(det_blur_1 <= 0.0f)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        float invdet_1 = 1.0f / (covar2d_0.rows[int(0)].x * covar2d_0.rows[int(1)].y - covar2d_0.rows[int(0)].y * covar2d_0.rows[int(1)].x);
        Matrix<float, 2, 2>  _S32 = makeMatrix<float, 2, 2> (covar2d_0.rows[int(1)].y * invdet_1, - covar2d_0.rows[int(0)].y * invdet_1, - covar2d_0.rows[int(1)].x * invdet_1, covar2d_0.rows[int(0)].x * invdet_1);
        *opacity_0 = 1.0f / (1.0f + (F32_exp((- in_opacity_0))));
        if(antialiased_0)
        {
            *opacity_0 = *opacity_0 * compensation_1;
        }
        if((*opacity_0) < 0.00392156885936856f)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        float extend_0 = (F32_min((3.32999992370605469f), ((F32_sqrt((2.0f * (F32_log((*opacity_0 / 0.00392156885936856f)))))))));
        float radius_x_0 = (F32_ceil((extend_0 * (F32_sqrt((covar2d_0[int(0)].x))))));
        float radius_y_0 = (F32_ceil((extend_0 * (F32_sqrt((covar2d_0[int(1)].y))))));
        if(radius_x_0 <= radius_clip_0)
        {
            _S25 = radius_y_0 <= radius_clip_0;
        }
        else
        {
            _S25 = false;
        }
        if(_S25)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        if(((*mean2d_4).x + radius_x_0) <= 0.0f)
        {
            _S25 = true;
        }
        else
        {
            _S25 = ((*mean2d_4).x - radius_x_0) >= _S27;
        }
        if(_S25)
        {
            _S25 = true;
        }
        else
        {
            _S25 = ((*mean2d_4).y + radius_y_0) <= 0.0f;
        }
        if(_S25)
        {
            _S25 = true;
        }
        else
        {
            _S25 = ((*mean2d_4).y - radius_y_0) >= _S28;
        }
        if(_S25)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        *radii_0 = make_int2 (int(radius_x_0), int(radius_y_0));
        *depth_0 = _S24;
        *conic_0 = make_float3 (_S32.rows[int(0)].x, _S32.rows[int(0)].y, _S32.rows[int(1)].y);
        break;
    }
    return;
}

inline __device__ void projection_3dgs_fisheye(bool antialiased_1, float3  mean_1, float4  quat_3, float3  scale_2, float in_opacity_1, Matrix<float, 3, 3>  R_3, float3  t_2, float fx_5, float fy_5, float cx_5, float cy_5, uint image_width_1, uint image_height_1, float eps2d_2, float near_plane_1, float far_plane_1, float radius_clip_1, int2  * radii_1, float * depth_1, float2  * mean2d_5, float3  * conic_1, float * opacity_1)
{
    for(;;)
    {
        float3  mean_c_1 = mul_0(R_3, mean_1) + t_2;
        float _S33 = mean_c_1.z;
        bool _S34;
        if(_S33 < near_plane_1)
        {
            _S34 = true;
        }
        else
        {
            _S34 = _S33 > far_plane_1;
        }
        if(_S34)
        {
            *radii_1 = make_int2 (int(0), int(0));
            break;
        }
        float3  _S35 = exp_0(scale_2);
        float x_12 = quat_3.y;
        float inv_norm_3 = (F32_rsqrt((x_12 * x_12 + quat_3.z * quat_3.z + quat_3.w * quat_3.w + quat_3.x * quat_3.x)));
        float x_13 = quat_3.y * inv_norm_3;
        float y_4 = quat_3.z * inv_norm_3;
        float z_3 = quat_3.w * inv_norm_3;
        float w_3 = quat_3.x * inv_norm_3;
        float x2_4 = x_13 * x_13;
        float y2_4 = y_4 * y_4;
        float z2_3 = z_3 * z_3;
        float xy_4 = x_13 * y_4;
        float xz_3 = x_13 * z_3;
        float yz_3 = y_4 * z_3;
        float wx_3 = w_3 * x_13;
        float wy_3 = w_3 * y_4;
        float wz_3 = w_3 * z_3;
        Matrix<float, 3, 3>  M_2 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_4 + z2_3), 2.0f * (xy_4 + wz_3), 2.0f * (xz_3 - wy_3), 2.0f * (xy_4 - wz_3), 1.0f - 2.0f * (x2_4 + z2_3), 2.0f * (yz_3 + wx_3), 2.0f * (xz_3 + wy_3), 2.0f * (yz_3 - wx_3), 1.0f - 2.0f * (x2_4 + y2_4))), makeMatrix<float, 3, 3> (_S35.x, 0.0f, 0.0f, 0.0f, _S35.y, 0.0f, 0.0f, 0.0f, _S35.z));
        Matrix<float, 3, 3>  covar_c_1 = mul_4(mul_4(R_3, mul_4(M_2, transpose_0(M_2))), transpose_0(R_3));
        Matrix<float, 2, 2>  covar2d_1;
        float xy_len_1 = length_0(make_float2 (mean_c_1.x, mean_c_1.y)) + 1.00000001168609742e-07f;
        float theta_1 = (F32_atan2((xy_len_1), (mean_c_1.z + 1.00000001168609742e-07f)));
        *mean2d_5 = make_float2 (mean_c_1.x * fx_5 * theta_1 / xy_len_1 + cx_5, mean_c_1.y * fy_5 * theta_1 / xy_len_1 + cy_5);
        float x2_5 = mean_c_1.x * mean_c_1.x + 1.00000001168609742e-07f;
        float y2_5 = mean_c_1.y * mean_c_1.y;
        float xy_5 = mean_c_1.x * mean_c_1.y;
        float x2y2_1 = x2_5 + y2_5;
        float x2y2z2_inv_1 = 1.0f / (x2y2_1 + mean_c_1.z * mean_c_1.z);
        float b_1 = (F32_atan2((xy_len_1), (mean_c_1.z))) / xy_len_1 / x2y2_1;
        float a_1 = mean_c_1.z * x2y2z2_inv_1 / x2y2_1;
        float _S36 = a_1 - b_1;
        Matrix<float, 2, 3>  J_5 = makeMatrix<float, 2, 3> (fx_5 * (x2_5 * a_1 + y2_5 * b_1), fx_5 * xy_5 * _S36, - fx_5 * mean_c_1.x * x2y2z2_inv_1, fy_5 * xy_5 * _S36, fy_5 * (y2_5 * a_1 + x2_5 * b_1), - fy_5 * mean_c_1.y * x2y2z2_inv_1);
        covar2d_1 = mul_6(mul_5(J_5, covar_c_1), transpose_1(J_5));
        float det_orig_2 = *&(((&covar2d_1)->rows + (int(0)))->x) * *&(((&covar2d_1)->rows + (int(1)))->y) - *&(((&covar2d_1)->rows + (int(0)))->y) * *&(((&covar2d_1)->rows + (int(1)))->x);
        *&(((&covar2d_1)->rows + (int(0)))->x) = *&(((&covar2d_1)->rows + (int(0)))->x) + eps2d_2;
        float _S37 = *&(((&covar2d_1)->rows + (int(1)))->y) + eps2d_2;
        *&(((&covar2d_1)->rows + (int(1)))->y) = _S37;
        float det_blur_2 = *&(((&covar2d_1)->rows + (int(0)))->x) * _S37 - *&(((&covar2d_1)->rows + (int(0)))->y) * *&(((&covar2d_1)->rows + (int(1)))->x);
        float compensation_2 = (F32_sqrt(((F32_max((0.0f), (det_orig_2 / det_blur_2))))));
        if(det_blur_2 <= 0.0f)
        {
            *radii_1 = make_int2 (int(0), int(0));
            break;
        }
        float invdet_2 = 1.0f / (covar2d_1.rows[int(0)].x * covar2d_1.rows[int(1)].y - covar2d_1.rows[int(0)].y * covar2d_1.rows[int(1)].x);
        Matrix<float, 2, 2>  _S38 = makeMatrix<float, 2, 2> (covar2d_1.rows[int(1)].y * invdet_2, - covar2d_1.rows[int(0)].y * invdet_2, - covar2d_1.rows[int(1)].x * invdet_2, covar2d_1.rows[int(0)].x * invdet_2);
        *opacity_1 = 1.0f / (1.0f + (F32_exp((- in_opacity_1))));
        if(antialiased_1)
        {
            *opacity_1 = *opacity_1 * compensation_2;
        }
        if((*opacity_1) < 0.00392156885936856f)
        {
            *radii_1 = make_int2 (int(0), int(0));
            break;
        }
        float extend_1 = (F32_min((3.32999992370605469f), ((F32_sqrt((2.0f * (F32_log((*opacity_1 / 0.00392156885936856f)))))))));
        float radius_x_1 = (F32_ceil((extend_1 * (F32_sqrt((covar2d_1[int(0)].x))))));
        float radius_y_1 = (F32_ceil((extend_1 * (F32_sqrt((covar2d_1[int(1)].y))))));
        if(radius_x_1 <= radius_clip_1)
        {
            _S34 = radius_y_1 <= radius_clip_1;
        }
        else
        {
            _S34 = false;
        }
        if(_S34)
        {
            *radii_1 = make_int2 (int(0), int(0));
            break;
        }
        if(((*mean2d_5).x + radius_x_1) <= 0.0f)
        {
            _S34 = true;
        }
        else
        {
            _S34 = ((*mean2d_5).x - radius_x_1) >= float(image_width_1);
        }
        if(_S34)
        {
            _S34 = true;
        }
        else
        {
            _S34 = ((*mean2d_5).y + radius_y_1) <= 0.0f;
        }
        if(_S34)
        {
            _S34 = true;
        }
        else
        {
            _S34 = ((*mean2d_5).y - radius_y_1) >= float(image_height_1);
        }
        if(_S34)
        {
            *radii_1 = make_int2 (int(0), int(0));
            break;
        }
        *radii_1 = make_int2 (int(radius_x_1), int(radius_y_1));
        *depth_1 = _S33;
        *conic_1 = make_float3 (_S38.rows[int(0)].x, _S38.rows[int(0)].y, _S38.rows[int(1)].y);
        break;
    }
    return;
}

inline __device__ void projection_3dgs_ortho(bool antialiased_2, float3  mean_2, float4  quat_4, float3  scale_3, float in_opacity_2, Matrix<float, 3, 3>  R_4, float3  t_3, float fx_6, float fy_6, float cx_6, float cy_6, uint image_width_2, uint image_height_2, float eps2d_3, float near_plane_2, float far_plane_2, float radius_clip_2, int2  * radii_2, float * depth_2, float2  * mean2d_6, float3  * conic_2, float * opacity_2)
{
    for(;;)
    {
        float3  mean_c_2 = mul_0(R_4, mean_2) + t_3;
        float _S39 = mean_c_2.z;
        bool _S40;
        if(_S39 < near_plane_2)
        {
            _S40 = true;
        }
        else
        {
            _S40 = _S39 > far_plane_2;
        }
        if(_S40)
        {
            *radii_2 = make_int2 (int(0), int(0));
            break;
        }
        float3  _S41 = exp_0(scale_3);
        float x_14 = quat_4.y;
        float inv_norm_4 = (F32_rsqrt((x_14 * x_14 + quat_4.z * quat_4.z + quat_4.w * quat_4.w + quat_4.x * quat_4.x)));
        float x_15 = quat_4.y * inv_norm_4;
        float y_5 = quat_4.z * inv_norm_4;
        float z_4 = quat_4.w * inv_norm_4;
        float w_4 = quat_4.x * inv_norm_4;
        float x2_6 = x_15 * x_15;
        float y2_6 = y_5 * y_5;
        float z2_4 = z_4 * z_4;
        float xy_6 = x_15 * y_5;
        float xz_4 = x_15 * z_4;
        float yz_4 = y_5 * z_4;
        float wx_4 = w_4 * x_15;
        float wy_4 = w_4 * y_5;
        float wz_4 = w_4 * z_4;
        Matrix<float, 3, 3>  M_3 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_6 + z2_4), 2.0f * (xy_6 + wz_4), 2.0f * (xz_4 - wy_4), 2.0f * (xy_6 - wz_4), 1.0f - 2.0f * (x2_6 + z2_4), 2.0f * (yz_4 + wx_4), 2.0f * (xz_4 + wy_4), 2.0f * (yz_4 - wx_4), 1.0f - 2.0f * (x2_6 + y2_6))), makeMatrix<float, 3, 3> (_S41.x, 0.0f, 0.0f, 0.0f, _S41.y, 0.0f, 0.0f, 0.0f, _S41.z));
        Matrix<float, 3, 3>  covar_c_2 = mul_4(mul_4(R_4, mul_4(M_3, transpose_0(M_3))), transpose_0(R_4));
        Matrix<float, 2, 2>  covar2d_2;
        Matrix<float, 2, 3>  J_6 = makeMatrix<float, 2, 3> (fx_6, 0.0f, 0.0f, 0.0f, fy_6, 0.0f);
        covar2d_2 = mul_6(mul_5(J_6, covar_c_2), transpose_1(J_6));
        *mean2d_6 = make_float2 (fx_6 * mean_c_2.x + cx_6, fy_6 * mean_c_2.y + cy_6);
        float det_orig_3 = *&(((&covar2d_2)->rows + (int(0)))->x) * *&(((&covar2d_2)->rows + (int(1)))->y) - *&(((&covar2d_2)->rows + (int(0)))->y) * *&(((&covar2d_2)->rows + (int(1)))->x);
        *&(((&covar2d_2)->rows + (int(0)))->x) = *&(((&covar2d_2)->rows + (int(0)))->x) + eps2d_3;
        float _S42 = *&(((&covar2d_2)->rows + (int(1)))->y) + eps2d_3;
        *&(((&covar2d_2)->rows + (int(1)))->y) = _S42;
        float det_blur_3 = *&(((&covar2d_2)->rows + (int(0)))->x) * _S42 - *&(((&covar2d_2)->rows + (int(0)))->y) * *&(((&covar2d_2)->rows + (int(1)))->x);
        float compensation_3 = (F32_sqrt(((F32_max((0.0f), (det_orig_3 / det_blur_3))))));
        if(det_blur_3 <= 0.0f)
        {
            *radii_2 = make_int2 (int(0), int(0));
            break;
        }
        float invdet_3 = 1.0f / (covar2d_2.rows[int(0)].x * covar2d_2.rows[int(1)].y - covar2d_2.rows[int(0)].y * covar2d_2.rows[int(1)].x);
        Matrix<float, 2, 2>  _S43 = makeMatrix<float, 2, 2> (covar2d_2.rows[int(1)].y * invdet_3, - covar2d_2.rows[int(0)].y * invdet_3, - covar2d_2.rows[int(1)].x * invdet_3, covar2d_2.rows[int(0)].x * invdet_3);
        *opacity_2 = 1.0f / (1.0f + (F32_exp((- in_opacity_2))));
        if(antialiased_2)
        {
            *opacity_2 = *opacity_2 * compensation_3;
        }
        if((*opacity_2) < 0.00392156885936856f)
        {
            *radii_2 = make_int2 (int(0), int(0));
            break;
        }
        float extend_2 = (F32_min((3.32999992370605469f), ((F32_sqrt((2.0f * (F32_log((*opacity_2 / 0.00392156885936856f)))))))));
        float radius_x_2 = (F32_ceil((extend_2 * (F32_sqrt((covar2d_2[int(0)].x))))));
        float radius_y_2 = (F32_ceil((extend_2 * (F32_sqrt((covar2d_2[int(1)].y))))));
        if(radius_x_2 <= radius_clip_2)
        {
            _S40 = radius_y_2 <= radius_clip_2;
        }
        else
        {
            _S40 = false;
        }
        if(_S40)
        {
            *radii_2 = make_int2 (int(0), int(0));
            break;
        }
        if(((*mean2d_6).x + radius_x_2) <= 0.0f)
        {
            _S40 = true;
        }
        else
        {
            _S40 = ((*mean2d_6).x - radius_x_2) >= float(image_width_2);
        }
        if(_S40)
        {
            _S40 = true;
        }
        else
        {
            _S40 = ((*mean2d_6).y + radius_y_2) <= 0.0f;
        }
        if(_S40)
        {
            _S40 = true;
        }
        else
        {
            _S40 = ((*mean2d_6).y - radius_y_2) >= float(image_height_2);
        }
        if(_S40)
        {
            *radii_2 = make_int2 (int(0), int(0));
            break;
        }
        *radii_2 = make_int2 (int(radius_x_2), int(radius_y_2));
        *depth_2 = _S39;
        *conic_2 = make_float3 (_S43.rows[int(0)].x, _S43.rows[int(0)].y, _S43.rows[int(1)].y);
        break;
    }
    return;
}

inline __device__ void _projection_3dgs_persp_differentiable(bool antialiased_3, float3  mean_3, float4  quat_5, float3  scale_4, float in_opacity_3, Matrix<float, 3, 3>  R_5, float3  t_4, float fx_7, float fy_7, float cx_7, float cy_7, uint image_width_3, uint image_height_3, float eps2d_4, float near_plane_3, float far_plane_3, float radius_clip_3, int2  * radii_3, float * depth_3, float2  * mean2d_7, float3  * conic_3, float * opacity_3)
{
    float3  mean_c_3 = mul_0(R_5, mean_3) + t_4;
    float3  _S44 = exp_0(scale_4);
    float x_16 = quat_5.y;
    float inv_norm_5 = (F32_rsqrt((x_16 * x_16 + quat_5.z * quat_5.z + quat_5.w * quat_5.w + quat_5.x * quat_5.x)));
    float x_17 = quat_5.y * inv_norm_5;
    float y_6 = quat_5.z * inv_norm_5;
    float z_5 = quat_5.w * inv_norm_5;
    float w_5 = quat_5.x * inv_norm_5;
    float x2_7 = x_17 * x_17;
    float y2_7 = y_6 * y_6;
    float z2_5 = z_5 * z_5;
    float xy_7 = x_17 * y_6;
    float xz_5 = x_17 * z_5;
    float yz_5 = y_6 * z_5;
    float wx_5 = w_5 * x_17;
    float wy_5 = w_5 * y_6;
    float wz_5 = w_5 * z_5;
    Matrix<float, 3, 3>  M_4 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_7 + z2_5), 2.0f * (xy_7 + wz_5), 2.0f * (xz_5 - wy_5), 2.0f * (xy_7 - wz_5), 1.0f - 2.0f * (x2_7 + z2_5), 2.0f * (yz_5 + wx_5), 2.0f * (xz_5 + wy_5), 2.0f * (yz_5 - wx_5), 1.0f - 2.0f * (x2_7 + y2_7))), makeMatrix<float, 3, 3> (_S44.x, 0.0f, 0.0f, 0.0f, _S44.y, 0.0f, 0.0f, 0.0f, _S44.z));
    float _S45 = float(image_width_3);
    float _S46 = float(image_height_3);
    float _S47 = 0.30000001192092896f * (0.5f * _S45 / fx_7);
    float _S48 = 0.30000001192092896f * (0.5f * _S46 / fy_7);
    float rz_3 = 1.0f / mean_c_3.z;
    float rz2_3 = rz_3 * rz_3;
    Matrix<float, 2, 3>  J_7 = makeMatrix<float, 2, 3> (fx_7 * rz_3, 0.0f, - fx_7 * (mean_c_3.z * (F32_min(((_S45 - cx_7) / fx_7 + _S47), ((F32_max((- (cx_7 / fx_7 + _S47)), (mean_c_3.x * rz_3))))))) * rz2_3, 0.0f, fy_7 * rz_3, - fy_7 * (mean_c_3.z * (F32_min(((_S46 - cy_7) / fy_7 + _S48), ((F32_max((- (cy_7 / fy_7 + _S48)), (mean_c_3.y * rz_3))))))) * rz2_3);
    Matrix<float, 2, 2>  covar2d_3 = mul_6(mul_5(J_7, mul_4(mul_4(R_5, mul_4(M_4, transpose_0(M_4))), transpose_0(R_5))), transpose_1(J_7));
    *mean2d_7 = make_float2 (fx_7 * mean_c_3.x * rz_3 + cx_7, fy_7 * mean_c_3.y * rz_3 + cy_7);
    float det_orig_4 = *&(((&covar2d_3)->rows + (int(0)))->x) * *&(((&covar2d_3)->rows + (int(1)))->y) - *&(((&covar2d_3)->rows + (int(0)))->y) * *&(((&covar2d_3)->rows + (int(1)))->x);
    *&(((&covar2d_3)->rows + (int(0)))->x) = *&(((&covar2d_3)->rows + (int(0)))->x) + eps2d_4;
    float _S49 = *&(((&covar2d_3)->rows + (int(1)))->y) + eps2d_4;
    *&(((&covar2d_3)->rows + (int(1)))->y) = _S49;
    float compensation_4 = (F32_sqrt(((F32_max((0.0f), (det_orig_4 / (*&(((&covar2d_3)->rows + (int(0)))->x) * _S49 - *&(((&covar2d_3)->rows + (int(0)))->y) * *&(((&covar2d_3)->rows + (int(1)))->x))))))));
    float invdet_4 = 1.0f / (covar2d_3.rows[int(0)].x * covar2d_3.rows[int(1)].y - covar2d_3.rows[int(0)].y * covar2d_3.rows[int(1)].x);
    Matrix<float, 2, 2>  _S50 = makeMatrix<float, 2, 2> (covar2d_3.rows[int(1)].y * invdet_4, - covar2d_3.rows[int(0)].y * invdet_4, - covar2d_3.rows[int(1)].x * invdet_4, covar2d_3.rows[int(0)].x * invdet_4);
    *opacity_3 = 1.0f / (1.0f + (F32_exp((- in_opacity_3))));
    if(antialiased_3)
    {
        *opacity_3 = *opacity_3 * compensation_4;
    }
    float extend_3 = (F32_min((3.32999992370605469f), ((F32_sqrt((2.0f * (F32_log((*opacity_3 / 0.00392156885936856f)))))))));
    *radii_3 = make_int2 (int((F32_ceil((extend_3 * (F32_sqrt((covar2d_3[int(0)].x))))))), int((F32_ceil((extend_3 * (F32_sqrt((covar2d_3[int(1)].y))))))));
    *depth_3 = mean_c_3.z;
    *conic_3 = make_float3 (_S50.rows[int(0)].x, _S50.rows[int(0)].y, _S50.rows[int(1)].y);
    return;
}

inline __device__ void _projection_3dgs_fisheye_differentiable(bool antialiased_4, float3  mean_4, float4  quat_6, float3  scale_5, float in_opacity_4, Matrix<float, 3, 3>  R_6, float3  t_5, float fx_8, float fy_8, float cx_8, float cy_8, uint image_width_4, uint image_height_4, float eps2d_5, float near_plane_4, float far_plane_4, float radius_clip_4, int2  * radii_4, float * depth_4, float2  * mean2d_8, float3  * conic_4, float * opacity_4)
{
    float3  mean_c_4 = mul_0(R_6, mean_4) + t_5;
    float3  _S51 = exp_0(scale_5);
    float x_18 = quat_6.y;
    float inv_norm_6 = (F32_rsqrt((x_18 * x_18 + quat_6.z * quat_6.z + quat_6.w * quat_6.w + quat_6.x * quat_6.x)));
    float x_19 = quat_6.y * inv_norm_6;
    float y_7 = quat_6.z * inv_norm_6;
    float z_6 = quat_6.w * inv_norm_6;
    float w_6 = quat_6.x * inv_norm_6;
    float x2_8 = x_19 * x_19;
    float y2_8 = y_7 * y_7;
    float z2_6 = z_6 * z_6;
    float xy_8 = x_19 * y_7;
    float xz_6 = x_19 * z_6;
    float yz_6 = y_7 * z_6;
    float wx_6 = w_6 * x_19;
    float wy_6 = w_6 * y_7;
    float wz_6 = w_6 * z_6;
    Matrix<float, 3, 3>  M_5 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_8 + z2_6), 2.0f * (xy_8 + wz_6), 2.0f * (xz_6 - wy_6), 2.0f * (xy_8 - wz_6), 1.0f - 2.0f * (x2_8 + z2_6), 2.0f * (yz_6 + wx_6), 2.0f * (xz_6 + wy_6), 2.0f * (yz_6 - wx_6), 1.0f - 2.0f * (x2_8 + y2_8))), makeMatrix<float, 3, 3> (_S51.x, 0.0f, 0.0f, 0.0f, _S51.y, 0.0f, 0.0f, 0.0f, _S51.z));
    Matrix<float, 3, 3>  covar_c_3 = mul_4(mul_4(R_6, mul_4(M_5, transpose_0(M_5))), transpose_0(R_6));
    float xy_len_2 = length_0(make_float2 (mean_c_4.x, mean_c_4.y)) + 1.00000001168609742e-07f;
    float theta_2 = (F32_atan2((xy_len_2), (mean_c_4.z + 1.00000001168609742e-07f)));
    *mean2d_8 = make_float2 (mean_c_4.x * fx_8 * theta_2 / xy_len_2 + cx_8, mean_c_4.y * fy_8 * theta_2 / xy_len_2 + cy_8);
    float x2_9 = mean_c_4.x * mean_c_4.x + 1.00000001168609742e-07f;
    float y2_9 = mean_c_4.y * mean_c_4.y;
    float xy_9 = mean_c_4.x * mean_c_4.y;
    float x2y2_2 = x2_9 + y2_9;
    float x2y2z2_inv_2 = 1.0f / (x2y2_2 + mean_c_4.z * mean_c_4.z);
    float b_2 = (F32_atan2((xy_len_2), (mean_c_4.z))) / xy_len_2 / x2y2_2;
    float a_2 = mean_c_4.z * x2y2z2_inv_2 / x2y2_2;
    float _S52 = a_2 - b_2;
    Matrix<float, 2, 3>  J_8 = makeMatrix<float, 2, 3> (fx_8 * (x2_9 * a_2 + y2_9 * b_2), fx_8 * xy_9 * _S52, - fx_8 * mean_c_4.x * x2y2z2_inv_2, fy_8 * xy_9 * _S52, fy_8 * (y2_9 * a_2 + x2_9 * b_2), - fy_8 * mean_c_4.y * x2y2z2_inv_2);
    Matrix<float, 2, 2>  covar2d_4 = mul_6(mul_5(J_8, covar_c_3), transpose_1(J_8));
    float det_orig_5 = *&(((&covar2d_4)->rows + (int(0)))->x) * *&(((&covar2d_4)->rows + (int(1)))->y) - *&(((&covar2d_4)->rows + (int(0)))->y) * *&(((&covar2d_4)->rows + (int(1)))->x);
    *&(((&covar2d_4)->rows + (int(0)))->x) = *&(((&covar2d_4)->rows + (int(0)))->x) + eps2d_5;
    float _S53 = *&(((&covar2d_4)->rows + (int(1)))->y) + eps2d_5;
    *&(((&covar2d_4)->rows + (int(1)))->y) = _S53;
    float compensation_5 = (F32_sqrt(((F32_max((0.0f), (det_orig_5 / (*&(((&covar2d_4)->rows + (int(0)))->x) * _S53 - *&(((&covar2d_4)->rows + (int(0)))->y) * *&(((&covar2d_4)->rows + (int(1)))->x))))))));
    float invdet_5 = 1.0f / (covar2d_4.rows[int(0)].x * covar2d_4.rows[int(1)].y - covar2d_4.rows[int(0)].y * covar2d_4.rows[int(1)].x);
    Matrix<float, 2, 2>  _S54 = makeMatrix<float, 2, 2> (covar2d_4.rows[int(1)].y * invdet_5, - covar2d_4.rows[int(0)].y * invdet_5, - covar2d_4.rows[int(1)].x * invdet_5, covar2d_4.rows[int(0)].x * invdet_5);
    *opacity_4 = 1.0f / (1.0f + (F32_exp((- in_opacity_4))));
    if(antialiased_4)
    {
        *opacity_4 = *opacity_4 * compensation_5;
    }
    float extend_4 = (F32_min((3.32999992370605469f), ((F32_sqrt((2.0f * (F32_log((*opacity_4 / 0.00392156885936856f)))))))));
    *radii_4 = make_int2 (int((F32_ceil((extend_4 * (F32_sqrt((covar2d_4[int(0)].x))))))), int((F32_ceil((extend_4 * (F32_sqrt((covar2d_4[int(1)].y))))))));
    *depth_4 = mean_c_4.z;
    *conic_4 = make_float3 (_S54.rows[int(0)].x, _S54.rows[int(0)].y, _S54.rows[int(1)].y);
    return;
}

inline __device__ void _projection_3dgs_ortho_differentiable(bool antialiased_5, float3  mean_5, float4  quat_7, float3  scale_6, float in_opacity_5, Matrix<float, 3, 3>  R_7, float3  t_6, float fx_9, float fy_9, float cx_9, float cy_9, uint image_width_5, uint image_height_5, float eps2d_6, float near_plane_5, float far_plane_5, float radius_clip_5, int2  * radii_5, float * depth_5, float2  * mean2d_9, float3  * conic_5, float * opacity_5)
{
    float3  mean_c_5 = mul_0(R_7, mean_5) + t_6;
    float3  _S55 = exp_0(scale_6);
    float x_20 = quat_7.y;
    float inv_norm_7 = (F32_rsqrt((x_20 * x_20 + quat_7.z * quat_7.z + quat_7.w * quat_7.w + quat_7.x * quat_7.x)));
    float x_21 = quat_7.y * inv_norm_7;
    float y_8 = quat_7.z * inv_norm_7;
    float z_7 = quat_7.w * inv_norm_7;
    float w_7 = quat_7.x * inv_norm_7;
    float x2_10 = x_21 * x_21;
    float y2_10 = y_8 * y_8;
    float z2_7 = z_7 * z_7;
    float xy_10 = x_21 * y_8;
    float xz_7 = x_21 * z_7;
    float yz_7 = y_8 * z_7;
    float wx_7 = w_7 * x_21;
    float wy_7 = w_7 * y_8;
    float wz_7 = w_7 * z_7;
    Matrix<float, 3, 3>  M_6 = mul_4(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_10 + z2_7), 2.0f * (xy_10 + wz_7), 2.0f * (xz_7 - wy_7), 2.0f * (xy_10 - wz_7), 1.0f - 2.0f * (x2_10 + z2_7), 2.0f * (yz_7 + wx_7), 2.0f * (xz_7 + wy_7), 2.0f * (yz_7 - wx_7), 1.0f - 2.0f * (x2_10 + y2_10))), makeMatrix<float, 3, 3> (_S55.x, 0.0f, 0.0f, 0.0f, _S55.y, 0.0f, 0.0f, 0.0f, _S55.z));
    Matrix<float, 2, 3>  J_9 = makeMatrix<float, 2, 3> (fx_9, 0.0f, 0.0f, 0.0f, fy_9, 0.0f);
    Matrix<float, 2, 2>  covar2d_5 = mul_6(mul_5(J_9, mul_4(mul_4(R_7, mul_4(M_6, transpose_0(M_6))), transpose_0(R_7))), transpose_1(J_9));
    *mean2d_9 = make_float2 (fx_9 * mean_c_5.x + cx_9, fy_9 * mean_c_5.y + cy_9);
    float det_orig_6 = *&(((&covar2d_5)->rows + (int(0)))->x) * *&(((&covar2d_5)->rows + (int(1)))->y) - *&(((&covar2d_5)->rows + (int(0)))->y) * *&(((&covar2d_5)->rows + (int(1)))->x);
    *&(((&covar2d_5)->rows + (int(0)))->x) = *&(((&covar2d_5)->rows + (int(0)))->x) + eps2d_6;
    float _S56 = *&(((&covar2d_5)->rows + (int(1)))->y) + eps2d_6;
    *&(((&covar2d_5)->rows + (int(1)))->y) = _S56;
    float compensation_6 = (F32_sqrt(((F32_max((0.0f), (det_orig_6 / (*&(((&covar2d_5)->rows + (int(0)))->x) * _S56 - *&(((&covar2d_5)->rows + (int(0)))->y) * *&(((&covar2d_5)->rows + (int(1)))->x))))))));
    float invdet_6 = 1.0f / (covar2d_5.rows[int(0)].x * covar2d_5.rows[int(1)].y - covar2d_5.rows[int(0)].y * covar2d_5.rows[int(1)].x);
    Matrix<float, 2, 2>  _S57 = makeMatrix<float, 2, 2> (covar2d_5.rows[int(1)].y * invdet_6, - covar2d_5.rows[int(0)].y * invdet_6, - covar2d_5.rows[int(1)].x * invdet_6, covar2d_5.rows[int(0)].x * invdet_6);
    *opacity_5 = 1.0f / (1.0f + (F32_exp((- in_opacity_5))));
    if(antialiased_5)
    {
        *opacity_5 = *opacity_5 * compensation_6;
    }
    float extend_5 = (F32_min((3.32999992370605469f), ((F32_sqrt((2.0f * (F32_log((*opacity_5 / 0.00392156885936856f)))))))));
    *radii_5 = make_int2 (int((F32_ceil((extend_5 * (F32_sqrt((covar2d_5[int(0)].x))))))), int((F32_ceil((extend_5 * (F32_sqrt((covar2d_5[int(1)].y))))))));
    *depth_5 = mean_c_5.z;
    *conic_5 = make_float3 (_S57.rows[int(0)].x, _S57.rows[int(0)].y, _S57.rows[int(1)].y);
    return;
}

inline __device__ float3  s_primal_ctx_mul_0(Matrix<float, 3, 3>  _S58, float3  _S59)
{
    return mul_0(_S58, _S59);
}

inline __device__ float3  s_primal_ctx_exp_0(float3  _S60)
{
    return exp_0(_S60);
}

inline __device__ float s_primal_ctx_rsqrt_0(float _S61)
{
    return (F32_rsqrt((_S61)));
}

inline __device__ Matrix<float, 3, 3>  s_primal_ctx_mul_1(Matrix<float, 3, 3>  _S62, Matrix<float, 3, 3>  _S63)
{
    return mul_4(_S62, _S63);
}

inline __device__ float s_primal_ctx_max_0(float _S64, float _S65)
{
    return (F32_max((_S64), (_S65)));
}

inline __device__ float s_primal_ctx_min_0(float _S66, float _S67)
{
    return (F32_min((_S66), (_S67)));
}

inline __device__ Matrix<float, 2, 3>  s_primal_ctx_mul_2(Matrix<float, 2, 3>  _S68, Matrix<float, 3, 3>  _S69)
{
    return mul_5(_S68, _S69);
}

inline __device__ Matrix<float, 2, 2>  s_primal_ctx_mul_3(Matrix<float, 2, 3>  _S70, Matrix<float, 3, 2>  _S71)
{
    return mul_6(_S70, _S71);
}

inline __device__ float s_primal_ctx_sqrt_0(float _S72)
{
    return (F32_sqrt((_S72)));
}

inline __device__ float s_primal_ctx_exp_1(float _S73)
{
    return (F32_exp((_S73)));
}

inline __device__ float s_primal_ctx_log_0(float _S74)
{
    return (F32_log((_S74)));
}

inline __device__ void s_bwd_prop_sqrt_0(DiffPair_float_0 * _S75, float _S76)
{
    _d_sqrt_0(_S75, _S76);
    return;
}

inline __device__ void s_bwd_prop_min_0(DiffPair_float_0 * _S77, DiffPair_float_0 * _S78, float _S79)
{
    _d_min_0(_S77, _S78, _S79);
    return;
}

inline __device__ void s_bwd_prop_log_0(DiffPair_float_0 * _S80, float _S81)
{
    _d_log_0(_S80, _S81);
    return;
}

inline __device__ void s_bwd_prop_exp_0(DiffPair_float_0 * _S82, float _S83)
{
    _d_exp_0(_S82, _S83);
    return;
}

inline __device__ void s_bwd_prop_max_0(DiffPair_float_0 * _S84, DiffPair_float_0 * _S85, float _S86)
{
    _d_max_0(_S84, _S85, _S86);
    return;
}

inline __device__ void s_bwd_prop_mul_0(DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 * _S87, DiffPair_matrixx3Cfloatx2C3x2C2x3E_0 * _S88, Matrix<float, 2, 2>  _S89)
{
    mul_3(_S87, _S88, _S89);
    return;
}

inline __device__ void s_bwd_prop_mul_1(DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 * _S90, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S91, Matrix<float, 2, 3>  _S92)
{
    mul_2(_S90, _S91, _S92);
    return;
}

inline __device__ void s_bwd_prop_mul_2(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S93, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S94, Matrix<float, 3, 3>  _S95)
{
    mul_1(_S93, _S94, _S95);
    return;
}

inline __device__ void s_bwd_prop_rsqrt_0(DiffPair_float_0 * _S96, float _S97)
{
    _d_rsqrt_0(_S96, _S97);
    return;
}

inline __device__ void s_bwd_prop_exp_1(DiffPair_vectorx3Cfloatx2C3x3E_0 * _S98, float3  _S99)
{
    _d_exp_vector_0(_S98, _S99);
    return;
}

inline __device__ void s_bwd_prop_mul_3(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S100, DiffPair_vectorx3Cfloatx2C3x3E_0 * _S101, float3  _S102)
{
    _d_mul_0(_S100, _S101, _S102);
    return;
}

inline __device__ void projection_3dgs_persp_vjp(bool antialiased_6, float3  mean_6, float4  quat_8, float3  scale_7, float in_opacity_6, Matrix<float, 3, 3>  R_8, float3  t_7, float fx_10, float fy_10, float cx_10, float cy_10, uint image_width_6, uint image_height_6, float eps2d_7, float v_depth_0, float2  v_mean2d_0, float3  v_conic_0, float v_opacity_0, float3  * v_mean_0, float4  * v_quat_0, float3  * v_scale_0, float * v_in_opacity_0, Matrix<float, 3, 3>  * v_R_0, float3  * v_t_0)
{
    float3  mean_c_6 = s_primal_ctx_mul_0(R_8, mean_6) + t_7;
    float3  _S103 = s_primal_ctx_exp_0(scale_7);
    float _S104 = quat_8.y;
    float _S105 = _S104 * _S104 + quat_8.z * quat_8.z + quat_8.w * quat_8.w + quat_8.x * quat_8.x;
    float _S106 = s_primal_ctx_rsqrt_0(_S105);
    float x_22 = quat_8.y * _S106;
    float y_9 = quat_8.z * _S106;
    float z_8 = quat_8.w * _S106;
    float w_8 = quat_8.x * _S106;
    float x2_11 = x_22 * x_22;
    float y2_11 = y_9 * y_9;
    float z2_8 = z_8 * z_8;
    float xy_11 = x_22 * y_9;
    float xz_8 = x_22 * z_8;
    float yz_8 = y_9 * z_8;
    float wx_8 = w_8 * x_22;
    float wy_8 = w_8 * y_9;
    float wz_8 = w_8 * z_8;
    Matrix<float, 3, 3>  _S107 = transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_11 + z2_8), 2.0f * (xy_11 + wz_8), 2.0f * (xz_8 - wy_8), 2.0f * (xy_11 - wz_8), 1.0f - 2.0f * (x2_11 + z2_8), 2.0f * (yz_8 + wx_8), 2.0f * (xz_8 + wy_8), 2.0f * (yz_8 - wx_8), 1.0f - 2.0f * (x2_11 + y2_11)));
    Matrix<float, 3, 3>  S_0 = makeMatrix<float, 3, 3> (_S103.x, 0.0f, 0.0f, 0.0f, _S103.y, 0.0f, 0.0f, 0.0f, _S103.z);
    Matrix<float, 3, 3>  _S108 = s_primal_ctx_mul_1(_S107, S_0);
    Matrix<float, 3, 3>  _S109 = transpose_0(_S108);
    Matrix<float, 3, 3>  _S110 = s_primal_ctx_mul_1(_S108, _S109);
    Matrix<float, 3, 3>  _S111 = s_primal_ctx_mul_1(R_8, _S110);
    Matrix<float, 3, 3>  _S112 = transpose_0(R_8);
    Matrix<float, 3, 3>  _S113 = s_primal_ctx_mul_1(_S111, _S112);
    float _S114 = float(image_width_6);
    float _S115 = float(image_height_6);
    float _S116 = 0.30000001192092896f * (0.5f * _S114 / fx_10);
    float lim_x_pos_0 = (_S114 - cx_10) / fx_10 + _S116;
    float _S117 = 0.30000001192092896f * (0.5f * _S115 / fy_10);
    float lim_y_pos_0 = (_S115 - cy_10) / fy_10 + _S117;
    float rz_4 = 1.0f / mean_c_6.z;
    float _S118 = mean_c_6.z * mean_c_6.z;
    float rz2_4 = rz_4 * rz_4;
    float _S119 = - (cx_10 / fx_10 + _S116);
    float _S120 = mean_c_6.x * rz_4;
    float _S121 = s_primal_ctx_max_0(_S119, _S120);
    float _S122 = s_primal_ctx_min_0(lim_x_pos_0, _S121);
    float _S123 = - (cy_10 / fy_10 + _S117);
    float _S124 = mean_c_6.y * rz_4;
    float _S125 = s_primal_ctx_max_0(_S123, _S124);
    float _S126 = s_primal_ctx_min_0(lim_y_pos_0, _S125);
    float _S127 = - fx_10;
    float _S128 = _S127 * (mean_c_6.z * _S122);
    float _S129 = - fy_10;
    float _S130 = _S129 * (mean_c_6.z * _S126);
    Matrix<float, 2, 3>  J_10 = makeMatrix<float, 2, 3> (fx_10 * rz_4, 0.0f, _S128 * rz2_4, 0.0f, fy_10 * rz_4, _S130 * rz2_4);
    Matrix<float, 2, 3>  _S131 = s_primal_ctx_mul_2(J_10, _S113);
    Matrix<float, 3, 2>  _S132 = transpose_1(J_10);
    Matrix<float, 2, 2>  _S133 = s_primal_ctx_mul_3(_S131, _S132);
    float _S134 = fx_10 * mean_c_6.x;
    float _S135 = fy_10 * mean_c_6.y;
    float _S136 = _S133.rows[int(0)].y * _S133.rows[int(1)].x;
    float det_orig_7 = _S133.rows[int(0)].x * _S133.rows[int(1)].y - _S136;
    float _S137 = _S133.rows[int(0)].x + eps2d_7;
    Matrix<float, 2, 2>  _S138 = _S133;
    *&(((&_S138)->rows + (int(0)))->x) = _S137;
    float _S139 = _S133.rows[int(1)].y + eps2d_7;
    *&(((&_S138)->rows + (int(1)))->y) = _S139;
    Matrix<float, 2, 2>  _S140 = _S138;
    Matrix<float, 2, 2>  _S141 = _S138;
    float det_blur_4 = _S137 * _S139 - _S136;
    float _S142 = det_orig_7 / det_blur_4;
    float _S143 = det_blur_4 * det_blur_4;
    float _S144 = s_primal_ctx_max_0(0.0f, _S142);
    float _S145 = s_primal_ctx_sqrt_0(_S144);
    float invdet_7 = 1.0f / det_blur_4;
    float _S146 = - _S133.rows[int(0)].y;
    float _S147 = - _S133.rows[int(1)].x;
    float _S148 = - in_opacity_6;
    float _S149 = 1.0f + s_primal_ctx_exp_1(_S148);
    float _S150 = 1.0f / _S149;
    float _S151 = _S149 * _S149;
    float _S152;
    if(antialiased_6)
    {
        _S152 = _S150 * _S145;
    }
    else
    {
        _S152 = _S150;
    }
    float _S153 = _S152 / 0.00392156885936856f;
    float _S154 = 2.0f * s_primal_ctx_log_0(_S153);
    float _S155 = s_primal_ctx_sqrt_0(_S154);
    float _S156 = _S140.rows[int(0)].x;
    float _S157 = _S141.rows[int(1)].y;
    float2  _S158 = make_float2 (0.0f);
    float2  _S159 = _S158;
    *&((&_S159)->y) = v_conic_0.z;
    float2  _S160 = _S158;
    *&((&_S160)->y) = v_conic_0.y;
    *&((&_S160)->x) = v_conic_0.x;
    DiffPair_float_0 _S161;
    (&_S161)->primal_0 = _S157;
    (&_S161)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S161, 0.0f);
    DiffPair_float_0 _S162;
    (&_S162)->primal_0 = _S156;
    (&_S162)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S162, 0.0f);
    DiffPair_float_0 _S163;
    (&_S163)->primal_0 = 3.32999992370605469f;
    (&_S163)->differential_0 = 0.0f;
    DiffPair_float_0 _S164;
    (&_S164)->primal_0 = _S155;
    (&_S164)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S163, &_S164, 0.0f);
    DiffPair_float_0 _S165;
    (&_S165)->primal_0 = _S154;
    (&_S165)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S165, _S164.differential_0);
    float _S166 = 2.0f * _S165.differential_0;
    DiffPair_float_0 _S167;
    (&_S167)->primal_0 = _S153;
    (&_S167)->differential_0 = 0.0f;
    s_bwd_prop_log_0(&_S167, _S166);
    float _S168 = v_opacity_0 + 254.9999847412109375f * _S167.differential_0;
    Matrix<float, 2, 2>  _S169 = makeMatrix<float, 2, 2> (0.0f);
    Matrix<float, 2, 2>  _S170 = _S169;
    _S170[int(1)] = _S159;
    _S170[int(0)] = _S160;
    Matrix<float, 2, 2>  _S171 = _S170;
    float3  _S172 = make_float3 (0.0f, 0.0f, v_depth_0);
    float2  _S173 = make_float2 (_S162.differential_0, 0.0f);
    float2  _S174 = make_float2 (0.0f, _S161.differential_0);
    float _S175;
    if(antialiased_6)
    {
        float _S176 = _S150 * _S168;
        _S152 = _S145 * _S168;
        _S175 = _S176;
    }
    else
    {
        _S152 = _S168;
        _S175 = 0.0f;
    }
    float _S177 = - (_S152 / _S151);
    DiffPair_float_0 _S178;
    (&_S178)->primal_0 = _S148;
    (&_S178)->differential_0 = 0.0f;
    s_bwd_prop_exp_0(&_S178, _S177);
    float _S179 = - _S178.differential_0;
    float _S180 = invdet_7 * _S171.rows[int(1)].y;
    float _S181 = - (invdet_7 * _S171.rows[int(1)].x);
    float _S182 = - (invdet_7 * _S171.rows[int(0)].y);
    float _S183 = invdet_7 * _S171.rows[int(0)].x;
    float _S184 = - ((_S137 * _S171.rows[int(1)].y + _S147 * _S171.rows[int(1)].x + _S146 * _S171.rows[int(0)].y + _S139 * _S171.rows[int(0)].x) / _S143);
    DiffPair_float_0 _S185;
    (&_S185)->primal_0 = _S144;
    (&_S185)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S185, _S175);
    DiffPair_float_0 _S186;
    (&_S186)->primal_0 = 0.0f;
    (&_S186)->differential_0 = 0.0f;
    DiffPair_float_0 _S187;
    (&_S187)->primal_0 = _S142;
    (&_S187)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S186, &_S187, _S185.differential_0);
    float _S188 = _S187.differential_0 / _S143;
    float s_diff_det_orig_T_0 = det_blur_4 * _S188;
    float _S189 = _S184 + det_orig_7 * - _S188;
    float _S190 = - _S189;
    float _S191 = _S137 * _S189;
    float _S192 = _S139 * _S189;
    Matrix<float, 2, 2>  _S193 = _S169;
    _S193[int(1)] = _S174;
    _S193[int(0)] = _S173;
    _S138 = _S193;
    *&(((&_S138)->rows + (int(1)))->y) = 0.0f;
    float _S194 = _S183 + _S191 + _S193.rows[int(1)].y;
    *&(((&_S138)->rows + (int(0)))->x) = 0.0f;
    float _S195 = _S180 + _S192 + _S193.rows[int(0)].x;
    float _S196 = _S190 + - s_diff_det_orig_T_0;
    float _S197 = _S181 + _S133.rows[int(0)].y * _S196;
    float _S198 = _S182 + _S133.rows[int(1)].x * _S196;
    float _S199 = _S133.rows[int(1)].y * s_diff_det_orig_T_0;
    float _S200 = _S194 + _S133.rows[int(0)].x * s_diff_det_orig_T_0;
    float2  _S201 = _S158;
    *&((&_S201)->x) = _S197;
    *&((&_S201)->y) = _S200;
    float _S202 = _S195 + _S199;
    float2  _S203 = _S158;
    *&((&_S203)->y) = _S198;
    *&((&_S203)->x) = _S202;
    float _S204 = _S135 * v_mean2d_0.y;
    float _S205 = fy_10 * (rz_4 * v_mean2d_0.y);
    float _S206 = _S134 * v_mean2d_0.x;
    float _S207 = fx_10 * (rz_4 * v_mean2d_0.x);
    Matrix<float, 2, 2>  _S208 = _S169;
    _S208[int(1)] = _S201;
    _S208[int(0)] = _S203;
    Matrix<float, 2, 2>  _S209 = _S138 + _S208;
    Matrix<float, 2, 3>  _S210 = makeMatrix<float, 2, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S211;
    (&_S211)->primal_0 = _S131;
    (&_S211)->differential_0 = _S210;
    Matrix<float, 3, 2>  _S212 = makeMatrix<float, 3, 2> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C2x3E_0 _S213;
    (&_S213)->primal_0 = _S132;
    (&_S213)->differential_0 = _S212;
    s_bwd_prop_mul_0(&_S211, &_S213, _S209);
    Matrix<float, 2, 3>  _S214 = transpose_2(_S213.differential_0);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S215;
    (&_S215)->primal_0 = J_10;
    (&_S215)->differential_0 = _S210;
    Matrix<float, 3, 3>  _S216 = makeMatrix<float, 3, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S217;
    (&_S217)->primal_0 = _S113;
    (&_S217)->differential_0 = _S216;
    s_bwd_prop_mul_1(&_S215, &_S217, _S211.differential_0);
    Matrix<float, 2, 3>  _S218 = _S214 + _S215.differential_0;
    float _S219 = _S130 * _S218.rows[int(1)].z;
    float s_diff_ty_T_0 = _S129 * (rz2_4 * _S218.rows[int(1)].z);
    float _S220 = fy_10 * _S218.rows[int(1)].y;
    float _S221 = _S128 * _S218.rows[int(0)].z;
    float s_diff_tx_T_0 = _S127 * (rz2_4 * _S218.rows[int(0)].z);
    float _S222 = fx_10 * _S218.rows[int(0)].x;
    float _S223 = mean_c_6.z * s_diff_ty_T_0;
    float _S224 = _S126 * s_diff_ty_T_0;
    DiffPair_float_0 _S225;
    (&_S225)->primal_0 = lim_y_pos_0;
    (&_S225)->differential_0 = 0.0f;
    DiffPair_float_0 _S226;
    (&_S226)->primal_0 = _S125;
    (&_S226)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S225, &_S226, _S223);
    DiffPair_float_0 _S227;
    (&_S227)->primal_0 = _S123;
    (&_S227)->differential_0 = 0.0f;
    DiffPair_float_0 _S228;
    (&_S228)->primal_0 = _S124;
    (&_S228)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S227, &_S228, _S226.differential_0);
    float _S229 = mean_c_6.y * _S228.differential_0;
    float _S230 = rz_4 * _S228.differential_0;
    float _S231 = mean_c_6.z * s_diff_tx_T_0;
    float _S232 = _S122 * s_diff_tx_T_0;
    DiffPair_float_0 _S233;
    (&_S233)->primal_0 = lim_x_pos_0;
    (&_S233)->differential_0 = 0.0f;
    DiffPair_float_0 _S234;
    (&_S234)->primal_0 = _S121;
    (&_S234)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S233, &_S234, _S231);
    DiffPair_float_0 _S235;
    (&_S235)->primal_0 = _S119;
    (&_S235)->differential_0 = 0.0f;
    DiffPair_float_0 _S236;
    (&_S236)->primal_0 = _S120;
    (&_S236)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S235, &_S236, _S234.differential_0);
    float _S237 = rz_4 * (_S219 + _S221);
    float _S238 = _S224 + _S232 + - ((_S204 + _S206 + _S220 + _S222 + _S229 + mean_c_6.x * _S236.differential_0 + _S237 + _S237) / _S118);
    float _S239 = _S205 + _S230;
    float _S240 = _S207 + rz_4 * _S236.differential_0;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S241;
    (&_S241)->primal_0 = _S111;
    (&_S241)->differential_0 = _S216;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S242;
    (&_S242)->primal_0 = _S112;
    (&_S242)->differential_0 = _S216;
    s_bwd_prop_mul_2(&_S241, &_S242, _S217.differential_0);
    Matrix<float, 3, 3>  _S243 = transpose_0(_S242.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S244;
    (&_S244)->primal_0 = R_8;
    (&_S244)->differential_0 = _S216;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S245;
    (&_S245)->primal_0 = _S110;
    (&_S245)->differential_0 = _S216;
    s_bwd_prop_mul_2(&_S244, &_S245, _S241.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S246;
    (&_S246)->primal_0 = _S108;
    (&_S246)->differential_0 = _S216;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S247;
    (&_S247)->primal_0 = _S109;
    (&_S247)->differential_0 = _S216;
    s_bwd_prop_mul_2(&_S246, &_S247, _S245.differential_0);
    Matrix<float, 3, 3>  _S248 = _S246.differential_0 + transpose_0(_S247.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S249;
    (&_S249)->primal_0 = _S107;
    (&_S249)->differential_0 = _S216;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S250;
    (&_S250)->primal_0 = S_0;
    (&_S250)->differential_0 = _S216;
    s_bwd_prop_mul_2(&_S249, &_S250, _S248);
    Matrix<float, 3, 3>  _S251 = transpose_0(_S249.differential_0);
    float _S252 = 2.0f * - _S251.rows[int(2)].z;
    float _S253 = 2.0f * _S251.rows[int(2)].y;
    float _S254 = 2.0f * _S251.rows[int(2)].x;
    float _S255 = 2.0f * _S251.rows[int(1)].z;
    float _S256 = 2.0f * - _S251.rows[int(1)].y;
    float _S257 = 2.0f * _S251.rows[int(1)].x;
    float _S258 = 2.0f * _S251.rows[int(0)].z;
    float _S259 = 2.0f * _S251.rows[int(0)].y;
    float _S260 = 2.0f * - _S251.rows[int(0)].x;
    float _S261 = - _S257 + _S259;
    float _S262 = _S254 + - _S258;
    float _S263 = - _S253 + _S255;
    float _S264 = _S253 + _S255;
    float _S265 = _S254 + _S258;
    float _S266 = _S257 + _S259;
    float _S267 = z_8 * (_S256 + _S260);
    float _S268 = y_9 * (_S252 + _S260);
    float _S269 = x_22 * (_S252 + _S256);
    float _S270 = z_8 * _S261 + y_9 * _S262 + x_22 * _S263;
    float _S271 = _S106 * _S270;
    float _S272 = w_8 * _S261 + y_9 * _S264 + x_22 * _S265 + _S267 + _S267;
    float _S273 = _S106 * _S272;
    float _S274 = w_8 * _S262 + z_8 * _S264 + x_22 * _S266 + _S268 + _S268;
    float _S275 = _S106 * _S274;
    float _S276 = w_8 * _S263 + z_8 * _S265 + y_9 * _S266 + _S269 + _S269;
    float _S277 = _S106 * _S276;
    float _S278 = quat_8.x * _S270 + quat_8.w * _S272 + quat_8.z * _S274 + quat_8.y * _S276;
    DiffPair_float_0 _S279;
    (&_S279)->primal_0 = _S105;
    (&_S279)->differential_0 = 0.0f;
    s_bwd_prop_rsqrt_0(&_S279, _S278);
    float _S280 = quat_8.x * _S279.differential_0;
    float _S281 = quat_8.w * _S279.differential_0;
    float _S282 = quat_8.z * _S279.differential_0;
    float _S283 = quat_8.y * _S279.differential_0;
    float _S284 = _S273 + _S281 + _S281;
    float _S285 = _S275 + _S282 + _S282;
    float _S286 = _S277 + _S283 + _S283;
    float _S287 = _S271 + _S280 + _S280;
    float3  _S288 = make_float3 (0.0f);
    float3  _S289 = _S288;
    *&((&_S289)->z) = _S250.differential_0.rows[int(2)].z;
    *&((&_S289)->y) = _S250.differential_0.rows[int(1)].y;
    *&((&_S289)->x) = _S250.differential_0.rows[int(0)].x;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S290;
    (&_S290)->primal_0 = scale_7;
    (&_S290)->differential_0 = _S288;
    s_bwd_prop_exp_1(&_S290, _S289);
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S291 = _S290;
    float3  _S292 = _S288;
    *&((&_S292)->z) = _S238;
    *&((&_S292)->y) = _S239;
    *&((&_S292)->x) = _S240;
    float3  _S293 = _S172 + _S292;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S294;
    (&_S294)->primal_0 = R_8;
    (&_S294)->differential_0 = _S216;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S295;
    (&_S295)->primal_0 = mean_6;
    (&_S295)->differential_0 = _S288;
    s_bwd_prop_mul_3(&_S294, &_S295, _S293);
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S296 = _S295;
    Matrix<float, 3, 3>  _S297 = _S243 + _S244.differential_0 + _S294.differential_0;
    float4  _S298 = make_float4 (0.0f);
    *&((&_S298)->w) = _S284;
    *&((&_S298)->z) = _S285;
    *&((&_S298)->y) = _S286;
    *&((&_S298)->x) = _S287;
    float4  _S299 = _S298;
    *v_mean_0 = _S296.differential_0;
    *v_quat_0 = _S299;
    *v_scale_0 = _S291.differential_0;
    *v_in_opacity_0 = _S179;
    *v_R_0 = _S297;
    *v_t_0 = _S293;
    return;
}

inline __device__ float s_primal_ctx_atan2_0(float _S300, float _S301)
{
    return (F32_atan2((_S300), (_S301)));
}

inline __device__ void s_bwd_prop_atan2_0(DiffPair_float_0 * _S302, DiffPair_float_0 * _S303, float _S304)
{
    _d_atan2_0(_S302, _S303, _S304);
    return;
}

struct DiffPair_vectorx3Cfloatx2C2x3E_0
{
    float2  primal_0;
    float2  differential_0;
};

inline __device__ void s_bwd_prop_length_impl_0(DiffPair_vectorx3Cfloatx2C2x3E_0 * dpx_8, float _s_dOut_0)
{
    float _S305 = (*dpx_8).primal_0.x;
    float _S306 = (*dpx_8).primal_0.y;
    DiffPair_float_0 _S307;
    (&_S307)->primal_0 = _S305 * _S305 + _S306 * _S306;
    (&_S307)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S307, _s_dOut_0);
    float _S308 = (*dpx_8).primal_0.y * _S307.differential_0;
    float _S309 = _S308 + _S308;
    float _S310 = (*dpx_8).primal_0.x * _S307.differential_0;
    float _S311 = _S310 + _S310;
    float2  _S312 = make_float2 (0.0f);
    *&((&_S312)->y) = _S309;
    *&((&_S312)->x) = _S311;
    dpx_8->primal_0 = (*dpx_8).primal_0;
    dpx_8->differential_0 = _S312;
    return;
}

inline __device__ void s_bwd_length_impl_0(DiffPair_vectorx3Cfloatx2C2x3E_0 * _S313, float _S314)
{
    s_bwd_prop_length_impl_0(_S313, _S314);
    return;
}

inline __device__ void projection_3dgs_fisheye_vjp(bool antialiased_7, float3  mean_7, float4  quat_9, float3  scale_8, float in_opacity_7, Matrix<float, 3, 3>  R_9, float3  t_8, float fx_11, float fy_11, float cx_11, float cy_11, uint image_width_7, uint image_height_7, float eps2d_8, float v_depth_1, float2  v_mean2d_1, float3  v_conic_1, float v_opacity_1, float3  * v_mean_1, float4  * v_quat_1, float3  * v_scale_1, float * v_in_opacity_1, Matrix<float, 3, 3>  * v_R_1, float3  * v_t_1)
{
    float3  mean_c_7 = s_primal_ctx_mul_0(R_9, mean_7) + t_8;
    float3  _S315 = s_primal_ctx_exp_0(scale_8);
    float _S316 = quat_9.y;
    float _S317 = _S316 * _S316 + quat_9.z * quat_9.z + quat_9.w * quat_9.w + quat_9.x * quat_9.x;
    float _S318 = s_primal_ctx_rsqrt_0(_S317);
    float x_23 = quat_9.y * _S318;
    float y_10 = quat_9.z * _S318;
    float z_9 = quat_9.w * _S318;
    float w_9 = quat_9.x * _S318;
    float x2_12 = x_23 * x_23;
    float y2_12 = y_10 * y_10;
    float z2_9 = z_9 * z_9;
    float xy_12 = x_23 * y_10;
    float xz_9 = x_23 * z_9;
    float yz_9 = y_10 * z_9;
    float wx_9 = w_9 * x_23;
    float wy_9 = w_9 * y_10;
    float wz_9 = w_9 * z_9;
    Matrix<float, 3, 3>  _S319 = transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_12 + z2_9), 2.0f * (xy_12 + wz_9), 2.0f * (xz_9 - wy_9), 2.0f * (xy_12 - wz_9), 1.0f - 2.0f * (x2_12 + z2_9), 2.0f * (yz_9 + wx_9), 2.0f * (xz_9 + wy_9), 2.0f * (yz_9 - wx_9), 1.0f - 2.0f * (x2_12 + y2_12)));
    Matrix<float, 3, 3>  S_1 = makeMatrix<float, 3, 3> (_S315.x, 0.0f, 0.0f, 0.0f, _S315.y, 0.0f, 0.0f, 0.0f, _S315.z);
    Matrix<float, 3, 3>  _S320 = s_primal_ctx_mul_1(_S319, S_1);
    Matrix<float, 3, 3>  _S321 = transpose_0(_S320);
    Matrix<float, 3, 3>  _S322 = s_primal_ctx_mul_1(_S320, _S321);
    Matrix<float, 3, 3>  _S323 = s_primal_ctx_mul_1(R_9, _S322);
    Matrix<float, 3, 3>  _S324 = transpose_0(R_9);
    Matrix<float, 3, 3>  _S325 = s_primal_ctx_mul_1(_S323, _S324);
    float2  _S326 = make_float2 (mean_c_7.x, mean_c_7.y);
    float xy_len_3 = length_0(_S326) + 1.00000001168609742e-07f;
    float _S327 = mean_c_7.z + 1.00000001168609742e-07f;
    float _S328 = s_primal_ctx_atan2_0(xy_len_3, _S327);
    float _S329 = mean_c_7.x * fx_11;
    float _S330 = _S329 * _S328;
    float _S331 = xy_len_3 * xy_len_3;
    float _S332 = mean_c_7.y * fy_11;
    float _S333 = _S332 * _S328;
    float x2_13 = mean_c_7.x * mean_c_7.x + 1.00000001168609742e-07f;
    float y2_13 = mean_c_7.y * mean_c_7.y;
    float xy_13 = mean_c_7.x * mean_c_7.y;
    float x2y2_3 = x2_13 + y2_13;
    float _S334 = x2y2_3 + mean_c_7.z * mean_c_7.z;
    float x2y2z2_inv_3 = 1.0f / _S334;
    float _S335 = _S334 * _S334;
    float _S336 = s_primal_ctx_atan2_0(xy_len_3, mean_c_7.z);
    float _S337 = _S336 / xy_len_3;
    float b_3 = _S337 / x2y2_3;
    float _S338 = x2y2_3 * x2y2_3;
    float _S339 = mean_c_7.z * x2y2z2_inv_3;
    float a_3 = _S339 / x2y2_3;
    float _S340 = fx_11 * xy_13;
    float _S341 = a_3 - b_3;
    float _S342 = - fx_11;
    float _S343 = _S342 * mean_c_7.x;
    float _S344 = fy_11 * xy_13;
    float _S345 = - fy_11;
    float _S346 = _S345 * mean_c_7.y;
    Matrix<float, 2, 3>  J_11 = makeMatrix<float, 2, 3> (fx_11 * (x2_13 * a_3 + y2_13 * b_3), _S340 * _S341, _S343 * x2y2z2_inv_3, _S344 * _S341, fy_11 * (y2_13 * a_3 + x2_13 * b_3), _S346 * x2y2z2_inv_3);
    Matrix<float, 2, 3>  _S347 = s_primal_ctx_mul_2(J_11, _S325);
    Matrix<float, 3, 2>  _S348 = transpose_1(J_11);
    Matrix<float, 2, 2>  _S349 = s_primal_ctx_mul_3(_S347, _S348);
    float _S350 = _S349.rows[int(0)].y * _S349.rows[int(1)].x;
    float det_orig_8 = _S349.rows[int(0)].x * _S349.rows[int(1)].y - _S350;
    float _S351 = _S349.rows[int(0)].x + eps2d_8;
    Matrix<float, 2, 2>  _S352 = _S349;
    *&(((&_S352)->rows + (int(0)))->x) = _S351;
    float _S353 = _S349.rows[int(1)].y + eps2d_8;
    *&(((&_S352)->rows + (int(1)))->y) = _S353;
    Matrix<float, 2, 2>  _S354 = _S352;
    Matrix<float, 2, 2>  _S355 = _S352;
    float det_blur_5 = _S351 * _S353 - _S350;
    float _S356 = det_orig_8 / det_blur_5;
    float _S357 = det_blur_5 * det_blur_5;
    float _S358 = s_primal_ctx_max_0(0.0f, _S356);
    float _S359 = s_primal_ctx_sqrt_0(_S358);
    float invdet_8 = 1.0f / det_blur_5;
    float _S360 = - _S349.rows[int(0)].y;
    float _S361 = - _S349.rows[int(1)].x;
    float _S362 = - in_opacity_7;
    float _S363 = 1.0f + s_primal_ctx_exp_1(_S362);
    float _S364 = 1.0f / _S363;
    float _S365 = _S363 * _S363;
    float _S366;
    if(antialiased_7)
    {
        _S366 = _S364 * _S359;
    }
    else
    {
        _S366 = _S364;
    }
    float _S367 = _S366 / 0.00392156885936856f;
    float _S368 = 2.0f * s_primal_ctx_log_0(_S367);
    float _S369 = s_primal_ctx_sqrt_0(_S368);
    float _S370 = _S354.rows[int(0)].x;
    float _S371 = _S355.rows[int(1)].y;
    float2  _S372 = make_float2 (0.0f);
    float2  _S373 = _S372;
    *&((&_S373)->y) = v_conic_1.z;
    float2  _S374 = _S372;
    *&((&_S374)->y) = v_conic_1.y;
    *&((&_S374)->x) = v_conic_1.x;
    DiffPair_float_0 _S375;
    (&_S375)->primal_0 = _S371;
    (&_S375)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S375, 0.0f);
    DiffPair_float_0 _S376;
    (&_S376)->primal_0 = _S370;
    (&_S376)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S376, 0.0f);
    DiffPair_float_0 _S377;
    (&_S377)->primal_0 = 3.32999992370605469f;
    (&_S377)->differential_0 = 0.0f;
    DiffPair_float_0 _S378;
    (&_S378)->primal_0 = _S369;
    (&_S378)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S377, &_S378, 0.0f);
    DiffPair_float_0 _S379;
    (&_S379)->primal_0 = _S368;
    (&_S379)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S379, _S378.differential_0);
    float _S380 = 2.0f * _S379.differential_0;
    DiffPair_float_0 _S381;
    (&_S381)->primal_0 = _S367;
    (&_S381)->differential_0 = 0.0f;
    s_bwd_prop_log_0(&_S381, _S380);
    float _S382 = v_opacity_1 + 254.9999847412109375f * _S381.differential_0;
    Matrix<float, 2, 2>  _S383 = makeMatrix<float, 2, 2> (0.0f);
    Matrix<float, 2, 2>  _S384 = _S383;
    _S384[int(1)] = _S373;
    _S384[int(0)] = _S374;
    Matrix<float, 2, 2>  _S385 = _S384;
    float3  _S386 = make_float3 (0.0f, 0.0f, v_depth_1);
    float2  _S387 = make_float2 (_S376.differential_0, 0.0f);
    float2  _S388 = make_float2 (0.0f, _S375.differential_0);
    float _S389;
    if(antialiased_7)
    {
        float _S390 = _S364 * _S382;
        _S366 = _S359 * _S382;
        _S389 = _S390;
    }
    else
    {
        _S366 = _S382;
        _S389 = 0.0f;
    }
    float _S391 = - (_S366 / _S365);
    DiffPair_float_0 _S392;
    (&_S392)->primal_0 = _S362;
    (&_S392)->differential_0 = 0.0f;
    s_bwd_prop_exp_0(&_S392, _S391);
    float _S393 = - _S392.differential_0;
    float _S394 = invdet_8 * _S385.rows[int(1)].y;
    float _S395 = - (invdet_8 * _S385.rows[int(1)].x);
    float _S396 = - (invdet_8 * _S385.rows[int(0)].y);
    float _S397 = invdet_8 * _S385.rows[int(0)].x;
    float _S398 = - ((_S351 * _S385.rows[int(1)].y + _S361 * _S385.rows[int(1)].x + _S360 * _S385.rows[int(0)].y + _S353 * _S385.rows[int(0)].x) / _S357);
    DiffPair_float_0 _S399;
    (&_S399)->primal_0 = _S358;
    (&_S399)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S399, _S389);
    DiffPair_float_0 _S400;
    (&_S400)->primal_0 = 0.0f;
    (&_S400)->differential_0 = 0.0f;
    DiffPair_float_0 _S401;
    (&_S401)->primal_0 = _S356;
    (&_S401)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S400, &_S401, _S399.differential_0);
    float _S402 = _S401.differential_0 / _S357;
    float s_diff_det_orig_T_1 = det_blur_5 * _S402;
    float _S403 = _S398 + det_orig_8 * - _S402;
    float _S404 = - _S403;
    float _S405 = _S351 * _S403;
    float _S406 = _S353 * _S403;
    Matrix<float, 2, 2>  _S407 = _S383;
    _S407[int(1)] = _S388;
    _S407[int(0)] = _S387;
    _S352 = _S407;
    *&(((&_S352)->rows + (int(1)))->y) = 0.0f;
    float _S408 = _S397 + _S405 + _S407.rows[int(1)].y;
    *&(((&_S352)->rows + (int(0)))->x) = 0.0f;
    float _S409 = _S394 + _S406 + _S407.rows[int(0)].x;
    float _S410 = _S404 + - s_diff_det_orig_T_1;
    float _S411 = _S395 + _S349.rows[int(0)].y * _S410;
    float _S412 = _S396 + _S349.rows[int(1)].x * _S410;
    float _S413 = _S349.rows[int(1)].y * s_diff_det_orig_T_1;
    float _S414 = _S408 + _S349.rows[int(0)].x * s_diff_det_orig_T_1;
    float2  _S415 = _S372;
    *&((&_S415)->x) = _S411;
    *&((&_S415)->y) = _S414;
    float _S416 = _S409 + _S413;
    float2  _S417 = _S372;
    *&((&_S417)->y) = _S412;
    *&((&_S417)->x) = _S416;
    Matrix<float, 2, 2>  _S418 = _S383;
    _S418[int(1)] = _S415;
    _S418[int(0)] = _S417;
    Matrix<float, 2, 2>  _S419 = _S352 + _S418;
    Matrix<float, 2, 3>  _S420 = makeMatrix<float, 2, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S421;
    (&_S421)->primal_0 = _S347;
    (&_S421)->differential_0 = _S420;
    Matrix<float, 3, 2>  _S422 = makeMatrix<float, 3, 2> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C2x3E_0 _S423;
    (&_S423)->primal_0 = _S348;
    (&_S423)->differential_0 = _S422;
    s_bwd_prop_mul_0(&_S421, &_S423, _S419);
    Matrix<float, 2, 3>  _S424 = transpose_2(_S423.differential_0);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S425;
    (&_S425)->primal_0 = J_11;
    (&_S425)->differential_0 = _S420;
    Matrix<float, 3, 3>  _S426 = makeMatrix<float, 3, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S427;
    (&_S427)->primal_0 = _S325;
    (&_S427)->differential_0 = _S426;
    s_bwd_prop_mul_1(&_S425, &_S427, _S421.differential_0);
    Matrix<float, 2, 3>  _S428 = _S424 + _S425.differential_0;
    float _S429 = _S346 * _S428.rows[int(1)].z;
    float _S430 = _S345 * (x2y2z2_inv_3 * _S428.rows[int(1)].z);
    float _S431 = fy_11 * _S428.rows[int(1)].y;
    float _S432 = b_3 * _S431;
    float _S433 = a_3 * _S431;
    float _S434 = fy_11 * (_S341 * _S428.rows[int(1)].x);
    float _S435 = _S343 * _S428.rows[int(0)].z;
    float _S436 = _S342 * (x2y2z2_inv_3 * _S428.rows[int(0)].z);
    float _S437 = _S344 * _S428.rows[int(1)].x + _S340 * _S428.rows[int(0)].y;
    float _S438 = fx_11 * (_S341 * _S428.rows[int(0)].y);
    float _S439 = fx_11 * _S428.rows[int(0)].x;
    float _S440 = b_3 * _S439;
    float _S441 = a_3 * _S439;
    float _S442 = (y2_13 * _S431 + _S437 + x2_13 * _S439) / _S338;
    float _S443 = _S339 * - _S442;
    float _S444 = x2y2_3 * _S442;
    float _S445 = mean_c_7.z * _S444;
    float _S446 = x2y2z2_inv_3 * _S444;
    float _S447 = (x2_13 * _S431 + - _S437 + y2_13 * _S439) / _S338;
    float _S448 = _S337 * - _S447;
    float _S449 = x2y2_3 * _S447 / _S331;
    float _S450 = _S336 * - _S449;
    float _S451 = xy_len_3 * _S449;
    DiffPair_float_0 _S452;
    (&_S452)->primal_0 = xy_len_3;
    (&_S452)->differential_0 = 0.0f;
    DiffPair_float_0 _S453;
    (&_S453)->primal_0 = mean_c_7.z;
    (&_S453)->differential_0 = 0.0f;
    s_bwd_prop_atan2_0(&_S452, &_S453, _S451);
    float _S454 = - ((_S429 + _S435 + _S445) / _S335);
    float _S455 = mean_c_7.z * _S454;
    float _S456 = _S443 + _S448 + _S454;
    float _S457 = _S434 + _S438;
    float _S458 = mean_c_7.x * _S457;
    float _S459 = mean_c_7.y * _S457;
    float _S460 = mean_c_7.y * (_S433 + _S440 + _S456);
    float _S461 = mean_c_7.x * (_S432 + _S441 + _S456);
    float _S462 = v_mean2d_1.y / _S331;
    float _S463 = _S333 * - _S462;
    float _S464 = xy_len_3 * _S462;
    float _S465 = fy_11 * (_S328 * _S464);
    float _S466 = v_mean2d_1.x / _S331;
    float _S467 = _S330 * - _S466;
    float _S468 = xy_len_3 * _S466;
    float _S469 = fx_11 * (_S328 * _S468);
    float _S470 = _S332 * _S464 + _S329 * _S468;
    DiffPair_float_0 _S471;
    (&_S471)->primal_0 = xy_len_3;
    (&_S471)->differential_0 = 0.0f;
    DiffPair_float_0 _S472;
    (&_S472)->primal_0 = _S327;
    (&_S472)->differential_0 = 0.0f;
    s_bwd_prop_atan2_0(&_S471, &_S472, _S470);
    float _S473 = _S450 + _S452.differential_0 + _S463 + _S467 + _S471.differential_0;
    DiffPair_vectorx3Cfloatx2C2x3E_0 _S474;
    (&_S474)->primal_0 = _S326;
    (&_S474)->differential_0 = _S372;
    s_bwd_length_impl_0(&_S474, _S473);
    float _S475 = _S446 + _S453.differential_0 + _S455 + _S455 + _S472.differential_0;
    float _S476 = _S430 + _S458 + _S460 + _S460 + _S465 + _S474.differential_0.y;
    float _S477 = _S436 + _S459 + _S461 + _S461 + _S469 + _S474.differential_0.x;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S478;
    (&_S478)->primal_0 = _S323;
    (&_S478)->differential_0 = _S426;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S479;
    (&_S479)->primal_0 = _S324;
    (&_S479)->differential_0 = _S426;
    s_bwd_prop_mul_2(&_S478, &_S479, _S427.differential_0);
    Matrix<float, 3, 3>  _S480 = transpose_0(_S479.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S481;
    (&_S481)->primal_0 = R_9;
    (&_S481)->differential_0 = _S426;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S482;
    (&_S482)->primal_0 = _S322;
    (&_S482)->differential_0 = _S426;
    s_bwd_prop_mul_2(&_S481, &_S482, _S478.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S483;
    (&_S483)->primal_0 = _S320;
    (&_S483)->differential_0 = _S426;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S484;
    (&_S484)->primal_0 = _S321;
    (&_S484)->differential_0 = _S426;
    s_bwd_prop_mul_2(&_S483, &_S484, _S482.differential_0);
    Matrix<float, 3, 3>  _S485 = _S483.differential_0 + transpose_0(_S484.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S486;
    (&_S486)->primal_0 = _S319;
    (&_S486)->differential_0 = _S426;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S487;
    (&_S487)->primal_0 = S_1;
    (&_S487)->differential_0 = _S426;
    s_bwd_prop_mul_2(&_S486, &_S487, _S485);
    Matrix<float, 3, 3>  _S488 = transpose_0(_S486.differential_0);
    float _S489 = 2.0f * - _S488.rows[int(2)].z;
    float _S490 = 2.0f * _S488.rows[int(2)].y;
    float _S491 = 2.0f * _S488.rows[int(2)].x;
    float _S492 = 2.0f * _S488.rows[int(1)].z;
    float _S493 = 2.0f * - _S488.rows[int(1)].y;
    float _S494 = 2.0f * _S488.rows[int(1)].x;
    float _S495 = 2.0f * _S488.rows[int(0)].z;
    float _S496 = 2.0f * _S488.rows[int(0)].y;
    float _S497 = 2.0f * - _S488.rows[int(0)].x;
    float _S498 = - _S494 + _S496;
    float _S499 = _S491 + - _S495;
    float _S500 = - _S490 + _S492;
    float _S501 = _S490 + _S492;
    float _S502 = _S491 + _S495;
    float _S503 = _S494 + _S496;
    float _S504 = z_9 * (_S493 + _S497);
    float _S505 = y_10 * (_S489 + _S497);
    float _S506 = x_23 * (_S489 + _S493);
    float _S507 = z_9 * _S498 + y_10 * _S499 + x_23 * _S500;
    float _S508 = _S318 * _S507;
    float _S509 = w_9 * _S498 + y_10 * _S501 + x_23 * _S502 + _S504 + _S504;
    float _S510 = _S318 * _S509;
    float _S511 = w_9 * _S499 + z_9 * _S501 + x_23 * _S503 + _S505 + _S505;
    float _S512 = _S318 * _S511;
    float _S513 = w_9 * _S500 + z_9 * _S502 + y_10 * _S503 + _S506 + _S506;
    float _S514 = _S318 * _S513;
    float _S515 = quat_9.x * _S507 + quat_9.w * _S509 + quat_9.z * _S511 + quat_9.y * _S513;
    DiffPair_float_0 _S516;
    (&_S516)->primal_0 = _S317;
    (&_S516)->differential_0 = 0.0f;
    s_bwd_prop_rsqrt_0(&_S516, _S515);
    float _S517 = quat_9.x * _S516.differential_0;
    float _S518 = quat_9.w * _S516.differential_0;
    float _S519 = quat_9.z * _S516.differential_0;
    float _S520 = quat_9.y * _S516.differential_0;
    float _S521 = _S510 + _S518 + _S518;
    float _S522 = _S512 + _S519 + _S519;
    float _S523 = _S514 + _S520 + _S520;
    float _S524 = _S508 + _S517 + _S517;
    float3  _S525 = make_float3 (0.0f);
    float3  _S526 = _S525;
    *&((&_S526)->z) = _S487.differential_0.rows[int(2)].z;
    *&((&_S526)->y) = _S487.differential_0.rows[int(1)].y;
    *&((&_S526)->x) = _S487.differential_0.rows[int(0)].x;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S527;
    (&_S527)->primal_0 = scale_8;
    (&_S527)->differential_0 = _S525;
    s_bwd_prop_exp_1(&_S527, _S526);
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S528 = _S527;
    float3  _S529 = _S525;
    *&((&_S529)->z) = _S475;
    *&((&_S529)->y) = _S476;
    *&((&_S529)->x) = _S477;
    float3  _S530 = _S386 + _S529;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S531;
    (&_S531)->primal_0 = R_9;
    (&_S531)->differential_0 = _S426;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S532;
    (&_S532)->primal_0 = mean_7;
    (&_S532)->differential_0 = _S525;
    s_bwd_prop_mul_3(&_S531, &_S532, _S530);
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S533 = _S532;
    Matrix<float, 3, 3>  _S534 = _S480 + _S481.differential_0 + _S531.differential_0;
    float4  _S535 = make_float4 (0.0f);
    *&((&_S535)->w) = _S521;
    *&((&_S535)->z) = _S522;
    *&((&_S535)->y) = _S523;
    *&((&_S535)->x) = _S524;
    float4  _S536 = _S535;
    *v_mean_1 = _S533.differential_0;
    *v_quat_1 = _S536;
    *v_scale_1 = _S528.differential_0;
    *v_in_opacity_1 = _S393;
    *v_R_1 = _S534;
    *v_t_1 = _S530;
    return;
}

inline __device__ void projection_3dgs_ortho_vjp(bool antialiased_8, float3  mean_8, float4  quat_10, float3  scale_9, float in_opacity_8, Matrix<float, 3, 3>  R_10, float3  t_9, float fx_12, float fy_12, float cx_12, float cy_12, uint image_width_8, uint image_height_8, float eps2d_9, float v_depth_2, float2  v_mean2d_2, float3  v_conic_2, float v_opacity_2, float3  * v_mean_2, float4  * v_quat_2, float3  * v_scale_2, float * v_in_opacity_2, Matrix<float, 3, 3>  * v_R_2, float3  * v_t_2)
{
    float3  _S537 = s_primal_ctx_exp_0(scale_9);
    float _S538 = quat_10.y;
    float _S539 = _S538 * _S538 + quat_10.z * quat_10.z + quat_10.w * quat_10.w + quat_10.x * quat_10.x;
    float _S540 = s_primal_ctx_rsqrt_0(_S539);
    float x_24 = quat_10.y * _S540;
    float y_11 = quat_10.z * _S540;
    float z_10 = quat_10.w * _S540;
    float w_10 = quat_10.x * _S540;
    float x2_14 = x_24 * x_24;
    float y2_14 = y_11 * y_11;
    float z2_10 = z_10 * z_10;
    float xy_14 = x_24 * y_11;
    float xz_10 = x_24 * z_10;
    float yz_10 = y_11 * z_10;
    float wx_10 = w_10 * x_24;
    float wy_10 = w_10 * y_11;
    float wz_10 = w_10 * z_10;
    Matrix<float, 3, 3>  _S541 = transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_14 + z2_10), 2.0f * (xy_14 + wz_10), 2.0f * (xz_10 - wy_10), 2.0f * (xy_14 - wz_10), 1.0f - 2.0f * (x2_14 + z2_10), 2.0f * (yz_10 + wx_10), 2.0f * (xz_10 + wy_10), 2.0f * (yz_10 - wx_10), 1.0f - 2.0f * (x2_14 + y2_14)));
    Matrix<float, 3, 3>  S_2 = makeMatrix<float, 3, 3> (_S537.x, 0.0f, 0.0f, 0.0f, _S537.y, 0.0f, 0.0f, 0.0f, _S537.z);
    Matrix<float, 3, 3>  _S542 = s_primal_ctx_mul_1(_S541, S_2);
    Matrix<float, 3, 3>  _S543 = transpose_0(_S542);
    Matrix<float, 3, 3>  _S544 = s_primal_ctx_mul_1(_S542, _S543);
    Matrix<float, 3, 3>  _S545 = s_primal_ctx_mul_1(R_10, _S544);
    Matrix<float, 3, 3>  _S546 = transpose_0(R_10);
    Matrix<float, 3, 3>  _S547 = s_primal_ctx_mul_1(_S545, _S546);
    Matrix<float, 2, 3>  J_12 = makeMatrix<float, 2, 3> (fx_12, 0.0f, 0.0f, 0.0f, fy_12, 0.0f);
    Matrix<float, 2, 3>  _S548 = s_primal_ctx_mul_2(J_12, _S547);
    Matrix<float, 3, 2>  _S549 = transpose_1(J_12);
    Matrix<float, 2, 2>  _S550 = s_primal_ctx_mul_3(_S548, _S549);
    float _S551 = _S550.rows[int(0)].y * _S550.rows[int(1)].x;
    float det_orig_9 = _S550.rows[int(0)].x * _S550.rows[int(1)].y - _S551;
    float _S552 = _S550.rows[int(0)].x + eps2d_9;
    Matrix<float, 2, 2>  _S553 = _S550;
    *&(((&_S553)->rows + (int(0)))->x) = _S552;
    float _S554 = _S550.rows[int(1)].y + eps2d_9;
    *&(((&_S553)->rows + (int(1)))->y) = _S554;
    Matrix<float, 2, 2>  _S555 = _S553;
    Matrix<float, 2, 2>  _S556 = _S553;
    float det_blur_6 = _S552 * _S554 - _S551;
    float _S557 = det_orig_9 / det_blur_6;
    float _S558 = det_blur_6 * det_blur_6;
    float _S559 = s_primal_ctx_max_0(0.0f, _S557);
    float _S560 = s_primal_ctx_sqrt_0(_S559);
    float invdet_9 = 1.0f / det_blur_6;
    float _S561 = - _S550.rows[int(0)].y;
    float _S562 = - _S550.rows[int(1)].x;
    float _S563 = - in_opacity_8;
    float _S564 = 1.0f + s_primal_ctx_exp_1(_S563);
    float _S565 = 1.0f / _S564;
    float _S566 = _S564 * _S564;
    float _S567;
    if(antialiased_8)
    {
        _S567 = _S565 * _S560;
    }
    else
    {
        _S567 = _S565;
    }
    float _S568 = _S567 / 0.00392156885936856f;
    float _S569 = 2.0f * s_primal_ctx_log_0(_S568);
    float _S570 = s_primal_ctx_sqrt_0(_S569);
    float _S571 = _S555.rows[int(0)].x;
    float _S572 = _S556.rows[int(1)].y;
    float2  _S573 = make_float2 (0.0f);
    float2  _S574 = _S573;
    *&((&_S574)->y) = v_conic_2.z;
    float2  _S575 = _S573;
    *&((&_S575)->y) = v_conic_2.y;
    *&((&_S575)->x) = v_conic_2.x;
    DiffPair_float_0 _S576;
    (&_S576)->primal_0 = _S572;
    (&_S576)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S576, 0.0f);
    DiffPair_float_0 _S577;
    (&_S577)->primal_0 = _S571;
    (&_S577)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S577, 0.0f);
    DiffPair_float_0 _S578;
    (&_S578)->primal_0 = 3.32999992370605469f;
    (&_S578)->differential_0 = 0.0f;
    DiffPair_float_0 _S579;
    (&_S579)->primal_0 = _S570;
    (&_S579)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S578, &_S579, 0.0f);
    DiffPair_float_0 _S580;
    (&_S580)->primal_0 = _S569;
    (&_S580)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S580, _S579.differential_0);
    float _S581 = 2.0f * _S580.differential_0;
    DiffPair_float_0 _S582;
    (&_S582)->primal_0 = _S568;
    (&_S582)->differential_0 = 0.0f;
    s_bwd_prop_log_0(&_S582, _S581);
    float _S583 = v_opacity_2 + 254.9999847412109375f * _S582.differential_0;
    Matrix<float, 2, 2>  _S584 = makeMatrix<float, 2, 2> (0.0f);
    Matrix<float, 2, 2>  _S585 = _S584;
    _S585[int(1)] = _S574;
    _S585[int(0)] = _S575;
    Matrix<float, 2, 2>  _S586 = _S585;
    float3  _S587 = make_float3 (0.0f, 0.0f, v_depth_2);
    float2  _S588 = make_float2 (_S577.differential_0, 0.0f);
    float2  _S589 = make_float2 (0.0f, _S576.differential_0);
    float _S590;
    if(antialiased_8)
    {
        float _S591 = _S565 * _S583;
        _S567 = _S560 * _S583;
        _S590 = _S591;
    }
    else
    {
        _S567 = _S583;
        _S590 = 0.0f;
    }
    float _S592 = - (_S567 / _S566);
    DiffPair_float_0 _S593;
    (&_S593)->primal_0 = _S563;
    (&_S593)->differential_0 = 0.0f;
    s_bwd_prop_exp_0(&_S593, _S592);
    float _S594 = - _S593.differential_0;
    float _S595 = invdet_9 * _S586.rows[int(1)].y;
    float _S596 = - (invdet_9 * _S586.rows[int(1)].x);
    float _S597 = - (invdet_9 * _S586.rows[int(0)].y);
    float _S598 = invdet_9 * _S586.rows[int(0)].x;
    float _S599 = - ((_S552 * _S586.rows[int(1)].y + _S562 * _S586.rows[int(1)].x + _S561 * _S586.rows[int(0)].y + _S554 * _S586.rows[int(0)].x) / _S558);
    DiffPair_float_0 _S600;
    (&_S600)->primal_0 = _S559;
    (&_S600)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S600, _S590);
    DiffPair_float_0 _S601;
    (&_S601)->primal_0 = 0.0f;
    (&_S601)->differential_0 = 0.0f;
    DiffPair_float_0 _S602;
    (&_S602)->primal_0 = _S557;
    (&_S602)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S601, &_S602, _S600.differential_0);
    float _S603 = _S602.differential_0 / _S558;
    float s_diff_det_orig_T_2 = det_blur_6 * _S603;
    float _S604 = _S599 + det_orig_9 * - _S603;
    float _S605 = - _S604;
    float _S606 = _S552 * _S604;
    float _S607 = _S554 * _S604;
    Matrix<float, 2, 2>  _S608 = _S584;
    _S608[int(1)] = _S589;
    _S608[int(0)] = _S588;
    _S553 = _S608;
    *&(((&_S553)->rows + (int(1)))->y) = 0.0f;
    float _S609 = _S598 + _S606 + _S608.rows[int(1)].y;
    *&(((&_S553)->rows + (int(0)))->x) = 0.0f;
    float _S610 = _S595 + _S607 + _S608.rows[int(0)].x;
    float _S611 = _S605 + - s_diff_det_orig_T_2;
    float _S612 = _S596 + _S550.rows[int(0)].y * _S611;
    float _S613 = _S597 + _S550.rows[int(1)].x * _S611;
    float _S614 = _S550.rows[int(1)].y * s_diff_det_orig_T_2;
    float _S615 = _S609 + _S550.rows[int(0)].x * s_diff_det_orig_T_2;
    float2  _S616 = _S573;
    *&((&_S616)->x) = _S612;
    *&((&_S616)->y) = _S615;
    float _S617 = _S610 + _S614;
    float2  _S618 = _S573;
    *&((&_S618)->y) = _S613;
    *&((&_S618)->x) = _S617;
    float _S619 = fy_12 * v_mean2d_2.y;
    float _S620 = fx_12 * v_mean2d_2.x;
    Matrix<float, 2, 2>  _S621 = _S584;
    _S621[int(1)] = _S616;
    _S621[int(0)] = _S618;
    Matrix<float, 2, 2>  _S622 = _S553 + _S621;
    Matrix<float, 2, 3>  _S623 = makeMatrix<float, 2, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S624;
    (&_S624)->primal_0 = _S548;
    (&_S624)->differential_0 = _S623;
    Matrix<float, 3, 2>  _S625 = makeMatrix<float, 3, 2> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C2x3E_0 _S626;
    (&_S626)->primal_0 = _S549;
    (&_S626)->differential_0 = _S625;
    s_bwd_prop_mul_0(&_S624, &_S626, _S622);
    DiffPair_matrixx3Cfloatx2C2x2C3x3E_0 _S627;
    (&_S627)->primal_0 = J_12;
    (&_S627)->differential_0 = _S623;
    Matrix<float, 3, 3>  _S628 = makeMatrix<float, 3, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S629;
    (&_S629)->primal_0 = _S547;
    (&_S629)->differential_0 = _S628;
    s_bwd_prop_mul_1(&_S627, &_S629, _S624.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S630;
    (&_S630)->primal_0 = _S545;
    (&_S630)->differential_0 = _S628;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S631;
    (&_S631)->primal_0 = _S546;
    (&_S631)->differential_0 = _S628;
    s_bwd_prop_mul_2(&_S630, &_S631, _S629.differential_0);
    Matrix<float, 3, 3>  _S632 = transpose_0(_S631.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S633;
    (&_S633)->primal_0 = R_10;
    (&_S633)->differential_0 = _S628;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S634;
    (&_S634)->primal_0 = _S544;
    (&_S634)->differential_0 = _S628;
    s_bwd_prop_mul_2(&_S633, &_S634, _S630.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S635;
    (&_S635)->primal_0 = _S542;
    (&_S635)->differential_0 = _S628;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S636;
    (&_S636)->primal_0 = _S543;
    (&_S636)->differential_0 = _S628;
    s_bwd_prop_mul_2(&_S635, &_S636, _S634.differential_0);
    Matrix<float, 3, 3>  _S637 = _S635.differential_0 + transpose_0(_S636.differential_0);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S638;
    (&_S638)->primal_0 = _S541;
    (&_S638)->differential_0 = _S628;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S639;
    (&_S639)->primal_0 = S_2;
    (&_S639)->differential_0 = _S628;
    s_bwd_prop_mul_2(&_S638, &_S639, _S637);
    Matrix<float, 3, 3>  _S640 = transpose_0(_S638.differential_0);
    float _S641 = 2.0f * - _S640.rows[int(2)].z;
    float _S642 = 2.0f * _S640.rows[int(2)].y;
    float _S643 = 2.0f * _S640.rows[int(2)].x;
    float _S644 = 2.0f * _S640.rows[int(1)].z;
    float _S645 = 2.0f * - _S640.rows[int(1)].y;
    float _S646 = 2.0f * _S640.rows[int(1)].x;
    float _S647 = 2.0f * _S640.rows[int(0)].z;
    float _S648 = 2.0f * _S640.rows[int(0)].y;
    float _S649 = 2.0f * - _S640.rows[int(0)].x;
    float _S650 = - _S646 + _S648;
    float _S651 = _S643 + - _S647;
    float _S652 = - _S642 + _S644;
    float _S653 = _S642 + _S644;
    float _S654 = _S643 + _S647;
    float _S655 = _S646 + _S648;
    float _S656 = z_10 * (_S645 + _S649);
    float _S657 = y_11 * (_S641 + _S649);
    float _S658 = x_24 * (_S641 + _S645);
    float _S659 = z_10 * _S650 + y_11 * _S651 + x_24 * _S652;
    float _S660 = _S540 * _S659;
    float _S661 = w_10 * _S650 + y_11 * _S653 + x_24 * _S654 + _S656 + _S656;
    float _S662 = _S540 * _S661;
    float _S663 = w_10 * _S651 + z_10 * _S653 + x_24 * _S655 + _S657 + _S657;
    float _S664 = _S540 * _S663;
    float _S665 = w_10 * _S652 + z_10 * _S654 + y_11 * _S655 + _S658 + _S658;
    float _S666 = _S540 * _S665;
    float _S667 = quat_10.x * _S659 + quat_10.w * _S661 + quat_10.z * _S663 + quat_10.y * _S665;
    DiffPair_float_0 _S668;
    (&_S668)->primal_0 = _S539;
    (&_S668)->differential_0 = 0.0f;
    s_bwd_prop_rsqrt_0(&_S668, _S667);
    float _S669 = quat_10.x * _S668.differential_0;
    float _S670 = quat_10.w * _S668.differential_0;
    float _S671 = quat_10.z * _S668.differential_0;
    float _S672 = quat_10.y * _S668.differential_0;
    float _S673 = _S662 + _S670 + _S670;
    float _S674 = _S664 + _S671 + _S671;
    float _S675 = _S666 + _S672 + _S672;
    float _S676 = _S660 + _S669 + _S669;
    float3  _S677 = make_float3 (0.0f);
    float3  _S678 = _S677;
    *&((&_S678)->z) = _S639.differential_0.rows[int(2)].z;
    *&((&_S678)->y) = _S639.differential_0.rows[int(1)].y;
    *&((&_S678)->x) = _S639.differential_0.rows[int(0)].x;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S679;
    (&_S679)->primal_0 = scale_9;
    (&_S679)->differential_0 = _S677;
    s_bwd_prop_exp_1(&_S679, _S678);
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S680 = _S679;
    float3  _S681 = _S677;
    *&((&_S681)->y) = _S619;
    *&((&_S681)->x) = _S620;
    float3  _S682 = _S587 + _S681;
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S683;
    (&_S683)->primal_0 = R_10;
    (&_S683)->differential_0 = _S628;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S684;
    (&_S684)->primal_0 = mean_8;
    (&_S684)->differential_0 = _S677;
    s_bwd_prop_mul_3(&_S683, &_S684, _S682);
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S685 = _S684;
    Matrix<float, 3, 3>  _S686 = _S632 + _S633.differential_0 + _S683.differential_0;
    float4  _S687 = make_float4 (0.0f);
    *&((&_S687)->w) = _S673;
    *&((&_S687)->z) = _S674;
    *&((&_S687)->y) = _S675;
    *&((&_S687)->x) = _S676;
    float4  _S688 = _S687;
    *v_mean_2 = _S685.differential_0;
    *v_quat_2 = _S688;
    *v_scale_2 = _S680.differential_0;
    *v_in_opacity_2 = _S594;
    *v_R_2 = _S686;
    *v_t_2 = _S682;
    return;
}

