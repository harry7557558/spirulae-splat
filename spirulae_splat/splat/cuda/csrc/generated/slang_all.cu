#include "hip/hip_runtime.h"
#include "slang.cuh"

struct DiffPair_float_0
{
    float primal_0;
    float differential_0;
};

__device__ void _d_exp_0(DiffPair_float_0 * dpx_0, float dOut_0)
{
    float _S1 = (F32_exp(((*dpx_0).primal_0))) * dOut_0;
    dpx_0->primal_0 = (*dpx_0).primal_0;
    dpx_0->differential_0 = _S1;
    return;
}

__device__ void _d_max_0(DiffPair_float_0 * dpx_1, DiffPair_float_0 * dpy_0, float dOut_1)
{
    DiffPair_float_0 _S2 = *dpx_1;
    float _S3;
    if(((*dpx_1).primal_0) > ((*dpy_0).primal_0))
    {
        _S3 = dOut_1;
    }
    else
    {
        if(((*dpx_1).primal_0) < ((*dpy_0).primal_0))
        {
            _S3 = 0.0f;
        }
        else
        {
            _S3 = 0.5f * dOut_1;
        }
    }
    dpx_1->primal_0 = _S2.primal_0;
    dpx_1->differential_0 = _S3;
    DiffPair_float_0 _S4 = *dpy_0;
    if(((*dpy_0).primal_0) > (_S2.primal_0))
    {
        _S3 = dOut_1;
    }
    else
    {
        if(((*dpy_0).primal_0) < ((*dpx_1).primal_0))
        {
            _S3 = 0.0f;
        }
        else
        {
            _S3 = 0.5f * dOut_1;
        }
    }
    dpy_0->primal_0 = _S4.primal_0;
    dpy_0->differential_0 = _S3;
    return;
}

__device__ void _d_sqrt_0(DiffPair_float_0 * dpx_2, float dOut_2)
{
    float _S5 = 0.5f / (F32_sqrt(((F32_max((1.00000001168609742e-07f), ((*dpx_2).primal_0)))))) * dOut_2;
    dpx_2->primal_0 = (*dpx_2).primal_0;
    dpx_2->differential_0 = _S5;
    return;
}

__device__ float dot_0(float4  x_0, float4  y_0)
{
    int i_0 = int(0);
    float result_0 = 0.0f;
    for(;;)
    {
        if(i_0 < int(4))
        {
        }
        else
        {
            break;
        }
        float result_1 = result_0 + _slang_vector_get_element(x_0, i_0) * _slang_vector_get_element(y_0, i_0);
        i_0 = i_0 + int(1);
        result_0 = result_1;
    }
    return result_0;
}

__device__ float length_0(float4  x_1)
{
    return (F32_sqrt((dot_0(x_1, x_1))));
}

__device__ void _d_log_0(DiffPair_float_0 * dpx_3, float dOut_3)
{
    float _S6 = 1.0f / (*dpx_3).primal_0 * dOut_3;
    dpx_3->primal_0 = (*dpx_3).primal_0;
    dpx_3->differential_0 = _S6;
    return;
}

struct DiffPair_vectorx3Cfloatx2C3x3E_0
{
    float3  primal_0;
    float3  differential_0;
};

__device__ float3  exp_0(float3  x_2)
{
    float3  result_2;
    int i_1 = int(0);
    for(;;)
    {
        if(i_1 < int(3))
        {
        }
        else
        {
            break;
        }
        *_slang_vector_get_element_ptr(&result_2, i_1) = (F32_exp((_slang_vector_get_element(x_2, i_1))));
        i_1 = i_1 + int(1);
    }
    return result_2;
}

__device__ void _d_exp_vector_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dpx_4, float3  dOut_4)
{
    float3  _S7 = exp_0((*dpx_4).primal_0) * dOut_4;
    dpx_4->primal_0 = (*dpx_4).primal_0;
    dpx_4->differential_0 = _S7;
    return;
}

struct DiffPair_vectorx3Cfloatx2C2x3E_0
{
    float2  primal_0;
    float2  differential_0;
};

__device__ float2  exp_1(float2  x_3)
{
    float2  result_3;
    int i_2 = int(0);
    for(;;)
    {
        if(i_2 < int(2))
        {
        }
        else
        {
            break;
        }
        *_slang_vector_get_element_ptr(&result_3, i_2) = (F32_exp((_slang_vector_get_element(x_3, i_2))));
        i_2 = i_2 + int(1);
    }
    return result_3;
}

__device__ void _d_exp_vector_1(DiffPair_vectorx3Cfloatx2C2x3E_0 * dpx_5, float2  dOut_5)
{
    float2  _S8 = exp_1((*dpx_5).primal_0) * dOut_5;
    dpx_5->primal_0 = (*dpx_5).primal_0;
    dpx_5->differential_0 = _S8;
    return;
}

__device__ void _d_min_0(DiffPair_float_0 * dpx_6, DiffPair_float_0 * dpy_1, float dOut_6)
{
    DiffPair_float_0 _S9 = *dpx_6;
    float _S10;
    if(((*dpx_6).primal_0) < ((*dpy_1).primal_0))
    {
        _S10 = dOut_6;
    }
    else
    {
        if(((*dpx_6).primal_0) > ((*dpy_1).primal_0))
        {
            _S10 = 0.0f;
        }
        else
        {
            _S10 = 0.5f * dOut_6;
        }
    }
    dpx_6->primal_0 = _S9.primal_0;
    dpx_6->differential_0 = _S10;
    DiffPair_float_0 _S11 = *dpy_1;
    if(((*dpy_1).primal_0) < (_S9.primal_0))
    {
        _S10 = dOut_6;
    }
    else
    {
        if(((*dpy_1).primal_0) > ((*dpx_6).primal_0))
        {
            _S10 = 0.0f;
        }
        else
        {
            _S10 = 0.5f * dOut_6;
        }
    }
    dpy_1->primal_0 = _S11.primal_0;
    dpy_1->differential_0 = _S10;
    return;
}

__device__ void per_splat_losses(bool is_3dgs_0, float3  scales_0, float opacity_0, float4  quat_0, float mcmc_opacity_reg_weight_0, float mcmc_scale_reg_weight_0, float max_gauss_ratio_0, float scale_regularization_weight_0, float erank_reg_weight_0, float erank_reg_weight_s3_0, float quat_norm_reg_weight_0, FixedArray<float, 5>  * _S12)
{
    FixedArray<float, 5>  losses_0;
    losses_0[int(0)] = mcmc_opacity_reg_weight_0 * (1.0f / (1.0f + (F32_exp((- opacity_0)))));
    float quat_norm_0 = length_0(quat_0);
    losses_0[int(4)] = quat_norm_reg_weight_0 * (quat_norm_0 - 1.0f - (F32_log((quat_norm_0))));
    if(is_3dgs_0)
    {
        float3  _S13 = exp_0(scales_0);
        float _S14 = _S13.x;
        float _S15 = _S13.y;
        float _S16 = _S13.z;
        losses_0[int(1)] = mcmc_scale_reg_weight_0 * (_S14 + _S15 + _S16) / 3.0f;
        losses_0[int(2)] = scale_regularization_weight_0 * ((F32_max(((F32_max(((F32_max((_S14), (_S15)))), (_S16))) / (F32_min(((F32_min((_S14), (_S15)))), (_S16)))), (max_gauss_ratio_0))) - max_gauss_ratio_0);
        float3  _S17 = exp_0(make_float3 (2.0f) * scales_0);
        float x_4 = _S17.x;
        float y_1 = _S17.y;
        float z_0 = _S17.z;
        float s_0 = x_4 + y_1 + z_0;
        float s1_0 = (F32_max(((F32_max((x_4), (y_1)))), (z_0))) / s_0;
        float s3_0 = (F32_min(((F32_min((x_4), (y_1)))), (z_0))) / s_0;
        float s2_0 = 1.0f - s1_0 - s3_0;
        losses_0[int(3)] = erank_reg_weight_0 * (F32_max((- (F32_log(((F32_exp((- s1_0 * (F32_log((s1_0))) - s2_0 * (F32_log((s2_0))) - s3_0 * (F32_log((s3_0)))))) - 0.99998998641967773f)))), (0.0f))) + erank_reg_weight_s3_0 * s3_0;
    }
    else
    {
        float2  _S18 = float2 {scales_0.x, scales_0.y};
        float2  _S19 = exp_1(_S18);
        float _S20 = _S19.x;
        float _S21 = _S19.y;
        losses_0[int(1)] = mcmc_scale_reg_weight_0 * (_S20 + _S21) / 2.0f;
        losses_0[int(2)] = scale_regularization_weight_0 * ((F32_max(((F32_max((_S20), (_S21))) / (F32_min((_S20), (_S21)))), (max_gauss_ratio_0))) - max_gauss_ratio_0);
        float2  _S22 = exp_1(make_float2 (2.0f) * _S18);
        float x_5 = _S22.x;
        float y_2 = _S22.y;
        float s_1 = x_5 + y_2;
        float s1_1 = (F32_max((x_5), (y_2))) / s_1;
        float s2_1 = (F32_min((x_5), (y_2))) / s_1;
        losses_0[int(3)] = erank_reg_weight_0 * (F32_max((- (F32_log(((F32_exp((- s1_1 * (F32_log((s1_1))) - s2_1 * (F32_log((s2_1)))))) - 0.99998998641967773f)))), (0.0f)));
    }
    *_S12 = losses_0;
    return;
}

struct DiffPair_vectorx3Cfloatx2C4x3E_0
{
    float4  primal_0;
    float4  differential_0;
};

__device__ float s_primal_ctx_exp_0(float _S23)
{
    return (F32_exp((_S23)));
}

__device__ float2  s_primal_ctx_exp_1(float2  _S24)
{
    return exp_1(_S24);
}

__device__ float s_primal_ctx_max_0(float _S25, float _S26)
{
    return (F32_max((_S25), (_S26)));
}

__device__ float s_primal_ctx_min_0(float _S27, float _S28)
{
    return (F32_min((_S27), (_S28)));
}

__device__ float s_primal_ctx_log_0(float _S29)
{
    return (F32_log((_S29)));
}

__device__ float3  s_primal_ctx_exp_2(float3  _S30)
{
    return exp_0(_S30);
}

__device__ void s_bwd_prop_max_0(DiffPair_float_0 * _S31, DiffPair_float_0 * _S32, float _S33)
{
    _d_max_0(_S31, _S32, _S33);
    return;
}

__device__ void s_bwd_prop_log_0(DiffPair_float_0 * _S34, float _S35)
{
    _d_log_0(_S34, _S35);
    return;
}

__device__ void s_bwd_prop_exp_0(DiffPair_float_0 * _S36, float _S37)
{
    _d_exp_0(_S36, _S37);
    return;
}

__device__ void s_bwd_prop_min_0(DiffPair_float_0 * _S38, DiffPair_float_0 * _S39, float _S40)
{
    _d_min_0(_S38, _S39, _S40);
    return;
}

__device__ void s_bwd_prop_exp_1(DiffPair_vectorx3Cfloatx2C2x3E_0 * _S41, float2  _S42)
{
    _d_exp_vector_1(_S41, _S42);
    return;
}

__device__ void s_bwd_prop_exp_2(DiffPair_vectorx3Cfloatx2C3x3E_0 * _S43, float3  _S44)
{
    _d_exp_vector_0(_S43, _S44);
    return;
}

__device__ void s_bwd_prop_sqrt_0(DiffPair_float_0 * _S45, float _S46)
{
    _d_sqrt_0(_S45, _S46);
    return;
}

__device__ void s_bwd_prop_length_impl_0(DiffPair_vectorx3Cfloatx2C4x3E_0 * dpx_7, float _s_dOut_0)
{
    float _S47 = (*dpx_7).primal_0.x;
    float _S48 = (*dpx_7).primal_0.y;
    float _S49 = (*dpx_7).primal_0.z;
    float _S50 = (*dpx_7).primal_0.w;
    DiffPair_float_0 _S51;
    (&_S51)->primal_0 = _S47 * _S47 + _S48 * _S48 + _S49 * _S49 + _S50 * _S50;
    (&_S51)->differential_0 = 0.0f;
    s_bwd_prop_sqrt_0(&_S51, _s_dOut_0);
    float _S52 = (*dpx_7).primal_0.w * _S51.differential_0;
    float _S53 = _S52 + _S52;
    float _S54 = (*dpx_7).primal_0.z * _S51.differential_0;
    float _S55 = _S54 + _S54;
    float _S56 = (*dpx_7).primal_0.y * _S51.differential_0;
    float _S57 = _S56 + _S56;
    float _S58 = (*dpx_7).primal_0.x * _S51.differential_0;
    float _S59 = _S58 + _S58;
    float4  _S60 = make_float4 (0.0f);
    *&((&_S60)->w) = _S53;
    *&((&_S60)->z) = _S55;
    *&((&_S60)->y) = _S57;
    *&((&_S60)->x) = _S59;
    dpx_7->primal_0 = (*dpx_7).primal_0;
    dpx_7->differential_0 = _S60;
    return;
}

__device__ void s_bwd_length_impl_0(DiffPair_vectorx3Cfloatx2C4x3E_0 * _S61, float _S62)
{
    s_bwd_prop_length_impl_0(_S61, _S62);
    return;
}

__device__ void s_bwd_prop_per_splat_losses_0(bool is_3dgs_1, DiffPair_vectorx3Cfloatx2C3x3E_0 * dpscales_0, DiffPair_float_0 * dpopacity_0, DiffPair_vectorx3Cfloatx2C4x3E_0 * dpquat_0, float mcmc_opacity_reg_weight_1, float mcmc_scale_reg_weight_1, float max_gauss_ratio_1, float scale_regularization_weight_1, float erank_reg_weight_1, float erank_reg_weight_s3_1, float quat_norm_reg_weight_1, FixedArray<float, 5>  _s_dOut_1)
{
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S63 = *dpscales_0;
    DiffPair_vectorx3Cfloatx2C4x3E_0 _S64 = *dpquat_0;
    float2  _S65 = make_float2 (0.0f);
    float3  _S66 = make_float3 (0.0f);
    float _S67 = - (*dpopacity_0).primal_0;
    float _S68 = 1.0f + s_primal_ctx_exp_0(_S67);
    float _S69 = _S68 * _S68;
    float _S70 = length_0((*dpquat_0).primal_0);
    float _S71;
    float _S72;
    float _S73;
    float _S74;
    float _S75;
    float _S76;
    float _S77;
    float _S78;
    float _S79;
    float _S80;
    float _S81;
    float _S82;
    float _S83;
    float _S84;
    float _S85;
    float _S86;
    float _S87;
    float _S88;
    float _S89;
    float _S90;
    float _S91;
    float _S92;
    float _S93;
    float _S94;
    float _S95;
    float _S96;
    float _S97;
    float _S98;
    float _S99;
    float _S100;
    float _S101;
    float _S102;
    float _S103;
    float _S104;
    float _S105;
    float _S106;
    float _S107;
    float _S108;
    float _S109;
    float _S110;
    float _S111;
    float _S112;
    float _S113;
    float _S114;
    float _S115;
    float _S116;
    float _S117;
    float _S118;
    float _S119;
    float _S120;
    float2  _S121;
    float2  _S122;
    float3  _S123;
    if(is_3dgs_1)
    {
        float3  _S124 = s_primal_ctx_exp_2(_S63.primal_0);
        float _S125 = _S124.x;
        float _S126 = _S124.y;
        float _S127 = _S124.z;
        float _S128 = s_primal_ctx_max_0(_S125, _S126);
        float _S129 = s_primal_ctx_max_0(_S128, _S127);
        float _S130 = s_primal_ctx_min_0(_S125, _S126);
        float _S131 = s_primal_ctx_min_0(_S130, _S127);
        float _S132 = _S129 / _S131;
        float _S133 = _S131 * _S131;
        float3  _S134 = make_float3 (2.0f) * _S63.primal_0;
        float3  _S135 = s_primal_ctx_exp_2(_S134);
        float x_6 = _S135.x;
        float y_3 = _S135.y;
        float z_1 = _S135.z;
        float s_2 = x_6 + y_3 + z_1;
        float _S136 = s_primal_ctx_max_0(x_6, y_3);
        float _S137 = s_primal_ctx_max_0(_S136, z_1);
        float s1_2 = _S137 / s_2;
        float _S138 = s_2 * s_2;
        float _S139 = s_primal_ctx_min_0(x_6, y_3);
        float _S140 = s_primal_ctx_min_0(_S139, z_1);
        float s3_1 = _S140 / s_2;
        float s2_2 = 1.0f - s1_2 - s3_1;
        float _S141 = - s1_2;
        float _S142 = s_primal_ctx_log_0(s1_2);
        float _S143 = s_primal_ctx_log_0(s2_2);
        float _S144 = s_primal_ctx_log_0(s3_1);
        float _S145 = _S141 * _S142 - s2_2 * _S143 - s3_1 * _S144;
        float _S146 = s_primal_ctx_exp_0(_S145) - 0.99998998641967773f;
        float _S147 = - s_primal_ctx_log_0(_S146);
        _S71 = 0.0f;
        _S72 = 0.0f;
        _S73 = 0.0f;
        _S74 = 0.0f;
        _S75 = 0.0f;
        _S76 = 0.0f;
        _S77 = 0.0f;
        _S78 = 0.0f;
        _S79 = 0.0f;
        _S80 = 0.0f;
        _S81 = 0.0f;
        _S82 = 0.0f;
        _S83 = 0.0f;
        _S84 = 0.0f;
        _S85 = 0.0f;
        _S121 = _S65;
        _S86 = 0.0f;
        _S87 = 0.0f;
        _S88 = 0.0f;
        _S89 = 0.0f;
        _S90 = 0.0f;
        _S91 = 0.0f;
        _S122 = _S65;
        _S92 = _S147;
        _S93 = _S146;
        _S94 = _S145;
        _S95 = s3_1;
        _S96 = _S144;
        _S97 = s2_2;
        _S98 = _S143;
        _S99 = _S141;
        _S100 = _S142;
        _S101 = s1_2;
        _S102 = _S138;
        _S103 = _S140;
        _S104 = s_2;
        _S105 = _S139;
        _S106 = z_1;
        _S107 = x_6;
        _S108 = y_3;
        _S109 = _S138;
        _S110 = _S137;
        _S111 = _S136;
        _S123 = _S134;
        _S112 = _S132;
        _S113 = _S133;
        _S114 = _S129;
        _S115 = _S131;
        _S116 = _S130;
        _S117 = _S127;
        _S118 = _S125;
        _S119 = _S126;
        _S120 = _S128;
    }
    else
    {
        float2  _S148 = float2 {_S63.primal_0.x, _S63.primal_0.y};
        float2  _S149 = s_primal_ctx_exp_1(_S148);
        float _S150 = _S149.x;
        float _S151 = _S149.y;
        float _S152 = s_primal_ctx_max_0(_S150, _S151);
        float _S153 = s_primal_ctx_min_0(_S150, _S151);
        float _S154 = _S152 / _S153;
        float _S155 = _S153 * _S153;
        float2  _S156 = make_float2 (2.0f) * _S148;
        float2  _S157 = s_primal_ctx_exp_1(_S156);
        float x_7 = _S157.x;
        float y_4 = _S157.y;
        float s_3 = x_7 + y_4;
        float _S158 = s_primal_ctx_max_0(x_7, y_4);
        float s1_3 = _S158 / s_3;
        float _S159 = s_3 * s_3;
        float _S160 = s_primal_ctx_min_0(x_7, y_4);
        float s2_3 = _S160 / s_3;
        float _S161 = - s1_3;
        float _S162 = s_primal_ctx_log_0(s1_3);
        float _S163 = s_primal_ctx_log_0(s2_3);
        float _S164 = _S161 * _S162 - s2_3 * _S163;
        float _S165 = s_primal_ctx_exp_0(_S164) - 0.99998998641967773f;
        _S71 = - s_primal_ctx_log_0(_S165);
        _S72 = _S165;
        _S73 = _S164;
        _S74 = s2_3;
        _S75 = _S163;
        _S76 = _S161;
        _S77 = _S162;
        _S78 = s1_3;
        _S79 = _S159;
        _S80 = _S160;
        _S81 = s_3;
        _S82 = x_7;
        _S83 = y_4;
        _S84 = _S159;
        _S85 = _S158;
        _S121 = _S156;
        _S86 = _S154;
        _S87 = _S155;
        _S88 = _S152;
        _S89 = _S153;
        _S90 = _S150;
        _S91 = _S151;
        _S122 = _S148;
        _S92 = 0.0f;
        _S93 = 0.0f;
        _S94 = 0.0f;
        _S95 = 0.0f;
        _S96 = 0.0f;
        _S97 = 0.0f;
        _S98 = 0.0f;
        _S99 = 0.0f;
        _S100 = 0.0f;
        _S101 = 0.0f;
        _S102 = 0.0f;
        _S103 = 0.0f;
        _S104 = 0.0f;
        _S105 = 0.0f;
        _S106 = 0.0f;
        _S107 = 0.0f;
        _S108 = 0.0f;
        _S109 = 0.0f;
        _S110 = 0.0f;
        _S111 = 0.0f;
        _S123 = _S66;
        _S112 = 0.0f;
        _S113 = 0.0f;
        _S114 = 0.0f;
        _S115 = 0.0f;
        _S116 = 0.0f;
        _S117 = 0.0f;
        _S118 = 0.0f;
        _S119 = 0.0f;
        _S120 = 0.0f;
    }
    if(is_3dgs_1)
    {
        float _S166 = erank_reg_weight_s3_1 * _s_dOut_1[int(3)];
        float _S167 = erank_reg_weight_1 * _s_dOut_1[int(3)];
        DiffPair_float_0 _S168;
        (&_S168)->primal_0 = _S92;
        (&_S168)->differential_0 = 0.0f;
        DiffPair_float_0 _S169;
        (&_S169)->primal_0 = 0.0f;
        (&_S169)->differential_0 = 0.0f;
        s_bwd_prop_max_0(&_S168, &_S169, _S167);
        float _S170 = - _S168.differential_0;
        DiffPair_float_0 _S171;
        (&_S171)->primal_0 = _S93;
        (&_S171)->differential_0 = 0.0f;
        s_bwd_prop_log_0(&_S171, _S170);
        DiffPair_float_0 _S172;
        (&_S172)->primal_0 = _S94;
        (&_S172)->differential_0 = 0.0f;
        s_bwd_prop_exp_0(&_S172, _S171.differential_0);
        float _S173 = - _S172.differential_0;
        float _S174 = _S95 * _S173;
        float _S175 = _S96 * _S173;
        DiffPair_float_0 _S176;
        (&_S176)->primal_0 = _S95;
        (&_S176)->differential_0 = 0.0f;
        s_bwd_prop_log_0(&_S176, _S174);
        float _S177 = _S97 * _S173;
        float _S178 = _S98 * _S173;
        DiffPair_float_0 _S179;
        (&_S179)->primal_0 = _S97;
        (&_S179)->differential_0 = 0.0f;
        s_bwd_prop_log_0(&_S179, _S177);
        float _S180 = _S99 * _S172.differential_0;
        float _S181 = _S100 * _S172.differential_0;
        DiffPair_float_0 _S182;
        (&_S182)->primal_0 = _S101;
        (&_S182)->differential_0 = 0.0f;
        s_bwd_prop_log_0(&_S182, _S180);
        float _S183 = - _S181;
        float _S184 = - (_S178 + _S179.differential_0);
        float _S185 = (_S166 + _S175 + _S176.differential_0 + _S184) / _S102;
        float _S186 = _S103 * - _S185;
        float _S187 = _S104 * _S185;
        DiffPair_float_0 _S188;
        (&_S188)->primal_0 = _S105;
        (&_S188)->differential_0 = 0.0f;
        DiffPair_float_0 _S189;
        (&_S189)->primal_0 = _S106;
        (&_S189)->differential_0 = 0.0f;
        s_bwd_prop_min_0(&_S188, &_S189, _S187);
        DiffPair_float_0 _S190;
        (&_S190)->primal_0 = _S107;
        (&_S190)->differential_0 = 0.0f;
        DiffPair_float_0 _S191;
        (&_S191)->primal_0 = _S108;
        (&_S191)->differential_0 = 0.0f;
        s_bwd_prop_min_0(&_S190, &_S191, _S188.differential_0);
        float _S192 = (_S182.differential_0 + _S183 + _S184) / _S109;
        float _S193 = _S110 * - _S192;
        float _S194 = _S104 * _S192;
        DiffPair_float_0 _S195;
        (&_S195)->primal_0 = _S111;
        (&_S195)->differential_0 = 0.0f;
        DiffPair_float_0 _S196;
        (&_S196)->primal_0 = _S106;
        (&_S196)->differential_0 = 0.0f;
        s_bwd_prop_max_0(&_S195, &_S196, _S194);
        DiffPair_float_0 _S197;
        (&_S197)->primal_0 = _S107;
        (&_S197)->differential_0 = 0.0f;
        DiffPair_float_0 _S198;
        (&_S198)->primal_0 = _S108;
        (&_S198)->differential_0 = 0.0f;
        s_bwd_prop_max_0(&_S197, &_S198, _S195.differential_0);
        float _S199 = _S186 + _S193;
        float3  _S200 = make_float3 (_S190.differential_0 + _S197.differential_0 + _S199, _S191.differential_0 + _S198.differential_0 + _S199, _S189.differential_0 + _S196.differential_0 + _S199);
        DiffPair_vectorx3Cfloatx2C3x3E_0 _S201;
        (&_S201)->primal_0 = _S123;
        (&_S201)->differential_0 = _S66;
        s_bwd_prop_exp_2(&_S201, _S200);
        float3  _S202 = make_float3 (2.0f) * _S201.differential_0;
        float s_diff_scale_reg_T_0 = scale_regularization_weight_1 * _s_dOut_1[int(2)];
        DiffPair_float_0 _S203;
        (&_S203)->primal_0 = _S112;
        (&_S203)->differential_0 = 0.0f;
        DiffPair_float_0 _S204;
        (&_S204)->primal_0 = max_gauss_ratio_1;
        (&_S204)->differential_0 = 0.0f;
        s_bwd_prop_max_0(&_S203, &_S204, s_diff_scale_reg_T_0);
        float _S205 = _S203.differential_0 / _S113;
        float _S206 = _S114 * - _S205;
        float _S207 = _S115 * _S205;
        DiffPair_float_0 _S208;
        (&_S208)->primal_0 = _S116;
        (&_S208)->differential_0 = 0.0f;
        DiffPair_float_0 _S209;
        (&_S209)->primal_0 = _S117;
        (&_S209)->differential_0 = 0.0f;
        s_bwd_prop_min_0(&_S208, &_S209, _S206);
        DiffPair_float_0 _S210;
        (&_S210)->primal_0 = _S118;
        (&_S210)->differential_0 = 0.0f;
        DiffPair_float_0 _S211;
        (&_S211)->primal_0 = _S119;
        (&_S211)->differential_0 = 0.0f;
        s_bwd_prop_min_0(&_S210, &_S211, _S208.differential_0);
        DiffPair_float_0 _S212;
        (&_S212)->primal_0 = _S120;
        (&_S212)->differential_0 = 0.0f;
        DiffPair_float_0 _S213;
        (&_S213)->primal_0 = _S117;
        (&_S213)->differential_0 = 0.0f;
        s_bwd_prop_max_0(&_S212, &_S213, _S207);
        DiffPair_float_0 _S214;
        (&_S214)->primal_0 = _S118;
        (&_S214)->differential_0 = 0.0f;
        DiffPair_float_0 _S215;
        (&_S215)->primal_0 = _S119;
        (&_S215)->differential_0 = 0.0f;
        s_bwd_prop_max_0(&_S214, &_S215, _S212.differential_0);
        float _S216 = mcmc_scale_reg_weight_1 * (0.3333333432674408f * _s_dOut_1[int(1)]);
        float3  _S217 = make_float3 (_S210.differential_0 + _S214.differential_0 + _S216, _S211.differential_0 + _S215.differential_0 + _S216, _S209.differential_0 + _S213.differential_0 + _S216);
        DiffPair_vectorx3Cfloatx2C3x3E_0 _S218;
        (&_S218)->primal_0 = _S63.primal_0;
        (&_S218)->differential_0 = _S66;
        s_bwd_prop_exp_2(&_S218, _S217);
        float3  _S219 = _S202 + _S218.differential_0;
        _S71 = _s_dOut_1[int(4)];
        _S72 = _s_dOut_1[int(0)];
        _S123 = _S219;
    }
    else
    {
        float _S220 = erank_reg_weight_1 * _s_dOut_1[int(3)];
        DiffPair_float_0 _S221;
        (&_S221)->primal_0 = _S71;
        (&_S221)->differential_0 = 0.0f;
        DiffPair_float_0 _S222;
        (&_S222)->primal_0 = 0.0f;
        (&_S222)->differential_0 = 0.0f;
        s_bwd_prop_max_0(&_S221, &_S222, _S220);
        float _S223 = - _S221.differential_0;
        DiffPair_float_0 _S224;
        (&_S224)->primal_0 = _S72;
        (&_S224)->differential_0 = 0.0f;
        s_bwd_prop_log_0(&_S224, _S223);
        DiffPair_float_0 _S225;
        (&_S225)->primal_0 = _S73;
        (&_S225)->differential_0 = 0.0f;
        s_bwd_prop_exp_0(&_S225, _S224.differential_0);
        float _S226 = - _S225.differential_0;
        float _S227 = _S74 * _S226;
        float _S228 = _S75 * _S226;
        DiffPair_float_0 _S229;
        (&_S229)->primal_0 = _S74;
        (&_S229)->differential_0 = 0.0f;
        s_bwd_prop_log_0(&_S229, _S227);
        float _S230 = _S76 * _S225.differential_0;
        float _S231 = _S77 * _S225.differential_0;
        DiffPair_float_0 _S232;
        (&_S232)->primal_0 = _S78;
        (&_S232)->differential_0 = 0.0f;
        s_bwd_prop_log_0(&_S232, _S230);
        float _S233 = - _S231;
        float _S234 = (_S228 + _S229.differential_0) / _S79;
        float _S235 = _S80 * - _S234;
        float _S236 = _S81 * _S234;
        DiffPair_float_0 _S237;
        (&_S237)->primal_0 = _S82;
        (&_S237)->differential_0 = 0.0f;
        DiffPair_float_0 _S238;
        (&_S238)->primal_0 = _S83;
        (&_S238)->differential_0 = 0.0f;
        s_bwd_prop_min_0(&_S237, &_S238, _S236);
        float _S239 = (_S232.differential_0 + _S233) / _S84;
        float _S240 = _S85 * - _S239;
        float _S241 = _S81 * _S239;
        DiffPair_float_0 _S242;
        (&_S242)->primal_0 = _S82;
        (&_S242)->differential_0 = 0.0f;
        DiffPair_float_0 _S243;
        (&_S243)->primal_0 = _S83;
        (&_S243)->differential_0 = 0.0f;
        s_bwd_prop_max_0(&_S242, &_S243, _S241);
        float _S244 = _S235 + _S240;
        float2  _S245 = make_float2 (_S237.differential_0 + _S242.differential_0 + _S244, _S238.differential_0 + _S243.differential_0 + _S244);
        DiffPair_vectorx3Cfloatx2C2x3E_0 _S246;
        (&_S246)->primal_0 = _S121;
        (&_S246)->differential_0 = _S65;
        s_bwd_prop_exp_1(&_S246, _S245);
        float2  _S247 = make_float2 (2.0f) * _S246.differential_0;
        float s_diff_scale_reg_T_1 = scale_regularization_weight_1 * _s_dOut_1[int(2)];
        DiffPair_float_0 _S248;
        (&_S248)->primal_0 = _S86;
        (&_S248)->differential_0 = 0.0f;
        DiffPair_float_0 _S249;
        (&_S249)->primal_0 = max_gauss_ratio_1;
        (&_S249)->differential_0 = 0.0f;
        s_bwd_prop_max_0(&_S248, &_S249, s_diff_scale_reg_T_1);
        float _S250 = _S248.differential_0 / _S87;
        float _S251 = _S88 * - _S250;
        float _S252 = _S89 * _S250;
        DiffPair_float_0 _S253;
        (&_S253)->primal_0 = _S90;
        (&_S253)->differential_0 = 0.0f;
        DiffPair_float_0 _S254;
        (&_S254)->primal_0 = _S91;
        (&_S254)->differential_0 = 0.0f;
        s_bwd_prop_min_0(&_S253, &_S254, _S251);
        DiffPair_float_0 _S255;
        (&_S255)->primal_0 = _S90;
        (&_S255)->differential_0 = 0.0f;
        DiffPair_float_0 _S256;
        (&_S256)->primal_0 = _S91;
        (&_S256)->differential_0 = 0.0f;
        s_bwd_prop_max_0(&_S255, &_S256, _S252);
        float _S257 = mcmc_scale_reg_weight_1 * (0.5f * _s_dOut_1[int(1)]);
        float2  _S258 = make_float2 (_S253.differential_0 + _S255.differential_0 + _S257, _S254.differential_0 + _S256.differential_0 + _S257);
        DiffPair_vectorx3Cfloatx2C2x3E_0 _S259;
        (&_S259)->primal_0 = _S122;
        (&_S259)->differential_0 = _S65;
        s_bwd_prop_exp_1(&_S259, _S258);
        float2  _S260 = _S247 + _S259.differential_0;
        float3  _S261 = make_float3 (_S260.x, _S260.y, 0.0f);
        _S71 = _s_dOut_1[int(4)];
        _S72 = _s_dOut_1[int(0)];
        _S123 = _S261;
    }
    float s_diff_quat_norm_reg_T_0 = quat_norm_reg_weight_1 * _S71;
    float _S262 = - s_diff_quat_norm_reg_T_0;
    DiffPair_float_0 _S263;
    (&_S263)->primal_0 = _S70;
    (&_S263)->differential_0 = 0.0f;
    s_bwd_prop_log_0(&_S263, _S262);
    float _S264 = _S263.differential_0 + s_diff_quat_norm_reg_T_0;
    float4  _S265 = make_float4 (0.0f);
    DiffPair_vectorx3Cfloatx2C4x3E_0 _S266;
    (&_S266)->primal_0 = _S64.primal_0;
    (&_S266)->differential_0 = _S265;
    s_bwd_length_impl_0(&_S266, _S264);
    float _S267 = - (mcmc_opacity_reg_weight_1 * _S72 / _S69);
    DiffPair_float_0 _S268;
    (&_S268)->primal_0 = _S67;
    (&_S268)->differential_0 = 0.0f;
    s_bwd_prop_exp_0(&_S268, _S267);
    float _S269 = - _S268.differential_0;
    dpquat_0->primal_0 = (*dpquat_0).primal_0;
    dpquat_0->differential_0 = _S266.differential_0;
    dpopacity_0->primal_0 = (*dpopacity_0).primal_0;
    dpopacity_0->differential_0 = _S269;
    dpscales_0->primal_0 = (*dpscales_0).primal_0;
    dpscales_0->differential_0 = _S123;
    return;
}

__device__ void s_bwd_per_splat_losses_0(bool _S270, DiffPair_vectorx3Cfloatx2C3x3E_0 * _S271, DiffPair_float_0 * _S272, DiffPair_vectorx3Cfloatx2C4x3E_0 * _S273, float _S274, float _S275, float _S276, float _S277, float _S278, float _S279, float _S280, FixedArray<float, 5>  _S281)
{
    s_bwd_prop_per_splat_losses_0(_S270, _S271, _S272, _S273, _S274, _S275, _S276, _S277, _S278, _S279, _S280, _S281);
    return;
}

__device__ void per_splat_losses_bwd(bool is_3dgs_2, float3  scales_1, float opacity_1, float4  quat_1, FixedArray<float, 5>  v_loss_0, float3  * v_scales_0, float * v_opacity_0, float4  * v_quat_0, float mcmc_opacity_reg_weight_2, float mcmc_scale_reg_weight_2, float max_gauss_ratio_2, float scale_regularization_weight_2, float erank_reg_weight_2, float erank_reg_weight_s3_2, float quat_norm_reg_weight_2)
{
    float3  _S282 = make_float3 (0.0f);
    DiffPair_vectorx3Cfloatx2C3x3E_0 p_scales_0;
    (&p_scales_0)->primal_0 = scales_1;
    (&p_scales_0)->differential_0 = _S282;
    DiffPair_float_0 p_opacity_0;
    (&p_opacity_0)->primal_0 = opacity_1;
    (&p_opacity_0)->differential_0 = 0.0f;
    float4  _S283 = make_float4 (0.0f);
    DiffPair_vectorx3Cfloatx2C4x3E_0 p_quat_0;
    (&p_quat_0)->primal_0 = quat_1;
    (&p_quat_0)->differential_0 = _S283;
    s_bwd_per_splat_losses_0(is_3dgs_2, &p_scales_0, &p_opacity_0, &p_quat_0, mcmc_opacity_reg_weight_2, mcmc_scale_reg_weight_2, max_gauss_ratio_2, scale_regularization_weight_2, erank_reg_weight_2, erank_reg_weight_s3_2, quat_norm_reg_weight_2, v_loss_0);
    *v_scales_0 = p_scales_0.differential_0;
    *v_opacity_0 = p_opacity_0.differential_0;
    *v_quat_0 = p_quat_0.differential_0;
    return;
}

__device__ float3  min_0(float3  x_8, float3  y_5)
{
    float3  result_4;
    int i_3 = int(0);
    for(;;)
    {
        if(i_3 < int(3))
        {
        }
        else
        {
            break;
        }
        *_slang_vector_get_element_ptr(&result_4, i_3) = (F32_min((_slang_vector_get_element(x_8, i_3)), (_slang_vector_get_element(y_5, i_3))));
        i_3 = i_3 + int(1);
    }
    return result_4;
}

__device__ float3  max_0(float3  x_9, float3  y_6)
{
    float3  result_5;
    int i_4 = int(0);
    for(;;)
    {
        if(i_4 < int(3))
        {
        }
        else
        {
            break;
        }
        *_slang_vector_get_element_ptr(&result_5, i_4) = (F32_max((_slang_vector_get_element(x_9, i_4)), (_slang_vector_get_element(y_6, i_4))));
        i_4 = i_4 + int(1);
    }
    return result_5;
}

__device__ void _d_clamp_0(DiffPair_float_0 * dpx_8, DiffPair_float_0 * dpMin_0, DiffPair_float_0 * dpMax_0, float dOut_7)
{
    DiffPair_float_0 _S284 = *dpx_8;
    bool _S285;
    if(((*dpx_8).primal_0) >= ((*dpMin_0).primal_0))
    {
        _S285 = ((*dpx_8).primal_0) <= ((*dpMax_0).primal_0);
    }
    else
    {
        _S285 = false;
    }
    float _S286;
    if(_S285)
    {
        _S286 = dOut_7;
    }
    else
    {
        _S286 = 0.0f;
    }
    dpx_8->primal_0 = _S284.primal_0;
    dpx_8->differential_0 = _S286;
    DiffPair_float_0 _S287 = *dpMin_0;
    if((_S284.primal_0) < ((*dpMin_0).primal_0))
    {
        _S286 = dOut_7;
    }
    else
    {
        _S286 = 0.0f;
    }
    dpMin_0->primal_0 = _S287.primal_0;
    dpMin_0->differential_0 = _S286;
    DiffPair_float_0 _S288 = *dpMax_0;
    if(((*dpx_8).primal_0) > ((*dpMax_0).primal_0))
    {
        _S286 = dOut_7;
    }
    else
    {
        _S286 = 0.0f;
    }
    dpMax_0->primal_0 = _S288.primal_0;
    dpMax_0->differential_0 = _S286;
    return;
}

__device__ void _d_clamp_vector_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dpx_9, DiffPair_vectorx3Cfloatx2C3x3E_0 * dpy_2, DiffPair_vectorx3Cfloatx2C3x3E_0 * dpz_0, float3  dOut_8)
{
    DiffPair_float_0 left_dp_0;
    (&left_dp_0)->primal_0 = (*dpx_9).primal_0.x;
    (&left_dp_0)->differential_0 = 0.0f;
    DiffPair_float_0 middle_dp_0;
    (&middle_dp_0)->primal_0 = (*dpy_2).primal_0.x;
    (&middle_dp_0)->differential_0 = 0.0f;
    DiffPair_float_0 right_dp_0;
    (&right_dp_0)->primal_0 = (*dpz_0).primal_0.x;
    (&right_dp_0)->differential_0 = 0.0f;
    _d_clamp_0(&left_dp_0, &middle_dp_0, &right_dp_0, dOut_8.x);
    float3  left_d_result_0;
    *&((&left_d_result_0)->x) = left_dp_0.differential_0;
    float3  middle_d_result_0;
    *&((&middle_d_result_0)->x) = middle_dp_0.differential_0;
    float3  right_d_result_0;
    *&((&right_d_result_0)->x) = right_dp_0.differential_0;
    DiffPair_float_0 left_dp_1;
    (&left_dp_1)->primal_0 = (*dpx_9).primal_0.y;
    (&left_dp_1)->differential_0 = 0.0f;
    DiffPair_float_0 middle_dp_1;
    (&middle_dp_1)->primal_0 = (*dpy_2).primal_0.y;
    (&middle_dp_1)->differential_0 = 0.0f;
    DiffPair_float_0 right_dp_1;
    (&right_dp_1)->primal_0 = (*dpz_0).primal_0.y;
    (&right_dp_1)->differential_0 = 0.0f;
    _d_clamp_0(&left_dp_1, &middle_dp_1, &right_dp_1, dOut_8.y);
    *&((&left_d_result_0)->y) = left_dp_1.differential_0;
    *&((&middle_d_result_0)->y) = middle_dp_1.differential_0;
    *&((&right_d_result_0)->y) = right_dp_1.differential_0;
    DiffPair_float_0 left_dp_2;
    (&left_dp_2)->primal_0 = (*dpx_9).primal_0.z;
    (&left_dp_2)->differential_0 = 0.0f;
    DiffPair_float_0 middle_dp_2;
    (&middle_dp_2)->primal_0 = (*dpy_2).primal_0.z;
    (&middle_dp_2)->differential_0 = 0.0f;
    DiffPair_float_0 right_dp_2;
    (&right_dp_2)->primal_0 = (*dpz_0).primal_0.z;
    (&right_dp_2)->differential_0 = 0.0f;
    _d_clamp_0(&left_dp_2, &middle_dp_2, &right_dp_2, dOut_8.z);
    *&((&left_d_result_0)->z) = left_dp_2.differential_0;
    *&((&middle_d_result_0)->z) = middle_dp_2.differential_0;
    *&((&right_d_result_0)->z) = right_dp_2.differential_0;
    dpx_9->primal_0 = (*dpx_9).primal_0;
    dpx_9->differential_0 = left_d_result_0;
    dpy_2->primal_0 = (*dpy_2).primal_0;
    dpy_2->differential_0 = middle_d_result_0;
    dpz_0->primal_0 = (*dpz_0).primal_0;
    dpz_0->differential_0 = right_d_result_0;
    return;
}

__device__ float3  clamp_0(float3  x_10, float3  minBound_0, float3  maxBound_0)
{
    return min_0(max_0(x_10, minBound_0), maxBound_0);
}

__device__ float3  blend_background(float3  rgb_0, float alpha_0, float3  background_0)
{
    return clamp_0(rgb_0 + make_float3 (1.0f - alpha_0) * background_0, make_float3 (0.0f), make_float3 (1.0f));
}

__device__ void s_bwd_prop_clamp_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * _S289, DiffPair_vectorx3Cfloatx2C3x3E_0 * _S290, DiffPair_vectorx3Cfloatx2C3x3E_0 * _S291, float3  _S292)
{
    _d_clamp_vector_0(_S289, _S290, _S291, _S292);
    return;
}

__device__ void s_bwd_prop_blend_background_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dprgb_0, DiffPair_float_0 * dpalpha_0, DiffPair_vectorx3Cfloatx2C3x3E_0 * dpbackground_0, float3  _s_dOut_2)
{
    float _S293 = 1.0f - (*dpalpha_0).primal_0;
    float3  _S294 = make_float3 (_S293);
    float3  _S295 = make_float3 (0.0f);
    float3  _S296 = make_float3 (1.0f);
    float3  _S297 = make_float3 (0.0f);
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S298;
    (&_S298)->primal_0 = (*dprgb_0).primal_0 + make_float3 (_S293) * (*dpbackground_0).primal_0;
    (&_S298)->differential_0 = _S297;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S299;
    (&_S299)->primal_0 = _S295;
    (&_S299)->differential_0 = _S297;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S300;
    (&_S300)->primal_0 = _S296;
    (&_S300)->differential_0 = _S297;
    s_bwd_prop_clamp_0(&_S298, &_S299, &_S300, _s_dOut_2);
    float3  _S301 = _S294 * _S298.differential_0;
    float3  _S302 = (*dpbackground_0).primal_0 * _S298.differential_0;
    float _S303 = - (_S302.x + _S302.y + _S302.z);
    dpbackground_0->primal_0 = (*dpbackground_0).primal_0;
    dpbackground_0->differential_0 = _S301;
    dpalpha_0->primal_0 = (*dpalpha_0).primal_0;
    dpalpha_0->differential_0 = _S303;
    dprgb_0->primal_0 = (*dprgb_0).primal_0;
    dprgb_0->differential_0 = _S298.differential_0;
    return;
}

__device__ void s_bwd_blend_background_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * _S304, DiffPair_float_0 * _S305, DiffPair_vectorx3Cfloatx2C3x3E_0 * _S306, float3  _S307)
{
    s_bwd_prop_blend_background_0(_S304, _S305, _S306, _S307);
    return;
}

__device__ void blend_background_bwd(float3  rgb_1, float alpha_1, float3  background_1, float3  v_out_rgb_0, float3  * v_rgb_0, float * v_alpha_0, float3  * v_background_0)
{
    float3  _S308 = make_float3 (0.0f);
    DiffPair_vectorx3Cfloatx2C3x3E_0 p_rgb_0;
    (&p_rgb_0)->primal_0 = rgb_1;
    (&p_rgb_0)->differential_0 = _S308;
    DiffPair_float_0 p_alpha_0;
    (&p_alpha_0)->primal_0 = alpha_1;
    (&p_alpha_0)->differential_0 = 0.0f;
    DiffPair_vectorx3Cfloatx2C3x3E_0 p_background_0;
    (&p_background_0)->primal_0 = background_1;
    (&p_background_0)->differential_0 = _S308;
    s_bwd_blend_background_0(&p_rgb_0, &p_alpha_0, &p_background_0, v_out_rgb_0);
    *v_rgb_0 = p_rgb_0.differential_0;
    *v_alpha_0 = p_alpha_0.differential_0;
    *v_background_0 = p_background_0.differential_0;
    return;
}

