#include "hip/hip_runtime.h"
#include "slang.cuh"

struct DiffPair_float_0
{
    float primal_0;
    float differential_0;
};

inline __device__ void _d_max_0(DiffPair_float_0 * dpx_0, DiffPair_float_0 * dpy_0, float dOut_0)
{
    DiffPair_float_0 _S1 = *dpx_0;
    float _S2;
    if(((*dpx_0).primal_0) > ((*dpy_0).primal_0))
    {
        _S2 = dOut_0;
    }
    else
    {
        if(((*dpx_0).primal_0) < ((*dpy_0).primal_0))
        {
            _S2 = 0.0f;
        }
        else
        {
            _S2 = 0.5f * dOut_0;
        }
    }
    dpx_0->primal_0 = _S1.primal_0;
    dpx_0->differential_0 = _S2;
    DiffPair_float_0 _S3 = *dpy_0;
    if(((*dpy_0).primal_0) > (_S1.primal_0))
    {
        _S2 = dOut_0;
    }
    else
    {
        if(((*dpy_0).primal_0) < ((*dpx_0).primal_0))
        {
            _S2 = 0.0f;
        }
        else
        {
            _S2 = 0.5f * dOut_0;
        }
    }
    dpy_0->primal_0 = _S3.primal_0;
    dpy_0->differential_0 = _S2;
    return;
}

inline __device__ Matrix<float, 3, 3>  transpose_0(Matrix<float, 3, 3>  x_0)
{
    Matrix<float, 3, 3>  result_0;
    int r_0 = int(0);
    for(;;)
    {
        if(r_0 < int(3))
        {
        }
        else
        {
            break;
        }
        int c_0 = int(0);
        for(;;)
        {
            if(c_0 < int(3))
            {
            }
            else
            {
                break;
            }
            *_slang_vector_get_element_ptr(((&result_0)->rows + (r_0)), c_0) = _slang_vector_get_element(x_0.rows[c_0], r_0);
            c_0 = c_0 + int(1);
        }
        r_0 = r_0 + int(1);
    }
    return result_0;
}

inline __device__ Matrix<float, 3, 3>  quat_to_rotmat(float4  quat_0)
{
    float x_1 = quat_0.y;
    float inv_norm_0 = (F32_rsqrt((x_1 * x_1 + quat_0.z * quat_0.z + quat_0.w * quat_0.w + quat_0.x * quat_0.x)));
    float x_2 = quat_0.y * inv_norm_0;
    float y_0 = quat_0.z * inv_norm_0;
    float z_0 = quat_0.w * inv_norm_0;
    float w_0 = quat_0.x * inv_norm_0;
    float x2_0 = x_2 * x_2;
    float y2_0 = y_0 * y_0;
    float z2_0 = z_0 * z_0;
    float xy_0 = x_2 * y_0;
    float xz_0 = x_2 * z_0;
    float yz_0 = y_0 * z_0;
    float wx_0 = w_0 * x_2;
    float wy_0 = w_0 * y_0;
    float wz_0 = w_0 * z_0;
    return transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_0 + z2_0), 2.0f * (xy_0 + wz_0), 2.0f * (xz_0 - wy_0), 2.0f * (xy_0 - wz_0), 1.0f - 2.0f * (x2_0 + z2_0), 2.0f * (yz_0 + wx_0), 2.0f * (xz_0 + wy_0), 2.0f * (yz_0 - wx_0), 1.0f - 2.0f * (x2_0 + y2_0)));
}

struct DiffPair_matrixx3Cfloatx2C3x2C3x3E_0
{
    Matrix<float, 3, 3>  primal_0;
    Matrix<float, 3, 3>  differential_0;
};

struct DiffPair_vectorx3Cfloatx2C3x3E_0
{
    float3  primal_0;
    float3  differential_0;
};

inline __device__ void _d_mul_0(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * left_0, DiffPair_vectorx3Cfloatx2C3x3E_0 * right_0, float3  dOut_1)
{
    float _S4 = (*left_0).primal_0.rows[int(0)].x * dOut_1.x;
    Matrix<float, 3, 3>  left_d_result_0;
    *&(((&left_d_result_0)->rows + (int(0)))->x) = (*right_0).primal_0.x * dOut_1.x;
    float sum_0 = _S4 + (*left_0).primal_0.rows[int(1)].x * dOut_1.y;
    *&(((&left_d_result_0)->rows + (int(1)))->x) = (*right_0).primal_0.x * dOut_1.y;
    float sum_1 = sum_0 + (*left_0).primal_0.rows[int(2)].x * dOut_1.z;
    *&(((&left_d_result_0)->rows + (int(2)))->x) = (*right_0).primal_0.x * dOut_1.z;
    float3  right_d_result_0;
    *&((&right_d_result_0)->x) = sum_1;
    float _S5 = (*left_0).primal_0.rows[int(0)].y * dOut_1.x;
    *&(((&left_d_result_0)->rows + (int(0)))->y) = (*right_0).primal_0.y * dOut_1.x;
    float sum_2 = _S5 + (*left_0).primal_0.rows[int(1)].y * dOut_1.y;
    *&(((&left_d_result_0)->rows + (int(1)))->y) = (*right_0).primal_0.y * dOut_1.y;
    float sum_3 = sum_2 + (*left_0).primal_0.rows[int(2)].y * dOut_1.z;
    *&(((&left_d_result_0)->rows + (int(2)))->y) = (*right_0).primal_0.y * dOut_1.z;
    *&((&right_d_result_0)->y) = sum_3;
    float _S6 = (*left_0).primal_0.rows[int(0)].z * dOut_1.x;
    *&(((&left_d_result_0)->rows + (int(0)))->z) = (*right_0).primal_0.z * dOut_1.x;
    float sum_4 = _S6 + (*left_0).primal_0.rows[int(1)].z * dOut_1.y;
    *&(((&left_d_result_0)->rows + (int(1)))->z) = (*right_0).primal_0.z * dOut_1.y;
    float sum_5 = sum_4 + (*left_0).primal_0.rows[int(2)].z * dOut_1.z;
    *&(((&left_d_result_0)->rows + (int(2)))->z) = (*right_0).primal_0.z * dOut_1.z;
    *&((&right_d_result_0)->z) = sum_5;
    left_0->primal_0 = (*left_0).primal_0;
    left_0->differential_0 = left_d_result_0;
    right_0->primal_0 = (*right_0).primal_0;
    right_0->differential_0 = right_d_result_0;
    return;
}

inline __device__ float3  mul_0(Matrix<float, 3, 3>  left_1, float3  right_1)
{
    float3  result_1;
    int i_0 = int(0);
    for(;;)
    {
        if(i_0 < int(3))
        {
        }
        else
        {
            break;
        }
        int j_0 = int(0);
        float sum_6 = 0.0f;
        for(;;)
        {
            if(j_0 < int(3))
            {
            }
            else
            {
                break;
            }
            float sum_7 = sum_6 + _slang_vector_get_element(left_1.rows[i_0], j_0) * _slang_vector_get_element(right_1, j_0);
            j_0 = j_0 + int(1);
            sum_6 = sum_7;
        }
        *_slang_vector_get_element_ptr(&result_1, i_0) = sum_6;
        i_0 = i_0 + int(1);
    }
    return result_1;
}

inline __device__ void posW2C(Matrix<float, 3, 3>  R_0, float3  t_0, float3  pW_0, float3  * pC_0)
{
    *pC_0 = mul_0(R_0, pW_0) + t_0;
    return;
}

inline __device__ Matrix<float, 3, 3>  mul_1(Matrix<float, 3, 3>  left_2, Matrix<float, 3, 3>  right_2)
{
    Matrix<float, 3, 3>  result_2;
    int r_1 = int(0);
    for(;;)
    {
        if(r_1 < int(3))
        {
        }
        else
        {
            break;
        }
        int c_1 = int(0);
        for(;;)
        {
            if(c_1 < int(3))
            {
            }
            else
            {
                break;
            }
            int i_1 = int(0);
            float sum_8 = 0.0f;
            for(;;)
            {
                if(i_1 < int(3))
                {
                }
                else
                {
                    break;
                }
                float sum_9 = sum_8 + _slang_vector_get_element(left_2.rows[r_1], i_1) * _slang_vector_get_element(right_2.rows[i_1], c_1);
                i_1 = i_1 + int(1);
                sum_8 = sum_9;
            }
            *_slang_vector_get_element_ptr(((&result_2)->rows + (r_1)), c_1) = sum_8;
            c_1 = c_1 + int(1);
        }
        r_1 = r_1 + int(1);
    }
    return result_2;
}

inline __device__ void covarW2C(Matrix<float, 3, 3>  R_1, Matrix<float, 3, 3>  covarW_0, Matrix<float, 3, 3>  * covarC_0)
{
    *covarC_0 = mul_1(mul_1(R_1, covarW_0), transpose_0(R_1));
    return;
}

inline __device__ void quat_scale_to_covar(float4  quat_1, float3  scale_0, Matrix<float, 3, 3>  * covar_0)
{
    float x_3 = quat_1.y;
    float inv_norm_1 = (F32_rsqrt((x_3 * x_3 + quat_1.z * quat_1.z + quat_1.w * quat_1.w + quat_1.x * quat_1.x)));
    float x_4 = quat_1.y * inv_norm_1;
    float y_1 = quat_1.z * inv_norm_1;
    float z_1 = quat_1.w * inv_norm_1;
    float w_1 = quat_1.x * inv_norm_1;
    float x2_1 = x_4 * x_4;
    float y2_1 = y_1 * y_1;
    float z2_1 = z_1 * z_1;
    float xy_1 = x_4 * y_1;
    float xz_1 = x_4 * z_1;
    float yz_1 = y_1 * z_1;
    float wx_1 = w_1 * x_4;
    float wy_1 = w_1 * y_1;
    float wz_1 = w_1 * z_1;
    Matrix<float, 3, 3>  M_0 = mul_1(transpose_0(makeMatrix<float, 3, 3> (1.0f - 2.0f * (y2_1 + z2_1), 2.0f * (xy_1 + wz_1), 2.0f * (xz_1 - wy_1), 2.0f * (xy_1 - wz_1), 1.0f - 2.0f * (x2_1 + z2_1), 2.0f * (yz_1 + wx_1), 2.0f * (xz_1 + wy_1), 2.0f * (yz_1 - wx_1), 1.0f - 2.0f * (x2_1 + y2_1))), makeMatrix<float, 3, 3> (scale_0.x, 0.0f, 0.0f, 0.0f, scale_0.y, 0.0f, 0.0f, 0.0f, scale_0.z));
    *covar_0 = mul_1(M_0, transpose_0(M_0));
    return;
}

inline __device__ Matrix<float, 2, 2>  inverse(Matrix<float, 2, 2>  m_0)
{
    float invdet_0 = 1.0f / (m_0.rows[int(0)].x * m_0.rows[int(1)].y - m_0.rows[int(0)].y * m_0.rows[int(1)].x);
    return makeMatrix<float, 2, 2> (m_0.rows[int(1)].y * invdet_0, - m_0.rows[int(0)].y * invdet_0, - m_0.rows[int(1)].x * invdet_0, m_0.rows[int(0)].x * invdet_0);
}

inline __device__ void _d_min_0(DiffPair_float_0 * dpx_1, DiffPair_float_0 * dpy_1, float dOut_2)
{
    DiffPair_float_0 _S7 = *dpx_1;
    float _S8;
    if(((*dpx_1).primal_0) < ((*dpy_1).primal_0))
    {
        _S8 = dOut_2;
    }
    else
    {
        if(((*dpx_1).primal_0) > ((*dpy_1).primal_0))
        {
            _S8 = 0.0f;
        }
        else
        {
            _S8 = 0.5f * dOut_2;
        }
    }
    dpx_1->primal_0 = _S7.primal_0;
    dpx_1->differential_0 = _S8;
    DiffPair_float_0 _S9 = *dpy_1;
    if(((*dpy_1).primal_0) < (_S7.primal_0))
    {
        _S8 = dOut_2;
    }
    else
    {
        if(((*dpy_1).primal_0) > ((*dpx_1).primal_0))
        {
            _S8 = 0.0f;
        }
        else
        {
            _S8 = 0.5f * dOut_2;
        }
    }
    dpy_1->primal_0 = _S9.primal_0;
    dpy_1->differential_0 = _S8;
    return;
}

inline __device__ void projection_opaque_triangle_persp(float3  vert0_0, float3  vert1_0, float3  vert2_0, float hardness_0, Matrix<float, 3, 3>  R_2, float3  t_1, float fx_0, float fy_0, float cx_0, float cy_0, uint image_width_0, uint image_height_0, float near_plane_0, float far_plane_0, float radius_clip_0, int2  * radii_0, float * depth_0, float2  * uv0_0, float2  * uv1_0, float2  * uv2_0, float * out_hardness_0)
{
    for(;;)
    {
        float3  vert0_c_0 = mul_0(R_2, vert0_0) + t_1;
        float3  vert1_c_0 = mul_0(R_2, vert1_0) + t_1;
        float3  vert2_c_0 = mul_0(R_2, vert2_0) + t_1;
        float3  vert_mean_c_0 = mul_0(R_2, (vert0_0 + vert1_0 + vert2_0) / make_float3 (3.0f)) + t_1;
        float _S10 = vert0_c_0.z;
        bool _S11;
        if(_S10 < near_plane_0)
        {
            _S11 = true;
        }
        else
        {
            _S11 = _S10 > far_plane_0;
        }
        if(_S11)
        {
            _S11 = true;
        }
        else
        {
            _S11 = (vert1_c_0.z) < near_plane_0;
        }
        if(_S11)
        {
            _S11 = true;
        }
        else
        {
            _S11 = (vert1_c_0.z) > far_plane_0;
        }
        if(_S11)
        {
            _S11 = true;
        }
        else
        {
            _S11 = (vert2_c_0.z) < near_plane_0;
        }
        if(_S11)
        {
            _S11 = true;
        }
        else
        {
            _S11 = (vert2_c_0.z) > far_plane_0;
        }
        if(_S11)
        {
            _S11 = true;
        }
        else
        {
            _S11 = (vert_mean_c_0.z) < near_plane_0;
        }
        if(_S11)
        {
            _S11 = true;
        }
        else
        {
            _S11 = (vert_mean_c_0.z) > far_plane_0;
        }
        if(_S11)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        *uv0_0 = float2 {vert0_c_0.x, vert0_c_0.y} / make_float2 (_S10);
        *uv1_0 = float2 {vert1_c_0.x, vert1_c_0.y} / make_float2 (vert1_c_0.z);
        *uv2_0 = float2 {vert2_c_0.x, vert2_c_0.y} / make_float2 (vert2_c_0.z);
        float2  _S12 = make_float2 (fx_0, fy_0);
        float2  _S13 = make_float2 (cx_0, cy_0);
        *uv0_0 = _S12 * *uv0_0 + _S13;
        *uv1_0 = _S12 * *uv1_0 + _S13;
        float2  _S14 = _S12 * *uv2_0 + _S13;
        *uv2_0 = _S14;
        float _S15 = _S14.x;
        float _S16 = float(image_width_0);
        float x_max_0 = (F32_min(((F32_max(((F32_max(((*uv0_0).x), ((*uv1_0).x)))), (_S15))) + 0.5f), (_S16)));
        float x_min_0 = (F32_max(((F32_min(((F32_min(((*uv0_0).x), ((*uv1_0).x)))), (_S15))) - 0.5f), (0.0f)));
        float _S17 = _S14.y;
        float _S18 = float(image_height_0);
        float y_max_0 = (F32_min(((F32_max(((F32_max(((*uv0_0).y), ((*uv1_0).y)))), (_S17))) + 0.5f), (_S18)));
        float y_min_0 = (F32_max(((F32_min(((F32_min(((*uv0_0).y), ((*uv1_0).y)))), (_S17))) - 0.5f), (0.0f)));
        float radius_x_0 = (F32_ceil((x_max_0 - x_min_0)));
        float radius_y_0 = (F32_ceil((y_max_0 - y_min_0)));
        if(radius_x_0 <= radius_clip_0)
        {
            _S11 = radius_y_0 <= radius_clip_0;
        }
        else
        {
            _S11 = false;
        }
        if(_S11)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        if(x_max_0 <= 0.0f)
        {
            _S11 = true;
        }
        else
        {
            _S11 = x_min_0 >= _S16;
        }
        if(_S11)
        {
            _S11 = true;
        }
        else
        {
            _S11 = y_max_0 <= 0.0f;
        }
        if(_S11)
        {
            _S11 = true;
        }
        else
        {
            _S11 = y_min_0 >= _S18;
        }
        if(_S11)
        {
            *radii_0 = make_int2 (int(0), int(0));
            break;
        }
        *radii_0 = make_int2 (int(radius_x_0), int(radius_y_0));
        *depth_0 = vert_mean_c_0.z;
        *out_hardness_0 = hardness_0;
        break;
    }
    return;
}

inline __device__ void _projection_opaque_triangle_persp_differentiable(float3  vert0_1, float3  vert1_1, float3  vert2_1, float hardness_1, Matrix<float, 3, 3>  R_3, float3  t_2, float fx_1, float fy_1, float cx_1, float cy_1, uint image_width_1, uint image_height_1, float near_plane_1, float far_plane_1, float radius_clip_1, int2  * radii_1, float * depth_1, float2  * uv0_1, float2  * uv1_1, float2  * uv2_1, float * out_hardness_1)
{
    float3  vert0_c_1 = mul_0(R_3, vert0_1) + t_2;
    float3  vert1_c_1 = mul_0(R_3, vert1_1) + t_2;
    float3  vert2_c_1 = mul_0(R_3, vert2_1) + t_2;
    float3  vert_mean_c_1 = mul_0(R_3, (vert0_1 + vert1_1 + vert2_1) / make_float3 (3.0f)) + t_2;
    *uv0_1 = float2 {vert0_c_1.x, vert0_c_1.y} / make_float2 (vert0_c_1.z);
    *uv1_1 = float2 {vert1_c_1.x, vert1_c_1.y} / make_float2 (vert1_c_1.z);
    *uv2_1 = float2 {vert2_c_1.x, vert2_c_1.y} / make_float2 (vert2_c_1.z);
    float2  _S19 = make_float2 (fx_1, fy_1);
    float2  _S20 = make_float2 (cx_1, cy_1);
    *uv0_1 = _S19 * *uv0_1 + _S20;
    *uv1_1 = _S19 * *uv1_1 + _S20;
    float2  _S21 = _S19 * *uv2_1 + _S20;
    *uv2_1 = _S21;
    float _S22 = _S21.x;
    float _S23 = _S21.y;
    *radii_1 = make_int2 (int((F32_ceil(((F32_min(((F32_max(((F32_max(((*uv0_1).x), ((*uv1_1).x)))), (_S22))) + 0.5f), (float(image_width_1)))) - (F32_max(((F32_min(((F32_min(((*uv0_1).x), ((*uv1_1).x)))), (_S22))) - 0.5f), (0.0f))))))), int((F32_ceil(((F32_min(((F32_max(((F32_max(((*uv0_1).y), ((*uv1_1).y)))), (_S23))) + 0.5f), (float(image_height_1)))) - (F32_max(((F32_min(((F32_min(((*uv0_1).y), ((*uv1_1).y)))), (_S23))) - 0.5f), (0.0f))))))));
    *depth_1 = vert_mean_c_1.z;
    *out_hardness_1 = hardness_1;
    return;
}

inline __device__ float3  s_primal_ctx_mul_0(Matrix<float, 3, 3>  _S24, float3  _S25)
{
    return mul_0(_S24, _S25);
}

inline __device__ float s_primal_ctx_max_0(float _S26, float _S27)
{
    return (F32_max((_S26), (_S27)));
}

inline __device__ float s_primal_ctx_min_0(float _S28, float _S29)
{
    return (F32_min((_S28), (_S29)));
}

inline __device__ void s_bwd_prop_max_0(DiffPair_float_0 * _S30, DiffPair_float_0 * _S31, float _S32)
{
    _d_max_0(_S30, _S31, _S32);
    return;
}

inline __device__ void s_bwd_prop_min_0(DiffPair_float_0 * _S33, DiffPair_float_0 * _S34, float _S35)
{
    _d_min_0(_S33, _S34, _S35);
    return;
}

inline __device__ void s_bwd_prop_mul_0(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S36, DiffPair_vectorx3Cfloatx2C3x3E_0 * _S37, float3  _S38)
{
    _d_mul_0(_S36, _S37, _S38);
    return;
}

inline __device__ void projection_opaque_triangle_persp_vjp(float3  vert0_2, float3  vert1_2, float3  vert2_2, float hardness_2, Matrix<float, 3, 3>  R_4, float3  t_3, float fx_2, float fy_2, float cx_2, float cy_2, uint image_width_2, uint image_height_2, float v_depth_0, float2  v_uv0_0, float2  v_uv1_0, float2  v_uv2_0, float v_out_hardness_0, float3  * v_vert0_0, float3  * v_vert1_0, float3  * v_vert2_0, float * v_hardness_0, Matrix<float, 3, 3>  * v_R_0, float3  * v_t_0)
{
    float3  vert0_c_2 = s_primal_ctx_mul_0(R_4, vert0_2) + t_3;
    float3  vert_mean_0 = (vert0_2 + vert0_2 + vert0_2) / make_float3 (3.0f);
    float2  _S39 = float2 {vert0_c_2.x, vert0_c_2.y};
    float _S40 = vert0_c_2.z;
    float2  _S41 = make_float2 (_S40);
    float2  _S42 = make_float2 (_S40 * _S40);
    float2  _S43 = make_float2 (fx_2, fy_2);
    float2  _S44 = make_float2 (cx_2, cy_2);
    float2  _S45 = _S43 * (_S39 / make_float2 (_S40)) + _S44;
    float2  _S46 = _S43 * (_S39 / make_float2 (_S40)) + _S44;
    float2  _S47 = _S43 * (_S39 / make_float2 (_S40)) + _S44;
    float _S48 = _S45.x;
    float _S49 = _S46.x;
    float _S50 = s_primal_ctx_max_0(_S48, _S49);
    float _S51 = _S47.x;
    float _S52 = s_primal_ctx_max_0(_S50, _S51) + 0.5f;
    float _S53 = float(image_width_2);
    float _S54 = s_primal_ctx_min_0(_S48, _S49);
    float _S55 = s_primal_ctx_min_0(_S54, _S51) - 0.5f;
    float _S56 = _S45.y;
    float _S57 = _S46.y;
    float _S58 = s_primal_ctx_max_0(_S56, _S57);
    float _S59 = _S47.y;
    float _S60 = s_primal_ctx_max_0(_S58, _S59) + 0.5f;
    float _S61 = float(image_height_2);
    float _S62 = s_primal_ctx_min_0(_S56, _S57);
    DiffPair_float_0 _S63;
    (&_S63)->primal_0 = s_primal_ctx_min_0(_S62, _S59) - 0.5f;
    (&_S63)->differential_0 = 0.0f;
    DiffPair_float_0 _S64;
    (&_S64)->primal_0 = 0.0f;
    (&_S64)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S63, &_S64, -0.0f);
    DiffPair_float_0 _S65;
    (&_S65)->primal_0 = _S62;
    (&_S65)->differential_0 = 0.0f;
    DiffPair_float_0 _S66;
    (&_S66)->primal_0 = _S59;
    (&_S66)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S65, &_S66, _S63.differential_0);
    DiffPair_float_0 _S67;
    (&_S67)->primal_0 = _S56;
    (&_S67)->differential_0 = 0.0f;
    DiffPair_float_0 _S68;
    (&_S68)->primal_0 = _S57;
    (&_S68)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S67, &_S68, _S65.differential_0);
    DiffPair_float_0 _S69;
    (&_S69)->primal_0 = _S60;
    (&_S69)->differential_0 = 0.0f;
    DiffPair_float_0 _S70;
    (&_S70)->primal_0 = _S61;
    (&_S70)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S69, &_S70, 0.0f);
    DiffPair_float_0 _S71;
    (&_S71)->primal_0 = _S58;
    (&_S71)->differential_0 = 0.0f;
    DiffPair_float_0 _S72;
    (&_S72)->primal_0 = _S59;
    (&_S72)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S71, &_S72, _S69.differential_0);
    float _S73 = _S66.differential_0 + _S72.differential_0;
    DiffPair_float_0 _S74;
    (&_S74)->primal_0 = _S56;
    (&_S74)->differential_0 = 0.0f;
    DiffPair_float_0 _S75;
    (&_S75)->primal_0 = _S57;
    (&_S75)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S74, &_S75, _S71.differential_0);
    float _S76 = _S68.differential_0 + _S75.differential_0;
    float _S77 = _S67.differential_0 + _S74.differential_0;
    DiffPair_float_0 _S78;
    (&_S78)->primal_0 = _S55;
    (&_S78)->differential_0 = 0.0f;
    DiffPair_float_0 _S79;
    (&_S79)->primal_0 = 0.0f;
    (&_S79)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S78, &_S79, -0.0f);
    DiffPair_float_0 _S80;
    (&_S80)->primal_0 = _S54;
    (&_S80)->differential_0 = 0.0f;
    DiffPair_float_0 _S81;
    (&_S81)->primal_0 = _S51;
    (&_S81)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S80, &_S81, _S78.differential_0);
    DiffPair_float_0 _S82;
    (&_S82)->primal_0 = _S48;
    (&_S82)->differential_0 = 0.0f;
    DiffPair_float_0 _S83;
    (&_S83)->primal_0 = _S49;
    (&_S83)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S82, &_S83, _S80.differential_0);
    DiffPair_float_0 _S84;
    (&_S84)->primal_0 = _S52;
    (&_S84)->differential_0 = 0.0f;
    DiffPair_float_0 _S85;
    (&_S85)->primal_0 = _S53;
    (&_S85)->differential_0 = 0.0f;
    s_bwd_prop_min_0(&_S84, &_S85, 0.0f);
    DiffPair_float_0 _S86;
    (&_S86)->primal_0 = _S50;
    (&_S86)->differential_0 = 0.0f;
    DiffPair_float_0 _S87;
    (&_S87)->primal_0 = _S51;
    (&_S87)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S86, &_S87, _S84.differential_0);
    float _S88 = _S81.differential_0 + _S87.differential_0;
    DiffPair_float_0 _S89;
    (&_S89)->primal_0 = _S48;
    (&_S89)->differential_0 = 0.0f;
    DiffPair_float_0 _S90;
    (&_S90)->primal_0 = _S49;
    (&_S90)->differential_0 = 0.0f;
    s_bwd_prop_max_0(&_S89, &_S90, _S86.differential_0);
    float2  _S91 = _S43 * (v_uv2_0 + make_float2 (_S88, _S73)) / _S42;
    float2  _S92 = _S39 * - _S91;
    float2  _S93 = _S41 * _S91;
    float _S94 = _S92.x + _S92.y;
    float2  _S95 = _S43 * (v_uv1_0 + make_float2 (_S83.differential_0 + _S90.differential_0, _S76)) / _S42;
    float2  _S96 = _S39 * - _S95;
    float2  _S97 = _S41 * _S95;
    float _S98 = _S96.x + _S96.y;
    float2  _S99 = _S43 * (v_uv0_0 + make_float2 (_S82.differential_0 + _S89.differential_0, _S77)) / _S42;
    float2  _S100 = _S39 * - _S99;
    float2  _S101 = _S41 * _S99;
    float _S102 = _S100.x + _S100.y;
    float3  s_diff_vert_mean_c_T_0 = make_float3 (0.0f, 0.0f, v_depth_0);
    Matrix<float, 3, 3>  _S103 = makeMatrix<float, 3, 3> (0.0f);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S104;
    (&_S104)->primal_0 = R_4;
    (&_S104)->differential_0 = _S103;
    float3  _S105 = make_float3 (0.0f);
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S106;
    (&_S106)->primal_0 = vert_mean_0;
    (&_S106)->differential_0 = _S105;
    s_bwd_prop_mul_0(&_S104, &_S106, s_diff_vert_mean_c_T_0);
    float3  _S107 = make_float3 (0.3333333432674408f) * _S106.differential_0;
    float3  s_diff_vert2_c_T_0 = make_float3 (_S93.x, _S93.y, _S94);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S108;
    (&_S108)->primal_0 = R_4;
    (&_S108)->differential_0 = _S103;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S109;
    (&_S109)->primal_0 = vert0_2;
    (&_S109)->differential_0 = _S105;
    s_bwd_prop_mul_0(&_S108, &_S109, s_diff_vert2_c_T_0);
    float3  s_diff_vert1_c_T_0 = make_float3 (_S97.x, _S97.y, _S98);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S110;
    (&_S110)->primal_0 = R_4;
    (&_S110)->differential_0 = _S103;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S111;
    (&_S111)->primal_0 = vert0_2;
    (&_S111)->differential_0 = _S105;
    s_bwd_prop_mul_0(&_S110, &_S111, s_diff_vert1_c_T_0);
    float3  s_diff_vert0_c_T_0 = make_float3 (_S101.x, _S101.y, _S102);
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S112;
    (&_S112)->primal_0 = R_4;
    (&_S112)->differential_0 = _S103;
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S113;
    (&_S113)->primal_0 = vert0_2;
    (&_S113)->differential_0 = _S105;
    s_bwd_prop_mul_0(&_S112, &_S113, s_diff_vert0_c_T_0);
    float3  _S114 = s_diff_vert_mean_c_T_0 + s_diff_vert2_c_T_0 + s_diff_vert1_c_T_0 + s_diff_vert0_c_T_0;
    Matrix<float, 3, 3>  _S115 = _S104.differential_0 + _S108.differential_0 + _S110.differential_0 + _S112.differential_0;
    float3  _S116 = _S107 + _S109.differential_0;
    float3  _S117 = _S107 + _S111.differential_0;
    *v_vert0_0 = _S107 + _S113.differential_0;
    *v_vert1_0 = _S117;
    *v_vert2_0 = _S116;
    *v_hardness_0 = v_out_hardness_0;
    *v_R_0 = _S115;
    *v_t_0 = _S114;
    return;
}

