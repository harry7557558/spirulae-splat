#include "hip/hip_runtime.h"
// Modified from https://github.com/nerfstudio-project/gsplat/blob/main/gsplat/cuda/csrc/RasterizeToPixels3DGSFwd.cu

#include "common.cuh"

#include <gsplat/Common.h>
#include <gsplat/Utils.cuh>


template <uint32_t CDIM>
__global__ void rasterize_to_pixels_3dgs_fwd_kernel(
    const uint32_t I,
    const uint32_t N,
    const uint32_t n_isects,
    const bool packed,
    const glm::vec2 *__restrict__ means2d,         // [I, N, 2] or [nnz, 2]
    const glm::vec3 *__restrict__ conics,          // [I, N, 3] or [nnz, 3]
    const float *__restrict__ colors,      // [I, N, CDIM] or [nnz, CDIM]
    const float *__restrict__ opacities,   // [I, N] or [nnz]
    const float *__restrict__ backgrounds, // [I, CDIM]
    const bool *__restrict__ masks,           // [I, tile_height, tile_width]
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    const uint32_t tile_width,
    const uint32_t tile_height,
    const int32_t *__restrict__ tile_offsets, // [I, tile_height, tile_width]
    const int32_t *__restrict__ flatten_ids,  // [n_isects]
    float
        *__restrict__ render_colors, // [I, image_height, image_width, CDIM]
    float *__restrict__ render_Ts, // [I, image_height, image_width, 1]
    int32_t *__restrict__ last_ids        // [I, image_height, image_width]
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t image_id = block.group_index().x;
    int32_t tile_id =
        block.group_index().y * tile_width + block.group_index().z;
    uint32_t i = block.group_index().y * tile_size + block.thread_index().y;
    uint32_t j = block.group_index().z * tile_size + block.thread_index().x;

    tile_offsets += image_id * tile_height * tile_width;
    render_colors += image_id * image_height * image_width * CDIM;
    render_Ts += image_id * image_height * image_width;
    last_ids += image_id * image_height * image_width;
    if (backgrounds != nullptr) {
        backgrounds += image_id * CDIM;
    }
    if (masks != nullptr) {
        masks += image_id * tile_height * tile_width;
    }

    float px = (float)j + 0.5f;
    float py = (float)i + 0.5f;
    int32_t pix_id = i * image_width + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < image_height && j < image_width);
    bool done = !inside;

    // when the mask is provided, render the background color and return
    // if this tile is labeled as False
    if (masks != nullptr && inside && !masks[tile_id]) {
#pragma unroll
        for (uint32_t k = 0; k < CDIM; ++k) {
            render_colors[pix_id * CDIM + k] =
                backgrounds == nullptr ? 0.0f : backgrounds[k];
        }
        return;
    }

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (image_id == I - 1) && (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const uint32_t block_size = block.size();
    uint32_t num_batches =
        (range_end - range_start + block_size - 1) / block_size;

    extern __shared__ int s[];
    int32_t *id_batch = (int32_t *)s; // [block_size]
    glm::vec3 *xy_opacity_batch =
        reinterpret_cast<glm::vec3 *>(&id_batch[block_size]); // [block_size]
    glm::vec3 *conic_batch =
        reinterpret_cast<glm::vec3 *>(&xy_opacity_batch[block_size]); // [block_size]

    // current visibility left to render
    // transmittance is gonna be used in the backward pass which requires a high
    // numerical precision so we use double for it. However double make bwd 1.5x
    // slower so we stick with float for now.
    float T = 1.0f;
    // index of most recent gaussian to write to this thread's pixel
    uint32_t cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    uint32_t tr = block.thread_rank();

    float pix_out[CDIM] = {0.f};
    for (uint32_t b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        uint32_t batch_start = range_start + block_size * b;
        uint32_t idx = batch_start + tr;
        if (idx < range_end) {
            int32_t g = flatten_ids[idx]; // flatten index in [I * N] or [nnz]
            id_batch[tr] = g;
            const glm::vec2 xy = means2d[g];
            const float opac = opacities[g];
            xy_opacity_batch[tr] = {xy.x, xy.y, opac};
            conic_batch[tr] = conics[g];
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        uint32_t batch_size = min(block_size, range_end - batch_start);
        for (uint32_t t = 0; (t < batch_size) && !done; ++t) {
            const glm::vec3 conic = conic_batch[t];
            const glm::vec3 xy_opac = xy_opacity_batch[t];
            const float opac = xy_opac.z;
            const glm::vec2 delta = {xy_opac.x - px, xy_opac.y - py};
            const float sigma = 0.5f * (conic.x * delta.x * delta.x +
                                        conic.z * delta.y * delta.y) +
                                conic.y * delta.x * delta.y;
            float alpha = min(0.999f, opac * __expf(-sigma));
            if (sigma < 0.f || alpha < ALPHA_THRESHOLD) {
                continue;
            }

            const float next_T = T * (1.0f - alpha);
            if (next_T <= 1e-4f) { // this pixel is done: exclusive
                done = true;
                break;
            }

            int32_t g = id_batch[t];
            const float vis = alpha * T;
            const float *c_ptr = colors + g * CDIM;
#pragma unroll
            for (uint32_t k = 0; k < CDIM; ++k) {
                pix_out[k] += c_ptr[k] * vis;
            }
            cur_idx = batch_start + t;

            T = next_T;
        }
    }

    if (inside) {
        render_Ts[pix_id] = T;
#pragma unroll
        for (uint32_t k = 0; k < CDIM; ++k) {
            render_colors[pix_id * CDIM + k] =
                backgrounds == nullptr ? pix_out[k]
                                       : (pix_out[k] + T * backgrounds[k]);
        }
        // index in bin of last gaussian in this pixel
        last_ids[pix_id] = static_cast<int32_t>(cur_idx);
    }
}

template <uint32_t CDIM>
void launch_rasterize_to_pixels_3dgs_fwd_kernel(
    // Gaussian parameters
    const at::Tensor means2d,   // [..., N, 2] or [nnz, 2]
    const at::Tensor conics,    // [..., N, 3] or [nnz, 3]
    const at::Tensor colors,    // [..., N, channels] or [nnz, channels]
    const at::Tensor opacities, // [..., N]  or [nnz]
    const at::optional<at::Tensor> backgrounds, // [..., channels]
    const at::optional<at::Tensor> masks,       // [..., tile_height, tile_width]
    // image size
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    // intersections
    const at::Tensor tile_offsets, // [..., tile_height, tile_width]
    const at::Tensor flatten_ids,  // [n_isects]
    // outputs
    at::Tensor renders, // [..., image_height, image_width, channels]
    at::Tensor transmittances,  // [..., image_height, image_width]
    at::Tensor last_ids // [..., image_height, image_width]
) {
    bool packed = means2d.dim() == 2;

    uint32_t N = packed ? 0 : means2d.size(-2); // number of gaussians
    uint32_t I = transmittances.numel() / (image_height * image_width); // number of images
    uint32_t tile_height = tile_offsets.size(-2);
    uint32_t tile_width = tile_offsets.size(-1);
    uint32_t n_isects = flatten_ids.size(0);

    // Each block covers a tile on the image. In total there are
    // I * tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 grid = {I, tile_height, tile_width};

    int64_t shmem_size =
        tile_size * tile_size * (sizeof(int32_t) + sizeof(glm::vec3) + sizeof(glm::vec3));

    // TODO: an optimization can be done by passing the actual number of
    // channels into the kernel functions and avoid necessary global memory
    // writes. This requires moving the channel padding from python to C side.
    if (hipFuncSetAttribute(reinterpret_cast<const void*>(
            rasterize_to_pixels_3dgs_fwd_kernel<CDIM>),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            shmem_size
        ) != hipSuccess) {
        AT_ERROR(
            "Failed to set maximum shared memory size (requested ",
            shmem_size,
            " bytes), try lowering tile_size."
        );
    }

    rasterize_to_pixels_3dgs_fwd_kernel<CDIM>
        <<<grid, threads, shmem_size, at::cuda::getCurrentCUDAStream()>>>(
            I,
            N,
            n_isects,
            packed,
            reinterpret_cast<glm::vec2 *>(means2d.data_ptr<float>()),
            reinterpret_cast<glm::vec3 *>(conics.data_ptr<float>()),
            colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                    : nullptr,
            masks.has_value() ? masks.value().data_ptr<bool>() : nullptr,
            image_width,
            image_height,
            tile_size,
            tile_width,
            tile_height,
            tile_offsets.data_ptr<int32_t>(),
            flatten_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(),
            transmittances.data_ptr<float>(),
            last_ids.data_ptr<int32_t>()
        );
}

// Explicit Instantiation: this should match how it is being called in .cpp
// file.
// TODO: this is slow to compile, can we do something about it?
#define __INS__(CDIM)                                                          \
    template void launch_rasterize_to_pixels_3dgs_fwd_kernel<CDIM>(            \
        const at::Tensor means2d,                                              \
        const at::Tensor conics,                                               \
        const at::Tensor colors,                                               \
        const at::Tensor opacities,                                            \
        const at::optional<at::Tensor> backgrounds,                            \
        const at::optional<at::Tensor> masks,                                  \
        uint32_t image_width,                                                  \
        uint32_t image_height,                                                 \
        uint32_t tile_size,                                                    \
        const at::Tensor tile_offsets,                                         \
        const at::Tensor flatten_ids,                                          \
        at::Tensor renders,                                                    \
        at::Tensor transmittances,                                             \
        at::Tensor last_ids                                                    \
    );

__INS__(1)
__INS__(2)
__INS__(3)
__INS__(4)
__INS__(5)
// __INS__(8)
// __INS__(9)
// __INS__(16)
// __INS__(17)
// __INS__(32)
// __INS__(33)
// __INS__(64)
// __INS__(65)
// __INS__(128)
// __INS__(129)
// __INS__(256)
// __INS__(257)
// __INS__(512)
// __INS__(513)
#undef __INS__


std::tuple<at::Tensor, at::Tensor, at::Tensor> rasterize_to_pixels_3dgs_fwd(
    // Gaussian parameters
    const at::Tensor means2d,   // [..., N, 2] or [nnz, 2]
    const at::Tensor conics,    // [..., N, 3] or [nnz, 3]
    const at::Tensor colors,    // [..., N, channels] or [nnz, channels]
    const at::Tensor opacities, // [..., N]  or [nnz]
    const at::optional<at::Tensor> backgrounds, // [..., channels]
    const at::optional<at::Tensor> masks,       // [..., tile_height, tile_width]
    // image size
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    // intersections
    const at::Tensor tile_offsets, // [..., tile_height, tile_width]
    const at::Tensor flatten_ids   // [n_isects]
) {
    DEVICE_GUARD(means2d);
    CHECK_INPUT(means2d);
    CHECK_INPUT(conics);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(tile_offsets);
    CHECK_INPUT(flatten_ids);
    if (backgrounds.has_value()) {
        CHECK_INPUT(backgrounds.value());
    }
    if (masks.has_value()) {
        CHECK_INPUT(masks.value());
    }

    auto opt = means2d.options();
    at::DimVector image_dims(tile_offsets.sizes().slice(0, tile_offsets.dim() - 2));
    uint32_t channels = colors.size(-1);

    at::DimVector renders_dims(image_dims);
    renders_dims.append({image_height, image_width, channels});
    at::Tensor renders = at::empty(renders_dims, opt);

    at::DimVector transmittance_dims(image_dims);
    transmittance_dims.append({image_height, image_width, 1});
    at::Tensor transmittances = at::empty(transmittance_dims, opt);

    at::DimVector last_ids_dims(image_dims);
    last_ids_dims.append({image_height, image_width});
    at::Tensor last_ids = at::empty(last_ids_dims, opt.dtype(at::kInt));

#define __LAUNCH_KERNEL__(N)                                                   \
    case N:                                                                    \
        launch_rasterize_to_pixels_3dgs_fwd_kernel<N>(                         \
            means2d,                                                           \
            conics,                                                            \
            colors,                                                            \
            opacities,                                                         \
            backgrounds,                                                       \
            masks,                                                             \
            image_width,                                                       \
            image_height,                                                      \
            tile_size,                                                         \
            tile_offsets,                                                      \
            flatten_ids,                                                       \
            renders,                                                           \
            transmittances,                                                    \
            last_ids                                                           \
        );                                                                     \
        break;

    // TODO: an optimization can be done by passing the actual number of
    // channels into the kernel functions and avoid necessary global memory
    // writes. This requires moving the channel padding from python to C side.
    switch (channels) {
        __LAUNCH_KERNEL__(1)
        __LAUNCH_KERNEL__(2)
        __LAUNCH_KERNEL__(3)
        __LAUNCH_KERNEL__(4)
        __LAUNCH_KERNEL__(5)
        // __LAUNCH_KERNEL__(8)
        // __LAUNCH_KERNEL__(9)
        // __LAUNCH_KERNEL__(16)
        // __LAUNCH_KERNEL__(17)
        // __LAUNCH_KERNEL__(32)
        // __LAUNCH_KERNEL__(33)
        // __LAUNCH_KERNEL__(64)
        // __LAUNCH_KERNEL__(65)
        // __LAUNCH_KERNEL__(128)
        // __LAUNCH_KERNEL__(129)
        // __LAUNCH_KERNEL__(256)
        // __LAUNCH_KERNEL__(257)
        // __LAUNCH_KERNEL__(512)
        // __LAUNCH_KERNEL__(513)
    default:
        AT_ERROR("Unsupported number of channels: ", channels);
    }
#undef __LAUNCH_KERNEL__

    return std::make_tuple(renders, transmittances, last_ids);
}
