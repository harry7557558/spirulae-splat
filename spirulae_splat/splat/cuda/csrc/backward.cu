#include "hip/hip_runtime.h"
#include "backward.cuh"
#include "helpers.cuh"
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

template<typename vec3>
inline __device__ void warpSum3(vec3& val, cg::thread_block_tile<32>& tile){
    val.x = cg::reduce(tile, val.x, cg::plus<float>());
    val.y = cg::reduce(tile, val.y, cg::plus<float>());
    val.z = cg::reduce(tile, val.z, cg::plus<float>());
}

template<typename vec2>
inline __device__ void warpSum2(vec2& val, cg::thread_block_tile<32>& tile){
    val.x = cg::reduce(tile, val.x, cg::plus<float>());
    val.y = cg::reduce(tile, val.y, cg::plus<float>());
}

inline __device__ void warpSum(float& val, cg::thread_block_tile<32>& tile){
    val = cg::reduce(tile, val, cg::plus<float>());
}

__global__ void rasterize_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float3& __restrict__ background,
    const float2* __restrict__ depth_grads,
    const float2* __restrict__ depth_normal_ref_im,
    const int* __restrict__ final_index,
    const float* __restrict__ output_alpha,
    const float3* __restrict__ output_depth_grad,
    const float* __restrict__ v_output_alpha,
    const float3* __restrict__ v_output,
    const float3* __restrict__ v_output_depth_grad,
    const float* __restrict__ v_output_reg_depth,
    const float* __restrict__ v_output_reg_normal,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float* __restrict__ v_opacities,
    float2* __restrict__ v_depth_grad,
    float2* __restrict__ v_depth_normal_ref
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec4 color_opacity_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec2 depth_grad_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    const float3 out_depth_grad = output_depth_grad[pix_id];
    const float3 v_out = v_output[pix_id];
    const float3 v_out_depth_grad = v_output_depth_grad[pix_id];
    const float v_out_alpha = v_output_alpha[pix_id];
    const float v_out_reg_depth = v_output_reg_depth[pix_id];
    const float v_out_reg_normal = v_output_reg_normal[pix_id];
    const glm::vec2 v_g_sum = {v_out_depth_grad.x, v_out_depth_grad.y};
    const float v_depth_sum = v_out_depth_grad.z;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();

    // regularization
    const float2 depth_normal_ref = inside ?
        depth_normal_ref_im[pix_id] : make_float2(0.f, 0.f);
    glm::vec2 n_bar = {depth_normal_ref.x, depth_normal_ref.y};
    glm::vec2 v_n_bar = {0.f, 0.f};

    const float vis_sum_final = 1.0f - T_final;
    const float depth_sum_final = out_depth_grad.z;
    float vis_sum = vis_sum_final;
    float depth_sum = depth_sum_final;
    glm::vec2 g_sum = {out_depth_grad.x, out_depth_grad.y};

    float3 buffer = {0.f, 0.f, 0.f};
    float3 buffer_depth = {0.f, 0.f, 0.f};
    float buffer_depth_reg = 0.f;
    float buffer_normal_reg = 0.f;
    
    float v_sum_vis = v_out_alpha;

    glm::vec2 v_g_bar = {v_out_depth_grad.x, v_out_depth_grad.y};
    float v_depth_out = v_out_depth_grad.z;

    // second run through, full gradient calculation
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    T = T_final;
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            const float2 depth_grad = depth_grads[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_opacity_batch[tr] = {color.x, color.y, color.z, opac};
            depth_grad_batch[tr] = {depth_grad.x, depth_grad.y};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            float opac = color_opacity_batch[t].w;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            glm::vec3 v_color_local = {0.f, 0.f, 0.f};
            glm::vec2 v_depth_grad_local = {0.f, 0.f};
            float v_opacity_local = 0.f;
            //initialize everything to 0, only set if the lane is valid
            if(valid){
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                // update depth regularizer
                const glm::vec2 depth_grad = depth_grad_batch[t];
                const float depth = pos.z;
                float vis_sum_next = vis_sum - vis;
                float depth_sum_next = depth_sum - vis*depth;
                v_position_local.z += v_out_reg_depth * vis * (vis_sum_next - (vis_sum_final-vis_sum));
                float reg_depth_i = (
                    depth * vis_sum_next - depth_sum_next +
                    (depth_sum_final-depth_sum) - depth * (vis_sum_final-vis_sum)
                );

                // update normal regularizer
                glm::vec2 g_i = {depth_grad.x, depth_grad.y};
                float g_i_norm = glm::length(g_i) + 1e-6f;
                glm::vec2 n_i = g_i / g_i_norm;
                glm::mat2 J_i = (glm::mat2(1.0f) - glm::outerProduct(n_i, n_i)) / g_i_norm;
                float reg_normal_i = 1.0f - dot(n_i, n_bar);
                glm::vec2 v_normal_glm = v_out_reg_normal * (-vis) * J_i * n_bar;
                v_depth_grad_local.x += v_normal_glm.x;
                v_depth_grad_local.y += v_normal_glm.y;
                v_n_bar += vis * (-n_i) * v_out_reg_normal;

                // update v_rgb for this gaussian
                v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};
                v_depth_grad_local.x += vis * v_g_sum.x;
                v_depth_grad_local.y += vis * v_g_sum.y;
                v_position_local.z += vis * v_depth_sum;

                float v_alpha = 0.0f;
                const glm::vec4 rgba = color_opacity_batch[t];
                // contribution from this pixel
                v_alpha += (rgba.x * T - buffer.x) * ra * v_out.x;
                v_alpha += (rgba.y * T - buffer.y) * ra * v_out.y;
                v_alpha += (rgba.z * T - buffer.z) * ra * v_out.z;
                v_alpha += (depth_grad.x * T - buffer_depth.x) * ra * v_g_sum.x;
                v_alpha += (depth_grad.y * T - buffer_depth.y) * ra * v_g_sum.y;
                v_alpha += (pos.z * T - buffer_depth.z) * ra * v_depth_sum;
                v_alpha += (reg_depth_i * T - buffer_depth_reg) * ra * v_out_reg_depth;
                v_alpha += (reg_normal_i * T - buffer_normal_reg) * ra * v_out_reg_normal;

                v_alpha += T_final * ra * v_out_alpha;
                // contribution from background pixel
                v_alpha += -T_final * ra * background.x * v_out.x;
                v_alpha += -T_final * ra * background.y * v_out.y;
                v_alpha += -T_final * ra * background.z * v_out.z;
                // update the running sum
                buffer.x += rgba.x * vis;
                buffer.y += rgba.y * vis;
                buffer.z += rgba.z * vis;
                buffer_depth.x += depth_grad.x * vis;
                buffer_depth.y += depth_grad.y * vis;
                buffer_depth.z += pos.z * vis;
                buffer_depth_reg += reg_depth_i * vis;
                buffer_normal_reg += reg_normal_i * vis;

                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, rgba.w, v_alpha,
                    v_uv, v_opacity_local
                );
                glm::mat2x3 v_axis_uv;
                glm::vec3 v_position_local_temp;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    glm::vec3(0), v_uv,
                    v_position_local_temp, v_axis_uv
                );
                v_position_local += v_position_local_temp;
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                // v_position_xy_abs_local /= pos.z;
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];

                T = next_T;
                vis_sum = vis_sum_next;
                depth_sum = depth_sum_next;
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum3(v_color_local, warp);
            warpSum(v_opacity_local, warp);
            warpSum2(v_depth_grad_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                float* v_color_ptr = (float*)(v_colors);
                atomicAdd(v_color_ptr + 3*g + 0, v_color_local.x);
                atomicAdd(v_color_ptr + 3*g + 1, v_color_local.y);
                atomicAdd(v_color_ptr + 3*g + 2, v_color_local.z);
                atomicAdd(v_opacities + g, v_opacity_local);

                float* v_depth_grad_ptr = (float*)(v_depth_grad);
                atomicAdd(v_depth_grad_ptr + 2*g + 0, v_depth_grad_local.x);
                atomicAdd(v_depth_grad_ptr + 2*g + 1, v_depth_grad_local.y);
            }
        }
    }

    if (inside) {
        v_depth_normal_ref[pix_id] = {v_n_bar.x, v_n_bar.y};
    }

}

__global__ void rasterize_simple_backward_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float3& __restrict__ background,
    const int* __restrict__ final_index,
    const float* __restrict__ output_alpha,
    const float3* __restrict__ v_output,
    const float* __restrict__ v_output_alpha,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float* __restrict__ v_opacities
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * img_size.x + j, img_size.x * img_size.y - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < img_size.y && j < img_size.x);

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // the contribution from gaussians behind the current one
    float3 buffer = {0.f, 0.f, 0.f};
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside? final_index[pix_id] : 0;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    const int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    const int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec4 color_opacity_batch[MAX_BLOCK_SIZE];

    // df/d_out for this pixel
    const float3 v_out = v_output[pix_id];
    const float v_out_alpha = v_output_alpha[pix_id];

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range.y - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range.x);
        const int idx = batch_end - tr;
        if (idx >= range.x) {
            int32_t g_id = gaussian_ids_sorted[idx];
            id_batch[tr] = g_id;
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float3 color = colors[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            color_opacity_batch[tr] = {color.x, color.y, color.z, opac};
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0,batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }

            glm::vec3 pos = position_batch[t];
            float opac = color_opacity_batch[t].w;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            if(valid){
                get_intersection(pos, axis_uv, pos_2d, poi, uv);
                if (glm::length(uv) > visibility_kernel_radius())
                    valid = 0;
            }
            float alpha;
            if (valid){
                if (!get_alpha(uv, opac, alpha))
                    valid = 0;
            }
            if(!warp.any(valid)){
                continue;
            }

            glm::vec3 v_position_local = {0.f, 0.f, 0.f};
            glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
            glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
            glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
            glm::vec3 v_color_local = {0.f, 0.f, 0.f};
            float v_opacity_local = 0.f;
            //initialize everything to 0, only set if the lane is valid
            if(valid){
                // compute the current T for this gaussian
                const float ra = 1.f / (1.f - alpha);
                const float next_T = T * ra;
                const float vis = alpha * next_T;

                // update v_rgb for this gaussian
                float v_alpha = 0.f;
                v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};

                const glm::vec4 rgba = color_opacity_batch[t];
                // contribution from this pixel
                v_alpha += (rgba.x * T - buffer.x) * ra * v_out.x;
                v_alpha += (rgba.y * T - buffer.y) * ra * v_out.y;
                v_alpha += (rgba.z * T - buffer.z) * ra * v_out.z;

                v_alpha += T_final * ra * v_out_alpha;
                // contribution from background pixel
                v_alpha += -T_final * ra * background.x * v_out.x;
                v_alpha += -T_final * ra * background.y * v_out.y;
                v_alpha += -T_final * ra * background.z * v_out.z;
                // update the running sum
                buffer.x += rgba.x * vis;
                buffer.y += rgba.y * vis;
                buffer.z += rgba.z * vis;

                glm::vec2 v_uv;
                get_alpha_vjp(
                    uv, rgba.w, v_alpha,
                    v_uv, v_opacity_local
                );
                glm::mat2x3 v_axis_uv;
                get_intersection_vjp(
                    pos, axis_uv, pos_2d,
                    glm::vec3(0), v_uv,
                    v_position_local, v_axis_uv
                );
                v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
                // v_position_xy_abs_local /= pos.z;
                v_axis_u_local = v_axis_uv[0];
                v_axis_v_local = v_axis_uv[1];

                T = next_T;
            }
            warpSum3(v_position_local, warp);
            warpSum2(v_position_xy_abs_local, warp);
            warpSum3(v_axis_u_local, warp);
            warpSum3(v_axis_v_local, warp);
            warpSum3(v_color_local, warp);
            warpSum(v_opacity_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];

                float* v_position_ptr = (float*)(v_positions);
                atomicAdd(v_position_ptr + 3*g + 0, v_position_local.x);
                atomicAdd(v_position_ptr + 3*g + 1, v_position_local.y);
                atomicAdd(v_position_ptr + 3*g + 2, v_position_local.z);
                float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 0, v_position_xy_abs_local.x);
                atomicAdd(v_positions_xy_abs_ptr + 2*g + 1, v_position_xy_abs_local.y);

                float* v_axis_u_ptr = (float*)(v_axes_u);
                atomicAdd(v_axis_u_ptr + 3*g + 0, v_axis_u_local.x);
                atomicAdd(v_axis_u_ptr + 3*g + 1, v_axis_u_local.y);
                atomicAdd(v_axis_u_ptr + 3*g + 2, v_axis_u_local.z);
                float* v_axis_v_ptr = (float*)(v_axes_v);
                atomicAdd(v_axis_v_ptr + 3*g + 0, v_axis_v_local.x);
                atomicAdd(v_axis_v_ptr + 3*g + 1, v_axis_v_local.y);
                atomicAdd(v_axis_v_ptr + 3*g + 2, v_axis_v_local.z);
                
                float* v_color_ptr = (float*)(v_colors);
                atomicAdd(v_color_ptr + 3*g + 0, v_color_local.x);
                atomicAdd(v_color_ptr + 3*g + 1, v_color_local.y);
                atomicAdd(v_color_ptr + 3*g + 2, v_color_local.z);
                atomicAdd(v_opacities + g, v_opacity_local);
            }
        }
    }
}

__global__ void project_gaussians_backward_kernel(
    const int num_points,
    const float3* __restrict__ means3d,
    const float2* __restrict__ scales,
    const float4* __restrict__ quats,
    const float* __restrict__ viewmat,
    const float4 intrins,
    const int* __restrict__ num_tiles_hit,
    const float3* __restrict__ v_positions,
    const float3* __restrict__ v_axes_u,
    const float3* __restrict__ v_axes_v,
    const float2* __restrict__ v_depth_grads,
    float3* __restrict__ v_means3d,
    float2* __restrict__ v_scales,
    float4* __restrict__ v_quats
) {
    unsigned idx = cg::this_grid().thread_rank(); // idx of thread within grid
    if (idx >= num_points || num_tiles_hit <= 0) {
        return;
    }

    // position
    float3 p_world = means3d[idx];
    float fx = intrins.x;
    float fy = intrins.y;
    float3 p_view = transform_4x3(viewmat, p_world);
    float3 v_p_view = v_positions[idx];

    // forward
    float2 scale = scales[idx];
    float4 quat = quats[idx];
    glm::mat3 R1 = glm::transpose(glm::mat3(
        viewmat[0], viewmat[1], viewmat[2],
        viewmat[4], viewmat[5], viewmat[6],
        viewmat[8], viewmat[9], viewmat[10]
    ));
    glm::mat3 R2 = quat_to_rotmat(quat);
    glm::mat3 R = R1 * R2;
    glm::vec3 V0 = scale.x * R[0];
    glm::vec3 V1 = scale.y * R[1];

    // scale
    glm::vec3 v_V0 = *(glm::vec3*)&v_axes_u[idx];
    glm::vec3 v_V1 = *(glm::vec3*)&v_axes_v[idx];
    float2 v_scale = {glm::dot(R[0], v_V0), glm::dot(R[1], v_V1)};

    // orientation
    glm::mat3 v_R = glm::mat3(0.0f);
    v_R[0] = scale.x * v_V0;
    v_R[1] = scale.y * v_V1;

    // depth_grad
    glm::mat3 v_R_dg;
    float3 v_p_view_dg = {0.f, 0.f, 0.f};
    projected_depth_grad_vjp(
        p_view, R, fx, fy, v_depth_grads[idx],
        v_p_view_dg, v_R_dg);
    v_R += v_R_dg;
    v_p_view.x += v_p_view_dg.x;
    v_p_view.y += v_p_view_dg.y;
    v_p_view.z += v_p_view_dg.z;

    float3 v_p_world = transform_4x3_rot_only_transposed(viewmat, v_p_view);
    v_means3d[idx] = v_p_world;
    v_scales[idx] = v_scale;
    glm::mat3 v_R2 = glm::transpose(R1) * v_R;
    float4 v_quat = quat_to_rotmat_vjp(quat, v_R2);
    v_quats[idx] = v_quat;
}

// output space: 2D covariance, input space: cov3d
__device__ void project_cov3d_ewa_vjp(
    const float3& __restrict__ mean3d,
    const float* __restrict__ cov3d,
    const float* __restrict__ viewmat,
    const float fx,
    const float fy,
    const float3& __restrict__ v_cov2d,
    float3& __restrict__ v_mean3d,
    float* __restrict__ v_cov3d
) {
    // viewmat is row major, glm is column major
    // upper 3x3 submatrix
    // clang-format off
    glm::mat3 W = glm::mat3(
        viewmat[0], viewmat[4], viewmat[8],
        viewmat[1], viewmat[5], viewmat[9],
        viewmat[2], viewmat[6], viewmat[10]
    );
    // clang-format on
    glm::vec3 p = glm::vec3(viewmat[3], viewmat[7], viewmat[11]);
    glm::vec3 t = W * glm::vec3(mean3d.x, mean3d.y, mean3d.z) + p;
    float rz = 1.f / t.z;
    float rz2 = rz * rz;

    // column major
    // we only care about the top 2x2 submatrix
    // clang-format off
    glm::mat3 J = glm::mat3(
        fx * rz,         0.f,             0.f,
        0.f,             fy * rz,         0.f,
        -fx * t.x * rz2, -fy * t.y * rz2, 0.f
    );
    glm::mat3 V = glm::mat3(
        cov3d[0], cov3d[1], cov3d[2],
        cov3d[1], cov3d[3], cov3d[4],
        cov3d[2], cov3d[4], cov3d[5]
    );
    // cov = T * V * Tt; G = df/dcov = v_cov
    // -> d/dV = Tt * G * T
    // -> df/dT = G * T * Vt + Gt * T * V
    glm::mat3 v_cov = glm::mat3(
        v_cov2d.x,        0.5f * v_cov2d.y, 0.f,
        0.5f * v_cov2d.y, v_cov2d.z,        0.f,
        0.f,              0.f,              0.f
    );
    // clang-format on

    glm::mat3 T = J * W;
    glm::mat3 Tt = glm::transpose(T);
    glm::mat3 Vt = glm::transpose(V);
    glm::mat3 v_V = Tt * v_cov * T;
    glm::mat3 v_T = v_cov * T * Vt + glm::transpose(v_cov) * T * V;

    // vjp of cov3d parameters
    // v_cov3d_i = v_V : dV/d_cov3d_i
    // where : is frobenius inner product
    v_cov3d[0] = v_V[0][0];
    v_cov3d[1] = v_V[0][1] + v_V[1][0];
    v_cov3d[2] = v_V[0][2] + v_V[2][0];
    v_cov3d[3] = v_V[1][1];
    v_cov3d[4] = v_V[1][2] + v_V[2][1];
    v_cov3d[5] = v_V[2][2];

    // compute df/d_mean3d
    // T = J * W
    glm::mat3 v_J = v_T * glm::transpose(W);
    float rz3 = rz2 * rz;
    glm::vec3 v_t = glm::vec3(
        -fx * rz2 * v_J[2][0],
        -fy * rz2 * v_J[2][1],
        -fx * rz2 * v_J[0][0] + 2.f * fx * t.x * rz3 * v_J[2][0] -
            fy * rz2 * v_J[1][1] + 2.f * fy * t.y * rz3 * v_J[2][1]
    );
    // printf("v_t %.2f %.2f %.2f\n", v_t[0], v_t[1], v_t[2]);
    // printf("W %.2f %.2f %.2f\n", W[0][0], W[0][1], W[0][2]);
    v_mean3d.x += (float)glm::dot(v_t, W[0]);
    v_mean3d.y += (float)glm::dot(v_t, W[1]);
    v_mean3d.z += (float)glm::dot(v_t, W[2]);
}

// given cotangent v in output space (e.g. d_L/d_cov3d) in R(6)
// compute vJp for scale and rotation
__device__ void scale_rot_to_cov3d_vjp(
    const float2 scale,
    const float4 quat,
    const float* __restrict__ v_cov3d,
    float2& __restrict__ v_scale,
    float4& __restrict__ v_quat
) {
    // cov3d is upper triangular elements of matrix
    // off-diagonal elements count grads from both ij and ji elements,
    // must halve when expanding back into symmetric matrix
    glm::mat3 v_V = glm::mat3(
        v_cov3d[0],
        0.5 * v_cov3d[1],
        0.5 * v_cov3d[2],
        0.5 * v_cov3d[1],
        v_cov3d[3],
        0.5 * v_cov3d[4],
        0.5 * v_cov3d[2],
        0.5 * v_cov3d[4],
        v_cov3d[5]
    );
    glm::mat3 R = quat_to_rotmat(quat);
    glm::mat3 S = scale_to_mat({ scale.x, scale.y, 0.0f });
    glm::mat3 M = R * S;
    // https://math.stackexchange.com/a/3850121
    // for D = W * X, G = df/dD
    // df/dW = G * XT, df/dX = WT * G
    glm::mat3 v_M = 2.f * v_V * M;
    // glm::mat3 v_S = glm::transpose(R) * v_M;
    v_scale.x = (float)glm::dot(R[0], v_M[0]);
    v_scale.y = (float)glm::dot(R[1], v_M[1]);
    // v_scale.z = (float)glm::dot(R[2], v_M[2]);

    glm::mat3 v_R = v_M * S;
    v_quat = quat_to_rotmat_vjp(quat, v_R);
}


__device__ void projected_depth_grad_vjp(
    const float3 p, const glm::mat3 R,
    const float fx, const float fy,
    const float2 v_depth_grad,
    float3 &v_p_view, glm::mat3 &v_R
) {
    // forward
    glm::vec3 n1 = R[2];
    glm::mat3 invJ = glm::mat3(
        p.z/fx, 0.0f, 0.0f,
        0.0f, p.z/fy, 0.0f,
        p.x/p.z, p.y/p.z, 1.0f
    );
    glm::vec3 n = glm::transpose(invJ) * n1;
    // n.z = safe_denom(n.z, 1e-2f);
    // glm::vec2 depth_grad = glm::vec2(-n.x/n.z, -n.y/n.z);
    glm::vec2 depth_grad = glm::vec2(-n.x*n.z, -n.y*n.z);

    // backward
    glm::vec3 v_n = glm::vec3(
        // -1.0f/n.z * v_depth_grad.x,
        // -1.0f/n.z * v_depth_grad.y,
        // (n.x*v_depth_grad.x + n.y*v_depth_grad.y) / safe_denom(n.z*n.z,1e-2f)
        -n.z * v_depth_grad.x,
        -n.z * v_depth_grad.y,
        -(n.x * v_depth_grad.x + n.y*v_depth_grad.y)
    );
    // rotation
    glm::vec3 v_n1 = invJ * v_n;
    v_R = glm::mat3(0.0);
    v_R[2] = v_n1;
    // view
    glm::mat3 v_invJ = glm::outerProduct(v_n, n1);
    glm::vec3 v_p = glm::vec3(
        v_invJ[0][2] / p.z,
        v_invJ[1][2] / p.z,
        v_invJ[0][0]/fx + v_invJ[1][1]/fy -
        (p.x*v_invJ[0][2]+p.y*v_invJ[1][2]) / safe_denom(p.z*p.z, 1e-2f)
    );
    v_p_view = {v_p.x, v_p.y, v_p.z};
}
