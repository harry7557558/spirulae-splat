#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include "ch.cuh"
#include "rasterization_sorted.cuh"
#include <algorithm>

#include "stdio.h"


__global__ void rasterize_indices_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    int* __restrict__ num_intersects,
    int32_t* __restrict__ sorted_indices_,
    float* __restrict__ sorted_depths_
) {
    // each thread draws one pixel, but also timeshares caching splats in a shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ float opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // outputs
    int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    float* sorted_depths = &sorted_depths_[pix_id*MAX_SORTED_SPLATS];
    int intersect_count = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            id_batch[tr] = g_id;
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = opac;
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            float opac = opacity_batch[t];
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            if (!std::isfinite(poi.z))
                continue;
            float alpha;
            if (!get_alpha(uv, opac, alpha))
                continue;

            sorted_indices[intersect_count] = (int32_t)id_batch[t];
            sorted_depths[intersect_count] = poi.z;
            if (++intersect_count >= MAX_SORTED_SPLATS){
                done = true;
                break;
            }

            const float next_T = T * (1.f - alpha);

            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        num_intersects[pix_id] = intersect_count;

        // mark this so no need for num_intersects in rendering
        if (intersect_count < MAX_SORTED_SPLATS)
            sorted_indices[intersect_count] = SORTED_INDEX_INF;

        #if 0
            while (intersect_count < MAX_SORTED_SPLATS)
                sorted_depths[intersect_count++] = 1e6f;
        #endif
    }

}




inline __device__ void sort_per_pixel_insertion(int n, int32_t* indices, float* depths) {
    for (int i = 1; i < n; ++i) {
        int gid = indices[i];
        float z = depths[i];
        int j = i-1;

        while (j >= 0 && depths[j] > z) {
            depths[j+1] = depths[j];
            indices[j+1] = indices[j];
            j--;
        }
        depths[j+1] = z;
        indices[j+1] = gid;
    }
}


inline __device__ void sort_per_pixel_quick(int32_t* indices, float* depths, int i0, int i1) {
    if (i1-i0+1 < 8) {
        return sort_per_pixel_insertion(i1-i0+1, indices+i0, depths+i0);
    }

    // split into two arrays by value at center
    int ic = (i0+i1) / 2;
    float z = depths[ic];
    int pi = i0-1;
    for (int j = i0; j <= i1; j++) {
        if (depths[j] < z) {
            if (++pi == j) continue;
            // now there's always pi < j and j != ic
            // swap pi and j
            float z_ = depths[pi]; depths[pi] = depths[j]; depths[j] = z_;
            int32_t i_ = indices[pi]; indices[pi] = indices[j]; indices[j] = i_;
            if (ic == pi) ic = j;
        }
    }
    ++pi;
    // move ic to pi
    if (pi != ic) {
        // float z_ = depths[pi]; depths[pi] = depths[ic]; depths[ic] = z_;
        depths[ic] = depths[pi]; depths[pi] = z;
        int32_t i_ = indices[pi]; indices[pi] = indices[ic]; indices[ic] = i_;
    }

    // debug
    #if 0
        if (depths[pi] != z) printf("@");
        if (pi > ic) printf("#");
        for (int i = i0; i <= pi-1; i++)
            if (depths[i] > depths[pi]) printf("l");
        for (int i = pi+1; i <= i1; i++)
            if (depths[i] < depths[pi]) printf("r");
    #endif

    // warning: may cause stack overflow on some devices in a worst case
    // i.e. "CUDA error: an illegal memory access was encountered"
    if ((pi-1) - i0 > 0)
        sort_per_pixel_quick(indices, depths, i0, pi-1);
    if (i1 - (pi+1) > 0)
        sort_per_pixel_quick(indices, depths, pi+1, i1);
}


inline __device__ void _sort_per_pixel_heap_heapify(int n, int32_t* indices, float* depths, int i) {
    float z_i = depths[i];

    while (true) {
        int largest = i;
        float z_largest = z_i;
        int l = 2*i+1;
        int r = 2*i+2;

        float z_l = depths[l];
        if (l < n && z_l > z_largest) {
            largest = l; z_largest = z_l;
        }

        float z_r = depths[r];
        if (r < n && z_r > z_largest) {
            largest = r; z_largest = z_r;
        }

        if (largest == i) break;

        // swap i and largest
        depths[i] = z_largest; depths[largest] = z_i;
        float i_ = indices[i]; indices[i] = indices[largest]; indices[largest] = i_;

        i = largest;
    }
}

inline __device__ void sort_per_pixel_heap(int n, int32_t* indices, float* depths) {
    for (int i = n/2-1; i >= 0; --i) {
        _sort_per_pixel_heap_heapify(n, indices, depths, i);
    }

    for (int i = n-1; i > 0; --i) {
        // swap 0 and i
        float z_ = depths[0]; depths[0] = depths[i]; depths[i] = z_;
        float i_ = indices[0]; indices[0] = indices[i]; indices[i] = i_;

        _sort_per_pixel_heap_heapify(i, indices, depths, 0);
    }
}


inline __device__ void sort_per_pixel_randomized_quick(int32_t* indices, float* depths, int i0, int i1) {
    if (i1-i0+1 < 8) {
        return sort_per_pixel_insertion(i1-i0+1, indices+i0, depths+i0);
    }

    // random number (FNV-1a hash)
    uint32_t hash = 2166136261u;
    hash ^= (uint32_t)i0;
    hash *= 16777619u;
    hash ^= (uint32_t)i1;
    hash *= 16777619u;
    uint32_t m = i1-i0+1;
    int ic = i0 + (int)(hash%m);

    // split into two arrays
    float z = depths[ic];
    int pi = i0-1;
    for (int j = i0; j <= i1; j++) {
        if (depths[j] < z) {
            if (++pi == j) continue;
            // now there's always pi < j and j != ic
            // swap pi and j
            float z_ = depths[pi]; depths[pi] = depths[j]; depths[j] = z_;
            int32_t i_ = indices[pi]; indices[pi] = indices[j]; indices[j] = i_;
            if (ic == pi) ic = j;
        }
    }
    ++pi;
    // move ic to pi
    if (pi != ic) {
        // float z_ = depths[pi]; depths[pi] = depths[ic]; depths[ic] = z_;
        depths[ic] = depths[pi]; depths[pi] = z;
        int32_t i_ = indices[pi]; indices[pi] = indices[ic]; indices[ic] = i_;
    }

    // warning: may cause stack overflow on some devices in a worst case
    // i.e. "CUDA error: an illegal memory access was encountered"
    if ((pi-1) - i0 > 0)
        sort_per_pixel_randomized_quick(indices, depths, i0, pi-1);
    if (i1 - (pi+1) > 0)
        sort_per_pixel_randomized_quick(indices, depths, pi+1, i1);
}


template<typename val4>
inline __device__ void _pps_memcpy(int n, val4* src, val4* dst) {
#if 0
    for (int i = 0; i < n; i++)
        dst[i] = src[i];
#elif 0
    int m = n / 4;
    float4* src4 = (float4*)src;
    float4* dst4 = (float4*)dst;
    for (int i = 0; i < m; i++) {
        dst4[i] = src4[i];
    }
    for (int i = 4*m; i < n; i++) {
        dst[i] = src[i];
    }
#else
    int m = (n + 3) / 4;
    float4* src4 = (float4*)src;
    float4* dst4 = (float4*)dst;
    for (int i = 0; i < m; i++) {
        dst4[i] = src4[i];
    }
#endif
}

template <PerPixelSortType SORT_TYPE>
__global__ void sort_per_pixel_kernel(
    const unsigned num_pixels,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= num_pixels)
        return;

    int n = num_intersects[idx];
    if (n <= 1)
        return;
    int32_t* indices_g = &indices_[idx*MAX_SORTED_SPLATS];
    float* depths_g = &depths_[idx*MAX_SORTED_SPLATS];
    int intersect_count = 0;

    __shared__ int32_t indices_s[MAX_SORTED_SPLATS*N_THREADS_PPS];
    __shared__ float depths_s[MAX_SORTED_SPLATS*N_THREADS_PPS];

    int32_t* indices = &indices_s[MAX_SORTED_SPLATS*threadIdx.x];
    float* depths = &depths_s[MAX_SORTED_SPLATS*threadIdx.x];
    _pps_memcpy<int32_t>(n, indices_g, indices);
    _pps_memcpy<float>(n, depths_g, depths);

    // int32_t* indices = indices_g;
    // float* depths = depths_g;

    switch (SORT_TYPE)
    {
    case PerPixelSortType::InsertionSort:
        sort_per_pixel_insertion(n, indices, depths);
        break;
    case PerPixelSortType::QuickSort:
        sort_per_pixel_quick(indices, depths, 0, n-1);
        break;
    case PerPixelSortType::HeapSort:
        sort_per_pixel_heap(n, indices, depths);
        break;
    case PerPixelSortType::RandomizedQuickSort:
        sort_per_pixel_randomized_quick(indices, depths, 0, n-1);
        break;
    }

    _pps_memcpy<int32_t>(n, indices, indices_g);
    _pps_memcpy<float>(n, depths, depths_g);
}



__global__ void rasterize_simple_sorted_forward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float3& __restrict__ background,
    float3* __restrict__ out_img,
    float* __restrict__ out_alpha
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];

    // current visibility left to render
    float T = 1.f;

    // rasterize
    float3 pix_out = {0.f, 0.f, 0.f};
    for (int cur_idx = 0; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        const float next_T = T * (1.f - alpha);

        const float vis = alpha * T;
        pix_out.x = pix_out.x + color.x * vis;
        pix_out.y = pix_out.y + color.y * vis;
        pix_out.z = pix_out.z + color.z * vis;
        T = next_T;
    }

    if (inside) {
        float3 final_color;
        final_color.x = pix_out.x + T * background.x;
        final_color.y = pix_out.y + T * background.y;
        final_color.z = pix_out.z + T * background.z;
        out_img[pix_id] = final_color;
        out_alpha[pix_id] = 1.0f - T;
    }
}


__global__ void rasterize_simple_sorted_backward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int* __restrict__ num_intersects,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float3& __restrict__ background,
    const float* __restrict__ output_alpha,
    const float3* __restrict__ v_output,
    const float* __restrict__ v_output_alpha,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float* __restrict__ v_opacities
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;
    
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;
    
    int n = num_intersects[pix_id];
    if (n == 0) return;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // the contribution from gaussians behind the current one
    float3 buffer = {0.f, 0.f, 0.f};
    // index of last gaussian to contribute to this pixel

    // df/d_out for this pixel
    const float3 v_out = nan_to_num(v_output[pix_id]);
    const float v_out_alpha = nan_to_num(v_output_alpha[pix_id]);

    // rasterize
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    for (int cur_idx = n-1; cur_idx >= 0; cur_idx--) {
        int g_id = sorted_indices[cur_idx];

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        glm::vec3 v_position_local = {0.f, 0.f, 0.f};
        glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
        glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
        glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
        glm::vec3 v_color_local = {0.f, 0.f, 0.f};
        float v_opacity_local = 0.f;

        // compute the current T for this gaussian
        const float ra = 1.f / (1.f - alpha);
        const float next_T = T * ra;
        const float vis = alpha * next_T;

        // update v_rgb for this gaussian
        float v_alpha = 0.f;
        v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};

        // contribution from this pixel
        v_alpha += (color.x * T - buffer.x) * ra * v_out.x;
        v_alpha += (color.y * T - buffer.y) * ra * v_out.y;
        v_alpha += (color.z * T - buffer.z) * ra * v_out.z;

        v_alpha += T_final * ra * v_out_alpha;
        // contribution from background pixel
        v_alpha += -T_final * ra * background.x * v_out.x;
        v_alpha += -T_final * ra * background.y * v_out.y;
        v_alpha += -T_final * ra * background.z * v_out.z;
        // update the running sum
        buffer.x += color.x * vis;
        buffer.y += color.y * vis;
        buffer.z += color.z * vis;

        glm::vec2 v_uv;
        get_alpha_vjp(
            uv, opac,
            v_alpha, v_uv, v_opacity_local
        );
        glm::mat2x3 v_axis_uv = glm::mat2x3(0.0f);
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            glm::vec3(0), v_uv,
            v_position_local, v_axis_uv
        );
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
        // v_position_xy_abs_local /= pos.z;
        v_axis_u_local = v_axis_uv[0];
        v_axis_v_local = v_axis_uv[1];

        T = next_T;

        float* v_position_ptr = (float*)(v_positions);
        atomicAdd(v_position_ptr + 3*g_id + 0, v_position_local.x);
        atomicAdd(v_position_ptr + 3*g_id + 1, v_position_local.y);
        atomicAdd(v_position_ptr + 3*g_id + 2, v_position_local.z);
        float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 0, v_position_xy_abs_local.x);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 1, v_position_xy_abs_local.y);

        float* v_axis_u_ptr = (float*)(v_axes_u);
        atomicAdd(v_axis_u_ptr + 3*g_id + 0, v_axis_u_local.x);
        atomicAdd(v_axis_u_ptr + 3*g_id + 1, v_axis_u_local.y);
        atomicAdd(v_axis_u_ptr + 3*g_id + 2, v_axis_u_local.z);
        float* v_axis_v_ptr = (float*)(v_axes_v);
        atomicAdd(v_axis_v_ptr + 3*g_id + 0, v_axis_v_local.x);
        atomicAdd(v_axis_v_ptr + 3*g_id + 1, v_axis_v_local.y);
        atomicAdd(v_axis_v_ptr + 3*g_id + 2, v_axis_v_local.z);

        float* v_color_ptr = (float*)(v_colors);
        atomicAdd(v_color_ptr + 3*g_id + 0, v_color_local.x);
        atomicAdd(v_color_ptr + 3*g_id + 1, v_color_local.y);
        atomicAdd(v_color_ptr + 3*g_id + 2, v_color_local.z);
        
        atomicAdd(v_opacities + g_id, v_opacity_local);
    }

}



template <DepthMode DEPTH_MODE>
__global__ void rasterize_depth_sorted_forward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    int* __restrict__ final_index,
    float* __restrict__ out_depth,
    float2* __restrict__ out_visibility
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];

    // current visibility left to render
    float T = 1.f;
    float interp = 1.0f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // rasterize
    float output_depth = 0.0f;
    float output_visibility = 0.0f;

    for (; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        const float next_T = T * (1.f - alpha);

        // mean depth
        if (DEPTH_MODE == DepthMode::Mean) {

            // const float depth_raw = pos.z;
            const float depth_raw = poi.z;
            const float depth = depth_map(depth_raw);
            float vis = alpha * T;
            output_depth += vis * depth;

        }  // DEPTH_MODE

        // median depth
        else if (DEPTH_MODE == DepthMode::Median) {

            const float next_depth_raw = poi.z;
            const float next_depth = depth_map(next_depth_raw);
            if (next_T < DEPTH_REG_MEDIAN_TH) {
                if (T < 0.99999f) {
                    // https://www.desmos.com/3d/fttajoozww
                    interp = (1.0f-alpha)/alpha * (T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH;
                    interp = glm::clamp(interp, 0.0f, 1.0f);
                    output_depth = output_depth + (next_depth-output_depth)*interp;
                }
                else {
                    output_depth = next_depth;
                }
                T = next_T;
                cur_idx++;
                break;
            }
            output_depth = next_depth;

        }  // DEPTH_MODE

        T = next_T;
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        if (DEPTH_MODE == DepthMode::Mean) {
            float depth = T == 1.0f ? output_depth : output_depth / (1.0f-T);
            // out_depth[pix_id] = depth_inv_map(depth);
            out_depth[pix_id] = depth;
            out_visibility[pix_id] = {T, 1.0f-T};
        }
        else if (DEPTH_MODE == DepthMode::Median) {
            // out_depth[pix_id] = depth_inv_map(output_depth);
            out_depth[pix_id] = output_depth;
            out_visibility[pix_id] = {T, interp};
        }
    }
}


template <DepthMode DEPTH_MODE>
__global__ void rasterize_depth_sorted_backward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int* __restrict__ final_index,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float* __restrict__ out_depth,
    const float2* __restrict__ out_visibility,
    const float* __restrict__ v_out_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float* __restrict__ v_opacities
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;
    
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;
    
    int n = final_index[pix_id];
    if (n == 0) return;

    // this is the T AFTER the last gaussian in this pixel
    glm::vec2 meta_out = *(glm::vec2*)&out_visibility[pix_id];
    float T_final = meta_out.x;
    float T = T_final;
    float v_T = 0.0f;
    const float interp = meta_out.y;

    // df/d_out for this pixel
    float output_depth = out_depth[pix_id];
    float v_output_depth = nan_to_num(v_out_depth[pix_id]);
    float v_out_alpha = 0.0f;
    float v_depth = 0.f;
    float v_depth_next = 0.f;
    float v_alpha = 0.f;
    float v_interp = 0.f;
    if (DEPTH_MODE == DepthMode::Mean) {
        if (T != 1.0f) {
            float alpha = 1.0f-T;
            v_out_alpha = -output_depth / fmax(alpha, 1e-4f) * v_output_depth;
            output_depth *= alpha;
            v_output_depth /= fmax(alpha, 1e-4);
        }
    }

    float depth_buffer = 0.0f;

    // rasterize
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    for (int cur_idx = n-1; cur_idx >= 0; cur_idx--) {
        int g_id = sorted_indices[cur_idx];

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        glm::vec3 v_position_local = {0.f, 0.f, 0.f};
        glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
        glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
        glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
        float v_opacity_local = 0.f;

        // compute the current T for this gaussian
        const float ra = 1.f / (1.f - alpha);
        const float next_T = T * ra;
        const float vis = alpha * next_T;

        float depth_raw = poi.z;
        float depth = depth_map(depth_raw);

        // mean depth
        if (DEPTH_MODE == DepthMode::Mean) {

            v_depth = vis * v_output_depth;
            v_alpha = (depth * T - depth_buffer) * ra * v_output_depth;
            v_alpha += T_final * ra * v_out_alpha;
            depth_buffer += depth * vis;

        }  // DEPTH_MODE

        // median depth
        else if (DEPTH_MODE == DepthMode::Median) {

            // depth gradient
            if (T == T_final) {
                v_depth = v_output_depth * interp;
                v_depth_next = v_output_depth * (1.0f-interp);
            }
            else {
                v_depth = v_depth_next;
                v_depth_next = 0.0f;
            }

            // alpha gradient
            if (T == T_final && interp < 1.0f && interp > 0.0f) {
                float depth_0 = (output_depth-depth*interp) / (1.0f-interp);
                v_interp = (depth-depth_0) * v_output_depth;
                v_alpha = (next_T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH * \
                    v_interp / safe_denom(-alpha*alpha, 1e-3);
                v_T = (1.0f-alpha)/alpha * v_interp / DEPTH_REG_MEDIAN_TH;
            }
            else {
                v_alpha = v_T * (-next_T);
                v_T = v_T * (1.0f-alpha);
            }

        }  // DEPTH_MODE

        T = next_T;

        // backward
        glm::vec2 v_uv;
        get_alpha_vjp(
            uv, opac,
            v_alpha, v_uv, v_opacity_local
        );
        glm::mat2x3 v_axis_uv = glm::mat2x3(0.0f);
        float v_depth_raw = depth_map_vjp(depth_raw, v_depth);
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            {0.f, 0.f, v_depth_raw}, v_uv,
            v_position_local, v_axis_uv
        );
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
        v_axis_u_local = v_axis_uv[0];
        v_axis_v_local = v_axis_uv[1];
        
        float* v_position_ptr = (float*)(v_positions);
        atomicAdd(v_position_ptr + 3*g_id + 0, v_position_local.x);
        atomicAdd(v_position_ptr + 3*g_id + 1, v_position_local.y);
        atomicAdd(v_position_ptr + 3*g_id + 2, v_position_local.z);
        float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 0, v_position_xy_abs_local.x);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 1, v_position_xy_abs_local.y);

        float* v_axis_u_ptr = (float*)(v_axes_u);
        atomicAdd(v_axis_u_ptr + 3*g_id + 0, v_axis_u_local.x);
        atomicAdd(v_axis_u_ptr + 3*g_id + 1, v_axis_u_local.y);
        atomicAdd(v_axis_u_ptr + 3*g_id + 2, v_axis_u_local.z);
        float* v_axis_v_ptr = (float*)(v_axes_v);
        atomicAdd(v_axis_v_ptr + 3*g_id + 0, v_axis_v_local.x);
        atomicAdd(v_axis_v_ptr + 3*g_id + 1, v_axis_v_local.y);
        atomicAdd(v_axis_v_ptr + 3*g_id + 2, v_axis_v_local.z);
        
        float v_opacity_local_ = (float)v_opacity_local;
        atomicAdd(v_opacities + g_id, v_opacity_local_);
    }
}




__global__ void rasterize_sorted_forward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const float depth_reg_pairwise_factor,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float3* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    // const float3& __restrict__ background,
    const float* __restrict__ depth_ref_im,
    float* __restrict__ out_alpha,
    float3* __restrict__ out_img,
    float2* __restrict__ out_depth,
    float3* __restrict__ out_normal,
    float* __restrict__ out_reg_depth
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);

    float T = 1.f;  // current/total visibility
    float3 normal_out = {0.f, 0.f, 0.f};  // sum of normals
    float3 pix_out = {0.f, 0.f, 0.f};  // output radiance
    float vis_sum = 0.f;  // output alpha
    float depth_sum = 0.f;  // output depth
    float depth_squared_sum = 0.f;  // for L2 depth regularizer
    const float depth_ref = inside ? depth_ref_im[pix_id] : 0.f;
    float reg_depth_p = 0.f, reg_depth_i = 0.f;  // output depth regularizer
    float reg_normal = 0.f;  // output normal regularizer

    // rasterize
    for (int cur_idx = 0; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color_0 = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        const float next_T = T * (1.f - alpha);

        glm::vec3 color;
        if (dim_ch > 0) {
            const glm::vec3* coeffs = (glm::vec3*)&ch_coeffs[dim_ch*g_id];
            glm::vec3 ch_color = ch_coeffs_to_color(
                ch_degree_r, ch_degree_r_to_use,
                ch_degree_phi, ch_degree_phi_to_use,
                coeffs, {uv.x, uv.y}
            );
            color = color_0 / (1.0f+glm::exp(-ch_color));
        }
        else color = color_0;

        const float vis = alpha * T;
        #if DEPTH_REG_L == 01 && false
        const float depth_raw = pos.z;
        #else
        const float depth_raw = poi.z;
        #endif
        const float depth = depth_map(depth_raw);

        pix_out.x = pix_out.x + color.x * vis;
        pix_out.y = pix_out.y + color.y * vis;
        pix_out.z = pix_out.z + color.z * vis;

        // depth regularization
        {
            float pairwise_l1 = vis*depth * vis_sum - vis * depth_sum;  // requires pos.z for depth
            float pairwise_l2 = vis * (vis_sum*depth*depth + depth_squared_sum - 2.0f*depth*depth_sum);
            float intersect_l1 = vis * abs(depth - depth_ref);
            float intersect_l2 = vis * (depth-depth_ref) * (depth-depth_ref);
            reg_depth_p += pairwise_l2;
            reg_depth_i += intersect_l1;
        }
        vis_sum += vis;
        depth_sum += vis*depth;
        depth_squared_sum += vis*depth*depth;

        // normal regularization
        glm::vec3 normal = get_normal_from_axisuv(axis_uv, poi);
        normal_out.x = normal_out.x + normal.x * vis;
        normal_out.y = normal_out.y + normal.y * vis;
        normal_out.z = normal_out.z + normal.z * vis;

        T = next_T;
    }

    if (inside) {
        out_alpha[pix_id] = 1.0f - T;
        float3 final_color;
        // final_color.x = pix_out.x + T * background.x;
        // final_color.y = pix_out.y + T * background.y;
        // final_color.z = pix_out.z + T * background.z;
        final_color.x = pix_out.x;
        final_color.y = pix_out.y;
        final_color.z = pix_out.z;
        out_img[pix_id] = final_color;
        out_depth[pix_id] = {depth_sum, depth_squared_sum};
        out_normal[pix_id] = normal_out;
        out_reg_depth[pix_id] = reg_depth_i + (reg_depth_p-reg_depth_i) * depth_reg_pairwise_factor;
    }
}


__global__ void rasterize_sorted_backward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float depth_reg_pairwise_factor,
    const int* __restrict__ num_intersects,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float3* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    // const float3& __restrict__ background,
    const float* __restrict__ depth_ref_im,
    const float* __restrict__ output_alpha,
    const float2* __restrict__ output_depth,
    const float* __restrict__ v_output_alpha,
    const float3* __restrict__ v_output,
    const float2* __restrict__ v_output_depth,
    const float3* __restrict__ v_output_normal,
    const float* __restrict__ v_output_reg_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float3* __restrict__ v_ch_coeffs,
    // float* __restrict__ v_ch_coeffs_abs,
    float* __restrict__ v_opacities,
    // float3* __restrict__ v_background,
    float* __restrict__ v_depth_ref_im
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;
    
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;
    
    int n = num_intersects[pix_id];
    if (n == 0) return;

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);
    assert(dim_ch <= MAX_CH_FLOAT3);

    // df/d_out for this pixel
    const float2 out_depth = output_depth[pix_id];
    const float3 v_out = nan_to_num(v_output[pix_id]);
    const float2 v_out_depth = nan_to_num(v_output_depth[pix_id]);
    const float3 v_out_normal = nan_to_num(v_output_normal[pix_id]);
    const float v_out_alpha = nan_to_num(v_output_alpha[pix_id]);
    const float v_out_reg_depth = nan_to_num(v_output_reg_depth[pix_id]);
    const float v_reg_depth_p = v_out_reg_depth * depth_reg_pairwise_factor;
    const float v_reg_depth_i = v_out_reg_depth * (1.0f-depth_reg_pairwise_factor);
    const float v_depth_sum = v_out_depth.x;
    const float v_depth_squared_sum = v_out_depth.y;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;

    // regularization
    const float depth_ref = inside ? depth_ref_im[pix_id] : 0.f;
    float v_depth_ref = 0.f;

    const float vis_sum_final = 1.0f - T_final;
    const float depth_sum_final = out_depth.x;
    const float depth_squared_sum_final = out_depth.y;
    float vis_sum = vis_sum_final;
    float depth_sum = depth_sum_final;
    float depth_squared_sum = depth_squared_sum_final;

    float3 buffer = {0.f, 0.f, 0.f};
    float2 buffer_depth = {0.f, 0.f};
    float3 buffer_normal = {0.f, 0.f, 0.f};
    float buffer_depth_reg = 0.f;
    
    float v_sum_vis = v_out_alpha;

    // rasterize
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    for (int cur_idx = n-1; cur_idx >= 0; cur_idx--) {
        int g_id = sorted_indices[cur_idx];

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color_0 = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        glm::vec3 v_position_local = {0.f, 0.f, 0.f};
        glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
        glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
        glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
        glm::vec3 v_color_local = {0.f, 0.f, 0.f};
        float v_opacity_local = 0.f;
        glm::vec3 v_ch_coeff_local[MAX_CH_FLOAT3];
        for (int i = 0; i < dim_ch; i++)
            v_ch_coeff_local[i] = {0.f, 0.f, 0.f};
        float v_ch_coeff_abs_local = 0.f;
        //initialize everything to 0, only set if the lane is valid

        // compute the current T for this gaussian
        const float ra = 1.f / (1.f - alpha);
        const float next_T = T * ra;
        const float vis = alpha * next_T;

        // update accumulation
        float v_depth = 0.0f;
        #if DEPTH_REG_L == 01 && false
        const float depth_raw = pos.z;
        const float depth = depth_map(depth_raw);
        v_depth += vis * v_depth_sum;
        v_depth += vis * 2.0f*depth * v_depth_squared_sum;
        #else
        const float depth_raw = poi.z;
        const float depth = depth_map(depth_raw);
        v_depth += vis * v_depth_sum;
        v_depth += vis * 2.0f*depth * v_depth_squared_sum;
        #endif

        // update depth regularizer
        float vis_sum_next = vis_sum - vis;
        float depth_sum_next = depth_sum - vis*depth;
        float depth_squared_sum_next = depth_squared_sum - vis*depth*depth;
        #if 0  // pairwise L1, requires pos.z for depth
        v_depth += v_reg_depth_p * vis * (vis_sum_next - (vis_sum_final-vis_sum));
        float reg_depth_i_p = (
            depth * vis_sum_next - depth_sum_next +
            (depth_sum_final-depth_sum) - depth * (vis_sum_final-vis_sum)
        );
        v_position_local.z = depth_map_vjp(depth_raw, v_depth);
        #else  // pairwise L2
        v_depth += v_reg_depth_p * vis * 2.0f * (
            vis_sum_final * depth - depth_sum_final);
        float reg_depth_i_p =
            vis_sum_final*depth*depth + depth_squared_sum_final
            - 2.0f*depth*depth_sum_final;
        #endif
        #if 1  // L1 with intersected depth
        float v_z = v_reg_depth_i * vis * glm::sign(depth-depth_ref);
        v_depth += v_z;
        v_depth_ref += (-v_z);
        float reg_depth_i_i = abs(depth-depth_ref);
        #else  // L2 with intersected depth
        float v_z = v_reg_depth_i * vis * 2.0f*(depth-depth_ref);
        v_depth += v_z;
        v_depth_ref += (-v_z);
        float reg_depth_i_i = (depth-depth_ref) * (depth-depth_ref);
        #endif
        float reg_depth_i = reg_depth_i_i + (reg_depth_i_p-reg_depth_i_i) * depth_reg_pairwise_factor;

        float v_depth_raw = depth_map_vjp(depth_raw, v_depth);
        glm::vec3 v_poi = {0.f, 0.f, v_depth_raw};

        // normal regularization
        glm::vec3 v_normal = {vis * v_out_normal.x, vis * v_out_normal.y, vis * v_out_normal.z};
        glm::mat2x3 v_axis_uv; glm::vec3 normal;
        get_normal_from_axisuv_vjp(axis_uv, poi, v_normal, normal, v_axis_uv);

        // update color
        glm::vec3 v_color_1 = {vis * v_out.x, vis * v_out.y, vis * v_out.z};
        glm::vec3 color_1;
        glm::vec2 v_uv_ch = {0.f, 0.f};
        if (dim_ch > 0) {
            glm::vec3 v_ch_color_sigmoid = v_color_1 * color_0;
            #if 0
            int32_t g_id = id_batch[t];
            glm::vec3 ch_color = ch_coeffs_to_color(
                ch_degree_r, ch_degree_r_to_use,
                ch_degree_phi, ch_degree_phi_to_use,
                (glm::vec3*)&ch_coeffs[dim_ch*g_id], {uv.x, uv.y}
            );
            glm::vec3 ch_color_sigmoid = 1.0f / (1.0f+glm::exp(-ch_color));
            glm::vec3 v_ch_color = v_ch_color_sigmoid * ch_color_sigmoid*(1.0f-ch_color_sigmoid);
            ch_coeffs_to_color_vjp(
                ch_degree_r, ch_degree_r_to_use,
                ch_degree_phi, ch_degree_phi_to_use,
                (glm::vec3*)&ch_coeffs[dim_ch*g_id],
                {uv.x, uv.y},
                v_ch_color,
                ch_color,
                v_ch_coeff_local, v_ch_coeff_abs_local,
                v_uv_ch
            );
            #else
            // makes overall training 0.1x faster
            glm::vec3 ch_color_sigmoid;
            ch_coeffs_to_color_sigmoid_vjp(
                ch_degree_r, ch_degree_r_to_use,
                ch_degree_phi, ch_degree_phi_to_use,
                (glm::vec3*)&ch_coeffs[dim_ch*g_id],
                {uv.x, uv.y},
                v_ch_color_sigmoid,
                ch_color_sigmoid,
                v_ch_coeff_local, v_ch_coeff_abs_local,
                v_uv_ch
            );
            #endif
            color_1 = color_0 * ch_color_sigmoid;
            v_color_local = v_color_1 * ch_color_sigmoid;
        }
        else {
            color_1 = color_0;
            v_color_local = v_color_1;
        }

        float v_alpha = 0.0f;
        // contribution from this pixel
        v_alpha += (color_1.x * T - buffer.x) * ra * v_out.x;
        v_alpha += (color_1.y * T - buffer.y) * ra * v_out.y;
        v_alpha += (color_1.z * T - buffer.z) * ra * v_out.z;
        v_alpha += T_final * ra * v_out_alpha;
        // v_alpha += -T_final * ra * background.x * v_out.x;
        // v_alpha += -T_final * ra * background.y * v_out.y;
        // v_alpha += -T_final * ra * background.z * v_out.z;
        float v_alpha_color_only = v_alpha;
        v_alpha += (depth * T - buffer_depth.x) * ra * v_depth_sum;
        v_alpha += (depth*depth * T - buffer_depth.y) * ra * v_depth_squared_sum;
        v_alpha += (reg_depth_i * T - buffer_depth_reg) * ra * v_out_reg_depth;
        v_alpha += (normal.x * T - buffer_normal.x) * ra * v_out_normal.x;
        v_alpha += (normal.y * T - buffer_normal.y) * ra * v_out_normal.y;
        v_alpha += (normal.z * T - buffer_normal.z) * ra * v_out_normal.z;

        // update the running sum
        buffer.x += color_1.x * vis;
        buffer.y += color_1.y * vis;
        buffer.z += color_1.z * vis;
        buffer_depth.x += depth * vis;
        buffer_depth.y += depth*depth * vis;
        buffer_depth_reg += reg_depth_i * vis;
        buffer_normal.x += normal.x * vis;
        buffer_normal.y += normal.y * vis;
        buffer_normal.z += normal.z * vis;

        // grad
        glm::vec2 v_uv;
        get_alpha_vjp(
            uv, opac,
            v_alpha, v_uv, v_opacity_local
        );
        v_uv += v_uv_ch;
        glm::vec3 v_position_local_temp;
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            v_poi, v_uv,
            v_position_local_temp, v_axis_uv
        );
        v_position_local += v_position_local_temp;
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
        v_axis_u_local = v_axis_uv[0];
        v_axis_v_local = v_axis_uv[1];

        // absgrad (color only)
        #if 0
        float v_opacity_local_1;
        get_alpha_vjp(
            uv, opac,
            v_alpha_color_only,
            v_uv, v_opacity_local_1
        );
        v_uv += v_uv_ch;
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            glm::vec3(0), v_uv,
            v_position_local_temp, v_axis_uv
        );
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local_temp));
        #endif

        // next loop
        T = next_T;
        vis_sum = vis_sum_next;
        depth_sum = depth_sum_next;
        depth_squared_sum = depth_squared_sum_next;

        float* v_position_ptr = (float*)(v_positions);
        atomicAdd(v_position_ptr + 3*g_id + 0, v_position_local.x);
        atomicAdd(v_position_ptr + 3*g_id + 1, v_position_local.y);
        atomicAdd(v_position_ptr + 3*g_id + 2, v_position_local.z);
        float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 0, v_position_xy_abs_local.x);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 1, v_position_xy_abs_local.y);

        float* v_axis_u_ptr = (float*)(v_axes_u);
        atomicAdd(v_axis_u_ptr + 3*g_id + 0, v_axis_u_local.x);
        atomicAdd(v_axis_u_ptr + 3*g_id + 1, v_axis_u_local.y);
        atomicAdd(v_axis_u_ptr + 3*g_id + 2, v_axis_u_local.z);
        float* v_axis_v_ptr = (float*)(v_axes_v);
        atomicAdd(v_axis_v_ptr + 3*g_id + 0, v_axis_v_local.x);
        atomicAdd(v_axis_v_ptr + 3*g_id + 1, v_axis_v_local.y);
        atomicAdd(v_axis_v_ptr + 3*g_id + 2, v_axis_v_local.z);
        
        float* v_color_ptr = (float*)(v_colors);
        atomicAdd(v_color_ptr + 3*g_id + 0, v_color_local.x);
        atomicAdd(v_color_ptr + 3*g_id + 1, v_color_local.y);
        atomicAdd(v_color_ptr + 3*g_id + 2, v_color_local.z);
        float* v_ch_coeffs_ptr = (float*)(v_ch_coeffs);
        for (int i = 0; i < dim_ch; i++) {
            atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g_id + 3*i + 0, v_ch_coeff_local[i].x);
            atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g_id + 3*i + 1, v_ch_coeff_local[i].y);
            atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g_id + 3*i + 2, v_ch_coeff_local[i].z);
        }
        // atomicAdd(v_ch_coeffs_abs + g, v_ch_coeff_abs_local);

        atomicAdd(v_opacities + g_id, v_opacity_local);
    }

    if (inside) {
        v_depth_ref_im[pix_id] = v_depth_ref;

        // background gradient
        #if 0
        float3 v_bkg = {
            v_out.x * T_final,
            v_out.y * T_final,
            v_out.z * T_final
        };
        atomicAdd((float*)v_background+0, v_bkg.x);
        atomicAdd((float*)v_background+1, v_bkg.y);
        atomicAdd((float*)v_background+2, v_bkg.z);
        #endif
    }

}



__global__ void rasterize_simplified_sorted_forward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    float* __restrict__ out_alpha,
    float3* __restrict__ out_img,
    float2* __restrict__ out_depth,  // { depth, depth^2 }
    float3* __restrict__ out_normal,
    float* __restrict__ out_depth_reg
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    float T = 1.f;  // current/total visibility
    float3 normal_out = {0.f, 0.f, 0.f};  // sum of normals
    float3 pix_out = {0.f, 0.f, 0.f};  // output radiance
    float vis_sum = 0.f;  // output alpha
    float depth_sum = 0.f;  // output depth
    float depth_squared_sum = 0.f;  // for L2 depth regularizer
    float reg_depth_p = 0.f;  // output depth regularizer

    // rasterize
    for (int cur_idx = 0; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        const float next_T = T * (1.f - alpha);
        const float vis = alpha * T;

        // color
        pix_out.x = pix_out.x + color.x * vis;
        pix_out.y = pix_out.y + color.y * vis;
        pix_out.z = pix_out.z + color.z * vis;

        // depth regularization
        const float depth_raw = poi.z;
        const float depth = depth_map(depth_raw);
        {
            float pairwise_l1 = vis*depth * vis_sum - vis * depth_sum;  // requires pos.z for depth
            float pairwise_l2 = vis * (vis_sum*depth*depth + depth_squared_sum - 2.0f*depth*depth_sum);
            reg_depth_p += pairwise_l2;
        }
        vis_sum += vis;
        depth_sum += vis*depth;
        depth_squared_sum += vis*depth*depth;

        // normal regularization
        glm::vec3 normal = get_normal_from_axisuv(axis_uv, poi);
        normal_out.x = normal_out.x + normal.x * vis;
        normal_out.y = normal_out.y + normal.y * vis;
        normal_out.z = normal_out.z + normal.z * vis;

        T = next_T;
    }

    if (inside) {
        out_alpha[pix_id] = 1.0f - T;
        out_img[pix_id] = pix_out;
        out_depth[pix_id] = { depth_sum, depth_squared_sum };
        out_normal[pix_id] = normal_out;
        out_depth_reg[pix_id] = reg_depth_p;
    }
}


__global__ void rasterize_simplified_sorted_backward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int* __restrict__ num_intersects,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float* __restrict__ output_alpha,
    const float2* __restrict__ output_depth,
    const float* __restrict__ v_output_alpha,
    const float3* __restrict__ v_output_img,
    const float2* __restrict__ v_output_depth,
    const float3* __restrict__ v_output_normal,
    const float* __restrict__ v_output_depth_reg,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float* __restrict__ v_opacities
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;
    
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;
    
    int n = num_intersects[pix_id];
    if (n == 0) return;

    // df/d_out for this pixel
    const float3 v_out = nan_to_num(v_output_img[pix_id]);
    const float2 v_out_depth = nan_to_num(v_output_depth[pix_id]);
    const float3 v_out_normal = nan_to_num(v_output_normal[pix_id]);
    const float v_out_alpha = nan_to_num(v_output_alpha[pix_id]);
    const float v_reg_depth_p = nan_to_num(v_output_depth_reg[pix_id]);
    const float v_depth_sum = v_out_depth.x;
    const float v_depth_squared_sum = v_out_depth.y;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();

    const float vis_sum_final = 1.0f - T_final;
    const float depth_sum_final = output_depth[pix_id].x;
    const float depth_squared_sum_final = output_depth[pix_id].y;
    float vis_sum = vis_sum_final;

    float3 buffer = {0.f, 0.f, 0.f};
    float2 buffer_depth = {0.f, 0.f};  // depth, depth^2
    float3 buffer_normal = {0.f, 0.f, 0.f};
    float buffer_depth_reg = 0.f;

    float v_sum_vis = v_out_alpha;

    // rasterize
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    for (int cur_idx = n-1; cur_idx >= 0; cur_idx--) {
        int g_id = sorted_indices[cur_idx];

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        glm::vec3 v_position_local = {0.f, 0.f, 0.f};
        glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
        glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
        glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
        glm::vec3 v_color_local = {0.f, 0.f, 0.f};
        float v_opacity_local = 0.f;
        //initialize everything to 0, only set if the lane is valid

        // compute the current T for this gaussian
        const float ra = 1.f / (1.f - alpha);
        const float next_T = T * ra;
        const float vis = alpha * next_T;

        // update accumulation
        const float depth_raw = poi.z;
        const float depth = depth_map(depth_raw);
        float v_depth = 0.0f;
        v_depth += vis * v_depth_sum;
        v_depth += vis * 2.0f*depth * v_depth_squared_sum;

        // update depth regularizer
        float vis_sum_next = vis_sum - vis;
        // pairwise L2
        v_depth += v_reg_depth_p * vis * 2.0f * (
            vis_sum_final * depth - depth_sum_final);
        float reg_depth_i =
            vis_sum_final*depth*depth + depth_squared_sum_final
            - 2.0f*depth*depth_sum_final;

        float v_depth_raw = depth_map_vjp(depth_raw, v_depth);
        glm::vec3 v_poi = {0.f, 0.f, v_depth_raw};

        // update color
        v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};

        // normal regularization
        glm::vec3 v_normal = {vis * v_out_normal.x, vis * v_out_normal.y, vis * v_out_normal.z};
        glm::mat2x3 v_axis_uv; glm::vec3 normal;
        get_normal_from_axisuv_vjp(axis_uv, poi, v_normal, normal, v_axis_uv);

        float v_alpha = 0.0f;
        // contribution from this pixel
        v_alpha += (color.x * T - buffer.x) * ra * v_out.x;
        v_alpha += (color.y * T - buffer.y) * ra * v_out.y;
        v_alpha += (color.z * T - buffer.z) * ra * v_out.z;
        v_alpha += T_final * ra * v_out_alpha;
        float v_alpha_color_only = v_alpha;
        v_alpha += (depth * T - buffer_depth.x) * ra * v_depth_sum;
        v_alpha += (depth*depth * T - buffer_depth.y) * ra * v_depth_squared_sum;
        v_alpha += (reg_depth_i * T - buffer_depth_reg) * ra * v_reg_depth_p;
        v_alpha += (normal.x * T - buffer_normal.x) * ra * v_out_normal.x;
        v_alpha += (normal.y * T - buffer_normal.y) * ra * v_out_normal.y;
        v_alpha += (normal.z * T - buffer_normal.z) * ra * v_out_normal.z;

        // update the running sum
        buffer.x += color.x * vis;
        buffer.y += color.y * vis;
        buffer.z += color.z * vis;
        buffer_depth.x += depth * vis;
        buffer_depth.y += depth*depth * vis;
        buffer_depth_reg += reg_depth_i * vis;
        buffer_normal.x += normal.x * vis;
        buffer_normal.y += normal.y * vis;
        buffer_normal.z += normal.z * vis;

        // grad
        glm::vec2 v_uv;
        get_alpha_vjp(
            uv, opac,
            v_alpha, v_uv, v_opacity_local
        );
        glm::vec3 v_position_local_temp;
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            v_poi, v_uv,
            v_position_local_temp, v_axis_uv
        );
        v_position_local += v_position_local_temp;
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
        v_axis_u_local = v_axis_uv[0];
        v_axis_v_local = v_axis_uv[1];

        // next loop
        T = next_T;
        vis_sum = vis_sum_next;

        float* v_position_ptr = (float*)(v_positions);
        atomicAdd(v_position_ptr + 3*g_id + 0, v_position_local.x);
        atomicAdd(v_position_ptr + 3*g_id + 1, v_position_local.y);
        atomicAdd(v_position_ptr + 3*g_id + 2, v_position_local.z);
        float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 0, v_position_xy_abs_local.x);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 1, v_position_xy_abs_local.y);

        float* v_axis_u_ptr = (float*)(v_axes_u);
        atomicAdd(v_axis_u_ptr + 3*g_id + 0, v_axis_u_local.x);
        atomicAdd(v_axis_u_ptr + 3*g_id + 1, v_axis_u_local.y);
        atomicAdd(v_axis_u_ptr + 3*g_id + 2, v_axis_u_local.z);
        float* v_axis_v_ptr = (float*)(v_axes_v);
        atomicAdd(v_axis_v_ptr + 3*g_id + 0, v_axis_v_local.x);
        atomicAdd(v_axis_v_ptr + 3*g_id + 1, v_axis_v_local.y);
        atomicAdd(v_axis_v_ptr + 3*g_id + 2, v_axis_v_local.z);
        
        float* v_color_ptr = (float*)(v_colors);
        atomicAdd(v_color_ptr + 3*g_id + 0, v_color_local.x);
        atomicAdd(v_color_ptr + 3*g_id + 1, v_color_local.y);
        atomicAdd(v_color_ptr + 3*g_id + 2, v_color_local.z);

        atomicAdd(v_opacities + g_id, v_opacity_local);
    }

}





template __global__ void sort_per_pixel_kernel<PerPixelSortType::InsertionSort>(
    const unsigned num_pixels,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void sort_per_pixel_kernel<PerPixelSortType::QuickSort>(
    const unsigned num_pixels,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void sort_per_pixel_kernel<PerPixelSortType::HeapSort>(
    const unsigned num_pixels,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void sort_per_pixel_kernel<PerPixelSortType::RandomizedQuickSort>(
    const unsigned num_pixels,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void rasterize_depth_sorted_forward_kernel<DepthMode::Mean>(
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    int* __restrict__ final_index,
    float* __restrict__ out_depth,
    float2* __restrict__ out_visibility
);

template __global__ void rasterize_depth_sorted_forward_kernel<DepthMode::Median>(
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    int* __restrict__ final_index,
    float* __restrict__ out_depth,
    float2* __restrict__ out_visibility
);

template __global__ void rasterize_depth_sorted_backward_kernel<DepthMode::Mean>(
    const dim3 img_size,
    const float4 intrins,
    const int* __restrict__ num_intersects,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float* __restrict__ out_depth,
    const float2* __restrict__ out_visibility,
    const float* __restrict__ v_out_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float* __restrict__ v_opacities
);

template __global__ void rasterize_depth_sorted_backward_kernel<DepthMode::Median>(
    const dim3 img_size,
    const float4 intrins,
    const int* __restrict__ num_intersects,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float* __restrict__ out_depth,
    const float2* __restrict__ out_visibility,
    const float* __restrict__ v_out_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float* __restrict__ v_opacities
);
