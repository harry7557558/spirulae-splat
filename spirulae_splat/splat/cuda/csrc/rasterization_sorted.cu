#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include "ch.cuh"
#include "rasterization_sorted.cuh"
#include <algorithm>

#include "stdio.h"


template<CameraType CAMERA_TYPE>
__global__ void rasterize_indices_kernel(
    _ARGS_rasterize_indices_kernel
) {
    // each thread draws one pixel, but also timeshares caching splats in a shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    if (CAMERA_TYPE == CameraType::GenericDistorted) {
        float2 pos_2d_u = undistortion_map[pix_id];
        if (isnan(pos_2d.x+pos_2d.y))
            done = true;
        else
            pos_2d = { pos_2d_u.x, pos_2d_u.y };
    }

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ float opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;

    // outputs
    int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    float* sorted_depths = &sorted_depths_[pix_id*MAX_SORTED_SPLATS];
    int intersect_count = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            id_batch[tr] = g_id;
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = opac;
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            float opac = opacity_batch[t];
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            if (!std::isfinite(poi.z))
                continue;
            float alpha;
            if (!get_alpha(uv, opac, alpha))
                continue;

            sorted_indices[intersect_count] = (int32_t)id_batch[t];
            sorted_depths[intersect_count] = poi.z;
            if (++intersect_count >= MAX_SORTED_SPLATS){
                done = true;
                break;
            }

            const float next_T = T * (1.f - alpha);

            T = next_T;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        num_intersects[pix_id] = intersect_count;

        // mark this so no need for num_intersects in rendering
        if (intersect_count < MAX_SORTED_SPLATS)
            sorted_indices[intersect_count] = SORTED_INDEX_INF;

        #if 0
            while (intersect_count < MAX_SORTED_SPLATS)
                sorted_depths[intersect_count++] = 1e6f;
        #endif
    }

}




inline __device__ void sort_per_pixel_insertion(int n, int32_t* indices, float* depths) {
    for (int i = 1; i < n; ++i) {
        int gid = indices[i];
        float z = depths[i];
        int j = i-1;

        while (j >= 0 && depths[j] > z) {
            depths[j+1] = depths[j];
            indices[j+1] = indices[j];
            j--;
        }
        depths[j+1] = z;
        indices[j+1] = gid;
    }
}


inline __device__ void sort_per_pixel_quick(int32_t* indices, float* depths, int i0, int i1) {
    if (i1-i0+1 < 8) {
        return sort_per_pixel_insertion(i1-i0+1, indices+i0, depths+i0);
    }

    // split into two arrays by value at center
    int ic = (i0+i1) / 2;
    float z = depths[ic];
    int pi = i0-1;
    for (int j = i0; j <= i1; j++) {
        if (depths[j] < z) {
            if (++pi == j) continue;
            // now there's always pi < j and j != ic
            // swap pi and j
            float z_ = depths[pi]; depths[pi] = depths[j]; depths[j] = z_;
            int32_t i_ = indices[pi]; indices[pi] = indices[j]; indices[j] = i_;
            if (ic == pi) ic = j;
        }
    }
    ++pi;
    // move ic to pi
    if (pi != ic) {
        // float z_ = depths[pi]; depths[pi] = depths[ic]; depths[ic] = z_;
        depths[ic] = depths[pi]; depths[pi] = z;
        int32_t i_ = indices[pi]; indices[pi] = indices[ic]; indices[ic] = i_;
    }

    // debug
    #if 0
        if (depths[pi] != z) printf("@");
        if (pi > ic) printf("#");
        for (int i = i0; i <= pi-1; i++)
            if (depths[i] > depths[pi]) printf("l");
        for (int i = pi+1; i <= i1; i++)
            if (depths[i] < depths[pi]) printf("r");
    #endif

    // warning: may cause stack overflow on some devices in a worst case
    // i.e. "CUDA error: an illegal memory access was encountered"
    if ((pi-1) - i0 > 0)
        sort_per_pixel_quick(indices, depths, i0, pi-1);
    if (i1 - (pi+1) > 0)
        sort_per_pixel_quick(indices, depths, pi+1, i1);
}


inline __device__ void _sort_per_pixel_heap_heapify(int n, int32_t* indices, float* depths, int i) {
    float z_i = depths[i];

    while (true) {
        int largest = i;
        float z_largest = z_i;
        int l = 2*i+1;
        int r = 2*i+2;

        float z_l = depths[l];
        if (l < n && z_l > z_largest) {
            largest = l; z_largest = z_l;
        }

        float z_r = depths[r];
        if (r < n && z_r > z_largest) {
            largest = r; z_largest = z_r;
        }

        if (largest == i) break;

        // swap i and largest
        depths[i] = z_largest; depths[largest] = z_i;
        float i_ = indices[i]; indices[i] = indices[largest]; indices[largest] = i_;

        i = largest;
    }
}

inline __device__ void sort_per_pixel_heap(int n, int32_t* indices, float* depths) {
    for (int i = n/2-1; i >= 0; --i) {
        _sort_per_pixel_heap_heapify(n, indices, depths, i);
    }

    for (int i = n-1; i > 0; --i) {
        // swap 0 and i
        float z_ = depths[0]; depths[0] = depths[i]; depths[i] = z_;
        float i_ = indices[0]; indices[0] = indices[i]; indices[i] = i_;

        _sort_per_pixel_heap_heapify(i, indices, depths, 0);
    }
}


inline __device__ void sort_per_pixel_randomized_quick(int32_t* indices, float* depths, int i0, int i1) {
    if (i1-i0+1 < 8) {
        return sort_per_pixel_insertion(i1-i0+1, indices+i0, depths+i0);
    }

    // random number (FNV-1a hash)
    uint32_t hash = 2166136261u;
    hash ^= (uint32_t)i0;
    hash *= 16777619u;
    hash ^= (uint32_t)i1;
    hash *= 16777619u;
    uint32_t m = i1-i0+1;
    int ic = i0 + (int)(hash%m);

    // split into two arrays
    float z = depths[ic];
    int pi = i0-1;
    for (int j = i0; j <= i1; j++) {
        if (depths[j] < z) {
            if (++pi == j) continue;
            // now there's always pi < j and j != ic
            // swap pi and j
            float z_ = depths[pi]; depths[pi] = depths[j]; depths[j] = z_;
            int32_t i_ = indices[pi]; indices[pi] = indices[j]; indices[j] = i_;
            if (ic == pi) ic = j;
        }
    }
    ++pi;
    // move ic to pi
    if (pi != ic) {
        // float z_ = depths[pi]; depths[pi] = depths[ic]; depths[ic] = z_;
        depths[ic] = depths[pi]; depths[pi] = z;
        int32_t i_ = indices[pi]; indices[pi] = indices[ic]; indices[ic] = i_;
    }

    // warning: may cause stack overflow on some devices in a worst case
    // i.e. "CUDA error: an illegal memory access was encountered"
    if ((pi-1) - i0 > 0)
        sort_per_pixel_randomized_quick(indices, depths, i0, pi-1);
    if (i1 - (pi+1) > 0)
        sort_per_pixel_randomized_quick(indices, depths, pi+1, i1);
}


inline __device__ void _sort_per_pixel_network_compare_swap(int32_t *indices, float* depths, int i, int j) {
    float u = depths[i], v = depths[j];
    if (u > v) {
        depths[j] = u, depths[i] = v;
        int32_t k = indices[i]; indices[i] = indices[j], indices[j] = k;
    }
}

inline __device__ void sort_per_pixel_network(int n, int32_t* indices, float* depths) {
    // https://bertdobbelaere.github.io/sorting_networks.html
    // s = """<paste sorting network>"""
    // for i, j in sum([eval(x) for x in s.strip().split('\n')], []):
    //     print(f"_({i},{j})", end='')
    // print(' ')
    #define _(i,j) _sort_per_pixel_network_compare_swap(indices,depths,i,j);
    if (n <= 4) {
        if (n == 2) { _(0,1) }
        else if (n == 3) { _(0,2)_(0,1)_(1,2) }
        else { _(0,2)_(1,3)_(0,1)_(2,3)_(1,2) }
    }
    else if (n <= 8) {
        if (n == 5) { _(0,3)_(1,4)_(0,2)_(1,3)_(0,1)_(2,4)_(1,2)_(3,4)_(2,3) }
        else if (n == 6) { _(0,5)_(1,3)_(2,4)_(1,2)_(3,4)_(0,3)_(2,5)_(0,1)_(2,3)_(4,5)_(1,2)_(3,4) }
        else if (n == 7) { _(0,6)_(2,3)_(4,5)_(0,2)_(1,4)_(3,6)_(0,1)_(2,5)_(3,4)_(1,2)_(4,6)_(2,3)_(4,5)_(1,2)_(3,4)_(5,6) }
        else { _(0,2)_(1,3)_(4,6)_(5,7)_(0,4)_(1,5)_(2,6)_(3,7)_(0,1)_(2,3)_(4,5)_(6,7)_(2,4)_(3,5)_(1,4)_(3,6)_(1,2)_(3,4)_(5,6) }
    }
#if MAX_SORTED_SPLATS > 8
    else if (n <= 12) {
        if (n == 9) { _(0,3)_(1,7)_(2,5)_(4,8)_(0,7)_(2,4)_(3,8)_(5,6)_(0,2)_(1,3)_(4,5)_(7,8)_(1,4)_(3,6)_(5,7)_(0,1)_(2,4)_(3,5)_(6,8)_(2,3)_(4,5)_(6,7)_(1,2)_(3,4)_(5,6) }
        else if (n == 10) { _(0,8)_(1,9)_(2,7)_(3,5)_(4,6)_(0,2)_(1,4)_(5,8)_(7,9)_(0,3)_(2,4)_(5,7)_(6,9)_(0,1)_(3,6)_(8,9)_(1,5)_(2,3)_(4,8)_(6,7)_(1,2)_(3,5)_(4,6)_(7,8)_(2,3)_(4,5)_(6,7)_(3,4)_(5,6) }
        else if (n == 11) { _(0,9)_(1,6)_(2,4)_(3,7)_(5,8)_(0,1)_(3,5)_(4,10)_(6,9)_(7,8)_(1,3)_(2,5)_(4,7)_(8,10)_(0,4)_(1,2)_(3,7)_(5,9)_(6,8)_(0,1)_(2,6)_(4,5)_(7,8)_(9,10)_(2,4)_(3,6)_(5,7)_(8,9)_(1,2)_(3,4)_(5,6)_(7,8)_(2,3)_(4,5)_(6,7) }
        else { _(0,8)_(1,7)_(2,6)_(3,11)_(4,10)_(5,9)_(0,1)_(2,5)_(3,4)_(6,9)_(7,8)_(10,11)_(0,2)_(1,6)_(5,10)_(9,11)_(0,3)_(1,2)_(4,6)_(5,7)_(8,11)_(9,10)_(1,4)_(3,5)_(6,8)_(7,10)_(1,3)_(2,5)_(6,9)_(8,10)_(2,3)_(4,5)_(6,7)_(8,9)_(4,6)_(5,7)_(3,4)_(5,6)_(7,8) }
    }
#endif
#if MAX_SORTED_SPLATS > 12
    else if (n <= 16) {
        if (n == 13) { _(0,11)_(1,7)_(2,4)_(3,5)_(8,9)_(10,12)_(0,2)_(3,6)_(4,12)_(5,7)_(8,10)_(0,8)_(1,3)_(2,5)_(4,9)_(6,11)_(7,12)_(0,1)_(2,10)_(3,8)_(4,6)_(9,11)_(1,3)_(2,4)_(5,10)_(6,8)_(7,9)_(11,12)_(1,2)_(3,4)_(5,8)_(6,9)_(7,10)_(2,3)_(4,7)_(5,6)_(8,11)_(9,10)_(4,5)_(6,7)_(8,9)_(10,11)_(3,4)_(5,6)_(7,8)_(9,10) }
        else if (n == 14) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(0,2)_(1,3)_(4,8)_(5,9)_(10,12)_(11,13)_(0,4)_(1,2)_(3,7)_(5,8)_(6,10)_(9,13)_(11,12)_(0,6)_(1,5)_(3,9)_(4,10)_(7,13)_(8,12)_(2,10)_(3,11)_(4,6)_(7,9)_(1,3)_(2,8)_(5,11)_(6,7)_(10,12)_(1,4)_(2,6)_(3,5)_(7,11)_(8,10)_(9,12)_(2,4)_(3,6)_(5,8)_(7,10)_(9,11)_(3,4)_(5,6)_(7,8)_(9,10)_(6,7) }
        else if (n == 15) { _(1,2)_(3,10)_(4,14)_(5,8)_(6,13)_(7,12)_(9,11)_(0,14)_(1,5)_(2,8)_(3,7)_(6,9)_(10,12)_(11,13)_(0,7)_(1,6)_(2,9)_(4,10)_(5,11)_(8,13)_(12,14)_(0,6)_(2,4)_(3,5)_(7,11)_(8,10)_(9,12)_(13,14)_(0,3)_(1,2)_(4,7)_(5,9)_(6,8)_(10,11)_(12,13)_(0,1)_(2,3)_(4,6)_(7,9)_(10,12)_(11,13)_(1,2)_(3,5)_(8,10)_(11,12)_(3,4)_(5,6)_(7,8)_(9,10)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(5,6)_(7,8) }
        else { _(0,13)_(1,12)_(2,15)_(3,14)_(4,8)_(5,6)_(7,11)_(9,10)_(0,5)_(1,7)_(2,9)_(3,4)_(6,13)_(8,14)_(10,15)_(11,12)_(0,1)_(2,3)_(4,5)_(6,8)_(7,9)_(10,11)_(12,13)_(14,15)_(0,2)_(1,3)_(4,10)_(5,11)_(6,7)_(8,9)_(12,14)_(13,15)_(1,2)_(3,12)_(4,6)_(5,7)_(8,10)_(9,11)_(13,14)_(1,4)_(2,6)_(5,8)_(7,10)_(9,13)_(11,14)_(2,4)_(3,6)_(9,12)_(11,13)_(3,5)_(6,8)_(7,9)_(10,12)_(3,4)_(5,6)_(7,8)_(9,10)_(11,12)_(6,7)_(8,9) }
    }
#endif
#if MAX_SORTED_SPLATS > 16
    else if (n <= 20) {
        if (n == 17) { _(0,11)_(1,15)_(2,10)_(3,5)_(4,6)_(8,12)_(9,16)_(13,14)_(0,6)_(1,13)_(2,8)_(4,14)_(5,15)_(7,11)_(0,8)_(3,7)_(4,9)_(6,16)_(10,11)_(12,14)_(0,2)_(1,4)_(5,6)_(7,13)_(8,9)_(10,12)_(11,14)_(15,16)_(0,3)_(2,5)_(6,11)_(7,10)_(9,13)_(12,15)_(14,16)_(0,1)_(3,4)_(5,10)_(6,9)_(7,8)_(11,15)_(13,14)_(1,2)_(3,7)_(4,8)_(6,12)_(11,13)_(14,15)_(1,3)_(2,7)_(4,5)_(9,11)_(10,12)_(13,14)_(2,3)_(4,6)_(5,7)_(8,10)_(3,4)_(6,8)_(7,9)_(10,12)_(5,6)_(7,8)_(9,10)_(11,12)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13) }
        else if (n == 18) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(0,2)_(1,3)_(4,12)_(5,13)_(6,8)_(9,11)_(14,16)_(15,17)_(0,14)_(1,16)_(2,15)_(3,17)_(0,6)_(1,10)_(2,9)_(7,16)_(8,15)_(11,17)_(1,4)_(3,9)_(5,7)_(8,14)_(10,12)_(13,16)_(0,1)_(2,5)_(3,13)_(4,14)_(7,9)_(8,10)_(12,15)_(16,17)_(1,2)_(3,5)_(4,6)_(11,13)_(12,14)_(15,16)_(4,8)_(5,12)_(6,10)_(7,11)_(9,13)_(1,4)_(2,8)_(3,6)_(5,7)_(9,15)_(10,12)_(11,14)_(13,16)_(2,4)_(5,8)_(6,10)_(7,11)_(9,12)_(13,15)_(3,5)_(6,8)_(7,10)_(9,11)_(12,14)_(3,4)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14) }
        else if (n == 19) { _(0,12)_(1,4)_(2,8)_(3,5)_(6,17)_(7,11)_(9,14)_(10,13)_(15,16)_(0,2)_(1,7)_(3,6)_(4,11)_(5,17)_(8,12)_(10,15)_(13,16)_(14,18)_(3,10)_(4,14)_(5,15)_(6,13)_(7,9)_(11,17)_(16,18)_(0,7)_(1,10)_(4,6)_(9,15)_(11,16)_(12,17)_(13,14)_(0,3)_(2,6)_(5,7)_(8,11)_(12,16)_(1,8)_(2,9)_(3,4)_(6,15)_(7,13)_(10,11)_(12,18)_(1,3)_(2,5)_(6,9)_(7,12)_(8,10)_(11,14)_(17,18)_(0,1)_(2,3)_(4,8)_(6,10)_(9,12)_(14,15)_(16,17)_(1,2)_(5,8)_(6,7)_(9,11)_(10,13)_(14,16)_(15,17)_(3,6)_(4,5)_(7,9)_(8,10)_(11,12)_(13,14)_(15,16)_(3,4)_(5,6)_(7,8)_(9,10)_(11,13)_(12,14)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15) }
        else { _(0,3)_(1,7)_(2,5)_(4,8)_(6,9)_(10,13)_(11,15)_(12,18)_(14,17)_(16,19)_(0,14)_(1,11)_(2,16)_(3,17)_(4,12)_(5,19)_(6,10)_(7,15)_(8,18)_(9,13)_(0,4)_(1,2)_(3,8)_(5,7)_(11,16)_(12,14)_(15,19)_(17,18)_(1,6)_(2,12)_(3,5)_(4,11)_(7,17)_(8,15)_(13,18)_(14,16)_(0,1)_(2,6)_(7,10)_(9,12)_(13,17)_(18,19)_(1,6)_(5,9)_(7,11)_(8,12)_(10,14)_(13,18)_(3,5)_(4,7)_(8,10)_(9,11)_(12,15)_(14,16)_(1,3)_(2,4)_(5,7)_(6,10)_(9,13)_(12,14)_(15,17)_(16,18)_(1,2)_(3,4)_(6,7)_(8,9)_(10,11)_(12,13)_(15,16)_(17,18)_(2,3)_(4,6)_(5,8)_(7,9)_(10,12)_(11,14)_(13,15)_(16,17)_(4,5)_(6,8)_(7,10)_(9,12)_(11,13)_(14,15)_(3,4)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14)_(15,16) }
    }
#endif
#if MAX_SORTED_SPLATS > 20
    else if (n <= 24) {
        if (n == 21) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(0,2)_(1,3)_(4,6)_(5,7)_(8,10)_(9,11)_(12,14)_(13,15)_(16,18)_(17,19)_(0,8)_(1,9)_(2,10)_(3,11)_(4,12)_(5,13)_(6,14)_(7,15)_(0,4)_(1,5)_(3,7)_(6,20)_(8,12)_(9,13)_(10,14)_(15,19)_(2,6)_(3,18)_(7,20)_(2,16)_(3,6)_(5,18)_(7,17)_(11,20)_(0,2)_(3,8)_(6,12)_(7,10)_(9,16)_(11,15)_(13,17)_(14,18)_(19,20)_(1,7)_(2,3)_(4,9)_(10,11)_(13,16)_(15,18)_(17,19)_(1,4)_(5,10)_(6,13)_(7,8)_(11,14)_(12,16)_(15,17)_(18,19)_(1,2)_(3,4)_(5,6)_(10,12)_(11,13)_(14,16)_(17,18)_(2,3)_(4,5)_(6,9)_(10,11)_(12,13)_(14,15)_(16,17)_(6,7)_(8,9)_(15,16)_(4,6)_(7,8)_(9,12)_(13,15)_(3,4)_(5,7)_(8,10)_(9,11)_(12,14)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14) }
        else if (n == 22) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21)_(0,2)_(1,3)_(4,6)_(5,7)_(8,12)_(9,13)_(14,16)_(15,17)_(18,20)_(19,21)_(0,4)_(1,5)_(2,6)_(3,7)_(8,10)_(9,12)_(11,13)_(14,18)_(15,19)_(16,20)_(17,21)_(0,14)_(1,15)_(2,18)_(3,19)_(4,16)_(5,17)_(6,20)_(7,21)_(9,11)_(10,12)_(2,8)_(3,11)_(6,9)_(10,18)_(12,15)_(13,19)_(0,2)_(1,10)_(3,16)_(5,18)_(6,14)_(7,15)_(8,12)_(9,13)_(11,20)_(19,21)_(2,6)_(3,10)_(4,8)_(5,12)_(9,16)_(11,18)_(13,17)_(15,19)_(1,4)_(7,13)_(8,14)_(9,12)_(17,20)_(1,2)_(3,8)_(4,6)_(7,11)_(10,14)_(13,18)_(15,17)_(19,20)_(2,4)_(5,10)_(7,9)_(11,16)_(12,14)_(17,19)_(5,6)_(7,8)_(9,11)_(10,12)_(13,14)_(15,16)_(3,5)_(6,7)_(8,10)_(9,12)_(11,13)_(14,15)_(16,18)_(3,4)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14)_(15,16)_(17,18) }
        else if (n == 23) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21)_(0,2)_(1,3)_(4,6)_(5,7)_(8,10)_(9,11)_(12,14)_(13,15)_(16,18)_(17,19)_(21,22)_(0,4)_(1,5)_(2,6)_(3,7)_(8,12)_(9,13)_(10,14)_(11,15)_(17,21)_(18,20)_(19,22)_(0,8)_(1,9)_(2,10)_(3,11)_(4,12)_(5,13)_(6,14)_(7,15)_(1,2)_(5,18)_(7,19)_(9,16)_(10,21)_(12,20)_(15,22)_(5,9)_(6,7)_(10,18)_(11,21)_(12,17)_(13,20)_(14,15)_(3,17)_(6,16)_(7,14)_(8,12)_(15,19)_(20,21)_(3,4)_(5,8)_(6,10)_(9,12)_(13,16)_(14,15)_(17,18)_(19,21)_(0,5)_(1,8)_(2,12)_(3,9)_(4,10)_(7,13)_(11,17)_(14,16)_(18,20)_(2,6)_(3,5)_(4,8)_(7,11)_(10,12)_(13,18)_(14,17)_(15,20)_(1,3)_(2,5)_(6,9)_(7,10)_(11,13)_(12,14)_(15,18)_(16,17)_(19,20)_(2,3)_(4,6)_(8,9)_(11,12)_(13,14)_(15,16)_(17,19)_(3,4)_(5,6)_(7,8)_(9,10)_(12,13)_(14,15)_(17,18)_(4,5)_(6,7)_(8,9)_(10,11)_(16,17) }
        else { _(0,20)_(1,12)_(2,16)_(3,23)_(4,6)_(5,10)_(7,21)_(8,14)_(9,15)_(11,22)_(13,18)_(17,19)_(0,3)_(1,11)_(2,7)_(4,17)_(5,13)_(6,19)_(8,9)_(10,18)_(12,22)_(14,15)_(16,21)_(20,23)_(0,1)_(2,4)_(3,12)_(5,8)_(6,9)_(7,10)_(11,20)_(13,16)_(14,17)_(15,18)_(19,21)_(22,23)_(2,5)_(4,8)_(6,11)_(7,14)_(9,16)_(12,17)_(15,19)_(18,21)_(1,8)_(3,14)_(4,7)_(9,20)_(10,12)_(11,13)_(15,22)_(16,19)_(0,7)_(1,5)_(3,4)_(6,11)_(8,15)_(9,14)_(10,13)_(12,17)_(16,23)_(18,22)_(19,20)_(0,2)_(1,6)_(4,7)_(5,9)_(8,10)_(13,15)_(14,18)_(16,19)_(17,22)_(21,23)_(2,3)_(4,5)_(6,8)_(7,9)_(10,11)_(12,13)_(14,16)_(15,17)_(18,19)_(20,21)_(1,2)_(3,6)_(4,10)_(7,8)_(9,11)_(12,14)_(13,19)_(15,16)_(17,20)_(21,22)_(2,3)_(5,10)_(6,7)_(8,9)_(13,18)_(14,15)_(16,17)_(20,21)_(3,4)_(5,7)_(10,12)_(11,13)_(16,18)_(19,20)_(4,6)_(8,10)_(9,12)_(11,14)_(13,15)_(17,19)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14)_(15,16)_(17,18) }
    }
#endif
#if MAX_SORTED_SPLATS > 24
    else if (n <= 28) {
        if (n == 25) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21)_(22,23)_(0,2)_(1,3)_(4,6)_(5,7)_(8,10)_(9,11)_(12,14)_(13,15)_(16,18)_(17,19)_(21,22)_(23,24)_(0,4)_(1,5)_(2,6)_(3,7)_(8,12)_(9,13)_(10,14)_(11,15)_(18,21)_(20,23)_(22,24)_(0,8)_(1,9)_(2,10)_(3,11)_(4,12)_(5,13)_(6,14)_(7,15)_(16,20)_(17,22)_(19,24)_(21,23)_(1,18)_(3,21)_(5,23)_(6,19)_(11,14)_(15,24)_(1,16)_(3,17)_(6,9)_(7,11)_(13,19)_(14,23)_(0,1)_(2,16)_(3,8)_(7,20)_(10,13)_(11,22)_(15,23)_(1,2)_(5,10)_(7,18)_(11,21)_(15,20)_(19,22)_(4,7)_(5,6)_(9,18)_(10,17)_(11,12)_(13,21)_(14,15)_(19,20)_(22,23)_(3,4)_(7,8)_(9,10)_(11,16)_(12,17)_(13,18)_(19,21)_(20,22)_(1,3)_(2,4)_(5,11)_(6,16)_(7,9)_(8,10)_(12,13)_(14,19)_(15,18)_(2,3)_(5,7)_(6,9)_(8,11)_(10,16)_(12,14)_(15,17)_(3,5)_(4,6)_(7,8)_(9,11)_(10,12)_(13,14)_(15,16)_(17,18)_(4,7)_(6,8)_(9,10)_(11,12)_(13,15)_(14,16)_(17,19)_(18,21)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21) }
        else if (n == 26) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21)_(22,23)_(24,25)_(0,2)_(1,3)_(4,6)_(5,7)_(8,10)_(9,11)_(14,16)_(15,17)_(18,20)_(19,21)_(22,24)_(23,25)_(0,4)_(1,6)_(2,5)_(3,7)_(8,14)_(9,16)_(10,15)_(11,17)_(18,22)_(19,24)_(20,23)_(21,25)_(0,18)_(1,19)_(2,20)_(3,21)_(4,22)_(5,23)_(6,24)_(7,25)_(9,12)_(13,16)_(3,11)_(8,9)_(10,13)_(12,15)_(14,22)_(16,17)_(0,8)_(1,9)_(2,14)_(6,12)_(7,15)_(10,18)_(11,23)_(13,19)_(16,24)_(17,25)_(1,2)_(3,18)_(4,8)_(7,22)_(17,21)_(23,24)_(3,14)_(4,10)_(5,18)_(7,20)_(8,13)_(11,22)_(12,17)_(15,21)_(1,4)_(5,6)_(7,9)_(8,10)_(15,17)_(16,18)_(19,20)_(21,24)_(2,5)_(3,10)_(6,14)_(9,13)_(11,19)_(12,16)_(15,22)_(20,23)_(2,8)_(5,7)_(6,9)_(11,12)_(13,14)_(16,19)_(17,23)_(18,20)_(2,4)_(3,5)_(6,11)_(7,10)_(9,16)_(12,13)_(14,19)_(15,18)_(20,22)_(21,23)_(3,4)_(5,8)_(6,7)_(9,11)_(10,12)_(13,15)_(14,16)_(17,20)_(18,19)_(21,22)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14)_(15,16)_(17,18)_(19,20)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21) }
        else if (n == 27) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,14)_(15,16)_(17,18)_(19,20)_(21,22)_(23,24)_(25,26)_(0,2)_(1,3)_(4,6)_(5,7)_(8,10)_(9,11)_(12,13)_(15,17)_(16,18)_(19,21)_(20,22)_(23,25)_(24,26)_(0,23)_(1,24)_(2,25)_(3,26)_(4,8)_(5,9)_(6,10)_(7,11)_(13,14)_(15,19)_(16,20)_(17,21)_(18,22)_(0,4)_(1,6)_(2,19)_(3,20)_(5,13)_(9,21)_(11,14)_(12,16)_(17,23)_(18,24)_(22,26)_(5,17)_(6,16)_(7,22)_(9,25)_(10,24)_(12,15)_(13,20)_(14,26)_(1,12)_(4,15)_(7,23)_(10,19)_(11,16)_(13,18)_(20,24)_(22,25)_(0,1)_(6,12)_(8,11)_(9,15)_(10,17)_(14,24)_(16,21)_(18,19)_(1,4)_(2,8)_(3,11)_(12,15)_(14,20)_(16,22)_(21,25)_(2,5)_(3,17)_(8,13)_(11,23)_(21,22)_(24,25)_(1,2)_(3,10)_(5,6)_(7,13)_(11,15)_(14,21)_(18,23)_(20,22)_(4,5)_(6,9)_(7,8)_(13,17)_(14,16)_(19,23)_(22,24)_(2,4)_(3,6)_(5,7)_(8,12)_(9,10)_(11,13)_(14,18)_(15,17)_(16,19)_(21,23)_(3,5)_(6,8)_(7,9)_(10,12)_(11,14)_(13,16)_(15,18)_(17,19)_(20,21)_(22,23)_(5,6)_(8,11)_(9,10)_(12,14)_(13,15)_(17,18)_(19,21)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,20)_(21,22)_(3,4)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14)_(15,16)_(17,18)_(19,20) }
        else { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21)_(22,23)_(24,25)_(26,27)_(0,2)_(1,3)_(4,6)_(5,7)_(8,10)_(9,11)_(12,14)_(13,15)_(16,18)_(17,19)_(20,22)_(21,23)_(24,26)_(25,27)_(0,4)_(1,5)_(2,6)_(3,7)_(8,12)_(9,13)_(14,18)_(15,19)_(20,24)_(21,25)_(22,26)_(23,27)_(0,20)_(1,21)_(2,22)_(3,23)_(4,24)_(5,25)_(6,26)_(7,27)_(9,17)_(10,18)_(11,15)_(12,16)_(1,2)_(4,20)_(5,6)_(7,23)_(8,12)_(9,16)_(10,14)_(11,18)_(13,17)_(15,19)_(21,22)_(25,26)_(0,8)_(1,9)_(2,12)_(3,5)_(4,10)_(6,16)_(7,13)_(11,21)_(14,20)_(15,25)_(17,23)_(18,26)_(19,27)_(22,24)_(2,4)_(3,7)_(5,17)_(8,14)_(9,11)_(10,22)_(13,19)_(16,18)_(20,24)_(23,25)_(1,8)_(3,9)_(5,11)_(6,10)_(7,15)_(12,20)_(16,22)_(17,21)_(18,24)_(19,26)_(1,2)_(4,6)_(5,9)_(10,16)_(11,17)_(12,14)_(13,15)_(18,22)_(21,23)_(25,26)_(4,8)_(6,12)_(7,11)_(10,14)_(13,17)_(15,21)_(16,20)_(19,23)_(2,4)_(6,8)_(7,16)_(9,14)_(10,12)_(11,20)_(13,18)_(15,17)_(19,21)_(23,25)_(3,10)_(5,12)_(7,9)_(11,13)_(14,16)_(15,22)_(17,24)_(18,20)_(3,6)_(5,8)_(7,10)_(9,12)_(11,14)_(13,16)_(15,18)_(17,20)_(19,22)_(21,24)_(3,4)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14)_(15,16)_(17,18)_(19,20)_(21,22)_(23,24) }
    }
    else {
        if (n == 29) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,28)_(17,26)_(18,25)_(19,23)_(21,27)_(22,24)_(0,2)_(1,3)_(4,6)_(5,7)_(8,10)_(9,11)_(12,14)_(13,15)_(17,22)_(18,19)_(20,27)_(23,25)_(24,26)_(0,4)_(1,5)_(2,6)_(3,7)_(8,12)_(9,13)_(10,14)_(11,15)_(16,20)_(17,18)_(19,22)_(23,24)_(25,26)_(27,28)_(0,8)_(1,9)_(2,10)_(3,11)_(4,12)_(5,13)_(6,14)_(7,15)_(16,18)_(20,22)_(21,25)_(24,27)_(26,28)_(1,8)_(2,24)_(4,19)_(5,20)_(6,21)_(7,27)_(9,18)_(10,23)_(11,26)_(13,22)_(14,25)_(15,28)_(16,17)_(0,6)_(2,4)_(3,24)_(5,10)_(12,19)_(13,18)_(14,21)_(15,25)_(20,23)_(26,27)_(0,16)_(1,6)_(3,12)_(4,8)_(5,17)_(7,24)_(14,20)_(15,26)_(18,21)_(19,23)_(25,27)_(1,5)_(2,16)_(3,10)_(6,9)_(7,18)_(8,17)_(11,19)_(13,14)_(15,22)_(21,23)_(25,26)_(1,2)_(3,5)_(4,8)_(6,16)_(7,11)_(9,17)_(10,12)_(14,20)_(15,18)_(19,24)_(22,27)_(4,6)_(9,16)_(10,13)_(11,19)_(12,14)_(20,21)_(22,26)_(23,24)_(2,4)_(3,6)_(7,16)_(8,9)_(11,17)_(15,19)_(18,23)_(24,25)_(3,4)_(5,9)_(7,10)_(11,13)_(12,16)_(14,17)_(15,20)_(19,21)_(22,24)_(5,8)_(6,7)_(9,12)_(10,11)_(13,14)_(15,16)_(17,20)_(18,19)_(21,23)_(24,25)_(5,6)_(7,8)_(9,10)_(11,12)_(13,15)_(14,16)_(17,18)_(19,20)_(21,22)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21)_(22,23) }
        else if (n == 30) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21)_(22,23)_(24,25)_(26,27)_(28,29)_(0,2)_(1,3)_(4,6)_(5,7)_(8,10)_(9,11)_(13,15)_(14,16)_(18,20)_(19,21)_(22,24)_(23,25)_(26,28)_(27,29)_(0,4)_(1,5)_(2,6)_(3,7)_(8,14)_(9,17)_(10,16)_(12,20)_(13,19)_(15,21)_(22,26)_(23,27)_(24,28)_(25,29)_(0,8)_(1,9)_(2,14)_(3,17)_(4,10)_(5,11)_(6,16)_(12,26)_(13,23)_(15,27)_(18,24)_(19,25)_(20,28)_(21,29)_(1,13)_(2,12)_(3,15)_(4,18)_(5,19)_(6,20)_(7,21)_(8,22)_(9,23)_(10,24)_(11,25)_(14,26)_(16,28)_(17,27)_(0,4)_(2,8)_(3,13)_(5,9)_(6,22)_(7,23)_(10,12)_(11,15)_(14,18)_(16,26)_(17,19)_(20,24)_(21,27)_(25,29)_(0,2)_(1,14)_(3,5)_(4,8)_(9,13)_(11,17)_(12,18)_(15,28)_(16,20)_(21,25)_(24,26)_(27,29)_(2,4)_(5,9)_(6,14)_(7,13)_(8,10)_(15,23)_(16,22)_(19,21)_(20,24)_(25,27)_(6,8)_(7,11)_(10,14)_(12,16)_(13,17)_(15,19)_(18,22)_(21,23)_(4,6)_(7,9)_(8,10)_(11,13)_(12,14)_(15,17)_(16,18)_(19,21)_(20,22)_(23,25)_(1,8)_(3,18)_(5,20)_(7,22)_(9,24)_(10,12)_(11,26)_(13,15)_(14,16)_(17,19)_(21,28)_(1,2)_(3,10)_(5,12)_(7,14)_(9,16)_(11,18)_(13,20)_(15,22)_(17,24)_(19,26)_(27,28)_(2,4)_(3,6)_(5,8)_(7,10)_(9,12)_(11,14)_(13,16)_(15,18)_(17,20)_(19,22)_(21,24)_(23,26)_(25,27)_(3,4)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14)_(15,16)_(17,18)_(19,20)_(21,22)_(23,24)_(25,26) }
        else if (n == 31) { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21)_(22,23)_(24,25)_(26,27)_(28,29)_(0,2)_(1,3)_(4,6)_(5,7)_(8,10)_(9,11)_(12,14)_(13,15)_(16,18)_(17,19)_(20,22)_(21,23)_(24,26)_(25,27)_(28,30)_(0,4)_(1,5)_(2,6)_(3,7)_(8,12)_(9,13)_(10,14)_(11,15)_(16,20)_(17,21)_(18,22)_(19,23)_(24,28)_(25,29)_(26,30)_(0,8)_(1,9)_(2,10)_(3,11)_(4,12)_(5,13)_(6,14)_(7,15)_(16,24)_(17,25)_(18,26)_(19,27)_(20,28)_(21,29)_(22,30)_(0,16)_(1,8)_(2,4)_(3,12)_(5,10)_(6,9)_(7,14)_(11,13)_(17,24)_(18,20)_(19,28)_(21,26)_(22,25)_(23,30)_(27,29)_(1,2)_(3,5)_(4,8)_(6,22)_(7,11)_(9,25)_(10,12)_(13,14)_(17,18)_(19,21)_(20,24)_(23,27)_(26,28)_(29,30)_(1,17)_(2,18)_(3,19)_(4,20)_(5,10)_(7,23)_(8,24)_(11,27)_(12,28)_(13,29)_(14,30)_(21,26)_(3,17)_(4,16)_(5,21)_(6,18)_(7,9)_(8,20)_(10,26)_(11,23)_(13,25)_(14,28)_(15,27)_(22,24)_(1,4)_(3,8)_(5,16)_(7,17)_(9,21)_(10,22)_(11,19)_(12,20)_(14,24)_(15,26)_(23,28)_(27,30)_(2,5)_(7,8)_(9,18)_(11,17)_(12,16)_(13,22)_(14,20)_(15,19)_(23,24)_(26,29)_(2,4)_(6,12)_(9,16)_(10,11)_(13,17)_(14,18)_(15,22)_(19,25)_(20,21)_(27,29)_(5,6)_(8,12)_(9,10)_(11,13)_(14,16)_(15,17)_(18,20)_(19,23)_(21,22)_(25,26)_(3,5)_(6,7)_(8,9)_(10,12)_(11,14)_(13,16)_(15,18)_(17,20)_(19,21)_(22,23)_(24,25)_(26,28)_(3,4)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14)_(15,16)_(17,18)_(19,20)_(21,22)_(23,24)_(25,26)_(27,28) }
        else { _(0,1)_(2,3)_(4,5)_(6,7)_(8,9)_(10,11)_(12,13)_(14,15)_(16,17)_(18,19)_(20,21)_(22,23)_(24,25)_(26,27)_(28,29)_(30,31)_(0,2)_(1,3)_(4,6)_(5,7)_(8,10)_(9,11)_(12,14)_(13,15)_(16,18)_(17,19)_(20,22)_(21,23)_(24,26)_(25,27)_(28,30)_(29,31)_(0,4)_(1,5)_(2,6)_(3,7)_(8,12)_(9,13)_(10,14)_(11,15)_(16,20)_(17,21)_(18,22)_(19,23)_(24,28)_(25,29)_(26,30)_(27,31)_(0,8)_(1,9)_(2,10)_(3,11)_(4,12)_(5,13)_(6,14)_(7,15)_(16,24)_(17,25)_(18,26)_(19,27)_(20,28)_(21,29)_(22,30)_(23,31)_(0,16)_(1,8)_(2,4)_(3,12)_(5,10)_(6,9)_(7,14)_(11,13)_(15,31)_(17,24)_(18,20)_(19,28)_(21,26)_(22,25)_(23,30)_(27,29)_(1,2)_(3,5)_(4,8)_(6,22)_(7,11)_(9,25)_(10,12)_(13,14)_(17,18)_(19,21)_(20,24)_(23,27)_(26,28)_(29,30)_(1,17)_(2,18)_(3,19)_(4,20)_(5,10)_(7,23)_(8,24)_(11,27)_(12,28)_(13,29)_(14,30)_(21,26)_(3,17)_(4,16)_(5,21)_(6,18)_(7,9)_(8,20)_(10,26)_(11,23)_(13,25)_(14,28)_(15,27)_(22,24)_(1,4)_(3,8)_(5,16)_(7,17)_(9,21)_(10,22)_(11,19)_(12,20)_(14,24)_(15,26)_(23,28)_(27,30)_(2,5)_(7,8)_(9,18)_(11,17)_(12,16)_(13,22)_(14,20)_(15,19)_(23,24)_(26,29)_(2,4)_(6,12)_(9,16)_(10,11)_(13,17)_(14,18)_(15,22)_(19,25)_(20,21)_(27,29)_(5,6)_(8,12)_(9,10)_(11,13)_(14,16)_(15,17)_(18,20)_(19,23)_(21,22)_(25,26)_(3,5)_(6,7)_(8,9)_(10,12)_(11,14)_(13,16)_(15,18)_(17,20)_(19,21)_(22,23)_(24,25)_(26,28)_(3,4)_(5,6)_(7,8)_(9,10)_(11,12)_(13,14)_(15,16)_(17,18)_(19,20)_(21,22)_(23,24)_(25,26)_(27,28) }
    }
#endif
    #undef _
}


template<typename val4>
inline __device__ void _pps_memcpy(int n, val4* src, val4* dst) {
#if 0
    for (int i = 0; i < n; i++)
        dst[i] = src[i];
#elif 0
    int m = n / 4;
    float4* src4 = (float4*)src;
    float4* dst4 = (float4*)dst;
    for (int i = 0; i < m; i++) {
        dst4[i] = src4[i];
    }
    for (int i = 4*m; i < n; i++) {
        dst[i] = src[i];
    }
#else
    int m = (n + 3) / 4;
    float4* src4 = (float4*)src;
    float4* dst4 = (float4*)dst;
    for (int i = 0; i < m; i++) {
        dst4[i] = src4[i];
    }
#endif
}

template <PerPixelSortType SORT_TYPE>
__global__ void sort_per_pixel_kernel(
    _ARGS_sort_per_pixel_kernel
) {
    unsigned i = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned j = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= img_height || j >= img_width)
        return;
    unsigned idx = i * img_width + j;
    unsigned tidx = threadIdx.y * blockDim.x + threadIdx.x;

    int n = num_intersects[idx];
    if (n <= 1)
        return;
    int32_t* indices_g = &indices_[idx*MAX_SORTED_SPLATS];
    float* depths_g = &depths_[idx*MAX_SORTED_SPLATS];

    __shared__ int32_t indices_s[MAX_SORTED_SPLATS*N_THREADS_PPS];
    __shared__ float depths_s[MAX_SORTED_SPLATS*N_THREADS_PPS];

    int32_t* indices = &indices_s[tidx*MAX_SORTED_SPLATS];
    float* depths = &depths_s[tidx*MAX_SORTED_SPLATS];
    _pps_memcpy<int32_t>(n, indices_g, indices);
    _pps_memcpy<float>(n, depths_g, depths);

    // int32_t* indices = indices_g;
    // float* depths = depths_g;

    switch (SORT_TYPE)
    {
    case PerPixelSortType::InsertionSort:
        sort_per_pixel_insertion(n, indices, depths);
        break;
    case PerPixelSortType::QuickSort:
        sort_per_pixel_quick(indices, depths, 0, n-1);
        break;
    case PerPixelSortType::HeapSort:
        sort_per_pixel_heap(n, indices, depths);
        break;
    case PerPixelSortType::RandomizedQuickSort:
        sort_per_pixel_randomized_quick(indices, depths, 0, n-1);
        break;
    case PerPixelSortType::NetworkSort:
        sort_per_pixel_network(n, indices, depths);
        break;
    }

    _pps_memcpy<int32_t>(n, indices, indices_g);
    _pps_memcpy<float>(n, depths, depths_g);
}



template<CameraType CAMERA_TYPE>
__global__ void rasterize_simple_sorted_forward_kernel(
    _ARGS_rasterize_simple_sorted_forward_kernel
) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= img_height || j >= img_width)
        return;
    int pix_id = i * img_width + j;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;

    glm::vec2 pos_2d = { (j + 0.5f - cx) / fx, (i + 0.5f - cy) / fy };
    if (CAMERA_TYPE == CameraType::GenericDistorted) {
        float2 pos_2d_u = undistortion_map[pix_id];
        if (isnan(pos_2d.x+pos_2d.y)) {
            out_img[pix_id] = {0.0f, 0.0f, 0.0f};
            out_alpha[pix_id] = 0.0f;
            return;
        }
        else
            pos_2d = { pos_2d_u.x, pos_2d_u.y };
    }

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];

    // current visibility left to render
    float T = 1.f;

    // rasterize
    float3 pix_out = {0.f, 0.f, 0.f};
    for (int cur_idx = 0; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        const float next_T = T * (1.f - alpha);

        const float vis = alpha * T;
        pix_out.x = pix_out.x + color.x * vis;
        pix_out.y = pix_out.y + color.y * vis;
        pix_out.z = pix_out.z + color.z * vis;
        T = next_T;
    }

    float3 final_color;
    final_color.x = pix_out.x + T * background.x;
    final_color.y = pix_out.y + T * background.y;
    final_color.z = pix_out.z + T * background.z;
    out_img[pix_id] = final_color;
    out_alpha[pix_id] = 1.0f - T;
}


__global__ void rasterize_simple_sorted_backward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int* __restrict__ num_intersects,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float3 __restrict__ background,
    const float* __restrict__ output_alpha,
    const float3* __restrict__ v_output,
    const float* __restrict__ v_output_alpha,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float* __restrict__ v_opacities
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;
    
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;
    
    int n = num_intersects[pix_id];
    if (n == 0) return;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // the contribution from gaussians behind the current one
    float3 buffer = {0.f, 0.f, 0.f};
    // index of last gaussian to contribute to this pixel

    // df/d_out for this pixel
    const float3 v_out = nan_to_num(v_output[pix_id]);
    const float v_out_alpha = nan_to_num(v_output_alpha[pix_id]);

    // rasterize
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    for (int cur_idx = n-1; cur_idx >= 0; cur_idx--) {
        int g_id = sorted_indices[cur_idx];

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        glm::vec3 v_position_local = {0.f, 0.f, 0.f};
        glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
        glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
        glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
        glm::vec3 v_color_local = {0.f, 0.f, 0.f};
        float v_opacity_local = 0.f;

        // compute the current T for this gaussian
        const float ra = 1.f / (1.f - alpha);
        const float next_T = T * ra;
        const float vis = alpha * next_T;

        // update v_rgb for this gaussian
        float v_alpha = 0.f;
        v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};

        // contribution from this pixel
        v_alpha += (color.x * T - buffer.x) * ra * v_out.x;
        v_alpha += (color.y * T - buffer.y) * ra * v_out.y;
        v_alpha += (color.z * T - buffer.z) * ra * v_out.z;

        v_alpha += T_final * ra * v_out_alpha;
        // contribution from background pixel
        v_alpha += -T_final * ra * background.x * v_out.x;
        v_alpha += -T_final * ra * background.y * v_out.y;
        v_alpha += -T_final * ra * background.z * v_out.z;
        // update the running sum
        buffer.x += color.x * vis;
        buffer.y += color.y * vis;
        buffer.z += color.z * vis;

        glm::vec2 v_uv;
        get_alpha_vjp(
            uv, opac,
            v_alpha, v_uv, v_opacity_local
        );
        glm::mat2x3 v_axis_uv = glm::mat2x3(0.0f);
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            glm::vec3(0), v_uv,
            v_position_local, v_axis_uv
        );
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local)) / glm::vec2(fx, fy);
        // v_position_xy_abs_local /= pos.z;
        v_axis_u_local = v_axis_uv[0];
        v_axis_v_local = v_axis_uv[1];

        T = next_T;

        float* v_position_ptr = (float*)(v_positions);
        atomicAdd(v_position_ptr + 3*g_id + 0, v_position_local.x);
        atomicAdd(v_position_ptr + 3*g_id + 1, v_position_local.y);
        atomicAdd(v_position_ptr + 3*g_id + 2, v_position_local.z);
        float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 0, v_position_xy_abs_local.x);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 1, v_position_xy_abs_local.y);

        float* v_axis_u_ptr = (float*)(v_axes_u);
        atomicAdd(v_axis_u_ptr + 3*g_id + 0, v_axis_u_local.x);
        atomicAdd(v_axis_u_ptr + 3*g_id + 1, v_axis_u_local.y);
        atomicAdd(v_axis_u_ptr + 3*g_id + 2, v_axis_u_local.z);
        float* v_axis_v_ptr = (float*)(v_axes_v);
        atomicAdd(v_axis_v_ptr + 3*g_id + 0, v_axis_v_local.x);
        atomicAdd(v_axis_v_ptr + 3*g_id + 1, v_axis_v_local.y);
        atomicAdd(v_axis_v_ptr + 3*g_id + 2, v_axis_v_local.z);

        float* v_color_ptr = (float*)(v_colors);
        atomicAdd(v_color_ptr + 3*g_id + 0, v_color_local.x);
        atomicAdd(v_color_ptr + 3*g_id + 1, v_color_local.y);
        atomicAdd(v_color_ptr + 3*g_id + 2, v_color_local.z);
        
        atomicAdd(v_opacities + g_id, v_opacity_local);
    }

}



template <DepthMode DEPTH_MODE, CameraType CAMERA_TYPE>
__global__ void rasterize_depth_sorted_forward_kernel(
    _ARGS_rasterize_depth_sorted_forward_kernel
) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= img_height || j >= img_width)
        return;
    int pix_id = i * img_width + j;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;

    glm::vec2 pos_2d = { (j + 0.5f - cx) / fx, (i + 0.5f - cy) / fy };
    if (CAMERA_TYPE == CameraType::GenericDistorted) {
        float2 pos_2d_u = undistortion_map[pix_id];
        if (isnan(pos_2d.x+pos_2d.y)) {
            out_depth[pix_id] = 0.0f;
            out_visibility[pix_id] = { 1.0f, 0.0f };
            return;
        }
        else
            pos_2d = { pos_2d_u.x, pos_2d_u.y };
    }

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];

    // current visibility left to render
    float T = 1.f;
    float interp = 1.0f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // rasterize
    float output_depth = 0.0f;

    for (; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        const float next_T = T * (1.f - alpha);

        // mean depth
        if (DEPTH_MODE == DepthMode::Mean) {

            // const float depth_raw = pos.z;
            const float depth_raw = poi.z;
            const float depth = depth_map(depth_raw);
            float vis = alpha * T;
            output_depth += vis * depth;

        }  // DEPTH_MODE

        // median depth
        else if (DEPTH_MODE == DepthMode::Median) {

            const float next_depth_raw = poi.z;
            const float next_depth = depth_map(next_depth_raw);
            if (next_T < DEPTH_REG_MEDIAN_TH) {
                if (T < 0.99999f) {
                    // https://www.desmos.com/3d/fttajoozww
                    interp = (1.0f-alpha)/alpha * (T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH;
                    interp = glm::clamp(interp, 0.0f, 1.0f);
                    output_depth = output_depth + (next_depth-output_depth)*interp;
                }
                else {
                    output_depth = next_depth;
                }
                T = next_T;
                cur_idx++;
                break;
            }
            output_depth = next_depth;

        }  // DEPTH_MODE

        T = next_T;
    }

    final_index[pix_id] = cur_idx;
    if (DEPTH_MODE == DepthMode::Mean) {
        float depth = T == 1.0f ? output_depth : output_depth / (1.0f-T);
        // out_depth[pix_id] = depth_inv_map(depth);
        out_depth[pix_id] = depth;
        out_visibility[pix_id] = {T, 1.0f-T};
    }
    else if (DEPTH_MODE == DepthMode::Median) {
        // out_depth[pix_id] = depth_inv_map(output_depth);
        out_depth[pix_id] = output_depth;
        out_visibility[pix_id] = {T, interp};
    }
}


template <DepthMode DEPTH_MODE>
__global__ void rasterize_depth_sorted_backward_kernel(
    _ARGS_rasterize_depth_sorted_backward_kernel
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;
    
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;
    
    int n = final_index[pix_id];
    if (n == 0) return;

    // this is the T AFTER the last gaussian in this pixel
    glm::vec2 meta_out = *(glm::vec2*)&out_visibility[pix_id];
    float T_final = meta_out.x;
    float T = T_final;
    float v_T = 0.0f;
    const float interp = meta_out.y;

    // df/d_out for this pixel
    float output_depth = out_depth[pix_id];
    float v_output_depth = nan_to_num(v_out_depth[pix_id]);
    float v_out_alpha = 0.0f;
    float v_depth = 0.f;
    float v_depth_next = 0.f;
    float v_alpha = 0.f;
    float v_interp = 0.f;
    if (DEPTH_MODE == DepthMode::Mean) {
        if (T != 1.0f) {
            float alpha = 1.0f-T;
            v_out_alpha = -output_depth / fmax(alpha, 1e-4f) * v_output_depth;
            output_depth *= alpha;
            v_output_depth /= fmax(alpha, 1e-4);
        }
    }

    float depth_buffer = 0.0f;

    // rasterize
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    for (int cur_idx = n-1; cur_idx >= 0; cur_idx--) {
        int g_id = sorted_indices[cur_idx];

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        glm::vec3 v_position_local = {0.f, 0.f, 0.f};
        glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
        glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
        glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
        float v_opacity_local = 0.f;

        // compute the current T for this gaussian
        const float ra = 1.f / (1.f - alpha);
        const float next_T = T * ra;
        const float vis = alpha * next_T;

        float depth_raw = poi.z;
        float depth = depth_map(depth_raw);

        // mean depth
        if (DEPTH_MODE == DepthMode::Mean) {

            v_depth = vis * v_output_depth;
            v_alpha = (depth * T - depth_buffer) * ra * v_output_depth;
            v_alpha += T_final * ra * v_out_alpha;
            depth_buffer += depth * vis;

        }  // DEPTH_MODE

        // median depth
        else if (DEPTH_MODE == DepthMode::Median) {

            // depth gradient
            if (T == T_final) {
                v_depth = v_output_depth * interp;
                v_depth_next = v_output_depth * (1.0f-interp);
            }
            else {
                v_depth = v_depth_next;
                v_depth_next = 0.0f;
            }

            // alpha gradient
            if (T == T_final && interp < 1.0f && interp > 0.0f) {
                float depth_0 = (output_depth-depth*interp) / (1.0f-interp);
                v_interp = (depth-depth_0) * v_output_depth;
                v_alpha = (next_T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH * \
                    v_interp / safe_denom(-alpha*alpha, 1e-3);
                v_T = (1.0f-alpha)/alpha * v_interp / DEPTH_REG_MEDIAN_TH;
            }
            else {
                v_alpha = v_T * (-next_T);
                v_T = v_T * (1.0f-alpha);
            }

        }  // DEPTH_MODE

        T = next_T;

        // backward
        glm::vec2 v_uv;
        get_alpha_vjp(
            uv, opac,
            v_alpha, v_uv, v_opacity_local
        );
        glm::mat2x3 v_axis_uv = glm::mat2x3(0.0f);
        float v_depth_raw = depth_map_vjp(depth_raw, v_depth);
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            {0.f, 0.f, v_depth_raw}, v_uv,
            v_position_local, v_axis_uv
        );
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local)) / glm::vec2(fx, fy);
        v_axis_u_local = v_axis_uv[0];
        v_axis_v_local = v_axis_uv[1];
        
        float* v_position_ptr = (float*)(v_positions);
        atomicAdd(v_position_ptr + 3*g_id + 0, v_position_local.x);
        atomicAdd(v_position_ptr + 3*g_id + 1, v_position_local.y);
        atomicAdd(v_position_ptr + 3*g_id + 2, v_position_local.z);
        float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 0, v_position_xy_abs_local.x);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 1, v_position_xy_abs_local.y);

        float* v_axis_u_ptr = (float*)(v_axes_u);
        atomicAdd(v_axis_u_ptr + 3*g_id + 0, v_axis_u_local.x);
        atomicAdd(v_axis_u_ptr + 3*g_id + 1, v_axis_u_local.y);
        atomicAdd(v_axis_u_ptr + 3*g_id + 2, v_axis_u_local.z);
        float* v_axis_v_ptr = (float*)(v_axes_v);
        atomicAdd(v_axis_v_ptr + 3*g_id + 0, v_axis_v_local.x);
        atomicAdd(v_axis_v_ptr + 3*g_id + 1, v_axis_v_local.y);
        atomicAdd(v_axis_v_ptr + 3*g_id + 2, v_axis_v_local.z);
        
        float v_opacity_local_ = (float)v_opacity_local;
        atomicAdd(v_opacities + g_id, v_opacity_local_);
    }
}




__global__ void rasterize_sorted_forward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const float depth_reg_pairwise_factor,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float3* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    // const float3 __restrict__ background,
    const float* __restrict__ depth_ref_im,
    float* __restrict__ out_alpha,
    float3* __restrict__ out_img,
    float2* __restrict__ out_depth,
    float3* __restrict__ out_normal,
    float* __restrict__ out_reg_depth
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);

    float T = 1.f;  // current/total visibility
    float3 normal_out = {0.f, 0.f, 0.f};  // sum of normals
    float3 pix_out = {0.f, 0.f, 0.f};  // output radiance
    float vis_sum = 0.f;  // output alpha
    float depth_sum = 0.f;  // output depth
    float depth_squared_sum = 0.f;  // for L2 depth regularizer
    const float depth_ref = inside ? depth_ref_im[pix_id] : 0.f;
    float reg_depth_p = 0.f, reg_depth_i = 0.f;  // output depth regularizer

    // rasterize
    for (int cur_idx = 0; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color_0 = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        const float next_T = T * (1.f - alpha);

        glm::vec3 color;
        if (dim_ch > 0) {
            const glm::vec3* coeffs = (glm::vec3*)&ch_coeffs[dim_ch*g_id];
            glm::vec3 ch_color = ch_coeffs_to_color(
                ch_degree_r, ch_degree_r_to_use,
                ch_degree_phi, ch_degree_phi_to_use,
                coeffs, {uv.x, uv.y}
            );
            color = color_0 / (1.0f+glm::exp(-ch_color));
        }
        else color = color_0;

        const float vis = alpha * T;
        #if DEPTH_REG_L == 01 && false
        const float depth_raw = pos.z;
        #else
        const float depth_raw = poi.z;
        #endif
        const float depth = depth_map(depth_raw);

        pix_out.x = pix_out.x + color.x * vis;
        pix_out.y = pix_out.y + color.y * vis;
        pix_out.z = pix_out.z + color.z * vis;

        // depth regularization
        {
            // float pairwise_l1 = vis*depth * vis_sum - vis * depth_sum;  // requires pos.z for depth
            float pairwise_l2 = vis * (vis_sum*depth*depth + depth_squared_sum - 2.0f*depth*depth_sum);
            float intersect_l1 = vis * abs(depth - depth_ref);
            // float intersect_l2 = vis * (depth-depth_ref) * (depth-depth_ref);
            reg_depth_p += pairwise_l2;
            reg_depth_i += intersect_l1;
        }
        vis_sum += vis;
        depth_sum += vis*depth;
        depth_squared_sum += vis*depth*depth;

        // normal regularization
        glm::vec3 normal = get_normal_from_axisuv(axis_uv, poi);
        normal_out.x = normal_out.x + normal.x * vis;
        normal_out.y = normal_out.y + normal.y * vis;
        normal_out.z = normal_out.z + normal.z * vis;

        T = next_T;
    }

    if (inside) {
        out_alpha[pix_id] = 1.0f - T;
        float3 final_color;
        // final_color.x = pix_out.x + T * background.x;
        // final_color.y = pix_out.y + T * background.y;
        // final_color.z = pix_out.z + T * background.z;
        final_color.x = pix_out.x;
        final_color.y = pix_out.y;
        final_color.z = pix_out.z;
        out_img[pix_id] = final_color;
        out_depth[pix_id] = {depth_sum, depth_squared_sum};
        out_normal[pix_id] = normal_out;
        out_reg_depth[pix_id] = reg_depth_i + (reg_depth_p-reg_depth_i) * depth_reg_pairwise_factor;
    }
}


__global__ void rasterize_sorted_backward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const unsigned ch_degree_r,
    const unsigned ch_degree_r_to_use,
    const unsigned ch_degree_phi,
    const unsigned ch_degree_phi_to_use,
    const float depth_reg_pairwise_factor,
    const int* __restrict__ num_intersects,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float3* __restrict__ ch_coeffs,
    const float* __restrict__ opacities,
    // const float3 __restrict__ background,
    const float* __restrict__ depth_ref_im,
    const float* __restrict__ output_alpha,
    const float2* __restrict__ output_depth,
    const float* __restrict__ v_output_alpha,
    const float3* __restrict__ v_output,
    const float2* __restrict__ v_output_depth,
    const float3* __restrict__ v_output_normal,
    const float* __restrict__ v_output_reg_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float3* __restrict__ v_ch_coeffs,
    // float* __restrict__ v_ch_coeffs_abs,
    float* __restrict__ v_opacities,
    // float3* __restrict__ v_background,
    float* __restrict__ v_depth_ref_im
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;
    
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;
    
    int n = num_intersects[pix_id];
    if (n == 0) return;

    const int dim_ch = ch_degree_r * (2*ch_degree_phi+1);
    assert(dim_ch <= MAX_CH_FLOAT3);

    // df/d_out for this pixel
    const float2 out_depth = output_depth[pix_id];
    const float3 v_out = nan_to_num(v_output[pix_id]);
    const float2 v_out_depth = nan_to_num(v_output_depth[pix_id]);
    const float3 v_out_normal = nan_to_num(v_output_normal[pix_id]);
    const float v_out_alpha = nan_to_num(v_output_alpha[pix_id]);
    const float v_out_reg_depth = nan_to_num(v_output_reg_depth[pix_id]);
    const float v_reg_depth_p = v_out_reg_depth * depth_reg_pairwise_factor;
    const float v_reg_depth_i = v_out_reg_depth * (1.0f-depth_reg_pairwise_factor);
    const float v_depth_sum = v_out_depth.x;
    const float v_depth_squared_sum = v_out_depth.y;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;

    // regularization
    const float depth_ref = inside ? depth_ref_im[pix_id] : 0.f;
    float v_depth_ref = 0.f;

    const float vis_sum_final = 1.0f - T_final;
    const float depth_sum_final = out_depth.x;
    const float depth_squared_sum_final = out_depth.y;
    float vis_sum = vis_sum_final;
    float depth_sum = depth_sum_final;
    float depth_squared_sum = depth_squared_sum_final;

    float3 buffer = {0.f, 0.f, 0.f};
    float2 buffer_depth = {0.f, 0.f};
    float3 buffer_normal = {0.f, 0.f, 0.f};
    float buffer_depth_reg = 0.f;
    
    // rasterize
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    for (int cur_idx = n-1; cur_idx >= 0; cur_idx--) {
        int g_id = sorted_indices[cur_idx];

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color_0 = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        glm::vec3 v_position_local = {0.f, 0.f, 0.f};
        glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
        glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
        glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
        glm::vec3 v_color_local = {0.f, 0.f, 0.f};
        float v_opacity_local = 0.f;
        glm::vec3 v_ch_coeff_local[MAX_CH_FLOAT3];
        for (int i = 0; i < dim_ch; i++)
            v_ch_coeff_local[i] = {0.f, 0.f, 0.f};
        float v_ch_coeff_abs_local = 0.f;
        //initialize everything to 0, only set if the lane is valid

        // compute the current T for this gaussian
        const float ra = 1.f / (1.f - alpha);
        const float next_T = T * ra;
        const float vis = alpha * next_T;

        // update accumulation
        float v_depth = 0.0f;
        #if DEPTH_REG_L == 01 && false
        const float depth_raw = pos.z;
        const float depth = depth_map(depth_raw);
        v_depth += vis * v_depth_sum;
        v_depth += vis * 2.0f*depth * v_depth_squared_sum;
        #else
        const float depth_raw = poi.z;
        const float depth = depth_map(depth_raw);
        v_depth += vis * v_depth_sum;
        v_depth += vis * 2.0f*depth * v_depth_squared_sum;
        #endif

        // update depth regularizer
        float vis_sum_next = vis_sum - vis;
        float depth_sum_next = depth_sum - vis*depth;
        float depth_squared_sum_next = depth_squared_sum - vis*depth*depth;
        #if 0  // pairwise L1, requires pos.z for depth
        v_depth += v_reg_depth_p * vis * (vis_sum_next - (vis_sum_final-vis_sum));
        float reg_depth_i_p = (
            depth * vis_sum_next - depth_sum_next +
            (depth_sum_final-depth_sum) - depth * (vis_sum_final-vis_sum)
        );
        v_position_local.z = depth_map_vjp(depth_raw, v_depth);
        #else  // pairwise L2
        v_depth += v_reg_depth_p * vis * 2.0f * (
            vis_sum_final * depth - depth_sum_final);
        float reg_depth_i_p =
            vis_sum_final*depth*depth + depth_squared_sum_final
            - 2.0f*depth*depth_sum_final;
        #endif
        #if 1  // L1 with intersected depth
        float v_z = v_reg_depth_i * vis * glm::sign(depth-depth_ref);
        v_depth += v_z;
        v_depth_ref += (-v_z);
        float reg_depth_i_i = abs(depth-depth_ref);
        #else  // L2 with intersected depth
        float v_z = v_reg_depth_i * vis * 2.0f*(depth-depth_ref);
        v_depth += v_z;
        v_depth_ref += (-v_z);
        float reg_depth_i_i = (depth-depth_ref) * (depth-depth_ref);
        #endif
        float reg_depth_i = reg_depth_i_i + (reg_depth_i_p-reg_depth_i_i) * depth_reg_pairwise_factor;

        float v_depth_raw = depth_map_vjp(depth_raw, v_depth);
        glm::vec3 v_poi = {0.f, 0.f, v_depth_raw};

        // normal regularization
        glm::vec3 v_normal = {vis * v_out_normal.x, vis * v_out_normal.y, vis * v_out_normal.z};
        glm::mat2x3 v_axis_uv; glm::vec3 normal;
        get_normal_from_axisuv_vjp(axis_uv, poi, v_normal, normal, v_axis_uv);

        // update color
        glm::vec3 v_color_1 = {vis * v_out.x, vis * v_out.y, vis * v_out.z};
        glm::vec3 color_1;
        glm::vec2 v_uv_ch = {0.f, 0.f};
        if (dim_ch > 0) {
            glm::vec3 v_ch_color_sigmoid = v_color_1 * color_0;
            #if 0
            int32_t g_id = id_batch[t];
            glm::vec3 ch_color = ch_coeffs_to_color(
                ch_degree_r, ch_degree_r_to_use,
                ch_degree_phi, ch_degree_phi_to_use,
                (glm::vec3*)&ch_coeffs[dim_ch*g_id], {uv.x, uv.y}
            );
            glm::vec3 ch_color_sigmoid = 1.0f / (1.0f+glm::exp(-ch_color));
            glm::vec3 v_ch_color = v_ch_color_sigmoid * ch_color_sigmoid*(1.0f-ch_color_sigmoid);
            ch_coeffs_to_color_vjp(
                ch_degree_r, ch_degree_r_to_use,
                ch_degree_phi, ch_degree_phi_to_use,
                (glm::vec3*)&ch_coeffs[dim_ch*g_id],
                {uv.x, uv.y},
                v_ch_color,
                ch_color,
                v_ch_coeff_local, v_ch_coeff_abs_local,
                v_uv_ch
            );
            #else
            // makes overall training 0.1x faster
            glm::vec3 ch_color_sigmoid;
            ch_coeffs_to_color_sigmoid_vjp(
                ch_degree_r, ch_degree_r_to_use,
                ch_degree_phi, ch_degree_phi_to_use,
                (glm::vec3*)&ch_coeffs[dim_ch*g_id],
                {uv.x, uv.y},
                v_ch_color_sigmoid,
                ch_color_sigmoid,
                v_ch_coeff_local, v_ch_coeff_abs_local,
                v_uv_ch
            );
            #endif
            color_1 = color_0 * ch_color_sigmoid;
            v_color_local = v_color_1 * ch_color_sigmoid;
        }
        else {
            color_1 = color_0;
            v_color_local = v_color_1;
        }

        float v_alpha = 0.0f;
        // contribution from this pixel
        v_alpha += (color_1.x * T - buffer.x) * ra * v_out.x;
        v_alpha += (color_1.y * T - buffer.y) * ra * v_out.y;
        v_alpha += (color_1.z * T - buffer.z) * ra * v_out.z;
        v_alpha += T_final * ra * v_out_alpha;
        // v_alpha += -T_final * ra * background.x * v_out.x;
        // v_alpha += -T_final * ra * background.y * v_out.y;
        // v_alpha += -T_final * ra * background.z * v_out.z;
        v_alpha += (depth * T - buffer_depth.x) * ra * v_depth_sum;
        v_alpha += (depth*depth * T - buffer_depth.y) * ra * v_depth_squared_sum;
        v_alpha += (reg_depth_i * T - buffer_depth_reg) * ra * v_out_reg_depth;
        v_alpha += (normal.x * T - buffer_normal.x) * ra * v_out_normal.x;
        v_alpha += (normal.y * T - buffer_normal.y) * ra * v_out_normal.y;
        v_alpha += (normal.z * T - buffer_normal.z) * ra * v_out_normal.z;

        // update the running sum
        buffer.x += color_1.x * vis;
        buffer.y += color_1.y * vis;
        buffer.z += color_1.z * vis;
        buffer_depth.x += depth * vis;
        buffer_depth.y += depth*depth * vis;
        buffer_depth_reg += reg_depth_i * vis;
        buffer_normal.x += normal.x * vis;
        buffer_normal.y += normal.y * vis;
        buffer_normal.z += normal.z * vis;

        // grad
        glm::vec2 v_uv;
        get_alpha_vjp(
            uv, opac,
            v_alpha, v_uv, v_opacity_local
        );
        v_uv += v_uv_ch;
        glm::vec3 v_position_local_temp;
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            v_poi, v_uv,
            v_position_local_temp, v_axis_uv
        );
        v_position_local += v_position_local_temp;
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local)) / glm::vec2(fx, fy);
        v_axis_u_local = v_axis_uv[0];
        v_axis_v_local = v_axis_uv[1];

        // next loop
        T = next_T;
        vis_sum = vis_sum_next;
        depth_sum = depth_sum_next;
        depth_squared_sum = depth_squared_sum_next;

        float* v_position_ptr = (float*)(v_positions);
        atomicAdd(v_position_ptr + 3*g_id + 0, v_position_local.x);
        atomicAdd(v_position_ptr + 3*g_id + 1, v_position_local.y);
        atomicAdd(v_position_ptr + 3*g_id + 2, v_position_local.z);
        float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 0, v_position_xy_abs_local.x);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 1, v_position_xy_abs_local.y);

        float* v_axis_u_ptr = (float*)(v_axes_u);
        atomicAdd(v_axis_u_ptr + 3*g_id + 0, v_axis_u_local.x);
        atomicAdd(v_axis_u_ptr + 3*g_id + 1, v_axis_u_local.y);
        atomicAdd(v_axis_u_ptr + 3*g_id + 2, v_axis_u_local.z);
        float* v_axis_v_ptr = (float*)(v_axes_v);
        atomicAdd(v_axis_v_ptr + 3*g_id + 0, v_axis_v_local.x);
        atomicAdd(v_axis_v_ptr + 3*g_id + 1, v_axis_v_local.y);
        atomicAdd(v_axis_v_ptr + 3*g_id + 2, v_axis_v_local.z);
        
        float* v_color_ptr = (float*)(v_colors);
        atomicAdd(v_color_ptr + 3*g_id + 0, v_color_local.x);
        atomicAdd(v_color_ptr + 3*g_id + 1, v_color_local.y);
        atomicAdd(v_color_ptr + 3*g_id + 2, v_color_local.z);
        float* v_ch_coeffs_ptr = (float*)(v_ch_coeffs);
        for (int i = 0; i < dim_ch; i++) {
            atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g_id + 3*i + 0, v_ch_coeff_local[i].x);
            atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g_id + 3*i + 1, v_ch_coeff_local[i].y);
            atomicAdd(v_ch_coeffs_ptr + 3*dim_ch*g_id + 3*i + 2, v_ch_coeff_local[i].z);
        }
        // atomicAdd(v_ch_coeffs_abs + g, v_ch_coeff_abs_local);

        atomicAdd(v_opacities + g_id, v_opacity_local);
    }

    if (inside) {
        v_depth_ref_im[pix_id] = v_depth_ref;

        // background gradient
        #if 0
        float3 v_bkg = {
            v_out.x * T_final,
            v_out.y * T_final,
            v_out.z * T_final
        };
        atomicAdd((float*)v_background+0, v_bkg.x);
        atomicAdd((float*)v_background+1, v_bkg.y);
        atomicAdd((float*)v_background+2, v_bkg.z);
        #endif
    }

}



template<CameraType CAMERA_TYPE>
__global__ void rasterize_simplified_sorted_forward_kernel(
    _ARGS_rasterize_simplified_sorted_forward_kernel
) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= img_height || j >= img_width)
        return;
    int pix_id = i * img_width + j;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;

    glm::vec2 pos_2d = { (j + 0.5f - cx) / fx, (i + 0.5f - cy) / fy };
    if (CAMERA_TYPE == CameraType::GenericDistorted) {
        float2 pos_2d_u = undistortion_map[pix_id];
        if (isnan(pos_2d.x+pos_2d.y)) {
            out_alpha[pix_id] = 0.0f;
            out_img[pix_id] = {0.0f, 0.0f, 0.0f};
            out_depth[pix_id] = {0.0f, 0.0f};
            out_normal[pix_id] = {0.0f, 0.0f, 0.0f};
            out_depth_reg[pix_id] = 0.0f;
            return;
        }
        else
            pos_2d = { pos_2d_u.x, pos_2d_u.y };
    }

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];

    float T = 1.f;  // current/total visibility
    float3 normal_out = {0.f, 0.f, 0.f};  // sum of normals
    float3 pix_out = {0.f, 0.f, 0.f};  // output radiance
    float vis_sum = 0.f;  // output alpha
    float depth_sum = 0.f;  // output depth
    float depth_squared_sum = 0.f;  // for L2 depth regularizer
    float reg_depth_p = 0.f;  // output depth regularizer

    // rasterize
    for (int cur_idx = 0; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        const float next_T = T * (1.f - alpha);
        const float vis = alpha * T;

        // color
        pix_out.x = pix_out.x + color.x * vis;
        pix_out.y = pix_out.y + color.y * vis;
        pix_out.z = pix_out.z + color.z * vis;

        // depth regularization
        const float depth_raw = poi.z;
        const float depth = depth_map(depth_raw);
        {
            // float pairwise_l1 = vis*depth * vis_sum - vis * depth_sum;  // requires pos.z for depth
            float pairwise_l2 = vis * (vis_sum*depth*depth + depth_squared_sum - 2.0f*depth*depth_sum);
            reg_depth_p += pairwise_l2;
        }
        vis_sum += vis;
        depth_sum += vis*depth;
        depth_squared_sum += vis*depth*depth;

        // normal regularization
        glm::vec3 normal = get_normal_from_axisuv(axis_uv, poi);
        normal_out.x = normal_out.x + normal.x * vis;
        normal_out.y = normal_out.y + normal.y * vis;
        normal_out.z = normal_out.z + normal.z * vis;

        T = next_T;
    }

    out_alpha[pix_id] = 1.0f - T;
    out_img[pix_id] = pix_out;
    out_depth[pix_id] = { depth_sum, depth_squared_sum };
    out_normal[pix_id] = normal_out;
    out_depth_reg[pix_id] = reg_depth_p;
}


template<CameraType CAMERA_TYPE>
__global__ void rasterize_simplified_sorted_backward_kernel(
    _ARGS_rasterize_simplified_sorted_backward_kernel
) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= img_height || j >= img_width)
        return;
    int pix_id = i * img_width + j;

    int n = num_intersects[pix_id];
    if (n == 0) return;
    
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;

    glm::vec2 pos_2d = { (j + 0.5f - cx) / fx, (i + 0.5f - cy) / fy };
    if (CAMERA_TYPE == CameraType::GenericDistorted) {
        float2 pos_2d_u = undistortion_map[pix_id];
        if (isnan(pos_2d.x+pos_2d.y))
            return;
        else
            pos_2d = { pos_2d_u.x, pos_2d_u.y };
    }

    // df/d_out for this pixel
    const float3 v_out = nan_to_num(v_output_img[pix_id]);
    const float2 v_out_depth = nan_to_num(v_output_depth[pix_id]);
    const float3 v_out_normal = nan_to_num(v_output_normal[pix_id]);
    const float v_out_alpha = nan_to_num(v_output_alpha[pix_id]);
    const float v_reg_depth_p = nan_to_num(v_output_depth_reg[pix_id]);
    const float v_depth_sum = v_out_depth.x;
    const float v_depth_squared_sum = v_out_depth.y;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;

    const float vis_sum_final = 1.0f - T_final;
    const float depth_sum_final = output_depth[pix_id].x;
    const float depth_squared_sum_final = output_depth[pix_id].y;
    float vis_sum = vis_sum_final;

    float3 buffer = {0.f, 0.f, 0.f};
    float2 buffer_depth = {0.f, 0.f};  // depth, depth^2
    float3 buffer_normal = {0.f, 0.f, 0.f};
    float buffer_depth_reg = 0.f;

    // rasterize
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    for (int cur_idx = n-1; cur_idx >= 0; cur_idx--) {
        int g_id = sorted_indices[cur_idx];

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        glm::vec3 v_position_local = {0.f, 0.f, 0.f};
        glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
        glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
        glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
        glm::vec3 v_color_local = {0.f, 0.f, 0.f};
        float v_opacity_local = 0.f;
        //initialize everything to 0, only set if the lane is valid

        // compute the current T for this gaussian
        const float ra = 1.f / (1.f - alpha);
        const float next_T = T * ra;
        const float vis = alpha * next_T;

        // update accumulation
        const float depth_raw = poi.z;
        const float depth = depth_map(depth_raw);
        float v_depth = 0.0f;
        v_depth += vis * v_depth_sum;
        v_depth += vis * 2.0f*depth * v_depth_squared_sum;

        // update depth regularizer
        float vis_sum_next = vis_sum - vis;
        // pairwise L2
        v_depth += v_reg_depth_p * vis * 2.0f * (
            vis_sum_final * depth - depth_sum_final);
        float reg_depth_i =
            vis_sum_final*depth*depth + depth_squared_sum_final
            - 2.0f*depth*depth_sum_final;

        float v_depth_raw = depth_map_vjp(depth_raw, v_depth);
        glm::vec3 v_poi = {0.f, 0.f, v_depth_raw};

        // update color
        v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};

        // normal regularization
        glm::vec3 v_normal = {vis * v_out_normal.x, vis * v_out_normal.y, vis * v_out_normal.z};
        glm::mat2x3 v_axis_uv; glm::vec3 normal;
        get_normal_from_axisuv_vjp(axis_uv, poi, v_normal, normal, v_axis_uv);

        float v_alpha = 0.0f;
        // contribution from this pixel
        v_alpha += (color.x * T - buffer.x) * ra * v_out.x;
        v_alpha += (color.y * T - buffer.y) * ra * v_out.y;
        v_alpha += (color.z * T - buffer.z) * ra * v_out.z;
        v_alpha += T_final * ra * v_out_alpha;
        v_alpha += (depth * T - buffer_depth.x) * ra * v_depth_sum;
        v_alpha += (depth*depth * T - buffer_depth.y) * ra * v_depth_squared_sum;
        v_alpha += (reg_depth_i * T - buffer_depth_reg) * ra * v_reg_depth_p;
        v_alpha += (normal.x * T - buffer_normal.x) * ra * v_out_normal.x;
        v_alpha += (normal.y * T - buffer_normal.y) * ra * v_out_normal.y;
        v_alpha += (normal.z * T - buffer_normal.z) * ra * v_out_normal.z;

        // update the running sum
        buffer.x += color.x * vis;
        buffer.y += color.y * vis;
        buffer.z += color.z * vis;
        buffer_depth.x += depth * vis;
        buffer_depth.y += depth*depth * vis;
        buffer_depth_reg += reg_depth_i * vis;
        buffer_normal.x += normal.x * vis;
        buffer_normal.y += normal.y * vis;
        buffer_normal.z += normal.z * vis;

        // grad
        glm::vec2 v_uv;
        get_alpha_vjp(
            uv, opac,
            v_alpha, v_uv, v_opacity_local
        );
        glm::vec3 v_position_local_temp;
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            v_poi, v_uv,
            v_position_local_temp, v_axis_uv
        );
        v_position_local += v_position_local_temp;
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local)) / glm::vec2(fx, fy);
        v_axis_u_local = v_axis_uv[0];
        v_axis_v_local = v_axis_uv[1];

        // next loop
        T = next_T;
        vis_sum = vis_sum_next;

        float* v_position_ptr = (float*)(v_positions);
        atomicAdd(v_position_ptr + 3*g_id + 0, v_position_local.x);
        atomicAdd(v_position_ptr + 3*g_id + 1, v_position_local.y);
        atomicAdd(v_position_ptr + 3*g_id + 2, v_position_local.z);
        float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 0, v_position_xy_abs_local.x);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 1, v_position_xy_abs_local.y);

        float* v_axis_u_ptr = (float*)(v_axes_u);
        atomicAdd(v_axis_u_ptr + 3*g_id + 0, v_axis_u_local.x);
        atomicAdd(v_axis_u_ptr + 3*g_id + 1, v_axis_u_local.y);
        atomicAdd(v_axis_u_ptr + 3*g_id + 2, v_axis_u_local.z);
        float* v_axis_v_ptr = (float*)(v_axes_v);
        atomicAdd(v_axis_v_ptr + 3*g_id + 0, v_axis_v_local.x);
        atomicAdd(v_axis_v_ptr + 3*g_id + 1, v_axis_v_local.y);
        atomicAdd(v_axis_v_ptr + 3*g_id + 2, v_axis_v_local.z);
        
        float* v_color_ptr = (float*)(v_colors);
        atomicAdd(v_color_ptr + 3*g_id + 0, v_color_local.x);
        atomicAdd(v_color_ptr + 3*g_id + 1, v_color_local.y);
        atomicAdd(v_color_ptr + 3*g_id + 2, v_color_local.z);

        atomicAdd(v_opacities + g_id, v_opacity_local);
    }

}




template __global__ void rasterize_indices_kernel<CameraType::Undistorted>(
    _ARGS_rasterize_indices_kernel
);
template __global__ void rasterize_indices_kernel<CameraType::GenericDistorted>(
    _ARGS_rasterize_indices_kernel
);

template __global__ void sort_per_pixel_kernel<PerPixelSortType::InsertionSort>(
    _ARGS_sort_per_pixel_kernel
);
template __global__ void sort_per_pixel_kernel<PerPixelSortType::QuickSort>(
    _ARGS_sort_per_pixel_kernel
);
template __global__ void sort_per_pixel_kernel<PerPixelSortType::HeapSort>(
    _ARGS_sort_per_pixel_kernel
);
template __global__ void sort_per_pixel_kernel<PerPixelSortType::RandomizedQuickSort>(
    _ARGS_sort_per_pixel_kernel
);
template __global__ void sort_per_pixel_kernel<PerPixelSortType::NetworkSort>(
    _ARGS_sort_per_pixel_kernel
);

template __global__ void rasterize_simple_sorted_forward_kernel<CameraType::Undistorted>(
    _ARGS_rasterize_simple_sorted_forward_kernel
);
template __global__ void rasterize_simple_sorted_forward_kernel<CameraType::GenericDistorted>(
    _ARGS_rasterize_simple_sorted_forward_kernel
);

template __global__ void rasterize_depth_sorted_forward_kernel
<DepthMode::Mean, CameraType::Undistorted>(
    _ARGS_rasterize_depth_sorted_forward_kernel
);
template __global__ void rasterize_depth_sorted_forward_kernel
<DepthMode::Mean, CameraType::GenericDistorted>(
    _ARGS_rasterize_depth_sorted_forward_kernel
);
template __global__ void rasterize_depth_sorted_forward_kernel
<DepthMode::Median, CameraType::Undistorted>(
    _ARGS_rasterize_depth_sorted_forward_kernel
);
template __global__ void rasterize_depth_sorted_forward_kernel
<DepthMode::Median, CameraType::GenericDistorted>(
    _ARGS_rasterize_depth_sorted_forward_kernel
);

template __global__ void rasterize_depth_sorted_backward_kernel<DepthMode::Mean>(
    _ARGS_rasterize_depth_sorted_backward_kernel
);
template __global__ void rasterize_depth_sorted_backward_kernel<DepthMode::Median>(
    _ARGS_rasterize_depth_sorted_backward_kernel
);

template __global__ void rasterize_simplified_sorted_forward_kernel<CameraType::Undistorted>(
    _ARGS_rasterize_simplified_sorted_forward_kernel
);

template __global__ void rasterize_simplified_sorted_forward_kernel<CameraType::GenericDistorted>(
    _ARGS_rasterize_simplified_sorted_forward_kernel
);

template __global__ void rasterize_simplified_sorted_backward_kernel<CameraType::Undistorted>(
    _ARGS_rasterize_simplified_sorted_backward_kernel
);

template __global__ void rasterize_simplified_sorted_backward_kernel<CameraType::GenericDistorted>(
    _ARGS_rasterize_simplified_sorted_backward_kernel
);
