#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include "rasterization_sorted.cuh"
#include <algorithm>

#include "stdio.h"


__global__ void rasterize_indices_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    int* __restrict__ num_intersects,
    int32_t* __restrict__ sorted_indices_,
    float* __restrict__ sorted_depths_
) {
    // each thread draws one pixel, but also timeshares caching splats in a shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // outputs
    int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    float* sorted_depths = &sorted_depths_[pix_id*MAX_SORTED_SPLATS];
    int intersect_count = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float2 aniso = anisotropies[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            id_batch[tr] = g_id;
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = {aniso.x, aniso.y, opac};
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            glm::vec2 aniso = {opacity_batch[t].x, opacity_batch[t].y};
            float opac = opacity_batch[t].z;
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            if (!std::isfinite(poi.z))
                continue;
            float alpha;
            if (!get_alpha(uv, opac, aniso, alpha))
                continue;

            sorted_indices[intersect_count] = (int32_t)id_batch[t];
            sorted_depths[intersect_count] = poi.z;
            if (++intersect_count >= MAX_SORTED_SPLATS){
                done = true;
                break;
            }

            const float next_T = T * (1.f - alpha);

            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    num_intersects[pix_id] = intersect_count;

    // mark this so no need for num_intersects in rendering
    if (intersect_count < MAX_SORTED_SPLATS)
        sorted_indices[intersect_count] = SORTED_INDEX_INF;

    #if 0
        while (intersect_count < MAX_SORTED_SPLATS)
            sorted_depths[intersect_count++] = 1e6f;
    #endif

}




inline __device__ void sort_per_pixel_insertion(int n, int32_t* indices, float* depths) {
    for (int i = 1; i < n; ++i) {
        int gid = indices[i];
        float z = depths[i];
        int j = i-1;

        while (j >= 0 && depths[j] > z) {
            depths[j+1] = depths[j];
            indices[j+1] = indices[j];
            j--;
        }
        depths[j+1] = z;
        indices[j+1] = gid;
    }
}


inline __device__ void sort_per_pixel_quick(int32_t* indices, float* depths, int i0, int i1) {
    if (i1-i0+1 < 8) {
        return sort_per_pixel_insertion(i1-i0+1, indices+i0, depths+i0);
    }

    // split into two arrays by value at center
    int ic = (i0+i1) / 2;
    float z = depths[ic];
    int pi = i0-1;
    for (int j = i0; j <= i1; j++) {
        if (depths[j] < z) {
            if (++pi == j) continue;
            // now there's always pi < j and j != ic
            // swap pi and j
            float z_ = depths[pi]; depths[pi] = depths[j]; depths[j] = z_;
            int32_t i_ = indices[pi]; indices[pi] = indices[j]; indices[j] = i_;
            if (ic == pi) ic = j;
        }
    }
    ++pi;
    // move ic to pi
    if (pi != ic) {
        // float z_ = depths[pi]; depths[pi] = depths[ic]; depths[ic] = z_;
        depths[ic] = depths[pi]; depths[pi] = z;
        int32_t i_ = indices[pi]; indices[pi] = indices[ic]; indices[ic] = i_;
    }

    // debug
    #if 0
        if (depths[pi] != z) printf("@");
        if (pi > ic) printf("#");
        for (int i = i0; i <= pi-1; i++)
            if (depths[i] > depths[pi]) printf("l");
        for (int i = pi+1; i <= i1; i++)
            if (depths[i] < depths[pi]) printf("r");
    #endif

    // warning: may cause stack overflow on some devices in a worst case
    // i.e. "CUDA error: an illegal memory access was encountered"
    if ((pi-1) - i0 > 0)
        sort_per_pixel_quick(indices, depths, i0, pi-1);
    if (i1 - (pi+1) > 0)
        sort_per_pixel_quick(indices, depths, pi+1, i1);
}


inline __device__ void _sort_per_pixel_heap_heapify(int n, int32_t* indices, float* depths, int i) {
    float z_i = depths[i];

    while (true) {
        int largest = i;
        float z_largest = z_i;
        int l = 2*i+1;
        int r = 2*i+2;

        float z_l = depths[l];
        if (l < n && z_l > z_largest) {
            largest = l; z_largest = z_l;
        }

        float z_r = depths[r];
        if (r < n && z_r > z_largest) {
            largest = r; z_largest = z_r;
        }

        if (largest == i) break;

        // swap i and largest
        depths[i] = z_largest; depths[largest] = z_i;
        float i_ = indices[i]; indices[i] = indices[largest]; indices[largest] = i_;

        i = largest;
    }
}

inline __device__ void sort_per_pixel_heap(int n, int32_t* indices, float* depths) {
    for (int i = n/2-1; i >= 0; --i) {
        _sort_per_pixel_heap_heapify(n, indices, depths, i);
    }

    for (int i = n-1; i > 0; --i) {
        // swap 0 and i
        float z_ = depths[0]; depths[0] = depths[i]; depths[i] = z_;
        float i_ = indices[0]; indices[0] = indices[i]; indices[i] = i_;

        _sort_per_pixel_heap_heapify(i, indices, depths, 0);
    }
}


inline __device__ void sort_per_pixel_randomized_quick(int32_t* indices, float* depths, int i0, int i1) {
    if (i1-i0+1 < 8) {
        return sort_per_pixel_insertion(i1-i0+1, indices+i0, depths+i0);
    }

    // random number (FNV-1a hash)
    uint32_t hash = 2166136261u;
    hash ^= (uint32_t)i0;
    hash *= 16777619u;
    hash ^= (uint32_t)i1;
    hash *= 16777619u;
    uint32_t m = i1-i0+1;
    int ic = i0 + (int)(hash%m);

    // split into two arrays
    float z = depths[ic];
    int pi = i0-1;
    for (int j = i0; j <= i1; j++) {
        if (depths[j] < z) {
            if (++pi == j) continue;
            // now there's always pi < j and j != ic
            // swap pi and j
            float z_ = depths[pi]; depths[pi] = depths[j]; depths[j] = z_;
            int32_t i_ = indices[pi]; indices[pi] = indices[j]; indices[j] = i_;
            if (ic == pi) ic = j;
        }
    }
    ++pi;
    // move ic to pi
    if (pi != ic) {
        // float z_ = depths[pi]; depths[pi] = depths[ic]; depths[ic] = z_;
        depths[ic] = depths[pi]; depths[pi] = z;
        int32_t i_ = indices[pi]; indices[pi] = indices[ic]; indices[ic] = i_;
    }

    // warning: may cause stack overflow on some devices in a worst case
    // i.e. "CUDA error: an illegal memory access was encountered"
    if ((pi-1) - i0 > 0)
        sort_per_pixel_randomized_quick(indices, depths, i0, pi-1);
    if (i1 - (pi+1) > 0)
        sort_per_pixel_randomized_quick(indices, depths, pi+1, i1);
}



template <PerPixelSortType SORT_TYPE>
__global__ void sort_per_pixel_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;

    int32_t pix_id = i * img_size.x + j;

    int n = num_intersects[pix_id];
    if (n <= 1)
        return;
    int32_t* indices = &indices_[pix_id*MAX_SORTED_SPLATS];
    float* depths = &depths_[pix_id*MAX_SORTED_SPLATS];
    int intersect_count = 0;

    switch (SORT_TYPE)
    {
    case PerPixelSortType::InsertionSort:
        sort_per_pixel_insertion(n, indices, depths);
        break;
    case PerPixelSortType::QuickSort:
        sort_per_pixel_quick(indices, depths, 0, n-1);
        break;
    case PerPixelSortType::HeapSort:
        sort_per_pixel_heap(n, indices, depths);
        break;
    case PerPixelSortType::RandomizedQuickSort:
        sort_per_pixel_randomized_quick(indices, depths, 0, n-1);
        break;
    }
}



__global__ void rasterize_simple_sorted_forward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float2* __restrict__ anisotropies,
    const float3& __restrict__ background,
    float3* __restrict__ out_img,
    float* __restrict__ out_alpha
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // current visibility left to render
    float T = 1.f;

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    int cur_idx = 0;

    // rasterize
    float3 pix_out = {0.f, 0.f, 0.f};
    for (int cur_idx = 0; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec2 aniso = *(glm::vec2*)&anisotropies[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, aniso, alpha))
            continue;

        const float next_T = T * (1.f - alpha);

        const float vis = alpha * T;
        pix_out.x = pix_out.x + color.x * vis;
        pix_out.y = pix_out.y + color.y * vis;
        pix_out.z = pix_out.z + color.z * vis;
        T = next_T;
        if (T <= 1e-3f) {
            done = true;
            break;
        }
    }

    if (inside) {
        float3 final_color;
        final_color.x = pix_out.x + T * background.x;
        final_color.y = pix_out.y + T * background.y;
        final_color.z = pix_out.z + T * background.z;
        out_img[pix_id] = final_color;
        out_alpha[pix_id] = 1.0f - T;
    }
}



template __global__ void sort_per_pixel_kernel<PerPixelSortType::InsertionSort>(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void sort_per_pixel_kernel<PerPixelSortType::QuickSort>(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void sort_per_pixel_kernel<PerPixelSortType::HeapSort>(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void sort_per_pixel_kernel<PerPixelSortType::RandomizedQuickSort>(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);
