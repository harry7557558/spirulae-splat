#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include "rasterization_sorted.cuh"
#include <algorithm>

#include "stdio.h"


__global__ void rasterize_indices_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ gaussian_ids_sorted,
    const int2* __restrict__ tile_bins,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    int* __restrict__ num_intersects,
    int32_t* __restrict__ sorted_indices_,
    float* __restrict__ sorted_depths_
) {
    // each thread draws one pixel, but also timeshares caching splats in a shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < img_size.y && j < img_size.x);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int2 range = tile_bins[tile_id];
    const int block_size = block.size();
    int num_batches = (range.y - range.x + block_size - 1) / block_size;

    __shared__ int id_batch[MAX_BLOCK_SIZE];
    __shared__ glm::vec3 position_batch[MAX_BLOCK_SIZE];
    __shared__ glm::mat2x3 axes_uv_batch[MAX_BLOCK_SIZE];
    __shared__ float opacity_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    float T = 1.f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // outputs
    int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    float* sorted_depths = &sorted_depths_[pix_id*MAX_SORTED_SPLATS];
    int intersect_count = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range.x + block_size * b;
        int idx = batch_start + tr;
        if (idx < range.y) {
            int32_t g_id = gaussian_ids_sorted[idx];
            const float3 pos = positions[g_id];
            const float opac = opacities[g_id];
            const float3 v0 = axes_u[g_id];
            const float3 v1 = axes_v[g_id];
            id_batch[tr] = g_id;
            position_batch[tr] = {pos.x, pos.y, pos.z};
            axes_uv_batch[tr] = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};
            opacity_batch[tr] = opac;
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range.y - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            glm::vec3 pos = position_batch[t];
            float opac = opacity_batch[t];
            glm::mat2x3 axis_uv = axes_uv_batch[t];

            glm::vec3 poi;
            glm::vec2 uv;
            // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
            //     continue;
            get_intersection(pos, axis_uv, pos_2d, poi, uv);
            if (glm::length(uv) > visibility_kernel_radius())
                continue;
            if (!std::isfinite(poi.z))
                continue;
            float alpha;
            if (!get_alpha(uv, opac, alpha))
                continue;

            sorted_indices[intersect_count] = (int32_t)id_batch[t];
            sorted_depths[intersect_count] = poi.z;
            if (++intersect_count >= MAX_SORTED_SPLATS){
                done = true;
                break;
            }

            const float next_T = T * (1.f - alpha);

            T = next_T;
            cur_idx = batch_start + t;
            if (T <= 1e-3f) {
                done = true;
                break;
            }
        }
    }

    if (inside) {
        num_intersects[pix_id] = intersect_count;

        // mark this so no need for num_intersects in rendering
        if (intersect_count < MAX_SORTED_SPLATS)
            sorted_indices[intersect_count] = SORTED_INDEX_INF;

        #if 0
            while (intersect_count < MAX_SORTED_SPLATS)
                sorted_depths[intersect_count++] = 1e6f;
        #endif
    }

}




inline __device__ void sort_per_pixel_insertion(int n, int32_t* indices, float* depths) {
    for (int i = 1; i < n; ++i) {
        int gid = indices[i];
        float z = depths[i];
        int j = i-1;

        while (j >= 0 && depths[j] > z) {
            depths[j+1] = depths[j];
            indices[j+1] = indices[j];
            j--;
        }
        depths[j+1] = z;
        indices[j+1] = gid;
    }
}


inline __device__ void sort_per_pixel_quick(int32_t* indices, float* depths, int i0, int i1) {
    if (i1-i0+1 < 8) {
        return sort_per_pixel_insertion(i1-i0+1, indices+i0, depths+i0);
    }

    // split into two arrays by value at center
    int ic = (i0+i1) / 2;
    float z = depths[ic];
    int pi = i0-1;
    for (int j = i0; j <= i1; j++) {
        if (depths[j] < z) {
            if (++pi == j) continue;
            // now there's always pi < j and j != ic
            // swap pi and j
            float z_ = depths[pi]; depths[pi] = depths[j]; depths[j] = z_;
            int32_t i_ = indices[pi]; indices[pi] = indices[j]; indices[j] = i_;
            if (ic == pi) ic = j;
        }
    }
    ++pi;
    // move ic to pi
    if (pi != ic) {
        // float z_ = depths[pi]; depths[pi] = depths[ic]; depths[ic] = z_;
        depths[ic] = depths[pi]; depths[pi] = z;
        int32_t i_ = indices[pi]; indices[pi] = indices[ic]; indices[ic] = i_;
    }

    // debug
    #if 0
        if (depths[pi] != z) printf("@");
        if (pi > ic) printf("#");
        for (int i = i0; i <= pi-1; i++)
            if (depths[i] > depths[pi]) printf("l");
        for (int i = pi+1; i <= i1; i++)
            if (depths[i] < depths[pi]) printf("r");
    #endif

    // warning: may cause stack overflow on some devices in a worst case
    // i.e. "CUDA error: an illegal memory access was encountered"
    if ((pi-1) - i0 > 0)
        sort_per_pixel_quick(indices, depths, i0, pi-1);
    if (i1 - (pi+1) > 0)
        sort_per_pixel_quick(indices, depths, pi+1, i1);
}


inline __device__ void _sort_per_pixel_heap_heapify(int n, int32_t* indices, float* depths, int i) {
    float z_i = depths[i];

    while (true) {
        int largest = i;
        float z_largest = z_i;
        int l = 2*i+1;
        int r = 2*i+2;

        float z_l = depths[l];
        if (l < n && z_l > z_largest) {
            largest = l; z_largest = z_l;
        }

        float z_r = depths[r];
        if (r < n && z_r > z_largest) {
            largest = r; z_largest = z_r;
        }

        if (largest == i) break;

        // swap i and largest
        depths[i] = z_largest; depths[largest] = z_i;
        float i_ = indices[i]; indices[i] = indices[largest]; indices[largest] = i_;

        i = largest;
    }
}

inline __device__ void sort_per_pixel_heap(int n, int32_t* indices, float* depths) {
    for (int i = n/2-1; i >= 0; --i) {
        _sort_per_pixel_heap_heapify(n, indices, depths, i);
    }

    for (int i = n-1; i > 0; --i) {
        // swap 0 and i
        float z_ = depths[0]; depths[0] = depths[i]; depths[i] = z_;
        float i_ = indices[0]; indices[0] = indices[i]; indices[i] = i_;

        _sort_per_pixel_heap_heapify(i, indices, depths, 0);
    }
}


inline __device__ void sort_per_pixel_randomized_quick(int32_t* indices, float* depths, int i0, int i1) {
    if (i1-i0+1 < 8) {
        return sort_per_pixel_insertion(i1-i0+1, indices+i0, depths+i0);
    }

    // random number (FNV-1a hash)
    uint32_t hash = 2166136261u;
    hash ^= (uint32_t)i0;
    hash *= 16777619u;
    hash ^= (uint32_t)i1;
    hash *= 16777619u;
    uint32_t m = i1-i0+1;
    int ic = i0 + (int)(hash%m);

    // split into two arrays
    float z = depths[ic];
    int pi = i0-1;
    for (int j = i0; j <= i1; j++) {
        if (depths[j] < z) {
            if (++pi == j) continue;
            // now there's always pi < j and j != ic
            // swap pi and j
            float z_ = depths[pi]; depths[pi] = depths[j]; depths[j] = z_;
            int32_t i_ = indices[pi]; indices[pi] = indices[j]; indices[j] = i_;
            if (ic == pi) ic = j;
        }
    }
    ++pi;
    // move ic to pi
    if (pi != ic) {
        // float z_ = depths[pi]; depths[pi] = depths[ic]; depths[ic] = z_;
        depths[ic] = depths[pi]; depths[pi] = z;
        int32_t i_ = indices[pi]; indices[pi] = indices[ic]; indices[ic] = i_;
    }

    // warning: may cause stack overflow on some devices in a worst case
    // i.e. "CUDA error: an illegal memory access was encountered"
    if ((pi-1) - i0 > 0)
        sort_per_pixel_randomized_quick(indices, depths, i0, pi-1);
    if (i1 - (pi+1) > 0)
        sort_per_pixel_randomized_quick(indices, depths, pi+1, i1);
}



template <PerPixelSortType SORT_TYPE>
__global__ void sort_per_pixel_kernel(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
) {
    auto block = cg::this_thread_block();
    int32_t tile_id =
        block.group_index().y * tile_bounds.x + block.group_index().x;
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;

    int32_t pix_id = i * img_size.x + j;

    int n = num_intersects[pix_id];
    if (n <= 1)
        return;
    int32_t* indices = &indices_[pix_id*MAX_SORTED_SPLATS];
    float* depths = &depths_[pix_id*MAX_SORTED_SPLATS];
    int intersect_count = 0;

    switch (SORT_TYPE)
    {
    case PerPixelSortType::InsertionSort:
        sort_per_pixel_insertion(n, indices, depths);
        break;
    case PerPixelSortType::QuickSort:
        sort_per_pixel_quick(indices, depths, 0, n-1);
        break;
    case PerPixelSortType::HeapSort:
        sort_per_pixel_heap(n, indices, depths);
        break;
    case PerPixelSortType::RandomizedQuickSort:
        sort_per_pixel_randomized_quick(indices, depths, 0, n-1);
        break;
    }
}



__global__ void rasterize_simple_sorted_forward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float3& __restrict__ background,
    float3* __restrict__ out_img,
    float* __restrict__ out_alpha
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];

    // current visibility left to render
    float T = 1.f;

    // rasterize
    float3 pix_out = {0.f, 0.f, 0.f};
    for (int cur_idx = 0; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        const float next_T = T * (1.f - alpha);

        const float vis = alpha * T;
        pix_out.x = pix_out.x + color.x * vis;
        pix_out.y = pix_out.y + color.y * vis;
        pix_out.z = pix_out.z + color.z * vis;
        T = next_T;
        if (T <= 1e-3f) break;
    }

    if (inside) {
        float3 final_color;
        final_color.x = pix_out.x + T * background.x;
        final_color.y = pix_out.y + T * background.y;
        final_color.z = pix_out.z + T * background.z;
        out_img[pix_id] = final_color;
        out_alpha[pix_id] = 1.0f - T;
    }
}


__global__ void rasterize_simple_sorted_backward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int* __restrict__ num_intersects,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float3* __restrict__ colors,
    const float* __restrict__ opacities,
    const float3& __restrict__ background,
    const float* __restrict__ output_alpha,
    const float3* __restrict__ v_output,
    const float* __restrict__ v_output_alpha,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float3* __restrict__ v_colors,
    float* __restrict__ v_opacities
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;
    
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;
    
    int n = num_intersects[pix_id];
    if (n == 0) return;

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - output_alpha[pix_id];
    float T = T_final;
    // the contribution from gaussians behind the current one
    float3 buffer = {0.f, 0.f, 0.f};
    // index of last gaussian to contribute to this pixel

    // df/d_out for this pixel
    const float3 v_out = nan_to_num(v_output[pix_id]);
    const float v_out_alpha = nan_to_num(v_output_alpha[pix_id]);

    // rasterize
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    for (int cur_idx = n-1; cur_idx >= 0; cur_idx--) {
        int g_id = sorted_indices[cur_idx];

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const glm::vec3 color = *(glm::vec3*)&colors[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        glm::vec3 v_position_local = {0.f, 0.f, 0.f};
        glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
        glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
        glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
        glm::vec3 v_color_local = {0.f, 0.f, 0.f};
        float v_opacity_local = 0.f;

        // compute the current T for this gaussian
        const float ra = 1.f / (1.f - alpha);
        const float next_T = T * ra;
        const float vis = alpha * next_T;

        // update v_rgb for this gaussian
        float v_alpha = 0.f;
        v_color_local = {vis * v_out.x, vis * v_out.y, vis * v_out.z};

        // contribution from this pixel
        v_alpha += (color.x * T - buffer.x) * ra * v_out.x;
        v_alpha += (color.y * T - buffer.y) * ra * v_out.y;
        v_alpha += (color.z * T - buffer.z) * ra * v_out.z;

        v_alpha += T_final * ra * v_out_alpha;
        // contribution from background pixel
        v_alpha += -T_final * ra * background.x * v_out.x;
        v_alpha += -T_final * ra * background.y * v_out.y;
        v_alpha += -T_final * ra * background.z * v_out.z;
        // update the running sum
        buffer.x += color.x * vis;
        buffer.y += color.y * vis;
        buffer.z += color.z * vis;

        glm::vec2 v_uv;
        get_alpha_vjp(
            uv, opac,
            v_alpha, v_uv, v_opacity_local
        );
        glm::mat2x3 v_axis_uv = glm::mat2x3(0.0f);
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            glm::vec3(0), v_uv,
            v_position_local, v_axis_uv
        );
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
        // v_position_xy_abs_local /= pos.z;
        v_axis_u_local = v_axis_uv[0];
        v_axis_v_local = v_axis_uv[1];

        T = next_T;

        float* v_position_ptr = (float*)(v_positions);
        atomicAdd(v_position_ptr + 3*g_id + 0, v_position_local.x);
        atomicAdd(v_position_ptr + 3*g_id + 1, v_position_local.y);
        atomicAdd(v_position_ptr + 3*g_id + 2, v_position_local.z);
        float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 0, v_position_xy_abs_local.x);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 1, v_position_xy_abs_local.y);

        float* v_axis_u_ptr = (float*)(v_axes_u);
        atomicAdd(v_axis_u_ptr + 3*g_id + 0, v_axis_u_local.x);
        atomicAdd(v_axis_u_ptr + 3*g_id + 1, v_axis_u_local.y);
        atomicAdd(v_axis_u_ptr + 3*g_id + 2, v_axis_u_local.z);
        float* v_axis_v_ptr = (float*)(v_axes_v);
        atomicAdd(v_axis_v_ptr + 3*g_id + 0, v_axis_v_local.x);
        atomicAdd(v_axis_v_ptr + 3*g_id + 1, v_axis_v_local.y);
        atomicAdd(v_axis_v_ptr + 3*g_id + 2, v_axis_v_local.z);

        float* v_color_ptr = (float*)(v_colors);
        atomicAdd(v_color_ptr + 3*g_id + 0, v_color_local.x);
        atomicAdd(v_color_ptr + 3*g_id + 1, v_color_local.y);
        atomicAdd(v_color_ptr + 3*g_id + 2, v_color_local.z);
        
        atomicAdd(v_opacities + g_id, v_opacity_local);
    }

}



template <DepthMode DEPTH_MODE>
__global__ void rasterize_depth_sorted_forward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    int* __restrict__ final_index,
    float* __restrict__ out_depth,
    float2* __restrict__ out_visibility
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;

    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;

    // list of indices
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];

    // current visibility left to render
    float T = 1.f;
    float interp = 1.0f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // rasterize
    float output_depth = 0.0f;
    float output_visibility = 0.0f;

    for (; cur_idx < MAX_SORTED_SPLATS; cur_idx++) {
        int g_id = sorted_indices[cur_idx];
        if (g_id == SORTED_INDEX_INF)
            break;

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        // if (!get_intersection(pos, axis_uv, pos_2d, poi, uv));
        //     continue;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        const float next_T = T * (1.f - alpha);

        // mean depth
        if (DEPTH_MODE == DepthMode::Mean) {

            // const float depth_raw = pos.z;
            const float depth_raw = poi.z;
            const float depth = depth_map(depth_raw);
            float vis = alpha * T;
            output_depth += vis * depth;

        }  // DEPTH_MODE

        // median depth
        else if (DEPTH_MODE == DepthMode::Median) {

            const float next_depth_raw = poi.z;
            const float next_depth = depth_map(next_depth_raw);
            if (next_T < DEPTH_REG_MEDIAN_TH) {
                if (T < 0.99999f) {
                    // https://www.desmos.com/3d/fttajoozww
                    interp = (1.0f-alpha)/alpha * (T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH;
                    interp = glm::clamp(interp, 0.0f, 1.0f);
                    output_depth = output_depth + (next_depth-output_depth)*interp;
                }
                else {
                    output_depth = next_depth;
                }
                T = next_T;
                cur_idx++;
                break;
            }
            output_depth = next_depth;

        }  // DEPTH_MODE

        T = next_T;
    }

    if (inside) {
        final_index[pix_id] = cur_idx;
        if (DEPTH_MODE == DepthMode::Mean) {
            float depth = T == 1.0f ? output_depth : output_depth / (1.0f-T);
            // out_depth[pix_id] = depth_inv_map(depth);
            out_depth[pix_id] = depth;
            out_visibility[pix_id] = {T, 1.0f-T};
        }
        else if (DEPTH_MODE == DepthMode::Median) {
            // out_depth[pix_id] = depth_inv_map(output_depth);
            out_depth[pix_id] = output_depth;
            out_visibility[pix_id] = {T, interp};
        }
    }
}


template <DepthMode DEPTH_MODE>
__global__ void rasterize_depth_sorted_backward_kernel(
    const dim3 img_size,
    const float4 intrins,
    const int* __restrict__ final_index,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float* __restrict__ out_depth,
    const float2* __restrict__ out_visibility,
    const float* __restrict__ v_out_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float* __restrict__ v_opacities
) {
    auto block = cg::this_thread_block();
    unsigned i =
        block.group_index().y * block.group_dim().y + block.thread_index().y;
    unsigned j =
        block.group_index().x * block.group_dim().x + block.thread_index().x;

    bool inside = (i < img_size.y && j < img_size.x);
    if (!inside) return;
    
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    glm::vec2 pos_screen = { (float)j + 0.5f, (float)i + 0.5f };
    glm::vec2 pos_2d = { (pos_screen.x-cx)/fx, (pos_screen.y-cy)/fy };
    int32_t pix_id = i * img_size.x + j;
    
    int n = final_index[pix_id];
    if (n == 0) return;

    // this is the T AFTER the last gaussian in this pixel
    glm::vec2 meta_out = *(glm::vec2*)&out_visibility[pix_id];
    float T_final = meta_out.x;
    float T = T_final;
    float v_T = 0.0f;
    const float interp = meta_out.y;

    // df/d_out for this pixel
    float output_depth = out_depth[pix_id];
    float v_output_depth = nan_to_num(v_out_depth[pix_id]);
    float v_out_alpha = 0.0f;
    float v_depth = 0.f;
    float v_depth_next = 0.f;
    float v_alpha = 0.f;
    float v_interp = 0.f;
    if (DEPTH_MODE == DepthMode::Mean) {
        if (T != 1.0f) {
            float alpha = 1.0f-T;
            v_out_alpha = -output_depth / fmax(alpha, 1e-4f) * v_output_depth;
            output_depth *= alpha;
            v_output_depth /= fmax(alpha, 1e-4);
        }
    }

    float depth_buffer = 0.0f;

    // rasterize
    const int32_t* sorted_indices = &sorted_indices_[pix_id*MAX_SORTED_SPLATS];
    for (int cur_idx = n-1; cur_idx >= 0; cur_idx--) {
        int g_id = sorted_indices[cur_idx];

        const glm::vec3 pos = *(glm::vec3*)&positions[g_id];
        const float opac = opacities[g_id];
        const float3 v0 = axes_u[g_id];
        const float3 v1 = axes_v[g_id];
        glm::mat2x3 axis_uv = {v0.x, v0.y, v0.z, v1.x, v1.y, v1.z};

        glm::vec3 poi;
        glm::vec2 uv;
        get_intersection(pos, axis_uv, pos_2d, poi, uv);
        if (glm::length(uv) > visibility_kernel_radius())
            continue;
        float alpha;
        if (!get_alpha(uv, opac, alpha))
            continue;

        glm::vec3 v_position_local = {0.f, 0.f, 0.f};
        glm::vec2 v_position_xy_abs_local = {0.f, 0.f};
        glm::vec3 v_axis_u_local = {0.f, 0.f, 0.f};
        glm::vec3 v_axis_v_local = {0.f, 0.f, 0.f};
        float v_opacity_local = 0.f;

        // compute the current T for this gaussian
        const float ra = 1.f / (1.f - alpha);
        const float next_T = T * ra;
        const float vis = alpha * next_T;

        float depth_raw = poi.z;
        float depth = depth_map(depth_raw);

        // mean depth
        if (DEPTH_MODE == DepthMode::Mean) {

            v_depth = vis * v_output_depth;
            v_alpha = (depth * T - depth_buffer) * ra * v_output_depth;
            v_alpha += T_final * ra * v_out_alpha;
            depth_buffer += depth * vis;

        }  // DEPTH_MODE

        // median depth
        else if (DEPTH_MODE == DepthMode::Median) {

            // depth gradient
            if (T == T_final) {
                v_depth = v_output_depth * interp;
                v_depth_next = v_output_depth * (1.0f-interp);
            }
            else {
                v_depth = v_depth_next;
                v_depth_next = 0.0f;
            }

            // alpha gradient
            if (T == T_final && interp < 1.0f && interp > 0.0f) {
                float depth_0 = (output_depth-depth*interp) / (1.0f-interp);
                v_interp = (depth-depth_0) * v_output_depth;
                v_alpha = (next_T-DEPTH_REG_MEDIAN_TH)/DEPTH_REG_MEDIAN_TH * \
                    v_interp / safe_denom(-alpha*alpha, 1e-3);
                v_T = (1.0f-alpha)/alpha * v_interp / DEPTH_REG_MEDIAN_TH;
            }
            else {
                v_alpha = v_T * (-next_T);
                v_T = v_T * (1.0f-alpha);
            }

        }  // DEPTH_MODE

        T = next_T;

        // backward
        glm::vec2 v_uv;
        get_alpha_vjp(
            uv, opac,
            v_alpha, v_uv, v_opacity_local
        );
        glm::mat2x3 v_axis_uv = glm::mat2x3(0.0f);
        float v_depth_raw = depth_map_vjp(depth_raw, v_depth);
        get_intersection_vjp(
            pos, axis_uv, pos_2d,
            {0.f, 0.f, v_depth_raw}, v_uv,
            v_position_local, v_axis_uv
        );
        v_position_xy_abs_local = glm::abs(glm::vec2(v_position_local));
        v_axis_u_local = v_axis_uv[0];
        v_axis_v_local = v_axis_uv[1];
        
        float* v_position_ptr = (float*)(v_positions);
        atomicAdd(v_position_ptr + 3*g_id + 0, v_position_local.x);
        atomicAdd(v_position_ptr + 3*g_id + 1, v_position_local.y);
        atomicAdd(v_position_ptr + 3*g_id + 2, v_position_local.z);
        float* v_positions_xy_abs_ptr = (float*)(v_positions_xy_abs);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 0, v_position_xy_abs_local.x);
        atomicAdd(v_positions_xy_abs_ptr + 2*g_id + 1, v_position_xy_abs_local.y);

        float* v_axis_u_ptr = (float*)(v_axes_u);
        atomicAdd(v_axis_u_ptr + 3*g_id + 0, v_axis_u_local.x);
        atomicAdd(v_axis_u_ptr + 3*g_id + 1, v_axis_u_local.y);
        atomicAdd(v_axis_u_ptr + 3*g_id + 2, v_axis_u_local.z);
        float* v_axis_v_ptr = (float*)(v_axes_v);
        atomicAdd(v_axis_v_ptr + 3*g_id + 0, v_axis_v_local.x);
        atomicAdd(v_axis_v_ptr + 3*g_id + 1, v_axis_v_local.y);
        atomicAdd(v_axis_v_ptr + 3*g_id + 2, v_axis_v_local.z);
        
        float v_opacity_local_ = (float)v_opacity_local;
        atomicAdd(v_opacities + g_id, v_opacity_local_);
    }
}




template __global__ void sort_per_pixel_kernel<PerPixelSortType::InsertionSort>(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void sort_per_pixel_kernel<PerPixelSortType::QuickSort>(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void sort_per_pixel_kernel<PerPixelSortType::HeapSort>(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void sort_per_pixel_kernel<PerPixelSortType::RandomizedQuickSort>(
    const dim3 tile_bounds,
    const dim3 img_size,
    const int* __restrict__ num_intersects,
    int32_t* __restrict__ indices_,
    float* __restrict__ depths_
);

template __global__ void rasterize_depth_sorted_forward_kernel<DepthMode::Mean>(
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    int* __restrict__ final_index,
    float* __restrict__ out_depth,
    float2* __restrict__ out_visibility
);

template __global__ void rasterize_depth_sorted_forward_kernel<DepthMode::Median>(
    const dim3 img_size,
    const float4 intrins,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    int* __restrict__ final_index,
    float* __restrict__ out_depth,
    float2* __restrict__ out_visibility
);

template __global__ void rasterize_depth_sorted_backward_kernel<DepthMode::Mean>(
    const dim3 img_size,
    const float4 intrins,
    const int* __restrict__ num_intersects,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float* __restrict__ out_depth,
    const float2* __restrict__ out_visibility,
    const float* __restrict__ v_out_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float* __restrict__ v_opacities
);

template __global__ void rasterize_depth_sorted_backward_kernel<DepthMode::Median>(
    const dim3 img_size,
    const float4 intrins,
    const int* __restrict__ num_intersects,
    const int32_t* __restrict__ sorted_indices_,
    const float3* __restrict__ positions,
    const float3* __restrict__ axes_u,
    const float3* __restrict__ axes_v,
    const float* __restrict__ opacities,
    const float* __restrict__ out_depth,
    const float2* __restrict__ out_visibility,
    const float* __restrict__ v_out_depth,
    float3* __restrict__ v_positions,
    float2* __restrict__ v_positions_xy_abs,
    float3* __restrict__ v_axes_u,
    float3* __restrict__ v_axes_v,
    float* __restrict__ v_opacities
);
