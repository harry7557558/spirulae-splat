#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;




// device helper to get screen space depth gradient
inline __device__ float2 projected_depth_grad(
    const glm::vec3 p, const glm::mat3 R,
    const float fx, const float fy
) {
    glm::vec3 n1 = R[2];
    glm::mat3 invJ = glm::mat3(
        p.z/fx, 0.0f, 0.0f,
        0.0f, p.z/fy, 0.0f,
        p.x/p.z, p.y/p.z, 1.0f
    );
    glm::vec3 n = glm::transpose(invJ) * n1;
    // n.z = safe_denom(n.z, 1e-3f);
    // return { -n.x/n.z, -n.y/n.z };
    return {-n.x*n.z, -n.y*n.z};
}

inline __device__ void projected_depth_grad_vjp(
    const glm::vec3 p, const glm::mat3 R,
    const float fx, const float fy,
    const float2 v_depth_grad,
    glm::vec3 &v_p_view, glm::mat3 &v_R
) {
    // forward
    glm::vec3 n1 = R[2];
    glm::mat3 invJ = glm::mat3(
        p.z/fx, 0.0f, 0.0f,
        0.0f, p.z/fy, 0.0f,
        p.x/p.z, p.y/p.z, 1.0f
    );
    glm::vec3 n = glm::transpose(invJ) * n1;
    // n.z = safe_denom(n.z, 1e-2f);
    // glm::vec2 depth_grad = glm::vec2(-n.x/n.z, -n.y/n.z);
    glm::vec2 depth_grad = glm::vec2(-n.x*n.z, -n.y*n.z);

    // backward
    glm::vec3 v_n = glm::vec3(
        // -1.0f/n.z * v_depth_grad.x,
        // -1.0f/n.z * v_depth_grad.y,
        // (n.x*v_depth_grad.x + n.y*v_depth_grad.y) / safe_denom(n.z*n.z,1e-2f)
        -n.z * v_depth_grad.x,
        -n.z * v_depth_grad.y,
        -(n.x * v_depth_grad.x + n.y*v_depth_grad.y)
    );
    // rotation
    glm::vec3 v_n1 = invJ * v_n;
    v_R = glm::mat3(0.0);
    v_R[2] = v_n1;
    // view
    glm::mat3 v_invJ = glm::outerProduct(v_n, n1);
    glm::vec3 v_p = glm::vec3(
        v_invJ[0][2] / p.z,
        v_invJ[1][2] / p.z,
        v_invJ[0][0]/fx + v_invJ[1][1]/fy -
        (p.x*v_invJ[0][2]+p.y*v_invJ[1][2]) / safe_denom(p.z*p.z, 1e-2f)
    );
    v_p_view = {v_p.x, v_p.y, v_p.z};
}



// kernel function for projecting each gaussian on device
// each thread processes one gaussian
__global__ void project_gaussians_forward_kernel(
    const int num_points,
    const float3* __restrict__ means3d,
    const float2* __restrict__ scales,
    const float4* __restrict__ quats,
    const float* __restrict__ viewmat,
    const float4 intrins,
    const dim3 tile_bounds,
    const unsigned block_width,
    const float clip_thresh,
    int4* __restrict__ bounds,
    int32_t* __restrict__ num_tiles_hit,
    float3* __restrict__ positions,
    float3* __restrict__ axes_u,
    float3* __restrict__ axes_v
    // float2* __restrict__ depth_grads
) {
    unsigned idx = cg::this_grid().thread_rank(); // idx of thread within grid
    if (idx >= num_points) {
        return;
    }
    bounds[idx] = {0, 0, 0, 0};
    num_tiles_hit[idx] = 0;

    glm::mat3 R0 = glm::mat3(
        viewmat[0], viewmat[4], viewmat[8],
        viewmat[1], viewmat[5], viewmat[9],
        viewmat[2], viewmat[6], viewmat[10]
    );
    glm::vec3 T0 = { viewmat[3], viewmat[7], viewmat[11] };

    // world to view
    glm::vec3 p_world = *(glm::vec3*)&means3d[idx];
    glm::vec3 p_view = R0 * p_world + T0;
    if (!(p_view.z >= clip_thresh))
        return;

    // patch orientation
    float2 scale = scales[idx];
    float4 quat = quats[idx];
    glm::mat3 Rq = quat_to_rotmat(quat);
    glm::mat3 R = R0 * Rq;
    glm::vec3 V0 = scale.x * R[0];
    glm::vec3 V1 = scale.y * R[1];

    // project to 2d
    const float fx = intrins.x;
    const float fy = intrins.y;
    const float cx = intrins.z;
    const float cy = intrins.w;
    float2 center;
    float3 bound;
    const float kr = visibility_kernel_radius();
    project_ellipse_bound(p_view, kr*V0, kr*V1, fx, fy, cx, cy, center, bound);

    // compute the projected area
    int2 tile_min, tile_max;
    get_tile_bbox(center, {bound.x,bound.y}, tile_bounds, tile_min, tile_max, block_width);
    int32_t tile_area = (tile_max.x - tile_min.x) * (tile_max.y - tile_min.y);
    if (tile_area <= 0)
        return;

    // compute the depth gradient
    // float2 depth_grad = projected_depth_grad(p_view, R, fx, fy);

    // output
    bounds[idx] = {tile_min.x, tile_min.y, tile_max.x, tile_max.y};
    num_tiles_hit[idx] = tile_area;
    positions[idx] = {p_view.x, p_view.y, p_view.z};
    axes_u[idx] = {V0.x, V0.y, V0.z};
    axes_v[idx] = {V1.x, V1.y, V1.z};
    // depth_grads[idx] = {depth_grad.x, depth_grad.y};
}


__global__ void project_gaussians_backward_kernel(
    const int num_points,
    const float3* __restrict__ means3d,
    const float2* __restrict__ scales,
    const float4* __restrict__ quats,
    const float* __restrict__ viewmat,  // 3x4 row major
    const float4 intrins,
    const int* __restrict__ num_tiles_hit,
    const float3* __restrict__ v_positions,
    const float3* __restrict__ v_axes_u,
    const float3* __restrict__ v_axes_v,
    // const float2* __restrict__ v_depth_grads,
    float3* __restrict__ v_means3d,
    float2* __restrict__ v_scales,
    float4* __restrict__ v_quats,
    float* __restrict__ v_viewmat
) {
    unsigned idx = cg::this_grid().thread_rank(); // idx of thread within grid
    if (idx >= num_points || num_tiles_hit <= 0) {
        return;
    }

    glm::mat3 R0 = glm::mat3(
        viewmat[0], viewmat[4], viewmat[8],
        viewmat[1], viewmat[5], viewmat[9],
        viewmat[2], viewmat[6], viewmat[10]
    );
    glm::vec3 T0 = { viewmat[3], viewmat[7], viewmat[11] };

    // position
    glm::vec3 p_world = *(glm::vec3*)&means3d[idx];
    float fx = intrins.x;
    float fy = intrins.y;
    glm::vec3 p_view = R0 * p_world + T0;
    glm::vec3 v_p_view = *(glm::vec3*)&v_positions[idx];

    // forward
    float2 scale = scales[idx];
    float4 quat = quats[idx];
    glm::mat3 Rq = quat_to_rotmat(quat);
    glm::mat3 R = R0 * Rq;
    glm::vec3 V0 = scale.x * R[0];
    glm::vec3 V1 = scale.y * R[1];

    // scale
    glm::vec3 v_V0 = *(glm::vec3*)&v_axes_u[idx];
    glm::vec3 v_V1 = *(glm::vec3*)&v_axes_v[idx];
    float2 v_scale = {glm::dot(R[0], v_V0), glm::dot(R[1], v_V1)};

    // orientation
    glm::mat3 v_R = glm::mat3(0.0f);
    v_R[0] = scale.x * v_V0;
    v_R[1] = scale.y * v_V1;

    // depth_grad
    #if 0
    glm::mat3 v_R_dg;
    glm::vec3 v_p_view_dg = {0.f, 0.f, 0.f};
    projected_depth_grad_vjp(
        p_view, R, fx, fy, v_depth_grads[idx],
        v_p_view_dg, v_R_dg);
    v_R += v_R_dg;
    v_p_view += v_p_view_dg;
    #endif

    glm::vec3 v_p_world = glm::transpose(R0) * v_p_view;
    v_means3d[idx] = {v_p_world.x, v_p_world.y, v_p_world.z};
    v_scales[idx] = v_scale;
    glm::mat3 v_Rq = glm::transpose(R0) * v_R;
    float4 v_quat = quat_to_rotmat_vjp(quat, v_Rq);
    v_quats[idx] = v_quat;
    glm::mat3 v_R0 = v_R * glm::transpose(Rq) +
        glm::outerProduct(v_p_view, p_world);
    glm::vec3 v_T0 = v_p_view;

    atomicAdd(&v_viewmat[0], v_R0[0][0]);
    atomicAdd(&v_viewmat[1], v_R0[1][0]);
    atomicAdd(&v_viewmat[2], v_R0[2][0]);
    atomicAdd(&v_viewmat[3], v_T0[0]);
    atomicAdd(&v_viewmat[4], v_R0[0][1]);
    atomicAdd(&v_viewmat[5], v_R0[1][1]);
    atomicAdd(&v_viewmat[6], v_R0[2][1]);
    atomicAdd(&v_viewmat[7], v_T0[1]);
    atomicAdd(&v_viewmat[8], v_R0[0][2]);
    atomicAdd(&v_viewmat[9], v_R0[1][2]);
    atomicAdd(&v_viewmat[10], v_R0[2][2]);
    atomicAdd(&v_viewmat[11], v_T0[2]);
}



// kernel to map each intersection from tile ID and depth to a gaussian
// writes output to isect_ids and gaussian_ids
__global__ void map_gaussian_to_intersects(
    const int num_points,
    const float3* __restrict__ positions,
    int4* __restrict__ bounds,
    const int32_t* __restrict__ cum_tiles_hit,
    const dim3 tile_bounds,
    const unsigned block_width,
    int64_t* __restrict__ isect_ids,
    int32_t* __restrict__ gaussian_ids
) {
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= num_points)
        return;
    int4 bound = bounds[idx];
    if (min(bound.z-bound.x, bound.w-bound.y) <= 0)
        return;

    // update the intersection info for all tiles this gaussian hits
    int32_t cur_idx = (idx == 0) ? 0 : cum_tiles_hit[idx - 1];
    // printf("point %d starting at %d\n", idx, cur_idx);
    int64_t depth_id = (int64_t) * (int32_t *)&(positions[idx].z);
    for (int i = bound.y; i < bound.w; ++i) {
        for (int j = bound.x; j < bound.z; ++j) {
            // isect_id is tile ID and depth as int32
            int64_t tile_id = i * tile_bounds.x + j; // tile within image
            isect_ids[cur_idx] = (tile_id << 32) | depth_id; // tile | depth id
            gaussian_ids[cur_idx] = idx;                     // 3D gaussian id
            ++cur_idx; // handles gaussians that hit more than one tile
        }
    }
    // printf("point %d ending at %d\n", idx, cur_idx);
}


// kernel to map sorted intersection IDs to tile bins
// expect that intersection IDs are sorted by increasing tile ID
// i.e. intersections of a tile are in contiguous chunks
__global__ void get_tile_bin_edges(
    const int num_intersects, const int64_t* __restrict__ isect_ids_sorted, int2* __restrict__ tile_bins
) {
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= num_intersects)
        return;
    // save the indices where the tile_id changes
    int32_t cur_tile_idx = (int32_t)(isect_ids_sorted[idx] >> 32);
    if (idx == 0 || idx == num_intersects - 1) {
        if (idx == 0)
            tile_bins[cur_tile_idx].x = 0;
        if (idx == num_intersects - 1)
            tile_bins[cur_tile_idx].y = num_intersects;
    }
    if (idx == 0)
        return;
    int32_t prev_tile_idx = (int32_t)(isect_ids_sorted[idx - 1] >> 32);
    if (prev_tile_idx != cur_tile_idx) {
        tile_bins[prev_tile_idx].y = idx;
        tile_bins[cur_tile_idx].x = idx;
        return;
    }
}


// Compute relocation for MCMC update
__global__ void compute_relocation_kernel(
    const int num_points,
    const int num_scales,
    const float *opacities,
    const float *scales,
    const int *ratios,
    float *binoms,
    int n_max,
    float *new_opacities,
    float *new_scales
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= num_points)
        return;

    int n_idx = ratios[idx];
    float denom_sum = 0.0f;

    float opac = opacities[idx];

#if 0
    // for Gaussian, minimize residual
    float new_opac = 1.0f - powf(1.0f - opac, 1.0f / n_idx);
    for (int i = 1; i <= n_idx; ++i) {
        for (int k = 0; k <= (i - 1); ++k) {
            float bin_coeff = binoms[(i - 1) * n_max + k];
            float term =
                (pow(-1, k) / sqrt(k + 1)) * pow(new_opac, k + 1);
            denom_sum += (bin_coeff * term);
        }
    }
    float sc = (opacities[idx] / denom_sum);
#else
    // for 1-r^2 splats, strictly less than

    // keep opacity, change scale
    //float new_opac = 1.0f - powf(1.0f - opac, 1.0f / n_idx);
    //float sc = sqrtf(n_idx/opac * new_opac * powf(1.0f-new_opac, n_idx-1.0f));

    // keep scale, change opacity
    float new_opac = opac / n_idx;
    float sc = 1.0f;

#endif

    new_opacities[idx] = new_opac;
    for (int i = 0; i < num_scales; ++i)
        new_scales[idx*num_scales+i] = sc * scales[idx*num_scales+i];
}


__global__ void compute_relocation_split_kernel(
    const int num_points,
    const float3 *positions,
    const float4 *quats,
    const float *opacities,
    const float2 *scales,
    float3 *new_position_offsets,
    float *new_opacities,
    float2 *new_scales
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= num_points)
        return;

    // split into 2 splats, compute position offset (plus/minus)
    // practically the effect of drawing more than 2 splats should be minimal

    // https://www.desmos.com/3d/0ov5xkvbqx

    const float a = 0.25f;

    float opac = opacities[idx];

    const float cx0 = 1.12234f;
    const float cx1 = 0.394553f;
    const float cx2 = -0.120572f;
    float sx1 = cx0 + opac * (cx1 + cx2 * opac);

    const float cy0 = 0.986738f;
    const float cy1 = 0.0304732f;
    const float cy2 = 0.0145201f;
    float sy1 = cy0 + opac * (cy1 + cy2 * opac);

    const float co1 = 0.554027f;
    const float co2 = 0.295756f;
    new_opacities[idx] = opac * (co1 + co2 * opac);

    float4 quat = quats[idx];
    glm::mat3 R = quat_to_rotmat(quat);

    float2 scale = scales[idx];
    if (scale.x > scale.y) {
        glm::vec3 offset = a * scale.x * R[0];
        new_position_offsets[idx] = { offset.x, offset.y, offset.z };
        new_scales[idx] = { scale.x / sx1, scale.y / sy1 };
    }
    else {
        glm::vec3 offset = a * scale.y * R[1];
        new_position_offsets[idx] = { offset.x, offset.y, offset.z };
        new_scales[idx] = { scale.x / sy1, scale.y / sx1 };
    }

}
